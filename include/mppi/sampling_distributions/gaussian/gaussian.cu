#include "hip/hip_runtime.h"
/**
 * Created by Bogdan Vlahov on 3/24/2023
 **/

#include <mppi/sampling_distributions/gaussian/gaussian.cuh>
#include <mppi/core/mppi_common_new.cuh>
#include <mppi/utils/cuda_math_utils.cuh>
#include <mppi/utils/math_utils.h>

namespace mppi
{
namespace sampling_distributions
{
#define GAUSSIAN_TEMPLATE template <class CLASS_T, template <int> class PARAMS_TEMPLATE, class DYN_PARAMS_T>
#define GAUSSIAN_CLASS GaussianDistributionImpl<CLASS_T, PARAMS_TEMPLATE, DYN_PARAMS_T>

__global__ void setGaussianControls(const float* __restrict__ mean_d, const float* __restrict__ std_dev_d,
                                    float* __restrict__ control_samples_d, const int control_dim,
                                    const int num_timesteps, const int num_rollouts, const int num_distributions,
                                    const int optimization_stride, const float std_dev_decay,
                                    const float pure_noise_percentage, const bool time_specific_std_dev)
{
  const int trajectory_index = threadIdx.x + blockDim.x * blockIdx.x;
  const int distribution_index = threadIdx.z + blockDim.z * blockIdx.z;
  const int time_index = threadIdx.y + blockDim.y * blockIdx.y;
  const bool valid_index =
      trajectory_index < num_rollouts && time_index < num_timesteps && distribution_index < num_distributions;
  const auto& num_timesteps_block = blockDim.y;
  const auto& num_rollouts_block = blockDim.x;
  const int shared_noise_index = threadIdx.z * num_timesteps_block * num_rollouts_block * control_dim +
                                 threadIdx.x * num_timesteps_block * control_dim + threadIdx.y * control_dim;
  const int global_noise_index =
      min(distribution_index, num_distributions) * num_timesteps * num_rollouts * control_dim +
      min(trajectory_index, num_rollouts) * num_timesteps * control_dim + min(time_index, num_timesteps) * control_dim;
  const int shared_mean_index = distribution_index * num_timesteps * control_dim + time_index * control_dim;
  // Std Deviation setup
  int std_dev_size = num_distributions * control_dim;
  int shared_std_dev_index = threadIdx.z * num_timesteps * control_dim + threadIdx.y * control_dim;
  int global_std_dev_index = min(distribution_index, num_distributions) * num_timesteps * control_dim +
                             min(time_index, num_timesteps) * control_dim;
  shared_std_dev_index = time_specific_std_dev ? shared_std_dev_index : 0;
  global_std_dev_index = time_specific_std_dev ? global_std_dev_index : 0;
  std_dev_size = time_specific_std_dev ? num_timesteps * std_dev_size : std_dev_size;

  // local variables
  int i, j, k;

  // Shared memory setup
  /**
   * @brief Shared memory setup
   * This kernel has three shared memory arrays, mean_shared, std_dev_shared, and control_samples_shared.
   * In order to prevent memory alignment issues, the memory is being over-allocated to ensure that they are start on
   * the float4 boundary mean_shared - size should be num_timesteps * num_distributions * control_dim std_dev_shared =
   * num_distributions * control_dim if time_specific_std_dev is false std_dev_shared = num_distributions *
   * num_timesteps * control_dim if time_specific_std_dev is true control_samples_shared = BLOCKSIZE_X * BLOCKSIZE_Y *
   * BLOCKSIZE_Z * control_dim
   *
   */
  extern __shared__ float entire_buffer[];
  // Create memory_aligned shared memory pointers
  float* mean_shared = entire_buffer;
  float* std_dev_shared = &mean_shared[mppi::math::nearest_multiple_4(num_timesteps * num_distributions * control_dim)];
  float* control_samples_shared = &std_dev_shared[mppi::math::nearest_multiple_4(std_dev_size)];
  if (control_dim % 4 == 0)
  {
    // Step 1: copy means into shared memory
    for (i = threadIdx.z; i < num_distributions; i += blockDim.z)
    {
      for (j = threadIdx.y; j < num_timesteps; j += blockDim.y)
      {
        const int mean_index = (i * num_timesteps + j) * control_dim;
        float4* mean_shared4 = reinterpret_cast<float4*>(&mean_shared[mean_index]);
        const float4* mean_d4 = reinterpret_cast<const float4*>(&mean_d[mean_index]);
        for (k = threadIdx.x; k < control_dim / 4; k += blockDim.x)
        {
          mean_shared4[k] = mean_d4[k];
        }
      }
    }

    // Step 2: load std_dev to shared memory
    const float4* std_dev_d4 = reinterpret_cast<const float4*>(&std_dev_d[global_std_dev_index]);
    float4* std_dev_shared4 = reinterpret_cast<float4*>(&std_dev_shared[shared_std_dev_index]);
    for (i = threadIdx.x; i < control_dim / 4; i += blockDim.x)
    {
      std_dev_shared4[i] = std_dev_decay * std_dev_d4[i];
    }

    // Step 3: load noise into shared memory
    float4* control_samples_shared4 = reinterpret_cast<float4*>(&control_samples_shared[shared_noise_index]);
    float4* control_samples_d4 = reinterpret_cast<float4*>(&control_samples_d[global_noise_index]);
    // Create const pointre to mean in shared memory as it shouldn't change henceforth
    const float4* mean_shared4 = reinterpret_cast<const float4*>(&mean_shared[shared_mean_index]);
    for (i = 0; valid_index && i < control_dim / 4; i++)
    {
      control_samples_shared4[i] = control_samples_d4[i];
    }

    __syncthreads();  // wait for all copying from global to shared memory to finish
    // Step 4: do mean + variance calculations
    if (valid_index && (trajectory_index == 0 || time_index < optimization_stride))
    {  // 0 noise trajectory
      for (i = 0; i < control_dim / 4; i++)
      {
        control_samples_shared4[i] = mean_shared4[i];
      }
    }
    else if (valid_index && trajectory_index >= (1.0f - pure_noise_percentage) * num_rollouts)
    {  // doing zero mean trajectories
      for (i = 0; i < control_dim / 4; i++)
      {
        control_samples_shared4[i] = std_dev_shared4[i] * control_samples_shared4[i];
      }
    }
    else if (valid_index)
    {
      for (i = 0; i < control_dim / 4; i++)
      {
        control_samples_shared4[i] = mean_shared4[i] + std_dev_shared4[i] * control_samples_shared4[i];
      }
    }

    // save back to global memory
    for (i = 0; valid_index && i < control_dim / 4; i++)
    {
      control_samples_d4[i] = control_samples_shared4[i];
    }
  }
  else if (control_dim % 2 == 0)
  {
    // Step 1: copy means into shared memory
    for (i = threadIdx.z; i < num_distributions; i += blockDim.z)
    {
      for (j = threadIdx.y; j < num_timesteps; j += blockDim.y)
      {
        const int mean_index = (i * num_timesteps + j) * control_dim;
        float2* mean_shared2 = reinterpret_cast<float2*>(&mean_shared[mean_index]);
        const float2* mean_d2 = reinterpret_cast<const float2*>(&mean_d[mean_index]);
        for (k = threadIdx.x; k < control_dim / 2; k += blockDim.x)
        {
          mean_shared2[k] = mean_d2[k];
        }
      }
    }

    // Step 2: load std_dev to shared memory
    const float2* std_dev_d2 = reinterpret_cast<const float2*>(&std_dev_d[global_std_dev_index]);
    float2* std_dev_shared2 = reinterpret_cast<float2*>(&std_dev_shared[shared_std_dev_index]);
    for (i = threadIdx.x; i < control_dim / 2; i += blockDim.x)
    {
      std_dev_shared2[i] = std_dev_decay * std_dev_d2[i];
    }

    // Step 3: load noise into shared memory
    float2* control_samples_shared2 = reinterpret_cast<float2*>(&control_samples_shared[shared_noise_index]);
    float2* control_samples_d2 = reinterpret_cast<float2*>(&control_samples_d[global_noise_index]);
    // Create const pointer to mean in shared memory as it shouldn't change henceforth
    const float2* mean_shared2 = reinterpret_cast<const float2*>(&mean_shared[shared_mean_index]);
    for (i = 0; valid_index && i < control_dim / 2; i++)
    {
      control_samples_shared2[i] = control_samples_d2[i];
    }

    __syncthreads();  // wait for all copying from global to shared memory to finish
    // Step 4: do mean + variance calculations
    if (valid_index && (trajectory_index == 0 || time_index < optimization_stride))
    {  // 0 noise trajectory
      for (i = 0; i < control_dim / 2; i++)
      {
        control_samples_shared2[i] = mean_shared2[i];
      }
    }
    else if (valid_index && trajectory_index >= (1.0f - pure_noise_percentage) * num_rollouts)
    {  // doing zero mean trajectories
      for (i = 0; i < control_dim / 2; i++)
      {
        control_samples_shared2[i] = std_dev_shared2[i] * control_samples_shared2[i];
      }
    }
    else if (valid_index)
    {
      for (i = 0; i < control_dim / 2; i++)
      {
        control_samples_shared2[i] = mean_shared2[i] + std_dev_shared2[i] * control_samples_shared2[i];
      }
    }

    // save back to global memory
    for (i = 0; valid_index && i < control_dim / 2; i++)
    {
      control_samples_d2[i] = control_samples_shared2[i];
    }
  }
  else
  {  // No memory alignment to take advantage of
    // Step 1: copy means into shared memory
    for (i = threadIdx.z; i < num_distributions; i += blockDim.z)
    {
      for (j = threadIdx.y; j < num_timesteps; j += blockDim.y)
      {
        const int mean_index = (i * num_timesteps + j) * control_dim;
        for (k = threadIdx.x; k < control_dim; k += blockDim.x)
        {
          mean_shared[mean_index + k] = mean_d[mean_index + k];
        }
      }
    }
    // __syncthreads();
    // if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
    // {
    //   printf("Mean: ");
    //   for (i = 0; i < num_timesteps; i++)
    //   {
    //     printf("%f, ", mean_d[i]);
    //   }
    //   printf("\n");
    // }
    // __syncthreads();

    // Step 2: load std_dev to shared memory
    for (i = threadIdx.x; i < control_dim; i += blockDim.x)
    {
      std_dev_shared[shared_std_dev_index + i] = std_dev_decay * std_dev_d[global_std_dev_index + i];
    }

    // Step 3: load noise into shared memory
    for (i = 0; valid_index && i < control_dim; i++)
    {
      control_samples_shared[shared_noise_index + i] = control_samples_d[global_noise_index + i];
    }
    // __syncthreads();
    // if (trajectory_index == 10 && time_index == 20)
    // {
    //   printf("Control noise %d at time %d: ", trajectory_index, time_index);
    //   for (i = 0; i < control_dim; i++)
    //   {
    //     printf("%f, ", control_samples_shared[shared_noise_index + i]);
    //   }
    //   printf("\n std_dev_decay: %f, optimization_stride: %d\n", std_dev_decay, optimization_stride);
    //   printf("Std Dev: %f\n", std_dev_shared[shared_std_dev_index]);
    // }
    // __syncthreads();

    __syncthreads();  // wait for all copying from global to shared memory to finish
    // Step 4: do mean + variance calculations
    if (valid_index && (trajectory_index == 0 || time_index < optimization_stride))
    {  // 0 noise trajectory
      for (i = 0; i < control_dim; i++)
      {
        control_samples_shared[shared_noise_index + i] = mean_shared[shared_mean_index + i];
      }
    }
    else if (valid_index && trajectory_index >= (1.0f - pure_noise_percentage) * num_rollouts)
    {  // doing zero mean trajectories
      for (i = 0; i < control_dim; i++)
      {
        control_samples_shared[shared_noise_index + i] =
            std_dev_shared[shared_std_dev_index + i] * control_samples_shared[shared_noise_index + i];
      }
    }
    else if (valid_index)
    {
      for (i = 0; i < control_dim; i++)
      {
        control_samples_shared[shared_noise_index + i] =
            mean_shared[shared_mean_index + i] +
            std_dev_shared[shared_std_dev_index + i] * control_samples_shared[shared_noise_index + i];
      }
    }
    __syncthreads();
    // save back to global memory
    for (i = 0; valid_index && i < control_dim; i++)
    {
      control_samples_d[global_noise_index + i] = control_samples_shared[shared_noise_index + i];
    }
  }
}

GAUSSIAN_TEMPLATE
GAUSSIAN_CLASS::GaussianDistributionImpl(hipStream_t stream) : PARENT_CLASS::SamplingDistribution(stream)
{
}

GAUSSIAN_TEMPLATE
GAUSSIAN_CLASS::GaussianDistributionImpl(const SAMPLING_PARAMS_T& params, hipStream_t stream)
  : PARENT_CLASS::SamplingDistribution(params, stream)
{
}

GAUSSIAN_TEMPLATE
__host__ void GAUSSIAN_CLASS::allocateCUDAMemoryHelper()
{
  if (this->GPUMemStatus_)
  {
    if (std_dev_d_)
    {
      HANDLE_ERROR(hipFreeAsync(std_dev_d_, this->stream_));
    }
    if (control_means_d_)
    {  // deallocate previous memory control trajectory means
      HANDLE_ERROR(hipFreeAsync(control_means_d_, this->stream_));
    }

    if (this->params_.time_specific_std_dev)
    {
      HANDLE_ERROR(hipMallocAsync((void**)&std_dev_d_,
                                   sizeof(float) * CONTROL_DIM * this->getNumTimesteps() * this->getNumDistributions(),
                                   this->stream_));
    }
    else
    {
      HANDLE_ERROR(hipMallocAsync((void**)&std_dev_d_, sizeof(float) * CONTROL_DIM * this->getNumDistributions(),
                                   this->stream_));
    }
    HANDLE_ERROR(hipMallocAsync((void**)&control_means_d_,
                                 sizeof(float) * this->getNumDistributions() * this->getNumTimesteps() * CONTROL_DIM,
                                 this->stream_));
    means_.resize(this->getNumDistributions() * this->getNumTimesteps() * CONTROL_DIM);
    // Ensure that the device side point knows where the the standard deviation memory is located
    HANDLE_ERROR(hipMemcpyAsync(&this->sampling_d_->std_dev_d_, &std_dev_d_, sizeof(float*), hipMemcpyHostToDevice,
                                 this->stream_));
    HANDLE_ERROR(hipMemcpyAsync(&this->sampling_d_->control_means_d_, &control_means_d_, sizeof(float*),
                                 hipMemcpyHostToDevice, this->stream_));
  }
}

GAUSSIAN_TEMPLATE
__host__ void GAUSSIAN_CLASS::freeCudaMem()
{
  if (this->GPUMemStatus_)
  {
    HANDLE_ERROR(hipFree(control_means_d_));
    HANDLE_ERROR(hipFree(std_dev_d_));
    control_means_d_ = nullptr;
    std_dev_d_ = nullptr;
  }
  PARENT_CLASS::freeCudaMem();
}

GAUSSIAN_TEMPLATE
void GAUSSIAN_CLASS::paramsToDevice(bool synchronize)
{
  PARENT_CLASS::paramsToDevice(false);
  if (this->GPUMemStatus_)
  {
    if (this->params_.time_specific_std_dev)
    {
      HANDLE_ERROR(hipMemcpyAsync(this->std_dev_d_, this->params_.std_dev,
                                   sizeof(float) * CONTROL_DIM * this->getNumTimesteps() * this->getNumDistributions(),
                                   hipMemcpyHostToDevice, this->stream_));
    }
    else
    {
      HANDLE_ERROR(hipMemcpyAsync(this->std_dev_d_, this->params_.std_dev,
                                   sizeof(float) * CONTROL_DIM * this->getNumDistributions(), hipMemcpyHostToDevice,
                                   this->stream_));
    }
    if (synchronize)
    {
      HANDLE_ERROR(hipStreamSynchronize(this->stream_));
    }
  }
}

GAUSSIAN_TEMPLATE
__host__ void GAUSSIAN_CLASS::generateSamples(const int& optimization_stride, const int& iteration_num,
                                              hiprandGenerator_t& gen, bool synchronize)
{
  if (this->params_.use_same_noise_for_all_distributions)
  {
    HANDLE_CURAND_ERROR(hiprandGenerateNormal(
        gen, this->control_samples_d_, this->getNumTimesteps() * this->getNumRollouts() * CONTROL_DIM, 0.0f, 1.0f));
    for (int i = 1; i < this->getNumDistributions(); i++)
    {
      HANDLE_ERROR(hipMemcpyAsync(
          &this->control_samples_d_[this->getNumRollouts() * this->getNumTimesteps() * CONTROL_DIM * i],
          this->control_samples_d_, sizeof(float) * this->getNumRollouts() * this->getNumTimesteps() * CONTROL_DIM,
          hipMemcpyDeviceToDevice, this->stream_));
    }
  }
  else
  {
    HANDLE_CURAND_ERROR(hiprandGenerateNormal(
        gen, this->control_samples_d_,
        this->getNumTimesteps() * this->getNumRollouts() * this->getNumDistributions() * CONTROL_DIM, 0.0f, 1.0f));
  }
  const int BLOCKSIZE_X = this->params_.rewrite_controls_block_dim.x;
  const int BLOCKSIZE_Y = this->params_.rewrite_controls_block_dim.y;
  const int BLOCKSIZE_Z = this->params_.rewrite_controls_block_dim.z;
  /**
   * Generate noise samples with mean added
   **/
  dim3 control_writing_grid;
  control_writing_grid.x = mppi::math::int_ceil(this->getNumRollouts(), BLOCKSIZE_X);
  control_writing_grid.y = mppi::math::int_ceil(this->getNumTimesteps(), BLOCKSIZE_Y);
  control_writing_grid.z = mppi::math::int_ceil(this->getNumDistributions(), BLOCKSIZE_Z);
  unsigned int std_dev_mem_size = this->getNumDistributions() * CONTROL_DIM;
  // Allocate shared memory for std_deviations per timestep or constant across the trajectory
  std_dev_mem_size = mppi::math::nearest_multiple_4(
      this->params_.time_specific_std_dev ? std_dev_mem_size * this->getNumTimesteps() : std_dev_mem_size);
  unsigned int shared_mem_size =
      std_dev_mem_size +
      mppi::math::nearest_multiple_4(this->getNumDistributions() * this->getNumTimesteps() * CONTROL_DIM) +
      mppi::math::nearest_multiple_4(BLOCKSIZE_X * BLOCKSIZE_Y * BLOCKSIZE_Z * CONTROL_DIM);
  shared_mem_size *= sizeof(float);
  // std::cout << "Shared mem size: " << shared_mem_size << " bytes. BLOCKSIZE_X: " << BLOCKSIZE_X
  //           << ", BLOCKSIZE_Y: " << BLOCKSIZE_Y << ", BLOCKSIZE_Z: " << BLOCKSIZE_Z
  //           << "Grid: (" << control_writing_grid.x << ", " << control_writing_grid.y
  //           << ", " << control_writing_grid.z << ")" << std::endl;
  setGaussianControls<<<control_writing_grid, this->params_.rewrite_controls_block_dim, shared_mem_size,
                        this->stream_>>>(
      this->control_means_d_, this->std_dev_d_, this->control_samples_d_, CONTROL_DIM, this->getNumTimesteps(),
      this->getNumRollouts(), this->getNumDistributions(), optimization_stride,
      powf(this->params_.std_dev_decay, iteration_num), this->params_.pure_noise_trajectories_percentage,
      this->params_.time_specific_std_dev);

  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));
  }
}

GAUSSIAN_TEMPLATE
__host__ void GAUSSIAN_CLASS::updateDistributionParamsFromDevice(const float* trajectory_weights_d, float normalizer,
                                                                 const int& distribution_i, bool synchronize)
{
  if (distribution_i >= this->getNumDistributions())
  {
    std::cerr << "Updating distributional params for distribution " << distribution_i << " out of "
              << this->getNumDistributions() << " total." << std::endl;
    return;
  }
  float* control_samples_i_d =
      &(this->control_samples_d_[distribution_i * this->getNumRollouts() * this->getNumTimesteps() * CONTROL_DIM]);
  float* control_mean_i_d = &(this->control_means_d_[distribution_i * this->getNumTimesteps() * CONTROL_DIM]);
  mppi::kernels::launchWeightedReductionKernel<CONTROL_DIM>(trajectory_weights_d, control_samples_i_d, control_mean_i_d,
                                                            normalizer, this->getNumTimesteps(), this->getNumRollouts(),
                                                            this->params_.sum_strides, this->stream_, synchronize);
  HANDLE_ERROR(hipMemcpyAsync(&means_[distribution_i * this->getNumTimesteps() * CONTROL_DIM], control_mean_i_d,
                               sizeof(float) * this->getNumTimesteps() * CONTROL_DIM, hipMemcpyDeviceToHost,
                               this->stream_));
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));
  }
}

GAUSSIAN_TEMPLATE
__host__ void GAUSSIAN_CLASS::setHostOptimalControlSequence(float* optimal_control_trajectory,
                                                            const int& distribution_i, bool synchronize)
{
  if (distribution_i >= this->getNumDistributions())
  {
    std::cerr << "Asking for optimal control sequence from distribution " << distribution_i << " out of "
              << this->getNumDistributions() << " total." << std::endl;
    return;
  }

  HANDLE_ERROR(hipMemcpyAsync(
      optimal_control_trajectory, &(this->control_means_d_[this->getNumTimesteps() * CONTROL_DIM * distribution_i]),
      sizeof(float) * this->getNumTimesteps() * CONTROL_DIM, hipMemcpyDeviceToHost, this->stream_));
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));
  }
}

GAUSSIAN_TEMPLATE
__host__ __device__ float GAUSSIAN_CLASS::computeLikelihoodRatioCost(const float* __restrict__ u,
                                                                     float* __restrict__ theta_d,
                                                                     const int sample_index, const int t,
                                                                     const int distribution_idx, const float lambda,
                                                                     const float alpha)
{
  SAMPLING_PARAMS_T* params_p = (SAMPLING_PARAMS_T*)theta_d;
  const int distribution_i = distribution_idx >= params_p->num_distributions ? 0 : distribution_idx;
  float* std_dev = &(params_p->std_dev[CONTROL_DIM * distribution_i]);
  if (params_p->time_specific_std_dev)
  {
    std_dev = &(params_p->std_dev[(distribution_i * params_p->num_timesteps + t) * CONTROL_DIM]);
  }
  float* mean = &(this->control_means_d_[(params_p->num_timesteps * distribution_i + t) * CONTROL_DIM]);
  float* control_cost_coeff = params_p->control_cost_coeff;

  float cost = 0;
#ifdef __CUDA_ARCH__
  int i = threadIdx.y;
  int step = blockDim.y;
#else
  int i = 0;
  int step = 1;
#endif

  if (CONTROL_DIM % 4 == 0)
  {
    float4 cost_i = make_float4(0, 0, 0, 0);
    float4 mean_i, std_dev_i, u_i, control_cost_coeff_i;
    for (; i < CONTROL_DIM / 4; i += step)
    {
      if (sample_index >= (1.0f - params_p->pure_noise_trajectories_percentage) * params_p->num_rollouts)
      {
        mean_i = make_float4(0, 0, 0, 0);
      }
      else
      {
        mean_i = reinterpret_cast<float4*>(mean)[i];  // read mean value from global memory only once
      }
      std_dev_i = reinterpret_cast<float4*>(std_dev)[i];
      u_i = reinterpret_cast<const float4*>(u)[i];
      control_cost_coeff_i = reinterpret_cast<float4*>(control_cost_coeff)[i];
      // cost_i += control_cost_coeff_i * mean_i * (mean_i + 2 * (u_i - mean_i)) / (std_dev_i * std_dev_i);
      cost_i += control_cost_coeff_i * mean_i * (mean_i + 2.0f * u_i) / (std_dev_i * std_dev_i);  // Proper way
    }
    cost += cost_i.x + cost_i.y + cost_i.z + cost_i.w;
  }
  else if (CONTROL_DIM % 2 == 0)
  {
    float2 cost_i = make_float2(0, 0);
    float2 mean_i, std_dev_i, u_i, control_cost_coeff_i;
    for (; i < CONTROL_DIM / 2; i += step)
    {
      if (sample_index >= (1.0f - params_p->pure_noise_trajectories_percentage) * params_p->num_rollouts)
      {
        mean_i = make_float2(0, 0);
      }
      else
      {
        mean_i = reinterpret_cast<float2*>(mean)[i];  // read mean value from global memory only once
      }
      std_dev_i = reinterpret_cast<float2*>(std_dev)[i];
      u_i = reinterpret_cast<const float2*>(u)[i];
      control_cost_coeff_i = reinterpret_cast<float2*>(control_cost_coeff)[i];
      // cost_i += control_cost_coeff_i * mean_i * (mean_i + 2 * (u_i - mean_i)) / (std_dev_i * std_dev_i);
      cost_i += control_cost_coeff_i * mean_i * (mean_i + 2.0f * u_i) / (std_dev_i * std_dev_i);  // Proper way
    }
    cost += cost_i.x + cost_i.y;
  }
  else
  {
    float mean_i;
    for (; i < CONTROL_DIM; i += step)
    {
      if (sample_index >= (1.0f - params_p->pure_noise_trajectories_percentage) * params_p->num_rollouts)
      {
        mean_i = 0.0f;
      }
      else
      {
        mean_i = mean[i];  // read mean value from global memory only once
      }
      cost += control_cost_coeff[i] * mean_i * (mean_i + 2.0f * u[i]) / (std_dev[i] * std_dev[i]);  // Proper way
      // float noise = u[i] - mean_i;
      // cost += control_cost_coeff[i] * mean_i * (u[i] + noise) / (std_dev[i] * std_dev[i]); // Way in cost kernel
    }
  }
  return 0.5 * lambda * (1 - alpha) * cost;
}

GAUSSIAN_TEMPLATE
__host__ __device__ float GAUSSIAN_CLASS::computeFeedbackCost(const float* __restrict__ u_fb,
                                                              float* __restrict__ theta_d, const int t,
                                                              const int distribution_idx, const float lambda,
                                                              const float alpha)
{
  SAMPLING_PARAMS_T* params_p = (SAMPLING_PARAMS_T*)theta_d;
  const int distribution_i = distribution_idx >= params_p->num_distributions ? 0 : distribution_idx;
  float* std_dev = &(params_p->std_dev[CONTROL_DIM * distribution_i]);
  if (params_p->time_specific_std_dev)
  {
    std_dev = &(params_p->std_dev[(distribution_i * params_p->num_timesteps + t) * CONTROL_DIM]);
  }
  float* control_cost_coeff = params_p->control_cost_coeff;

  float cost = 0;
#ifdef __CUDA_ARCH__
  int i = threadIdx.y;
  int step = blockDim.y;
#else
  int i = 0;
  int step = 1;
#endif

  if (CONTROL_DIM % 4 == 0)
  {
    float4 cost_i = make_float4(0, 0, 0, 0);
    float4 std_dev_i, control_cost_coeff_i, u_fb_i;
    for (; i < CONTROL_DIM / 4; i += step)
    {
      std_dev_i = reinterpret_cast<float4*>(std_dev)[i];
      u_fb_i = reinterpret_cast<const float4*>(u_fb)[i];
      control_cost_coeff_i = reinterpret_cast<float4*>(control_cost_coeff)[i];
      cost_i += control_cost_coeff_i * (u_fb_i * u_fb_i) / (std_dev_i * std_dev_i);
    }
    cost += cost_i.x + cost_i.y + cost_i.z + cost_i.w;
  }
  else if (CONTROL_DIM % 2 == 0)
  {
    float2 cost_i = make_float2(0, 0);
    float2 std_dev_i, control_cost_coeff_i, u_fb_i;
    for (; i < CONTROL_DIM / 2; i += step)
    {
      std_dev_i = reinterpret_cast<float2*>(std_dev)[i];
      control_cost_coeff_i = reinterpret_cast<float2*>(control_cost_coeff)[i];
      u_fb_i = reinterpret_cast<const float2*>(u_fb)[i];
      cost_i += control_cost_coeff_i * (u_fb_i * u_fb_i) / (std_dev_i * std_dev_i);
    }
    cost += cost_i.x + cost_i.y;
  }
  else
  {
    for (; i < CONTROL_DIM; i += step)
    {
      cost += control_cost_coeff[i] * (u_fb[i] * u_fb[i]) / (std_dev[i] * std_dev[i]);
    }
  }
  return 0.5 * lambda * (1 - alpha) * cost;
}

GAUSSIAN_TEMPLATE
__host__ float GAUSSIAN_CLASS::computeLikelihoodRatioCost(const Eigen::Ref<const control_array>& u, const int t,
                                                          const int distribution_idx, const float lambda,
                                                          const float alpha)
{
  float cost = 0.0f;
  const int distribution_i = distribution_idx >= this->params_->num_distributions ? 0 : distribution_idx;
  const int mean_index = (distribution_i * this->getNumTimesteps() + t) * CONTROL_DIM;
  float* mean = &(this->means_[mean_index]);
  float* std_dev = &(this->params_.std_dev[CONTROL_DIM * distribution_i]);
  if (this->params_.time_specific_std_dev)
  {
    std_dev = &(this->params_.std_dev[(distribution_i * this->getNumTimesteps() + t) * CONTROL_DIM]);
  }
  for (int i = 0; i < CONTROL_DIM; i++)
  {
    cost += this->params_.control_cost_coeff[i] * mean[i] * (mean[i] + 2.0f * u(i)) /
            (std_dev[i] * std_dev[i]);  // Proper way
  }
  return cost;
}

GAUSSIAN_TEMPLATE
__host__ void GAUSSIAN_CLASS::copyImportanceSamplerToDevice(const float* importance_sampler,
                                                            const int& distribution_idx, bool synchronize)
{
  HANDLE_ERROR(hipMemcpyAsync(&control_means_d_[this->getNumTimesteps() * CONTROL_DIM * distribution_idx],
                               importance_sampler, sizeof(float) * this->getNumTimesteps() * CONTROL_DIM,
                               hipMemcpyHostToDevice, this->stream_));
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));
  }
}
#undef GAUSSIAN_TEMPLATE
#undef GAUSSIAN_CLASS

}  // namespace sampling_distributions
}  // namespace mppi
