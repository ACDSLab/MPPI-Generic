#include <mppi/feedback_controllers/feedback.cuh>

// ===================== GPUFeedbackController ========================
template<class CLASS_T, class DYN_T>
void GPUFeedbackController<CLASS_T, DYN_T>::freeCudaMem(){
  if(GPUMemStatus_) {
    CLASS_T* derived = static_cast<CLASS_T*>(this);
    derived->deallocateCUDAMemory();
    hipFree(feedback_d_);
    GPUMemStatus_ = false;
    feedback_d_ = nullptr;
  }
}

template<class CLASS_T, class DYN_T>
void GPUFeedbackController<CLASS_T, DYN_T>::GPUSetup(){
  CLASS_T* derived = static_cast<CLASS_T*>(this);
  if(!GPUMemStatus_) {
    feedback_d_ = Managed::GPUSetup(derived);
    derived->allocateCUDAMemory();
  } else {
    std::cout << "GPU Memory already set" << std::endl;
  }
  // derived->copyToDevice();
}

// ===================== FeedbackController ========================
