#include "hip/hip_runtime.h"
#include <mppi/feedback_controllers/DDP/ddp.cuh>

template <class GPU_FB_T, class DYN_T, int NUM_TIMESTEPS>
DeviceDDPImpl<GPU_FB_T, DYN_T, NUM_TIMESTEPS>::DeviceDDPImpl(int num_timesteps, hipStream_t stream) :
  num_timesteps_(num_timesteps), GPUFeedbackController<DeviceDDPImpl<GPU_FB_T, DYN_T, NUM_TIMESTEPS>, DYN_T, DDPFeedbackState<DYN_T, NUM_TIMESTEPS>>(stream) {

}

// template <class GPU_FB_T, class DYN_T>
// void DeviceDDPImpl<GPU_FB_T, DYN_T>::allocateCUDAMemory() {
//   int fb_size = DYN_T::STATE_DIM * DYN_T::CONTROL_DIM * num_timesteps_;

//   // Allocate CPU side memery
//   fb_gains_ = new float[fb_size];
//   std::cout << "fb_size: " << fb_size << std::endl;
//   // std::cout << "feedback_d_: " << this->feedback_d_->fb_gains_ << std::endl;
//   // float * fb_gains_d_;
//   // Allocate GPU side memory
//   hipMalloc((void**)&fb_gains_d_, fb_size * sizeof(float));
//   // hipMalloc((void**)&this->feedback_d_->fb_gains_, fb_size * sizeof(float));
//   HANDLE_ERROR(hipMemcpyAsync(&this->feedback_d_->fb_gains_,
//                                &fb_gains_d_,
//                                sizeof(float*),
//                                hipMemcpyHostToDevice,
//                                this->stream_));
//   HANDLE_ERROR( hipStreamSynchronize(this->stream_) );
// }

// template <class GPU_FB_T, class DYN_T>
// void DeviceDDPImpl<GPU_FB_T, DYN_T>::deallocateCUDAMemory() {
//   if (this->fb_gains_d_ != nullptr) {
//     // float* fb_gains_d_;
//     // HANDLE_ERROR(hipMemcpyAsync(&fb_gains_d_,
//     //                              &this->feedback_d_->fb_gains_,
//     //                             sizeof(float*),
//     //                             hipMemcpyDeviceToHost,
//     //                             this->stream_));
//     // HANDLE_ERROR( hipStreamSynchronize(this->stream_) );
//     hipFree(fb_gains_d_);
//     fb_gains_d_ = nullptr;
//   }
//   if (fb_gains_ != nullptr) {
//     delete [] fb_gains_;
//     fb_gains_ = nullptr;
//   }
// }

template <class GPU_FB_T, class DYN_T, int NUM_TIMESTEPS>
__device__ void DeviceDDPImpl<GPU_FB_T, DYN_T, NUM_TIMESTEPS>::k(const float * x_act, const float * x_goal,
                                       const float t, float * theta,
                                       float* control_output) {
  float * fb_gain_t = &(this->params_.fb_gain_traj_[DYN_T::STATE_DIM * DYN_T::CONTROL_DIM * (int) t]);
  float e = 0;
  for (int i = 0; i < DYN_T::STATE_DIM; i++) {
    e = x_act[i] - x_goal[i];
    for(int j = 0; j < DYN_T::CONTROL_DIM; j++) {
      control_output[j] += fb_gain_t[i * DYN_T::CONTROL_DIM + j] * e;
    }
  }
}

// template <class GPU_FB_T, class DYN_T>
// void DeviceDDPImpl<GPU_FB_T, DYN_T>::copyToDevice() {
//   int fb_size = DYN_T::STATE_DIM * DYN_T::CONTROL_DIM * num_timesteps_;
//   HANDLE_ERROR(hipMemcpyAsync(fb_gains_d_,
//                                fb_gains_,
//                                sizeof(float) * fb_size,
//                                hipMemcpyHostToDevice,
//                                this->stream_));
//   HANDLE_ERROR( hipStreamSynchronize(this->stream_) );
// }

template <class DYN_T, int NUM_TIMESTEPS>
DDPFeedback<DYN_T, NUM_TIMESTEPS>::DDPFeedback(DYN_T* model, float dt,
                                               int num_timesteps,
                                               hipStream_t stream) {
  model_ = model;
  this->dt_ = dt;
  this->num_timesteps_ = std::max(num_timesteps, NUM_TIMESTEPS);
  this->gpu_controller_->freeCudaMem(); // Remove allocated CUDA mem from default constructor
  this->gpu_controller_ = std::make_shared<DeviceDDP<DYN_T, NUM_TIMESTEPS>>(this->num_timesteps_, stream);
}

template <class DYN_T, int NUM_TIMESTEPS>
void DDPFeedback<DYN_T, NUM_TIMESTEPS>::initTrackingController() {
  util::DefaultLogger logger;
  bool verbose = false;
  ddp_model_  = std::make_shared<ModelWrapperDDP<DYN_T>>(model_);
  ddp_solver_ = std::make_shared< DDP<ModelWrapperDDP<DYN_T>>>(this->dt_,
                                                               this->num_timesteps_,
                                                               this->params_.num_iterations,
                                                               &logger,
                                                               verbose);

  result_ = OptimizerResult<ModelWrapperDDP<DYN_T>>();
  result_.feedback_gain = feedback_gain_trajectory(NUM_TIMESTEPS);
  for(int i = 0; i < NUM_TIMESTEPS; i++) {
    result_.feedback_gain[i] = DYN_T::feedback_matrix::Zero();
  }

  for (int i = 0; i < DYN_T::CONTROL_DIM; i++) {
    control_min_(i) = model_->control_rngs_[i].x;
    control_max_(i) = model_->control_rngs_[i].y;
  }

  run_cost_ = std::make_shared<TrackingCostDDP<ModelWrapperDDP<DYN_T>>>(this->params_.Q,
                                                                        this->params_.R,
                                                                        NUM_TIMESTEPS);
  terminal_cost_ = std::make_shared<TrackingTerminalCost<ModelWrapperDDP<DYN_T>>>(this->params_.Q_f);
}

template <class DYN_T, int NUM_TIMESTEPS>
void DDPFeedback<DYN_T, NUM_TIMESTEPS>::setParams(DDPParams<DYN_T>& params) {
  this->params_ = params;
  run_cost_ = std::make_shared<TrackingCostDDP<ModelWrapperDDP<DYN_T>>>(this->params_.Q,
                                                                        this->params_.R,
                                                                        NUM_TIMESTEPS);
  terminal_cost_ = std::make_shared<TrackingTerminalCost<ModelWrapperDDP<DYN_T>>>(this->params_.Q_f);
}

// template <class DYN_T, int NUM_TIMESTEPS>
// DDPFeedback<DYN_T, NUM_TIMESTEPS>::control_array DDPFeedback<DYN_T, NUM_TIMESTEPS>::k(
//     const Eigen::Ref<state_array>& x_act,
//     const Eigen::Ref<state_array>& x_goal,
//     float t,
//     INTERNAL_STATE_T& fb_state) {
//   // TODO INTERNAL_STATE_T probably won't compile
//   control_array u_output = fb_state.fb_gain_traj_[t] * (x_act - x_goal);
//   return u_output;
// }

template <class DYN_T, int NUM_TIMESTEPS>
void DDPFeedback<DYN_T, NUM_TIMESTEPS>::computeFeedbackGains(
    const Eigen::Ref<const state_array>& init_state,
    const Eigen::Ref<const state_trajectory>& goal_traj,
    const Eigen::Ref<const control_trajectory>& control_traj) {

  run_cost_->setTargets(goal_traj.data(), control_traj.data(),
                        NUM_TIMESTEPS);

  terminal_cost_->xf = run_cost_->traj_target_x_.col(NUM_TIMESTEPS - 1);
  result_ = ddp_solver_->run(init_state, control_traj,
                             *ddp_model_, *run_cost_, *terminal_cost_,
                             control_min_, control_max_);
  // this->feedback_state_.fb_gain_traj_ = result_.feedback_gain;

  // Copy Feedback Gains into GPU state
  // Doesn't work because feedback_gain is a vector of matrices
  // this->gpu_controller_->fb_gains_ = result_.feedback_gain.data();
  for (size_t i = 0; i < result_.feedback_gain.size(); i++) {
    int i_index = i * DYN_T::STATE_DIM * DYN_T::CONTROL_DIM;
    for (size_t j = 0; j < DYN_T::CONTROL_DIM * DYN_T::STATE_DIM; j++) {
      // this->gpu_controller_->fb_gains_[i_index + j] = result_.feedback_gain[i].data()[j];
      // this->gpu_controller_->params_.fb_gain_traj_[i_index + j] = result_.feedback_gain[i].data()[j];
      // this->feedback_state_.fb_gain_traj_[i_index + j] = result_.feedback_gain[i].data()[j];
      this->getFeedbackStatePointer()->fb_gain_traj_[i_index + j] = result_.feedback_gain[i].data()[j];
    }
  }
  // Actually put new feedback gain trajectory onto the GPU
  this->gpu_controller_->copyToDevice();
}

// template <class DYN_T, int NUM_TIMESTEPS>
// control_array DDPFeedback<DYN_T, NUM_TIMESTEPS>::interpolateFeedback(
//     state_array& state,
//     state_array& target_nominal_state,
//     double rel_time,
//     INTERNAL_STATE_T& fb_state) {
//   // TODO call the feedback controller version directly
//   int lower_idx = (int) (rel_time / dt_);
//   int upper_idx = lower_idx + 1;
//   double alpha = (rel_time - lower_idx * dt_) / dt_;

//   control_array u_fb = (1 - alpha) * k(state, target_nominal_state, lower_idx)
//       + alpha*k(state, target_nominal_state, upper_idx);

//   return u_fb;
// }
