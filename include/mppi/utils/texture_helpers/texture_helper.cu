#include "hip/hip_runtime.h"
#include "texture_helper.cuh"

template <class TEX_T, class DATA_T>
TextureHelper<TEX_T, DATA_T>::TextureHelper(int number, hipStream_t stream) : Managed(stream)
{
  textures_.resize(number);
  textures_buffer_.resize(number);
  cpu_values_.resize(number);
  cpu_buffer_values_.resize(number);
  textures_d_ = textures_.data();
}

template <class TEX_T, class DATA_T>
TextureHelper<TEX_T, DATA_T>::~TextureHelper()
{
  freeCudaMem();
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::GPUSetup()
{
  if (!GPUMemStatus_)
  {
    TEX_T* derived = static_cast<TEX_T*>(this);
    ptr_d_ = Managed::GPUSetup<TEX_T>(derived);
    // allocates memory to access params on the GPU by pointer
    HANDLE_ERROR(hipMalloc(&params_d_, sizeof(TextureParams<DATA_T>) * textures_.size()));
    HANDLE_ERROR(hipMemcpyAsync(&(ptr_d_->textures_d_), &(params_d_), sizeof(TextureParams<DATA_T>*),
                                 hipMemcpyHostToDevice, this->stream_));
    copyToDevice(true);
  }
  else
  {
    std::cout << "GPU Memory already set" << std::endl;
  }
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::freeCudaMem()
{
  if (this->GPUMemStatus_)
  {
    for (int index = 0; index < textures_.size(); index++)
    {
      freeCudaMem(textures_[index]);
    }
    if (params_d_ != nullptr)
    {
      HANDLE_ERROR(hipFree(params_d_));
    }
    if (ptr_d_ != nullptr)
    {
      HANDLE_ERROR(hipFree(ptr_d_));
    }
  }
  this->GPUMemStatus_ = false;
  this->params_d_ = nullptr;
  this->ptr_d_ = nullptr;
  CudaCheckError();
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::freeCudaMem(TextureParams<DATA_T>& texture)
{
  if (texture.allocated)
  {
    HANDLE_ERROR(hipFreeArray(texture.array_d));
    HANDLE_ERROR(hipDestroyTextureObject(texture.tex_d));
    texture.allocated = false;
    texture.array_d = nullptr;
    texture.tex_d = 0;
  }
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::allocateCudaTexture(int index)
{
  // if already allocated, deallocate
  if (this->GPUMemStatus_ && textures_[index].allocated)
  {
    freeCudaMem(textures_[index]);
  }
}

template <class TEX_T, class DATA_T>
__host__ __device__ void TextureHelper<TEX_T, DATA_T>::worldPoseToMapPose(const int index, const float3& input,
                                                                          float3& output)
{
  float3 diff = make_float3(input.x - textures_d_[index].origin.x, input.y - textures_d_[index].origin.y,
                            input.z - textures_d_[index].origin.z);
  float3* rotation_mat_ptr = textures_d_[index].rotations;
  output.x = (rotation_mat_ptr[0].x * diff.x + rotation_mat_ptr[0].y * diff.y + rotation_mat_ptr[0].z * diff.z);
  output.y = (rotation_mat_ptr[1].x * diff.x + rotation_mat_ptr[1].y * diff.y + rotation_mat_ptr[1].z * diff.z);
  output.z = (rotation_mat_ptr[2].x * diff.x + rotation_mat_ptr[2].y * diff.y + rotation_mat_ptr[2].z * diff.z);
}

template <class TEX_T, class DATA_T>
__host__ __device__ void TextureHelper<TEX_T, DATA_T>::mapPoseToTexCoord(const int index, const float3& input,
                                                                         float3& output)
{
  // from map frame to pixels [m] -> [px]
  output.x = input.x / textures_d_[index].resolution.x;
  output.y = input.y / textures_d_[index].resolution.y;
  output.z = input.z / textures_d_[index].resolution.z;

  // normalize pixel values
  output.x /= textures_d_[index].extent.width;
  output.y /= textures_d_[index].extent.height;
  if (textures_d_[index].extent.depth != 0)
  {
    output.z /= textures_d_[index].extent.depth;
  }
}

template <class TEX_T, class DATA_T>
__host__ __device__ void TextureHelper<TEX_T, DATA_T>::worldPoseToTexCoord(const int index, const float3& input,
                                                                           float3& output)
{
  float3 map;
  worldPoseToMapPose(index, input, map);
  mapPoseToTexCoord(index, map, output);
  // printf("world to map (%f, %f, %f) -> (%f, %f, %f) -> (%f, %f, %f)\n", input.x, input.y, input.z, map.x, map.y,
  // map.z, output.x, output.y, output.z);
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::copyToDevice(bool synchronize)
{
  // TODO lock the buffer
  // copies the buffer to the CPU side version
  for (int i = 0; i < textures_buffer_.size(); i++)
  {
    // copy the relevant things over from buffer
    textures_[i].use = textures_buffer_[i].use;
    if (textures_buffer_[i].update_data)
    {
      // moves data from cpu buffer to cpu side
      cpu_values_[i] = std::move(cpu_buffer_values_[i]);
      // TODO should I resize the buffer vector?
      textures_[i].update_data = true;
      textures_buffer_[i].update_data = false;
    }
    if (textures_buffer_[i].update_mem)
    {
      textures_[i].extent = textures_buffer_[i].extent;
      textures_[i].update_mem = true;
      textures_buffer_[i].update_mem = false;
    }
    if (textures_buffer_[i].update_params)
    {
      // copy over params from buffer to object
      textures_[i].origin = textures_buffer_[i].origin;
      textures_[i].rotations[0] = textures_buffer_[i].rotations[0];
      textures_[i].rotations[1] = textures_buffer_[i].rotations[1];
      textures_[i].rotations[2] = textures_buffer_[i].rotations[2];
      textures_[i].resolution = textures_buffer_[i].resolution;
      textures_[i].update_params = true;
      textures_buffer_[i].update_params = false;
    }
  }
  // TODO unlock buffer

  if (!this->GPUMemStatus_)
  {
    return;
  }

  // goes through and checks what needs to be copied and does it
  TEX_T* derived = static_cast<TEX_T*>(this);
  for (int i = 0; i < textures_.size(); i++)
  {
    TextureParams<DATA_T>* param = &textures_[i];

    // do the allocation and texture creation
    if (param->update_mem)
    {
      derived->allocateCudaTexture(i);
      derived->createCudaTexture(i, false);
    }
    // if allocated
    if (param->allocated)
    {
      // if we have new parameter values copy it over
      if (param->update_params)
      {
        derived->copyParamsToGPU(i, false);
      }

      // if we have updated data copy it over
      if (param->update_data)
      {
        // copies data to the GPU
        derived->copyDataToGPU(i, false);
      }
    }
  }
  if (synchronize)
  {
    hipStreamSynchronize(this->stream_);
  }
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::createCudaTexture(int index, bool sync)
{
  TextureParams<DATA_T>* cpu_param = &textures_[index];
  cpu_param->resDesc.res.array.array = cpu_param->array_d;

  HANDLE_ERROR(hipCreateTextureObject(&(cpu_param->tex_d), &cpu_param->resDesc, &cpu_param->texDesc, NULL));

  cpu_param->allocated = true;
  cpu_param->update_mem = false;

  copyParamsToGPU(index, sync);
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::addNewTexture(const hipExtent& extent)
{
  // update the buffer not the actual textures
  textures_buffer_.resize(textures_buffer_.size() + 1);
  textures_.resize(textures_.size() + 1);
  textures_buffer_.back().extent = extent;
  textures_.back().extent = extent;

  if (this->GPUMemStatus_)
  {
    TEX_T* derived = static_cast<TEX_T*>(this);
    int index = textures_.size() - 1;

    // TODO resize the device side array that stores textures

    derived->allocateCudaTexture(index);
    derived->createCudaTexture(index);
    textures_.back().allocated = true;
  }
}

template <class TEX_T, class DATA_T>
__host__ __device__ DATA_T TextureHelper<TEX_T, DATA_T>::queryTextureAtWorldPose(const int index, const float3& input)
{
  float3 tex_coords;
  worldPoseToTexCoord(index, input, tex_coords);
  TEX_T* derived = static_cast<TEX_T*>(this);
  return derived->queryTexture(index, tex_coords);
}

template <class TEX_T, class DATA_T>
__host__ __device__ DATA_T TextureHelper<TEX_T, DATA_T>::queryTextureAtMapPose(const int index, const float3& input)
{
  float3 tex_coords;
  mapPoseToTexCoord(index, input, tex_coords);
  TEX_T* derived = static_cast<TEX_T*>(this);
  return derived->queryTexture(index, tex_coords);
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::updateOrigin(int index, float3 new_origin)
{
  this->textures_buffer_[index].origin = new_origin;
  this->textures_buffer_[index].update_params = true;
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::updateRotation(int index, std::array<float3, 3>& new_rotation)
{
  this->textures_buffer_[index].rotations[0] = new_rotation[0];
  this->textures_buffer_[index].rotations[1] = new_rotation[1];
  this->textures_buffer_[index].rotations[2] = new_rotation[2];
  this->textures_buffer_[index].update_params = true;
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::updateResolution(int index, float resolution)
{
  this->textures_buffer_[index].resolution.x = resolution;
  this->textures_buffer_[index].resolution.y = resolution;
  this->textures_buffer_[index].resolution.z = resolution;
  this->textures_buffer_[index].update_params = true;
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::updateResolution(int index, float3 resolution)
{
  this->textures_buffer_[index].resolution.x = resolution.x;
  this->textures_buffer_[index].resolution.y = resolution.y;
  this->textures_buffer_[index].resolution.z = resolution.z;
  this->textures_buffer_[index].update_params = true;
}

template <class TEX_T, class DATA_T>
bool TextureHelper<TEX_T, DATA_T>::setExtent(int index, hipExtent& extent)
{
  // checks if the extent has changed and reallocates if yes
  TextureParams<DATA_T>* param = &textures_buffer_[index];
  if (param->extent.width != extent.width || param->extent.height != extent.height ||
      param->extent.depth != extent.depth)
  {
    // flag to update mem next time we should
    param->update_mem = true;
    this->textures_buffer_[index].extent = extent;
    return true;
  }
  return false;
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::copyParamsToGPU(int index, bool sync)
{
  TextureParams<DATA_T>* cpu_param = &textures_[index];

  // Copy entire param structure over from CPU to GPU
  HANDLE_ERROR(hipMemcpyAsync(&(params_d_[index]), cpu_param, sizeof(TextureParams<DATA_T>), hipMemcpyHostToDevice,
                               this->stream_));
  cpu_param->update_params = false;
  if (sync)
  {
    hipStreamSynchronize(this->stream_);
  }
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::updateAddressMode(int index, hipTextureAddressMode mode)
{
  this->textures_buffer_[index].texDesc.addressMode[0] = mode;
  this->textures_buffer_[index].texDesc.addressMode[1] = mode;
  this->textures_buffer_[index].texDesc.addressMode[2] = mode;
  this->textures_buffer_[index].update_mem = true;
}

template <class TEX_T, class DATA_T>
void TextureHelper<TEX_T, DATA_T>::updateAddressMode(int index, int layer, hipTextureAddressMode mode)
{
  this->textures_buffer_[index].texDesc.addressMode[layer] = mode;
}
