#include "hip/hip_runtime.h"
//
// Created by jason on 1/5/22.
//

#include "two_d_texture_helper.cuh"

template <class DATA_T>
void TwoDTextureHelper<DATA_T>::allocateCudaTexture(int index)
{
  TextureHelper<TwoDTextureHelper<DATA_T>, DATA_T>::allocateCudaTexture(index);

  TextureParams<DATA_T>* param = &this->textures_[index];

  int w = param->extent.width;
  int h = param->extent.height;

  HANDLE_ERROR(hipMallocArray(&(param->array_d), &(param->channelDesc), w, h));
}

template <class DATA_T>
void TwoDTextureHelper<DATA_T>::updateTexture(const int index, std::vector<DATA_T>& values, bool column_major)
{
  TextureParams<DATA_T>* param = &this->textures_buffer_[index];
  int w = param->extent.width;
  int h = param->extent.height;

  // check that the sizes are correct
  if (values.size() != w * h)
  {
    throw std::runtime_error(std::string("Error: invalid size to updateTexture ") + std::to_string(values.size()) +
                             " != " + std::to_string(w * h));
  }

  // copy over values to cpu side holder
  this->cpu_buffer_values_[index].resize(w * h);
  if (column_major)
  {
    for (int j = 0; j < w; j++)
    {
      for (int i = 0; i < h; i++)
      {
        int columnMajorIndex = j * h + i;
        int rowMajorIndex = i * w + j;
        this->cpu_buffer_values_[index][rowMajorIndex] = values[columnMajorIndex];
      }
    }
  }
  else
  {
    // std::copy(values.begin(), values.end(), cpu_buffer_values_[index].begin());
    this->cpu_buffer_values_[index] = std::move(values);
  }
  // tells the object to copy it over next time that happens
  param->update_data = true;
}

template <class DATA_T>
void TwoDTextureHelper<DATA_T>::updateTexture(const int index, std::vector<DATA_T>& data, hipExtent& extent,
                                              bool column_major)
{
  setExtent(index, extent);
  updateTexture(index, data, column_major);
}

template <class DATA_T>
__host__ __device__ DATA_T TwoDTextureHelper<DATA_T>::queryTexture(const int index, const float3& point)
{
#ifdef __CUDA_ARCH__
  return tex2D<DATA_T>(this->textures_d_[index].tex_d, point.x, point.y);
#else
  return queryTextureCPU(index, point);
#endif
}

template <class DATA_T>
bool TwoDTextureHelper<DATA_T>::setExtent(int index, hipExtent& extent)
{
  if (extent.depth != 0)
  {
    throw std::runtime_error(std::string("Error: extent in setExtent invalid,"
                                         " cannot use depth != 0 in 2D texture: using ") +
                             std::to_string(extent.depth));
  }

  return TextureHelper<TwoDTextureHelper<DATA_T>, DATA_T>::setExtent(index, extent);
}

template <class DATA_T>
void TwoDTextureHelper<DATA_T>::copyDataToGPU(int index, bool sync)
{
  TextureParams<DATA_T>* param = &this->textures_[index];
  int w = param->extent.width;
  int h = param->extent.height;
  HANDLE_ERROR(hipMemcpy2DToArrayAsync(param->array_d, 0, 0, this->cpu_values_[index].data(), w * sizeof(DATA_T),
                                        w * sizeof(DATA_T), h, hipMemcpyHostToDevice, this->stream_));
  if (sync)
  {
    hipStreamSynchronize(this->stream_);
  }
  param->update_data = false;
}

template <class DATA_T>
void TwoDTextureHelper<DATA_T>::updateTexture(
    const int index,
    const Eigen::Ref<const Eigen::Matrix<DATA_T, Eigen::Dynamic, Eigen::Dynamic>, 0,
                     Eigen::Stride<Eigen::Dynamic, Eigen::Dynamic>>
        values,
    bool column_major)
{
  TextureParams<DATA_T>* param = &this->textures_buffer_[index];
  int w = param->extent.width;
  int h = param->extent.height;
  this->cpu_buffer_values_[index].resize(w * h);

  if (column_major)
  {
    // if we are column major transform to row major
    for (int j = 0; j < w; j++)
    {
      for (int i = 0; i < h; i++)
      {
        int columnMajorIndex = j * h + i;
        int rowMajorIndex = i * w + j;
        this->cpu_buffer_values_[index][rowMajorIndex] = values.data()[columnMajorIndex];
      }
    }
  }
  else
  {
    // if we row major copy directly
    memcpy(this->cpu_buffer_values_[index].data(), values.data(), values.size() * sizeof(DATA_T));
  }
  // tells the object to copy it over next time that happens
  param->update_data = true;
}

template <class DATA_T>
void TwoDTextureHelper<DATA_T>::updateTexture(
    int index,
    const Eigen::Ref<const Eigen::Matrix<DATA_T, Eigen::Dynamic, Eigen::Dynamic>, 0,
                     Eigen::Stride<Eigen::Dynamic, Eigen::Dynamic>>
        values,
    hipExtent& extent, bool column_major)
{
  setExtent(index, extent);
  updateTexture(index, values, column_major);
}

template <class DATA_T>
DATA_T TwoDTextureHelper<DATA_T>::queryTextureCPU(const int index, const float3& point)
{
  TextureParams<DATA_T>* param = &this->textures_[index];

  // convert normalized to array index
  float2 query = make_float2(point.x * param->extent.width, point.y * param->extent.height);

  // we subtract half a grid cell since the elevation map is the elevation at the center of the grid cell
  query.x = query.x - 0.5f;
  query.y = query.y - 0.5f;
  // if (this->cpu_values_[index].size() < param->extent.width * param->extent.height)
  // {
  //   return DATA_T();
  // }
  if (param->texDesc.addressMode[0] == hipAddressModeClamp)
  {
    if (query.x > param->extent.width - 1)
    {
      query.x = param->extent.width - 1;
    }
    else if (query.x <= 0.0)
    {
      query.x = 0.0;
    }
  }
  else if (param->texDesc.addressMode[0] == hipAddressModeBorder)
  {
    if (query.x > param->extent.width - 1 || query.x <= 0.0)
    {
      return createPartialCudaTuple<DATA_T>(param->texDesc.borderColor[0], param->texDesc.borderColor[1],
                                            param->texDesc.borderColor[2], param->texDesc.borderColor[3]);
    }
  }
  else
  {
    throw std::runtime_error(std::string("using unsupported address mode on the CPU in texture utils"));
  }
  if (param->texDesc.addressMode[1] == hipAddressModeClamp)
  {
    if (query.y > param->extent.height - 1)
    {
      query.y = param->extent.height - 1;
    }
    else if (query.y <= 0.0)
    {
      query.y = 0.0;
    }
  }
  else if (param->texDesc.addressMode[1] == hipAddressModeBorder)
  {
    if (query.y > param->extent.height - 1 || query.y <= 0.0)
    {
      return createPartialCudaTuple<DATA_T>(param->texDesc.borderColor[0], param->texDesc.borderColor[1],
                                            param->texDesc.borderColor[2], param->texDesc.borderColor[3]);
    }
  }
  else
  {
    throw std::runtime_error(std::string("using unsupported address mode on the CPU in texture utils"));
  }
  int w = param->extent.width;
  if (param->texDesc.filterMode == hipFilterModeLinear)
  {
    // the value is distributed evenly in the space starting at half a cell from 0.0
    int x_min = std::min((int)std::floor(query.x), w - 2);
    int x_max = x_min + 1;
    int y_min = std::min((int)std::floor(query.y), (int)param->extent.height - 2);
    int y_max = y_min + 1;

    // does bilinear interpolation https://en.wikipedia.org/wiki/Bilinear_interpolation

    DATA_T Q_11 = this->cpu_values_[index][y_min * w + x_min];
    DATA_T Q_12 = this->cpu_values_[index][y_min * w + x_max];
    DATA_T Q_21 = this->cpu_values_[index][y_max * w + x_min];
    DATA_T Q_22 = this->cpu_values_[index][y_max * w + x_max];

    DATA_T y_min_interp = Q_11 * ((x_max - query.x) / (x_max - x_min)) + Q_12 * ((query.x - x_min) / (x_max - x_min));
    DATA_T y_max_interp = Q_21 * ((x_max - query.x) / (x_max - x_min)) + Q_22 * ((query.x - x_min) / (x_max - x_min));

    DATA_T result =
        y_min_interp * ((y_max - query.y) / (y_max - y_min)) + y_max_interp * ((query.y - y_min) / (y_max - y_min));

    // does the actual interpolation
    return result;
  }
  else if (param->texDesc.filterMode == hipFilterModePoint)
  {
    int rowMajorIndex = std::round(query.y) * w + std::round(query.x);
    return this->cpu_values_[index][rowMajorIndex];
  }
  else
  {
    throw std::runtime_error(std::string("using unsupported filter mode on the CPU in texture utils"));
  }
}
