#include "hip/hip_runtime.h"
#include "three_d_texture_helper.cuh"

template <class DATA_T>
ThreeDTextureHelper<DATA_T>::ThreeDTextureHelper(int number, bool synched, hipStream_t stream)
  : TextureHelper<ThreeDTextureHelper<DATA_T>, DATA_T>(number, stream)
{
  layer_copy_.resize(number);
  this->synched_ = synched;
  for (std::vector<bool>& layer : layer_copy_)
  {
    // sets all current indexes to be true
    std::fill(layer.begin(), layer.end(), false);
  }
}

template <class DATA_T>
void ThreeDTextureHelper<DATA_T>::allocateCudaTexture(int index)
{
  TextureHelper<ThreeDTextureHelper<DATA_T>, DATA_T>::allocateCudaTexture(index);

  TextureParams<DATA_T>* param = &this->textures_[index];

  // TODO check to make sure our alloc is correct, i.e. extent is valid
  HANDLE_ERROR(hipMalloc3DArray(&(param->array_d), &(param->channelDesc), param->extent));
}

template <class DATA_T>
void ThreeDTextureHelper<DATA_T>::updateTexture(const int index, const int z_index, std::vector<DATA_T>& values,
                                                bool column_major)
{
  TextureParams<DATA_T>* param = &this->textures_buffer_[index];
  int w = param->extent.width;
  int h = param->extent.height;
  int d = param->extent.depth;

  // check that the sizes are correct
  if (values.size() != w * h)
  {
    throw std::runtime_error(std::string("Error: invalid size to updateTexture ") + std::to_string(values.size()) +
                             " != " + std::to_string(w * h));
  }

  // TODO needs to be in the data format used for textures

  if (this->cpu_buffer_values_[index].size() != w * h * d)
  {
    this->cpu_buffer_values_[index].resize(w * h * d);
    // copies values back to the buffer if it has been recently moved
    std::copy(this->cpu_values_[index].begin(), this->cpu_values_[index].end(),
              this->cpu_buffer_values_[index].begin());
  }
  // copy over values to cpu side holder
  if (column_major)
  {
    for (int j = 0; j < w; j++)
    {
      for (int i = 0; i < h; i++)
      {
        int columnMajorIndex = j * h + i;
        int rowMajorIndex = (w * h * z_index) + i * w + j;
        this->cpu_buffer_values_[index][rowMajorIndex] = values[columnMajorIndex];
      }
    }
  }
  else
  {
    auto start = this->cpu_buffer_values_[index].begin() + (w * h * z_index);
    std::copy(values.begin(), values.end(), start);
  }
  // tells the object to copy it over next time that happens
  layer_copy_[index][z_index] = true;
  if (!synched_)
  {
    param->update_data = true;
  }
}

template <class DATA_T>
void ThreeDTextureHelper<DATA_T>::updateTexture(
    const int index, const int z_index,
    const Eigen::Ref<const Eigen::Matrix<DATA_T, Eigen::Dynamic, Eigen::Dynamic>, 0,
                     Eigen::Stride<Eigen::Dynamic, Eigen::Dynamic>>
        values,
    bool column_major)
{
  TextureParams<DATA_T>* param = &this->textures_buffer_[index];
  int w = param->extent.width;
  int h = param->extent.height;
  int d = param->extent.depth;

  // check that the sizes are correct
  if (values.size() != w * h)
  {
    throw std::runtime_error(std::string("Error: invalid size to updateTexture ") + std::to_string(values.size()) +
                             " != " + std::to_string(w * h));
  }

  if (this->cpu_buffer_values_[index].size() != w * h * d)
  {
    this->cpu_buffer_values_[index].resize(w * h * d);
    // copies values back to the buffer if it has been recently moved
    std::copy(this->cpu_values_[index].begin(), this->cpu_values_[index].end(),
              this->cpu_buffers_values_[index].begin());
  }

  // copy over values to cpu side holder
  if (column_major)
  {
    for (int j = 0; j < w; j++)
    {
      for (int i = 0; i < h; i++)
      {
        int columnMajorIndex = j * h + i;
        int rowMajorIndex = (w * h * z_index) + i * w + j;
        this->cpu_buffer_values_[index][rowMajorIndex] = values.data()[columnMajorIndex];
      }
    }
  }
  else
  {
    auto start = this->cpu_buffer_values_[index].data() + (w * h * z_index);
    memcpy(start, values.data(), values.size() * sizeof(DATA_T));
  }
  // tells the object to copy it over next time that happens
  layer_copy_[index][z_index] = true;
  if (!synched_)
  {
    param->update_data = true;
  }
}

// TODO update texture where everything is copied over in one go

template <class DATA_T>
__host__ __device__ DATA_T ThreeDTextureHelper<DATA_T>::queryTexture(const int index, const float3& point)
{
#ifdef __CUDA_ARCH__
  return tex3D<DATA_T>(this->textures_d_[index].tex_d, point.x, point.y, point.z);
#else
  TextureParams<DATA_T>* param = &this->textures_[index];
  float3 query =
      make_float3(point.x * param->extent.width, point.y * param->extent.height, point.z * param->extent.depth);
  query.x = query.x - 0.5f;
  query.y = query.y - 0.5f;
  query.z = query.z - 0.5f;
  if (param->texDesc.addressMode[0] == hipAddressModeClamp)
  {
    if (query.x > param->extent.width - 1)
    {
      query.x = param->extent.width - 1;
    }
    else if (query.x <= 0.0f)
    {
      query.x = 0.0;
    }
  }
  else if (param->texDesc.addressMode[0] == hipAddressModeBorder)
  {
    if (query.x > param->extent.width - 1 || query.x <= 0.0)
    {
      return createPartialCudaTuple<DATA_T>(param->texDesc.borderColor[0], param->texDesc.borderColor[1],
                                            param->texDesc.borderColor[2], param->texDesc.borderColor[3]);
    }
  }
  else
  {
    throw std::runtime_error(std::string("using unsupported address mode on the CPU in texture utils"));
  }
  if (param->texDesc.addressMode[1] == hipAddressModeClamp)
  {
    if (query.y > param->extent.height - 1)
    {
      query.y = param->extent.height - 1;
    }
    else if (query.y <= 0.0f)
    {
      query.y = 0.0;
    }
  }
  else if (param->texDesc.addressMode[1] == hipAddressModeBorder)
  {
    if (query.y > param->extent.height - 1 || query.y <= 0.0)
    {
      return createPartialCudaTuple<DATA_T>(param->texDesc.borderColor[0], param->texDesc.borderColor[1],
                                            param->texDesc.borderColor[2], param->texDesc.borderColor[3]);
    }
  }
  else
  {
    throw std::runtime_error(std::string("using unsupported address mode on the CPU in texture utils"));
  }
  if (param->texDesc.addressMode[2] == hipAddressModeClamp)
  {
    if (query.z > param->extent.depth - 1)
    {
      query.z = param->extent.depth - 1;
    }
    else if (query.z <= 0.0f)
    {
      query.z = 0.0;
    }
  }
  if (param->texDesc.addressMode[2] == hipAddressModeBorder)
  {
    if (query.z > param->extent.depth - 1 || query.z <= 0.0)
    {
      return createPartialCudaTuple<DATA_T>(param->texDesc.borderColor[0], param->texDesc.borderColor[1],
                                            param->texDesc.borderColor[2], param->texDesc.borderColor[3]);
    }
  }
  else if (param->texDesc.addressMode[2] == hipAddressModeWrap)
  {
    while (query.z > param->extent.depth - 1)
    {
      query.z -= param->extent.depth - 1;
    }
    while (query.z < 0.0f)
    {
      query.z += param->extent.depth - 1;
    }
  }
  else
  {
    throw std::runtime_error(std::string("using unsupported address mode on the CPU in texture utils"));
  }

  const int w = param->extent.width;
  const int h = param->extent.height;
  if (param->texDesc.filterMode == hipFilterModeLinear)
  {
    // the value is distributed evenly in the space starting at half a cell from 0.0
    int x_min = std::min((int)std::floor(query.x), w - 2);
    int x_max = x_min + 1;
    int y_min = std::min((int)std::floor(query.y), h - 2);
    int y_max = y_min + 1;
    int z_min = std::min((int)std::floor(query.z), (int)param->extent.depth - 2);
    int z_max = z_min + 1;

    float x_d = (query.x - x_min) / (x_max - x_min);
    float y_d = (query.y - y_min) / (y_max - y_min);
    float z_d = (query.z - z_min) / (z_max - z_min);

    /**
     * does trilinear interpolation https://en.wikipedia.org/wiki/Trilinear_interpolation
     */

    // Query corners of a cube
    DATA_T c_000 = this->cpu_values_[index][(z_min * h + y_min) * w + x_min];
    DATA_T c_100 = this->cpu_values_[index][(z_min * h + y_min) * w + x_max];
    DATA_T c_010 = this->cpu_values_[index][(z_min * h + y_max) * w + x_min];
    DATA_T c_001 = this->cpu_values_[index][(z_max * h + y_min) * w + x_min];
    DATA_T c_110 = this->cpu_values_[index][(z_min * h + y_max) * w + x_max];
    DATA_T c_101 = this->cpu_values_[index][(z_max * h + y_min) * w + x_max];
    DATA_T c_011 = this->cpu_values_[index][(z_max * h + y_max) * w + x_min];
    DATA_T c_111 = this->cpu_values_[index][(z_max * h + y_max) * w + x_max];

    // interpolate along x to make a square
    DATA_T c_00 = c_000 * (1 - x_d) + c_100 * x_d;
    DATA_T c_01 = c_001 * (1 - x_d) + c_101 * x_d;
    DATA_T c_10 = c_010 * (1 - x_d) + c_110 * x_d;
    DATA_T c_11 = c_011 * (1 - x_d) + c_111 * x_d;

    // inperpolate along y to make a line
    DATA_T c_0 = c_00 * (1 - y_d) + c_10 * y_d;
    DATA_T c_1 = c_01 * (1 - y_d) + c_11 * y_d;

    // interpolate along z to get the point
    DATA_T result = c_0 * (1 - z_d) + c_1 * z_d;

    // does the actual interpolation
    return result;
  }
  else if (param->texDesc.filterMode == hipFilterModePoint)
  {
    int rowMajorIndex = (std::round(query.z) * h + std::round(query.y)) * w + std::round(query.x);
    return this->cpu_values_[index][rowMajorIndex];
  }
  else
  {
    throw std::runtime_error(std::string("using unsupported filter mode on the CPU in texture utils"));
  }
#endif
}

template <class DATA_T>
bool ThreeDTextureHelper<DATA_T>::setExtent(int index, hipExtent& extent)
{
  if (extent.depth == 0)
  {
    throw std::runtime_error(std::string("Error: extent in setExtent invalid,"
                                         " cannot use depth == 0 in 3D texture: using ") +
                             std::to_string(extent.depth));
  }

  if (!TextureHelper<ThreeDTextureHelper<DATA_T>, DATA_T>::setExtent(index, extent))
  {
    return false;
  }

  this->cpu_buffer_values_[index].resize(extent.width * extent.height * extent.depth);
  this->cpu_values_[index].resize(extent.width * extent.height * extent.depth);

  // TODO recopy better when depth changes if possible

  // this means we have changed our extent so we need to copy over all the data layers again
  for (std::vector<bool>& layer : layer_copy_)
  {
    // resizes the array to account for change in depth
    layer.resize(extent.depth);
    // sets all current indexes to be true
    std::fill(layer.begin(), layer.end(), true);
  }

  return true;
}

template <class DATA_T>
void ThreeDTextureHelper<DATA_T>::copyDataToGPU(int index, bool sync)
{
  TextureParams<DATA_T>* param = &this->textures_[index];
  auto copy_vec = layer_copy_[index].begin();

  int w = param->extent.width;
  int h = param->extent.height;
  int d = param->extent.depth;

  hipMemcpy3DParms params = { 0 };
  params.srcPtr = make_hipPitchedPtr(this->cpu_values_[index].data(), w * sizeof(DATA_T), w, h);
  params.dstArray = param->array_d;
  params.kind = hipMemcpyHostToDevice;
  params.dstPos = make_hipPos(0, 0, 0);
  params.srcPos = make_hipPos(0, 0, 0);

  // TODO check if we just need to copy it all and do that

  // current index of z we are looking at
  int cur_z_index = 0;
  int prev_z_pos = -1;

  // TODO since we cannot have an extent of zero depth this is fine
  while (cur_z_index + 1 < d)
  {
    if (*(copy_vec + 1) and *copy_vec)
    {
      // if next index is true and cur is true keep building up copy
      copy_vec++;
      cur_z_index++;
      continue;
    }
    else if (!*(copy_vec + 1) and *copy_vec)
    {
      // if the next one is false and current is true begin a copy
      params.extent = make_hipExtent(w, h, cur_z_index - prev_z_pos);

      HANDLE_ERROR(hipMemcpy3DAsync(&params, this->stream_));
      prev_z_pos = cur_z_index;
    }
    else if (*(copy_vec + 1) and !*copy_vec)
    {
      // if the next one is true and cur is false start building up copy
      params.dstPos = make_hipPos(0, 0, cur_z_index + 1);
      params.srcPos = make_hipPos(0, 0, cur_z_index + 1);

      prev_z_pos = cur_z_index;
    }

    // increment counters
    copy_vec++;
    cur_z_index++;
  }

  // execute whatever copy is left
  if (prev_z_pos + 1 != cur_z_index)
  {
    params.extent = make_hipExtent(w, h, cur_z_index - prev_z_pos);
    HANDLE_ERROR(hipMemcpy3DAsync(&params, this->stream_));
  }

  for (std::vector<bool>& layer : layer_copy_)
  {
    std::fill(layer.begin(), layer.end(), false);
  }

  if (sync)
  {
    hipStreamSynchronize(this->stream_);
  }

  param->update_data = false;
}
