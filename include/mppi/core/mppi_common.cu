#include "hip/hip_runtime.h"
#include <mppi/core/mppi_common.cuh>
#include <hiprand.h>
#include <mppi/utils/gpu_err_chk.cuh>

namespace mppi_common
{
/*******************************************************************************************************************
 * Kernel Functions
 *******************************************************************************************************************/
// TODO remove dt
template <class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int NUM_ROLLOUTS, int BLOCKSIZE_Z>
__global__ void rolloutKernel(DYN_T* dynamics, COST_T* costs, float dt, int num_timesteps, int optimization_stride,
                              float lambda, float alpha, float* x_d, float* u_d, float* du_d, float* sigma_u_d,
                              float* trajectory_costs_d)
{
  // Get thread and block id
  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

  // Create shared state and control arrays
  __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float du_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float sigma_u[DYN_T::CONTROL_DIM];
  __shared__ int crash_status_shared[BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the dynamics model to use
  __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD + DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create local state, state dot and controls
  float* x;
  float* xdot;
  float* u;
  float* du;
  int* crash_status;

  // Initialize running cost and total cost
  float running_cost = 0;
  // Load global array to shared array
  if (global_idx < NUM_ROLLOUTS)
  {
    x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    du = &du_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
    crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.
  }
  //__syncthreads();
  loadGlobalToShared(DYN_T::STATE_DIM, DYN_T::CONTROL_DIM, NUM_ROLLOUTS, BLOCKSIZE_Y, global_idx, thread_idy,
                     thread_idz, x_d, sigma_u_d, x, xdot, u, du, sigma_u);
  __syncthreads();

  if (global_idx < NUM_ROLLOUTS)
  {
    /*<----Start of simulation loop-----> */
    dynamics->initializeDynamics(x, u, theta_s, 0.0, dt);
    __syncthreads();
    for (int t = 0; t < num_timesteps; t++)
    {
      // Load noise trajectories scaled by the exploration factor
      injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps, t, global_idx, thread_idy,
                         optimization_stride, u_d, du_d, sigma_u, u, du);
      // du_d is now v
      __syncthreads();

      // applies constraints as defined in dynamics.cuh see specific dynamics class for what happens here
      // usually just control clamping
      // calls enforceConstraints on both since one is used later on in kernel (u), du_d is what is sent back to the CPU
      dynamics->enforceConstraints(x, &du_d[(NUM_ROLLOUTS * num_timesteps * threadIdx.z +  // z part
                                             global_idx * num_timesteps + t) *
                                            DYN_T::CONTROL_DIM]);  //
      dynamics->enforceConstraints(x, u);
      __syncthreads();

      // Accumulate running cost
      if (thread_idy == 0 && t > 0)
      {
        running_cost +=
            (costs->computeRunningCost(x, u, du, sigma_u, lambda, alpha, t, crash_status) - running_cost) / (1.0 * t);
      }

      if (thread_idy == 0 && t == num_timesteps / 2){
        running_cost += costs->terminalCost(x);
      }

      // Compute state derivatives
      dynamics->computeStateDeriv(x, u, xdot, theta_s);
      __syncthreads();

      // Increment states
      dynamics->updateState(x, xdot, dt);
      __syncthreads();
    }
    // Compute terminal cost and the final cost for each thread
    computeAndSaveCost(NUM_ROLLOUTS, global_idx, costs, x, running_cost, trajectory_costs_d);
  }
}

__global__ void normExpKernel(int num_rollouts, float* trajectory_costs_d, float lambda_inv, float baseline)
{
  int global_idx = (blockDim.x * blockIdx.x + threadIdx.x) * blockDim.z + threadIdx.z;

  if (global_idx < num_rollouts * blockDim.z)
  {
    float cost_dif = trajectory_costs_d[global_idx] - baseline;
    trajectory_costs_d[global_idx] = expf(-lambda_inv * cost_dif);
  }
}

template <int CONTROL_DIM, int NUM_ROLLOUTS, int SUM_STRIDE>
__global__ void weightedReductionKernel(float* exp_costs_d, float* du_d, float* du_new_d, float normalizer,
                                        int num_timesteps)
{
  int thread_idx = threadIdx.x;  // Rollout index
  int block_idx = blockIdx.x;    // Timestep

  // Create a shared array for intermediate sums: CONTROL_DIM x NUM_THREADS
  __shared__ float u_intermediate[CONTROL_DIM * ((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1)];

  float u[CONTROL_DIM];
  setInitialControlToZero(CONTROL_DIM, thread_idx, u, u_intermediate);

  __syncthreads();

  // Sum the weighted control variations at a desired stride
  strideControlWeightReduction(NUM_ROLLOUTS, num_timesteps, SUM_STRIDE, thread_idx, block_idx, CONTROL_DIM, exp_costs_d,
                               normalizer, du_d, u, u_intermediate);

  __syncthreads();

  // Sum all weighted control variations
  rolloutWeightReductionAndSaveControl(thread_idx, block_idx, NUM_ROLLOUTS, num_timesteps, CONTROL_DIM, SUM_STRIDE, u,
                                       u_intermediate, du_new_d);

  __syncthreads();
}

/*******************************************************************************************************************
 * Rollout Kernel Helpers
 *******************************************************************************************************************/
__device__ void loadGlobalToShared(int state_dim, int control_dim, int num_rollouts, int blocksize_y, int global_idx,
                                   int thread_idy, int thread_idz, const float* x_device, const float* sigma_u_device,
                                   float* x_thread, float* xdot_thread, float* u_thread, float* du_thread,
                                   float* sigma_u_thread)
{
  // Transfer to shared memory
  int i;
  if (global_idx < num_rollouts)
  {
    for (i = thread_idy; i < state_dim; i += blocksize_y)
    {
      x_thread[i] = x_device[i + state_dim * thread_idz];
      xdot_thread[i] = 0;
    }
    for (i = thread_idy; i < control_dim; i += blocksize_y)
    {
      u_thread[i] = 0;
      du_thread[i] = 0;
      // Only do in threadIdx.x and parallelize along threadIdx.y
      // sigma_u_thread[i] = sigma_u_device[i];
    }
  }
  if (threadIdx.x == 0 /*&& threadIdx.z == 0*/)
  {
    for (i = thread_idy; i < control_dim; i += blocksize_y)
    {
      sigma_u_thread[i] = sigma_u_device[i];
    }
  }
}

// TODO generalize the trim control
// The zero control trajectory should be an equilbrium control defined in the dynamics.
__device__ void injectControlNoise(int control_dim, int blocksize_y, int num_rollouts, int num_timesteps,
                                   int current_timestep, int global_idx, int thread_idy, int optimization_stride,
                                   const float* u_traj_device, float* ep_v_device, const float* sigma_u_thread,
                                   float* u_thread, float* du_thread)
{
  // this is a global index
  int control_index = (num_rollouts * num_timesteps * threadIdx.z +  // z part
                       global_idx * num_timesteps + current_timestep) *
                      control_dim;  // normal part
  // Load the noise trajectory scaled by the exploration factor
  // The prior loop already guarantees that the global index is less than the number of rollouts

  for (int i = thread_idy; i < control_dim; i += blocksize_y)
  {
    // Keep one noise free trajectory
    if (global_idx == 0 || current_timestep < optimization_stride)
    {
      du_thread[i] = 0;
      u_thread[i] = u_traj_device[current_timestep * control_dim + i];
    }
    // Generate 1% zero control trajectory
    else if (global_idx >= 0.99 * num_rollouts)
    {
      du_thread[i] = ep_v_device[control_index + i] * sigma_u_thread[i];
      u_thread[i] = du_thread[i];
    }
    else
    {
      du_thread[i] = ep_v_device[control_index + i] * sigma_u_thread[i];
      u_thread[i] = u_traj_device[current_timestep * control_dim + i] + du_thread[i];
    }
    // Saves the control but doesn't clamp it.
    ep_v_device[control_index + i] = u_thread[i];
  }
}

template <class COST_T>
__device__ void computeAndSaveCost(int num_rollouts, int global_idx, COST_T* costs, float* x_thread, float running_cost,
                                   float* cost_rollouts_device)
{
  // only want to save 1 cost per trajectory
  if (threadIdx.y == 0 && global_idx < num_rollouts)
  {
    cost_rollouts_device[global_idx + num_rollouts * threadIdx.z] = running_cost; // + costs->terminalCost(x_thread);
  }
}

/*******************************************************************************************************************
 * NormExp Kernel Helpers
 *******************************************************************************************************************/
float computeBaselineCost(float* cost_rollouts_host, int num_rollouts)
{  // TODO if we use standard containers in MPPI, should this be replaced with a min algorithm?
  int best_idx = computeBestIndex(cost_rollouts_host, num_rollouts);
  return cost_rollouts_host[best_idx];
}

float constructBestWeights(float* cost_rollouts_host, int num_rollouts)
{
  int best_idx = computeBestIndex(cost_rollouts_host, num_rollouts);
  float best_cost = cost_rollouts_host[best_idx];

  for (int i = 0; i < num_rollouts; i++)
  {
    if (i == best_idx)
    {
      cost_rollouts_host[i] = 1.0;
    }
    else
    {
      cost_rollouts_host[i] = 0.0;
    }
  }

  // printf("Best idx: %d, cost: %f\n", best_cost_idx, best_cost);
  return best_cost;
}

int computeBestIndex(float* cost_rollouts_host, int num_rollouts)
{
  float best_cost = cost_rollouts_host[0];
  int best_cost_idx = 0;
  for (int i = 0; i < num_rollouts; i++)
  {
    if (cost_rollouts_host[i] < best_cost)
    {
      best_cost = cost_rollouts_host[i];
      best_cost_idx = i;
    }
  }

  // printf("Best idx: %d, cost: %f\n", best_cost_idx, best_cost);
  return best_cost_idx;
}

float computeNormalizer(float* cost_rollouts_host, int num_rollouts)
{
  float normalizer = 0.f;
  for (int i = 0; i < num_rollouts; ++i)
  {
    normalizer += cost_rollouts_host[i];
  }
  return normalizer;
}

void computeFreeEnergy(float& free_energy, float& free_energy_var, float& free_energy_modified,
                       float* cost_rollouts_host, int num_rollouts, float baseline, float lambda)
{
  float var = 0;
  float norm = 0;
  for (int i = 0; i < num_rollouts; i++)
  {
    norm += cost_rollouts_host[i];
    var += powf(cost_rollouts_host[i], 2);
  }
  norm /= num_rollouts;
  free_energy = -lambda * logf(norm) + baseline;
  free_energy_var = lambda * (var / num_rollouts - powf(norm, 2));
  // TODO Figure out the point of the following lines
  float weird_term = free_energy_var / (norm * sqrtf(1.0 * num_rollouts));
  free_energy_modified = lambda * (weird_term + 0.5 * powf(weird_term, 2));
}

/*******************************************************************************************************************
 * Weighted Reduction Kernel Helpers
 *******************************************************************************************************************/
__device__ void setInitialControlToZero(int control_dim, int thread_idx, float* u, float* u_intermediate)
{
  for (int i = 0; i < control_dim; i++)
  {
    u[i] = 0;
    u_intermediate[thread_idx * control_dim + i] = 0;
  }
}

__device__ void strideControlWeightReduction(int num_rollouts, int num_timesteps, int sum_stride, int thread_idx,
                                             int block_idx, int control_dim, float* exp_costs_d, float normalizer,
                                             float* du_d, float* u, float* u_intermediate)
{
  // int index = thread_idx * sum_stride + i;
  for (int i = 0; i < sum_stride; ++i)
  {  // Iterate through the size of the subsection
    if ((thread_idx * sum_stride + i) < num_rollouts)
    {                                                                        // Ensure we do not go out of bounds
      float weight = exp_costs_d[thread_idx * sum_stride + i] / normalizer;  // compute the importance sampling weight
      for (int j = 0; j < control_dim; ++j)
      {  // Iterate through the control dimensions
        // Rollout index: (thread_idx*sum_stride + i)*(num_timesteps*control_dim)
        // Current timestep: block_idx*control_dim
        u[j] = du_d[(thread_idx * sum_stride + i) * (num_timesteps * control_dim) + block_idx * control_dim + j];
        u_intermediate[thread_idx * control_dim + j] += weight * u[j];
      }
    }
  }
}

__device__ void rolloutWeightReductionAndSaveControl(int thread_idx, int block_idx, int num_rollouts, int num_timesteps,
                                                     int control_dim, int sum_stride, float* u, float* u_intermediate,
                                                     float* du_new_d)
{
  if (thread_idx == 0 && block_idx < num_timesteps)
  {  // block index refers to the current timestep
    for (int i = 0; i < control_dim; ++i)
    {  // TODO replace with memset?
      u[i] = 0;
    }
    for (int i = 0; i < ((num_rollouts - 1) / sum_stride + 1); ++i)
    {  // iterate through the each subsection
      for (int j = 0; j < control_dim; ++j)
      {
        u[j] += u_intermediate[i * control_dim + j];
      }
    }
    for (int i = 0; i < control_dim; i++)
    {
      du_new_d[block_idx * control_dim + i] = u[i];
    }
  }
}

template <class DYN_T, class COST_T, class FB_T, int BLOCKSIZE_X, int BLOCKSIZE_Z>
__global__ void stateAndCostTrajectoryKernel(DYN_T* dynamics, COST_T* costs, FB_T* fb_controller, float* control,
                                             float* state, float* state_traj_d, float* cost_traj_d, int* crash_status_d,
                                             int num_rollouts, int num_timesteps, float dt, float value_func_threshold)
{
  // Get thread and block id
  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

  // Create shared state and control arrays
  __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];

  // Create a shared array for the nominal costs calculations
  __shared__ int crash_status_shared[BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the dynamics model to use
  __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD + DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];
  __shared__ float theta_fb[FB_T::SHARED_MEM_SIZE];

  // Create local state, state dot and controls
  float* x;
  float* x_other;
  float* xdot;
  float* u;
  int* crash_status;
  float fb_control[DYN_T::CONTROL_DIM];
  int t_index = 0;
  int cost_index = 0;

  if (global_idx < num_rollouts)
  {
    // Actual or nominal
    x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    // The opposite state from above
    x_other = &x_shared[(blockDim.x * (1 - thread_idz) + thread_idx) * DYN_T::STATE_DIM];
    xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    // Base trajectory
    u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    // Nominal State Cost
    crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
    crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.

    // Load memory into appropriate arrays
    for (int i = thread_idy; i < DYN_T::STATE_DIM; i += blockDim.y)
    {
      x[i] = state[DYN_T::STATE_DIM * threadIdx.z + i];
      xdot[i] = 0.0;
    }
    __syncthreads();
    float curr_state_cost = 0.0;

    dynamics->initializeDynamics(x, u, theta_s, 0.0, dt);

    for (int t = 0; t < num_timesteps; t++)
    {
      t_index = threadIdx.z * num_rollouts * num_timesteps + global_idx * num_timesteps + t;
      cost_index = threadIdx.z * num_rollouts * (num_timesteps + 1) + global_idx * (num_timesteps + 1) + t;
      // get next u
      for (int i = thread_idy; i < DYN_T::CONTROL_DIM; i += blockDim.y)
      {
        u[i] = control[global_idx * num_timesteps * DYN_T::CONTROL_DIM + t * DYN_T::CONTROL_DIM + i];
      }

      // only apply feedback if enabled
      // feedback is only applied on real state in RMPPI
      if (BLOCKSIZE_Z > 1 && value_func_threshold == -1 && thread_idz == 0)
      {
        fb_controller->k(x, x_other, t, theta_fb, fb_control);

        for (int i = thread_idy; i < DYN_T::CONTROL_DIM; i += blockDim.y)
        {
          u[i] += fb_control[i];
        }
      }
      __syncthreads();

      dynamics->enforceConstraints(x, u);
      __syncthreads();

      if (thread_idy == 0)
      {
        curr_state_cost = costs->computeStateCost(x, t, crash_status);
        crash_status_d[t_index] = crash_status[0];
        cost_traj_d[cost_index] = curr_state_cost;
      }
      __syncthreads();
      // Nominal system is where thread_idz == 1
      if (thread_idz == 1 && thread_idy == 0)
      {
        // compute the nominal system cost
        cost_traj_d[cost_index] =
            0.5 * curr_state_cost +
            // here we know threadIdx.z == 0 since we are only talking about the real system
            fmaxf(fminf(cost_traj_d[global_idx * (num_timesteps + 1) + t], value_func_threshold), curr_state_cost);
      }
      __syncthreads();
      // reset crash status in case initial location is actually a crash cost
      if (t == 0)
      {
        crash_status[0] = 0;
      }

      // Compute state derivatives
      dynamics->computeStateDeriv(x, u, xdot, theta_s);
      __syncthreads();

      // Increment states
      dynamics->updateState(x, xdot, dt);
      __syncthreads();

      // save results, skips the first state location since that is known
      for (int i = thread_idy; i < DYN_T::STATE_DIM; i += blockDim.y)
      {
        state_traj_d[t_index * DYN_T::STATE_DIM + i] = x[i];
      }
    }
    // get cost traj at +1
    cost_index = threadIdx.z * num_rollouts * (num_timesteps + 1) + global_idx * (num_timesteps + 1) + num_timesteps;
    cost_traj_d[cost_index] = costs->terminalCost(x);
  }
}

/*******************************************************************************************************************
 * Launch Functions
 *******************************************************************************************************************/
template <class DYN_T, class COST_T, int NUM_ROLLOUTS, int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z = 1>
void launchRolloutKernel(DYN_T* dynamics, COST_T* costs, float dt, int num_timesteps, int optimization_stride,
                         float lambda, float alpha, float* x_d, float* u_d, float* du_d, float* sigma_u_d,
                         float* trajectory_costs, hipStream_t stream)
{
  const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(gridsize_x, 1, 1);
  rolloutKernel<DYN_T, COST_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS, BLOCKSIZE_Z>
      <<<dimGrid, dimBlock, 0, stream>>>(dynamics, costs, dt, num_timesteps, optimization_stride, lambda, alpha, x_d,
                                         u_d, du_d, sigma_u_d, trajectory_costs);
  // CudaCheckError();
  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipStreamSynchronize(stream));
}

void launchNormExpKernel(int num_rollouts, int blocksize_x, float* trajectory_costs_d, float lambda_inv, float baseline,
                         hipStream_t stream)
{
  dim3 dimBlock(blocksize_x, 1, 1);
  dim3 dimGrid((num_rollouts - 1) / blocksize_x + 1, 1, 1);
  normExpKernel<<<dimGrid, dimBlock, 0, stream>>>(num_rollouts, trajectory_costs_d, lambda_inv, baseline);
  // CudaCheckError();
  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipStreamSynchronize(stream));
}

template <class DYN_T, int NUM_ROLLOUTS, int SUM_STRIDE>
void launchWeightedReductionKernel(float* exp_costs_d, float* du_d, float* du_new_d, float normalizer,
                                   int num_timesteps, hipStream_t stream)
{
  dim3 dimBlock((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1, 1, 1);
  dim3 dimGrid(num_timesteps, 1, 1);
  weightedReductionKernel<DYN_T::CONTROL_DIM, NUM_ROLLOUTS, SUM_STRIDE>
      <<<dimGrid, dimBlock, 0, stream>>>(exp_costs_d, du_d, du_new_d, normalizer, num_timesteps);
  CudaCheckError();
  HANDLE_ERROR(hipStreamSynchronize(stream));
}

template <class DYN_T, class COST_T, class FB_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z = 1>
void launchStateAndCostTrajectoryKernel(DYN_T* dynamics, COST_T* cost, FB_T* fb_controller, float* control_trajectories,
                                        float* state, float* state_traj_result, float* cost_traj_result,
                                        int* crash_status_result, int num_rollouts, int num_timesteps, float dt,
                                        hipStream_t stream, float value_func_threshold = -1)
{
  const int gridsize_x = (num_rollouts - 1) / BLOCKSIZE_X + 1;
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(gridsize_x, 1, 1);
  stateAndCostTrajectoryKernel<DYN_T, COST_T, FB_T, BLOCKSIZE_X, BLOCKSIZE_Z><<<dimGrid, dimBlock, 0, stream>>>(
      dynamics, cost, fb_controller, control_trajectories, state, state_traj_result, cost_traj_result,
      crash_status_result, num_rollouts, num_timesteps, dt, value_func_threshold);
}
}  // namespace mppi_common

namespace rmppi_kernels
{
template <class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int SAMPLES_PER_CONDITION>
__global__ void initEvalKernel(DYN_T* dynamics, COST_T* costs, int num_timesteps, float lambda, float alpha,
                               int ctrl_stride, float dt, int* strides_d, float* exploration_std_dev_d, float* states_d,
                               float* control_d, float* control_noise_d, float* costs_d)
{
  int i, j;
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  int bdx = blockIdx.x;

  // Initialize the local state, controls, and noise
  float* state;
  float* state_der;
  float* control;
  float* control_noise;  // du
  int* crash_status;

  // Create shared arrays for holding state and control data.
  __shared__ float state_shared[BLOCKSIZE_X * DYN_T::STATE_DIM];
  __shared__ float state_der_shared[BLOCKSIZE_X * DYN_T::STATE_DIM];
  __shared__ float control_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM];
  __shared__ float control_noise_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM];
  __shared__ float exploration_std_dev[DYN_T::CONTROL_DIM];  // Each thread only reads
  __shared__ int crash_status_shared[BLOCKSIZE_X];

  // Create a shared array for the dynamics model to use
  __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD + DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X];

  float running_cost = 0;  // Initialize trajectory cost

  int global_idx = BLOCKSIZE_X * bdx + tdx;                // Set the global index for CUDA threads
  int condition_idx = global_idx / SAMPLES_PER_CONDITION;  // Set the index for our candidate
  int stride = strides_d[condition_idx];                   // Each candidate can have a different starting stride

  // Get the pointer that belongs to the current thread with respect to the shared arrays
  state = &state_shared[tdx * DYN_T::STATE_DIM];
  state_der = &state_der_shared[tdx * DYN_T::STATE_DIM];
  control = &control_shared[tdx * DYN_T::CONTROL_DIM];
  control_noise = &control_noise_shared[tdx * DYN_T::CONTROL_DIM];
  crash_status = &crash_status_shared[tdx];
  crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.

  // Copy the state to the thread
  for (i = tdy; i < DYN_T::STATE_DIM; i += blockDim.y)
  {
    state[i] = states_d[condition_idx * DYN_T::STATE_DIM + i];  // states_d holds each condition
  }

  // Copy the exploration noise std_dev to the thread
  for (i = tdy; i < DYN_T::CONTROL_DIM; i += blockDim.y)
  {
    control[i] = 0.0;
    control_noise[i] = 0.0;
    exploration_std_dev[i] = exploration_std_dev_d[i];
  }

  __syncthreads();
  dynamics->initializeDynamics(state, control, theta_s, 0.0, dt);
  for (i = 0; i < num_timesteps; ++i)
  {  // Outer loop iterates on timesteps
    // Inject the control noise
    for (j = tdy; j < DYN_T::CONTROL_DIM; j += blockDim.y)
    {
      if ((i + stride) >= num_timesteps)
      {  // Pad the end of the controls with the last control
        control[j] = control_d[(num_timesteps - 1) * DYN_T::CONTROL_DIM + j];
      }
      else
      {
        control[j] = control_d[(i + stride) * DYN_T::CONTROL_DIM + j];
      }

      // First rollout is noise free
      if (global_idx % SAMPLES_PER_CONDITION == 0 || i < ctrl_stride)
      {
        control_noise[j] = 0.0;
      }
      else
      {
        control_noise[j] =
            control_noise_d[num_timesteps * DYN_T::CONTROL_DIM * global_idx + i * DYN_T::CONTROL_DIM + j] *
            exploration_std_dev[j];
      }

      // Sum the control and the noise
      control[j] += control_noise[j];
    }  // End inject control noise

    __syncthreads();

    dynamics->enforceConstraints(state, control);
    __syncthreads();
    if (tdy == 0 && i > 0)
    {  // Only compute once per global index, make sure that we don't divide by zero
      running_cost += (costs->computeRunningCost(state, control, control_noise, exploration_std_dev, lambda, alpha, i,
                                                 crash_status) -
                       running_cost) /
                      (1.0 * i);
    }
    __syncthreads();

    // Compute state derivatives
    dynamics->computeStateDeriv(state, control, state_der, theta_s);
    __syncthreads();

    // Increment states
    dynamics->updateState(state, state_der, dt);
    __syncthreads();
  }
  // End loop outer loop on timesteps

  if (tdy == 0)
  {  // Only save the costs once per global idx (thread y is only for parallelization)
    costs_d[global_idx] = running_cost;  // This is the running average of the costs along the trajectory
  }
}

template <class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int SAMPLES_PER_CONDITION>
void launchInitEvalKernel(DYN_T* dynamics, COST_T* costs, int num_candidates, int num_timesteps, float lambda,
                          float alpha, int ctrl_stride, float dt, int* strides_d, float* exploration_std_dev_d,
                          float* states_d, float* control_d, float* control_noise_d, float* costs_d,
                          hipStream_t stream)
{
  int GRIDSIZE_X = num_candidates * SAMPLES_PER_CONDITION / BLOCKSIZE_X;
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
  dim3 dimGrid(GRIDSIZE_X, 1, 1);
  initEvalKernel<DYN_T, COST_T, BLOCKSIZE_X, BLOCKSIZE_Y, SAMPLES_PER_CONDITION>
      <<<dimGrid, dimBlock, 0, stream>>>(dynamics, costs, num_timesteps, lambda, alpha, ctrl_stride, dt, strides_d,
                                         exploration_std_dev_d, states_d, control_d, control_noise_d, costs_d);
}

// Newly Written
template <class DYN_T, class COST_T, class FB_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int NUM_ROLLOUTS, int BLOCKSIZE_Z>
__global__ void RMPPIRolloutKernel(DYN_T* dynamics, COST_T* costs, FB_T* fb_controller, float dt, int num_timesteps,
                                   int optimization_stride, float lambda, float alpha, float value_func_threshold,
                                   float* x_d, float* u_d, float* du_d, float* sigma_u_d, float* trajectory_costs_d)
{
  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

  // Create shared memory for state and control
  __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float du_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float sigma_u[DYN_T::CONTROL_DIM];

  // Create a shared array for the nominal costs calculations
  __shared__ float running_state_cost_nom_shared[BLOCKSIZE_X];
  __shared__ float running_control_cost_nom_shared[BLOCKSIZE_X];
  __shared__ int crash_status_shared[BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the dynamics model to use
  __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD + DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the feedback controller to use
  __shared__ float theta_fb[FB_T::SHARED_MEM_SIZE];

  // Create local state, state dot and controls
  float* x;
  float* x_other;
  float* xdot;
  float* u;
  float* du;
  int* crash_status;
  // The array to hold K(x,x*)
  float fb_control[DYN_T::CONTROL_DIM];

  int t = 0;
  int i = 0;
  // int j = 0;

  // Initialize running costs
  float running_state_cost_real = 0;
  float running_control_cost_real = 0;
  float* running_state_cost_nom;
  float running_tracking_cost_real = 0;
  float* running_control_cost_nom;

  // Load global array into shared memory
  if (global_idx < NUM_ROLLOUTS)
  {
    // Actual or nominal
    x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    // The opposite state from above
    x_other = &x_shared[(blockDim.x * (1 - thread_idz) + thread_idx) * DYN_T::STATE_DIM];
    xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    // Base trajectory
    u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    // Noise added to trajectory
    du = &du_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    // Nominal State Cost
    running_state_cost_nom = &running_state_cost_nom_shared[thread_idx];
    running_control_cost_nom = &running_control_cost_nom_shared[thread_idx];
    crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
    crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.

    // Load memory into appropriate arrays
    mppi_common::loadGlobalToShared(DYN_T::STATE_DIM, DYN_T::CONTROL_DIM, NUM_ROLLOUTS, BLOCKSIZE_Y, global_idx,
                                    thread_idy, thread_idz, x_d, sigma_u_d, x, xdot, u, du, sigma_u);
    __syncthreads();
    *running_state_cost_nom = 0;
    *running_control_cost_nom = 0;
    float curr_state_cost = 0.0;
    dynamics->initializeDynamics(x, u, theta_s, 0.0, dt);
    for (t = 0; t < num_timesteps; t++)
    {
      mppi_common::injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps, t, global_idx,
                                      thread_idy, optimization_stride, u_d, du_d, sigma_u, u, du);
      __syncthreads();

      // Now find feedback control
      for (i = 0; i < DYN_T::CONTROL_DIM; i++)
      {
        fb_control[i] = 0;
      }

      // we do not apply feedback on the nominal state z == 1
      if (thread_idz == 0)
      {
        fb_controller->k(x, x_other, t, theta_fb, fb_control);
      }

      for (i = thread_idy; i < DYN_T::CONTROL_DIM; i += BLOCKSIZE_Y)
      {
        u[i] += fb_control[i];
        // Make sure feedback is added to the modified control noise pointer
        // du_d[control_index + i] += fb_control[i];
      }

      __syncthreads();
      // Clamp the control in both the importance sampling sequence and the disturbed sequence.
      dynamics->enforceConstraints(x, u);

      __syncthreads();
      // Calculate All the costs
      if (t > 0)
      {
        curr_state_cost = costs->computeStateCost(x, t, crash_status);
      }

      // Nominal system is where thread_idz == 1
      if (thread_idz == 1 && thread_idy == 0 && t > 0)
      {
        // This memory is shared in the y direction so limit which threads can write to it
        *running_state_cost_nom += curr_state_cost;
        *running_control_cost_nom += costs->computeLikelihoodRatioCost(u, du, sigma_u, lambda, alpha);
      }
      // Real system cost update when thread_idz == 0
      if (thread_idz == 0 && t > 0)
      {
        running_state_cost_real += curr_state_cost;
        running_control_cost_real += costs->computeLikelihoodRatioCost(u, du, sigma_u, lambda, alpha);

        running_tracking_cost_real +=
            (curr_state_cost + costs->computeFeedbackCost(fb_control, sigma_u, lambda, alpha));
      }

      //        if (global_idx == 29 && thread_idy == 0 && thread_idz == 0 && t > 0) {
      //          printf("RMPPI Current state real: [%f, %f, %f, %f]\n", x[0], x[1], x[2], x[3]);
      //          printf("RMPPI Current state cost real: [%f]\n",
      //          (running_state_cost_real+running_control_cost_real)/t);
      //        }
      __syncthreads();
      // dynamics update
      dynamics->computeStateDeriv(x, u, xdot, theta_s);
      __syncthreads();
      dynamics->updateState(x, xdot, dt);
      __syncthreads();
    }

    // Compute average cost per timestep
    if (thread_idz == 1 && thread_idy == 0)
    {
      *running_state_cost_nom /= ((float)num_timesteps - 1);
      *running_control_cost_nom /= ((float)num_timesteps - 1);
    }

    if (thread_idz == 0)
    {
      running_state_cost_real /= ((float)num_timesteps - 1);
      running_tracking_cost_real /= ((float)num_timesteps - 1);
      running_control_cost_real /= ((float)num_timesteps - 1);
    }

    // calculate terminal costs
    if (thread_idz == 1 && thread_idy == 0)
    {  // Thread y required to prevent double addition
      *running_state_cost_nom += costs->terminalCost(x);
    }

    if (thread_idz == 0)
    {
      running_state_cost_real += costs->terminalCost(x);
      running_tracking_cost_real += costs->terminalCost(x);
    }

    // Figure out final form of nominal cost
    float running_cost_nom = 0;
    if (thread_idz == 0)
    {
      running_cost_nom = 0.5 * (*running_state_cost_nom) +
                         0.5 * fmaxf(fminf(running_tracking_cost_real, value_func_threshold), *running_state_cost_nom);

      running_cost_nom += *running_control_cost_nom;

      // Copy costs over to global memory
      // Actual System cost
      trajectory_costs_d[global_idx] = running_state_cost_real + running_control_cost_real;
      // Nominal System Cost - Again this is actaully only  known on real system threads
      trajectory_costs_d[global_idx + NUM_ROLLOUTS] = running_cost_nom;
    }
  }
  __syncthreads();
}

/*******************************************************************************************************************
 * Launch Functions
 *******************************************************************************************************************/
template <class DYN_T, class COST_T, class FB_T, int NUM_ROLLOUTS, int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
void launchRMPPIRolloutKernel(DYN_T* dynamics, COST_T* costs, FB_T* fb_controller, float dt, int num_timesteps,
                              int optimization_stride, float lambda, float alpha, float value_func_threshold,
                              float* x_d, float* u_d, float* du_d, float* sigma_u_d, float* trajectory_costs,
                              hipStream_t stream)
{
  const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(gridsize_x, 1, 1);
  RMPPIRolloutKernel<DYN_T, COST_T, FB_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS, BLOCKSIZE_Z>
      <<<dimGrid, dimBlock, 0, stream>>>(dynamics, costs, fb_controller, dt, num_timesteps, optimization_stride, lambda,
                                         alpha, value_func_threshold, x_d, u_d, du_d, sigma_u_d, trajectory_costs);
  CudaCheckError();
  HANDLE_ERROR(hipStreamSynchronize(stream));
}
}  // namespace rmppi_kernels
