#include "hip/hip_runtime.h"
#include <mppi/core/mppi_common.cuh>
#include <hiprand.h>
#include <mppi/utils/gpu_err_chk.cuh>

namespace mppi_common {
  /*******************************************************************************************************************
  * Kernel Functions
  *******************************************************************************************************************/
  // TODO remove dt
  template<class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y,
         int NUM_ROLLOUTS, int BLOCKSIZE_Z>
  __global__ void rolloutKernel(DYN_T* dynamics, COST_T* costs,
                               float dt,
                               int num_timesteps,
                               float* x_d,
                               float* u_d,
                               float* du_d,
                               float* sigma_u_d,
                               float* trajectory_costs_d) {
    // Get thread and block id
    int thread_idx = threadIdx.x;
    int thread_idy = threadIdx.y;
    int thread_idz = threadIdx.z;
    int block_idx = blockIdx.x;
    int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

    // Create shared state and control arrays
    __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
    __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
    __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
    __shared__ float du_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
    __shared__ float sigma_u[DYN_T::CONTROL_DIM];

    // Create a shared array for the dynamics model to use
    __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD + DYN_T::SHARED_MEM_REQUEST_BLK*BLOCKSIZE_X];

    // Create local state, state dot and controls
    float* x;
    float* xdot;
    float* u;
    float* du;
    // float* sigma_u;

    //Initialize running cost and total cost
    float running_cost = 0;
    //Load global array to shared array
    if (global_idx < NUM_ROLLOUTS) {
      x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
      xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
      u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
      du = &du_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
      // sigma_u = &sigma_u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    }
    __syncthreads();
    loadGlobalToShared(DYN_T::STATE_DIM, DYN_T::CONTROL_DIM, NUM_ROLLOUTS,
                       BLOCKSIZE_Y, global_idx, thread_idy,
                       thread_idz, x_d, sigma_u_d, x, xdot, u, du, sigma_u);
    __syncthreads();


    if (global_idx < NUM_ROLLOUTS) {
      /*<----Start of simulation loop-----> */
      for (int t = 0; t < num_timesteps; t++) {
        //Load noise trajectories scaled by the exploration factor
        injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps,
                          t, global_idx, thread_idy, u_d, du_d, sigma_u, u, du);
        __syncthreads();

        // applies constraints as defined in dynamics.cuh see specific dynamics class for what happens here
        // usually just control clamping
        // Clamp the control in both the importance sampling sequence and the disturbed sequence. TODO remove extraneous call?
        dynamics->enforceConstraints(x, &du_d[global_idx*num_timesteps*DYN_T::CONTROL_DIM + t*DYN_T::CONTROL_DIM]);
        dynamics->enforceConstraints(x, u);

        __syncthreads();

        //Accumulate running cost
        running_cost += costs->computeRunningCost(x, u, du, sigma_u, t)*dt;
        __syncthreads();

        //Compute state derivatives
        dynamics->computeStateDeriv(x, u, xdot, theta_s);
        __syncthreads();

        //Increment states
        dynamics->updateState(x, xdot, dt);
        __syncthreads();
      }
      //Compute terminal cost and the final cost for each thread
      computeAndSaveCost(NUM_ROLLOUTS, global_idx, costs, x, running_cost,
                        trajectory_costs_d + thread_idz * NUM_ROLLOUTS);
    }

    __syncthreads();
  }

  __global__ void normExpKernel(int num_rollouts,
                                float* trajectory_costs_d,
                                float gamma,
                                float baseline) {
    int global_idx = (blockDim.x * blockIdx.x + threadIdx.x) * blockDim.z + \
      threadIdx.z;

    if (global_idx < num_rollouts * blockDim.z) {
      float cost_dif = trajectory_costs_d[global_idx] - baseline;
      trajectory_costs_d[global_idx] = expf(-gamma*cost_dif);
    }
  }

  template<int CONTROL_DIM, int NUM_ROLLOUTS, int SUM_STRIDE>
  __global__ void weightedReductionKernel(float*  exp_costs_d,
                                          float* du_d,
                                          float* du_new_d,
                                          float normalizer,
                                          int num_timesteps) {
    int thread_idx = threadIdx.x;  // Rollout index
    int block_idx = blockIdx.x; // Timestep

    //Create a shared array for intermediate sums: CONTROL_DIM x NUM_THREADS
    __shared__ float u_intermediate[CONTROL_DIM * ((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1)];

    float u[CONTROL_DIM];
    setInitialControlToZero(CONTROL_DIM, thread_idx, u, u_intermediate);

    __syncthreads();

    //Sum the weighted control variations at a desired stride
    strideControlWeightReduction(NUM_ROLLOUTS, num_timesteps, SUM_STRIDE,
                                 thread_idx,
                                 block_idx, CONTROL_DIM,
                                 exp_costs_d, normalizer,
                                 du_d, u, u_intermediate);

    __syncthreads();

    //Sum all weighted control variations
    rolloutWeightReductionAndSaveControl(thread_idx, block_idx,
                                         NUM_ROLLOUTS, num_timesteps,
                                         CONTROL_DIM, SUM_STRIDE,
                                         u, u_intermediate, du_new_d);

    __syncthreads();
  }


    /*******************************************************************************************************************
     * Rollout Kernel Helpers
    *******************************************************************************************************************/
    __device__ void loadGlobalToShared(int state_dim, int control_dim,
                                       int num_rollouts, int blocksize_y,
                                       int global_idx, int thread_idy,
                                       int thread_idz,
                                       const float* x_device,
                                       const float* sigma_u_device,
                                       float* x_thread,
                                       float* xdot_thread,
                                       float* u_thread,
                                       float* du_thread,
                                       float* sigma_u_thread) {
      //Transfer to shared memory
      int i;
      if (global_idx < num_rollouts) {
        for (i = thread_idy; i < state_dim; i += blocksize_y) {
          x_thread[i] = x_device[i + state_dim * thread_idz];
          xdot_thread[i] = 0;
        }
        for (i = thread_idy; i < control_dim; i += blocksize_y) {
          u_thread[i] = 0;
          du_thread[i] = 0;
          // Only do in threadIdx.x and parallelize along threadIdx.y
          // sigma_u_thread[i] = sigma_u_device[i];
        }
      }
      if (threadIdx.x == 0 /*&& threadIdx.z == 0*/) {
        for(i = thread_idy; i < control_dim; i +=blocksize_y){
          sigma_u_thread[i] = sigma_u_device[i];
        }
      }
      // for (i = blockDim.y*blockDim.x*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x; i < control_dim; i+= blockDim.z*blockDim.x*blockDim.y){

      // }
    }

    __device__ void injectControlNoise(int control_dim,
                                       int blocksize_y, int num_rollouts,
                                       int num_timesteps,
                                       int current_timestep,
                                       int global_idx,
                                       int thread_idy,
                                       const float* u_traj_device,
                                       float* ep_v_device,
                                       const float* sigma_u_thread,
                                       float* u_thread, float* du_thread) {
        int control_index = global_idx*control_dim*num_timesteps + current_timestep * control_dim;
        //Load the noise trajectory scaled by the exploration factor
        // The prior loop already guarantees that the global index is less than the number of rollouts
        for (int i = thread_idy; i < control_dim; i += blocksize_y) {
            //Keep one noise free trajectory
            if (global_idx == 0){
                du_thread[i] = 0;
                u_thread[i] = u_traj_device[current_timestep * control_dim + i];
            }
            //Generate 1% zero control trajectory
            else if (global_idx >= 0.99*num_rollouts) {
                du_thread[i] = ep_v_device[control_index + i] * sigma_u_thread[i];
                u_thread[i] = du_thread[i];
            }
            else {
                du_thread[i] = ep_v_device[control_index + i] * sigma_u_thread[i];
                u_thread[i] = u_traj_device[current_timestep * control_dim + i] + du_thread[i];
            }
            // Saves the control but doesn't clamp it.
            ep_v_device[control_index + i] = u_thread[i];
        }
    }

    template<class COST_T>
    __device__ void computeAndSaveCost(int num_rollouts, int global_idx, COST_T* costs, float* x_thread,
                                        float running_cost, float* cost_rollouts_device) {
        if (global_idx < num_rollouts) {
            cost_rollouts_device[global_idx] = running_cost + costs->terminalCost(x_thread);
        }
    }

    /*******************************************************************************************************************
     * NormExp Kernel Helpers
    *******************************************************************************************************************/
    float computeBaselineCost(float* cost_rollouts_host, int num_rollouts) { // TODO if we use standard containers in MPPI, should this be replaced with a min algorithm?
        float baseline = cost_rollouts_host[0];
        // Find the minimum cost trajectory
        for (int i = 0; i < num_rollouts; ++i) {
            if (cost_rollouts_host[i] < baseline) {
                baseline = cost_rollouts_host[i];
            }
        }
        return baseline;
    }

    float computeNormalizer(float* cost_rollouts_host, int num_rollouts) {
        float normalizer = 0.f;
        for (int i = 0; i < num_rollouts; ++i) {
            normalizer += cost_rollouts_host[i];
        }
        return normalizer;
    }

    /*******************************************************************************************************************
     * Weighted Reduction Kernel Helpers
    *******************************************************************************************************************/
    __device__ void setInitialControlToZero(int control_dim, int thread_idx, float* u, float* u_intermediate) {
        // TODO replace with memset?
        for (int i = 0; i < control_dim; i++) {
            u[i] = 0;
            u_intermediate[thread_idx * control_dim + i] = 0;
        }
    }

    __device__ void strideControlWeightReduction(int num_rollouts,
                                                 int num_timesteps,
                                                 int sum_stride,
                                                 int thread_idx,
                                                 int block_idx,
                                                 int control_dim,
                                                 float* exp_costs_d,
                                                 float normalizer,
                                                 float* du_d,
                                                 float* u,
                                                 float* u_intermediate) {
        // int index = thread_idx * sum_stride + i;
        for (int i = 0; i < sum_stride; ++i) { // Iterate through the size of the subsection
            if ((thread_idx * sum_stride + i) < num_rollouts) { //Ensure we do not go out of bounds
                float weight = exp_costs_d[thread_idx * sum_stride + i] / normalizer; // compute the importance sampling weight
                for (int j = 0; j < control_dim; ++j) { // Iterate through the control dimensions
                    // Rollout index: (thread_idx*sum_stride + i)*(num_timesteps*control_dim)
                    // Current timestep: block_idx*control_dim
                    u[j] = du_d[(thread_idx * sum_stride + i)*(num_timesteps*control_dim) + block_idx*control_dim + j];
                    u_intermediate[thread_idx * control_dim + j] += weight * u[j];
                }
            }
        }
    }

    __device__ void rolloutWeightReductionAndSaveControl(int thread_idx, int block_idx, int num_rollouts, int num_timesteps,
            int control_dim, int sum_stride, float* u, float* u_intermediate, float* du_new_d) {
        if (thread_idx == 0 && block_idx < num_timesteps) { //block index refers to the current timestep
            for (int i = 0; i < control_dim; ++i) { // TODO replace with memset?
                u[i] = 0;
            }
            for (int i = 0; i < ((num_rollouts - 1) / sum_stride + 1); ++i) { // iterate through the each subsection
                for (int j = 0; j < control_dim; ++j) {
                    u[j] += u_intermediate[i * control_dim + j];
                }
            }
            for (int i = 0; i < control_dim; i++) {
                du_new_d[block_idx * control_dim + i] = u[i];
            }
        }
    }

    /*******************************************************************************************************************
     * Launch Functions
    *******************************************************************************************************************/
    template<class DYN_T, class COST_T, int NUM_ROLLOUTS, int BLOCKSIZE_X,
             int BLOCKSIZE_Y, int BLOCKSIZE_Z = 1>
    void launchRolloutKernel(DYN_T* dynamics, COST_T* costs, float dt,
                             int num_timesteps, float* x_d, float* u_d,
                             float* du_d, float* sigma_u_d,
                             float* trajectory_costs, hipStream_t stream) {
      const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
      dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
      dim3 dimGrid(gridsize_x, 1, 1);
      rolloutKernel<DYN_T, COST_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS, BLOCKSIZE_Z><<<dimGrid, dimBlock, 0, stream>>>(dynamics, costs, dt,
              num_timesteps, x_d, u_d, du_d, sigma_u_d, trajectory_costs);
      CudaCheckError();
      HANDLE_ERROR( hipStreamSynchronize(stream) );
    }

    void launchNormExpKernel(int num_rollouts, int blocksize_x, float* trajectory_costs_d, float gamma, float baseline, hipStream_t stream) {
        dim3 dimBlock(blocksize_x, 1, 1);
        dim3 dimGrid((num_rollouts - 1) / blocksize_x + 1, 1, 1);
        normExpKernel<<<dimGrid, dimBlock, 0, stream>>>(num_rollouts, trajectory_costs_d, gamma, baseline);
        CudaCheckError();
        HANDLE_ERROR( hipStreamSynchronize(stream) );
    }

    template<class DYN_T, int NUM_ROLLOUTS, int SUM_STRIDE >
    void launchWeightedReductionKernel(float* exp_costs_d, float* du_d, float* du_new_d, float normalizer,
            int num_timesteps, hipStream_t stream) {
        dim3 dimBlock((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1, 1, 1);
        dim3 dimGrid(num_timesteps, 1, 1);
        weightedReductionKernel<DYN_T::CONTROL_DIM, NUM_ROLLOUTS, SUM_STRIDE><<<dimGrid, dimBlock, 0, stream>>>
                (exp_costs_d, du_d, du_new_d, normalizer, num_timesteps);
        CudaCheckError();
        HANDLE_ERROR( hipStreamSynchronize(stream) );
    }

}

namespace rmppi_kernels {
  template <class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y>
  __global__ void initEvalKernel(DYN_T* dynamics,
                                 COST_T* costs,
                                 int samples_per_condition,
                                 int num_timesteps,
                                 int ctrl_stride,
                                 float dt,
                                 int* strides_d,
                                 float* exploration_var_d,
                                 float* states_d,
                                 float* control_d,
                                 float* control_noise_d,
                                 float* costs_d) {
    int i,j;
    int tdx = threadIdx.x;
    int tdy = threadIdx.y;
    int bdx = blockIdx.x;

    //Initialize the local state, controls, and noise
    float* state;
    float* state_der;
    float* control;
    float* control_noise;  // du
    float* exploration_var;  //nu


    //Create shared arrays for holding state and control data.
    __shared__ float state_shared[BLOCKSIZE_X*DYN_T::STATE_DIM];
    __shared__ float state_der_shared[BLOCKSIZE_X*DYN_T::STATE_DIM];
    __shared__ float control_shared[BLOCKSIZE_X*DYN_T::CONTROL_DIM];
    __shared__ float control_noise_shared[BLOCKSIZE_X*DYN_T::CONTROL_DIM];
    __shared__ float exploration_variance[BLOCKSIZE_X*DYN_T::CONTROL_DIM]; // Each thread has its own copy

    //Create a shared array for the dynamics model to use
    __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD + DYN_T::SHARED_MEM_REQUEST_BLK*BLOCKSIZE_X];


    float running_cost = 0;  //Initialize trajectory cost

    int global_idx = BLOCKSIZE_X*bdx + tdx;  // Set the global index for CUDA threads
    int condition_idx = global_idx / samples_per_condition; // Set the index for our candidate
    int stride = strides_d[condition_idx];  // Each candidate can have a different starting stride

    // Get the pointer that belongs to the current thread with respect to the shared arrays
    state = &state_shared[tdx*DYN_T::STATE_DIM];
    state_der = &state_der_shared[tdx*DYN_T::STATE_DIM];
    control = &control[tdx*DYN_T::CONTROL_DIM];
    control_noise = &control_noise_shared[tdx*DYN_T::CONTROL_DIM];
    exploration_var = &exploration_variance[tdx*DYN_T::CONTROL_DIM];

    // Copy the state to the thread
    for (i = tdy; i < DYN_T::STATE_DIM; i+= blockDim.y) {
      state[i] = states_d[condition_idx*DYN_T::STATE_DIM + i]; // states_d holds each condition
    }

    // Copy the exploration noise to the thread
    for (i = tdy; i < DYN_T::CONTROL_DIM; i += blockDim.y) {
      control[i] = 0;
      control_noise[i] = 0;
      exploration_var[i] = exploration_var_d[i];
    }

    __syncthreads();

    for (i = 0; i < num_timesteps; ++i) { // Outer loop iterates on timesteps
      // Inject the control noise
      for (j = tdy; j < DYN_T::CONTROL_DIM; j += blockDim.y) {
        if (i + stride >= num_timesteps) {  // Pad the end of the controls with the last control
          control[j] = control_d[num_timesteps*DYN_T::CONTROL_DIM + j];
        } else {
          control[j] = control_d[(i + stride)*DYN_T::CONTROL_DIM + j];
        }

        // First rollout is noise free
        if (global_idx % samples_per_condition == 0 || i < ctrl_stride) {
          control_noise[j] = 0.0;
        } else {
          control_noise[j] = control_noise_d[num_timesteps*DYN_T::CONTROL_DIM*global_idx +
                                             i*DYN_T::CONTROL_DIM + j]*exploration_var[j];
        }

        // Sum the control and the noise
        control[j] += control_noise[j];
      } // End inject control noise

      __syncthreads();
      if (tdy == 0) {
        dynamics->enforceConstraints(state, &control_noise_d[num_timesteps*DYN_T::CONTROL_DIM*global_idx +
                                                             i*DYN_T::CONTROL_DIM]);
        dynamics->enforceConstraints(state, control);
      }

      __syncthreads();
      if (tdy == 0) { // Only compute once per global index.
        running_cost +=
                (costs->computeCost(state, control, control_noise, exploration_var, i) * dt - running_cost) / (1.0 * i);
      }
      __syncthreads();

      //Compute state derivatives
      dynamics->computeStateDeriv(state, control, state_der, theta_s);
      __syncthreads();

      //Increment states
      dynamics->updateState(state, state_der, dt);
      __syncthreads();
      }
    // End loop outer loop on timesteps

    if (tdy == 0) {  // Only save the costs once per global idx (thread y is only for parallelization)
      costs_d[global_idx] = running_cost; // This is the running average of the costs along the trajectory
    }
  }

  template<class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y>
  void launchInitEvalKernel(DYN_T* dynamics,
                            COST_T* costs,
                            int samples_per_condition,
                            int num_candidates,
                            int num_timesteps,
                            int ctrl_stride,
                            float dt,
                            int* strides_d,
                            float* exploration_var_d,
                            float* states_d,
                            float* control_d,
                            float* control_noise_d,
                            float* costs_d) {

    int GRIDSIZE_X = num_candidates * samples_per_condition / BLOCKSIZE_X;
    dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    dim3 dimGrid(GRIDSIZE_X, 1, 1);
    initEvalKernel<DYN_T, COST_T, BLOCKSIZE_X, BLOCKSIZE_Y><<<dimGrid, dimBlock, 0>>>(dynamics, costs,
            samples_per_condition, num_timesteps, ctrl_stride, dt, strides_d, exploration_var_d, states_d,
            control_d, control_noise_d, costs_d);

  }

  // Newly Written
  template<class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y,
         int NUM_ROLLOUTS, int BLOCKSIZE_Z>
  __global__ void RMPPIRolloutKernel(DYN_T * dynamics, COST_T* costs,
                                     float dt,
                                     int num_timesteps,
                                     float* x_d,
                                     float* u_d,
                                     float* du_d,
                                     float* feedback_gains_d,
                                     float* sigma_u_d,
                                     float* trajectory_costs_d,
                                     float lambda) {
    int thread_idx = threadIdx.x;
    int thread_idy = threadIdx.y;
    int thread_idz = threadIdx.z;
    int block_idx = blockIdx.x;
    int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

    // Create shared memory for state and control
    __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
    __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
    __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
    __shared__ float du_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
    __shared__ float sigma_u[DYN_T::CONTROL_DIM];

    // Create a shared array for the nominal costs calculations
    __shared__ float running_state_cost_nom_shared[BLOCKSIZE_X];

    // Create a shared array for the dynamics model to use
    __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD + DYN_T::SHARED_MEM_REQUEST_BLK*BLOCKSIZE_X];

    // Create local state, state dot and controls
    float* x;
    float* x_other;
    float* xdot;
    float* u;
    float* du;
    float* fb_gain;
    // The array to hold K(x,x*)
    float* fb_control[DYN_T::CONTROL_DIM];

    int t = 0;
    int i = 0;
    int j = 0;

    // Initialize running costs
    float running_cost_real = 0;
    float* running_state_cost_nom;
    float running_tracking_cost_real = 0;

    // Load global array into shared memory
    if (global_idx < NUM_ROLLOUTS) {
      // Actual or nominal
      x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
      // The opposite state from above
      x_other = &x_shared[(blockDim.x * (1 - thread_idz) + thread_idx) * DYN_T::STATE_DIM];
      xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
      // Base trajectory
      u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
      // Noise added to trajectory
      du = &du_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
      // Nominal State Cost
      running_state_cost_nom = &running_state_cost_nom_shared[thread_idx];
    }

    *running_state_cost_nom = 0;

    __syncthreads();
    // Load memory into appropriate arrays
    loadGlobalToShared(DYN_T::STATE_DIM, DYN_T::CONTROL_DIM, NUM_ROLLOUTS,
                      BLOCKSIZE_Y, global_idx, thread_idy,
                      thread_idz, x_d, sigma_u_d, x, xdot, u, du, sigma_u);
    __syncthreads();
    //TODO: Need to load feedback gains as well
    for (t = 0; t < num_timesteps; t++) {
      injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps,
                        t, global_idx, thread_idy, u_d, du_d, sigma_u, u, du);
      // Now find feedback control
      float e;
      // Feedback gains at time t
      fb_gain = &feedback_gains_d[t * DYN_T::CONTROL_DIM * DYN_T::STATE_DIM];
      for (i = 0; i < DYN_T::CONTROL_DIM; i++) {
        fb_control[i] = 0;
      }
      // Don't enter for loop if in nominal states (thread_idz == 1)
      for (i = 0; i < DYN_T::STATE_DIM * (1 - thread_idz); i++) {
        // Find difference between nominal and actual
        e = (x - x_other);
        for (j = 0; j < DYN_T::CONTROL_DIM; j++) {
          // Assuming column major storage atm. TODO Double check storage option
          fb_control[j] += fb_gain[i * DYN_T::CONTROL_DIM + j] * e;
        }
      }
      for (i = 0; i < DYN_T::CONTROL_DIM; i++) {
        u[i] += fb_control[i];
      }

      __syncthreads();
      // Clamp the control in both the importance sampling sequence and the disturbed sequence. TODO remove extraneous call?
      dynamics->enforceConstraints(x, du);
      dynamics->enforceConstraints(x, u);

      __syncthreads();
      // Calculate All the costs
      float curr_state_cost =  costs->computeStateCost(x);

      // Nominal system is where thread_idz == 1
      if (thread_idz == 1) {
        *running_state_cost_nom += curr_state_cost;
      }
      // Real system cost update when thread_idz == 0
      if (thread_idz == 0) {
        running_cost_real += (curr_state_cost +
          costs->computeLikelihoodRatioCost(u, du, sigma_u, lambda));

        running_tracking_cost_real += (curr_state_cost +
          costs->computeFeedbackCost(fb_control, sigma_u, lambda));
      }

      // Non if statement version
      // running_cost_real += (1 - thread_idz) * (curr_state_cost +
      //   costs->computeLikelihoodRatioCost(u, du, sigma_u, t));
      // running_tracking_cost_real += (1 - thread_idz) * (curr_state_cost +
      //   costs->computeFeedbackCost(fb_control, sigma_u));

      __syncthreads();
      // dynamics update
      dynamics->computeStateDeriv(x, u, xdot, theta_s);
      __syncthreads();
      dynamics->updateState(x, xdot, dt);
      __syncthreads();
    }
    // Choose which cost to use for nominal cost
    // TODO: Replace with parameter passed in
    float value_func_threshold_ = 10;
    /** TODO: This will not work in current setup because running_state_cost_nom
    * and running_tracking_cost_real are calculated by different threads (thread_z 1 or 0)
    * Need to create shared memory for some parts of it
    **/
    float running_cost_nom  = 0;
    if (thread_idz == 0) {
      running_cost_nom = 0.5 * (*running_state_cost_nom) + 0.5 *
        fmaxf(fminf(running_tracking_cost_real, value_func_threshold_), *running_state_cost_nom);

      for(t = 0; t < num_timesteps - 1; t++) {
        // Get u(t) and noise at time t
        injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps,
          t, global_idx, thread_idy, u_d, du_d, sigma_u, u, du);
        __syncthreads();
        running_cost_nom += costs->computeLikelihoodRatioCost(u, du, sigma_u, lambda);
      }
    }
    __syncthreads();
    // Copy costs over to correct locations
    /** TODO: Right now copying can only occur from the real system threads
    * We can leave it as is or we could copy the nominal trajectory costs
    * into a shared memory location so that we could try to use computeAndSaveCost
    */
    if (thread_idz == 0) {
      // Only the threadds running the actual system have the final running costs for both
      // real and nominal
      if (global_idx < NUM_ROLLOUTS) {
        // Actual System cost
        trajectory_costs_d[global_idx] = running_cost_real;
        // Nominal System Cost - Again this is actaully only  known on real system threads
        trajectory_costs_d[global_idx + NUM_ROLLOUTS] = running_cost_nom;
      }
    }
    __syncthreads();
  }

  /*******************************************************************************************************************
   * Launch Functions
   *******************************************************************************************************************/
  template<class DYN_T, class COST_T, int NUM_ROLLOUTS, int BLOCKSIZE_X,
            int BLOCKSIZE_Y, int BLOCKSIZE_Z>
  void launchRMPPIRolloutKernel(DYN_T* dynamics, COST_T* costs,
                                float dt,
                                int num_timesteps,
                                float* x_d,
                                float* u_d,
                                float* du_d,
                                float* feedback_gains_d,
                                float* sigma_u_d,
                                float* trajectory_costs,
                                float lambda,
                                hipStream_t stream) {
    const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
    dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    dim3 dimGrid(gridsize_x, 1, 1);
    RMPPIRolloutKernel<DYN_T, COST_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS,
                      BLOCKSIZE_Z><<<dimGrid, dimBlock, 0, stream>>>(
                        dynamics, costs, dt, num_timesteps, x_d, u_d, du_d,
                        feedback_gains_d, sigma_u_d, trajectory_costs, lambda);
    CudaCheckError();
    HANDLE_ERROR( hipStreamSynchronize(stream) );
  }

}
