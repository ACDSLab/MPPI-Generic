#include "hip/hip_runtime.h"
#include <mppi/core/mppi_common.cuh>
#include <hiprand.h>
#include <mppi/utils/gpu_err_chk.cuh>
#include <mppi/utils/math_utils.h>
#include <mppi/utils/cuda_math_utils.cuh>

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
namespace mp1 = mppi::p1;

namespace mppi_common
{
/*******************************************************************************************************************
 * Kernel Functions
 *******************************************************************************************************************/
// TODO remove dt
template <class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int NUM_ROLLOUTS, int BLOCKSIZE_Z>
__global__ void rolloutKernel(DYN_T* dynamics, COST_T* costs, float dt, int num_timesteps, int optimization_stride,
                              float lambda, float alpha, float* x_d, float* u_d, float* du_d, float* sigma_u_d,
                              float* trajectory_costs_d)
{
  // Get thread and block id
  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

  // Create shared state and control arrays
  __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float x_next_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float y_shared[BLOCKSIZE_X * DYN_T::OUTPUT_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float du_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float sigma_u[DYN_T::CONTROL_DIM];
  __shared__ int crash_status_shared[BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the dynamics model to use
  __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD / sizeof(float) + 1 +
                           DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];
  __shared__ float theta_c[COST_T::SHARED_MEM_REQUEST_GRD + COST_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create local state, state dot and controls
  float* x;
  float* x_next;
  float* x_temp;
  float* y;
  float* xdot;
  float* u;
  float* du;
  int* crash_status;

  // Initialize running cost and total cost
  float running_cost = 0;
  // Load global array to shared array
  if (global_idx < NUM_ROLLOUTS)
  {
    x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    x_next = &x_next_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    y = &y_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::OUTPUT_DIM];
    xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    du = &du_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
    crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.
  }
  loadGlobalToShared<DYN_T::STATE_DIM, DYN_T::CONTROL_DIM>(NUM_ROLLOUTS, BLOCKSIZE_Y, global_idx, thread_idy,
                                                           thread_idz, x_d, sigma_u_d, x, xdot, u, du, sigma_u);
  __syncthreads();

  if (global_idx < NUM_ROLLOUTS)
  {
    /*<----Start of simulation loop-----> */
    dynamics->initializeDynamics(x, u, y, theta_s, 0.0, dt);
    costs->initializeCosts(y, u, theta_c, 0.0, dt);
    __syncthreads();
    for (int t = 0; t < num_timesteps; t++)
    {
      // Load noise trajectories scaled by the exploration factor
      injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps, t, global_idx, thread_idy,
                         optimization_stride, u_d, du_d, sigma_u, u, du);
      // du_d is now v
      __syncthreads();

      // applies constraints as defined in dynamics.cuh see specific dynamics class for what happens here
      // usually just control clamping
      // calls enforceConstraints on both since one is used later on in kernel (u), du_d is what is sent back to the CPU
      dynamics->enforceConstraints(x, &du_d[(NUM_ROLLOUTS * num_timesteps * threadIdx.z +  // z part
                                             global_idx * num_timesteps + t) *
                                            DYN_T::CONTROL_DIM]);
      dynamics->enforceConstraints(x, u);
      __syncthreads();

      // Accumulate running cost
      if (thread_idy == 0 && t > 0)
      {
        running_cost +=
            (costs->computeRunningCost(y, u, du, sigma_u, lambda, alpha, t, theta_c, crash_status) - running_cost) /
            (t);
        // running_cost +=
        //     costs->computeRunningCost(y, u, du, sigma_u, lambda, alpha, t, theta_c, crash_status) / (num_timesteps -
        //     1);
      }

      // Compute state derivatives
      __syncthreads();  // Ensure step doesn't change y while cost above is caclulating
      dynamics->step(x, x_next, xdot, u, y, theta_s, t, dt);
      __syncthreads();
      x_temp = x;
      x = x_next;
      x_next = x_temp;
    }
    // Compute terminal cost and the final cost for each thread
    computeAndSaveCost(NUM_ROLLOUTS, num_timesteps, global_idx, costs, y, running_cost, theta_c, trajectory_costs_d);
  }
}

template <class DYN_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int NUM_ROLLOUTS, int BLOCKSIZE_Z>
__global__ void rolloutDynamicsKernel(DYN_T* __restrict__ dynamics, float dt, int num_timesteps,
                                      int optimization_stride, const float* __restrict__ init_x_d,
                                      const float* __restrict__ u_d, float* __restrict__ du_d,
                                      const float* __restrict__ sigma_u_d, float* __restrict__ y_d)
{
  // Get thread and block id
  const int thread_idx = threadIdx.x;
  const int thread_idy = threadIdx.y;
  const int thread_idz = threadIdx.z;
  const int block_idx = blockIdx.x;
  const int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

  // Create shared state and control arrays
  __shared__ float4 x_shared[mppi::math::int_ceil(BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z, 4)];
  __shared__ float4 x_next_shared[mppi::math::int_ceil(BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z, 4)];
  __shared__ float4 y_shared[mppi::math::int_ceil(BLOCKSIZE_X * DYN_T::OUTPUT_DIM * BLOCKSIZE_Z, 4)];
  __shared__ float4 xdot_shared[mppi::math::int_ceil(BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z, 4)];
  __shared__ float4 u_shared[mppi::math::int_ceil(BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z, 4)];
  __shared__ float4 du_shared[mppi::math::int_ceil(BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z, 4)];
  __shared__ float4 sigma_u[mppi::math::int_ceil(DYN_T::CONTROL_DIM, 4)];

  // Create a shared array for the dynamics model to use
  __shared__ float4 theta_s4[mppi::math::int_ceil(DYN_T::SHARED_MEM_REQUEST_GRD / sizeof(float) + 1 +
                                                      DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z,
                                                  4)];

  float* theta_s = reinterpret_cast<float*>(theta_s4);
  // Create local state, state dot and controls
  float* x;
  float* x_next;
  float* x_temp;
  float* xdot;
  float* u;
  float* du;
  float* y;

  // Load global array to shared array
  if (global_idx < NUM_ROLLOUTS)
  {
    x = &(reinterpret_cast<float*>(x_shared)[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM]);
    x_next = &(reinterpret_cast<float*>(x_next_shared)[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM]);
    y = &(reinterpret_cast<float*>(y_shared)[(blockDim.x * thread_idz + thread_idx) * DYN_T::OUTPUT_DIM]);
    xdot = &(reinterpret_cast<float*>(xdot_shared)[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM]);
    u = &(reinterpret_cast<float*>(u_shared)[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM]);
    du = &(reinterpret_cast<float*>(du_shared)[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM]);
  }
  loadGlobalToShared<DYN_T::STATE_DIM, DYN_T::CONTROL_DIM>(NUM_ROLLOUTS, BLOCKSIZE_Y, global_idx, thread_idy,
                                                           thread_idz, init_x_d, sigma_u_d, x, xdot, u, du,
                                                           reinterpret_cast<float*>(sigma_u));
  __syncthreads();

  if (global_idx < NUM_ROLLOUTS)
  {
    /*<----Start of simulation loop-----> */
    dynamics->initializeDynamics(x, u, y, theta_s, 0.0, dt);
    for (int t = 0; t < num_timesteps; t++)
    {
      // Load noise trajectories scaled by the exploration factor
      injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps, t, global_idx, thread_idy,
                         optimization_stride, u_d, du_d, reinterpret_cast<float*>(sigma_u), u, du);
      // du_d is now v
      __syncthreads();

      // applies constraints as defined in dynamics.cuh see specific dynamics class for what happens here
      // usually just control clamping
      // calls enforceConstraints on both since one is used later on in kernel (u), du_d is what is sent back to the CPU
      dynamics->enforceConstraints(x, &du_d[(NUM_ROLLOUTS * num_timesteps * threadIdx.z +  // z part
                                             global_idx * num_timesteps + t) *
                                            DYN_T::CONTROL_DIM]);
      dynamics->enforceConstraints(x, u);
      __syncthreads();

      // Increment states
      dynamics->step(x, x_next, xdot, u, y, theta_s, t, dt);
      __syncthreads();
      x_temp = x;
      x = x_next;
      x_next = x_temp;
      // Copy state to global memory
      int sample_time_offset = (NUM_ROLLOUTS * thread_idz + global_idx) * num_timesteps + t;
      mp1::loadArrayParallel<DYN_T::OUTPUT_DIM>(y_d, sample_time_offset * DYN_T::OUTPUT_DIM, y, 0);
    }
  }
}

template <class DYN_T, class COST_T, int NUM_ROLLOUTS, int BLOCKSIZE_X, bool COALESCE = false>
__global__ void rolloutCostKernel(DYN_T* dynamics, COST_T* costs, float dt, const int num_timesteps, float lambda,
                                  float alpha, const float* __restrict__ init_x_d, const float* __restrict__ u_d,
                                  const float* __restrict__ du_d, const float* __restrict__ sigma_u_d,
                                  const float* __restrict__ y_d, float* __restrict__ trajectory_costs_d)
{
  // Get thread and block id
  const int thread_idx = threadIdx.x;
  const int thread_idy = threadIdx.y;
  const int thread_idz = threadIdx.z;
  const int global_idx = blockIdx.x;

  // Create shared state and control arrays
  extern __shared__ float entire_buffer[];
  float* y_shared = entire_buffer;
  float* u_shared = &y_shared[blockDim.x * blockDim.z * DYN_T::OUTPUT_DIM];
  float* du_shared = &u_shared[blockDim.x * blockDim.z * DYN_T::CONTROL_DIM];
  float* sigma_u = &du_shared[blockDim.x * blockDim.z * DYN_T::CONTROL_DIM];
  float* running_cost_shared = &sigma_u[DYN_T::CONTROL_DIM];
  int* crash_status_shared = (int*)&running_cost_shared[blockDim.x * blockDim.z];
  float* theta_c = (float*)&crash_status_shared[blockDim.x * blockDim.z];

  // Create local state, state dot and controls
  float* y;
  float* u;
  float* du;
  int* crash_status;

  // Initialize running cost and total cost
  float* running_cost;
  int sample_time_offset = 0;
  int j = 0;

  // Load global array to shared array
  y = &y_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::OUTPUT_DIM];
  u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
  du = &du_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
  crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
  crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.
  running_cost = &running_cost_shared[thread_idz * blockDim.x + thread_idx];
  running_cost[0] = 0;
  if (thread_idx == 0)
  {
    mp1::loadArrayParallel<DYN_T::CONTROL_DIM>(sigma_u, 0, sigma_u_d, 0);
  }

  /*<----Start of simulation loop-----> */
  const int max_time_iters = ceilf((float)num_timesteps / BLOCKSIZE_X);
  costs->initializeCosts(y, u, theta_c, 0.0, dt);
  __syncthreads();
  for (int time_iter = 0; time_iter < max_time_iters; ++time_iter)
  {
    int t = thread_idx + time_iter * blockDim.x + 1;
    if (t <= num_timesteps)
    {  // t = num_timesteps is the terminal state for outside this for-loop
      if (COALESCE)
      {  // Fill entire shared mem sequentially using sequential threads_idx
        mp1::loadArrayParallel<DYN_T::OUTPUT_DIM * BLOCKSIZE_X, mp1::Parallel1Dir::THREAD_X>(
            y_shared, blockDim.x * thread_idz, y_d,
            ((NUM_ROLLOUTS * thread_idz + global_idx) * num_timesteps + time_iter * blockDim.x) * DYN_T::OUTPUT_DIM);
      }
      else
      {
        sample_time_offset = (NUM_ROLLOUTS * thread_idz + global_idx) * num_timesteps + t - 1;
        mp1::loadArrayParallel<DYN_T::OUTPUT_DIM>(y, 0, y_d, sample_time_offset * DYN_T::OUTPUT_DIM);
      }
    }
    if (t < num_timesteps)
    {  // load controls from t = 0 to t = num_timesteps - 1
      // Have to do similar steps as injectControlNoise but using the already transformed cost samples
      readControlsFromGlobal(DYN_T::CONTROL_DIM, blockDim.y, NUM_ROLLOUTS, num_timesteps, t, global_idx, thread_idy,
                             u_d, du_d, u, du);
    }
    __syncthreads();

    // dynamics->enforceConstraints(x, u);
    // __syncthreads();
    // Compute cost
    if (thread_idy == 0 && t < num_timesteps)
    {
      running_cost[0] += costs->computeRunningCost(y, u, du, sigma_u, lambda, alpha, t, theta_c, crash_status);
    }
    __syncthreads();
  }

  // Add all costs together
  int prev_size = BLOCKSIZE_X;
  running_cost = &running_cost_shared[blockDim.x * thread_idz];
#pragma unroll
  for (int size = prev_size / 2; size > 0; size /= 2)
  {
    if (thread_idy == 0)
    {
      for (j = thread_idx; j < size; j += blockDim.x)
      {
        running_cost[j] += running_cost[j + size];
      }
    }
    __syncthreads();
    if (prev_size - 2 * size == 1 && threadIdx.x == blockDim.x - 1 && thread_idy == 0)
    {
      running_cost[size - 1] += running_cost[prev_size - 1];
    }
    __syncthreads();
    prev_size = size;
  }
  __syncthreads();
  // point every thread to the last output at t = NUM_TIMESTEPS for terminal cost calculation
  const int last_y_index = (num_timesteps - 1) % BLOCKSIZE_X;
  y = &y_shared[(blockDim.x * thread_idz + last_y_index) * DYN_T::OUTPUT_DIM];
  // Compute terminal cost and the final cost for each thread
  computeAndSaveCost(NUM_ROLLOUTS, num_timesteps, global_idx, costs, y, running_cost[0] / (num_timesteps - 1), theta_c,
                     trajectory_costs_d);
}

__global__ void normExpKernel(int num_rollouts, float* trajectory_costs_d, float lambda_inv, float baseline)
{
  int global_idx = (blockDim.x * blockIdx.x + threadIdx.x) * blockDim.z + threadIdx.z;
  int global_step = blockDim.x * gridDim.x * blockDim.z * gridDim.z;
  // #if defined(CUDA_VERSION) && CUDA_VERSION > 11060
  //   auto block = cg::this_grid();
  //   int global_idx_b = block.thread_rank() + block.block_rank() * block.num_threads();
  //   int global_step_b = block.num_threads() * block.num_blocks();
  //   if (global_idx == 200 && threadIdx.y == 0 && threadIdx.z == 0)
  //   {
  //     printf("Global ind: %d, thread_rank: %d\n", global_idx, global_idx_b);
  //     printf("Global step: %d, thread_rank: %d\n", global_step, global_step_b);
  //   }
  // #endif
  normExpTransform(num_rollouts * blockDim.z, trajectory_costs_d, lambda_inv, baseline, global_idx, global_step);
}

__global__ void TsallisKernel(int num_rollouts, float* trajectory_costs_d, float gamma, float r, float baseline)
{
  int global_idx = (blockDim.x * blockIdx.x + threadIdx.x) * blockDim.z + threadIdx.z;
  int global_step = blockDim.x * gridDim.x * blockDim.z * gridDim.z;
  TsallisTransform(num_rollouts * blockDim.z, trajectory_costs_d, gamma, r, baseline, global_idx, global_step);
}

template <int CONTROL_DIM, int NUM_ROLLOUTS, int SUM_STRIDE>
__global__ void weightedReductionKernel(float* exp_costs_d, float* du_d, float* du_new_d, float normalizer,
                                        int num_timesteps)
{
  int thread_idx = threadIdx.x;  // Rollout index
  int block_idx = blockIdx.x;    // Timestep

  // Create a shared array for intermediate sums: CONTROL_DIM x NUM_THREADS
  __shared__ float u_intermediate[CONTROL_DIM * ((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1)];

  float u[CONTROL_DIM];
  setInitialControlToZero(CONTROL_DIM, thread_idx, u, u_intermediate);

  __syncthreads();

  // Sum the weighted control variations at a desired stride
  strideControlWeightReduction(NUM_ROLLOUTS, num_timesteps, SUM_STRIDE, thread_idx, block_idx, CONTROL_DIM, exp_costs_d,
                               normalizer, du_d, u, u_intermediate);

  __syncthreads();

  // Sum all weighted control variations
  rolloutWeightReductionAndSaveControl(thread_idx, block_idx, NUM_ROLLOUTS, num_timesteps, CONTROL_DIM, SUM_STRIDE, u,
                                       u_intermediate, du_new_d);

  __syncthreads();
}

template <int CONTROL_DIM, int NUM_ROLLOUTS, int SUM_STRIDE>
__global__ void weightedReductionKernel(float* exp_costs_d, float* du_d, float* du_new_d,
                                        float2* baseline_and_normalizer_d, int num_timesteps)
{
  int thread_idx = threadIdx.x;  // Rollout index
  int block_idx = blockIdx.x;    // Timestep

  // Create a shared array for intermediate sums: CONTROL_DIM x NUM_THREADS
  __shared__ float u_intermediate[CONTROL_DIM * ((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1)];

  float u[CONTROL_DIM];
  setInitialControlToZero(CONTROL_DIM, thread_idx, u, u_intermediate);

  __syncthreads();

  // Sum the weighted control variations at a desired stride
  strideControlWeightReduction(NUM_ROLLOUTS, num_timesteps, SUM_STRIDE, thread_idx, block_idx, CONTROL_DIM, exp_costs_d,
                               baseline_and_normalizer_d->y, du_d, u, u_intermediate);

  __syncthreads();

  // Sum all weighted control variations
  rolloutWeightReductionAndSaveControl(thread_idx, block_idx, NUM_ROLLOUTS, num_timesteps, CONTROL_DIM, SUM_STRIDE, u,
                                       u_intermediate, du_new_d);

  __syncthreads();
}

/*******************************************************************************************************************
 * Rollout Kernel Helpers
 *******************************************************************************************************************/
template <int STATE_DIM, int CONTROL_DIM>
__device__ void loadGlobalToShared(const int num_rollouts, const int blocksize_y, const int global_idx,
                                   const int thread_idy, const int thread_idz, const float* x_device,
                                   const float* sigma_u_device, float* x_thread, float* xdot_thread, float* u_thread,
                                   float* du_thread, float* sigma_u_thread)
{
  // Transfer to shared memory
  int i;
  // float zero_state[STATE_DIM] = { 0 };
  if (global_idx < num_rollouts)
  {
#if false
    mp1::loadArrayParallel<STATE_DIM>(x_thread, 0, x_device, STATE_DIM * thread_idz);
    if (STATE_DIM % 4 == 0)
    {
      float4* xdot4_t = reinterpret_cast<float4*>(xdot_thread);
      for (i = thread_idy; i < STATE_DIM / 4; i +=blocksize_y)
      {
        xdot4_t[i] = make_float4(0, 0, 0, 0);
      }
    }
    else if (STATE_DIM % 2 == 0)
    {
      float2* xdot2_t = reinterpret_cast<float2*>(xdot_thread);
      for (i = thread_idy; i < STATE_DIM / 2; i +=blocksize_y)
      {
        xdot2_t[i] = make_float2(0, 0);
      }
    }
    else
    {
      for (i = thread_idy; i < STATE_DIM; i +=blocksize_y)
      {
        xdot_thread[i] = 0;
      }
    }

    if (CONTROL_DIM % 4 == 0)
    {
      float4* u4_t = reinterpret_cast<float4*>(u_thread);
      float4* du4_t = reinterpret_cast<float4*>(du_thread);
      for (i = thread_idy; i < CONTROL_DIM / 4; i +=blocksize_y)
      {
        u4_t[i] = make_float4(0, 0, 0, 0);
        du4_t[i] = make_float4(0, 0, 0, 0);
      }
    }
    else if (CONTROL_DIM % 2 == 0)
    {
      float2* u2_t = reinterpret_cast<float2*>(u_thread);
      float2* du2_t = reinterpret_cast<float2*>(du_thread);
      for (i = thread_idy; i < CONTROL_DIM / 2; i +=blocksize_y)
      {
        u2_t[i] = make_float2(0, 0);
        du2_t[i] = make_float2(0, 0);
      }
    }
    else
    {
      for (i = thread_idy; i < CONTROL_DIM; i += blocksize_y)
      {
        u_thread[i] = 0;
        du_thread[i] = 0;
      }
    }
#else
    for (i = thread_idy; i < STATE_DIM; i += blocksize_y)
    {
      x_thread[i] = x_device[i + STATE_DIM * thread_idz];
      xdot_thread[i] = 0;
    }
    for (i = thread_idy; i < CONTROL_DIM; i += blocksize_y)
    {
      u_thread[i] = 0;
      du_thread[i] = 0;
    }
#endif
  }
  if (threadIdx.x == 0 /*&& threadIdx.z == 0*/)
  {
    mp1::loadArrayParallel<CONTROL_DIM>(sigma_u_thread, 0, sigma_u_device, 0);
    // for (i = thread_idy; i < control_dim; i += blocksize_y)
    // {
    //   sigma_u_thread[i] = sigma_u_device[i];
    // }
  }
}

__device__ void readControlsFromGlobal(const int control_dim, const int blocksize_y, const int num_rollouts,
                                       const int num_timesteps, const int t, const int global_idx, const int thread_idy,
                                       const float* u_d, const float* du_d, float* u_thread, float* du_thread)
{
  const int control_index = ((num_rollouts * threadIdx.z + global_idx) * num_timesteps + t) * control_dim;
  if (control_dim % 4 == 0)
  {
    float4* du4 = reinterpret_cast<float4*>(du_thread);
    float4* u4 = reinterpret_cast<float4*>(u_thread);
    const float4* u4_mean_d = reinterpret_cast<const float4*>(&u_d[t * control_dim]);
    const float4* du4_d = reinterpret_cast<const float4*>(&du_d[control_index]);
    for (int j = thread_idy; j < control_dim / 4; j += blocksize_y)
    {
      if (global_idx == 0)
      {
        du4[j] = make_float4(0, 0, 0, 0);
        u4[j] = u4_mean_d[j];
      }
      else if (global_idx >= 0.99 * num_rollouts)
      {
        du4[j] = du4_d[j];
        u4[j] = du4[j];
      }
      else
      {
        u4[j] = du4_d[j];
        du4[j] = u4[j] - u4_mean_d[j];
      }
    }
  }
  else if (control_dim % 2 == 0)
  {
    float2* du2 = reinterpret_cast<float2*>(du_thread);
    float2* u2 = reinterpret_cast<float2*>(u_thread);
    const float2* u2_mean_d = reinterpret_cast<const float2*>(&u_d[t * control_dim]);
    const float2* du2_d = reinterpret_cast<const float2*>(&du_d[control_index]);
    for (int j = thread_idy; j < control_dim / 2; j += blocksize_y)
    {
      if (global_idx == 0)
      {
        du2[j] = make_float2(0, 0);
        u2[j] = u2_mean_d[j];
      }
      else if (global_idx >= 0.99 * num_rollouts)
      {
        du2[j] = du2_d[j];
        u2[j] = du2[j];
      }
      else
      {
        u2[j] = du2_d[j];
        du2[j] = u2[j] - u2_mean_d[j];
      }
    }
  }
  else
  {
    for (int j = thread_idy; j < control_dim; j += blockDim.y)
    {
      if (global_idx == 0)
      {
        du_thread[j] = 0;
        u_thread[j] = u_d[t * control_dim + j];
      }
      else if (global_idx >= 0.99 * num_rollouts)
      {
        du_thread[j] = du_d[control_index + j];
        u_thread[j] = du_thread[j];
      }
      else
      {
        u_thread[j] = du_d[control_index + j];
        du_thread[j] = u_thread[j] - u_d[t * control_dim + j];
      }
    }
  }
}

// TODO generalize the trim control
// The zero control trajectory should be an equilbrium control defined in the dynamics.
__device__ void injectControlNoise(int control_dim, int blocksize_y, int num_rollouts, int num_timesteps,
                                   int current_timestep, int global_idx, int thread_idy, int optimization_stride,
                                   const float* u_traj_device, float* ep_v_device, const float* sigma_u_thread,
                                   float* u_thread, float* du_thread)
{
  // this is a global index
  int control_index = (num_rollouts * num_timesteps * threadIdx.z +  // z part
                       global_idx * num_timesteps + current_timestep) *
                      control_dim;  // normal part
  // Load the noise trajectory scaled by the exploration factor
  // The prior loop already guarantees that the global index is less than the number of rollouts
  if (control_dim % 4 == 0)
  {
    float4* u4_thread = reinterpret_cast<float4*>(u_thread);
    float4* du4_thread = reinterpret_cast<float4*>(du_thread);
    const float4* sigma_u4_thread = reinterpret_cast<const float4*>(sigma_u_thread);
    const float4* u4_traj_device = reinterpret_cast<const float4*>(&u_traj_device[current_timestep * control_dim]);
    float4* ep4_v_device = reinterpret_cast<float4*>(&ep_v_device[control_index]);
    for (int i = thread_idy; i < control_dim / 4; i += blocksize_y)
    {
      // Keep one noise free trajectory
      if (global_idx == 0 || current_timestep < optimization_stride)
      {
        du4_thread[i] = make_float4(0, 0, 0, 0);
        u4_thread[i] = u4_traj_device[i];
      }
      // Generate 1% zero control trajectory
      else if (global_idx >= 0.99 * num_rollouts)
      {
        du4_thread[i] = ep4_v_device[i] * sigma_u4_thread[i];
        u4_thread[i] = du4_thread[i];
      }
      else
      {
        du4_thread[i] = ep4_v_device[i] * sigma_u4_thread[i];
        u4_thread[i] = u4_traj_device[i] + du4_thread[i];
      }
      // Saves the control but doesn't clamp it.
      ep4_v_device[i] = u4_thread[i];
    }
  }
  else if (control_dim % 2 == 0)
  {
    float2* u2_thread = reinterpret_cast<float2*>(u_thread);
    float2* du2_thread = reinterpret_cast<float2*>(du_thread);
    const float2* sigma_u2_thread = reinterpret_cast<const float2*>(sigma_u_thread);
    const float2* u2_traj_device = reinterpret_cast<const float2*>(&u_traj_device[current_timestep * control_dim]);
    float2* ep2_v_device = reinterpret_cast<float2*>(&ep_v_device[control_index]);
    for (int i = thread_idy; i < control_dim / 2; i += blocksize_y)
    {
      // Keep one noise free trajectory
      if (global_idx == 0 || current_timestep < optimization_stride)
      {
        du2_thread[i] = make_float2(0, 0);
        u2_thread[i] = u2_traj_device[i];
      }
      // Generate 1% zero control trajectory
      else if (global_idx >= 0.99 * num_rollouts)
      {
        du2_thread[i] = ep2_v_device[i] * sigma_u2_thread[i];
        u2_thread[i] = du2_thread[i];
      }
      else
      {
        du2_thread[i] = ep2_v_device[i] * sigma_u2_thread[i];
        u2_thread[i] = u2_traj_device[i] + du2_thread[i];
      }
      // Saves the control but doesn't clamp it.
      ep2_v_device[i] = u2_thread[i];
    }
  }
  else
  {
    for (int i = thread_idy; i < control_dim; i += blocksize_y)
    {
      // Keep one noise free trajectory
      if (global_idx == 0 || current_timestep < optimization_stride)
      {
        du_thread[i] = 0;
        u_thread[i] = u_traj_device[current_timestep * control_dim + i];
      }
      // Generate 1% zero control trajectory
      else if (global_idx >= 0.99 * num_rollouts)
      {
        du_thread[i] = ep_v_device[control_index + i] * sigma_u_thread[i];
        u_thread[i] = du_thread[i];
      }
      else
      {
        du_thread[i] = ep_v_device[control_index + i] * sigma_u_thread[i];
        u_thread[i] = u_traj_device[current_timestep * control_dim + i] + du_thread[i];
      }
      // Saves the control but doesn't clamp it.
      ep_v_device[control_index + i] = u_thread[i];
    }
  }
}

template <class COST_T>
__device__ void computeAndSaveCost(int num_rollouts, int num_timesteps, int global_idx, COST_T* costs, float* output,
                                   float running_cost, float* theta_c, float* cost_rollouts_device)
{
  // only want to save 1 cost per trajectory
  if (threadIdx.y == 0 && global_idx < num_rollouts)
  {
    cost_rollouts_device[global_idx + num_rollouts * threadIdx.z] =
        running_cost + costs->terminalCost(output, theta_c) / (num_timesteps - 1);
  }
}

/*******************************************************************************************************************
 * NormExp Kernel Helpers
 *******************************************************************************************************************/
float computeBaselineCost(float* cost_rollouts_host, int num_rollouts)
{  // TODO if we use standard containers in MPPI, should this be replaced with a min algorithm?
  int best_idx = computeBestIndex(cost_rollouts_host, num_rollouts);
  return cost_rollouts_host[best_idx];
}

float constructBestWeights(float* cost_rollouts_host, int num_rollouts)
{
  int best_idx = computeBestIndex(cost_rollouts_host, num_rollouts);
  float best_cost = cost_rollouts_host[best_idx];

  for (int i = 0; i < num_rollouts; i++)
  {
    if (i == best_idx)
    {
      cost_rollouts_host[i] = 1.0;
    }
    else
    {
      cost_rollouts_host[i] = 0.0;
    }
  }

  // printf("Best idx: %d, cost: %f\n", best_cost_idx, best_cost);
  return best_cost;
}

int computeBestIndex(float* cost_rollouts_host, int num_rollouts)
{
  float best_cost = cost_rollouts_host[0];
  int best_cost_idx = 0;
  for (int i = 1; i < num_rollouts; i++)
  {
    if (cost_rollouts_host[i] < best_cost)
    {
      best_cost = cost_rollouts_host[i];
      best_cost_idx = i;
    }
  }

  // printf("Best idx: %d, cost: %f\n", best_cost_idx, best_cost);
  return best_cost_idx;
}

__device__ inline float computeBaselineCost(int num_rollouts, const float* __restrict__ trajectory_costs_d,
                                            float* __restrict__ reduction_buffer, int rollout_idx_global,
                                            int rollout_idx_step)
{
  // Copy costs to shared memory
  float min_cost = 0.0;
#if false
  // potential method to speed up copying costs
  int prev_size = min(blockDim.x, num_rollouts);
  float my_val = (rollout_idx_global < num_rollouts) ? trajectory_costs_d[rollout_idx_global] : INFINITY;
  for (int i = rollout_idx_global + rollout_idx_step; i < num_rollouts; i += rollout_idx_step)
  {
    my_val = min(trajectory_costs_d[i], my_val);
  }
  reduction_buffer[rollout_idx_global] = my_val;
  // __syncthreads();
  // if (threadIdx.x == 0)
  // {
  //   for (int i = 0; i < min(blockDim.x, num_rollouts); i++)
  //   {
  //     printf("buff %d: %f\n", i, reduction_buffer[i]);
  //   }
  //   printf("Num rollouts; %d\n", num_rollouts);
  // }
#else
  int prev_size = num_rollouts / 2;
  for (int i = rollout_idx_global; i < prev_size; i += rollout_idx_step)
  {
    reduction_buffer[i] = min(trajectory_costs_d[i], trajectory_costs_d[i + prev_size]);
  }
  if (num_rollouts - 2 * prev_size == 1 && threadIdx.x == blockDim.x - 1)
  {
    reduction_buffer[prev_size - 1] = min(reduction_buffer[num_rollouts - 1], reduction_buffer[prev_size - 1]);
  }
#endif

  __syncthreads();
  // find min along the entire array
  for (int size = prev_size / 2; size > 0; size /= 2)
  {
    for (int i = rollout_idx_global; i < size; i += rollout_idx_step)
    {
      reduction_buffer[i] = min(reduction_buffer[i], reduction_buffer[i + size]);
    }
    __syncthreads();
    if (prev_size - 2 * size == 1 && threadIdx.x == blockDim.x - 1)
    {
      reduction_buffer[size - 1] = min(reduction_buffer[size - 1], reduction_buffer[prev_size - 1]);
    }
    __syncthreads();
    prev_size = size;
  }
  min_cost = reduction_buffer[0];
  return min_cost;
}

__device__ __host__ inline void normExpTransform(int num_rollouts, float* __restrict__ trajectory_costs_d,
                                                 float lambda_inv, float baseline, int global_idx, int rollout_idx_step)
{
  for (int i = global_idx; i < num_rollouts; i += rollout_idx_step)
  {
    float cost_dif = trajectory_costs_d[i] - baseline;
    trajectory_costs_d[i] = expf(-lambda_inv * cost_dif);
  }
}

__device__ __host__ inline void TsallisTransform(int num_rollouts, float* __restrict__ trajectory_costs_d, float gamma,
                                                 float r, float baseline, int global_idx, int rollout_idx_step)
{
  for (int i = global_idx; i < num_rollouts; i += rollout_idx_step)
  {
    float cost_dif = trajectory_costs_d[i] - baseline;
    // trajectory_costs_d[i] = mppi::math::expr(-lambda_bar_inv * cost_dif);
    // trajectory_costs_d[i] = (cost_dif < gamma) * expf(logf(1.0 - cost_dif / gamma) / (r - 1));
    if (cost_dif < gamma)
    {
      trajectory_costs_d[i] = expf(logf(1.0 - cost_dif / gamma) / (r - 1));
    }
    else
    {
      trajectory_costs_d[i] = 0;
    }
  }
}

__device__ inline float computeNormalizer(int num_rollouts, const float* __restrict__ trajectory_costs_d,
                                          float* __restrict__ reduction_buffer, int rollout_idx_global,
                                          int rollout_idx_step)
{
  // Copy costs to shared memory
#if false
  // potential method to speed up copying costs
  int prev_size = min(blockDim.x, num_rollouts);
  float my_val = (rollout_idx_global < num_rollouts) ? trajectory_costs_d[rollout_idx_global] : 0;
  for (int i = rollout_idx_global + rollout_idx_step; i < num_rollouts; i += rollout_idx_step)
  {
    my_val += trajectory_costs_d[i];
  }
  reduction_buffer[rollout_idx_global] = my_val;
#else
  int prev_size = num_rollouts / 2;
  for (int i = rollout_idx_global; i < prev_size; i += rollout_idx_step)
  {
    reduction_buffer[i] = trajectory_costs_d[i] + trajectory_costs_d[i + prev_size];
  }
  if (num_rollouts - 2 * prev_size == 1 && threadIdx.x == blockDim.x - 1)
  {
    reduction_buffer[prev_size - 1] += reduction_buffer[num_rollouts - 1];
  }
#endif
  __syncthreads();
  // sum the entire array
  for (int size = prev_size / 2; size > 0; size /= 2)
  {
    for (int i = rollout_idx_global; i < size; i += rollout_idx_step)
    {
      reduction_buffer[i] += reduction_buffer[i + size];
    }
    __syncthreads();
    if (prev_size - 2 * size == 1 && threadIdx.x == blockDim.x - 1)
    {
      reduction_buffer[size - 1] += reduction_buffer[prev_size - 1];
    }
    __syncthreads();
    prev_size = size;
  }
  return reduction_buffer[0];
}

template <int NUM_ROLLOUTS, int BLOCKSIZE_X = 1024>
__global__ void fullGPUcomputeWeights(float* __restrict__ trajectory_costs_d, float lambda_inv,
                                      float2* __restrict__ output)
{
  __shared__ float reduction_buffer[NUM_ROLLOUTS];
  // int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  // int better_global_idx = (blockIdx.x * blockDim.x + threadIdx.x) * blockDim.y + threadIdx.y;
  // int global_idx = (blockDim.x * blockIdx.x + threadIdx.x) * blockDim.z + threadIdx.z;
  // int global_step = blockDim.x * gridDim.x;
  // int better_global_step = blockDim.x * gridDim.x  * blockDim.y * gridDim.y;
  int global_idx = threadIdx.x;
  int global_step = blockDim.x;

  float baseline = computeBaselineCost(NUM_ROLLOUTS, trajectory_costs_d, reduction_buffer, global_idx, global_step);
  normExpTransform(NUM_ROLLOUTS, trajectory_costs_d, lambda_inv, baseline, global_idx, global_step);
  __syncthreads();
  float normalizer = computeNormalizer(NUM_ROLLOUTS, trajectory_costs_d, reduction_buffer, global_idx, global_step);
  __syncthreads();
  if (threadIdx.x == 0)
  {
    *output = make_float2(baseline, normalizer);
  }
}

float computeNormalizer(float* cost_rollouts_host, int num_rollouts)
{
  double normalizer = 0.0;
  for (int i = 0; i < num_rollouts; ++i)
  {
    normalizer += cost_rollouts_host[i];
  }
  return normalizer;
}

void computeFreeEnergy(float& free_energy, float& free_energy_var, float& free_energy_modified,
                       float* cost_rollouts_host, int num_rollouts, float baseline, float lambda)
{
  float var = 0;
  float norm = 0;
  for (int i = 0; i < num_rollouts; i++)
  {
    norm += cost_rollouts_host[i];
    var += powf(cost_rollouts_host[i], 2);
  }
  norm /= num_rollouts;
  free_energy = -lambda * logf(norm) + baseline;
  free_energy_var = lambda * (var / num_rollouts - powf(norm, 2));
  // TODO Figure out the point of the following lines
  float weird_term = free_energy_var / (norm * sqrtf(1.0 * num_rollouts));
  free_energy_modified = lambda * (weird_term + 0.5 * powf(weird_term, 2));
}

/*******************************************************************************************************************
 * Weighted Reduction Kernel Helpers
 *******************************************************************************************************************/
__device__ void setInitialControlToZero(int control_dim, int thread_idx, float* u, float* u_intermediate)
{
  for (int i = 0; i < control_dim; i++)
  {
    u[i] = 0;
    u_intermediate[thread_idx * control_dim + i] = 0;
  }
}

__device__ void strideControlWeightReduction(int num_rollouts, int num_timesteps, int sum_stride, int thread_idx,
                                             int block_idx, int control_dim, float* exp_costs_d, float normalizer,
                                             float* du_d, float* u, float* u_intermediate)
{
  // int index = thread_idx * sum_stride + i;
  for (int i = 0; i < sum_stride; ++i)
  {  // Iterate through the size of the subsection
    if ((thread_idx * sum_stride + i) < num_rollouts)
    {                                                                        // Ensure we do not go out of bounds
      float weight = exp_costs_d[thread_idx * sum_stride + i] / normalizer;  // compute the importance sampling weight
      for (int j = 0; j < control_dim; ++j)
      {  // Iterate through the control dimensions
        // Rollout index: (thread_idx*sum_stride + i)*(num_timesteps*control_dim)
        // Current timestep: block_idx*control_dim
        u[j] = du_d[(thread_idx * sum_stride + i) * (num_timesteps * control_dim) + block_idx * control_dim + j];
        u_intermediate[thread_idx * control_dim + j] += weight * u[j];
      }
    }
  }
}

__device__ void rolloutWeightReductionAndSaveControl(int thread_idx, int block_idx, int num_rollouts, int num_timesteps,
                                                     int control_dim, int sum_stride, float* u, float* u_intermediate,
                                                     float* du_new_d)
{
  if (thread_idx == 0 && block_idx < num_timesteps)
  {  // block index refers to the current timestep
    for (int i = 0; i < control_dim; ++i)
    {  // TODO replace with memset?
      u[i] = 0;
    }
    for (int i = 0; i < ((num_rollouts - 1) / sum_stride + 1); ++i)
    {  // iterate through the each subsection
      for (int j = 0; j < control_dim; ++j)
      {
        u[j] += u_intermediate[i * control_dim + j];
      }
    }
    for (int i = 0; i < control_dim; i++)
    {
      du_new_d[block_idx * control_dim + i] = u[i];
    }
  }
}

template <class COST_T, int BLOCKSIZE_X, bool COALESCE = false>
__global__ void visualizeCostKernel(COST_T* costs, float dt, const int num_timesteps, const int num_rollouts,
                                    const float lambda, float alpha, const float* __restrict__ u_d,
                                    const float* __restrict__ du_d, const float* __restrict__ sigma_u_d,
                                    const float* __restrict__ y_d, float* __restrict__ cost_traj_d,
                                    int* __restrict__ crash_status_d)
{
  // Get thread and block id
  const int thread_idx = threadIdx.x;
  const int thread_idy = threadIdx.y;
  const int thread_idz = threadIdx.z;
  const int global_idx = blockIdx.x;

  // Create shared state and control arrays
  extern __shared__ float entire_buffer[];
  float* y_shared = entire_buffer;
  float* u_shared = &y_shared[blockDim.x * blockDim.z * COST_T::OUTPUT_DIM];
  float* du_shared = &u_shared[blockDim.x * blockDim.z * COST_T::CONTROL_DIM];
  float* sigma_u = &du_shared[blockDim.x * blockDim.z * COST_T::CONTROL_DIM];
  float* running_cost_shared = &sigma_u[COST_T::CONTROL_DIM];
  int* crash_status_shared = (int*)&running_cost_shared[blockDim.z * num_timesteps];
  float* theta_c = (float*)&crash_status_shared[blockDim.z * blockDim.x];

  // Create local state, state dot and controls
  float* y;
  float* u;
  float* du;
  int* crash_status;

  // Initialize running cost and total cost
  float* running_cost;
  int sample_time_offset = 0;
  int cost_index = 0;

  // Load global array to shared array
  y = &y_shared[(blockDim.x * thread_idz + thread_idx) * COST_T::OUTPUT_DIM];
  u = &u_shared[(blockDim.x * thread_idz + thread_idx) * COST_T::CONTROL_DIM];
  du = &du_shared[(blockDim.x * thread_idz + thread_idx) * COST_T::CONTROL_DIM];
  crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
  crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.
  // running_cost = &running_cost_shared[thread_idz * blockDim.x + thread_idx];
  // running_cost[0] = 0;
  if (thread_idx == 0)
  {
    mp1::loadArrayParallel<COST_T::CONTROL_DIM>(sigma_u, 0, sigma_u_d, 0);
  }

  /*<----Start of simulation loop-----> */
  const int max_time_iters = ceilf((float)num_timesteps / BLOCKSIZE_X);
  costs->initializeCosts(y, u, theta_c, 0.0, dt);
  __syncthreads();
  for (int time_iter = 0; time_iter < max_time_iters; ++time_iter)
  {
    int t = thread_idx + time_iter * blockDim.x + 1;
    cost_index = (thread_idz * num_rollouts + global_idx) * (num_timesteps) + t - 1;
    running_cost = &running_cost_shared[thread_idz * blockDim.x + t - 1];
    if (t <= num_timesteps)
    {  // t = num_timesteps is the terminal state for outside this for-loop
      if (COALESCE)
      {  // Fill entire shared mem sequentially using sequential threads_idx
        mp1::loadArrayParallel<COST_T::OUTPUT_DIM * BLOCKSIZE_X, mp1::Parallel1Dir::THREAD_X>(
            y_shared, blockDim.x * thread_idz, y_d,
            ((num_rollouts * thread_idz + global_idx) * num_timesteps + time_iter * blockDim.x) * COST_T::OUTPUT_DIM);
      }
      else
      {
        sample_time_offset = (num_rollouts * thread_idz + global_idx) * num_timesteps + t - 1;
        mp1::loadArrayParallel<COST_T::OUTPUT_DIM>(y, 0, y_d, sample_time_offset * COST_T::OUTPUT_DIM);
      }
    }
    if (t < num_timesteps)
    {  // load controls from t = 0 to t = num_timesteps - 1
      // Have to do similar steps as injectControlNoise but using the already transformed cost samples
      readControlsFromGlobal(COST_T::CONTROL_DIM, blockDim.y, num_rollouts, num_timesteps, t, global_idx, thread_idy,
                             u_d, du_d, u, du);
    }
    __syncthreads();

    // Compute cost
    if (thread_idy == 0 && t < num_timesteps)
    {
      // cost_traj_d[cost_index] =
      running_cost[0] =
          costs->computeRunningCost(y, u, du, sigma_u, lambda, alpha, t, theta_c, crash_status) / (num_timesteps - 1);
      crash_status_d[global_idx * num_timesteps + t] = crash_status[0];
    }
    __syncthreads();
  }
  // point every thread to the last output at t = NUM_TIMESTEPS for terminal cost calculation
  const int last_y_index = (num_timesteps - 1) % BLOCKSIZE_X;
  y = &y_shared[(blockDim.x * thread_idz + last_y_index) * COST_T::OUTPUT_DIM];
  // Compute terminal cost for each thread
  if (threadIdx.x == 0 && threadIdx.y == 0)
  {
    cost_index = (threadIdx.z * num_rollouts + global_idx) * (num_timesteps + 1) + num_timesteps;
    cost_traj_d[cost_index] = costs->terminalCost(y, theta_c) / (num_timesteps - 1);
    // running_cost = &running_cost_shared[thread_idz * blockDim.x + num_timesteps];
    // running_cost[0] = costs->terminalCost(y, theta_c) / (num_timesteps - 1);
  }
  __syncthreads();
  // Copy to global memory
  if (((unsigned long)running_cost_shared & 15) == 0 && (num_timesteps) % 4 == 0)
  {
    for (int i = thread_idx; i < num_timesteps / 4; i += blockDim.x)
    {
      reinterpret_cast<float4*>(&cost_traj_d[(thread_idz * num_rollouts + global_idx) * num_timesteps])[i] =
          reinterpret_cast<float4*>(&running_cost_shared[thread_idz * num_timesteps])[i];
    }
  }
  else if (((unsigned long)running_cost_shared & 7) == 0 && (num_timesteps) % 2 == 0)
  {
    for (int i = thread_idx; i < num_timesteps / 2; i += blockDim.x)
    {
      reinterpret_cast<float2*>(&cost_traj_d[(thread_idz * num_rollouts + global_idx) * num_timesteps])[i] =
          reinterpret_cast<float2*>(&running_cost_shared[thread_idz * num_timesteps])[i];
    }
  }
  else
  {
    for (int i = thread_idx; i < num_timesteps; i += blockDim.x)
    {
      cost_traj_d[(thread_idz * num_rollouts + global_idx) * num_timesteps + i] =
          running_cost_shared[thread_idz * num_timesteps + i];
    }
  }
}

template <class DYN_T, class COST_T, class FB_T, int BLOCKSIZE_X, int BLOCKSIZE_Z>
__global__ void stateAndCostTrajectoryKernel(DYN_T* dynamics, COST_T* costs, FB_T* fb_controller, float* control,
                                             float* state, float* output_traj_d, float* cost_traj_d,
                                             int* crash_status_d, int num_rollouts, int num_timesteps, float dt,
                                             float value_func_threshold)
{
  // Get thread and block id
  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

  // Create shared state and control arrays
  __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float x_next_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float y_shared[BLOCKSIZE_X * DYN_T::OUTPUT_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];

  // Create a shared array for the nominal costs calculations
  __shared__ int crash_status_shared[BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the dynamics model to use
  __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD / sizeof(float) + 1 +
                           DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];
  __shared__ float
      theta_c[COST_T::SHARED_MEM_REQUEST_GRD / sizeof(float) + 1 + COST_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X];
  __shared__ float theta_fb[FB_T::SHARED_MEM_SIZE];

  // Create local state, state dot and controls
  float* x;
  float* x_next;
  float* x_temp;
  float* y;
  float* x_other;
  float* xdot;
  float* u;
  int* crash_status;
  float fb_control[DYN_T::CONTROL_DIM];
  int t_index = 0;
  int cost_index = 0;

  if (global_idx < num_rollouts)
  {
    // Actual or nominal
    x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    x_next = &x_next_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    y = &y_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::OUTPUT_DIM];
    // The opposite state from above
    x_other = &x_shared[(blockDim.x * (1 - thread_idz) + thread_idx) * DYN_T::STATE_DIM];
    xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    // Base trajectory
    u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    // Nominal State Cost
    crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
    crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.

    // Load memory into appropriate arrays
    for (int i = thread_idy; i < DYN_T::STATE_DIM; i += blockDim.y)
    {
      x[i] = state[DYN_T::STATE_DIM * threadIdx.z + i];
      xdot[i] = 0.0;
    }
    __syncthreads();
    float curr_state_cost = 0.0;

    dynamics->initializeDynamics(x, u, y, theta_s, 0.0, dt);
    costs->initializeCosts(y, u, theta_c, 0.0, dt);
    for (int t = 0; t < num_timesteps; t++)
    {
      t_index = threadIdx.z * num_rollouts * num_timesteps + global_idx * num_timesteps + t;
      cost_index = threadIdx.z * num_rollouts * (num_timesteps + 1) + global_idx * (num_timesteps + 1) + t;
      // get next u
      for (int i = thread_idy; i < DYN_T::CONTROL_DIM; i += blockDim.y)
      {
        u[i] = control[global_idx * num_timesteps * DYN_T::CONTROL_DIM + t * DYN_T::CONTROL_DIM + i];
      }

      // only apply feedback if enabled
      // feedback is only applied on real state in RMPPI
      if (BLOCKSIZE_Z > 1 && value_func_threshold == -1 && thread_idz == 0)
      {
        fb_controller->k(x, x_other, t, theta_fb, fb_control);

        for (int i = thread_idy; i < DYN_T::CONTROL_DIM; i += blockDim.y)
        {
          u[i] += fb_control[i];
        }
      }
      __syncthreads();

      dynamics->enforceConstraints(x, u);
      __syncthreads();

      if (thread_idy == 0 && t > 0)
      {
        curr_state_cost = costs->computeStateCost(y, t, theta_c, crash_status);
        crash_status_d[t_index] = crash_status[0];
        cost_traj_d[cost_index] = curr_state_cost;
      }
      __syncthreads();
      // Nominal system is where thread_idz == 1
      if (thread_idz == 1 && thread_idy == 0 && t > 0)
      {
        // compute the nominal system cost
        cost_traj_d[cost_index] =
            0.5 * curr_state_cost +
            // here we know threadIdx.z == 0 since we are only talking about the real system
            fmaxf(fminf(cost_traj_d[global_idx * (num_timesteps + 1) + t], value_func_threshold), curr_state_cost);
      }
      __syncthreads();
      // reset crash status in case initial location is actually a crash cost
      if (t == 0)
      {
        crash_status[0] = 0;
      }

      // Increment states
      dynamics->step(x, x_next, xdot, u, y, theta_s, t, dt);
      __syncthreads();
      x_temp = x;
      x = x_next;
      x_next = x_temp;

      // save results, skips the first state location since that is known
      for (int i = thread_idy; i < DYN_T::OUTPUT_DIM; i += blockDim.y)
      {
        output_traj_d[t_index * DYN_T::OUTPUT_DIM + i] = y[i];
      }
    }
    // get cost traj at +1
    cost_index = threadIdx.z * num_rollouts * (num_timesteps + 1) + global_idx * (num_timesteps + 1) + num_timesteps;
    cost_traj_d[cost_index] = costs->terminalCost(y, theta_c) / (num_timesteps - 1);
  }
}

/*******************************************************************************************************************
 * Launch Functions
 *******************************************************************************************************************/
template <class DYN_T, class COST_T, int NUM_ROLLOUTS, int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
void launchRolloutKernel(DYN_T* dynamics, COST_T* costs, float dt, int num_timesteps, int optimization_stride,
                         float lambda, float alpha, float* x_d, float* u_d, float* du_d, float* sigma_u_d,
                         float* trajectory_costs, hipStream_t stream, bool synchronize)
{
  const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
  static_assert(NUM_ROLLOUTS % BLOCKSIZE_X == 0, "NUM_ROLLOUTS must be evenly divided by BLOCKSIZE_X");
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(gridsize_x, 1, 1);
  rolloutKernel<DYN_T, COST_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS, BLOCKSIZE_Z>
      <<<dimGrid, dimBlock, 0, stream>>>(dynamics, costs, dt, num_timesteps, optimization_stride, lambda, alpha, x_d,
                                         u_d, du_d, sigma_u_d, trajectory_costs);
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

template <class DYN_T, class COST_T, int NUM_ROLLOUTS, int DYN_BLOCK_X, int DYN_BLOCK_Y, int BLOCKSIZE_Z,
          int COST_BLOCK_X, int COST_BLOCK_Y>
void launchFastRolloutKernel(DYN_T* dynamics, COST_T* costs, float dt, const int num_timesteps, int optimization_stride,
                             float lambda, float alpha, float* init_x_d, float* x_d, float* u_d, float* du_d,
                             float* sigma_u_d, float* trajectory_costs, hipStream_t stream, bool synchronize)
{
  // Run Dynamics
  const int gridsize_x = (NUM_ROLLOUTS - 1) / DYN_BLOCK_X + 1;
  dim3 dimBlock(DYN_BLOCK_X, DYN_BLOCK_Y, BLOCKSIZE_Z);
  dim3 dimGrid(gridsize_x, 1, 1);
  rolloutDynamicsKernel<DYN_T, DYN_BLOCK_X, DYN_BLOCK_Y, NUM_ROLLOUTS, BLOCKSIZE_Z><<<dimGrid, dimBlock, 0, stream>>>(
      dynamics, dt, num_timesteps, optimization_stride, init_x_d, u_d, du_d, sigma_u_d, x_d);

  // Run Costs
  dim3 dimCostBlock(COST_BLOCK_X, COST_BLOCK_Y, BLOCKSIZE_Z);
  dim3 dimCostGrid(NUM_ROLLOUTS, 1, 1);
  unsigned shared_mem_size =
      ((COST_BLOCK_X * BLOCKSIZE_Z) * (DYN_T::OUTPUT_DIM + 2 * DYN_T::CONTROL_DIM + 1) + DYN_T::CONTROL_DIM) *
          sizeof(float) +
      (COST_BLOCK_X * BLOCKSIZE_Z) * sizeof(int) + COST_T::SHARED_MEM_REQUEST_GRD +
      COST_T::SHARED_MEM_REQUEST_BLK * COST_BLOCK_X * BLOCKSIZE_Z * sizeof(float);
  rolloutCostKernel<DYN_T, COST_T, NUM_ROLLOUTS, COST_BLOCK_X><<<dimCostGrid, dimCostBlock, shared_mem_size, stream>>>(
      dynamics, costs, dt, num_timesteps, lambda, alpha, init_x_d, u_d, du_d, sigma_u_d, x_d, trajectory_costs);
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

void launchNormExpKernel(int num_rollouts, int blocksize_x, float* trajectory_costs_d, float lambda_inv, float baseline,
                         hipStream_t stream, bool synchronize)
{
  dim3 dimBlock(blocksize_x, 1, 1);
  dim3 dimGrid((num_rollouts - 1) / blocksize_x + 1, 1, 1);
  normExpKernel<<<dimGrid, dimBlock, 0, stream>>>(num_rollouts, trajectory_costs_d, lambda_inv, baseline);
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

void launchTsallisKernel(int num_rollouts, int blocksize_x, float* trajectory_costs_d, float gamma, float r,
                         float baseline, hipStream_t stream, bool synchronize)
{
  dim3 dimBlock(blocksize_x, 1, 1);
  dim3 dimGrid((num_rollouts - 1) / blocksize_x + 1, 1, 1);
  TsallisKernel<<<dimGrid, dimBlock, 0, stream>>>(num_rollouts, trajectory_costs_d, gamma, r, baseline);
  // CudaCheckError();
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

template <int NUM_ROLLOUTS>
void launchWeightTransformKernel(float* __restrict__ costs_d, float2* __restrict__ baseline_and_norm_d,
                                 const float lambda_inv, const int num_systems, hipStream_t stream, bool synchronize)
{
  // Figure out max size of threads from the device properties (slows down this method a lot)
  // int device_id = 0;
  // hipDeviceProp_t deviceProp;
  // hipGetDeviceProperties(&deviceProp, device_id);
  // int blocksize_x = deviceProp.maxThreadsDim[0];
  const int blocksize_x = 1024;
  dim3 dimBlock(blocksize_x, 1, 1);
  // Can't be split into multiple blocks because we want to do all the math in shared memory
  dim3 dimGrid(1, 1, 1);
  for (int i = 0; i < num_systems; i++)
  {
    fullGPUcomputeWeights<NUM_ROLLOUTS>
        <<<dimGrid, dimBlock, 0, stream>>>(costs_d + i * NUM_ROLLOUTS, lambda_inv, baseline_and_norm_d + i);
    HANDLE_ERROR(hipGetLastError());
  }
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

template <class DYN_T, int NUM_ROLLOUTS, int SUM_STRIDE>
void launchWeightedReductionKernel(float* exp_costs_d, float* du_d, float* du_new_d, float normalizer,
                                   int num_timesteps, hipStream_t stream, bool synchronize)
{
  dim3 dimBlock((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1, 1, 1);
  dim3 dimGrid(num_timesteps, 1, 1);
  weightedReductionKernel<DYN_T::CONTROL_DIM, NUM_ROLLOUTS, SUM_STRIDE>
      <<<dimGrid, dimBlock, 0, stream>>>(exp_costs_d, du_d, du_new_d, normalizer, num_timesteps);
  // CudaCheckError();
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

template <class DYN_T, int NUM_ROLLOUTS, int SUM_STRIDE>
void launchweightedReductionKernel(float* exp_costs_d, float* du_d, float* du_new_d, float2* baseline_and_normalizer_d,
                                   int num_timesteps, hipStream_t stream, bool synchronize)
{
  dim3 dimBlock((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1, 1, 1);
  dim3 dimGrid(num_timesteps, 1, 1);
  weightedReductionKernel<DYN_T::CONTROL_DIM, NUM_ROLLOUTS, SUM_STRIDE>
      <<<dimGrid, dimBlock, 0, stream>>>(exp_costs_d, du_d, du_new_d, baseline_and_normalizer_d, num_timesteps);
  // CudaCheckError();
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

template <class DYN_T, class COST_T, class FB_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
void launchStateAndCostTrajectoryKernel(DYN_T* dynamics, COST_T* cost, FB_T* fb_controller, float* control_trajectories,
                                        float* state, float* output_traj_result, float* cost_traj_result,
                                        int* crash_status_result, int num_rollouts, int num_timesteps, float dt,
                                        hipStream_t stream, float value_func_threshold, bool synchronize)
{
  const int gridsize_x = (num_rollouts - 1) / BLOCKSIZE_X + 1;
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(gridsize_x, 1, 1);
  stateAndCostTrajectoryKernel<DYN_T, COST_T, FB_T, BLOCKSIZE_X, BLOCKSIZE_Z><<<dimGrid, dimBlock, 0, stream>>>(
      dynamics, cost, fb_controller, control_trajectories, state, output_traj_result, cost_traj_result,
      crash_status_result, num_rollouts, num_timesteps, dt, value_func_threshold);
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

template <class COST_T, int COST_BLOCK_X, int COST_BLOCK_Y, int BLOCKSIZE_Z>
void launchVisualizeCostKernel(COST_T* costs, float dt, const int num_timesteps, const int num_rollouts, float lambda,
                               float alpha, float* y_d, float* sampled_controls_d, int* sampled_crash_status_d,
                               float* sigma_u_d, float* cost_traj_result, hipStream_t stream, bool synchronize)
{
  if (num_rollouts <= 1)
  {  // Not enough samples to visualize
    std::cerr << "Not enough samples to visualize" << std::endl;
    return;
  }
  // Run Costs
  dim3 dimCostBlock(COST_BLOCK_X, COST_BLOCK_Y, BLOCKSIZE_Z);
  dim3 dimCostGrid(num_rollouts, 1, 1);
  unsigned shared_mem_size =
      ((COST_BLOCK_X * BLOCKSIZE_Z) * (COST_T::OUTPUT_DIM + 2 * COST_T::CONTROL_DIM) + COST_T::CONTROL_DIM) *
          sizeof(float) +
      sizeof(float) * (num_timesteps * BLOCKSIZE_Z) + (COST_BLOCK_X * BLOCKSIZE_Z) * sizeof(int) +
      COST_T::SHARED_MEM_REQUEST_GRD + COST_T::SHARED_MEM_REQUEST_BLK * COST_BLOCK_X * BLOCKSIZE_Z * sizeof(float);
  visualizeCostKernel<COST_T, COST_BLOCK_X><<<dimCostGrid, dimCostBlock, shared_mem_size, stream>>>(
      costs, dt, num_timesteps, num_rollouts - 1, lambda, alpha, sampled_controls_d,
      sampled_controls_d + COST_T::CONTROL_DIM * num_timesteps, sigma_u_d, y_d, cost_traj_result,
      sampled_crash_status_d);
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}
}  // namespace mppi_common

namespace rmppi_kernels
{
template <class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int SAMPLES_PER_CONDITION>
__global__ void initEvalKernel(DYN_T* dynamics, COST_T* costs, int num_timesteps, float lambda, float alpha,
                               int ctrl_stride, float dt, int* strides_d, float* exploration_std_dev_d, float* states_d,
                               float* control_d, float* control_noise_d, float* costs_d)
{
  int i, j;
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  int bdx = blockIdx.x;

  // Initialize the local state, controls, and noise
  float* state;
  float* state_next;
  float* state_temp;
  float* output;
  float* state_der;
  float* control;
  float* control_noise;  // du
  int* crash_status;

  // Create shared arrays for holding state and control data.
  __shared__ float state_shared[BLOCKSIZE_X * DYN_T::STATE_DIM];
  __shared__ float state_next_shared[BLOCKSIZE_X * DYN_T::STATE_DIM];
  __shared__ float output_shared[BLOCKSIZE_X * DYN_T::OUTPUT_DIM];
  __shared__ float state_der_shared[BLOCKSIZE_X * DYN_T::STATE_DIM];
  __shared__ float control_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM];
  __shared__ float control_noise_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM];
  __shared__ float exploration_std_dev[DYN_T::CONTROL_DIM];  // Each thread only reads
  __shared__ int crash_status_shared[BLOCKSIZE_X];

  // Create a shared array for the dynamics model to use
  __shared__ float
      theta_s[DYN_T::SHARED_MEM_REQUEST_GRD / sizeof(float) + 1 + DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X];
  __shared__ float theta_c[COST_T::SHARED_MEM_REQUEST_GRD + COST_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X];

  float running_cost = 0;  // Initialize trajectory cost

  int global_idx = BLOCKSIZE_X * bdx + tdx;                // Set the global index for CUDA threads
  int condition_idx = global_idx / SAMPLES_PER_CONDITION;  // Set the index for our candidate
  int stride = strides_d[condition_idx];                   // Each candidate can have a different starting stride

  // Get the pointer that belongs to the current thread with respect to the shared arrays
  state = &state_shared[tdx * DYN_T::STATE_DIM];
  state_next = &state_next_shared[tdx * DYN_T::STATE_DIM];
  output = &output_shared[tdx * DYN_T::OUTPUT_DIM];
  state_der = &state_der_shared[tdx * DYN_T::STATE_DIM];
  control = &control_shared[tdx * DYN_T::CONTROL_DIM];
  control_noise = &control_noise_shared[tdx * DYN_T::CONTROL_DIM];
  crash_status = &crash_status_shared[tdx];
  crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.

  // Copy the state to the thread
  for (i = tdy; i < DYN_T::STATE_DIM; i += blockDim.y)
  {
    state[i] = states_d[condition_idx * DYN_T::STATE_DIM + i];  // states_d holds each condition
  }

  // Copy the exploration noise std_dev to the thread
  for (i = tdy; i < DYN_T::CONTROL_DIM; i += blockDim.y)
  {
    control[i] = 0.0;
    control_noise[i] = 0.0;
    exploration_std_dev[i] = exploration_std_dev_d[i];
  }

  __syncthreads();
  dynamics->initializeDynamics(state, control, output, theta_s, 0.0, dt);
  costs->initializeCosts(output, control, theta_c, 0.0, dt);
  for (i = 0; i < num_timesteps; ++i)
  {  // Outer loop iterates on timesteps
    // Inject the control noise
    for (j = tdy; j < DYN_T::CONTROL_DIM; j += blockDim.y)
    {
      if ((i + stride) >= num_timesteps)
      {  // Pad the end of the controls with the last control
        control[j] = control_d[(num_timesteps - 1) * DYN_T::CONTROL_DIM + j];
      }
      else
      {
        control[j] = control_d[(i + stride) * DYN_T::CONTROL_DIM + j];
      }

      // First rollout is noise free
      if (global_idx % SAMPLES_PER_CONDITION == 0 || i < ctrl_stride)
      {
        control_noise[j] = 0.0;
      }
      else
      {
        control_noise[j] =
            control_noise_d[num_timesteps * DYN_T::CONTROL_DIM * global_idx + i * DYN_T::CONTROL_DIM + j] *
            exploration_std_dev[j];
      }

      // Sum the control and the noise
      control[j] += control_noise[j];
    }  // End inject control noise

    __syncthreads();

    dynamics->enforceConstraints(state, control);
    __syncthreads();
    if (tdy == 0 && i > 0)
    {  // Only compute once per global index, make sure that we don't divide by zero
      running_cost += (costs->computeRunningCost(output, control, control_noise, exploration_std_dev, lambda, alpha, i,
                                                 theta_c, crash_status) -
                       running_cost) /
                      (1.0 * i);
    }
    __syncthreads();

    // Increment states
    dynamics->step(state, state_next, state_der, control, output, theta_s, i, dt);
    __syncthreads();
    state_temp = state;
    state = state_next;
    state_next = state_temp;
  }
  // End loop outer loop on timesteps

  if (tdy == 0)
  {  // Only save the costs once per global idx (thread y is only for parallelization)
    costs_d[global_idx] = running_cost + costs->terminalCost(output, theta_c) / (num_timesteps - 1);
  }
}

// Newly Written
template <class DYN_T, class COST_T, class FB_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int NUM_ROLLOUTS, int BLOCKSIZE_Z>
__global__ void RMPPIRolloutKernel(DYN_T* dynamics, COST_T* costs, FB_T* fb_controller, float dt, int num_timesteps,
                                   int optimization_stride, float lambda, float alpha, float value_func_threshold,
                                   float* x_d, float* u_d, float* du_d, float* sigma_u_d, float* trajectory_costs_d)
{
  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

  // Create shared memory for state and control
  __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float x_next_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float y_shared[BLOCKSIZE_X * DYN_T::OUTPUT_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float du_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float sigma_u[DYN_T::CONTROL_DIM];

  // Create a shared array for the nominal costs calculations
  __shared__ float running_state_cost_nom_shared[BLOCKSIZE_X];
  __shared__ float running_control_cost_nom_shared[BLOCKSIZE_X];
  __shared__ int crash_status_shared[BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the dynamics model to use
  __shared__ float theta_s[DYN_T::SHARED_MEM_REQUEST_GRD / sizeof(float) + 1 +
                           DYN_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];
  __shared__ float theta_c[COST_T::SHARED_MEM_REQUEST_GRD + COST_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X * BLOCKSIZE_Z];

  // Create a shared array for the feedback controller to use
  __shared__ float theta_fb[FB_T::SHARED_MEM_SIZE];

  // Create local state, state dot and controls
  float* x;
  float* x_next;
  float* x_next_other;
  float* x_temp;
  float* y;
  float* x_other;
  float* xdot;
  float* u;
  float* du;
  int* crash_status;
  // The array to hold K(x,x*)
  float fb_control[DYN_T::CONTROL_DIM];

  int t = 0;
  int i = 0;
  // int j = 0;

  // Initialize running costs
  float running_state_cost_real = 0;
  float running_control_cost_real = 0;
  float* running_state_cost_nom;
  float running_tracking_cost_real = 0;
  float* running_control_cost_nom;

  // Load global array into shared memory
  if (global_idx < NUM_ROLLOUTS)
  {
    // Actual or nominal
    x = &x_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    x_next = &x_next_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    y = &y_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::OUTPUT_DIM];
    // The opposite state from above
    x_other = &x_shared[(blockDim.x * (1 - thread_idz) + thread_idx) * DYN_T::STATE_DIM];
    x_next_other = &x_next_shared[(blockDim.x * (1 - thread_idz) + thread_idx) * DYN_T::STATE_DIM];
    xdot = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::STATE_DIM];
    // Base trajectory
    u = &u_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    // Noise added to trajectory
    du = &du_shared[(blockDim.x * thread_idz + thread_idx) * DYN_T::CONTROL_DIM];
    // Nominal State Cost
    running_state_cost_nom = &running_state_cost_nom_shared[thread_idx];
    running_control_cost_nom = &running_control_cost_nom_shared[thread_idx];
    crash_status = &crash_status_shared[thread_idz * blockDim.x + thread_idx];
    crash_status[0] = 0;  // We have not crashed yet as of the first trajectory.

    // Load memory into appropriate arrays
    mppi_common::loadGlobalToShared<DYN_T::STATE_DIM, DYN_T::CONTROL_DIM>(
        NUM_ROLLOUTS, BLOCKSIZE_Y, global_idx, thread_idy, thread_idz, x_d, sigma_u_d, x, xdot, u, du, sigma_u);
    __syncthreads();
    *running_state_cost_nom = 0;
    *running_control_cost_nom = 0;
    float curr_state_cost = 0.0;
    dynamics->initializeDynamics(x, u, y, theta_s, 0.0, dt);
    costs->initializeCosts(y, u, theta_c, 0.0, dt);
    for (t = 0; t < num_timesteps; t++)
    {
      mppi_common::injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps, t, global_idx,
                                      thread_idy, optimization_stride, u_d, du_d, sigma_u, u, du);
      __syncthreads();

      // Now find feedback control
      for (i = 0; i < DYN_T::CONTROL_DIM; i++)
      {
        fb_control[i] = 0;
      }

      // we do not apply feedback on the nominal state z == 1
      if (thread_idz == 0)
      {
        fb_controller->k(x, x_other, t, theta_fb, fb_control);
      }

      for (i = thread_idy; i < DYN_T::CONTROL_DIM; i += BLOCKSIZE_Y)
      {
        u[i] += fb_control[i];
        // Make sure feedback is added to the modified control noise pointer
        // du_d[control_index + i] += fb_control[i];
      }

      __syncthreads();
      // Clamp the control in both the importance sampling sequence and the disturbed sequence.
      dynamics->enforceConstraints(x, u);
      __syncthreads();

      // Calculate All the costs
      if (t > 0)
      {
        curr_state_cost = costs->computeStateCost(y, t, theta_c, crash_status);
      }

      // Nominal system is where thread_idz == 1
      if (thread_idz == 1 && thread_idy == 0 && t > 0)
      {
        // This memory is shared in the y direction so limit which threads can write to it
        *running_state_cost_nom += curr_state_cost;
        *running_control_cost_nom += costs->computeLikelihoodRatioCost(u, du, sigma_u, lambda, alpha);
      }
      // Real system cost update when thread_idz == 0
      if (thread_idz == 0 && t > 0)
      {
        running_state_cost_real += curr_state_cost;
        running_control_cost_real += costs->computeLikelihoodRatioCost(u, du, sigma_u, lambda, alpha);

        running_tracking_cost_real +=
            (curr_state_cost + costs->computeFeedbackCost(fb_control, sigma_u, lambda, alpha));
      }

      __syncthreads();
      // Dynamics update
      dynamics->step(x, x_next, xdot, u, y, theta_s, t, dt);
      x_temp = x;
      x = x_next;
      x_next = x_temp;
      x_temp = x_other;
      x_other = x_next_other;
      x_next_other = x_temp;
    }

    // Compute average cost per timestep
    if (thread_idz == 1 && thread_idy == 0)
    {
      *running_state_cost_nom /= ((float)num_timesteps - 1);
      *running_control_cost_nom /= ((float)num_timesteps - 1);
    }

    if (thread_idz == 0)
    {
      running_state_cost_real /= ((float)num_timesteps - 1);
      running_tracking_cost_real /= ((float)num_timesteps - 1);
      running_control_cost_real /= ((float)num_timesteps - 1);
    }

    // calculate terminal costs
    if (thread_idz == 1 && thread_idy == 0)
    {  // Thread y required to prevent double addition
      *running_state_cost_nom += costs->terminalCost(y, theta_c) / (num_timesteps - 1);
    }

    if (thread_idz == 0)
    {
      running_state_cost_real += costs->terminalCost(y, theta_c) / (num_timesteps - 1);
      running_tracking_cost_real += costs->terminalCost(y, theta_c) / (num_timesteps - 1);
    }

    // Figure out final form of nominal cost
    __syncthreads();
    float running_cost_nom = 0;
    if (thread_idz == 0)
    {
      running_cost_nom = 0.5 * (*running_state_cost_nom) +
                         0.5 * fmaxf(fminf(running_tracking_cost_real, value_func_threshold), *running_state_cost_nom);

      running_cost_nom += *running_control_cost_nom;

      // Copy costs over to global memory
      // Actual System cost
      trajectory_costs_d[global_idx] = running_state_cost_real + running_control_cost_real;
      // Nominal System Cost - Again this is actaully only  known on real system threads
      trajectory_costs_d[global_idx + NUM_ROLLOUTS] = running_cost_nom;
    }
  }
}

/*******************************************************************************************************************
 * Launch Functions
 *******************************************************************************************************************/

template <class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int SAMPLES_PER_CONDITION>
void launchInitEvalKernel(DYN_T* dynamics, COST_T* costs, int num_candidates, int num_timesteps, float lambda,
                          float alpha, int ctrl_stride, float dt, int* strides_d, float* exploration_std_dev_d,
                          float* states_d, float* control_d, float* control_noise_d, float* costs_d,
                          hipStream_t stream, bool synchronize)
{
  int GRIDSIZE_X = num_candidates * SAMPLES_PER_CONDITION / BLOCKSIZE_X;
  static_assert(SAMPLES_PER_CONDITION % BLOCKSIZE_X == 0,
                "SAMPLES_PER_CONDITION must be evenly divided by "
                "BLOCKSIZE_X");
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
  dim3 dimGrid(GRIDSIZE_X, 1, 1);
  initEvalKernel<DYN_T, COST_T, BLOCKSIZE_X, BLOCKSIZE_Y, SAMPLES_PER_CONDITION>
      <<<dimGrid, dimBlock, 0, stream>>>(dynamics, costs, num_timesteps, lambda, alpha, ctrl_stride, dt, strides_d,
                                         exploration_std_dev_d, states_d, control_d, control_noise_d, costs_d);
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}

template <class DYN_T, class COST_T, class FB_T, int NUM_ROLLOUTS, int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
void launchRMPPIRolloutKernel(DYN_T* dynamics, COST_T* costs, FB_T* fb_controller, float dt, int num_timesteps,
                              int optimization_stride, float lambda, float alpha, float value_func_threshold,
                              float* x_d, float* u_d, float* du_d, float* sigma_u_d, float* trajectory_costs,
                              hipStream_t stream, bool synchronize)
{
  const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
  static_assert(NUM_ROLLOUTS % BLOCKSIZE_X == 0, "NUM_ROLLOUTS must be evenly divided by BLOCKSIZE_X");
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(gridsize_x, 1, 1);
  RMPPIRolloutKernel<DYN_T, COST_T, FB_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS, BLOCKSIZE_Z>
      <<<dimGrid, dimBlock, 0, stream>>>(dynamics, costs, fb_controller, dt, num_timesteps, optimization_stride, lambda,
                                         alpha, value_func_threshold, x_d, u_d, du_d, sigma_u_d, trajectory_costs);
  HANDLE_ERROR(hipGetLastError());
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(stream));
  }
}
}  // namespace rmppi_kernels
