#include "hip/hip_runtime.h"
#include "tube_mppi_controller.cuh"

#define TubeMPPI TubeMPPIController<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
TubeMPPI::TubeMPPIController(DYN_T* model, COST_T* cost, FB_T* fb_controller, float dt, int max_iter, float lambda,
                             float alpha, const Eigen::Ref<const control_array>& control_std_dev, int num_timesteps,
                             const Eigen::Ref<const control_trajectory>& init_control_traj, hipStream_t stream)
  : Controller<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
        model, cost, fb_controller, dt, max_iter, lambda, alpha, control_std_dev, num_timesteps, init_control_traj,
        stream)
{
  nominal_control_trajectory_ = init_control_traj;

  // Allocate CUDA memory for the controller
  allocateCUDAMemory();

  // Copy the noise std_dev to the device
  this->copyControlStdDevToDevice();

  // Initialize Feedback
  this->fb_controller_->initTrackingController();
  this->enable_feedback_ = true;
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::computeControl(const Eigen::Ref<const state_array>& state, int optimization_stride)
{
  if (!nominalStateInit_)
  {
    // set the nominal state to the actual state
    nominal_state_trajectory_.col(0) = state;
    nominalStateInit_ = true;
  }

  this->free_energy_statistics_.real_sys.previousBaseline = this->baseline_;
  this->free_energy_statistics_.nominal_sys.previousBaseline = this->baseline_nominal_;

  //  std::cout << "Post disturbance Actual State: "; this->model_->printState(state.data());
  //  std::cout << "                Nominal State: "; this->model_->printState(nominal_state_trajectory_.col(0).data());

  // Handy reference pointers to the nominal state
  float* trajectory_costs_nominal_d = this->trajectory_costs_d_ + NUM_ROLLOUTS;
  float* initial_state_nominal_d = this->initial_state_d_ + DYN_T::STATE_DIM;

  float* control_noise_nominal_d = this->control_noise_d_ + NUM_ROLLOUTS * this->num_timesteps_ * DYN_T::CONTROL_DIM;
  float* control_nominal_d = this->control_d_ + this->num_timesteps_ * DYN_T::CONTROL_DIM;

  for (int opt_iter = 0; opt_iter < this->num_iters_; opt_iter++)
  {
    // Send the initial condition to the device
    HANDLE_ERROR(hipMemcpyAsync(this->initial_state_d_, state.data(), DYN_T::STATE_DIM * sizeof(float),
                                 hipMemcpyHostToDevice, this->stream_));
    HANDLE_ERROR(hipMemcpyAsync(initial_state_nominal_d, nominal_state_trajectory_.data(),
                                 DYN_T::STATE_DIM * sizeof(float), hipMemcpyHostToDevice, this->stream_));

    // Send the nominal control to the device
    copyControlToDevice();

    // Generate noise data
    hiprandGenerateNormal(this->gen_, this->control_noise_d_, NUM_ROLLOUTS * this->num_timesteps_ * DYN_T::CONTROL_DIM,
                         0.0, 1.0);

    HANDLE_ERROR(hipMemcpyAsync(control_noise_nominal_d, this->control_noise_d_,
                                 NUM_ROLLOUTS * this->num_timesteps_ * DYN_T::CONTROL_DIM * sizeof(float),
                                 hipMemcpyDeviceToDevice, this->stream_));
    hipDeviceSynchronize();

    // call rollout kernel with z = 2 since we have a nominal state
    mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y, 2>(
        this->model_->model_d_, this->cost_->cost_d_, this->dt_, this->num_timesteps_, optimization_stride,
        this->lambda_, this->alpha_, this->initial_state_d_, this->control_d_, this->control_noise_d_,
        this->control_std_dev_d_, this->trajectory_costs_d_, this->stream_);

    // Copy back sampled trajectories
    this->copySampledControlFromDevice();

    // Copy the costs back to the host
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));

    HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nominal_.data(), trajectory_costs_nominal_d,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    this->baseline_ = mppi_common::computeBaselineCost(this->trajectory_costs_.data(), NUM_ROLLOUTS);

    baseline_nominal_ = mppi_common::computeBaselineCost(this->trajectory_costs_nominal_.data(), NUM_ROLLOUTS);

    // Launch the norm exponential kernel for both actual and nominal
    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X, this->trajectory_costs_d_, 1.0 / this->lambda_,
                                     this->baseline_, this->stream_);

    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X, trajectory_costs_nominal_d, 1.0 / this->lambda_,
                                     this->baseline_nominal_, this->stream_);

    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nominal_.data(), trajectory_costs_nominal_d,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    // Compute the normalizer
    this->normalizer_ = mppi_common::computeNormalizer(this->trajectory_costs_.data(), NUM_ROLLOUTS);
    normalizer_nominal_ = mppi_common::computeNormalizer(this->trajectory_costs_nominal_.data(), NUM_ROLLOUTS);

    // Compute real free energy
    mppi_common::computeFreeEnergy(this->free_energy_statistics_.real_sys.freeEnergyMean,
                                   this->free_energy_statistics_.real_sys.freeEnergyVariance,
                                   this->free_energy_statistics_.real_sys.freeEnergyModifiedVariance,
                                   this->trajectory_costs_.data(), NUM_ROLLOUTS, this->baseline_, this->lambda_);

    // Compute Nominal State free Energy
    mppi_common::computeFreeEnergy(this->free_energy_statistics_.nominal_sys.freeEnergyMean,
                                   this->free_energy_statistics_.nominal_sys.freeEnergyVariance,
                                   this->free_energy_statistics_.nominal_sys.freeEnergyModifiedVariance,
                                   this->trajectory_costs_nominal_.data(), NUM_ROLLOUTS, this->baseline_nominal_,
                                   this->lambda_);

    // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
        this->trajectory_costs_d_, this->control_noise_d_, this->control_d_, this->normalizer_, this->num_timesteps_,
        this->stream_);
    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
        trajectory_costs_nominal_d, control_noise_nominal_d, control_nominal_d, this->normalizer_nominal_,
        this->num_timesteps_, this->stream_);

    // Transfer the new control to the host
    HANDLE_ERROR(hipMemcpyAsync(this->control_.data(), this->control_d_,
                                 sizeof(float) * this->num_timesteps_ * DYN_T::CONTROL_DIM, hipMemcpyDeviceToHost,
                                 this->stream_));
    HANDLE_ERROR(hipMemcpyAsync(nominal_control_trajectory_.data(), control_nominal_d,
                                 sizeof(float) * this->num_timesteps_ * DYN_T::CONTROL_DIM, hipMemcpyDeviceToHost,
                                 this->stream_));
    hipStreamSynchronize(this->stream_);

    // Compute the nominal and actual state trajectories
    computeStateTrajectory(state);  // Input is the actual state

    //    std::cout << "Actual baseline: " << this->baseline_ << std::endl;
    //    std::cout << "Nominal baseline: " << baseline_nominal_ << std::endl;

    if (this->baseline_ < baseline_nominal_ + nominal_threshold_)
    {
      // In this case, the disturbance the made the nominal and actual states differ improved the cost.
      // std::copy(state_trajectory.begin(), state_trajectory.end(), nominal_state_trajectory_.begin());
      // std::copy(control_trajectory.begin(), control_trajectory.end(), nominal_control_.begin());
      this->free_energy_statistics_.nominal_state_used = 0;
      nominal_state_trajectory_ = this->state_;
      nominal_control_trajectory_ = this->control_;
    }
    else
    {
      this->free_energy_statistics_.nominal_state_used = 1;
    }

    // Outside of this loop, we will utilize the nominal state trajectory and the nominal control trajectory to compute
    // the optimal feedback gains using our ancillary controller, then apply feedback inside our main while loop at the
    // same rate as our state estimator.
  }
  smoothControlTrajectory();
  computeStateTrajectory(state);  // Input is the actual state

  this->free_energy_statistics_.real_sys.normalizerPercent = this->normalizer_ / NUM_ROLLOUTS;
  this->free_energy_statistics_.real_sys.increase =
      this->baseline_ - this->free_energy_statistics_.real_sys.previousBaseline;
  this->free_energy_statistics_.nominal_sys.normalizerPercent = this->normalizer_nominal_ / NUM_ROLLOUTS;
  this->free_energy_statistics_.nominal_sys.increase =
      this->baseline_nominal_ - this->free_energy_statistics_.nominal_sys.previousBaseline;
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::copyControlToDevice()
{
  HANDLE_ERROR(hipMemcpyAsync(this->control_d_, this->control_.data(), sizeof(float) * this->control_.size(),
                               hipMemcpyHostToDevice, this->stream_));

  HANDLE_ERROR(hipMemcpyAsync(this->control_d_ + this->control_.size(), nominal_control_trajectory_.data(),
                               sizeof(float) * nominal_control_trajectory_.size(), hipMemcpyHostToDevice,
                               this->stream_));
  HANDLE_ERROR(hipStreamSynchronize(this->stream_));
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::allocateCUDAMemory()
{
  Controller<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>::allocateCUDAMemoryHelper(1);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::slideControlSequence(int steps)
{
  // Propagate the nominal trajectory forward
  updateNominalState(nominal_control_trajectory_.col(0));

  // Save the control history
  this->saveControlHistoryHelper(steps, nominal_control_trajectory_, this->control_history_);

  this->slideControlSequenceHelper(steps, nominal_control_trajectory_);
  this->slideControlSequenceHelper(steps, this->control_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::smoothControlTrajectory()
{
  this->smoothControlTrajectoryHelper(nominal_control_trajectory_, this->control_history_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::computeStateTrajectory(const Eigen::Ref<const state_array>& x0_actual)
{
  // update the nominal state
  this->computeStateTrajectoryHelper(nominal_state_trajectory_, nominal_state_trajectory_.col(0),
                                     nominal_control_trajectory_);
  // update the actual state
  this->computeStateTrajectoryHelper(this->state_, x0_actual, this->control_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::updateNominalState(const Eigen::Ref<const control_array>& u)
{
  state_array xdot;
  this->model_->computeDynamics(nominal_state_trajectory_.col(0), u, xdot);
  this->model_->updateState(nominal_state_trajectory_.col(0), xdot, this->dt_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void TubeMPPI::calculateSampledStateTrajectories()
{
  int num_sampled_trajectories = this->perc_sampled_control_trajectories * NUM_ROLLOUTS;
  std::vector<int> samples = mppi_math::sample_without_replacement(num_sampled_trajectories, NUM_ROLLOUTS);

  // TODO hipMalloc and free
  // get the current controls at sampled locations

  float* sampled_noise_d_nom =
      this->sampled_noise_d_ + num_sampled_trajectories * this->num_timesteps_ * DYN_T::CONTROL_DIM;
  int nom_corrector = NUM_ROLLOUTS * this->num_timesteps_ * DYN_T::CONTROL_DIM;
  if (this->baseline_ < baseline_nominal_ + nominal_threshold_)
  {
    nom_corrector = 0;
    // initial nominal state needs to be real state when we switch to real
    HANDLE_ERROR(hipMemcpyAsync(this->initial_state_d_ + DYN_T::STATE_DIM, this->initial_state_d_,
                                 sizeof(float) * DYN_T::STATE_DIM, hipMemcpyDeviceToDevice, this->stream_));
  }
  for (int i = 0; i < num_sampled_trajectories; i++)
  {
    // copy real over
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_noise_d_ + i * this->num_timesteps_ * DYN_T::CONTROL_DIM,
                                 this->control_noise_d_ + samples[i] * this->num_timesteps_ * DYN_T::CONTROL_DIM,
                                 sizeof(float) * this->num_timesteps_ * DYN_T::CONTROL_DIM, hipMemcpyDeviceToDevice,
                                 this->stream_));
    // copy nominal over
    HANDLE_ERROR(hipMemcpyAsync(
        sampled_noise_d_nom + i * this->num_timesteps_ * DYN_T::CONTROL_DIM,
        this->control_noise_d_ + nom_corrector + samples[i] * this->num_timesteps_ * DYN_T::CONTROL_DIM,
        sizeof(float) * this->num_timesteps_ * DYN_T::CONTROL_DIM, hipMemcpyDeviceToDevice, this->stream_));
  }
  HANDLE_ERROR(hipStreamSynchronize(this->stream_));

  // run kernel
  mppi_common::launchStateTrajectoryKernel<DYN_T, FEEDBACK_GPU, BDIM_X, BDIM_Y, 2, false>(
      this->model_->model_d_, this->fb_controller_->getDevicePointer(), this->sampled_noise_d_, this->initial_state_d_,
      this->sampled_states_d_, num_sampled_trajectories, this->num_timesteps_, this->dt_, this->stream_);

  // copy back results
  for (int i = 0; i < num_sampled_trajectories * 2; i++)
  {
    HANDLE_ERROR(hipMemcpyAsync(
        this->sampled_trajectories_[i].data(), this->sampled_states_d_ + i * this->num_timesteps_ * DYN_T::STATE_DIM,
        this->num_timesteps_ * DYN_T::STATE_DIM * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
  }
  HANDLE_ERROR(hipStreamSynchronize(this->stream_));
}
