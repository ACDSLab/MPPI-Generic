#include "hip/hip_runtime.h"
#include <mppi/controllers/MPPI/mppi_controller.cuh>
#include <mppi/core/mppi_common.cuh>
#include <algorithm>
#include <iostream>

#define VanillaMPPI VanillaMPPIController<DYN_T, COST_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
VanillaMPPI::VanillaMPPIController(DYN_T* model, COST_T* cost,
                                   float dt,
                                   int max_iter,
                                   float lambda,
                                   float alpha,
                                   const Eigen::Ref<const control_array>& control_std_dev,
                                   int num_timesteps,
                                   const Eigen::Ref<const control_trajectory>& init_control_traj,
                                   hipStream_t stream) :
Controller<DYN_T, COST_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(model, cost, dt,
        max_iter, lambda, alpha, control_std_dev, num_timesteps, init_control_traj, stream) {
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();

  // Copy the noise std_dev to the device
  this->copyControlStdDevToDevice();
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
VanillaMPPI::~VanillaMPPIController() {
  // all implemented in standard controller
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::computeControl(const Eigen::Ref<const state_array>& state, int optimization_stride) {

  // Send the initial condition to the device
  HANDLE_ERROR( hipMemcpyAsync(this->initial_state_d_, state.data(),
      DYN_T::STATE_DIM*sizeof(float), hipMemcpyHostToDevice, this->stream_));

  float baseline_prev = 1e8;

  for (int opt_iter = 0; opt_iter < this->num_iters_; opt_iter++) {
    // Send the nominal control to the device
    this->copyNominalControlToDevice();

    //Generate noise data
    hiprandGenerateNormal(this->gen_, this->control_noise_d_,
                         NUM_ROLLOUTS*this->num_timesteps_*DYN_T::CONTROL_DIM,
                         0.0, 1.0);
    /*
    std::vector<float> noise = this->getSampledNoise();
    float mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    float std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 1 side N(%f, %f)\n", mean, std_dev);
     */

    //Launch the rollout kernel
    mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
        this->model_->model_d_, this->cost_->cost_d_, this->dt_, this->num_timesteps_,
        optimization_stride, this->lambda_, this->alpha_,
        this->initial_state_d_, this->control_d_, this->control_noise_d_,
        this->control_std_dev_d_, this->trajectory_costs_d_, this->stream_);
    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 2 side N(%f, %f)\n", mean, std_dev);
     */

    // Copy back sampled trajectories
    this->copySampledControlFromDevice();
    // Copy the costs back to the host
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(),
        this->trajectory_costs_d_,
        NUM_ROLLOUTS*sizeof(float),
        hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR( hipStreamSynchronize(this->stream_) );

    this->baseline_ = mppi_common::computeBaselineCost(this->trajectory_costs_.data(),
        NUM_ROLLOUTS);

    if (this->baseline_ > baseline_prev + 1) {
      // TODO handle printing
      if (this->debug_) {
        std::cout << "Previous Baseline: " << baseline_prev << std::endl;
        std::cout << "         Baseline: " << this->baseline_ << std::endl;
      }

    }

    baseline_prev = this->baseline_;

    // Launch the norm exponential kernel
    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X,
        this->trajectory_costs_d_, 1.0/this->lambda_, this->baseline_, this->stream_);
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(),
        this->trajectory_costs_d_,
        NUM_ROLLOUTS*sizeof(float),
        hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    // Compute the normalizer
    this->normalizer_ = mppi_common::computeNormalizer(this->trajectory_costs_.data(),
        NUM_ROLLOUTS);

    // TODO Find lambda and also add it to this method call
    mppi_common::computeFreeEnergy(this->free_energy_, this->free_energy_var_,
                                   this->trajectory_costs_.data(), NUM_ROLLOUTS,
                                   this->baseline_);

    // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
            this->trajectory_costs_d_, this->control_noise_d_, this->control_d_,
            this->normalizer_, this->num_timesteps_, this->stream_);

    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 3 side N(%f, %f)\n", mean, std_dev);
     */

    // Transfer the new control to the host
    HANDLE_ERROR( hipMemcpyAsync(this->control_.data(), this->control_d_,
            sizeof(float)*this->num_timesteps_*DYN_T::CONTROL_DIM,
            hipMemcpyDeviceToHost, this->stream_));
    hipStreamSynchronize(this->stream_);

    }
  smoothControlTrajectory();
  computeStateTrajectory(state);

}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::allocateCUDAMemory() {
  Controller<DYN_T, COST_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>::allocateCUDAMemoryHelper();
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::computeStateTrajectory(const Eigen::Ref<const state_array>& x0) {
  this->computeStateTrajectoryHelper(this->state_, x0, this->control_);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::slideControlSequence(int steps) {
  // TODO does the logic of handling control history reasonable?

  // Save the control history
  this->saveControlHistoryHelper(steps, this->control_, this->control_history_);

  this->slideControlSequenceHelper(steps, this->control_);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void VanillaMPPI::smoothControlTrajectory() {
  this->smoothControlTrajectoryHelper(this->control_, this->control_history_);
}

#undef VanillaMPPI
