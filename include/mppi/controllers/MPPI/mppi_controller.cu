#include "hip/hip_runtime.h"
#include <mppi/controllers/MPPI/mppi_controller.cuh>
#include <mppi/core/mppi_common.cuh>
#include <algorithm>
#include <iostream>

#define VanillaMPPI VanillaMPPIController<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y, PARAMS_T>

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
VanillaMPPI::VanillaMPPIController(DYN_T* model, COST_T* cost, FB_T* fb_controller, float dt, int max_iter,
                                   float lambda, float alpha, const Eigen::Ref<const control_array>& control_std_dev,
                                   int num_timesteps, const Eigen::Ref<const control_trajectory>& init_control_traj,
                                   hipStream_t stream)
  : PARENT_CLASS(model, cost, fb_controller, dt, max_iter, lambda, alpha, control_std_dev, num_timesteps,
                 init_control_traj, stream)
{
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();

  // Copy the noise std_dev to the device
  this->copyControlStdDevToDevice();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
VanillaMPPI::VanillaMPPIController(DYN_T* model, COST_T* cost, FB_T* fb_controller, PARAMS_T& params,
                                   hipStream_t stream)
  : PARENT_CLASS(model, cost, fb_controller, params, stream)
{
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();

  // Copy the noise std_dev to the device
  this->copyControlStdDevToDevice();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
VanillaMPPI::~VanillaMPPIController()
{
  // all implemented in standard controller
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void VanillaMPPI::computeControl(const Eigen::Ref<const state_array>& state, int optimization_stride)
{
  this->free_energy_statistics_.real_sys.previousBaseline = this->baseline_;

  // Send the initial condition to the device
  HANDLE_ERROR(hipMemcpyAsync(this->initial_state_d_, state.data(), DYN_T::STATE_DIM * sizeof(float),
                               hipMemcpyHostToDevice, this->stream_));

  float baseline_prev = 1e8;

  for (int opt_iter = 0; opt_iter < this->getNumIters(); opt_iter++)
  {
    // Send the nominal control to the device
    this->copyNominalControlToDevice();

    // Generate noise data
    hiprandGenerateNormal(this->gen_, this->control_noise_d_,
                         NUM_ROLLOUTS * this->getNumTimesteps() * DYN_T::CONTROL_DIM, 0.0, 1.0);
    /*
    std::vector<float> noise = this->getSampledNoise();
    float mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    float std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 1 side N(%f, %f)\n", mean, std_dev);
     */

    // Launch the rollout kernel
    mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
        this->model_->model_d_, this->cost_->cost_d_, this->getDt(), this->getNumTimesteps(), optimization_stride,
        this->getLambda(), this->getAlpha(), this->initial_state_d_, this->control_d_, this->control_noise_d_,
        this->control_std_dev_d_, this->trajectory_costs_d_, this->stream_);
    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 2 side N(%f, %f)\n", mean, std_dev);
     */

    // Copy the costs back to the host
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    this->baseline_ = mppi_common::computeBaselineCost(this->trajectory_costs_.data(), NUM_ROLLOUTS);

    if (this->baseline_ > baseline_prev + 1)
    {
      // TODO handle printing
      if (this->debug_)
      {
        std::cout << "Previous Baseline: " << baseline_prev << std::endl;
        std::cout << "         Baseline: " << this->baseline_ << std::endl;
      }
    }

    baseline_prev = this->baseline_;

    // Launch the norm exponential kernel
    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X, this->trajectory_costs_d_, 1.0 / this->getLambda(),
                                     this->baseline_, this->stream_);
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    // Compute the normalizer
    this->normalizer_ = mppi_common::computeNormalizer(this->trajectory_costs_.data(), NUM_ROLLOUTS);

    mppi_common::computeFreeEnergy(this->free_energy_statistics_.real_sys.freeEnergyMean,
                                   this->free_energy_statistics_.real_sys.freeEnergyVariance,
                                   this->free_energy_statistics_.real_sys.freeEnergyModifiedVariance,
                                   this->trajectory_costs_.data(), NUM_ROLLOUTS, this->baseline_, this->getLambda());

    // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
        this->trajectory_costs_d_, this->control_noise_d_, this->control_d_, this->normalizer_, this->getNumTimesteps(),
        this->stream_);

    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 3 side N(%f, %f)\n", mean, std_dev);
     */

    // Transfer the new control to the host
    HANDLE_ERROR(hipMemcpyAsync(this->control_.data(), this->control_d_,
                                 sizeof(float) * this->getNumTimesteps() * DYN_T::CONTROL_DIM, hipMemcpyDeviceToHost,
                                 this->stream_));
    hipStreamSynchronize(this->stream_);
  }

  this->free_energy_statistics_.real_sys.normalizerPercent = this->normalizer_ / NUM_ROLLOUTS;
  this->free_energy_statistics_.real_sys.increase =
      this->baseline_ - this->free_energy_statistics_.real_sys.previousBaseline;
  smoothControlTrajectory();
  computeStateTrajectory(state);
  state_array zero_state = state_array::Zero();
  for (int i = 0; i < this->getNumTimesteps(); i++)
  {
    this->model_->enforceConstraints(zero_state, this->control_.col(i));
  }

  // Copy back sampled trajectories
  this->copySampledControlFromDevice();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void VanillaMPPI::allocateCUDAMemory()
{
  PARENT_CLASS::allocateCUDAMemoryHelper();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void VanillaMPPI::computeStateTrajectory(const Eigen::Ref<const state_array>& x0)
{
  this->computeStateTrajectoryHelper(this->state_, x0, this->control_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void VanillaMPPI::slideControlSequence(int steps)
{
  // TODO does the logic of handling control history reasonable?

  // Save the control history
  this->saveControlHistoryHelper(steps, this->control_, this->control_history_);

  this->slideControlSequenceHelper(steps, this->control_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void VanillaMPPI::smoothControlTrajectory()
{
  this->smoothControlTrajectoryHelper(this->control_, this->control_history_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void VanillaMPPI::calculateSampledStateTrajectories()
{
  int num_sampled_trajectories =
      this->perc_sampled_control_trajectories_ * NUM_ROLLOUTS + this->num_top_control_trajectories_;

  // control already copied in compute control, so run kernel
  mppi_common::launchStateAndCostTrajectoryKernel<DYN_T, COST_T, FEEDBACK_GPU, BDIM_X, BDIM_Y>(
      this->model_->model_d_, this->cost_->cost_d_, this->fb_controller_->getDevicePointer(), this->sampled_noise_d_,
      this->initial_state_d_, this->sampled_states_d_, this->sampled_costs_d_, this->sampled_crash_status_d_,
      num_sampled_trajectories, this->getNumTimesteps(), this->getDt(), this->vis_stream_);

  for (int i = 0; i < num_sampled_trajectories; i++)
  {
    // set initial state to the first location
    this->sampled_trajectories_[i].col(0) = this->state_.col(0);
    // shifted by one since we do not save the initial state
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_trajectories_[i].data() + (DYN_T::STATE_DIM),
                                 this->sampled_states_d_ + i * this->getNumTimesteps() * DYN_T::STATE_DIM,
                                 (this->getNumTimesteps() - 1) * DYN_T::STATE_DIM * sizeof(float),
                                 hipMemcpyDeviceToHost, this->vis_stream_));
    HANDLE_ERROR(
        hipMemcpyAsync(this->sampled_costs_[i].data(), this->sampled_costs_d_ + (i * (this->getNumTimesteps() + 1)),
                        (this->getNumTimesteps() + 1) * sizeof(float), hipMemcpyDeviceToHost, this->vis_stream_));
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_crash_status_[i].data(),
                                 this->sampled_crash_status_d_ + (i * this->getNumTimesteps()),
                                 this->getNumTimesteps() * sizeof(int), hipMemcpyDeviceToHost, this->vis_stream_));
  }
  HANDLE_ERROR(hipStreamSynchronize(this->vis_stream_));
}

#undef VanillaMPPI
