#include "hip/hip_runtime.h"
#include <mppi/controllers/MPPI/mppi_controller.cuh>
#include <mppi/core/mppi_common.cuh>
#include <algorithm>
#include <iostream>
#include <mppi/sampling_distributions/colored_noise/colored_noise.cuh>

#define VanillaMPPI VanillaMPPIController<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
VanillaMPPI::VanillaMPPIController(DYN_T* model, COST_T* cost, FB_T* fb_controller, float dt, int max_iter,
                                   float lambda, float alpha, const Eigen::Ref<const control_array>& control_std_dev,
                                   int num_timesteps, const Eigen::Ref<const control_trajectory>& init_control_traj,
                                   hipStream_t stream)
  : Controller<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
        model, cost, fb_controller, dt, max_iter, lambda, alpha, control_std_dev, num_timesteps, init_control_traj,
        stream)
{
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();

  // Copy the noise std_dev to the device
  this->copyControlStdDevToDevice();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
VanillaMPPI::~VanillaMPPIController()
{
  // all implemented in standard controller
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void VanillaMPPI::computeControl(const Eigen::Ref<const state_array>& state, int optimization_stride)
{
  this->free_energy_statistics_.real_sys.previousBaseline = this->baseline_;

  // Send the initial condition to the device
  HANDLE_ERROR(hipMemcpyAsync(this->initial_state_d_, state.data(), DYN_T::STATE_DIM * sizeof(float),
                               hipMemcpyHostToDevice, this->stream_));

  float baseline_prev = 1e8;

  for (int opt_iter = 0; opt_iter < this->num_iters_; opt_iter++)
  {
    // Send the nominal control to the device
    this->copyNominalControlToDevice();

    // Generate noise data
    powerlaw_psd_gaussian(1.0, this->num_timesteps_, NUM_ROLLOUTS, DYN_T::CONTROL_DIM, this->control_noise_d_,
                          this->gen_, this->stream_);
    // hiprandGenerateNormal(this->gen_, this->control_noise_d_, NUM_ROLLOUTS * this->num_timesteps_ *
    // DYN_T::CONTROL_DIM,
    //                      0.0, 1.0);
    /*
    std::vector<float> noise = this->getSampledNoise();
    float mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    float std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 1 side N(%f, %f)\n", mean, std_dev);
     */

    // Launch the rollout kernel
    mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
        this->model_->model_d_, this->cost_->cost_d_, this->dt_, this->num_timesteps_, optimization_stride,
        this->lambda_, this->alpha_, this->initial_state_d_, this->control_d_, this->control_noise_d_,
        this->control_std_dev_d_, this->trajectory_costs_d_, this->stream_);
    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 2 side N(%f, %f)\n", mean, std_dev);
     */

    // Copy the costs back to the host
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    this->baseline_ = mppi_common::computeBaselineCost(this->trajectory_costs_.data(), NUM_ROLLOUTS);

    if (this->baseline_ > baseline_prev + 1)
    {
      // TODO handle printing
      if (this->debug_)
      {
        std::cout << "Previous Baseline: " << baseline_prev << std::endl;
        std::cout << "         Baseline: " << this->baseline_ << std::endl;
      }
    }

    baseline_prev = this->baseline_;

    // Launch the norm exponential kernel
    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X, this->trajectory_costs_d_, 1.0 / this->lambda_,
                                     this->baseline_, this->stream_);
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    // Compute the normalizer
    this->normalizer_ = mppi_common::computeNormalizer(this->trajectory_costs_.data(), NUM_ROLLOUTS);

    mppi_common::computeFreeEnergy(this->free_energy_statistics_.real_sys.freeEnergyMean,
                                   this->free_energy_statistics_.real_sys.freeEnergyVariance,
                                   this->free_energy_statistics_.real_sys.freeEnergyModifiedVariance,
                                   this->trajectory_costs_.data(), NUM_ROLLOUTS, this->baseline_, this->lambda_);

    // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
        this->trajectory_costs_d_, this->control_noise_d_, this->control_d_, this->normalizer_, this->num_timesteps_,
        this->stream_);

    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 3 side N(%f, %f)\n", mean, std_dev);
     */

    // Transfer the new control to the host
    HANDLE_ERROR(hipMemcpyAsync(this->control_.data(), this->control_d_,
                                 sizeof(float) * this->num_timesteps_ * DYN_T::CONTROL_DIM, hipMemcpyDeviceToHost,
                                 this->stream_));
    hipStreamSynchronize(this->stream_);
  }

  this->free_energy_statistics_.real_sys.normalizerPercent = this->normalizer_ / NUM_ROLLOUTS;
  this->free_energy_statistics_.real_sys.increase =
      this->baseline_ - this->free_energy_statistics_.real_sys.previousBaseline;
  smoothControlTrajectory();
  computeStateTrajectory(state);
  state_array zero_state = state_array::Zero();
  // for (int i = 0; i < this->num_timesteps_; i++) {
  //   this->model_->enforceConstraints(zero_state, this->control_.col(i));
  // }

  // Copy back sampled trajectories
  this->copySampledControlFromDevice();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void VanillaMPPI::allocateCUDAMemory()
{
  Controller<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>::allocateCUDAMemoryHelper();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void VanillaMPPI::computeStateTrajectory(const Eigen::Ref<const state_array>& x0)
{
  this->computeStateTrajectoryHelper(this->state_, x0, this->control_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void VanillaMPPI::slideControlSequence(int steps)
{
  // TODO does the logic of handling control history reasonable?

  // Save the control history
  this->saveControlHistoryHelper(steps, this->control_, this->control_history_);

  this->slideControlSequenceHelper(steps, this->control_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void VanillaMPPI::smoothControlTrajectory()
{
  this->smoothControlTrajectoryHelper(this->control_, this->control_history_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
void VanillaMPPI::calculateSampledStateTrajectories()
{
  int num_sampled_trajectories = this->perc_sampled_control_trajectories_ * NUM_ROLLOUTS;
  std::vector<int> samples = mppi_math::sample_without_replacement(num_sampled_trajectories, NUM_ROLLOUTS);

  // TODO hipMalloc and free
  // get the current controls at sampled locations

  // controls already copied in compute control

  mppi_common::launchStateAndCostTrajectoryKernel<DYN_T, COST_T, FEEDBACK_GPU, BDIM_X, BDIM_Y>(
      this->model_->model_d_, this->cost_->cost_d_, this->fb_controller_->getDevicePointer(), this->sampled_noise_d_,
      this->initial_state_d_, this->sampled_states_d_, this->sampled_costs_d_, this->sampled_crash_status_d_,
      num_sampled_trajectories, this->num_timesteps_, this->dt_, this->vis_stream_);

  for (int i = 0; i < num_sampled_trajectories; i++)
  {
    // set initial state to the first location
    this->sampled_trajectories_[i].col(0) = this->state_.col(0);
    // shifted by one since we do not save the initial state
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_trajectories_[i].data() + (DYN_T::STATE_DIM),
                                 this->sampled_states_d_ + i * this->num_timesteps_ * DYN_T::STATE_DIM,
                                 (this->num_timesteps_ - 1) * DYN_T::STATE_DIM * sizeof(float), hipMemcpyDeviceToHost,
                                 this->vis_stream_));
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_costs_[i].data(), this->sampled_costs_d_ + (i * this->num_timesteps_),
                                 this->num_timesteps_ * sizeof(float), hipMemcpyDeviceToHost, this->vis_stream_));
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_crash_status_[i].data(),
                                 this->sampled_crash_status_d_ + (i * this->num_timesteps_),
                                 this->num_timesteps_ * sizeof(float), hipMemcpyDeviceToHost, this->vis_stream_));
  }
  HANDLE_ERROR(hipStreamSynchronize(this->vis_stream_));
}

#undef VanillaMPPI
