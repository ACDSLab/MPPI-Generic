#include "hip/hip_runtime.h"
#include <mppi/controllers/ColoredMPPI/colored_mppi_controller.cuh>
#include <mppi/core/mppi_common_new.cuh>
#include <mppi/core/mppi_common.cuh>
#include <algorithm>
#include <iostream>
#include <mppi/sampling_distributions/colored_noise/colored_noise.cuh>

#define ColoredMPPI_TEMPLATE                                                                                           \
  template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, class SAMPLING_T,              \
            class PARAMS_T>
#define ColoredMPPI ColoredMPPIController<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, SAMPLING_T, PARAMS_T>

ColoredMPPI_TEMPLATE ColoredMPPI::ColoredMPPIController(DYN_T* model, COST_T* cost, FB_T* fb_controller,
                                                        SAMPLING_T* sampler, float dt, int max_iter, float lambda,
                                                        float alpha, int num_timesteps,
                                                        const Eigen::Ref<const control_trajectory>& init_control_traj,
                                                        hipStream_t stream)
  : PARENT_CLASS(model, cost, fb_controller, sampler, dt, max_iter, lambda, alpha, num_timesteps, init_control_traj,
                 stream)
{
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();

  // Copy the noise std_dev to the device
  // this->copyControlStdDevToDevice();
  chooseAppropriateKernel();
}

ColoredMPPI_TEMPLATE ColoredMPPI::ColoredMPPIController(DYN_T* model, COST_T* cost, FB_T* fb_controller,
                                                        SAMPLING_T* sampler, PARAMS_T& params, hipStream_t stream)
  : PARENT_CLASS(model, cost, fb_controller, sampler, params, stream)
{
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();

  // // Copy the noise std_dev to the device
  // this->copyControlStdDevToDevice();
  chooseAppropriateKernel();
}

ColoredMPPI_TEMPLATE void ColoredMPPI::chooseAppropriateKernel()
{
  hipDeviceProp_t deviceProp;
  HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, 0));
  unsigned single_kernel_byte_size = mppi::kernels::calcRolloutCombinedKernelSharedMemSize(
      this->model_, this->cost_, this->sampler_, this->params_.dynamics_rollout_dim_);
  unsigned split_dyn_kernel_byte_size = mppi::kernels::calcRolloutDynamicsKernelSharedMemSize(
      this->model_, this->sampler_, this->params_.dynamics_rollout_dim_);
  unsigned split_cost_kernel_byte_size =
      mppi::kernels::calcRolloutCostKernelSharedMemSize(this->cost_, this->sampler_, this->params_.cost_rollout_dim_);
  unsigned vis_single_kernel_byte_size = mppi::kernels::calcVisualizeKernelSharedMemSize(
      this->model_, this->cost_, this->sampler_, this->getNumTimesteps(), this->params_.visualize_dim_);

  bool too_much_mem_single_kernel = single_kernel_byte_size > deviceProp.sharedMemPerBlock;
  bool too_much_mem_vis_kernel = vis_single_kernel_byte_size > deviceProp.sharedMemPerBlock;
  bool too_much_mem_split_kernel = split_dyn_kernel_byte_size > deviceProp.sharedMemPerBlock;
  too_much_mem_split_kernel = too_much_mem_split_kernel || split_cost_kernel_byte_size > deviceProp.sharedMemPerBlock;
  too_much_mem_single_kernel = too_much_mem_single_kernel || too_much_mem_vis_kernel;

  if (too_much_mem_split_kernel && too_much_mem_single_kernel)
  {
    std::string error_msg =
        "There is not enough shared memory on the GPU for either rollout kernel option. The combined rollout kernel "
        "takes " +
        std::to_string(single_kernel_byte_size) + " bytes, the cost rollout kernel takes " +
        std::to_string(split_cost_kernel_byte_size) + " bytes, the dynamics rollout kernel takes " +
        std::to_string(split_dyn_kernel_byte_size) + " bytes, the combined visualization kernel takes " +
        std::to_string(vis_single_kernel_byte_size) + " bytes, and the max is " +
        std::to_string(deviceProp.sharedMemPerBlock) +
        " bytes. Considering lowering the corresponding thread block sizes.";
    throw std::runtime_error(error_msg);
  }
  else if (too_much_mem_single_kernel)
  {
    this->setKernelChoice(kernelType::USE_SPLIT_KERNELS);
    return;
  }
  else if (too_much_mem_split_kernel)
  {
    this->setKernelChoice(kernelType::USE_SINGLE_KERNEL);
    return;
  }

  // Send the nominal control to the device
  this->copyNominalControlToDevice(false);
  state_array zero_state = this->model_->getZeroState();
  // Send zero state to the device
  HANDLE_ERROR(hipMemcpyAsync(this->initial_state_d_, zero_state.data(), DYN_T::STATE_DIM * sizeof(float),
                               hipMemcpyHostToDevice, this->stream_));
  // Generate noise data
  this->sampler_->generateSamples(1, 0, this->gen_, true);

  float single_kernel_time_ms = std::numeric_limits<float>::infinity();
  float split_kernel_time_ms = std::numeric_limits<float>::infinity();

  // Evaluate each kernel that is applicable
  auto start_single_kernel_time = std::chrono::steady_clock::now();
  for (int i = 0; i < this->getNumKernelEvaluations() && !too_much_mem_single_kernel; i++)
  {
    mppi::kernels::launchRolloutKernel<DYN_T, COST_T, SAMPLING_T>(
        this->model_->model_d_, this->cost_->cost_d_, this->sampler_->sampling_d_, this->getDt(),
        this->getNumTimesteps(), NUM_ROLLOUTS, this->getLambda(), this->getAlpha(), this->initial_state_d_,
        this->trajectory_costs_d_, this->params_.dynamics_rollout_dim_, this->stream_, true);
  }
  auto end_single_kernel_time = std::chrono::steady_clock::now();
  auto start_split_kernel_time = std::chrono::steady_clock::now();
  for (int i = 0; i < this->getNumKernelEvaluations() && !too_much_mem_split_kernel; i++)
  {
    mppi::kernels::launchSplitRolloutKernel<DYN_T, COST_T, SAMPLING_T>(
        this->model_->model_d_, this->cost_->cost_d_, this->sampler_->sampling_d_, this->getDt(),
        this->getNumTimesteps(), NUM_ROLLOUTS, this->getLambda(), this->getAlpha(), this->initial_state_d_,
        this->output_d_, this->trajectory_costs_d_, this->params_.dynamics_rollout_dim_,
        this->params_.cost_rollout_dim_, this->stream_, true);
  }
  auto end_split_kernel_time = std::chrono::steady_clock::now();

  // calc times
  if (!too_much_mem_single_kernel)
  {
    single_kernel_time_ms = mppi::math::timeDiffms(end_single_kernel_time, start_single_kernel_time);
  }
  if (!too_much_mem_split_kernel)
  {
    split_kernel_time_ms = mppi::math::timeDiffms(end_split_kernel_time, start_split_kernel_time);
  }
  std::string kernel_choice = "";
  if (split_kernel_time_ms < single_kernel_time_ms)
  {
    this->setKernelChoice(kernelType::USE_SPLIT_KERNELS);
    kernel_choice = "split ";
  }
  else
  {
    this->setKernelChoice(kernelType::USE_SINGLE_KERNEL);
    kernel_choice = "single";
  }
  this->logger_->info("Choosing %s kernel based on split taking %f ms and single taking %f ms after %d iterations\n",
                     kernel_choice.c_str(), split_kernel_time_ms, single_kernel_time_ms,
                     this->getNumKernelEvaluations());
}

ColoredMPPI_TEMPLATE ColoredMPPI::~ColoredMPPIController()
{
  // all implemented in standard controller
}

ColoredMPPI_TEMPLATE void ColoredMPPI::computeControl(const Eigen::Ref<const state_array>& state,
                                                      int optimization_stride)
{
  this->free_energy_statistics_.real_sys.previousBaseline = this->getBaselineCost();
  state_array local_state = state;

  if (getLeashActive())
  {
    this->model_->enforceLeash(state, this->state_.col(leash_jump_), this->params_.state_leash_dist_, local_state);
  }

  // Send the initial condition to the device
  HANDLE_ERROR(hipMemcpyAsync(this->initial_state_d_, local_state.data(), DYN_T::STATE_DIM * sizeof(float),
                               hipMemcpyHostToDevice, this->stream_));

  float baseline_prev = 1e8;
  for (int opt_iter = 0; opt_iter < this->getNumIters(); opt_iter++)
  {
    // Send the nominal control to the device
    this->copyNominalControlToDevice(false);

    // Generate noise data
    this->sampler_->generateSamples(optimization_stride, opt_iter, this->gen_, false);
    // Launch the rollout kernel
    if (this->getKernelChoiceAsEnum() == kernelType::USE_SPLIT_KERNELS)
    {
      mppi::kernels::launchSplitRolloutKernel<DYN_T, COST_T, SAMPLING_T>(
          this->model_->model_d_, this->cost_->cost_d_, this->sampler_->sampling_d_, this->getDt(),
          this->getNumTimesteps(), NUM_ROLLOUTS, this->getLambda(), this->getAlpha(), this->initial_state_d_,
          this->output_d_, this->trajectory_costs_d_, this->params_.dynamics_rollout_dim_,
          this->params_.cost_rollout_dim_, this->stream_, false);
    }
    else if (this->getKernelChoiceAsEnum() == kernelType::USE_SINGLE_KERNEL)
    {
      mppi::kernels::launchRolloutKernel<DYN_T, COST_T, SAMPLING_T>(
          this->model_->model_d_, this->cost_->cost_d_, this->sampler_->sampling_d_, this->getDt(),
          this->getNumTimesteps(), NUM_ROLLOUTS, this->getLambda(), this->getAlpha(), this->initial_state_d_,
          this->trajectory_costs_d_, this->params_.dynamics_rollout_dim_, this->stream_, false);
    }

    // Copy the costs back to the host
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    this->setBaseline(mppi_common::computeBaselineCost(this->trajectory_costs_.data(), NUM_ROLLOUTS));

    if (this->getBaselineCost() > baseline_prev + 1)
    {
      this->logger_->debug("Previous Baseline: %f\n         Baseline: %f\n", baseline_prev, this->getBaselineCost());
    }

    baseline_prev = this->getBaselineCost();

    // Launch the norm exponential kernel
    if (getGamma() == 0 || getRExp() == 0)
    {
      mppi_common::launchNormExpKernel(NUM_ROLLOUTS, this->getNormExpThreads(), this->trajectory_costs_d_,
                                       1.0 / this->getLambda(), this->getBaselineCost(), this->stream_, false);
    }
    else
    {
      mppi_common::launchTsallisKernel(NUM_ROLLOUTS, this->getNormExpThreads(), this->trajectory_costs_d_, getGamma(),
                                       getRExp(), this->getBaselineCost(), this->stream_, false);
    }
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));
    // Compute the normalizer
    this->setNormalizer(mppi_common::computeNormalizer(this->trajectory_costs_.data(), NUM_ROLLOUTS));

    mppi_common::computeFreeEnergy(this->free_energy_statistics_.real_sys.freeEnergyMean,
                                   this->free_energy_statistics_.real_sys.freeEnergyVariance,
                                   this->free_energy_statistics_.real_sys.freeEnergyModifiedVariance,
                                   this->trajectory_costs_.data(), NUM_ROLLOUTS, this->getBaselineCost(),
                                   this->getLambda());

    // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
    this->sampler_->updateDistributionParamsFromDevice(this->trajectory_costs_d_, this->getNormalizerCost(), 0, false);

    // Transfer the new control to the host
    this->sampler_->setHostOptimalControlSequence(this->control_.data(), 0, true);
  }

  this->free_energy_statistics_.real_sys.normalizerPercent = this->getNormalizerCost() / NUM_ROLLOUTS;
  this->free_energy_statistics_.real_sys.increase =
      this->getBaselineCost() - this->free_energy_statistics_.real_sys.previousBaseline;
  smoothControlTrajectory();
  computeStateTrajectory(local_state);
  state_array zero_state = this->model_->getZeroState();
  for (int i = 0; i < this->getNumTimesteps(); i++)
  {
    // this->model_->enforceConstraints(zero_state, this->control_.col(i));
    this->control_.col(i)[1] =
        fminf(fmaxf(this->control_.col(i)[1], this->model_->control_rngs_[1].x), this->model_->control_rngs_[1].y);
  }

  // Copy back sampled trajectories
  this->copySampledControlFromDevice(false);
  if (this->getKernelChoiceAsEnum() == kernelType::USE_SINGLE_KERNEL)
  {  // copy initial state to vis initial state for use with visualizeKernel
    HANDLE_ERROR(hipMemcpyAsync(this->vis_initial_state_d_, this->initial_state_d_, sizeof(float) * DYN_T::STATE_DIM,
                                 hipMemcpyDeviceToDevice, this->vis_stream_));
  }
  this->copyTopControlFromDevice(true);
}

ColoredMPPI_TEMPLATE void ColoredMPPI::allocateCUDAMemory()
{
  PARENT_CLASS::allocateCUDAMemoryHelper();
}

ColoredMPPI_TEMPLATE void ColoredMPPI::computeStateTrajectory(const Eigen::Ref<const state_array>& x0)
{
  this->computeOutputTrajectoryHelper(this->output_, this->state_, x0, this->control_);
}

ColoredMPPI_TEMPLATE void ColoredMPPI::slideControlSequence(int steps)
{
  // TODO does the logic of handling control history reasonable?
  leash_jump_ = steps;
  // Save the control history
  this->saveControlHistoryHelper(steps, this->control_, this->control_history_);

  this->slideControlSequenceHelper(steps, this->control_);
}

ColoredMPPI_TEMPLATE void ColoredMPPI::smoothControlTrajectory()
{
  this->smoothControlTrajectoryHelper(this->control_, this->control_history_);
}

ColoredMPPI_TEMPLATE void ColoredMPPI::calculateSampledStateTrajectories()
{
  int num_sampled_trajectories = this->getTotalSampledTrajectories();
  // controls already copied in compute control
  if (this->getKernelChoiceAsEnum() == kernelType::USE_SPLIT_KERNELS)
  {
    mppi::kernels::launchVisualizeCostKernel<COST_T, SAMPLING_T>(
        this->cost_->cost_d_, this->sampler_->sampling_d_, this->getDt(), this->getNumTimesteps(),
        num_sampled_trajectories, this->getLambda(), this->getAlpha(), this->sampled_outputs_d_,
        this->sampled_crash_status_d_, this->sampled_costs_d_, this->params_.cost_rollout_dim_, this->stream_, false);
  }
  else if (this->getKernelChoiceAsEnum() == kernelType::USE_SINGLE_KERNEL)
  {
    mppi::kernels::launchVisualizeKernel<DYN_T, COST_T, SAMPLING_T>(
        this->model_->model_d_, this->cost_->cost_d_, this->sampler_->sampling_d_, this->getDt(),
        this->getNumTimesteps(), num_sampled_trajectories, this->getLambda(), this->getAlpha(),
        this->vis_initial_state_d_, this->sampled_outputs_d_, this->sampled_costs_d_, this->sampled_crash_status_d_,
        this->params_.visualize_dim_, this->stream_, false);
  }
  // #if true
  //   mppi::kernels::launchVisualizeCostKernel<COST_T, SAMPLING_T>(
  //       this->cost_->cost_d_, this->sampler_->sampling_d_, this->getDt(), this->getNumTimesteps(),
  //       num_sampled_trajectories, this->getLambda(), this->getAlpha(), this->sampled_outputs_d_,
  //       this->sampled_crash_status_d_, this->sampled_costs_d_, this->params_.cost_rollout_dim_, this->vis_stream_,
  //       false);
  // #else
  //   mppi_common::launchVisualizeCostKernel<COST_T, 128, COST_B_Y, 1>(
  //       this->cost_->cost_d_, this->getDt(), this->getNumTimesteps(), num_sampled_trajectories, this->getLambda(),
  //       this->getAlpha(), this->sampled_outputs_d_, this->sampled_noise_d_, this->sampled_crash_status_d_,
  //       this->control_std_dev_d_, this->sampled_costs_d_, this->vis_stream_, false);
  // #endif
  for (int i = 0; i < num_sampled_trajectories; i++)
  {
    // set initial state to the first location
    // shifted by one since we do not save the initial state
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_trajectories_[i].data(),
                                 this->sampled_outputs_d_ + i * this->getNumTimesteps() * DYN_T::OUTPUT_DIM,
                                 (this->getNumTimesteps() - 1) * DYN_T::OUTPUT_DIM * sizeof(float),
                                 hipMemcpyDeviceToHost, this->vis_stream_));
    HANDLE_ERROR(
        hipMemcpyAsync(this->sampled_costs_[i].data(), this->sampled_costs_d_ + (i * (this->getNumTimesteps() + 1)),
                        (this->getNumTimesteps() + 1) * sizeof(float), hipMemcpyDeviceToHost, this->vis_stream_));
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_crash_status_[i].data(),
                                 this->sampled_crash_status_d_ + (i * this->getNumTimesteps()),
                                 this->getNumTimesteps() * sizeof(float), hipMemcpyDeviceToHost, this->vis_stream_));
  }
  HANDLE_ERROR(hipStreamSynchronize(this->vis_stream_));
}

#undef ColoredMPPI
