#include "hip/hip_runtime.h"
#include <mppi/controllers/Primitives/primitives_controller.cuh>
#include <mppi/core/mppi_common.cuh>
#include <algorithm>
#include <iostream>
#include <mppi/sampling_distributions/piecewise_linear/piecewise_linear_noise.cuh>
#include <mppi/sampling_distributions/colored_noise/colored_noise.cuh>

#define Primitives PrimitivesController<DYN_T, COST_T, FB_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y, PARAMS_T>

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
Primitives::PrimitivesController(DYN_T* model, COST_T* cost, FB_T* fb_controller, float dt, int max_iter, float lambda,
                                 float alpha, const Eigen::Ref<const control_array>& control_std_dev, int num_timesteps,
                                 const Eigen::Ref<const control_trajectory>& init_control_traj, hipStream_t stream)
  : PARENT_CLASS(model, cost, fb_controller, dt, max_iter, lambda, alpha, control_std_dev, num_timesteps,
                 init_control_traj, stream)
{
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();
  std::vector<float> tmp_vec(DYN_T::CONTROL_DIM, 0.0);
  getColoredNoiseExponentsLValue() = std::move(tmp_vec);
  getScalePiecewiseNoiseLValue() = std::move(tmp_vec);

  // Copy the noise std_dev to the device
  this->copyControlStdDevToDevice();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
Primitives::PrimitivesController(DYN_T* model, COST_T* cost, FB_T* fb_controller, PARAMS_T& params, hipStream_t stream)
  : PARENT_CLASS(model, cost, fb_controller, params, stream)
{
  // Allocate CUDA memory for the controller
  allocateCUDAMemory();
  if (this->getColoredNoiseExponentsLValue().size() == 0)
  {
    std::vector<float> tmp_vec(DYN_T::CONTROL_DIM, 0.0);
    getColoredNoiseExponentsLValue() = std::move(tmp_vec);
  }
  if (this->getScalePiecewiseNoiseLValue().size() == 0)
  {
    std::vector<float> tmp_vec(DYN_T::CONTROL_DIM, 0.0);
    getScalePiecewiseNoiseLValue() = std::move(tmp_vec);
  }

  // Copy the noise std_dev to the device
  this->copyControlStdDevToDevice();
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
Primitives::~PrimitivesController()
{
  // all implemented in standard controller
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::computeControl(const Eigen::Ref<const state_array>& state, int optimization_stride)
{
  // this->free_energy_statistics_.real_sys.previousBaseline = this->getBaselineCost();
  state_array local_state = state;
  for (int i = 0; i < DYN_T::STATE_DIM; i++)
  {
    float diff = fabsf(this->state_.col(leash_jump_)[i] - state[i]);
    if (getStateLeashLength(i) < diff)
    {
      local_state[i] = state[i];
    }
    else
    {
      local_state[i] = this->state_.col(leash_jump_)[i];
    }
  }

  // Send the initial condition to the device
  HANDLE_ERROR(hipMemcpyAsync(this->initial_state_d_, local_state.data(), DYN_T::STATE_DIM * sizeof(float),
                               hipMemcpyHostToDevice, this->stream_));

  /////////////////
  // BEGIN INTERMEDIATE PLANNER
  // Compute intermediate plan using piecewise linear noise, and choosing the best

  int prev_controls_idx = 1;
  float primitives_baseline = 0.0;
  float baseline_prev = 0.0;

  // Send the nominal control to the device
  this->copyNominalControlToDevice(false);

  for (int opt_iter = 0; opt_iter < getNumPrimitiveIterations(); opt_iter++)
  {
    powerlaw_psd_gaussian(getColoredNoiseExponentsLValue(), this->getNumTimesteps(), NUM_ROLLOUTS,
                          this->control_noise_d_, this->gen_, this->stream_);

    // Generate piecewise linear noise data, update control_noise_d_
    piecewise_linear_noise(this->getNumTimesteps(), NUM_ROLLOUTS, DYN_T::CONTROL_DIM, getPiecewiseSegments(),
                           optimization_stride, getScalePiecewiseNoiseLValue(), getFracRandomNoiseTrajLValue(),
                           getScaleAddNominalNoiseLValue(), this->control_d_, this->control_noise_d_,
                           this->control_std_dev_d_, this->gen_, this->stream_);

    // Set nominal controls to zero because we want to use the noise directly
    this->control_ = control_trajectory::Zero();

    // Send the zero nominal control to the device
    this->copyNominalControlToDevice();

    // Launch the rollout kernel
    mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
        this->model_->model_d_, this->cost_->cost_d_, this->getDt(), this->getNumTimesteps(),
        /*optimization_stride = */ 0, this->getLambda(), this->getAlpha(), this->initial_state_d_, this->control_d_,
        this->control_noise_d_, this->control_std_dev_d_, this->trajectory_costs_d_, this->stream_, false);

    // Copy the costs back to the host
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    primitives_baseline = mppi_common::computeBaselineCost(this->trajectory_costs_.data(), NUM_ROLLOUTS);

    // get previous control cost (at index = 1, since index = 0 is zero control traj)
    baseline_prev = this->trajectory_costs_.data()[prev_controls_idx];
    if (this->debug_)
    {
      std::cerr << "Previous Baseline: " << baseline_prev << "         Baseline: " << this->getBaselineCost()
                << std::endl;
    }

    // if baseline is too high and trajectory is unsafe, create and issue a stopping trajectory
    // reminder:  baseline_ is the average cost along trajectory
    if (getStoppingCostThreshold() > 0 && primitives_baseline * this->getNumTimesteps() > getStoppingCostThreshold())
    {
      std::cerr << "Baseline is too high, issuing stopping trajectory!" << std::endl;
      computeStoppingTrajectory(local_state);
      primitives_baseline = std::numeric_limits<float>::min();
    }
    else if (primitives_baseline > baseline_prev - getHysteresisCostThreshold())
    {
      // baseline is not decreasing enough, use controls from the previous iteration
      if (this->debug_)
      {
        std::cerr << "Not enough improvement, use prev controls." << std::endl;
      }
      HANDLE_ERROR(hipMemcpyAsync(
          this->control_.data(),
          this->control_noise_d_ + prev_controls_idx * this->getNumTimesteps() * DYN_T::CONTROL_DIM,
          sizeof(float) * this->getNumTimesteps() * DYN_T::CONTROL_DIM, hipMemcpyDeviceToHost, this->stream_));

      primitives_baseline = baseline_prev;
    }
    else
    {  // otherwise, update the nominal control
      // Copy best control from device to the host
      int best_idx = mppi_common::computeBestIndex(this->trajectory_costs_.data(), NUM_ROLLOUTS);
      HANDLE_ERROR(hipMemcpyAsync(
          this->control_.data(), this->control_noise_d_ + best_idx * this->getNumTimesteps() * DYN_T::CONTROL_DIM,
          sizeof(float) * this->getNumTimesteps() * DYN_T::CONTROL_DIM, hipMemcpyDeviceToHost, this->stream_));
    }

    this->copyNominalControlToDevice(false);

    hipStreamSynchronize(this->stream_);
  }

  // Copy back sampled trajectories for visualization
  if (getVisualizePrimitives())
  {
    this->copySampledControlFromDevice(false);
    this->copyTopControlFromDevice(true);
  }

  //  END INTERMEDIATE PLANNER
  ////////////////

  ////////////////
  // BEGIN MPPI
  for (int opt_iter = 0; opt_iter < this->getNumIters(); opt_iter++)
  {
    // Send the nominal control to the device
    copyMPPIControlToDevice(false);

    // Generate noise data
    powerlaw_psd_gaussian(getColoredNoiseExponentsLValue(), this->getNumTimesteps(), NUM_ROLLOUTS,
                          this->control_noise_d_, this->gen_, this->stream_);
    // hiprandGenerateNormal(this->gen_, this->control_noise_d_, NUM_ROLLOUTS * this->getNumTimesteps() *
    // DYN_T::CONTROL_DIM,
    //                      0.0, 1.0);
    /*
    std::vector<float> noise = this->getSampledNoise();
    float mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    float std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 1 side N(%f, %f)\n", mean, std_dev);
     */

    // Launch the rollout kernel
    mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
        this->model_->model_d_, this->cost_->cost_d_, this->getDt(), this->getNumTimesteps(), optimization_stride,
        this->getLambda(), this->getAlpha(), this->initial_state_d_, control_mppi_d_, this->control_noise_d_,
        this->control_std_dev_d_, this->trajectory_costs_d_, this->stream_, false);
    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 2 side N(%f, %f)\n", mean, std_dev);
     */

    // Copy the costs back to the host
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    this->setBaseline(mppi_common::computeBaselineCost(this->trajectory_costs_.data(), NUM_ROLLOUTS));

    // if (this->getBaselineCost() > baseline_prev + 1)
    // {
    //   // TODO handle printing
    //   if (this->debug_)
    //   {
    //     std::cout << "Previous Baseline: " << baseline_prev << std::endl;
    //     std::cout << "         Baseline: " << this->getBaselineCost() << std::endl;
    //   }
    // }

    // baseline_prev = this->getBaselineCost();

    // Launch the norm exponential kernel
    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X, this->trajectory_costs_d_, 1.0 / this->getLambda(),
                                     this->getBaselineCost(), this->stream_, false);
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(), this->trajectory_costs_d_,
                                 NUM_ROLLOUTS * sizeof(float), hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    // Compute the normalizer
    this->setNormalizer(mppi_common::computeNormalizer(this->trajectory_costs_.data(), NUM_ROLLOUTS));

    mppi_common::computeFreeEnergy(this->free_energy_statistics_.real_sys.freeEnergyMean,
                                   this->free_energy_statistics_.real_sys.freeEnergyVariance,
                                   this->free_energy_statistics_.real_sys.freeEnergyModifiedVariance,
                                   this->trajectory_costs_.data(), NUM_ROLLOUTS, this->getBaselineCost(),
                                   this->getLambda());

    // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
        this->trajectory_costs_d_, this->control_noise_d_, control_mppi_d_, this->getNormalizerCost(),
        this->getNumTimesteps(), this->stream_, false);

    /*
    noise = this->getSampledNoise();
    mean = 0;
    for(int k = 0; k < noise.size(); k++) {
      mean += (noise[k]/noise.size());
    }

    std_dev = 0;
    for(int k = 0; k < noise.size(); k++) {
      std_dev += powf(noise[k] - mean, 2);
    }
    std_dev = sqrt(std_dev/noise.size());
    printf("CPU 3 side N(%f, %f)\n", mean, std_dev);
     */

    // Transfer the new control to the host
    HANDLE_ERROR(hipMemcpyAsync(control_mppi_.data(), control_mppi_d_,
                                 sizeof(float) * this->getNumTimesteps() * DYN_T::CONTROL_DIM, hipMemcpyDeviceToHost,
                                 this->stream_));
    hipStreamSynchronize(this->stream_);
  }

  this->free_energy_statistics_.real_sys.normalizerPercent = this->getNormalizerCost() / NUM_ROLLOUTS;
  this->free_energy_statistics_.real_sys.increase =
      this->getBaselineCost() - this->free_energy_statistics_.real_sys.previousBaseline;

  // END MPPI
  ////////////////////////

  // decide between using the MPPI control or the primitives control
  if (this->debug_)
  {
    std::cerr << "mppi baseline: " << this->getBaselineCost() << ", primitives baseline: " << primitives_baseline
              << ", prev baseline: " << baseline_prev << std::endl;
  }
  if ((getNumPrimitiveIterations() == 0 && this->getNumIters() > 0) ||
      ((getNumPrimitiveIterations() > 0 && this->getNumIters() > 0) &&
       (this->getBaselineCost() < primitives_baseline - getHysteresisCostThreshold())))
  {
    this->control_ = control_mppi_;
    this->copyNominalControlToDevice();
    if (this->debug_)
    {
      std::cerr << "Using MPPI control" << std::endl;
    }
  }

  // smoothControlTrajectory();
  computeStateTrajectory(local_state);
  state_array zero_state = state_array::Zero();
  for (int i = 0; i < this->getNumTimesteps(); i++)
  {
    this->model_->enforceConstraints(zero_state, this->control_.col(i));
    this->model_->enforceConstraints(zero_state, control_mppi_.col(i));
    // this->control_.col(i)[1] =
    //     fminf(fmaxf(this->control_.col(i)[1], this->model_->control_rngs_[1].x), this->model_->control_rngs_[1].y);
  }

  // Copy back sampled trajectories for visualization
  if (!getVisualizePrimitives())
  {
    this->copySampledControlFromDevice(false);
    this->copyTopControlFromDevice(true);
  }
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::allocateCUDAMemory()
{
  PARENT_CLASS::allocateCUDAMemoryHelper();
  HANDLE_ERROR(hipMalloc((void**)&control_mppi_d_, sizeof(float) * DYN_T::CONTROL_DIM * MAX_TIMESTEPS));
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::copyMPPIControlToDevice(bool synchronize)
{
  HANDLE_ERROR(hipMemcpyAsync(control_mppi_d_, control_mppi_.data(), sizeof(float) * control_mppi_.size(),
                               hipMemcpyHostToDevice, this->stream_));
  if (synchronize)
  {
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));
  }
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::computeStateTrajectory(const Eigen::Ref<const state_array>& x0)
{
  this->computeStateTrajectoryHelper(this->state_, x0, this->control_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::computeStoppingTrajectory(const Eigen::Ref<const state_array>& x0)
{
  state_array xdot;
  state_array state = x0;
  state_array xnext;
  output_array output;
  control_array u_i = control_array::Zero();
  this->model_->initializeDynamics(state, u_i, 0, this->getDt());
  for (int i = 0; i < this->getNumTimesteps() - 1; ++i)
  {
    this->model_->getStoppingControl(state, u_i);
    this->model_->enforceConstraints(state, u_i);
    this->control_.col(i) = u_i;
    this->model_->step(state, xnext, xdot, u_i, output, i, this->getDt());
    state = xnext;
    // this->model_->computeStateDeriv(state, u_i, xdot);
    // this->model_->updateState(state, xdot, this->getDt());
  }
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::slideControlSequence(int steps)
{
  // TODO does the logic of handling control history reasonable?
  leash_jump_ = steps;
  // Save the control history
  this->saveControlHistoryHelper(steps, this->control_, this->control_history_);

  this->slideControlSequenceHelper(steps, this->control_);
  this->slideControlSequenceHelper(steps, control_mppi_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::smoothControlTrajectory()
{
  this->smoothControlTrajectoryHelper(this->control_, this->control_history_);
}

template <class DYN_T, class COST_T, class FB_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y,
          class PARAMS_T>
void Primitives::calculateSampledStateTrajectories()
{
  int num_sampled_trajectories = this->getTotalSampledTrajectories();
  // controls already copied in compute control

  mppi_common::launchStateAndCostTrajectoryKernel<DYN_T, COST_T, FEEDBACK_GPU, BDIM_X, BDIM_Y>(
      this->model_->model_d_, this->cost_->cost_d_, this->fb_controller_->getDevicePointer(), this->sampled_noise_d_,
      this->initial_state_d_, this->sampled_states_d_, this->sampled_costs_d_, this->sampled_crash_status_d_,
      num_sampled_trajectories, this->getNumTimesteps(), this->getDt(), this->vis_stream_);

  for (int i = 0; i < num_sampled_trajectories; i++)
  {
    // set initial state to the first location
    this->sampled_trajectories_[i].col(0) = this->state_.col(0);
    // shifted by one since we do not save the initial state
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_trajectories_[i].data() + (DYN_T::STATE_DIM),
                                 this->sampled_states_d_ + i * this->getNumTimesteps() * DYN_T::STATE_DIM,
                                 (this->getNumTimesteps() - 1) * DYN_T::STATE_DIM * sizeof(float),
                                 hipMemcpyDeviceToHost, this->vis_stream_));
    HANDLE_ERROR(
        hipMemcpyAsync(this->sampled_costs_[i].data(), this->sampled_costs_d_ + (i * (this->getNumTimesteps() + 1)),
                        (this->getNumTimesteps() + 1) * sizeof(float), hipMemcpyDeviceToHost, this->vis_stream_));
    HANDLE_ERROR(hipMemcpyAsync(this->sampled_crash_status_[i].data(),
                                 this->sampled_crash_status_d_ + (i * this->getNumTimesteps()),
                                 this->getNumTimesteps() * sizeof(float), hipMemcpyDeviceToHost, this->vis_stream_));
  }
  HANDLE_ERROR(hipStreamSynchronize(this->vis_stream_));
}

#undef Primitives
