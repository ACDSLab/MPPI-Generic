#include <mppi/cost_functions/cost.cuh>

template<class CLASS_T, class PARAMS_T, int S_DIM, int C_DIM>
void Cost<CLASS_T, PARAMS_T, S_DIM, C_DIM>::paramsToDevice() {
  if(GPUMemStatus_){
    HANDLE_ERROR(hipMemcpyAsync(&cost_d_->params_, &params_,
                                 sizeof(PARAMS_T), hipMemcpyHostToDevice,
                                 stream_));
    HANDLE_ERROR(hipStreamSynchronize(stream_));
  }
}


template<class CLASS_T, class PARAMS_T, int S_DIM, int C_DIM>
void Cost<CLASS_T, PARAMS_T, S_DIM, C_DIM>::freeCudaMem() {
  if(GPUMemStatus_) {
    hipFree(cost_d_);
    GPUMemStatus_ = false;
    cost_d_ = nullptr;
  }
}

template<class CLASS_T, class PARAMS_T, int S_DIM, int C_DIM>
void Cost<CLASS_T, PARAMS_T, S_DIM, C_DIM>::GPUSetup() {
  CLASS_T* derived = static_cast<CLASS_T*>(this);
  if (!GPUMemStatus_) {
    cost_d_ = Managed::GPUSetup<CLASS_T>(derived);
  } else {
    std::cout << "GPU Memory already set" << std::endl; //TODO should this be an exception?
  }
  derived->paramsToDevice();
}
