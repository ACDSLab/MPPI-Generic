#include "hip/hip_runtime.h"
#include <mppi/cost_functions/autorally/ar_standard_cost.cuh>

template <class CLASS_T, class PARAMS_T>
ARStandardCost<CLASS_T, PARAMS_T>::ARStandardCost(hipStream_t stream) {
  //std::cout << __PRETTY_FUNCTION__ << std::endl;
  this->bindToStream(stream);
}

template <class CLASS_T, class PARAMS_T>
ARStandardCost<CLASS_T, PARAMS_T>::~ARStandardCost() {
  //std::cout << __PRETTY_FUNCTION__ << std::endl;
  if(this->GPUMemStatus_) {
    freeCudaMem();
    this->GPUMemStatus_ = false;
  }
}

template <class CLASS_T, class PARAMS_T>
void ARStandardCost<CLASS_T, PARAMS_T>::freeCudaMem() {
  // TODO free everything
  Cost<ARStandardCost<CLASS_T, PARAMS_T>, PARAMS_T>::freeCudaMem();
}

template <class CLASS_T, class PARAMS_T>
void ARStandardCost<CLASS_T, PARAMS_T>::paramsToDevice() {
  HANDLE_ERROR( hipMemcpyAsync(&this->cost_d_->params_, &this->params_, sizeof(PARAMS_T), hipMemcpyHostToDevice, this->stream_));
  HANDLE_ERROR( hipMemcpyAsync(&this->cost_d_->width_, &width_, sizeof(float), hipMemcpyHostToDevice, this->stream_));
  HANDLE_ERROR( hipMemcpyAsync(&this->cost_d_->height_, &height_, sizeof(float), hipMemcpyHostToDevice, this->stream_));
  HANDLE_ERROR( hipStreamSynchronize(this->stream_));
}

template <class CLASS_T, class PARAMS_T>
bool ARStandardCost<CLASS_T, PARAMS_T>::changeCostmapSize(int width, int height) {
  // TODO set flag at top that indicates memory allocation changes
  if(height < 0 && width < 0) {
    std::cerr << "ERROR: cannot resize costmap to size less than 1" << std::endl;
    return false;
  }
  if(height != height_ || width != width_) {
    track_costs_.resize(width * height);

    //Allocate memory for the cuda array which is bound the costmap_tex_
    // has been allocated in the past, must be freed
    if(height_ > 0 && width_ > 0) {
      HANDLE_ERROR(hipFreeArray(costmapArray_d_));
    }
    // 4 floats of size 32 bits
    channelDesc_ = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    HANDLE_ERROR(hipMallocArray(&costmapArray_d_, &channelDesc_, width, height));

    // set all of the elements in the array to be zero
    std::vector<float4> zero_array(width_*height_);
    zero_array.resize(width*height, make_float4(0,0,0,0));
    HANDLE_ERROR(hipMemcpyToArray(costmapArray_d_, 0, 0, zero_array.data(), width*height*sizeof(float4), hipMemcpyHostToDevice));
  }

  width_ = width;
  height_ = height;
  return true;
}

template <class CLASS_T, class PARAMS_T>
void ARStandardCost<CLASS_T, PARAMS_T>::clearCostmapCPU(int width, int height) {
  changeCostmapSize(width, height);

  if(width_ < 0 && height_ < 0) {
    return;
  }

  for (int i = 0; i < width_*height_; i++){
    track_costs_[i].x = 0;
    track_costs_[i].y = 0;
    track_costs_[i].z = 0;
    track_costs_[i].w = 0;
  }
}

template <class CLASS_T, class PARAMS_T>
std::vector<float4> ARStandardCost<CLASS_T, PARAMS_T>::loadTrackData(std::string map_path) {
  // check if file exists
  if(!fileExists(map_path)) {
    std::cerr << "ERROR: map path invalid, " << map_path << std::endl;
    return std::vector<float4>();
  }

  // load the npz file
  cnpy::npz_t map_dict = cnpy::npz_load(map_path);
  float x_min, x_max, y_min, y_max, ppm;
  float* xBounds = map_dict["xBounds"].data<float>();
  float* yBounds = map_dict["yBounds"].data<float>();
  float* pixelsPerMeter = map_dict["pixelsPerMeter"].data<float>();
  x_min = xBounds[0];
  x_max = xBounds[1];
  y_min = yBounds[0];
  y_max = yBounds[1];
  ppm = pixelsPerMeter[0];

  int width = int((x_max - x_min)*ppm);
  int height = int((y_max - y_min)*ppm);

  if(!changeCostmapSize(width, height)) {
    std::cerr << "ERROR: load track has invalid sizes" << std::endl;
    return std::vector<float4>();
  }

  float* channel0 = map_dict["channel0"].data<float>();
  float* channel1 = map_dict["channel1"].data<float>();
  float* channel2 = map_dict["channel2"].data<float>();
  float* channel3 = map_dict["channel3"].data<float>();

  // copy the track data into CPU side storage
  for (int i = 0; i < width_*height_; i++){
    //std::cout << i << " = " << channel0[i] << ", " << channel1[i] << ", " << channel2[i] << ", " << channel3[i] << std::endl;
    track_costs_[i].x = channel0[i];
    track_costs_[i].y = channel1[i];
    track_costs_[i].z = channel2[i];
    track_costs_[i].w = channel3[i];
  }

  Eigen::Matrix3f R;
  Eigen::Array3f trs;

  //Save the scaling and offset
  R << 1./(x_max - x_min), 0,                  0,
          0,                  1./(y_max - y_min), 0,
          0,                  0,                  1;
  trs << -x_min/(x_max - x_min), -y_min/(y_max - y_min), 1;

  updateTransform(R, trs);
  costmapToTexture();

  return track_costs_;
}

template <class CLASS_T, class PARAMS_T>
void ARStandardCost<CLASS_T, PARAMS_T>::costmapToTexture() {
  if(width_ < 0 || height_ < 0) {
    std::cerr << "ERROR: cannot allocate texture with zero size" << std::endl;
    return;
  }

  // transfer CPU version of costmap to GPU
  float4* costmap_ptr = track_costs_.data();
  HANDLE_ERROR(hipMemcpyToArray(costmapArray_d_ , 0, 0, costmap_ptr, width_*height_*sizeof(float4), hipMemcpyHostToDevice));
  hipStreamSynchronize(this->stream_);

  //Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = costmapArray_d_;

  //Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  //Destroy current texture and create new texture object
  HANDLE_ERROR(hipDestroyTextureObject(costmap_tex_d_));
  HANDLE_ERROR(hipCreateTextureObject(&costmap_tex_d_, &resDesc, &texDesc, NULL) );

  // copy over pointers setup up on CPU code to GPU
  HANDLE_ERROR( hipMemcpyAsync(&this->cost_d_->costmapArray_d_, &costmapArray_d_, sizeof(hipArray*), hipMemcpyHostToDevice, this->stream_));
  HANDLE_ERROR( hipMemcpyAsync(&this->cost_d_->costmap_tex_d_, &costmap_tex_d_, sizeof(hipTextureObject_t), hipMemcpyHostToDevice, this->stream_));
  hipStreamSynchronize(this->stream_);
}

template <class CLASS_T, class PARAMS_T>
inline __device__ float4 ARStandardCost<CLASS_T, PARAMS_T>::queryTexture(float x, float y) const {
  //printf("\nquerying point (%f, %f)", x, y);
  return tex2D<float4>(costmap_tex_d_, x, y);
}

template <class CLASS_T, class PARAMS_T>
void ARStandardCost<CLASS_T, PARAMS_T>::updateTransform(Eigen::MatrixXf m, Eigen::ArrayXf trs) {
  this->params_.r_c1.x = m(0,0);
  this->params_.r_c1.y = m(1,0);
  this->params_.r_c1.z = m(2,0);
  this->params_.r_c2.x = m(0,1);
  this->params_.r_c2.y = m(1,1);
  this->params_.r_c2.z = m(2,1);
  this->params_.trs.x = trs(0);
  this->params_.trs.y = trs(1);
  this->params_.trs.z = trs(2);
  //Move the updated parameters to gpu memory
  if(this->GPUMemStatus_) {
    paramsToDevice();
  }
}

template <class CLASS_T, class PARAMS_T>
__host__ __device__ void ARStandardCost<CLASS_T, PARAMS_T>::coorTransform(float x, float y, float* u, float* v, float* w) {
  ////Compute a projective transform of (x, y, 0, 1)
  //printf("coordiante transform %f, %f, %f\n", params_.r_c1.x, params_.r_c2.x, params_.trs.x);
  // converts to the texture [0-1] coordinate system
  u[0] = this->params_.r_c1.x*x + this->params_.r_c2.x*y + this->params_.trs.x;
  v[0] = this->params_.r_c1.y*x + this->params_.r_c2.y*y + this->params_.trs.y;
  w[0] = this->params_.r_c1.z*x + this->params_.r_c2.z*y + this->params_.trs.z;
}

template <class CLASS_T, class PARAMS_T>
__device__ float4 ARStandardCost<CLASS_T, PARAMS_T>::queryTextureTransformed(float x, float y) {
  float u, v, w;
  coorTransform(x, y, &u, &v, &w);
  printf("input coordinates: %f, %f\n", x, y);
  //printf("\nu = %f, v = %f, w = %f", u, v, w);
  printf("transformed coordinates %f, %f = %f\n", u/w, v/w, tex2D<float4>(costmap_tex_d_, u/w, v/w).x);
  return tex2D<float4>(costmap_tex_d_, u/w, v/w);
}

template <class CLASS_T, class PARAMS_T>
Eigen::Matrix3f ARStandardCost<CLASS_T, PARAMS_T>::getRotation() {
  Eigen::Matrix3f m;
  m(0,0) = this->params_.r_c1.x;
  m(1,0) = this->params_.r_c1.y;
  m(2,0) = this->params_.r_c1.z;
  m(0,1) = this->params_.r_c2.x;
  m(1,1) = this->params_.r_c2.y;
  m(2,1) = this->params_.r_c2.z;
  m(0,2) = 0.0;
  m(1,2) = 0.0;
  m(2,2) = 1.0;
  return m;
}

template <class CLASS_T, class PARAMS_T>
Eigen::Array3f ARStandardCost<CLASS_T, PARAMS_T>::getTranslation() {
  Eigen::Array3f array;
  array(0) = this->params_.trs.x;
  array(1) = this->params_.trs.y;
  array(2) = this->params_.trs.z;
  return array;
}

template <class CLASS_T, class PARAMS_T>
inline __host__ __device__ float ARStandardCost<CLASS_T, PARAMS_T>::terminalCost(float *s) {
  return 0.0;
}

template <class CLASS_T, class PARAMS_T>
inline __host__ __device__ float ARStandardCost<CLASS_T, PARAMS_T>::getControlCost(float *u, float *du, float *vars) {
  float control_cost = 0.0;
  //printf("du %f, %f\n", du[0], du[1]);
  //printf("vars %f, %f\n", vars[0], vars[1]);
  //printf("vars %f, %f\n", u[0], u[1]);
  control_cost += this->params_.steering_coeff*du[0]*(u[0] - du[0])/(vars[0]*vars[0]);
  control_cost += this->params_.throttle_coeff*du[1]*(u[1] - du[1])/(vars[1]*vars[1]);
  return control_cost;
}

template <class CLASS_T, class PARAMS_T>
inline __host__ __device__ float ARStandardCost<CLASS_T, PARAMS_T>::getSpeedCost(float *s, int *crash) {
  float cost = 0;
  float error = s[4] - this->params_.desired_speed;
  if (l1_cost_){
    cost = fabs(error);
  }
  else {
    cost = error*error;
  }
  return (this->params_.speed_coeff*cost);
}

template <class CLASS_T, class PARAMS_T>
inline __host__ __device__ float ARStandardCost<CLASS_T, PARAMS_T>::getStabilizingCost(float *s) {
  float stabilizing_cost = 0;
  if (fabs(s[4]) > 0.001) {
    float slip = -atan(s[5]/fabs(s[4]));
    stabilizing_cost = this->params_.slip_penalty*powf(slip,2);
    if (fabs(-atan(s[5]/fabs(s[4]))) > this->params_.max_slip_ang) {
      //If the slip angle is above the max slip angle kill the trajectory.
      stabilizing_cost += this->params_.crash_coeff;
    }
  }
  return stabilizing_cost;
}

template <class CLASS_T, class PARAMS_T>
inline __host__ __device__ float ARStandardCost<CLASS_T, PARAMS_T>::getCrashCost(float *s, int *crash, int num_timestep) {
  float crash_cost = 0;
  if (crash[0] > 0) {
    crash_cost = this->params_.crash_coeff;
  }
  return crash_cost;
}

template <class CLASS_T, class PARAMS_T>
inline __device__ float ARStandardCost<CLASS_T, PARAMS_T>::getTrackCost(float *s, int *crash) {
  float track_cost = 0;

  //Compute a transformation to get the (x,y) positions of the front and back of the car.
  float x_front = s[0] + FRONT_D*__cosf(s[2]);
  float y_front = s[1] + FRONT_D*__sinf(s[2]);
  float x_back = s[0] + BACK_D*__cosf(s[2]);
  float y_back = s[1] + BACK_D*__sinf(s[2]);

  //Cost of front of the car
  printf("front before %f, %f\n", x_front, y_front);
  float track_cost_front = queryTextureTransformed(x_front, y_front).x;
  printf("front after %f, %f = %f\n", x_front, y_front, track_cost_front);
  //Cost for back of the car
  printf("back before %f, %f\n", x_back, y_back);
  float track_cost_back = queryTextureTransformed(x_back, y_back).x;
  printf("back after %f, %f = %f\n", x_back, y_back, track_cost_back);

  track_cost = (fabs(track_cost_front) + fabs(track_cost_back) )/2.0;
  if (fabs(track_cost) < this->params_.track_slop) {
    track_cost = 0;
  }
  else {
    track_cost = this->params_.track_coeff*track_cost;
  }
  if (track_cost_front >= this->params_.boundary_threshold || track_cost_back >= this->params_.boundary_threshold) {
    crash[0] = 1;
  }
  return track_cost;
}

template <class CLASS_T, class PARAMS_T>
inline __device__ float ARStandardCost<CLASS_T, PARAMS_T>::computeCost(float *s, float *u, float *du, float *vars, int *crash, int timestep) {
  float control_cost = getControlCost(u, du, vars);
  float track_cost = getTrackCost(s, crash);
  float speed_cost = getSpeedCost(s, crash);
  float crash_cost = powf(this->params_.discount, timestep)*getCrashCost(s, crash, timestep);
  float stabilizing_cost = getStabilizingCost(s);
  float cost = control_cost + speed_cost + crash_cost + track_cost + stabilizing_cost;
  if (cost > MAX_COST_VALUE || isnan(cost)) {
    cost = MAX_COST_VALUE;
  }
  return cost;
}

template <class CLASS_T, class PARAMS_T>
inline __device__ float ARStandardCost<CLASS_T, PARAMS_T>::computeRunningCost(float *s, float *u, float *du, float *vars, int timestep) {
  float control_cost = getControlCost(u, du, vars);
  int crash[1] = {0};
  float track_cost = getTrackCost(s, crash);
  float speed_cost = getSpeedCost(s, crash);
  float crash_cost = powf(this->params_.discount, timestep)*getCrashCost(s, crash, timestep);
  float stabilizing_cost = getStabilizingCost(s);
  float cost = control_cost + speed_cost + crash_cost + track_cost + stabilizing_cost;
  if (cost > MAX_COST_VALUE || isnan(cost)) {
    cost = MAX_COST_VALUE;
  }
  return cost;
}
