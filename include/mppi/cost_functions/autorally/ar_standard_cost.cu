#include "hip/hip_runtime.h"
#include <mppi/cost_functions/autorally/ar_standard_cost.cuh>

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::ARStandardCostImpl(hipStream_t stream)
{
  this->bindToStream(stream);
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
void ARStandardCostImpl<CLASS_T, PARAMS_T,  DYN_PARAMS_T>::freeCudaMem() {
  if (this->GPUMemStatus_)
  {
    HANDLE_ERROR(hipFreeArray(costmapArray_d_));
  }
  PARENT_CLASS::freeCudaMem();
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
void ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::paramsToDevice()
{
  HANDLE_ERROR(hipMemcpyAsync(&this->cost_d_->params_, &this->params_, sizeof(PARAMS_T), hipMemcpyHostToDevice,
                               this->stream_));
  HANDLE_ERROR(hipMemcpyAsync(&this->cost_d_->width_, &width_, sizeof(float), hipMemcpyHostToDevice, this->stream_));
  HANDLE_ERROR(
      hipMemcpyAsync(&this->cost_d_->height_, &height_, sizeof(float), hipMemcpyHostToDevice, this->stream_));
  HANDLE_ERROR(hipStreamSynchronize(this->stream_));
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
bool ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::changeCostmapSize(int width, int height)
{
  // TODO set flag at top that indicates memory allocation changes
  if (height < 0 && width < 0)
  {
    std::cerr << "ERROR: cannot resize costmap to size less than 1" << std::endl;
    return false;
  }
  if (height != height_ || width != width_)
  {
    track_costs_.resize(width * height);

    // Allocate memory for the cuda array which is bound the costmap_tex_
    // has been allocated in the past, must be freed
    if (height_ > 0 && width_ > 0)
    {
      HANDLE_ERROR(hipFreeArray(costmapArray_d_));
    }
    // 4 floats of size 32 bits
    channelDesc_ = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    HANDLE_ERROR(hipMallocArray(&costmapArray_d_, &channelDesc_, width, height));

    // set all of the elements in the array to be zero
    std::vector<float4> zero_array(width_ * height_);
    zero_array.resize(width * height, make_float4(0, 0, 0, 0));
    HANDLE_ERROR(hipMemcpyToArray(costmapArray_d_, 0, 0, zero_array.data(), width * height * sizeof(float4),
                                   hipMemcpyHostToDevice));
  }

  width_ = width;
  height_ = height;
  return true;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
void ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::clearCostmapCPU(int width, int height)
{
  changeCostmapSize(width, height);

  if (width_ < 0 && height_ < 0)
  {
    return;
  }

  for (int i = 0; i < width_ * height_; i++)
  {
    track_costs_[i].x = 0;
    track_costs_[i].y = 0;
    track_costs_[i].z = 0;
    track_costs_[i].w = 0;
  }
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
std::vector<float4> ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::loadTrackData(std::string map_path)
{
  // check if file exists
  if (!fileExists(map_path))
  {
    std::cerr << "ERROR: map path invalid, " << map_path << std::endl;
    return std::vector<float4>();
  }

  // load the npz file
  cnpy::npz_t map_dict = cnpy::npz_load(map_path);
  float x_min, x_max, y_min, y_max, ppm;
  float* xBounds = map_dict["xBounds"].data<float>();
  float* yBounds = map_dict["yBounds"].data<float>();
  float* pixelsPerMeter = map_dict["pixelsPerMeter"].data<float>();
  x_min = xBounds[0];
  x_max = xBounds[1];
  y_min = yBounds[0];
  y_max = yBounds[1];
  ppm = pixelsPerMeter[0];

  int width = int((x_max - x_min) * ppm);
  int height = int((y_max - y_min) * ppm);

  if (!changeCostmapSize(width, height))
  {
    std::cerr << "ERROR: load track has invalid sizes" << std::endl;
    return std::vector<float4>();
  }

  float* channel0 = map_dict["channel0"].data<float>();
  float* channel1 = map_dict["channel1"].data<float>();
  float* channel2 = map_dict["channel2"].data<float>();
  float* channel3 = map_dict["channel3"].data<float>();

  // copy the track data into CPU side storage
  for (int i = 0; i < width_ * height_; i++)
  {
    // std::cout << i << " = " << channel0[i] << ", " << channel1[i] << ", " << channel2[i] << ", " << channel3[i] <<
    // std::endl;
    track_costs_[i].x = channel0[i];
    track_costs_[i].y = channel1[i];
    track_costs_[i].z = channel2[i];
    track_costs_[i].w = channel3[i];
  }

  Eigen::Matrix3f R;
  Eigen::Array3f trs;

  // Save the scaling and offset
  R << 1. / (x_max - x_min), 0, 0, 0, 1. / (y_max - y_min), 0, 0, 0, 1;
  trs << -x_min / (x_max - x_min), -y_min / (y_max - y_min), 1;

  updateTransform(R, trs);
  costmapToTexture();

  return track_costs_;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
void ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::costmapToTexture()
{
  if (width_ < 0 || height_ < 0)
  {
    std::cerr << "ERROR: cannot allocate texture with zero size" << std::endl;
    return;
  }

  // transfer CPU version of costmap to GPU
  float4* costmap_ptr = track_costs_.data();
  HANDLE_ERROR(
      hipMemcpyToArray(costmapArray_d_, 0, 0, costmap_ptr, width_ * height_ * sizeof(float4), hipMemcpyHostToDevice));
  hipStreamSynchronize(this->stream_);

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = costmapArray_d_;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  // Destroy current texture and create new texture object
  HANDLE_ERROR(hipDestroyTextureObject(costmap_tex_d_));
  HANDLE_ERROR(hipCreateTextureObject(&costmap_tex_d_, &resDesc, &texDesc, NULL));

  // copy over pointers setup up on CPU code to GPU
  HANDLE_ERROR(hipMemcpyAsync(&this->cost_d_->costmapArray_d_, &costmapArray_d_, sizeof(hipArray*),
                               hipMemcpyHostToDevice, this->stream_));
  HANDLE_ERROR(hipMemcpyAsync(&this->cost_d_->costmap_tex_d_, &costmap_tex_d_, sizeof(hipTextureObject_t),
                               hipMemcpyHostToDevice, this->stream_));
  hipStreamSynchronize(this->stream_);
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
inline __device__ float4 ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::queryTexture(float x, float y) const
{
  // printf("\nquerying point (%f, %f)", x, y);
  return tex2D<float4>(costmap_tex_d_, x, y);
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
void ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::updateTransform(Eigen::MatrixXf m, Eigen::ArrayXf trs)
{
  this->params_.r_c1.x = m(0, 0);
  this->params_.r_c1.y = m(1, 0);
  this->params_.r_c1.z = m(2, 0);
  this->params_.r_c2.x = m(0, 1);
  this->params_.r_c2.y = m(1, 1);
  this->params_.r_c2.z = m(2, 1);
  this->params_.trs.x = trs(0);
  this->params_.trs.y = trs(1);
  this->params_.trs.z = trs(2);
  // Move the updated parameters to gpu memory
  if (this->GPUMemStatus_)
  {
    paramsToDevice();
  }
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
__host__ __device__ void ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::coorTransform(float x, float y, float* u,
                                                                                            float* v, float* w)
{
  ////Compute a projective transform of (x, y, 0, 1)
  // printf("coordiante transform %f, %f, %f\n", params_.r_c1.x, params_.r_c2.x, params_.trs.x);
  // converts to the texture [0-1] coordinate system
  u[0] = this->params_.r_c1.x * x + this->params_.r_c2.x * y + this->params_.trs.x;
  v[0] = this->params_.r_c1.y * x + this->params_.r_c2.y * y + this->params_.trs.y;
  w[0] = this->params_.r_c1.z * x + this->params_.r_c2.z * y + this->params_.trs.z;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
__host__ __device__ float4 ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::queryTextureTransformed(float x,
                                                                                                        float y)
{
  float u, v, w;
  coorTransform(x, y, &u, &v, &w);
  // printf("input coordinates: %f, %f\n", x, y);
  // printf("\nu = %f, v = %f, w = %f", u, v, w);
  // printf("transformed coordinates %f, %f = %f\n", u/w, v/w, tex2D<float4>(costmap_tex_d_, u/w, v/w).x);
#ifdef __CUDA_ARCH__
  return tex2D<float4>(costmap_tex_d_, u / w, v / w);
#else
  float2 query = make_float2(u / w * width_, v / w * height_);
  query.x = query.x - 0.5f;
  query.y = query.y - 0.5f;
  query.x = fmaxf(0.0f, fminf(width_ - 1, query.x));
  query.y = fmaxf(0.0f, fminf(height_ - 1, query.y));
  return this->track_costs_[std::round(query.y) * width_ + std::round(query.x)];
#endif
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
Eigen::Matrix3f ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::getRotation()
{
  Eigen::Matrix3f m;
  m(0, 0) = this->params_.r_c1.x;
  m(1, 0) = this->params_.r_c1.y;
  m(2, 0) = this->params_.r_c1.z;
  m(0, 1) = this->params_.r_c2.x;
  m(1, 1) = this->params_.r_c2.y;
  m(2, 1) = this->params_.r_c2.z;
  m(0, 2) = 0.0;
  m(1, 2) = 0.0;
  m(2, 2) = 1.0;
  return m;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
Eigen::Array3f ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::getTranslation()
{
  Eigen::Array3f array;
  array(0) = this->params_.trs.x;
  array(1) = this->params_.trs.y;
  array(2) = this->params_.trs.z;
  return array;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
inline __device__ float ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::terminalCost(float* s, float* theta_c)
{
  return 0.0;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
float ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::terminalCost(const Eigen::Ref<const output_array> y)
{
  return 0.0;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
inline __host__ __device__ float ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::getSpeedCost(float* s, int* crash)
{
  float cost = 0;
  float error = s[4] - this->params_.desired_speed;
  if (l1_cost_)
  {
    cost = fabs(error);
  }
  else
  {
    cost = error * error;
  }
  return (this->params_.speed_coeff * cost);
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
inline __host__ __device__ float
ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::getStabilizingCost(float* s, int* crash_status)
{
  float stabilizing_cost = 0;
  if (fabs(s[4]) > 0.001)
  {
    float slip = -atan(s[5] / fabs(s[4]));
    stabilizing_cost = this->params_.slip_coeff * powf(slip, 2);
    if (fabs(-atan(s[5] / fabs(s[4]))) > this->params_.max_slip_ang)
    {
      // If the slip angle is above the max slip angle kill the trajectory.
      stabilizing_cost += this->params_.crash_coeff;
    }
  }
  // if we roll over kill the trajectory
  if (fabs(s[3]) > M_PI_2)
  {
    crash_status[0] = 1;
  }
  // printf("stabilizing %f\n", stabilizing_cost);
  return stabilizing_cost;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
inline __host__ __device__ float ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::getCrashCost(float* s, int* crash,
                                                                                                   int num_timestep)
{
  float crash_cost = 0;
  if (crash[0] > 0)
  {
    crash_cost = this->params_.crash_coeff;
  }
  // printf("crash_cost %f\n", crash_cost);
  return crash_cost;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
inline __host__ __device__ float ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::getTrackCost(float* s, int* crash)
{
  float track_cost = 0;

  // Compute a transformation to get the (x,y) positions of the front and back of the car.
#ifdef __CUDA_ARCH__
  float x_front = s[0] + this->FRONT_D * __cosf(s[2]);
  float y_front = s[1] + this->FRONT_D * __sinf(s[2]);
  float x_back = s[0] + this->BACK_D * __cosf(s[2]);
  float y_back = s[1] + this->BACK_D * __sinf(s[2]);
#else
  float x_front = s[0] + this->FRONT_D * cosf(s[2]);
  float y_front = s[1] + this->FRONT_D * sinf(s[2]);
  float x_back = s[0] + this->BACK_D * cosf(s[2]);
  float y_back = s[1] + this->BACK_D * sinf(s[2]);
#endif

  // Cost of front of the car
  // printf("front before %f, %f\n", x_front, y_front);
  float track_cost_front = queryTextureTransformed(x_front, y_front).x;
  // printf("front after %f, %f = %f\n", x_front, y_front, track_cost_front);
  // Cost for back of the car
  // printf("back before %f, %f\n", x_back, y_back);
  float track_cost_back = queryTextureTransformed(x_back, y_back).x;
  // printf("back after %f, %f = %f\n", x_back, y_back, track_cost_back);

  track_cost = (fabs(track_cost_front) + fabs(track_cost_back)) / 2.0;
  if (fabs(track_cost) < this->params_.track_slop)
  {
    track_cost = 0;
  }
  else
  {
    track_cost = this->params_.track_coeff * track_cost;
  }
  if (track_cost_front >= this->params_.boundary_threshold || track_cost_back >= this->params_.boundary_threshold)
  {
    crash[0] = 1;
  }
  // printf("track_cost %f\n", track_cost);
  return track_cost;
}

template <class CLASS_T, class PARAMS_T, class DYN_PARAMS_T>
inline __device__ float ARStandardCostImpl<CLASS_T, PARAMS_T, DYN_PARAMS_T>::computeStateCost(float* s, int timestep,
                                                                                              float* theta_c,
                                                                                              int* crash_status)
{
  // printf("input state %f %f %f %f %f %f %f\n", s[0], s[1], s[2], s[3], s[4], s[5], s[6]);
  /*
  int global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(global_idx == 0) {
    printf("desired_speed %f\n", this->params_.desired_speed);
    printf("speed_coeff %f\n", this->params_.speed_coeff);
    printf("track_coeff %f\n", this->params_.track_coeff);
    printf("max_slip_angle %f\n", this->params_.max_slip_ang);
    printf("slip_coeff %f\n", this->params_.slip_coeff);
    printf("track_slop %f\n", this->params_.track_slop);
    printf("crash_coeff %f\n", this->params_.crash_coeff);
    printf("discount %f\n", this->params_.discount);
    printf("boundary_threshold %f\n", this->params_.boundary_threshold);
    printf("grid_res %d\n", this->params_.grid_res);
    printf("control_cost_coeff[0] %f\n", this->params_.control_cost_coeff[0]);
    printf("control_cost_coeff[1] %f\n", this->params_.control_cost_coeff[1]);
  }*/
  float track_cost = getTrackCost(s, crash_status);
  float speed_cost = getSpeedCost(s, crash_status);
  // printf("speed %f\n", speed_cost);
  float stabilizing_cost = getStabilizingCost(s, crash_status);
  float crash_cost = powf(this->params_.discount, timestep) * getCrashCost(s, crash_status, timestep);
  float cost = speed_cost + crash_cost + track_cost + stabilizing_cost;
  if (cost > MAX_COST_VALUE || isnan(cost))
  {  // TODO Handle max cost value in a generic way
    cost = MAX_COST_VALUE;
  }
  return cost;
}
