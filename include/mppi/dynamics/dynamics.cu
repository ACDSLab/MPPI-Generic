#include "hip/hip_runtime.h"
#include <mppi/dynamics/dynamics.cuh>

template <class CLASS_T, class PARAMS_T>
void Dynamics<CLASS_T, PARAMS_T>::paramsToDevice(bool synchronize)
{
  if (GPUMemStatus_)
  {
    HANDLE_ERROR(hipMemcpyAsync(&model_d_->params_, &params_, sizeof(PARAMS_T), hipMemcpyHostToDevice, stream_));

    HANDLE_ERROR(hipMemcpyAsync(&model_d_->control_rngs_, &control_rngs_, CONTROL_DIM * sizeof(float2),
                                 hipMemcpyHostToDevice, stream_));
    if (synchronize)
    {
      HANDLE_ERROR(hipStreamSynchronize(stream_));
    }
  }
}

template <class CLASS_T, class PARAMS_T>
void Dynamics<CLASS_T, PARAMS_T>::setControlRanges(std::array<float2, CONTROL_DIM>& control_rngs, bool synchronize)
{
  for (int i = 0; i < CONTROL_DIM; i++)
  {
    control_rngs_[i].x = control_rngs[i].x;
    control_rngs_[i].y = control_rngs[i].y;
  }
  if (GPUMemStatus_)
  {
    HANDLE_ERROR(hipMemcpyAsync(this->model_d_->control_rngs_, this->control_rngs_, CONTROL_DIM * sizeof(float2),
                                 hipMemcpyHostToDevice, stream_));
    if (synchronize)
    {
      HANDLE_ERROR(hipStreamSynchronize(stream_));
    }
  }
}

template <class CLASS_T, class PARAMS_T>
void Dynamics<CLASS_T, PARAMS_T>::setControlDeadbands(std::array<float, CONTROL_DIM>& control_deadband,
                                                      bool synchronize)
{
  for (int i = 0; i < CONTROL_DIM; i++)
  {
    control_deadband_[i] = control_deadband[i];
  }
  if (GPUMemStatus_)
  {
    HANDLE_ERROR(hipMemcpyAsync(this->model_d_->control_deadband_, this->control_deadband_,
                                 CONTROL_DIM * sizeof(float), hipMemcpyHostToDevice, stream_));
    if (synchronize)
    {
      HANDLE_ERROR(hipStreamSynchronize(stream_));
    }
  }
}

template <class CLASS_T, class PARAMS_T>
void Dynamics<CLASS_T, PARAMS_T>::GPUSetup()
{
  CLASS_T* derived = static_cast<CLASS_T*>(this);
  if (!GPUMemStatus_)
  {
    model_d_ = Managed::GPUSetup(derived);
  }
  else
  {
    this->logger_->debug("%s: GPU Memory already set\n", derived->getDynamicsModelName().c_str());
  }
  derived->paramsToDevice();
}

template <class CLASS_T, class PARAMS_T>
void Dynamics<CLASS_T, PARAMS_T>::freeCudaMem()
{
  if (GPUMemStatus_)
  {
    HANDLE_ERROR(hipFree(model_d_));
    GPUMemStatus_ = false;
    model_d_ = nullptr;
  }
}

template <class CLASS_T, class PARAMS_T>
__device__ inline void Dynamics<CLASS_T, PARAMS_T>::computeStateDeriv(float* state, float* control, float* state_der,
                                                                      float* theta_s)
{
  CLASS_T* derived = static_cast<CLASS_T*>(this);
  // only propagate a single state, i.e. thread.y = 0
  // find the change in x,y,theta based off of the rest of the state
  if (threadIdx.y == 0)
  {
    derived->computeKinematics(state, state_der);
  }
  derived->computeDynamics(state, control, state_der, theta_s);
}

template <class CLASS_T, class PARAMS_T>
__device__ void Dynamics<CLASS_T, PARAMS_T>::enforceConstraints(float* state, float* control)
{
  // TODO should control_rngs_ be a constant memory parameter
  int i, p_index, step;
  mppi::p1::getParallel1DIndex<mppi::p1::Parallel1Dir::THREAD_Y>(p_index, step);
  // parallelize setting the constraints with y dim
  for (i = p_index; i < CONTROL_DIM; i += step)
  {
    if (fabsf(control[i]) < this->control_deadband_[i])
    {
      control[i] = this->zero_control_[i];
    }
    else
    {
      control[i] += this->control_deadband_[i] * -mppi::math::sign(control[i]);
    }
    control[i] = fminf(fmaxf(this->control_rngs_[i].x, control[i]), this->control_rngs_[i].y);
  }
}

template <class CLASS_T, class PARAMS_T>
__device__ void Dynamics<CLASS_T, PARAMS_T>::updateState(float* state, float* next_state, float* state_der,
                                                         const float dt)
{
  int i, p_index, step;
  mppi::p1::getParallel1DIndex<mppi::p1::Parallel1Dir::THREAD_Y>(p_index, step);
  // Add the state derivative time dt to the current state.
  for (i = p_index; i < STATE_DIM; i += step)
  {
    next_state[i] = state[i] + state_der[i] * dt;
  }
}

template <class CLASS_T, class PARAMS_T>
__device__ inline void Dynamics<CLASS_T, PARAMS_T>::step(float* state, float* next_state, float* state_der,
                                                         float* control, float* output, float* theta_s, const float t,
                                                         const float dt)
{
  CLASS_T* derived = static_cast<CLASS_T*>(this);
  derived->computeStateDeriv(state, control, state_der, theta_s);
  __syncthreads();
  derived->updateState(state, next_state, state_der, dt);
  __syncthreads();
  derived->stateToOutput(next_state, output);
}

template <class CLASS_T, class PARAMS_T>
__host__ __device__ inline void Dynamics<CLASS_T, PARAMS_T>::stateToOutput(const float* __restrict__ state,
                                                                           float* __restrict__ output)
{
  // TODO this is a hack
  int p_index, step;
  mppi::p1::getParallel1DIndex<mppi::p1::Parallel1Dir::THREAD_Y>(p_index, step);
  for (int i = p_index; i < OUTPUT_DIM && i < STATE_DIM; i += step)
  {
    output[i] = state[i];
  }
}

template <class CLASS_T, class PARAMS_T>
__host__ __device__ inline void Dynamics<CLASS_T, PARAMS_T>::outputToState(const float* __restrict__ output,
                                                                           float* __restrict__ state)
{
  // TODO this is a hack
  int p_index, step;
  mppi::p1::getParallel1DIndex<mppi::p1::Parallel1Dir::THREAD_Y>(p_index, step);
  for (int i = p_index; i < OUTPUT_DIM && i < STATE_DIM; i += step)
  {
    state[i] = output[i];
  }
}

template <class CLASS_T, class PARAMS_T>
Dynamics<CLASS_T, PARAMS_T>::state_array Dynamics<CLASS_T, PARAMS_T>::getZeroState() const
{
  return state_array::Zero();
}

template <class CLASS_T, class PARAMS_T>
__host__ __device__ inline void Dynamics<CLASS_T, PARAMS_T>::getZeroState(float* state) const
{
  int p_index, step;
  mppi::p1::getParallel1DIndex<mppi::p1::Parallel1Dir::THREAD_Y>(p_index, step);
  for (int i = p_index; i < STATE_DIM; i += step)
  {
    state[i] = 0.0f;
  }
}
