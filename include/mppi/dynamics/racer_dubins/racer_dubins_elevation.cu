#include "hip/hip_runtime.h"
#include <mppi/dynamics/racer_dubins/racer_dubins_elevation.cuh>
#include <mppi/utils/math_utils.h>

void RacerDubinsElevation::GPUSetup()
{
  PARENT_CLASS* derived = static_cast<PARENT_CLASS*>(this);
  CudaCheckError();
  tex_helper_->GPUSetup();
  CudaCheckError();
  derived->GPUSetup();
  CudaCheckError();
}

void RacerDubinsElevation::freeCudaMem()
{
  tex_helper_->freeCudaMem();
}

void RacerDubinsElevation::paramsToDevice()
{
  if (this->GPUMemStatus_)
  {
    // does all the internal texture updates
    tex_helper_->copyToDevice();
    // makes sure that the device ptr sees the correct texture object
    HANDLE_ERROR(hipMemcpyAsync(&(this->model_d_->tex_helper_), &(tex_helper_->ptr_d_),
                                 sizeof(TwoDTextureHelper<float>*), hipMemcpyHostToDevice, this->stream_));
  }
  PARENT_CLASS::paramsToDevice();
}

void RacerDubinsElevation::updateState(const Eigen::Ref<const state_array> state, Eigen::Ref<state_array> next_state,
                                       Eigen::Ref<state_array> state_der, const float dt)
{
  next_state = state + state_der * dt;
  next_state(S_INDEX(YAW)) = angle_utils::normalizeAngle(next_state(S_INDEX(YAW)));
  next_state(S_INDEX(STEER_ANGLE)) =
      max(min(next_state(S_INDEX(STEER_ANGLE)), this->params_.max_steer_angle), -this->params_.max_steer_angle);
  next_state(S_INDEX(STEER_ANGLE_RATE)) = state_der(S_INDEX(STEER_ANGLE));
  next_state(S_INDEX(ACCEL_X)) = state_der(S_INDEX(VEL_X));
}

// __device__ void RacerDubinsElevation::updateState(float* state, float* next_state, float* state_der, const float dt)
// {
//   int i;
//   int tdy = threadIdx.y;
//   // Add the state derivative time dt to the current state.
//   // printf("updateState thread %d, %d = %f, %f\n", threadIdx.x, threadIdx.y, state[0], state_der[0]);
//   for (i = tdy; i < 5; i += blockDim.y)
//   {
//     next_state[i] = state[i] + state_der[i] * dt;
//     if (i == S_INDEX(VEL_X))
//     {
//       next_state[S_INDEX(ACCEL_X)] = state_der[S_INDEX(VEL_X)];
//     }
//     if (i == S_INDEX(YAW))
//     {
//       next_state[i] = angle_utils::normalizeAngle(next_state[i]);
//     }
//     if (i == S_INDEX(STEER_ANGLE))
//     {
//       next_state[S_INDEX(STEER_ANGLE)] =
//           max(min(next_state[S_INDEX(STEER_ANGLE)], this->params_.max_steer_angle), -this->params_.max_steer_angle);
//       next_state[S_INDEX(STEER_ANGLE_RATE)] = state_der[S_INDEX(STEER_ANGLE)];
//     }
//   }
// }

void RacerDubinsElevation::step(Eigen::Ref<state_array> state, Eigen::Ref<state_array> next_state,
                                Eigen::Ref<state_array> state_der, const Eigen::Ref<const control_array>& control,
                                Eigen::Ref<output_array> output, const float t, const float dt)
{
  // computeStateDeriv(state, control, state_der);
  bool enable_brake = control(0) < 0;
  int index = (abs(state(0)) > 0.2 && abs(state(0)) <= 3.0) + (abs(state(0)) > 3.0) * 2;
  // applying position throttle
  float throttle = this->params_.c_t[index] * control(0);
  float brake = this->params_.c_b[index] * control(0) * (state(0) >= 0 ? 1 : -1);
  float linear_brake_slope = 0.9f * (2 / dt);
  if (abs(state(0)) <= this->params_.c_b[index] / linear_brake_slope)
  {
    throttle = this->params_.c_t[index] * max(control(0) - this->params_.low_min_throttle, 0.0f);
    brake = linear_brake_slope * control(0) * state(0);
  }

  state_der(0) =
      (!enable_brake) * throttle + (enable_brake)*brake - this->params_.c_v[index] * state(0) + this->params_.c_0;
  if (abs(state[6]) < M_PI_2)
  {
    state_der[0] -= this->params_.gravity * sinf(state[6]);
  }
  state_der(1) = (state(0) / this->params_.wheel_base) * tan(state(4) / this->params_.steer_angle_scale[index]);
  state_der(2) = state(0) * cosf(state(1));
  state_der(3) = state(0) * sinf(state(1));
  state_der(4) = (control(1) * this->params_.steer_command_angle_scale - state(4)) * this->params_.steering_constant;
  state_der(4) = max(min(state_der(4), this->params_.max_steer_rate), -this->params_.max_steer_rate);
  // state(8) = state_der(0);

  // Integrate using racer_dubins upddateState
  updateState(state, next_state, state_der, dt);

  float pitch = 0;
  float roll = 0;

  float3 front_left = make_float3(2.981, 0.737, 0);
  float3 front_right = make_float3(2.981, -0.737, 0);
  float3 rear_left = make_float3(0, 0.737, 0);
  float3 rear_right = make_float3(0, -0.737, 0);
  front_left = make_float3(front_left.x * cosf(state(1)) - front_left.y * sinf(state(1)) + state(2),
                           front_left.x * sinf(state(1)) + front_left.y * cosf(state(1)) + state(3), 0);
  front_right = make_float3(front_right.x * cosf(state(1)) - front_right.y * sinf(state(1)) + state(2),
                            front_right.x * sinf(state(1)) + front_right.y * cosf(state(1)) + state(3), 0);
  rear_left = make_float3(rear_left.x * cosf(state(1)) - rear_left.y * sinf(state(1)) + state(2),
                          rear_left.x * sinf(state(1)) + rear_left.y * cosf(state(1)) + state(3), 0);
  rear_right = make_float3(rear_right.x * cosf(state(1)) - rear_right.y * sinf(state(1)) + state(2),
                           rear_right.x * sinf(state(1)) + rear_right.y * cosf(state(1)) + state(3), 0);
  float front_left_height = 0;
  float front_right_height = 0;
  float rear_left_height = 0;
  float rear_right_height = 0;

  if (this->tex_helper_->checkTextureUse(0))
  {
    front_left_height = this->tex_helper_->queryTextureAtWorldPose(0, front_left);
    front_right_height = this->tex_helper_->queryTextureAtWorldPose(0, front_right);
    rear_left_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_left);
    rear_right_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_right);

    float front_diff = front_left_height - front_right_height;
    front_diff = max(min(front_diff, 0.736 * 2), -0.736 * 2);
    float rear_diff = rear_left_height - rear_right_height;
    rear_diff = max(min(rear_diff, 0.736 * 2), -0.736 * 2);
    float front_roll = asinf(front_diff / (0.737 * 2));
    float rear_roll = asinf(rear_diff / (0.737 * 2));
    next_state(5) = (front_roll + rear_roll) / 2;

    float left_diff = rear_left_height - front_left_height;
    left_diff = max(min(left_diff, 2.98), -2.98);
    float right_diff = rear_right_height - front_right_height;
    right_diff = max(min(right_diff, 2.98), -2.98);
    float left_pitch = asinf((left_diff) / 2.981);
    float right_pitch = asinf((right_diff) / 2.981);
    next_state(6) = (left_pitch + right_pitch) / 2;
  }
  else
  {
    next_state(5) = 0;
    next_state(6) = 0;
  }

  if (isnan(next_state(5)) || isinf(next_state(5)) || abs(next_state(5)) > M_PI)
  {
    next_state(5) = 4.0;
  }
  if (isnan(next_state(6)) || isinf(next_state(6)) || abs(next_state(6)) > M_PI)
  {
    next_state(6) = 4.0;
  }

  // Setup output
  float yaw = next_state[S_INDEX(YAW)];
  output[O_INDEX(BASELINK_VEL_B_X)] = next_state[S_INDEX(VEL_X)];
  output[O_INDEX(BASELINK_VEL_B_Y)] = 0;
  output[O_INDEX(BASELINK_VEL_B_Z)] = 0;
  output[O_INDEX(BASELINK_POS_I_X)] = next_state[S_INDEX(POS_X)];
  output[O_INDEX(BASELINK_POS_I_Y)] = next_state[S_INDEX(POS_Y)];
  output[O_INDEX(BASELINK_POS_I_Z)] = 0;
  output[O_INDEX(YAW)] = yaw;
  output[O_INDEX(PITCH)] = pitch;
  output[O_INDEX(ROLL)] = roll;
  output[O_INDEX(STEER_ANGLE)] = next_state[S_INDEX(STEER_ANGLE)];
  output[O_INDEX(STEER_ANGLE_RATE)] = 0;
  output[O_INDEX(WHEEL_POS_I_FL_X)] = front_left.x;
  output[O_INDEX(WHEEL_POS_I_FL_Y)] = front_left.y;
  output[O_INDEX(WHEEL_POS_I_FR_X)] = front_right.x;
  output[O_INDEX(WHEEL_POS_I_FR_Y)] = front_right.y;
  output[O_INDEX(WHEEL_POS_I_RL_X)] = rear_left.x;
  output[O_INDEX(WHEEL_POS_I_RL_Y)] = rear_left.y;
  output[O_INDEX(WHEEL_POS_I_RR_X)] = rear_right.x;
  output[O_INDEX(WHEEL_POS_I_RR_Y)] = rear_right.y;
  output[O_INDEX(WHEEL_FORCE_B_FL)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_FR)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RL)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RR)] = 10000;
  // output[O_INDEX(CENTER_POS_I_X)] = output[O_INDEX(BASELINK_POS_I_X)];  // TODO
  // output[O_INDEX(CENTER_POS_I_Y)] = output[O_INDEX(BASELINK_POS_I_Y)];
  // output[O_INDEX(CENTER_POS_I_Z)] = 0;
  output[O_INDEX(ACCEL_X)] = next_state[S_INDEX(ACCEL_X)];
}

__device__ void RacerDubinsElevation::initializeDynamics(float* state, float* control, float* output, float* theta_s,
                                                         float t_0, float dt)
{
  PARENT_CLASS::initializeDynamics(state, control, output, theta_s, t_0, dt);
  if (SHARED_MEM_REQUEST_GRD != 1)
  {  // Allows us to turn on or off global or shared memory version of params
    DYN_PARAMS_T* shared_params = (DYN_PARAMS_T*)theta_s;
    *shared_params = this->params_;
  }
}

__device__ inline void RacerDubinsElevation::step(float* state, float* next_state, float* state_der, float* control,
                                                  float* output, float* theta_s, const float t, const float dt)
{
  DYN_PARAMS_T* params_p;
  if (SHARED_MEM_REQUEST_GRD != 1)
  {  // Allows us to turn on or off global or shared memory version of params
    params_p = (DYN_PARAMS_T*)theta_s;
  }
  else
  {
    params_p = &(this->params_);
  }
  const int tdy = threadIdx.y;

  // Compute dynamics
  bool enable_brake = control[0] < 0;
  int index = (fabsf(state[S_INDEX(VEL_X)]) > 0.2 && fabsf(state[S_INDEX(VEL_X)]) <= 3.0) +
              (fabsf(state[S_INDEX(VEL_X)]) > 3.0) * 2;
  // applying position throttle
  float throttle = params_p->c_t[index] * control[0];
  float brake = params_p->c_b[index] * control[0] * (state[S_INDEX(VEL_X)] >= 0 ? 1 : -1);
  float linear_brake_slope = 0.9f * (2 / dt);
  if (abs(state[S_INDEX(VEL_X)]) <= params_p->c_b[index] / linear_brake_slope)
  {
    throttle = params_p->c_t[index] * max(control[0] - params_p->low_min_throttle, 0.0f);
    brake = linear_brake_slope * control[0] * state[S_INDEX(VEL_X)];
  }

  state_der[S_INDEX(VEL_X)] =
      (!enable_brake) * throttle + (enable_brake)*brake - params_p->c_v[index] * state[S_INDEX(VEL_X)] + params_p->c_0;
  if (fabsf(state[S_INDEX(PITCH)]) < M_PI_2f32)
  {
    state_der[S_INDEX(VEL_X)] -= params_p->gravity * sinf(state[S_INDEX(PITCH)]);
  }
  state_der[S_INDEX(YAW)] = (state[S_INDEX(VEL_X)] / params_p->wheel_base) *
                            tan(state[S_INDEX(STEER_ANGLE)] / params_p->steer_angle_scale[index]);
  state_der[S_INDEX(POS_X)] = state[S_INDEX(VEL_X)] * cosf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(POS_Y)] = state[S_INDEX(VEL_X)] * sinf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(STEER_ANGLE)] =
      (control[1] * params_p->steer_command_angle_scale - state[S_INDEX(STEER_ANGLE)]) * params_p->steering_constant;
  state_der[S_INDEX(STEER_ANGLE)] =
      max(min(state_der[S_INDEX(STEER_ANGLE)], params_p->max_steer_rate), -params_p->max_steer_rate);

  // Calculate the next state
  float pitch = 0;
  float roll = 0;

  float3 front_left = make_float3(2.981, 0.737, 0);
  float3 front_right = make_float3(2.981, -0.737, 0);
  float3 rear_left = make_float3(0, 0.737, 0);
  float3 rear_right = make_float3(0, -0.737, 0);

  float front_left_height = 0;
  float front_right_height = 0;
  float rear_left_height = 0;
  float rear_right_height = 0;
  front_left = make_float3(front_left.x * cosf(state[1]) - front_left.y * sinf(state[1]) + state[2],
                           front_left.x * sinf(state[1]) + front_left.y * cosf(state[1]) + state[3], 0);
  front_right = make_float3(front_right.x * cosf(state[1]) - front_right.y * sinf(state[1]) + state[2],
                            front_right.x * sinf(state[1]) + front_right.y * cosf(state[1]) + state[3], 0);
  rear_left = make_float3(rear_left.x * cosf(state[1]) - rear_left.y * sinf(state[1]) + state[2],
                          rear_left.x * sinf(state[1]) + rear_left.y * cosf(state[1]) + state[3], 0);
  rear_right = make_float3(rear_right.x * cosf(state[1]) - rear_right.y * sinf(state[1]) + state[2],
                           rear_right.x * sinf(state[1]) + rear_right.y * cosf(state[1]) + state[3], 0);

  // Set to 7 as the last 2 states do not do euler integration
  for (int i = tdy; i < 7; i += blockDim.y)
  {
    next_state[i] = state[i] + state_der[i] * dt;
    switch (i)
    {
      case S_INDEX(VEL_X):
        next_state[S_INDEX(ACCEL_X)] = state_der[S_INDEX(VEL_X)];
        break;
      case S_INDEX(YAW):
        next_state[i] = angle_utils::normalizeAngle(next_state[i]);
        break;
      case S_INDEX(STEER_ANGLE):
        next_state[S_INDEX(STEER_ANGLE)] =
            max(min(next_state[S_INDEX(STEER_ANGLE)], this->params_.max_steer_angle), -this->params_.max_steer_angle);
        next_state[S_INDEX(STEER_ANGLE_RATE)] = state_der[S_INDEX(STEER_ANGLE)];
        break;
    }
    if (i == S_INDEX(ROLL) || i == S_INDEX(PITCH))
    {
      if (this->tex_helper_->checkTextureUse(0))
      {
        front_left_height = this->tex_helper_->queryTextureAtWorldPose(0, front_left);
        front_right_height = this->tex_helper_->queryTextureAtWorldPose(0, front_right);
        rear_left_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_left);
        rear_right_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_right);

        // max magnitude
        if (i == S_INDEX(ROLL))
        {
          float front_diff = front_left_height - front_right_height;
          front_diff = max(min(front_diff, 0.736 * 2), -0.736 * 2);
          float rear_diff = rear_left_height - rear_right_height;
          rear_diff = max(min(rear_diff, 0.736 * 2), -0.736 * 2);
          float front_roll = asinf(front_diff / (0.737 * 2));
          float rear_roll = asinf(rear_diff / (0.737 * 2));
          next_state[5] = (front_roll + rear_roll) / 2;
        }
        if (i == S_INDEX(PITCH))
        {
          float left_diff = rear_left_height - front_left_height;
          left_diff = max(min(left_diff, 2.98), -2.98);
          float right_diff = rear_right_height - front_right_height;
          right_diff = max(min(right_diff, 2.98), -2.98);
          float left_pitch = asinf((left_diff) / 2.981);
          float right_pitch = asinf((right_diff) / 2.981);
          next_state[6] = (left_pitch + right_pitch) / 2;
          if (isnan(next_state[6]) || isinf(next_state[6]) || fabsf(next_state[6]) > M_PIf32)
          {
            next_state[6] = 4.0;
          }
        }
        if (isnan(next_state[i]) || isinf(next_state[i]) || fabsf(next_state[i]) > M_PIf32)
        {
          next_state[i] = 4.0;
        }
      }
      else
      {
        next_state[5] = 0;
        next_state[6] = 0;
      }
    }
  }
  __syncthreads();

  // Fill in output
  float yaw = next_state[S_INDEX(YAW)];
  output[O_INDEX(BASELINK_VEL_B_X)] = next_state[S_INDEX(VEL_X)];
  output[O_INDEX(BASELINK_VEL_B_Y)] = 0;
  output[O_INDEX(BASELINK_VEL_B_Z)] = 0;
  output[O_INDEX(BASELINK_POS_I_X)] = next_state[S_INDEX(POS_X)];
  output[O_INDEX(BASELINK_POS_I_Y)] = next_state[S_INDEX(POS_Y)];
  output[O_INDEX(BASELINK_POS_I_Z)] = 0;
  output[O_INDEX(YAW)] = next_state[S_INDEX(YAW)];
  output[O_INDEX(PITCH)] = next_state[S_INDEX(PITCH)];
  output[O_INDEX(ROLL)] = next_state[S_INDEX(ROLL)];
  output[O_INDEX(STEER_ANGLE)] = next_state[S_INDEX(STEER_ANGLE)];
  output[O_INDEX(STEER_ANGLE_RATE)] = next_state[S_INDEX(STEER_ANGLE_RATE)];
  output[O_INDEX(WHEEL_POS_I_FL_X)] = front_left.x;
  output[O_INDEX(WHEEL_POS_I_FL_Y)] = front_left.y;
  output[O_INDEX(WHEEL_POS_I_FR_X)] = front_right.x;
  output[O_INDEX(WHEEL_POS_I_FR_Y)] = front_right.y;
  output[O_INDEX(WHEEL_POS_I_RL_X)] = rear_left.x;
  output[O_INDEX(WHEEL_POS_I_RL_Y)] = rear_left.y;
  output[O_INDEX(WHEEL_POS_I_RR_X)] = rear_right.x;
  output[O_INDEX(WHEEL_POS_I_RR_Y)] = rear_right.y;
  output[O_INDEX(WHEEL_FORCE_B_FL)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_FR)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RL)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RR)] = 10000;
  // output[O_INDEX(CENTER_POS_I_X)] = output[O_INDEX(BASELINK_POS_I_X)];  // TODO
  // output[O_INDEX(CENTER_POS_I_Y)] = output[O_INDEX(BASELINK_POS_I_Y)];
  // output[O_INDEX(CENTER_POS_I_Z)] = 0;
  output[O_INDEX(ACCEL_X)] = next_state[S_INDEX(ACCEL_X)];
}
