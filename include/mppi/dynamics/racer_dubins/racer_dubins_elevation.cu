#include "hip/hip_runtime.h"
#include <mppi/dynamics/racer_dubins/racer_dubins_elevation.cuh>
#include <mppi/utils/math_utils.h>

void RacerDubinsElevation::GPUSetup()
{
  PARENT_CLASS* derived = static_cast<PARENT_CLASS*>(this);
  CudaCheckError();
  tex_helper_->GPUSetup();
  CudaCheckError();
  derived->GPUSetup();
  CudaCheckError();
}

void RacerDubinsElevation::freeCudaMem()
{
  tex_helper_->freeCudaMem();
}

void RacerDubinsElevation::paramsToDevice()
{
  if (this->GPUMemStatus_)
  {
    // does all the internal texture updates
    tex_helper_->copyToDevice();
    // makes sure that the device ptr sees the correct texture object
    HANDLE_ERROR(hipMemcpyAsync(&(this->model_d_->tex_helper_), &(tex_helper_->ptr_d_),
                                 sizeof(TwoDTextureHelper<float>*), hipMemcpyHostToDevice, this->stream_));
  }
  PARENT_CLASS::paramsToDevice();
}

void RacerDubinsElevation::updateState(const Eigen::Ref<const state_array> state, Eigen::Ref<state_array> next_state,
                                       Eigen::Ref<state_array> state_der, const float dt)
{
  next_state = state + state_der * dt;
  next_state(S_INDEX(YAW)) = angle_utils::normalizeAngle(next_state(S_INDEX(YAW)));
  next_state(S_INDEX(STEER_ANGLE)) =
      max(min(next_state(S_INDEX(STEER_ANGLE)), this->params_.max_steer_angle), -this->params_.max_steer_angle);
  next_state(7) = state_der(S_INDEX(STEER_ANGLE));
  next_state(S_INDEX(ACCEL_X)) = state_der(S_INDEX(VEL_X));
}

__device__ void RacerDubinsElevation::updateState(float* state, float* next_state, float* state_der, const float dt)
{
  int i;
  int tdy = threadIdx.y;
  // Add the state derivative time dt to the current state.
  // printf("updateState thread %d, %d = %f, %f\n", threadIdx.x, threadIdx.y, state[0], state_der[0]);
  for (i = tdy; i < 5; i += blockDim.y)
  {
    next_state[i] = state[i] + state_der[i] * dt;
    if (i == S_INDEX(VEL_X))
    {
      next_state[S_INDEX(ACCEL_X)] = state_der[S_INDEX(VEL_X)];
    }
    if (i == S_INDEX(YAW))
    {
      next_state[i] = angle_utils::normalizeAngle(next_state[i]);
    }
    if (i == S_INDEX(STEER_ANGLE))
    {
      next_state[S_INDEX(STEER_ANGLE)] =
          max(min(next_state[S_INDEX(STEER_ANGLE)], this->params_.max_steer_angle), -this->params_.max_steer_angle);
      next_state[S_INDEX(STEER_ANGLE_RATE)] = state_der[S_INDEX(STEER_ANGLE)];
    }
  }
}

void RacerDubinsElevation::computeStateDeriv(const Eigen::Ref<const state_array>& state,
                                             const Eigen::Ref<const control_array>& control,
                                             Eigen::Ref<state_array> state_der)
{
}

void RacerDubinsElevation::step(Eigen::Ref<state_array> state, Eigen::Ref<state_array> next_state,
                                Eigen::Ref<state_array> state_der, const Eigen::Ref<const control_array>& control,
                                Eigen::Ref<output_array> output, const float t, const float dt)
{
  // computeStateDeriv(state, control, state_der);
  bool enable_brake = control(0) < 0;
  int index = (abs(state(0)) > 0.5 && abs(state(0)) <= 6.0) + (abs(state(0)) > 6.0) * 2;
  // applying position throttle
  float throttle = this->params_.c_t[index] * control(0);
  float brake = this->params_.c_b[index] * control(0) * (state(0) >= 0 ? 1 : -1);
  if (abs(state(0)) <= 0.5)
  {
    throttle = this->params_.c_t[index] * max(control(0) - this->params_.low_min_throttle, 0.0f);
    brake = this->params_.c_b[index] * control(0) * state(0);
  }

  state_der(0) =
      (!enable_brake) * throttle + (enable_brake)*brake - this->params_.c_v[index] * state(0) + this->params_.c_0;
  if (abs(state[6]) < M_PI_2)
  {
    state_der[0] -= this->params_.gravity * sinf(state[6]);
  }
  state_der(1) = (state(0) / this->params_.wheel_base) * tan(state(4) / this->params_.steer_angle_scale[index]);
  state_der(2) = state(0) * cosf(state(1));
  state_der(3) = state(0) * sinf(state(1));
  state_der(4) = (control(1) * this->params_.steer_command_angle_scale - state(4)) * this->params_.steering_constant;
  state_der(4) = max(min(state_der(4), this->params_.max_steer_rate), -this->params_.max_steer_rate);
  // state(8) = state_der(0);

  // Integrate using racer_dubins upddateState
  updateState(state, next_state, state_der, dt);

  float pitch = 0;
  float roll = 0;

  float3 front_left = make_float3(2.981, 0.737, 0);
  float3 front_right = make_float3(2.981, -0.737, 0);
  float3 rear_left = make_float3(0, 0.737, 0);
  float3 rear_right = make_float3(0, -0.737, 0);
  front_left = make_float3(front_left.x * cosf(state(1)) - front_left.y * sinf(state(1)) + state(2),
                           front_left.x * sinf(state(1)) + front_left.y * cosf(state(1)) + state(3), 0);
  front_right = make_float3(front_right.x * cosf(state(1)) - front_right.y * sinf(state(1)) + state(2),
                            front_right.x * sinf(state(1)) + front_right.y * cosf(state(1)) + state(3), 0);
  rear_left = make_float3(rear_left.x * cosf(state(1)) - rear_left.y * sinf(state(1)) + state(2),
                          rear_left.x * sinf(state(1)) + rear_left.y * cosf(state(1)) + state(3), 0);
  rear_right = make_float3(rear_right.x * cosf(state(1)) - rear_right.y * sinf(state(1)) + state(2),
                           rear_right.x * sinf(state(1)) + rear_right.y * cosf(state(1)) + state(3), 0);
  float front_left_height = 0;
  float front_right_height = 0;
  float rear_left_height = 0;
  float rear_right_height = 0;

  if (this->tex_helper_->checkTextureUse(0))
  {
    front_left_height = this->tex_helper_->queryTextureAtWorldPose(0, front_left);
    front_right_height = this->tex_helper_->queryTextureAtWorldPose(0, front_right);
    rear_left_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_left);
    rear_right_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_right);

    float front_diff = front_left_height - front_right_height;
    front_diff = max(min(front_diff, 0.736 * 2), -0.736 * 2);
    float rear_diff = rear_left_height - rear_right_height;
    rear_diff = max(min(rear_diff, 0.736 * 2), -0.736 * 2);
    float front_roll = asinf(front_diff / (0.737 * 2));
    float rear_roll = asinf(rear_diff / (0.737 * 2));
    next_state(5) = (front_roll + rear_roll) / 2;

    float left_diff = rear_left_height - front_left_height;
    left_diff = max(min(left_diff, 2.98), -2.98);
    float right_diff = rear_right_height - front_right_height;
    right_diff = max(min(right_diff, 2.98), -2.98);
    float left_pitch = asinf((left_diff) / 2.981);
    float right_pitch = asinf((right_diff) / 2.981);
    next_state(6) = (left_pitch + right_pitch) / 2;
  }
  else
  {
    next_state(5) = 0;
    next_state(6) = 0;
  }

  if (isnan(next_state(5)) || isinf(next_state(5)) || abs(next_state(5)) > M_PI)
  {
    next_state(5) = 4.0;
  }
  if (isnan(next_state(6)) || isinf(next_state(6)) || abs(next_state(6)) > M_PI)
  {
    next_state(6) = 4.0;
  }
  // state_der[5] = (next_state[5] - state[5]) / dt;
  // state_der[6] = (next_state[6] - state[6]) / dt;

  // Setup output
  float yaw = next_state[S_INDEX(YAW)];
  output[O_INDEX(BASELINK_VEL_B_X)] = next_state[S_INDEX(VEL_X)];
  output[O_INDEX(BASELINK_VEL_B_Y)] = 0;
  output[O_INDEX(BASELINK_VEL_B_Z)] = 0;
  output[O_INDEX(BASELINK_POS_I_X)] = next_state[S_INDEX(POS_X)];
  output[O_INDEX(BASELINK_POS_I_Y)] = next_state[S_INDEX(POS_Y)];
  output[O_INDEX(BASELINK_POS_I_Z)] = 0;
  output[O_INDEX(OMEGA_B_X)] = 0;
  output[O_INDEX(OMEGA_B_Y)] = 0;
  output[O_INDEX(OMEGA_B_Z)] = 0;
  output[O_INDEX(YAW)] = yaw;
  output[O_INDEX(PITCH)] = pitch;
  output[O_INDEX(ROLL)] = roll;
  Eigen::Quaternionf q;
  mppi::math::Euler2QuatNWU(roll, pitch, yaw, q);
  output[O_INDEX(ATTITUDE_QW)] = q.w();
  output[O_INDEX(ATTITUDE_QX)] = q.x();
  output[O_INDEX(ATTITUDE_QY)] = q.y();
  output[O_INDEX(ATTITUDE_QZ)] = q.z();
  output[O_INDEX(STEER_ANGLE)] = next_state[S_INDEX(STEER_ANGLE)];
  output[O_INDEX(STEER_ANGLE_RATE)] = 0;
  output[O_INDEX(WHEEL_POS_I_FL_X)] = front_left.x;
  output[O_INDEX(WHEEL_POS_I_FL_Y)] = front_left.y;
  output[O_INDEX(WHEEL_POS_I_FR_X)] = front_right.x;
  output[O_INDEX(WHEEL_POS_I_FR_Y)] = front_right.y;
  output[O_INDEX(WHEEL_POS_I_RL_X)] = rear_left.x;
  output[O_INDEX(WHEEL_POS_I_RL_Y)] = rear_left.y;
  output[O_INDEX(WHEEL_POS_I_RR_X)] = rear_right.x;
  output[O_INDEX(WHEEL_POS_I_RR_Y)] = rear_right.y;
  output[O_INDEX(WHEEL_FORCE_B_FL_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FL_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FL_Z)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_FR_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FR_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FR_Z)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RL_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RL_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RL_Z)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RR_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RR_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RR_Z)] = 10000;
  output[O_INDEX(CENTER_POS_I_X)] = output[O_INDEX(BASELINK_POS_I_X)];  // TODO
  output[O_INDEX(CENTER_POS_I_Y)] = output[O_INDEX(BASELINK_POS_I_Y)];
  output[O_INDEX(CENTER_POS_I_Z)] = 0;
}

__device__ void RacerDubinsElevation::initializeDynamics(float* state, float* control, float* theta_s, float t_0,
                                                         float dt)
{
  if (SHARED_MEM_REQUEST_GRD != 1)
  {  // Allows us to turn on or off global or shared memory version of params
    DYN_PARAMS_T* shared_params = (DYN_PARAMS_T*)theta_s;
    *shared_params = this->params_;
  }
}

__device__ inline void RacerDubinsElevation::step(float* state, float* next_state, float* state_der, float* control,
                                                  float* output, float* theta_s, const float t, const float dt)
{
  DYN_PARAMS_T* params_p;
  if (SHARED_MEM_REQUEST_GRD != 1)
  {  // Allows us to turn on or off global or shared memory version of params
    params_p = (DYN_PARAMS_T*)theta_s;
  }
  else
  {
    params_p = &(this->params_);
  }
  // computeStateDeriv(state, control, state_der, theta_s);
  const int tdy = threadIdx.y;
  // if (tdy == 1 % blockDim.y)
  // { // calculate on separate thread from cost to potentially have this happen in conjunction
  bool enable_brake = control[0] < 0;
  int index = (fabsf(state[S_INDEX(VEL_X)]) > 0.5 && fabsf(state[S_INDEX(VEL_X)]) <= 6.0) +
              (fabsf(state[S_INDEX(VEL_X)]) > 6.0) * 2;
  // applying position throttle
  float throttle = params_p->c_t[index] * control[0];
  float brake = params_p->c_b[index] * control[0] * (state[S_INDEX(VEL_X)] >= 0 ? 1 : -1);
  if (abs(state[S_INDEX(VEL_X)]) <= 0.5)
  {
    throttle = params_p->c_t[index] * max(control[0] - params_p->low_min_throttle, 0.0f);
    brake = params_p->c_b[index] * control[0] * state[S_INDEX(VEL_X)];
  }

  state_der[S_INDEX(VEL_X)] =
      (!enable_brake) * throttle + (enable_brake)*brake - params_p->c_v[index] * state[S_INDEX(VEL_X)] + params_p->c_0;
  if (fabsf(state[S_INDEX(PITCH)]) < M_PI_2f32)
  {
    state_der[S_INDEX(VEL_X)] -= params_p->gravity * sinf(state[S_INDEX(PITCH)]);
  }
  state_der[S_INDEX(YAW)] = (state[S_INDEX(VEL_X)] / params_p->wheel_base) *
                            tan(state[S_INDEX(STEER_ANGLE)] / params_p->steer_angle_scale[index]);
  state_der[S_INDEX(POS_X)] = state[S_INDEX(VEL_X)] * cosf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(POS_Y)] = state[S_INDEX(VEL_X)] * sinf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(STEER_ANGLE)] =
      (control[1] * params_p->steer_command_angle_scale - state[S_INDEX(STEER_ANGLE)]) * params_p->steering_constant;
  state_der[S_INDEX(STEER_ANGLE)] =
      max(min(state_der[S_INDEX(STEER_ANGLE)], params_p->max_steer_rate), -params_p->max_steer_rate);
  // }
  __syncthreads();
  // Use Euler Integration from racer_dubins parent class
  // updateState(state, next_state, state_der, dt);
  float pitch = 0;
  float roll = 0;

  float3 front_left = make_float3(2.981, 0.737, 0);
  float3 front_right = make_float3(2.981, -0.737, 0);
  float3 rear_left = make_float3(0, 0.737, 0);
  float3 rear_right = make_float3(0, -0.737, 0);

  float front_left_height = 0;
  float front_right_height = 0;
  float rear_left_height = 0;
  float rear_right_height = 0;
  for (int i = tdy; i < 5; i += blockDim.y)
  {
    next_state[i] = state[i] + state_der[i] * dt;
    switch (i)
    {
      case S_INDEX(VEL_X):
        next_state[S_INDEX(ACCEL_X)] = state_der[S_INDEX(VEL_X)];
        break;
      case S_INDEX(YAW):
        next_state[i] = angle_utils::normalizeAngle(next_state[i]);
        break;
      case S_INDEX(STEER_ANGLE):
        next_state[S_INDEX(STEER_ANGLE)] =
            max(min(next_state[S_INDEX(STEER_ANGLE)], this->params_.max_steer_angle), -this->params_.max_steer_angle);
        next_state[S_INDEX(STEER_ANGLE_RATE)] = state_der[S_INDEX(STEER_ANGLE)];
        break;
    }
    if (i == S_INDEX(ROLL) || i == S_INDEX(PITCH))
    {
      if (this->tex_helper_->checkTextureUse(0))
      {
        front_left =
            make_float3(front_left.x * cosf(next_state[1]) - front_left.y * sinf(next_state[1]) + next_state[2],
                        front_left.x * sinf(next_state[1]) + front_left.y * cosf(next_state[1]) + next_state[3], 0);
        front_right =
            make_float3(front_right.x * cosf(next_state[1]) - front_right.y * sinf(next_state[1]) + next_state[2],
                        front_right.x * sinf(next_state[1]) + front_right.y * cosf(next_state[1]) + next_state[3], 0);
        rear_left =
            make_float3(rear_left.x * cosf(next_state[1]) - rear_left.y * sinf(next_state[1]) + next_state[2],
                        rear_left.x * sinf(next_state[1]) + rear_left.y * cosf(next_state[1]) + next_state[3], 0);
        rear_right =
            make_float3(rear_right.x * cosf(next_state[1]) - rear_right.y * sinf(next_state[1]) + next_state[2],
                        rear_right.x * sinf(next_state[1]) + rear_right.y * cosf(next_state[1]) + next_state[3], 0);
        front_left_height = this->tex_helper_->queryTextureAtWorldPose(0, front_left);
        front_right_height = this->tex_helper_->queryTextureAtWorldPose(0, front_right);
        rear_left_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_left);
        rear_right_height = this->tex_helper_->queryTextureAtWorldPose(0, rear_right);

        // max magnitude
        if (i == S_INDEX(ROLL))
        {
          float front_diff = front_left_height - front_right_height;
          front_diff = max(min(front_diff, 0.736 * 2), -0.736 * 2);
          float rear_diff = rear_left_height - rear_right_height;
          rear_diff = max(min(rear_diff, 0.736 * 2), -0.736 * 2);
          float front_roll = asinf(front_diff / (0.737 * 2));
          float rear_roll = asinf(rear_diff / (0.737 * 2));
          next_state[5] = (front_roll + rear_roll) / 2;
        }
        if (i == S_INDEX(PITCH))
        {
          float left_diff = rear_left_height - front_left_height;
          left_diff = max(min(left_diff, 2.98), -2.98);
          float right_diff = rear_right_height - front_right_height;
          right_diff = max(min(right_diff, 2.98), -2.98);
          float left_pitch = asinf((left_diff) / 2.981);
          float right_pitch = asinf((right_diff) / 2.981);
          next_state[6] = (left_pitch + right_pitch) / 2;
          if (isnan(next_state[6]) || isinf(next_state[6]) || fabsf(next_state[6]) > M_PIf32)
          {
            next_state[6] = 4.0;
          }
        }
        if (isnan(next_state[i]) || isinf(next_state[i]) || fabsf(next_state[i]) > M_PIf32)
        {
          next_state[i] = 4.0;
        }
      }
      else
      {
        next_state[5] = 0;
        next_state[6] = 0;
      }
    }
  }
  __syncthreads();

  float yaw = next_state[S_INDEX(YAW)];
  float q[4];
  mppi::math::Euler2QuatNWU(roll, pitch, yaw, q);
#if false
  for (int i = tdy; i < PARENT_CLASS::OUTPUT_DIM; i += blockDim.y)
  {
    switch (i)
    {
      case O_INDEX(BASELINK_VEL_B_X):
        output[i] = next_state[S_INDEX(VEL_X)];
        break;
      case O_INDEX(BASELINK_POS_I_X):
        output[i] = next_state[S_INDEX(POS_X)];
        break;
      case O_INDEX(BASELINK_POS_I_Y):
        output[i] = next_state[S_INDEX(POS_Y)];
        break;
      case O_INDEX(YAW):
        output[i] = next_state[S_INDEX(YAW)];
        break;
      case O_INDEX(PITCH):
        output[i] = next_state[S_INDEX(PITCH)];
        break;
      case O_INDEX(ROLL):
        output[i] = next_state[S_INDEX(ROLL)];
        break;
      case O_INDEX(ATTITUDE_QW):
        output[i] = q[0];
        break;
      case O_INDEX(ATTITUDE_QX):
        output[i] = q[1];
        break;
      case O_INDEX(ATTITUDE_QY):
        output[i] = q[2];
        break;
      case O_INDEX(ATTITUDE_QZ):
        output[i] = q[3];
        break;
      case O_INDEX(STEER_ANGLE):
        output[i] = next_state[S_INDEX(STEER_ANGLE)];
        break;
      case O_INDEX(STEER_ANGLE_RATE):
        output[i] = next_state[S_INDEX(STEER_ANGLE_RATE)];
        break;
      case O_INDEX(CENTER_POS_I_X):
        output[i] = next_state[S_INDEX(POS_X)];
        break;
      case O_INDEX(CENTER_POS_I_Y):
        output[i] = next_state[S_INDEX(POS_Y)];
        break;
      case O_INDEX(WHEEL_POS_I_FL_X):
        output[i] = front_left.x;
        break;
      case O_INDEX(WHEEL_POS_I_FL_Y):
        output[i] = front_left.y;
        break;
      case O_INDEX(WHEEL_POS_I_FR_X):
        output[i] = front_right.x;
        break;
      case O_INDEX(WHEEL_POS_I_FR_Y):
        output[i] = front_right.y;
        break;
      case O_INDEX(WHEEL_POS_I_RR_X):
        output[i] = rear_right.x;
        break;
      case O_INDEX(WHEEL_POS_I_RR_Y):
        output[i] = rear_right.y;
        break;
      case O_INDEX(WHEEL_POS_I_RL_X):
        output[i] = rear_left.x;
        break;
      case O_INDEX(WHEEL_POS_I_RL_Y):
        output[i] = rear_left.y;
        break;
      case O_INDEX(WHEEL_FORCE_B_FL_Z):
      case O_INDEX(WHEEL_FORCE_B_FR_Z):
      case O_INDEX(WHEEL_FORCE_B_RR_Z):
      case O_INDEX(WHEEL_FORCE_B_RL_Z):
        output[i] = 1000;
        break;
      default:
        output[i] = 0;
    }
  }
#else
  output[O_INDEX(BASELINK_VEL_B_X)] = next_state[S_INDEX(VEL_X)];
  output[O_INDEX(BASELINK_VEL_B_Y)] = 0;
  output[O_INDEX(BASELINK_VEL_B_Z)] = 0;
  output[O_INDEX(BASELINK_POS_I_X)] = next_state[S_INDEX(POS_X)];
  output[O_INDEX(BASELINK_POS_I_Y)] = next_state[S_INDEX(POS_Y)];
  output[O_INDEX(BASELINK_POS_I_Z)] = 0;
  output[O_INDEX(OMEGA_B_X)] = 0;
  output[O_INDEX(OMEGA_B_Y)] = 0;
  output[O_INDEX(OMEGA_B_Z)] = 0;
  output[O_INDEX(YAW)] = next_state[S_INDEX(YAW)];
  output[O_INDEX(PITCH)] = next_state[S_INDEX(PITCH)];
  output[O_INDEX(ROLL)] = next_state[S_INDEX(ROLL)];
  output[O_INDEX(ATTITUDE_QW)] = q[0];
  output[O_INDEX(ATTITUDE_QX)] = q[1];
  output[O_INDEX(ATTITUDE_QY)] = q[2];
  output[O_INDEX(ATTITUDE_QZ)] = q[3];
  output[O_INDEX(STEER_ANGLE)] = next_state[S_INDEX(STEER_ANGLE)];
  output[O_INDEX(STEER_ANGLE_RATE)] = next_state[S_INDEX(STEER_ANGLE_RATE)];
  output[O_INDEX(WHEEL_POS_I_FL_X)] = front_left.x;
  output[O_INDEX(WHEEL_POS_I_FL_Y)] = front_left.y;
  output[O_INDEX(WHEEL_POS_I_FR_X)] = front_right.x;
  output[O_INDEX(WHEEL_POS_I_FR_Y)] = front_right.y;
  output[O_INDEX(WHEEL_POS_I_RL_X)] = rear_left.x;
  output[O_INDEX(WHEEL_POS_I_RL_Y)] = rear_left.y;
  output[O_INDEX(WHEEL_POS_I_RR_X)] = rear_right.x;
  output[O_INDEX(WHEEL_POS_I_RR_Y)] = rear_right.y;
  output[O_INDEX(WHEEL_FORCE_B_FL_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FL_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FL_Z)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_FR_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FR_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_FR_Z)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RL_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RL_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RL_Z)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RR_X)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RR_Y)] = 0;
  output[O_INDEX(WHEEL_FORCE_B_RR_Z)] = 10000;
  output[O_INDEX(CENTER_POS_I_X)] = output[O_INDEX(BASELINK_POS_I_X)];  // TODO
  output[O_INDEX(CENTER_POS_I_Y)] = output[O_INDEX(BASELINK_POS_I_Y)];
  output[O_INDEX(CENTER_POS_I_Z)] = 0;
#endif
}
