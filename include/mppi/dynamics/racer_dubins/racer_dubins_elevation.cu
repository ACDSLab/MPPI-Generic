#include "hip/hip_runtime.h"
#include <mppi/dynamics/racer_dubins/racer_dubins_elevation.cuh>

void RacerDubinsElevation::GPUSetup()
{
  RacerDubinsImpl<RacerDubinsElevation, 9>* derived = static_cast<RacerDubinsImpl<RacerDubinsElevation, 9>*>(this);
  CudaCheckError();
  tex_helper_->GPUSetup();
  CudaCheckError();
  derived->GPUSetup();
  CudaCheckError();
}

void RacerDubinsElevation::freeCudaMem()
{
  tex_helper_->freeCudaMem();
}

void RacerDubinsElevation::paramsToDevice()
{
  if (this->GPUMemStatus_)
  {
    // does all the internal texture updates
    tex_helper_->copyToDevice();
    // makes sure that the device ptr sees the correct texture object
    HANDLE_ERROR(hipMemcpyAsync(&(this->model_d_->tex_helper_), &(tex_helper_->ptr_d_),
                                 sizeof(TwoDTextureHelper<float>*), hipMemcpyHostToDevice, this->stream_));
  }
  RacerDubinsImpl<RacerDubinsElevation, 9>::paramsToDevice();
}

void RacerDubinsElevation::updateState(Eigen::Ref<state_array> state, Eigen::Ref<state_array> state_der, const float dt)
{
  state += state_der * dt;
  state(1) = angle_utils::normalizeAngle(state(1));
  state(4) = max(min(state(4), this->params_.max_steer_angle), -this->params_.max_steer_angle);
  state(7) = state_der(4);

  if (this->tex_helper_->checkTextureUse(0))
  {
    float3 front_left = make_float3(2.981, 0.737, 0);
    float3 front_right = make_float3(2.981, -0.737, 0);
    float3 back_left = make_float3(0, 0.737, 0);
    float3 back_right = make_float3(0, -0.737, 0);
    front_left = make_float3(front_left.x * cosf(state(1)) - front_left.y * sinf(state(1)) + state(2),
                             front_left.x * sinf(state(1)) + front_left.y * cosf(state(1)) + state(3), 0);
    front_right = make_float3(front_right.x * cosf(state(1)) - front_right.y * sinf(state(1)) + state(2),
                              front_right.x * sinf(state(1)) + front_right.y * cosf(state(1)) + state(3), 0);
    back_left = make_float3(back_left.x * cosf(state(1)) - back_left.y * sinf(state(1)) + state(2),
                            back_left.x * sinf(state(1)) + back_left.y * cosf(state(1)) + state(3), 0);
    back_right = make_float3(back_right.x * cosf(state(1)) - back_right.y * sinf(state(1)) + state(2),
                             back_right.x * sinf(state(1)) + back_right.y * cosf(state(1)) + state(3), 0);
    float front_left_height = this->tex_helper_->queryTextureAtWorldPose(0, front_left);
    float front_right_height = this->tex_helper_->queryTextureAtWorldPose(0, front_right);
    float back_left_height = this->tex_helper_->queryTextureAtWorldPose(0, back_left);
    float back_right_height = this->tex_helper_->queryTextureAtWorldPose(0, back_right);

    float front_diff = front_left_height - front_right_height;
    front_diff = max(min(front_diff, 0.736 * 2), -0.736 * 2);
    float back_diff = back_left_height - back_right_height;
    back_diff = max(min(back_diff, 0.736 * 2), -0.736 * 2);
    float front_roll = asinf(front_diff / (0.737 * 2));
    float back_roll = asinf(back_diff / (0.737 * 2));
    state(5) = (front_roll + back_roll) / 2;

    float left_diff = back_left_height - front_left_height;
    left_diff = max(min(left_diff, 2.98), -2.98);
    float right_diff = back_right_height - front_right_height;
    right_diff = max(min(right_diff, 2.98), -2.98);
    float left_pitch = asinf((left_diff) / 2.981);
    float right_pitch = asinf((right_diff) / 2.981);
    state(6) = (left_pitch + right_pitch) / 2;
  }
  else
  {
    state(5) = 0;
    state(6) = 0;
  }

  if (isnan(state(5)) || isinf(state(5)) || abs(state(5)) > M_PI)
  {
    state(5) = 4.0;
  }
  if (isnan(state(6)) || isinf(state(6)) || abs(state(6)) > M_PI)
  {
    state(6) = 4.0;
  }

  state(8) = state_der(0);

  state_der.setZero();
}

__device__ void RacerDubinsElevation::updateState(float* state, float* state_der, const float dt)
{
  int i;
  int tdy = threadIdx.y;
  // Add the state derivative time dt to the current state.
  // printf("updateState thread %d, %d = %f, %f\n", threadIdx.x, threadIdx.y, state[0], state_der[0]);
  for (i = tdy; i < 7; i += blockDim.y)
  {
    state[i] += state_der[i] * dt;
    if (i == 0)
    {
      state[8] = state_der[i];
    }
    if (i == 1)
    {
      state[i] = angle_utils::normalizeAngle(state[i]);
    }
    if (i == 4)
    {
      state[4] = max(min(state[4], this->params_.max_steer_angle), -this->params_.max_steer_angle);
      state[7] = state_der[i];
    }
    if (i == 5 || i == 6)
    {
      // roll
      if (this->tex_helper_->checkTextureUse(0))
      {
        float3 front_left = make_float3(2.981, 0.737, 0);
        float3 front_right = make_float3(2.981, -0.737, 0);
        float3 back_left = make_float3(0, 0.737, 0);
        float3 back_right = make_float3(0, -0.737, 0);
        front_left = make_float3(front_left.x * cosf(state[1]) - front_left.y * sinf(state[1]) + state[2],
                                 front_left.x * sinf(state[1]) + front_left.y * cosf(state[1]) + state[3], 0);
        front_right = make_float3(front_right.x * cosf(state[1]) - front_right.y * sinf(state[1]) + state[2],
                                  front_right.x * sinf(state[1]) + front_right.y * cosf(state[1]) + state[3], 0);
        back_left = make_float3(back_left.x * cosf(state[1]) - back_left.y * sinf(state[1]) + state[2],
                                back_left.x * sinf(state[1]) + back_left.y * cosf(state[1]) + state[3], 0);
        back_right = make_float3(back_right.x * cosf(state[1]) - back_right.y * sinf(state[1]) + state[2],
                                 back_right.x * sinf(state[1]) + back_right.y * cosf(state[1]) + state[3], 0);
        float front_left_height = this->tex_helper_->queryTextureAtWorldPose(0, front_left);
        float front_right_height = this->tex_helper_->queryTextureAtWorldPose(0, front_right);
        float back_left_height = this->tex_helper_->queryTextureAtWorldPose(0, back_left);
        float back_right_height = this->tex_helper_->queryTextureAtWorldPose(0, back_right);

        // max magnitude
        if (i == 5)
        {
          float front_diff = front_left_height - front_right_height;
          front_diff = max(min(front_diff, 0.736 * 2), -0.736 * 2);
          float back_diff = back_left_height - back_right_height;
          back_diff = max(min(back_diff, 0.736 * 2), -0.736 * 2);
          float front_roll = asinf(front_diff / (0.737 * 2));
          float back_roll = asinf(back_diff / (0.737 * 2));
          if (abs(front_roll) > abs(back_roll))
          {
            state[i] = front_roll;
          }
          else
          {
            state[i] = back_roll;
          }
        }
        if (i == 6)
        {
          float left_diff = back_left_height - front_left_height;
          left_diff = max(min(left_diff, 2.98), -2.98);
          float right_diff = back_right_height - front_right_height;
          right_diff = max(min(right_diff, 2.98), -2.98);
          float left_pitch = asinf((left_diff) / 2.981);
          float right_pitch = asinf((right_diff) / 2.981);
          if (abs(left_pitch) > abs(right_pitch))
          {
            state[i] = left_pitch;
          }
          else
          {
            state[i] = right_pitch;
          }
        }

        if (isnan(state[i]) || isinf(state[i]) || abs(state[i]) > M_PI)
        {
          // printf("got invalid roll %f from %f %f diff %f %f\n", state[i], front_left_height, front_right_height,
          // diff, (diff) / (0.737 * 2)); printf("got invalid roll at points (%f %f) (%f, %f)\n", front_left.x,
          // front_left.y, front_right.x, front_right.y);
          state[i] = 4.0;
        }
      }
      else
      {
        state[i] = 0;
      }
    }
    state_der[i] = 0;  // Important: reset the state derivative to zero.
  }
}

void RacerDubinsElevation::computeDynamics(const Eigen::Ref<const state_array>& state,
                                           const Eigen::Ref<const control_array>& control,
                                           Eigen::Ref<state_array> state_der)
{
  bool enable_brake = control(0) < 0;
  // applying position throttle
  state_der(0) = (!enable_brake) * this->params_.c_t * control(0) +
                 (enable_brake) * this->params_.c_b * control(0) * (state(0) >= 0 ? 1 : -1) -
                 this->params_.c_v * state(0) + this->params_.c_0;
  if (abs(state[6]) < M_PI)
  {
    state_der[0] -= this->params_.gravity * sinf(state[6]);
  }
  state_der(1) = (state(0) / this->params_.wheel_base) * tan(state(4));
  state_der(2) = state(0) * cosf(state(1));
  state_der(3) = state(0) * sinf(state(1));
  state_der(4) = (control(1) / this->params_.steer_command_angle_scale - state(4)) * this->params_.steering_constant;
  state_der(4) = max(min(state_der(4), this->params_.max_steer_rate), -this->params_.max_steer_rate);
}

__device__ void RacerDubinsElevation::computeDynamics(float* state, float* control, float* state_der, float* theta_s)
{
  bool enable_brake = control[0] < 0;
  // applying position throttle
  state_der[0] = (!enable_brake) * this->params_.c_t * control[0] +
                 (enable_brake) * this->params_.c_b * control[0] * (state[0] >= 0 ? 1 : -1) -
                 this->params_.c_v * state[0] + this->params_.c_0;
  if (abs(state[6]) < M_PI)
  {
    state_der[0] -= this->params_.gravity * sinf(state[6]);
  }
  state_der[1] = (state[0] / this->params_.wheel_base) * tan(state[4]);
  state_der[2] = state[0] * cosf(state[1]);
  state_der[3] = state[0] * sinf(state[1]);
  state_der[4] = (control[1] / this->params_.steer_command_angle_scale - state[4]) * this->params_.steering_constant;
  state_der[4] = max(min(state_der[4], this->params_.max_steer_rate), -this->params_.max_steer_rate);
}
