#include "hip/hip_runtime.h"
#include <mppi/dynamics/racer_dubins/racer_dubins.cuh>
#include <mppi/utils/math_utils.h>

template <class CLASS_T, class PARAMS_T>
void RacerDubinsImpl<CLASS_T, PARAMS_T>::computeDynamics(const Eigen::Ref<const state_array>& state,
                                                         const Eigen::Ref<const control_array>& control,
                                                         Eigen::Ref<state_array> state_der)
{
  bool enable_brake = control(C_INDEX(THROTTLE_BRAKE)) < 0;

  state_der(S_INDEX(BRAKE_STATE)) =
      min(max((enable_brake * -control(C_INDEX(THROTTLE_BRAKE)) - state(S_INDEX(BRAKE_STATE))) *
                  this->params_.brake_delay_constant,
              -this->params_.max_brake_rate_neg),
          this->params_.max_brake_rate_pos);
  // applying position throttle
  state_der(S_INDEX(VEL_X)) =
      (!enable_brake) * this->params_.c_t[0] * control(C_INDEX(THROTTLE_BRAKE)) * this->params_.gear_sign +
      this->params_.c_b[0] * state(S_INDEX(BRAKE_STATE)) * (state(S_INDEX(VEL_X)) >= 0 ? -1 : 1) -
      this->params_.c_v[0] * state(S_INDEX(VEL_X)) + this->params_.c_0;
  state_der(S_INDEX(YAW)) = (state(S_INDEX(VEL_X)) / this->params_.wheel_base) *
                            tan(state(S_INDEX(STEER_ANGLE)) / this->params_.steer_angle_scale);
  state_der(S_INDEX(POS_X)) = state(S_INDEX(VEL_X)) * cosf(state(S_INDEX(YAW)));
  state_der(S_INDEX(POS_Y)) = state(S_INDEX(VEL_X)) * sinf(state(S_INDEX(YAW)));
  state_der(S_INDEX(STEER_ANGLE)) =
      (control(C_INDEX(STEER_CMD)) * this->params_.steer_command_angle_scale - state(S_INDEX(STEER_ANGLE))) *
      this->params_.steering_constant;
  state_der(S_INDEX(STEER_ANGLE)) =
      max(min(state_der(S_INDEX(STEER_ANGLE)), this->params_.max_steer_rate), -this->params_.max_steer_rate);
}

template <class CLASS_T, class PARAMS_T>
bool RacerDubinsImpl<CLASS_T, PARAMS_T>::computeGrad(const Eigen::Ref<const state_array>& state,
                                                     const Eigen::Ref<const control_array>& control, Eigen::Ref<dfdx> A,
                                                     Eigen::Ref<dfdu> B)
{
  return false;
}

template <class CLASS_T, class PARAMS_T>
void RacerDubinsImpl<CLASS_T, PARAMS_T>::updateState(const Eigen::Ref<const state_array> state,
                                                     Eigen::Ref<state_array> next_state,
                                                     Eigen::Ref<state_array> state_der, const float dt)
{
  // Segmented it to ensure that roll and pitch don't get overwritten
  for (int i = 0; i < 6; i++)
  {
    next_state[i] = state[i] + state_der[i] * dt;
  }
  next_state(S_INDEX(YAW)) = angle_utils::normalizeAngle(next_state(S_INDEX(YAW)));
  next_state(S_INDEX(STEER_ANGLE)) =
      max(min(next_state(S_INDEX(STEER_ANGLE)), this->params_.max_steer_angle), -this->params_.max_steer_angle);
  next_state(S_INDEX(STEER_ANGLE_RATE)) = state_der(S_INDEX(STEER_ANGLE));
  next_state(S_INDEX(BRAKE_STATE)) =
      min(max(next_state(S_INDEX(BRAKE_STATE)), 0.0f), -this->control_rngs_[C_INDEX(THROTTLE_BRAKE)].x);
}

template <class CLASS_T, class PARAMS_T>
RacerDubinsImpl<CLASS_T, PARAMS_T>::state_array RacerDubinsImpl<CLASS_T, PARAMS_T>::interpolateState(
    const Eigen::Ref<state_array> state_1, const Eigen::Ref<state_array> state_2, const float alpha)
{
  state_array result = (1 - alpha) * state_1 + alpha * state_2;
  result(S_INDEX(YAW)) = angle_utils::interpolateEulerAngleLinear(state_1(S_INDEX(YAW)), state_2(S_INDEX(YAW)), alpha);
  return result;
}

template <class CLASS_T, class PARAMS_T>
__device__ void RacerDubinsImpl<CLASS_T, PARAMS_T>::updateState(float* state, float* next_state, float* state_der,
                                                                const float dt)
{
  int i;
  int tdy = threadIdx.y;
  // Add the state derivative time dt to the current state.
  // printf("updateState thread %d, %d = %f, %f\n", threadIdx.x, threadIdx.y, state[0], state_der[0]);
  for (i = tdy; i < 6; i += blockDim.y)
  {
    next_state[i] = state[i] + state_der[i] * dt;
    if (i == S_INDEX(YAW))
    {
      next_state[i] = angle_utils::normalizeAngle(next_state[i]);
    }
    if (i == S_INDEX(STEER_ANGLE))
    {
      next_state[i] = max(min(next_state[i], this->params_.max_steer_angle), -this->params_.max_steer_angle);
      next_state[S_INDEX(STEER_ANGLE_RATE)] = state_der[i];
    }
    if (i == S_INDEX(BRAKE_STATE))
    {
      next_state[i] = min(max(next_state[i], 0.0f), 1.0f);
    }
  }
}

template <class CLASS_T, class PARAMS_T>
Eigen::Quaternionf RacerDubinsImpl<CLASS_T, PARAMS_T>::attitudeFromState(const Eigen::Ref<const state_array>& state)
{
  float theta = state[S_INDEX(YAW)];
  return Eigen::Quaternionf(cos(theta / 2), 0, 0, sin(theta / 2));
}

template <class CLASS_T, class PARAMS_T>
Eigen::Vector3f RacerDubinsImpl<CLASS_T, PARAMS_T>::positionFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[S_INDEX(POS_X)], state[S_INDEX(POS_Y)], 0);
}

template <class CLASS_T, class PARAMS_T>
Eigen::Vector3f RacerDubinsImpl<CLASS_T, PARAMS_T>::velocityFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[S_INDEX(VEL_X)], 0, 0);
}

template <class CLASS_T, class PARAMS_T>
Eigen::Vector3f RacerDubinsImpl<CLASS_T, PARAMS_T>::angularRateFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(0, 0, 0);  // TODO compute yaw rate from steering angle & vel
}

template <class CLASS_T, class PARAMS_T>
RacerDubinsImpl<CLASS_T, PARAMS_T>::state_array RacerDubinsImpl<CLASS_T, PARAMS_T>::stateFromOdometry(
    const Eigen::Quaternionf& q, const Eigen::Vector3f& pos, const Eigen::Vector3f& vel, const Eigen::Vector3f& omega)
{
  state_array s;
  s.setZero();
  s[S_INDEX(POS_X)] = pos[0];
  s[S_INDEX(POS_Y)] = pos[1];
  s[S_INDEX(VEL_X)] = vel[0];
  float _roll, _pitch, yaw;
  mppi::math::Quat2EulerNWU(q, _roll, _pitch, yaw);
  s[S_INDEX(YAW)] = yaw;
  return s;
}

template <class CLASS_T, class PARAMS_T>
__device__ void RacerDubinsImpl<CLASS_T, PARAMS_T>::computeDynamics(float* state, float* control, float* state_der,
                                                                    float* theta_s)
{
  bool enable_brake = control[C_INDEX(THROTTLE_BRAKE)] < 0;

  state_der[S_INDEX(BRAKE_STATE)] =
      min(max((enable_brake * -control[C_INDEX(THROTTLE_BRAKE)] - state[S_INDEX(BRAKE_STATE)]) *
                  this->params_.brake_delay_constant,
              -this->params_.max_brake_rate_neg),
          this->params_.max_brake_rate_pos);

  // applying position throttle
  state_der[S_INDEX(VEL_X)] =
      (!enable_brake) * this->params_.c_t[0] * control[0] * this->params_.gear_sign +
      this->params_.c_b[0] * state[S_INDEX(BRAKE_STATE)] * (state[S_INDEX(VEL_X)] >= 0 ? -1 : 1) -
      this->params_.c_v[0] * state[S_INDEX(VEL_X)] + this->params_.c_0;
  state_der[S_INDEX(YAW)] = (state[S_INDEX(VEL_X)] / this->params_.wheel_base) *
                            tan(state[S_INDEX(STEER_ANGLE)] / this->params_.steer_angle_scale);
  state_der[S_INDEX(POS_X)] = state[S_INDEX(VEL_X)] * cosf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(POS_Y)] = state[S_INDEX(VEL_X)] * sinf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(STEER_ANGLE)] =
      max(min((control[1] * this->params_.steer_command_angle_scale - state[S_INDEX(STEER_ANGLE)]) *
                  this->params_.steering_constant,
              this->params_.max_steer_rate),
          -this->params_.max_steer_rate);
}

template <class CLASS_T, class PARAMS_T>
void RacerDubinsImpl<CLASS_T, PARAMS_T>::getStoppingControl(const Eigen::Ref<const state_array>& state,
                                                            Eigen::Ref<control_array> u)
{
  u[0] = -1.0;  // full brake
  u[1] = 0.0;   // no steering
}

template <class CLASS_T, class PARAMS_T>
void RacerDubinsImpl<CLASS_T, PARAMS_T>::enforceLeash(const Eigen::Ref<const state_array>& state_true,
                                                      const Eigen::Ref<const state_array>& state_nominal,
                                                      const Eigen::Ref<const state_array>& leash_values,
                                                      Eigen::Ref<state_array> state_output)
{
  state_output = state_true;

  // update state_output for leash, need to handle x and y positions specially, convert to body frame and leash in body
  // frame. transform x and y into body frame
  float dx = state_nominal[S_INDEX(POS_X)] - state_true[S_INDEX(POS_X)];
  float dy = state_nominal[S_INDEX(POS_Y)] - state_true[S_INDEX(POS_Y)];
  float dx_body = dx * cos(state_true[S_INDEX(YAW)]) + dy * sin(state_true[S_INDEX(YAW)]);
  float dy_body = -dx * sin(state_true[S_INDEX(YAW)]) + dy * cos(state_true[S_INDEX(YAW)]);

  // determine leash in body frame
  float y_leash = leash_values[S_INDEX(POS_Y)];
  float x_leash = leash_values[S_INDEX(POS_X)];
  dx_body = fminf(fmaxf(dx_body, -x_leash), x_leash);
  dy_body = fminf(fmaxf(dy_body, -y_leash), y_leash);

  // transform back to map frame
  dx = dx_body * cos(state_true[S_INDEX(YAW)]) + -dy_body * sin(state_true[S_INDEX(YAW)]);
  dy = dx_body * sin(state_true[S_INDEX(YAW)]) + dy_body * cos(state_true[S_INDEX(YAW)]);

  // apply leash
  state_output[S_INDEX(POS_X)] += dx;
  state_output[S_INDEX(POS_Y)] += dy;

  // handle leash for rest of states
  float diff;
  for (int i = 0; i < PARENT_CLASS::STATE_DIM; i++)
  {
    // use body x and y for leash
    if (i == S_INDEX(POS_X) || i == S_INDEX(POS_Y))
    {
      // handle outside for loop
      continue;
    }
    else if (i == S_INDEX(YAW))
    {
      diff = angle_utils::shortestAngularDistance(state_true[i], state_nominal[i]);
    }
    else
    {
      diff = state_nominal[i] - state_true[i];
    }

    if (leash_values[i] < fabsf(diff))
    {
      float leash_dir = fminf(fmaxf(diff, -leash_values[i]), leash_values[i]);
      state_output[i] = state_true[i] + leash_dir;
      if (i == S_INDEX(YAW))
      {
        state_output[i] = angle_utils::normalizeAngle(state_output[i]);
      }
    }
    else
    {
      state_output[i] = state_nominal[i];
    }
  }
}

template <class CLASS_T, class PARAMS_T>
RacerDubinsImpl<CLASS_T, PARAMS_T>::state_array
RacerDubinsImpl<CLASS_T, PARAMS_T>::stateFromMap(const std::map<std::string, float>& map)
{
  state_array s = state_array::Zero();
  if (map.find("VEL_X") == map.end() || map.find("VEL_Y") == map.end() || map.find("POS_X") == map.end() ||
      map.find("POS_Y") == map.end())
  {
    return s;
  }
  s(S_INDEX(POS_X)) = map.at("POS_X");
  s(S_INDEX(POS_Y)) = map.at("POS_Y");
  s(S_INDEX(VEL_X)) = map.at("VEL_X");
  s(S_INDEX(YAW)) = map.at("YAW");
  if (map.find("STEER_ANGLE") != map.end())
  {
    s(S_INDEX(STEER_ANGLE)) = map.at("STEER_ANGLE");
    s(S_INDEX(STEER_ANGLE_RATE)) = map.at("STEER_ANGLE_RATE");
  }
  else
  {
    s(S_INDEX(STEER_ANGLE)) = 0;
    s(S_INDEX(STEER_ANGLE_RATE)) = 0;
  }
  if (map.find("BRAKE_STATE") != map.end())
  {
    s(S_INDEX(BRAKE_STATE)) = map.at("BRAKE_STATE");
  }
  else if (map.find("BRAKE_CMD") != map.end())
  {
    s(S_INDEX(BRAKE_STATE)) = map.at("BRAKE_CMD");
  }
  else
  {
    s(S_INDEX(BRAKE_STATE)) = 0;
  }

  return s;
}

template <class OUTPUT_T, class TEX_T>
__device__ __host__ void RACER::computeStaticSettling(TEX_T* tex_helper, const float yaw, const float x, const float y,
                                                      float& roll, float& pitch, float* output)
{
  float height = 0.0f;

  float3 front_left = make_float3(2.981f, 0.737f, 0.0f);
  float3 front_right = make_float3(2.981f, -0.737f, 0.f);
  float3 rear_left = make_float3(0.0f, 0.737f, 0.0f);
  float3 rear_right = make_float3(0.0f, -0.737f, 0.0f);
  float3 body_pose = make_float3(x, y, 0.0f);
  float3 rotation = make_float3(roll, pitch, yaw);
  // front_left = make_float3(front_left.x * cosf(yaw) - front_left.y * sinf(yaw) + x,
  //                          front_left.x * sinf(yaw) + front_left.y * cosf(yaw) + y, 0.0f);
  // front_right = make_float3(front_right.x * cosf(yaw) - front_right.y * sinf(yaw) + x,
  //                           front_right.x * sinf(yaw) + front_right.y * cosf(yaw) + y, 0.0f);
  // rear_left = make_float3(rear_left.x * cosf(yaw) - rear_left.y * sinf(yaw) + x,
  //                         rear_left.x * sinf(yaw) + rear_left.y * cosf(yaw) + y, 0.0f);
  // rear_right = make_float3(rear_right.x * cosf(yaw) - rear_right.y * sinf(yaw) + x,
  //                          rear_right.x * sinf(yaw) + rear_right.y * cosf(yaw) + y, 0.0f);
  float front_left_height = 0.0f;
  float front_right_height = 0.0f;
  float rear_left_height = 0.0f;
  float rear_right_height = 0.0f;

  if (tex_helper->checkTextureUse(0))
  {
    front_left_height = tex_helper->queryTextureAtWorldOffsetPose(0, body_pose, front_left, rotation);
    front_right_height = tex_helper->queryTextureAtWorldOffsetPose(0, body_pose, front_right, rotation);
    rear_left_height = tex_helper->queryTextureAtWorldOffsetPose(0, body_pose, rear_left, rotation);
    rear_right_height = tex_helper->queryTextureAtWorldOffsetPose(0, body_pose, rear_right, rotation);

    float front_diff = front_left_height - front_right_height;
    front_diff = max(min(front_diff, 0.736f * 2.0f), -0.736f * 2.0f);
    float rear_diff = rear_left_height - rear_right_height;
    rear_diff = max(min(rear_diff, 0.736f * 2.0f), -0.736f * 2.0f);
    float front_roll = asinf(front_diff / (0.737f * 2.0f));
    float rear_roll = asinf(rear_diff / (0.737f * 2.0f));
    roll = (front_roll + rear_roll) / 2.0f;

    float left_diff = rear_left_height - front_left_height;
    left_diff = max(min(left_diff, 2.98f), -2.98f);
    float right_diff = rear_right_height - front_right_height;
    right_diff = max(min(right_diff, 2.98f), -2.98f);
    float left_pitch = asinf((left_diff) / 2.981f);
    float right_pitch = asinf((right_diff) / 2.981f);
    pitch = (left_pitch + right_pitch) / 2.0f;

    height = (rear_left_height + rear_right_height) / 2.0f;
  }
  else
  {
    roll = 0.0f;
    pitch = 0.0f;
    height = 0.0f;
  }

  // using 2pi so any rotation that accidently uses this will be using identity
  if (isnan(roll) || isinf(roll) || fabsf(roll) > M_PIf32)
  {
    roll = 2.0f * M_PIf32;
  }
  if (isnan(pitch) || isinf(pitch) || fabsf(pitch) > M_PIf32)
  {
    pitch = 2.0f * M_PIf32;
  }
  if (isnan(height) || isinf(height))
  {
    height = 0.0f;
  }

  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_FL_X)] = front_left.x;
  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_FL_Y)] = front_left.y;
  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_FR_X)] = front_right.x;
  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_FR_Y)] = front_right.y;
  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_RL_X)] = rear_left.x;
  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_RL_Y)] = rear_left.y;
  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_RR_X)] = rear_right.x;
  output[E_INDEX(OUTPUT_T, WHEEL_POS_I_RR_Y)] = rear_right.y;
  output[E_INDEX(OUTPUT_T, BASELINK_POS_I_Z)] = height;
}
