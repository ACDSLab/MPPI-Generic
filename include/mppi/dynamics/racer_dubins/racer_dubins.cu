#include "hip/hip_runtime.h"
#include <mppi/dynamics/racer_dubins/racer_dubins.cuh>

void RacerDubins::computeDynamics(const Eigen::Ref<const state_array>& state,
                                  const Eigen::Ref<const control_array>& control, Eigen::Ref<state_array> state_der)
{
  state_der(0) = this->params_.c_t * control(0) - this->params_.c_b * control(1) - this->params_.c_v * state(0) +
                 this->params_.c_0;
  state_der(1) = (state(0) / this->params_.wheel_base) * tan(state(4));
  state_der(2) = state(0) * cosf(state(1));
  state_der(3) = state(0) * sinf(state(1));
  // state_der(4) = (control(2) - state(4));
  state_der(4) = control(2) * this->params_.max_steer_angle;
}

bool RacerDubins::computeGrad(const Eigen::Ref<const state_array>& state,
                              const Eigen::Ref<const control_array>& control, Eigen::Ref<dfdx> A, Eigen::Ref<dfdu> B)
{
  return false;
}

void RacerDubins::updateState(Eigen::Ref<state_array> state, Eigen::Ref<state_array> state_der, const float dt)
{
  state += state_der * dt;
  state(1) = angle_utils::normalizeAngle(state(1));
  state(4) -= state_der(4) * dt;
  state(4) = state_der(4) + (state(4) - state_der(4)) * expf(-this->params_.steering_constant * dt);
  state_der.setZero();
}

RacerDubins::state_array RacerDubins::interpolateState(const Eigen::Ref<state_array> state_1,
                                                       const Eigen::Ref<state_array> state_2, const double alpha)
{
  state_array result = (1 - alpha) * state_1 + alpha * state_2;
  result(1) = angle_utils::interpolateEulerAngleLinear(state_1(1), state_2(1), alpha);
  return result;
}

__device__ void RacerDubins::updateState(float* state, float* state_der, const float dt)
{
  int i;
  int tdy = threadIdx.y;
  // Add the state derivative time dt to the current state.
  // printf("updateState thread %d, %d = %f, %f\n", threadIdx.x, threadIdx.y, state[0], state_der[0]);
  for (i = tdy; i < STATE_DIM; i += blockDim.y)
  {
    state[i] += state_der[i] * dt;
    if (i == 1)
    {
      state[i] = angle_utils::normalizeAngle(state[i]);
    }
    if (i == 4)
    {
      state[i] -= state_der[i] * dt;
      state[i] = state_der[i] + (state[i] - state_der[i]) * expf(-this->params_.steering_constant * dt);
      // state[i] += state_der[i] * expf(-this->params_.steering_constant * dt);
    }
    state_der[i] = 0;  // Important: reset the state derivative to zero.
  }
}

__device__ void RacerDubins::computeDynamics(float* state, float* control, float* state_der, float* theta_s)
{
  state_der[0] = this->params_.c_t * control[0] - this->params_.c_b * control[1] - this->params_.c_v * state[0] +
                 this->params_.c_0;
  state_der[1] = (state[0] / this->params_.wheel_base) * tan(state[4]);
  state_der[2] = state[0] * cosf(state[1]);
  state_der[3] = state[0] * sinf(state[1]);
  // state_der[4] = (control[2] - state[4]);
  state_der[4] = control[2] * this->params_.max_steer_angle;
}
