#include "hip/hip_runtime.h"
#include <mppi/dynamics/racer_dubins/racer_dubins.cuh>

template <class CLASS_T>
void RacerDubinsImpl<CLASS_T>::computeDynamics(const Eigen::Ref<const state_array>& state,
                                               const Eigen::Ref<const control_array>& control,
                                               Eigen::Ref<state_array> state_der)
{
  bool enable_brake = control(CTRL_THROTTLE_BRAKE) < 0;
  // applying position throttle
  state_der(STATE_V) =
      (!enable_brake) * this->params_.c_t * control(CTRL_THROTTLE_BRAKE) +
      (enable_brake) * this->params_.c_b * control(CTRL_THROTTLE_BRAKE) * (state(STATE_V) >= 0 ? 1 : -1) -
      this->params_.c_v * state(STATE_V) + this->params_.c_0;
  state_der(STATE_YAW) = (state(STATE_V) / this->params_.wheel_base) * tan(state(4));
  state_der(STATE_PX) = state(STATE_V) * cosf(state(STATE_YAW));
  state_der(STATE_PY) = state(STATE_V) * sinf(state(STATE_YAW));
  state_der(STATE_STEER) = control(CTRL_STEER_CMD) / this->params_.steer_command_angle_scale;
}

template <class CLASS_T>
bool RacerDubinsImpl<CLASS_T>::computeGrad(const Eigen::Ref<const state_array>& state,
                                           const Eigen::Ref<const control_array>& control, Eigen::Ref<dfdx> A,
                                           Eigen::Ref<dfdu> B)
{
  return false;
}

template <class CLASS_T>
void RacerDubinsImpl<CLASS_T>::updateState(Eigen::Ref<state_array> state, Eigen::Ref<state_array> state_der,
                                           const float dt)
{
  state += state_der * dt;
  state(STATE_YAW) = angle_utils::normalizeAngle(state(STATE_YAW));
  state(STATE_STEER) -= state_der(STATE_STEER) * dt;
  state(STATE_STEER) = state_der(STATE_STEER) +
                       (state(STATE_STEER) - state_der(STATE_STEER)) * expf(-this->params_.steering_constant * dt);
  state_der.setZero();
}

template <class CLASS_T>
RacerDubinsImpl<CLASS_T>::state_array RacerDubinsImpl<CLASS_T>::interpolateState(const Eigen::Ref<state_array> state_1,
                                                                                 const Eigen::Ref<state_array> state_2,
                                                                                 const float alpha)
{
  state_array result = (1 - alpha) * state_1 + alpha * state_2;
  result(1) = angle_utils::interpolateEulerAngleLinear(state_1(1), state_2(1), alpha);
  return result;
}

template <class CLASS_T>
__device__ void RacerDubinsImpl<CLASS_T>::updateState(float* state, float* state_der, const float dt)
{
  int i;
  int tdy = threadIdx.y;
  // Add the state derivative time dt to the current state.
  // printf("updateState thread %d, %d = %f, %f\n", threadIdx.x, threadIdx.y, state[0], state_der[0]);
  for (i = tdy; i < PARENT_CLASS::STATE_DIM; i += blockDim.y)
  {
    state[i] += state_der[i] * dt;
    if (i == STATE_YAW)
    {
      state[i] = angle_utils::normalizeAngle(state[i]);
    }
    if (i == STATE_STEER)
    {
      state[i] -= state_der[i] * dt;
      state[i] = state_der[i] + (state[i] - state_der[i]) * expf(-this->params_.steering_constant * dt);
      // state[i] += state_der[i] * expf(-this->params_.steering_constant * dt);
    }
    state_der[i] = 0;  // Important: reset the state derivative to zero.
  }
}

template <class CLASS_T>
Eigen::Quaternionf RacerDubinsImpl<CLASS_T>::attitudeFromState(const Eigen::Ref<const state_array>& state)
{
  float theta = state[STATE_YAW];
  return Eigen::Quaternionf(cos(theta / 2), 0, 0, sin(theta / 2));
}

template <class CLASS_T>
Eigen::Vector3f RacerDubinsImpl<CLASS_T>::positionFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[STATE_PX], state[STATE_PY], 0);
}

template <class CLASS_T>
Eigen::Vector3f RacerDubinsImpl<CLASS_T>::velocityFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[STATE_V], 0, 0);
}

template <class CLASS_T>
Eigen::Vector3f RacerDubinsImpl<CLASS_T>::angularRateFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(0, 0, 0);  // TODO compute yaw rate from steering angle & vel
}

template <class CLASS_T>
RacerDubinsImpl<CLASS_T>::state_array RacerDubinsImpl<CLASS_T>::stateFromOdometry(const Eigen::Quaternionf& q,
                                                                                  const Eigen::Vector3f& pos,
                                                                                  const Eigen::Vector3f& vel,
                                                                                  const Eigen::Vector3f& omega)
{
  state_array s;
  s.setZero();
  s[STATE_PX] = pos[0];
  s[STATE_PY] = pos[1];
  s[STATE_V] = vel[0];
  float _roll, _pitch, yaw;
  mppi::math::Quat2EulerNWU(q, _roll, _pitch, yaw);
  s[STATE_YAW] = yaw;
  return s;
}

template <class CLASS_T>
__device__ void RacerDubinsImpl<CLASS_T>::computeDynamics(float* state, float* control, float* state_der,
                                                          float* theta_s)
{
  bool enable_brake = control[CTRL_THROTTLE_BRAKE] < 0;
  // applying position throttle
  state_der[STATE_V] =
      (!enable_brake) * this->params_.c_t * control[CTRL_THROTTLE_BRAKE] +
      (enable_brake) * this->params_.c_b * control[CTRL_THROTTLE_BRAKE] * (state[STATE_V] >= 0 ? 1 : -1) -
      this->params_.c_v * state[STATE_V] + this->params_.c_0;
  state_der[STATE_YAW] = (state[STATE_V] / this->params_.wheel_base) * tan(state[4]);
  state_der[STATE_PX] = state[STATE_V] * cosf(state[STATE_YAW]);
  state_der[STATE_PY] = state[STATE_V] * sinf(state[STATE_YAW]);
  state_der[STATE_STEER] = control[CTRL_STEER_CMD] / this->params_.steer_command_angle_scale;
}

template <class CLASS_T>
void RacerDubinsImpl<CLASS_T>::getStoppingControl(const Eigen::Ref<const state_array>& state,
                                                  Eigen::Ref<control_array> u)
{
  u[0] = -1.0;  // full brake
  u[1] = 0.0;   // no steering
}
