#include "hip/hip_runtime.h"
#include <mppi/dynamics/racer_dubins/racer_dubins.cuh>

template <class CLASS_T, int STATE_DIM>
void RacerDubinsImpl<CLASS_T, STATE_DIM>::computeDynamics(const Eigen::Ref<const state_array>& state,
                                                          const Eigen::Ref<const control_array>& control,
                                                          Eigen::Ref<state_array> state_der)
{
  bool enable_brake = control(0) < 0;
  // applying position throttle
  state_der(0) = (!enable_brake) * this->params_.c_t * control(0) +
                 (enable_brake) * this->params_.c_b * control(0) * (state(0) >= 0 ? 1 : -1) -
                 this->params_.c_v * state(0) + this->params_.c_0;
  state_der(1) = (state(0) / this->params_.wheel_base) * tan(state(4));
  state_der(2) = state(0) * cosf(state(1));
  state_der(3) = state(0) * sinf(state(1));
  state_der(4) = control(1) / this->params_.steer_command_angle_scale;
}

template <class CLASS_T, int STATE_DIM>
bool RacerDubinsImpl<CLASS_T, STATE_DIM>::computeGrad(const Eigen::Ref<const state_array>& state,
                                                      const Eigen::Ref<const control_array>& control,
                                                      Eigen::Ref<dfdx> A, Eigen::Ref<dfdu> B)
{
  return false;
}

template <class CLASS_T, int STATE_DIM>
void RacerDubinsImpl<CLASS_T, STATE_DIM>::updateState(Eigen::Ref<state_array> state, Eigen::Ref<state_array> state_der,
                                                      const float dt)
{
  state += state_der * dt;
  state(1) = angle_utils::normalizeAngle(state(1));
  state(4) -= state_der(4) * dt;
  state(4) = state_der(4) + (state(4) - state_der(4)) * expf(-this->params_.steering_constant * dt);
  state_der.setZero();
}

template <class CLASS_T, int STATE_DIM>
RacerDubinsImpl<CLASS_T, STATE_DIM>::state_array RacerDubinsImpl<CLASS_T, STATE_DIM>::interpolateState(
    const Eigen::Ref<state_array> state_1, const Eigen::Ref<state_array> state_2, const float alpha)
{
  state_array result = (1 - alpha) * state_1 + alpha * state_2;
  result(1) = angle_utils::interpolateEulerAngleLinear(state_1(1), state_2(1), alpha);
  return result;
}

template <class CLASS_T, int STATE_DIM>
__device__ void RacerDubinsImpl<CLASS_T, STATE_DIM>::updateState(float* state, float* state_der, const float dt)
{
  int i;
  int tdy = threadIdx.y;
  // Add the state derivative time dt to the current state.
  // printf("updateState thread %d, %d = %f, %f\n", threadIdx.x, threadIdx.y, state[0], state_der[0]);
  for (i = tdy; i < STATE_DIM; i += blockDim.y)
  {
    state[i] += state_der[i] * dt;
    if (i == 1)
    {
      state[i] = angle_utils::normalizeAngle(state[i]);
    }
    if (i == 4)
    {
      state[i] -= state_der[i] * dt;
      state[i] = state_der[i] + (state[i] - state_der[i]) * expf(-this->params_.steering_constant * dt);
      // state[i] += state_der[i] * expf(-this->params_.steering_constant * dt);
    }
    state_der[i] = 0;  // Important: reset the state derivative to zero.
  }
}

template <class CLASS_T, int STATE_DIM>
Eigen::Quaternionf RacerDubinsImpl<CLASS_T, STATE_DIM>::attitudeFromState(const Eigen::Ref<const state_array>& state)
{
  float theta = state[STATE_YAW];
  return Eigen::Quaternionf(cos(theta / 2), 0, 0, sin(theta / 2));
}

template <class CLASS_T, int STATE_DIM>
Eigen::Vector3f RacerDubinsImpl<CLASS_T, STATE_DIM>::positionFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[STATE_PX], state[STATE_PY], 0);
}

template <class CLASS_T, int STATE_DIM>
Eigen::Vector3f RacerDubinsImpl<CLASS_T, STATE_DIM>::velocityFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[STATE_V], 0, 0);
}

template <class CLASS_T, int STATE_DIM>
Eigen::Vector3f RacerDubinsImpl<CLASS_T, STATE_DIM>::angularRateFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(0, 0, 0);  // TODO compute yaw rate from steering angle & vel
}

template <class CLASS_T, int STATE_DIM>
RacerDubinsImpl<CLASS_T, STATE_DIM>::state_array RacerDubinsImpl<CLASS_T, STATE_DIM>::stateFromOdometry(
    const Eigen::Quaternionf& q, const Eigen::Vector3f& pos, const Eigen::Vector3f& vel, const Eigen::Vector3f& omega)
{
  state_array s;
  s.setZero();
  s[STATE_PX] = pos[0];
  s[STATE_PY] = pos[1];
  s[STATE_V] = vel[0];
  float _roll, _pitch, yaw;
  mppi_math::Quat2EulerNWU(q, _roll, _pitch, yaw);
  s[STATE_YAW] = yaw;
  return s;
}

template <class CLASS_T, int STATE_DIM>
__device__ void RacerDubinsImpl<CLASS_T, STATE_DIM>::computeDynamics(float* state, float* control, float* state_der,
                                                                     float* theta_s)
{
  bool enable_brake = control[0] < 0;
  // applying position throttle
  state_der[0] = (!enable_brake) * this->params_.c_t * control[0] +
                 (enable_brake) * this->params_.c_b * control[0] * (state[0] >= 0 ? 1 : -1) -
                 this->params_.c_v * state[0] + this->params_.c_0;
  state_der[1] = (state[0] / this->params_.wheel_base) * tan(state[4]);
  state_der[2] = state[0] * cosf(state[1]);
  state_der[3] = state[0] * sinf(state[1]);
  state_der[4] = control[1] / this->params_.steer_command_angle_scale;
}

template <class CLASS_T, int STATE_DIM>
void RacerDubinsImpl<CLASS_T, STATE_DIM>::getStoppingControl(const Eigen::Ref<const state_array>& state,
                                                             Eigen::Ref<control_array> u)
{
  u[0] = -1.0;  // full brake
  u[1] = 0.0;   // no steering
}
