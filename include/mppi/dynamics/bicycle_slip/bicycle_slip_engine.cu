#include "hip/hip_runtime.h"
//
// Created by jason on 9/7/22.
//

#include "bicycle_slip_engine.cuh"

BicycleSlipEngine::BicycleSlipEngine(hipStream_t stream)
  : MPPI_internal::Dynamics<BicycleSlipEngine, BicycleSlipEngineParams>(stream)
{
  this->requires_buffer_ = true;
  tex_helper_ = new TwoDTextureHelper<float>(1, stream);
  steer_lstm_lstm_helper_ = std::make_shared<STEER_NN>(stream);
  delay_lstm_lstm_helper_ = std::make_shared<DELAY_NN>(stream);
  engine_lstm_lstm_helper_ = std::make_shared<ENGINE_NN>(stream);
  terra_lstm_lstm_helper_ = std::make_shared<TERRA_NN>(stream);
}

BicycleSlipEngine::BicycleSlipEngine(std::string ackerman_path, hipStream_t stream)
  : MPPI_internal::Dynamics<BicycleSlipEngine, BicycleSlipEngineParams>(stream)
{
  this->requires_buffer_ = true;
  tex_helper_ = new TwoDTextureHelper<float>(1, stream);

  delay_lstm_lstm_helper_ = std::make_shared<DELAY_NN>(stream);
  engine_lstm_lstm_helper_ = std::make_shared<ENGINE_NN>(stream);
  terra_lstm_lstm_helper_ = std::make_shared<TERRA_NN>(stream);
  steer_lstm_lstm_helper_ = std::make_shared<STEER_NN>(stream);

  if (!fileExists(ackerman_path))
  {
    std::cerr << "Could not load neural net model at ackerman_path: " << ackerman_path.c_str();
    exit(-1);
  }
  cnpy::npz_t param_dict = cnpy::npz_load(ackerman_path);
  this->params_.gravity = param_dict.at("params/gravity").data<float>()[0];
  this->params_.wheel_angle_scale = param_dict.at("params/wheel_angle_scale").data<float>()[0];
  this->params_.steer_angle_scale = param_dict.at("params/steer_angle_scale").data<float>()[0];

  // load the delay params
  this->params_.brake_delay_constant = param_dict.at("delay_model/params/brake_constant").data<float>()[0];
  this->params_.max_brake_rate_neg = param_dict.at("delay_model/params/max_brake_rate_neg").data<float>()[0];
  this->params_.max_brake_rate_pos = param_dict.at("delay_model/params/max_brake_rate_pos").data<float>()[0];

  // load the steering parameters
  this->params_.max_steer_rate = param_dict.at("steer_model/params/max_steer_rate").data<float>()[0];
  this->params_.steering_constant = param_dict.at("steer_model/params/steering_constant").data<float>()[0];

  delay_lstm_lstm_helper_->loadParams("delay_model/model", ackerman_path);
  terra_lstm_lstm_helper_->loadParams("terra_model", ackerman_path);
  engine_lstm_lstm_helper_->loadParams("engine_model", ackerman_path);
  steer_lstm_lstm_helper_->loadParams("steer_model/model", ackerman_path);
}

void BicycleSlipEngine::initializeDynamics(const Eigen::Ref<const state_array>& state,
                                           const Eigen::Ref<const control_array>& control,
                                           Eigen::Ref<output_array> output, float t_0, float dt)
{
  this->steer_lstm_lstm_helper_->resetLSTMHiddenCellCPU();
  this->delay_lstm_lstm_helper_->resetLSTMHiddenCellCPU();
  this->engine_lstm_lstm_helper_->resetLSTMHiddenCellCPU();
  this->terra_lstm_lstm_helper_->resetLSTMHiddenCellCPU();
  PARENT_CLASS::initializeDynamics(state, control, output, t_0, dt);
}

MPPI_internal::Dynamics<BicycleSlipEngine, BicycleSlipEngineParams>::state_array
BicycleSlipEngine::stateFromMap(const std::map<std::string, float>& map)
{
  state_array s = state_array::Zero();
  if (map.find("VEL_X") == map.end() || map.find("VEL_Y") == map.end() || map.find("POS_X") == map.end() ||
      map.find("POS_Y") == map.end() || map.find("ROLL") == map.end() || map.find("PITCH") == map.end())
  {
    std::cout << "WARNING: could not find all keys for ackerman slip dynamics" << std::endl;
    for (const auto& it : map)
    {
      std::cout << "got key " << it.first << std::endl;
    }
    return s;
  }
  s(S_INDEX(POS_X)) = map.at("POS_X");
  s(S_INDEX(POS_Y)) = map.at("POS_Y");
  s(S_INDEX(VEL_X)) = map.at("VEL_X");
  s(S_INDEX(VEL_Y)) = map.at("VEL_Y");
  s(S_INDEX(OMEGA_Z)) = map.at("OMEGA_Z");
  s(S_INDEX(YAW)) = map.at("YAW");
  s(S_INDEX(ROLL)) = map.at("ROLL");
  s(S_INDEX(PITCH)) = map.at("PITCH");
  if (map.find("STEER_ANGLE") != map.end())
  {
    s(S_INDEX(STEER_ANGLE)) = map.at("STEER_ANGLE");
    s(S_INDEX(STEER_ANGLE_RATE)) = map.at("STEER_ANGLE_RATE");
  }
  else
  {
    std::cout << "WARNING: unable to find BRAKE_STATE or STEER_ANGLE_RATE, using 0" << std::endl;
    s(S_INDEX(STEER_ANGLE)) = 0;
    s(S_INDEX(STEER_ANGLE_RATE)) = 0;
  }
  if (map.find("BRAKE_STATE") != map.end())
  {
    s(S_INDEX(BRAKE_STATE)) = map.at("BRAKE_STATE");
  }
  else if (map.find("BRAKE_CMD") != map.end())
  {
    std::cout << "WARNING: unable to find BRAKE_STATE" << std::endl;
    s(S_INDEX(BRAKE_STATE)) = map.at("BRAKE_CMD");
  }
  else
  {
    std::cout << "WARNING: unable to find BRAKE_CMD or BRAKE_STATE" << std::endl;
    s(S_INDEX(BRAKE_STATE)) = 0;
  }
  return s;
}

void BicycleSlipEngine::updateFromBuffer(const buffer_trajectory& buffer)
{
  if (buffer.find("VEL_X") == buffer.end() || buffer.find("VEL_Y") == buffer.end() ||
      buffer.find("STEER_ANGLE") == buffer.end() || buffer.find("STEER_ANGLE_RATE") == buffer.end() ||
      buffer.find("STEER_CMD") == buffer.end() || buffer.find("BRAKE_STATE") == buffer.end())
  {
    std::cout << "WARNING: not using init buffer" << std::endl;
    for (const auto& it : buffer)
    {
      std::cout << "got key " << it.first << std::endl;
    }
    return;
  }

  STEER_NN::init_buffer steer_init_buffer;
  steer_init_buffer.row(0) = buffer.at("VEL_X");
  steer_init_buffer.row(1) = buffer.at("STEER_ANGLE");
  steer_init_buffer.row(2) = buffer.at("STEER_ANGLE_RATE");
  steer_init_buffer.row(3) = buffer.at("STEER_CMD");
  steer_lstm_lstm_helper_->initializeLSTM(steer_init_buffer);

  DELAY_NN::init_buffer delay_init_buffer;
  delay_init_buffer.row(0) = buffer.at("BRAKE_STATE");
  delay_init_buffer.row(1) = buffer.at("BRAKE_CMD");
  delay_lstm_lstm_helper_->initializeLSTM(delay_init_buffer);

  ENGINE_NN::init_buffer engine_init_buffer;
  engine_init_buffer.row(0) = buffer.at("VEL_X");
  engine_init_buffer.row(1) = buffer.at("THROTTLE_CMD");
  engine_init_buffer.row(2) = buffer.at("BRAKE_STATE");
  engine_lstm_lstm_helper_->initializeLSTM(engine_init_buffer);

  TERRA_NN::init_buffer terra_init_buffer;
  terra_init_buffer.row(0) = buffer.at("VEL_X");
  terra_init_buffer.row(1) = buffer.at("VEL_Y");
  terra_init_buffer.row(2) = buffer.at("OMEGA_Z");
  terra_init_buffer.row(3) = buffer.at("STEER_ANGLE");
  terra_init_buffer.row(4) = buffer.at("STEER_ANGLE_RATE");
  // TODO should be pulled from elevation map to be entirely correct
  terra_init_buffer.row(5) = buffer.at("PITCH").unaryExpr([](float x) { return sinf(x); }) * this->params_.gravity;
  terra_init_buffer.row(6) = buffer.at("ROLL").unaryExpr([](float x) { return sinf(x); }) * this->params_.gravity;
  terra_init_buffer.row(7) =
      buffer.at("STEER_ANGLE").unaryExpr([this](float x) { return tanf(x / this->params_.wheel_angle_scale); });
  terra_lstm_lstm_helper_->initializeLSTM(terra_init_buffer);
}

void BicycleSlipEngine::GPUSetup()
{
  steer_lstm_lstm_helper_->GPUSetup();
  delay_lstm_lstm_helper_->GPUSetup();
  engine_lstm_lstm_helper_->GPUSetup();
  terra_lstm_lstm_helper_->GPUSetup();

  // makes sure that the device ptr sees the correct lstm model
  this->steer_network_d_ = steer_lstm_lstm_helper_->getLSTMDevicePtr();
  this->delay_network_d_ = delay_lstm_lstm_helper_->getLSTMDevicePtr();
  this->engine_network_d_ = engine_lstm_lstm_helper_->getLSTMDevicePtr();
  this->terra_network_d_ = terra_lstm_lstm_helper_->getLSTMDevicePtr();

  PARENT_CLASS::GPUSetup();
  tex_helper_->GPUSetup();
  // makes sure that the device ptr sees the correct texture object
  HANDLE_ERROR(hipMemcpyAsync(&(this->model_d_->tex_helper_), &(tex_helper_->ptr_d_),
                               sizeof(TwoDTextureHelper<float>*), hipMemcpyHostToDevice, this->stream_));
}

void BicycleSlipEngine::freeCudaMem()
{
  steer_lstm_lstm_helper_->freeCudaMem();
  delay_lstm_lstm_helper_->freeCudaMem();
  engine_lstm_lstm_helper_->freeCudaMem();
  terra_lstm_lstm_helper_->freeCudaMem();
  tex_helper_->freeCudaMem();
  Dynamics::freeCudaMem();
}

void BicycleSlipEngine::paramsToDevice()
{
  // does all the internal texture updates
  tex_helper_->copyToDevice();
  PARENT_CLASS::paramsToDevice();
}

void BicycleSlipEngine::computeDynamics(const Eigen::Ref<const state_array>& state,
                                        const Eigen::Ref<const control_array>& control,
                                        Eigen::Ref<state_array> state_der)
{
  state_der = state_array::Zero();
  bool enable_brake = control[C_INDEX(THROTTLE_BRAKE)] < 0;
  float brake_cmd = -enable_brake * control(C_INDEX(THROTTLE_BRAKE));
  float throttle_cmd = !enable_brake * control(C_INDEX(THROTTLE_BRAKE));

  state_der(S_INDEX(BRAKE_STATE)) =
      min(max((brake_cmd - state(S_INDEX(BRAKE_STATE))) * this->params_.brake_delay_constant,
              -this->params_.max_brake_rate_neg),
          this->params_.max_brake_rate_pos);
  // TODO if low speed allow infinite brake, not sure if needed
  // TODO need parametric reverse

  // kinematics component
  state_der(S_INDEX(POS_X)) =
      state(S_INDEX(VEL_X)) * cosf(state(S_INDEX(YAW))) - state(S_INDEX(VEL_Y)) * sinf(state(S_INDEX(YAW)));
  state_der(S_INDEX(POS_Y)) =
      state(S_INDEX(VEL_X)) * sinf(state(S_INDEX(YAW))) + state(S_INDEX(VEL_Y)) * cosf(state(S_INDEX(YAW)));

  // runs the brake model
  DELAY_LSTM::input_array brake_input;
  brake_input(0) = state(S_INDEX(BRAKE_STATE));
  brake_input(1) = brake_cmd;
  brake_input(2) = state_der(S_INDEX(BRAKE_STATE));  // stand in for y velocity
  DELAY_LSTM::output_array brake_output = DELAY_LSTM::output_array::Zero();
  delay_lstm_lstm_helper_->forward(brake_input, brake_output);
  state_der(S_INDEX(BRAKE_STATE)) += brake_output(0);

  // runs the engine model
  ENGINE_LSTM::input_array engine_input;
  engine_input(0) = throttle_cmd;
  engine_input(1) = state(S_INDEX(VEL_X));
  engine_input(2) = state(S_INDEX(BRAKE_STATE));
  ENGINE_LSTM::output_array engine_output_arr = ENGINE_LSTM::output_array::Zero();
  engine_lstm_lstm_helper_->forward(engine_input, engine_output_arr);
  const float engine_output = engine_output_arr(0) * 10;

  // runs the parametric part of the steering model
  state_der(S_INDEX(STEER_ANGLE)) =
      (control(C_INDEX(STEER_CMD)) * this->params_.steer_command_angle_scale - state(S_INDEX(STEER_ANGLE))) *
      this->params_.steering_constant;
  state_der(S_INDEX(STEER_ANGLE)) =
      max(min(state_der(S_INDEX(STEER_ANGLE)), this->params_.max_steer_rate), -this->params_.max_steer_rate);

  // runs the steering model
  STEER_LSTM::input_array steer_input;
  steer_input(0) = state(S_INDEX(VEL_X));
  steer_input(1) = state(S_INDEX(STEER_ANGLE));
  steer_input(2) = state(S_INDEX(STEER_ANGLE_RATE));
  steer_input(3) = control(C_INDEX(STEER_CMD));
  steer_input(4) = state_der(S_INDEX(STEER_ANGLE));  // this is the parametric part as input
  STEER_LSTM::output_array steer_output = STEER_LSTM::output_array::Zero();
  steer_lstm_lstm_helper_->forward(steer_input, steer_output);
  state_der(S_INDEX(STEER_ANGLE)) += steer_output(0) * 10;

  const float delta = tanf(state(S_INDEX(STEER_ANGLE)) / this->params_.wheel_angle_scale);
  const float param_yaw_rate = (state(S_INDEX(VEL_X)) / this->params_.wheel_base) *
                               tan(state(S_INDEX(STEER_ANGLE)) / this->params_.steer_angle_scale);

  // runs the terra dynamics model
  TERRA_LSTM::input_array terra_input;
  terra_input(0) = state(S_INDEX(VEL_X));
  terra_input(1) = state(S_INDEX(VEL_Y));
  terra_input(2) = state(S_INDEX(OMEGA_Z));
  terra_input(3) = state(S_INDEX(STEER_ANGLE));
  terra_input(4) = state(S_INDEX(STEER_ANGLE_RATE));
  terra_input(5) = sinf(state(S_INDEX(PITCH))) * this->params_.gravity;
  terra_input(6) = sinf(state(S_INDEX(ROLL))) * this->params_.gravity;
  terra_input(7) = engine_output;
  terra_input(8) = delta;
  terra_input(9) = param_yaw_rate;
  TERRA_LSTM::output_array terra_output = TERRA_LSTM::output_array::Zero();
  terra_lstm_lstm_helper_->forward(terra_input, terra_output);

  const float c_delta = cosf(delta);
  const float s_delta = sinf(delta);
  const float drag_x = terra_output(0) * 10.0f;
  const float drag_y = terra_output(1) * 10.0f;
  const float drag_yaw = terra_output(2) * 10.0f;

  // combine to compute state derivative
  state_der(S_INDEX(VEL_X)) = c_delta * engine_output + engine_output - drag_x * c_delta + drag_y * s_delta - drag_x;
  state_der(S_INDEX(VEL_Y)) = s_delta * engine_output - drag_x * s_delta - drag_y * c_delta - drag_y;
  state_der(S_INDEX(YAW)) = param_yaw_rate - drag_yaw;
}

void BicycleSlipEngine::updateState(const Eigen::Ref<const state_array> state, Eigen::Ref<state_array> next_state,
                                    Eigen::Ref<state_array> state_der, const float dt)
{
  next_state = state + state_der * dt;
  next_state(S_INDEX(YAW)) = angle_utils::normalizeAngle(next_state(S_INDEX(YAW)));
  next_state(S_INDEX(STEER_ANGLE)) =
      max(min(next_state(S_INDEX(STEER_ANGLE)), this->params_.max_steer_angle), -this->params_.max_steer_angle);
  next_state(S_INDEX(STEER_ANGLE_RATE)) = state_der(S_INDEX(STEER_ANGLE));
  next_state(S_INDEX(OMEGA_Z)) = state_der(S_INDEX(YAW));
  next_state(S_INDEX(BRAKE_STATE)) =
      min(max(next_state(S_INDEX(BRAKE_STATE)), 0.0f), -this->control_rngs_[C_INDEX(THROTTLE_BRAKE)].x);
}

void BicycleSlipEngine::step(Eigen::Ref<state_array> state, Eigen::Ref<state_array> next_state,
                             Eigen::Ref<state_array> state_der, const Eigen::Ref<const control_array>& control,
                             Eigen::Ref<output_array> output, const float t, const float dt)
{
  computeDynamics(state, control, state_der);
  updateState(state, next_state, state_der, dt);

  float roll = state(S_INDEX(ROLL));
  float pitch = state(S_INDEX(PITCH));
  RACER::computeStaticSettling<DYN_PARAMS_T::OutputIndex, TwoDTextureHelper<float>>(
      this->tex_helper_, next_state(S_INDEX(YAW)), next_state(S_INDEX(POS_X)), next_state(S_INDEX(POS_Y)), roll, pitch,
      output.data());
  next_state[S_INDEX(PITCH)] = pitch;
  next_state[S_INDEX(ROLL)] = roll;

  output = output_array::Zero();

  output[O_INDEX(BASELINK_VEL_B_X)] = next_state[S_INDEX(VEL_X)];
  output[O_INDEX(BASELINK_VEL_B_Y)] = next_state[S_INDEX(VEL_Y)];
  output[O_INDEX(BASELINK_VEL_B_Z)] = 0;
  output[O_INDEX(BASELINK_POS_I_X)] = next_state[S_INDEX(POS_X)];
  output[O_INDEX(BASELINK_POS_I_Y)] = next_state[S_INDEX(POS_Y)];
  output[O_INDEX(YAW)] = next_state[S_INDEX(YAW)];
  output[O_INDEX(PITCH)] = next_state[S_INDEX(PITCH)];
  output[O_INDEX(ROLL)] = next_state[S_INDEX(ROLL)];
  output[O_INDEX(STEER_ANGLE)] = next_state[S_INDEX(STEER_ANGLE)];
  output[O_INDEX(STEER_ANGLE_RATE)] = next_state[S_INDEX(STEER_ANGLE_RATE)];
  output[O_INDEX(WHEEL_FORCE_B_FL)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_FR)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RL)] = 10000;
  output[O_INDEX(WHEEL_FORCE_B_RR)] = 10000;
  output[O_INDEX(ACCEL_X)] = state_der[S_INDEX(VEL_X)];
  output[O_INDEX(ACCEL_Y)] = state_der[S_INDEX(VEL_Y)];
  output[O_INDEX(OMEGA_Z)] = state_der[S_INDEX(YAW)];
}

__device__ void BicycleSlipEngine::initializeDynamics(float* state, float* control, float* output, float* theta_s,
                                                      float t_0, float dt)
{
  const int shift = PARENT_CLASS::SHARED_MEM_REQUEST_GRD_BYTES / 4 + 1;
  if (PARENT_CLASS::SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {  // Allows us to turn on or off global or shared memory version of params
    DYN_PARAMS_T* dyn_params = (DYN_PARAMS_T*)theta_s;
    *dyn_params = this->params_;
  }
  SHARED_MEM_GRD_PARAMS* shared_params = (SHARED_MEM_GRD_PARAMS*)(theta_s + shift);

  // setup memory for hidden/cell state memory
  //
  // if we are using shared memory load in the parameters
  if (SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {
    SHARED_MEM_BLK_PARAMS* blk_params = (SHARED_MEM_BLK_PARAMS*)(shared_params + 1);
    blk_params += blockDim.x * threadIdx.z + threadIdx.x;
    steer_network_d_->initialize(&shared_params->steer_lstm_params, &shared_params->steer_output_params,
                                 &blk_params->steer_hidden_cell[0]);
    delay_network_d_->initialize(&shared_params->delay_lstm_params, &shared_params->delay_output_params,
                                 &blk_params->delay_hidden_cell[0]);
    terra_network_d_->initialize(&shared_params->terra_lstm_params, &shared_params->terra_output_params,
                                 &blk_params->terra_hidden_cell[0]);
    engine_network_d_->initialize(&shared_params->engine_lstm_params, &shared_params->engine_output_params,
                                  &blk_params->engine_hidden_cell[0]);
  }
  else
  {
    SHARED_MEM_BLK_PARAMS* blk_params = (SHARED_MEM_BLK_PARAMS*)(shared_params);
    blk_params += blockDim.x * threadIdx.z + threadIdx.x;
    // only setup the hidden/cell states
    steer_network_d_->initialize(nullptr, nullptr, &blk_params->steer_hidden_cell[0]);
    delay_network_d_->initialize(nullptr, nullptr, &blk_params->delay_hidden_cell[0]);
    terra_network_d_->initialize(nullptr, nullptr, &blk_params->terra_hidden_cell[0]);
    engine_network_d_->initialize(nullptr, nullptr, &blk_params->engine_hidden_cell[0]);
    __syncthreads();
  }
  for (int i = 0; i < OUTPUT_DIM && i < STATE_DIM; i++)
  {
    output[i] = state[i];
  }
}

__device__ void BicycleSlipEngine::updateState(float* state, float* next_state, float* state_der, const float dt,
                                               DYN_PARAMS_T* params_p)
{
  for (int i = threadIdx.y; i < 7; i += blockDim.y)
  {
    next_state[i] = state[i] + state_der[i] * dt;
    switch (i)
    {
      case S_INDEX(YAW):
        next_state[i] = angle_utils::normalizeAngle(next_state[i]);
        break;
      case S_INDEX(OMEGA_Z):
        next_state[i] = state_der[S_INDEX(YAW)];
      case S_INDEX(STEER_ANGLE):
        next_state[S_INDEX(STEER_ANGLE)] =
            max(min(next_state[S_INDEX(STEER_ANGLE)], params_p->max_steer_angle), -params_p->max_steer_angle);
        next_state[S_INDEX(STEER_ANGLE_RATE)] = state_der[S_INDEX(STEER_ANGLE)];
        break;
      case S_INDEX(BRAKE_STATE):
        next_state[S_INDEX(BRAKE_STATE)] =
            min(max(next_state[S_INDEX(BRAKE_STATE)], 0.0f), -this->control_rngs_[C_INDEX(THROTTLE_BRAKE)].x);
    }
  }

  __syncthreads();
}

__device__ void BicycleSlipEngine::computeDynamics(float* state, float* control, float* state_der, float* theta)
{
  DYN_PARAMS_T* params_p = nullptr;

  const int shift = PARENT_CLASS::SHARED_MEM_REQUEST_GRD_BYTES / 4 + 1;
  if (PARENT_CLASS::SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {  // Allows us to turn on or off global or shared memory version of params
    params_p = (DYN_PARAMS_T*)theta;
  }
  else
  {
    params_p = &(this->params_);
  }

  // nullptr if not shared memory
  SHARED_MEM_GRD_PARAMS* params = (SHARED_MEM_GRD_PARAMS*)(theta + shift);
  SHARED_MEM_BLK_PARAMS* blk_params = (SHARED_MEM_BLK_PARAMS*)params;
  if (SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {
    // if GRD in shared them
    blk_params = (SHARED_MEM_BLK_PARAMS*)(params + 1);
  }
  blk_params = blk_params + blockDim.x * threadIdx.z + threadIdx.x;
  float* theta_s_shifted = &blk_params->theta_s[0];

  bool enable_brake = control[C_INDEX(THROTTLE_BRAKE)] < 0;
  const float brake_cmd = -enable_brake * control[C_INDEX(THROTTLE_BRAKE)];
  const float throttle_cmd = !enable_brake * control[C_INDEX(THROTTLE_BRAKE)];
  const float delta = tanf(state[S_INDEX(STEER_ANGLE)] / params_p->wheel_angle_scale);
  const float param_yaw_rate =
      (state[S_INDEX(VEL_X)] / params_p->wheel_base) * tan(state[S_INDEX(STEER_ANGLE)] / params_p->steer_angle_scale);

  // parametric part of the brake
  state_der[S_INDEX(BRAKE_STATE)] = min(
      max((brake_cmd - state[S_INDEX(BRAKE_STATE)]) * params_p->brake_delay_constant, -params_p->max_brake_rate_neg),
      params_p->max_brake_rate_pos);

  // kinematics component
  state_der[S_INDEX(POS_X)] =
      state[S_INDEX(VEL_X)] * cosf(state[S_INDEX(YAW)]) - state[S_INDEX(VEL_Y)] * sinf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(POS_Y)] =
      state[S_INDEX(VEL_X)] * sinf(state[S_INDEX(YAW)]) + state[S_INDEX(VEL_Y)] * cosf(state[S_INDEX(YAW)]);
  state_der[S_INDEX(OMEGA_Z)] = 0.0f;

  // runs the parametric part of the steering model
  state_der[S_INDEX(STEER_ANGLE)] =
      max(min((control[C_INDEX(STEER_CMD)] * params_p->steer_command_angle_scale - state[S_INDEX(STEER_ANGLE)]) *
                  params_p->steering_constant,
              params_p->max_steer_rate),
          -params_p->max_steer_rate);

  // runs the brake model
  float* input_loc = &theta_s_shifted[DELAY_LSTM::HIDDEN_DIM];
  float* output = nullptr;
  input_loc[0] = state[S_INDEX(BRAKE_STATE)];
  input_loc[1] = brake_cmd;
  input_loc[2] = state_der[S_INDEX(BRAKE_STATE)];  // stand in for y velocity

  if (SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {
    output = delay_network_d_->forward(nullptr, theta_s_shifted, &blk_params->delay_hidden_cell[0],
                                       &params->delay_lstm_params, &params->delay_output_params, 0);
  }
  else
  {
    output =
        delay_network_d_->forward(nullptr, theta_s_shifted, &blk_params->delay_hidden_cell[0],
                                  &delay_network_d_->params_, delay_network_d_->getOutputModel()->getParamsPtr(), 0);
  }
  if (threadIdx.y == 0)
  {
    state_der[S_INDEX(BRAKE_STATE)] += output[0];
  }

  // runs the engine model
  input_loc = &theta_s_shifted[ENGINE_LSTM::HIDDEN_DIM];
  input_loc[0] = throttle_cmd;
  input_loc[1] = state[S_INDEX(VEL_X)];
  input_loc[2] = state[S_INDEX(BRAKE_STATE)];
  if (SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {
    output = engine_network_d_->forward(nullptr, theta_s_shifted, &blk_params->engine_hidden_cell[0],
                                        &params->engine_lstm_params, &params->engine_output_params, 0);
  }
  else
  {
    output =
        engine_network_d_->forward(nullptr, theta_s_shifted, &blk_params->engine_hidden_cell[0],
                                   &engine_network_d_->params_, engine_network_d_->getOutputModel()->getParamsPtr(), 0);
  }
  const float engine_output = output[0] * 10.0f;

  // runs the steering model
  __syncthreads();  // required since we can overwrite the output before grabbing it
  input_loc = &theta_s_shifted[STEER_LSTM::HIDDEN_DIM];
  input_loc[0] = state[S_INDEX(VEL_X)];
  input_loc[1] = state[S_INDEX(STEER_ANGLE)];
  input_loc[2] = state[S_INDEX(STEER_ANGLE_RATE)];
  input_loc[3] = control[C_INDEX(STEER_CMD)];
  input_loc[4] = state_der[S_INDEX(STEER_ANGLE)];  // this is the parametric part as input
  if (SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {
    output = steer_network_d_->forward(nullptr, theta_s_shifted, &blk_params->steer_hidden_cell[0],
                                       &params->steer_lstm_params, &params->steer_output_params, 0);
  }
  else
  {
    output =
        steer_network_d_->forward(nullptr, theta_s_shifted, &blk_params->steer_hidden_cell[0],
                                  &steer_network_d_->params_, steer_network_d_->getOutputModel()->getParamsPtr(), 0);
  }
  if (threadIdx.y == 0)
  {
    state_der[S_INDEX(STEER_ANGLE)] += output[0] * 10.0f;
  }
  __syncthreads();  // required since we can overwrite the output before grabbing it

  // runs the terra dynamics model
  input_loc = &theta_s_shifted[TERRA_LSTM::HIDDEN_DIM];
  input_loc[0] = state[S_INDEX(VEL_X)];
  input_loc[1] = state[S_INDEX(VEL_Y)];
  input_loc[2] = state[S_INDEX(OMEGA_Z)];
  input_loc[3] = state[S_INDEX(STEER_ANGLE)];
  input_loc[4] = state[S_INDEX(STEER_ANGLE_RATE)];
  input_loc[5] = sinf(state[S_INDEX(PITCH)]) * params_p->gravity;
  input_loc[6] = sinf(state[S_INDEX(ROLL)]) * params_p->gravity;
  input_loc[7] = engine_output;
  input_loc[8] = delta;
  input_loc[9] = param_yaw_rate;
  if (SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {
    output = terra_network_d_->forward(nullptr, theta_s_shifted, &blk_params->terra_hidden_cell[0],
                                       &params->terra_lstm_params, &params->terra_output_params, 0);
  }
  else
  {
    output =
        terra_network_d_->forward(nullptr, theta_s_shifted, &blk_params->terra_hidden_cell[0],
                                  &terra_network_d_->params_, terra_network_d_->getOutputModel()->getParamsPtr(), 0);
  }

  const float c_delta = cosf(delta);
  const float s_delta = sinf(delta);
  const float drag_x = output[0] * 10.0f;
  const float drag_y = output[1] * 10.0f;
  const float drag_yaw = output[2] * 10.0f;

  // combine to compute state derivative
  state_der[S_INDEX(VEL_X)] = c_delta * engine_output + engine_output - drag_x * c_delta + drag_y * s_delta - drag_x;
  state_der[S_INDEX(VEL_Y)] = s_delta * engine_output - drag_x * s_delta - drag_y * c_delta - drag_y;
  state_der[S_INDEX(YAW)] = param_yaw_rate - drag_yaw;
}

__device__ void BicycleSlipEngine::step(float* state, float* next_state, float* state_der, float* control,
                                        float* output, float* theta_s, const float t, const float dt)
{
  DYN_PARAMS_T* params_p;
  if (PARENT_CLASS::SHARED_MEM_REQUEST_GRD_BYTES != 0)
  {  // Allows us to turn on or off global or shared memory version of params
    params_p = (DYN_PARAMS_T*)theta_s;
  }
  else
  {
    params_p = &(this->params_);
  }
  const uint tdy = threadIdx.y;

  computeDynamics(state, control, state_der, theta_s);
  updateState(state, next_state, state_der, dt, params_p);

  if (tdy == 0)
  {
    float roll = state[S_INDEX(ROLL)];
    float pitch = state[S_INDEX(PITCH)];
    RACER::computeStaticSettling<DYN_PARAMS_T::OutputIndex, TwoDTextureHelper<float>>(
        this->tex_helper_, next_state[S_INDEX(YAW)], next_state[S_INDEX(POS_X)], next_state[S_INDEX(POS_Y)], roll,
        pitch, output);
    next_state[S_INDEX(PITCH)] = pitch;
    next_state[S_INDEX(ROLL)] = roll;

    output[O_INDEX(BASELINK_VEL_B_X)] = next_state[S_INDEX(VEL_X)];
    output[O_INDEX(BASELINK_VEL_B_Y)] = next_state[S_INDEX(VEL_Y)];
    output[O_INDEX(BASELINK_POS_I_X)] = next_state[S_INDEX(POS_X)];
    output[O_INDEX(BASELINK_POS_I_Y)] = next_state[S_INDEX(POS_Y)];
    output[O_INDEX(YAW)] = next_state[S_INDEX(YAW)];
    output[O_INDEX(PITCH)] = next_state[S_INDEX(PITCH)];
    output[O_INDEX(ROLL)] = next_state[S_INDEX(ROLL)];
    output[O_INDEX(STEER_ANGLE)] = next_state[S_INDEX(STEER_ANGLE)];
    output[O_INDEX(STEER_ANGLE_RATE)] = next_state[S_INDEX(STEER_ANGLE_RATE)];
    output[O_INDEX(WHEEL_FORCE_B_FL)] = 10000;
    output[O_INDEX(WHEEL_FORCE_B_FR)] = 10000;
    output[O_INDEX(WHEEL_FORCE_B_RL)] = 10000;
    output[O_INDEX(WHEEL_FORCE_B_RR)] = 10000;
    output[O_INDEX(ACCEL_X)] = state_der[S_INDEX(VEL_X)];
    output[O_INDEX(ACCEL_Y)] = state_der[S_INDEX(VEL_Y)];
    output[O_INDEX(OMEGA_Z)] = state_der[S_INDEX(YAW)];
    next_state[S_INDEX(OMEGA_Z)] = state_der[S_INDEX(YAW)];
  }
}

void BicycleSlipEngine::getStoppingControl(const Eigen::Ref<const state_array>& state, Eigen::Ref<control_array> u)
{
  u[0] = -1.0;  // full brake
  u[1] = 0.0;   // no steering
}

Eigen::Quaternionf BicycleSlipEngine::attitudeFromState(const Eigen::Ref<const state_array>& state)
{
  Eigen::Quaternionf q;
  mppi::math::Euler2QuatNWU(state(S_INDEX(ROLL)), state(S_INDEX(PITCH)), state(S_INDEX(YAW)), q);
  return q;
}

Eigen::Vector3f BicycleSlipEngine::positionFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[S_INDEX(POS_X)], state[S_INDEX(POS_Y)], 0);
}

Eigen::Vector3f BicycleSlipEngine::velocityFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(state[S_INDEX(VEL_X)], state(S_INDEX(VEL_Y)), 0);
}

Eigen::Vector3f BicycleSlipEngine::angularRateFromState(const Eigen::Ref<const state_array>& state)
{
  return Eigen::Vector3f(0, 0, state[S_INDEX(OMEGA_Z)]);
}
