#include "hip/hip_runtime.h"
//
// Created by jason on 12/12/22.
//

#include "bicycle_slip_hybrid.cuh"

BicycleSlipHybrid::BicycleSlipHybrid(hipStream_t stream) : BicycleSlipKinematicImpl<BicycleSlipHybrid>(stream)
{
}

BicycleSlipHybrid::BicycleSlipHybrid(std::string model_path, hipStream_t stream)
  : BicycleSlipHybrid::BicycleSlipHybrid(stream)
{
  if (!fileExists(model_path))
  {
    std::cerr << "Could not load neural net model at model_path: " << model_path.c_str();
    exit(-1);
  }
  cnpy::npz_t param_dict = cnpy::npz_load(model_path);
  // this->params_.wheel_angle_scale = param_dict.at("bicycle_model/params/wheel_angle_scale").data<float>()[0];

  // load the delay params
  this->params_.brake_delay_constant = param_dict.at("delay_model/params/constant").data<float>()[0];
  this->params_.max_brake_rate_neg = param_dict.at("delay_model/params/max_rate_neg").data<float>()[0];
  this->params_.max_brake_rate_pos = param_dict.at("delay_model/params/max_rate_pos").data<float>()[0];

  // load the steering parameters
  this->params_.max_steer_rate = param_dict.at("steer_model/params/max_rate_pos").data<float>()[0];
  this->params_.steering_constant = param_dict.at("steer_model/params/constant").data<float>()[0];

  delay_lstm_lstm_helper_->loadParams("delay_model/model", model_path);
  terra_lstm_lstm_helper_->loadParams("bicycle_model/terra_model", model_path);
  steer_lstm_lstm_helper_->loadParams("steer_model/model", model_path);
}

void BicycleSlipHybrid::computeDynamics(const Eigen::Ref<const state_array>& state,
                                        const Eigen::Ref<const control_array>& control,
                                        Eigen::Ref<state_array> state_der)
{
  // state_der = state_array::Zero();
  // bool enable_brake = control[C_INDEX(THROTTLE_BRAKE)] < 0;
  // float brake_cmd = -enable_brake * control(C_INDEX(THROTTLE_BRAKE));
  // float throttle_cmd = !enable_brake * control(C_INDEX(THROTTLE_BRAKE));

  // state_der(S_INDEX(BRAKE_STATE)) =
  //     min(max((brake_cmd - state(S_INDEX(BRAKE_STATE))) * this->params_.brake_delay_constant,
  //             -this->params_.max_brake_rate_neg),
  //         this->params_.max_brake_rate_pos);
  // // TODO if low speed allow infinite brake, not sure if needed
  // // TODO need parametric reverse

  // // kinematics component
  // state_der(S_INDEX(POS_X)) =
  //     state(S_INDEX(VEL_X)) * cosf(state(S_INDEX(YAW))) - state(S_INDEX(VEL_Y)) * sinf(state(S_INDEX(YAW)));
  // state_der(S_INDEX(POS_Y)) =
  //     state(S_INDEX(VEL_X)) * sinf(state(S_INDEX(YAW))) + state(S_INDEX(VEL_Y)) * cosf(state(S_INDEX(YAW)));
  // state_der(S_INDEX(YAW)) = state(S_INDEX(OMEGA_Z));

  // // runs the brake model
  // DELAY_LSTM::input_array brake_input;
  // brake_input(0) = state(S_INDEX(BRAKE_STATE));
  // brake_input(1) = brake_cmd;
  // brake_input(2) = state_der(S_INDEX(BRAKE_STATE));  // stand in for y velocity
  // DELAY_LSTM::output_array brake_output = DELAY_LSTM::output_array::Zero();
  // delay_lstm_lstm_helper_->forward(brake_input, brake_output);
  // state_der(S_INDEX(BRAKE_STATE)) += brake_output(0);

  // // runs the parametric part of the steering model
  // state_der(S_INDEX(STEER_ANGLE)) =
  //     (control(C_INDEX(STEER_CMD)) * this->params_.steer_command_angle_scale - state(S_INDEX(STEER_ANGLE))) *
  //     this->params_.steering_constant;
  // state_der(S_INDEX(STEER_ANGLE)) =
  //     max(min(state_der(S_INDEX(STEER_ANGLE)), this->params_.max_steer_rate), -this->params_.max_steer_rate);

  // // runs the steering model
  // STEER_LSTM::input_array steer_input;
  // steer_input(0) = state(S_INDEX(VEL_X)) / 20.0f;
  // steer_input(1) = state(S_INDEX(STEER_ANGLE)) / 5.0f;
  // steer_input(2) = state(S_INDEX(STEER_ANGLE_RATE)) / 5.0f;
  // steer_input(3) = control(C_INDEX(STEER_CMD));
  // steer_input(4) = state_der(S_INDEX(STEER_ANGLE));  // this is the parametric part as input
  // STEER_LSTM::output_array steer_output = STEER_LSTM::output_array::Zero();
  // steer_lstm_lstm_helper_->forward(steer_input, steer_output);
  // state_der(S_INDEX(STEER_ANGLE)) += steer_output(0) * 10;

  // const float delta = tanf(state(S_INDEX(STEER_ANGLE)) / this->params_.wheel_angle_scale);

  // // runs the terra dynamics model
  // TERRA_LSTM::input_array terra_input;
  // terra_input(0) = state(S_INDEX(VEL_X)) / 20.0f;
  // terra_input(1) = state(S_INDEX(VEL_Y)) / 5.0f;
  // terra_input(2) = state(S_INDEX(OMEGA_Z)) / 5.0f;
  // terra_input(3) = throttle_cmd;
  // terra_input(4) = state(S_INDEX(BRAKE_STATE));
  // terra_input(5) = state(S_INDEX(STEER_ANGLE)) / 5.0f;
  // terra_input(6) = state(S_INDEX(STEER_ANGLE_RATE)) / 5.0f;
  // terra_input(7) = state(S_INDEX(PITCH));
  // terra_input(8) = state(S_INDEX(ROLL));
  // terra_input(9) = this->params_.environment;
  // TERRA_LSTM::output_array terra_output = TERRA_LSTM::output_array::Zero();
  // terra_lstm_lstm_helper_->forward(terra_input, terra_output);

  // const float c_delta = cosf(delta + terra_output(3));
  // const float s_delta = sinf(delta + terra_output(3));
  // const float x_accel = terra_output(0) * 10.0f;
  // const float y_accel = terra_output(1) * 5.0f;
  // const float yaw_accel = terra_output(2) * 5.0f;

  // // combine to compute state derivative
  // state_der(S_INDEX(VEL_X)) = x_accel * c_delta - y_accel * s_delta + x_accel;
  // state_der(S_INDEX(VEL_Y)) = x_accel * s_delta + y_accel * c_delta + y_accel;
  // state_der(S_INDEX(OMEGA_Z)) = yaw_accel;
}

__device__ void BicycleSlipHybrid::computeDynamics(float* state, float* control, float* state_der, float* theta)
{
  // DYN_PARAMS_T* params_p = nullptr;

  // const int shift = PARENT_CLASS::SHARED_MEM_REQUEST_GRD / 4 + 1;
  // if (PARENT_CLASS::SHARED_MEM_REQUEST_GRD != 1)
  // {  // Allows us to turn on or off global or shared memory version of params
  //   params_p = (DYN_PARAMS_T*)theta;
  // }
  // else
  // {
  //   params_p = &(this->params_);
  // }

  // // nullptr if not shared memory
  // SHARED_MEM_GRD_PARAMS* params = (SHARED_MEM_GRD_PARAMS*)(theta + shift);
  // SHARED_MEM_BLK_PARAMS* blk_params = (SHARED_MEM_BLK_PARAMS*)params;
  // if (SHARED_MEM_REQUEST_GRD != 0)
  // {
  //   // if GRD in shared them
  //   blk_params = (SHARED_MEM_BLK_PARAMS*)(params + 1);
  // }
  // blk_params = blk_params + blockDim.x * threadIdx.z + threadIdx.x;
  // float* theta_s_shifted = &blk_params->theta_s[0];

  // bool enable_brake = control[C_INDEX(THROTTLE_BRAKE)] < 0;
  // const float brake_cmd = -enable_brake * control[C_INDEX(THROTTLE_BRAKE)];
  // const float throttle_cmd = !enable_brake * control[C_INDEX(THROTTLE_BRAKE)];
  // const float delta = tanf(state[S_INDEX(STEER_ANGLE)] / params_p->wheel_angle_scale);

  // // parametric part of the brake
  // state_der[S_INDEX(BRAKE_STATE)] = min(
  //     max((brake_cmd - state[S_INDEX(BRAKE_STATE)]) * params_p->brake_delay_constant, -params_p->max_brake_rate_neg),
  //     params_p->max_brake_rate_pos);

  // // kinematics component
  // state_der[S_INDEX(POS_X)] =
  //     state[S_INDEX(VEL_X)] * cosf(state[S_INDEX(YAW)]) - state[S_INDEX(VEL_Y)] * sinf(state[S_INDEX(YAW)]);
  // state_der[S_INDEX(POS_Y)] =
  //     state[S_INDEX(VEL_X)] * sinf(state[S_INDEX(YAW)]) + state[S_INDEX(VEL_Y)] * cosf(state[S_INDEX(YAW)]);
  // state_der[S_INDEX(YAW)] = state[S_INDEX(OMEGA_Z)];

  // // runs the parametric part of the steering model
  // state_der[S_INDEX(STEER_ANGLE)] =
  //     max(min((control[C_INDEX(STEER_CMD)] * params_p->steer_command_angle_scale - state[S_INDEX(STEER_ANGLE)]) *
  //                 params_p->steering_constant,
  //             params_p->max_steer_rate),
  //         -params_p->max_steer_rate);

  // // runs the brake model
  // float* input_loc = &theta_s_shifted[DELAY_LSTM::HIDDEN_DIM];
  // float* output = nullptr;
  // input_loc[0] = state[S_INDEX(BRAKE_STATE)];
  // input_loc[1] = brake_cmd;
  // input_loc[2] = state_der[S_INDEX(BRAKE_STATE)];  // stand in for y velocity

  // if (SHARED_MEM_REQUEST_GRD != 0)
  // {
  //   output = delay_network_d_->forward(nullptr, theta_s_shifted, &blk_params->delay_hidden_cell[0],
  //                                      &params->delay_lstm_params, &params->delay_output_params, 0);
  // }
  // else
  // {
  //   output =
  //       delay_network_d_->forward(nullptr, theta_s_shifted, &blk_params->delay_hidden_cell[0],
  //                                 &delay_network_d_->params_, delay_network_d_->getOutputModel()->getParamsPtr(), 0);
  // }
  // if (threadIdx.y == 0)
  // {
  //   state_der[S_INDEX(BRAKE_STATE)] += output[0];
  // }

  // // runs the steering model
  // __syncthreads();  // required since we can overwrite the output before grabbing it
  // input_loc = &theta_s_shifted[STEER_LSTM::HIDDEN_DIM];
  // input_loc[0] = state[S_INDEX(VEL_X)] / 20.0f;
  // input_loc[1] = state[S_INDEX(STEER_ANGLE)] / 5.0f;
  // input_loc[2] = state[S_INDEX(STEER_ANGLE_RATE)] / 5.0f;
  // input_loc[3] = control[C_INDEX(STEER_CMD)];
  // input_loc[4] = state_der[S_INDEX(STEER_ANGLE)];  // this is the parametric part as input
  // if (SHARED_MEM_REQUEST_GRD != 0)
  // {
  //   output = steer_network_d_->forward(nullptr, theta_s_shifted, &blk_params->steer_hidden_cell[0],
  //                                      &params->steer_lstm_params, &params->steer_output_params, 0);
  // }
  // else
  // {
  //   output =
  //       steer_network_d_->forward(nullptr, theta_s_shifted, &blk_params->steer_hidden_cell[0],
  //                                 &steer_network_d_->params_, steer_network_d_->getOutputModel()->getParamsPtr(), 0);
  // }
  // if (threadIdx.y == 0)
  // {
  //   state_der[S_INDEX(STEER_ANGLE)] += output[0] * 10.0f;
  // }
  // __syncthreads();  // required since we can overwrite the output before grabbing it

  // // runs the terra dynamics model
  // input_loc = &theta_s_shifted[TERRA_LSTM::HIDDEN_DIM];
  // input_loc[0] = state[S_INDEX(VEL_X)] / 20.0f;
  // input_loc[1] = state[S_INDEX(VEL_Y)] / 5.0f;
  // input_loc[2] = state[S_INDEX(OMEGA_Z)] / 5.0f;
  // input_loc[3] = throttle_cmd;
  // input_loc[4] = state[S_INDEX(BRAKE_STATE)];
  // input_loc[5] = state[S_INDEX(STEER_ANGLE)] / 5.0f;
  // input_loc[6] = state[S_INDEX(STEER_ANGLE_RATE)] / 5.0f;
  // input_loc[7] = state[S_INDEX(PITCH)];
  // input_loc[8] = state[S_INDEX(ROLL)];
  // input_loc[9] = this->params_.environment;

  // if (SHARED_MEM_REQUEST_GRD != 0)
  // {
  //   output = terra_network_d_->forward(nullptr, theta_s_shifted, &blk_params->terra_hidden_cell[0],
  //                                      &params->terra_lstm_params, &params->terra_output_params, 0);
  // }
  // else
  // {
  //   output =
  //       terra_network_d_->forward(nullptr, theta_s_shifted, &blk_params->terra_hidden_cell[0],
  //                                 &terra_network_d_->params_, terra_network_d_->getOutputModel()->getParamsPtr(), 0);
  // }

  // const float c_delta = cosf(delta + output[3]);
  // const float s_delta = sinf(delta + output[3]);
  // const float x_accel = output[0] * 10.0f;
  // const float y_accel = output[1] * 5.0f;
  // const float yaw_accel = output[2] * 5.0f;

  // // combine to compute state derivative
  // state_der[S_INDEX(VEL_X)] = x_accel * c_delta - y_accel * s_delta + x_accel;
  // state_der[S_INDEX(VEL_Y)] = x_accel * s_delta + y_accel * c_delta + y_accel;
  // state_der[S_INDEX(OMEGA_Z)] = yaw_accel;
}
