#include "hip/hip_runtime.h"
template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::LSTMModel(std::array<float2, C_DIM> control_rngs, hipStream_t stream)
                  : Dynamics<LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>, LSTMDynamicsParams<S_DIM, C_DIM, K_DIM, layer_args...>, S_DIM, C_DIM>(control_rngs, stream) {
  CPUSetup();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::LSTMModel(hipStream_t stream)
                  : Dynamics<LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>, LSTMDynamicsParams<S_DIM, C_DIM, K_DIM, layer_args...>, S_DIM, C_DIM>(stream) {
  CPUSetup();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::~LSTMModel() {
  if(weights_ != nullptr) {
    delete[] weights_;
  }
  if(biases_ != nullptr) {
    delete[] biases_;
  }
  if(weighted_in_ != nullptr) {
    delete[] weighted_in_;
  }
  freeCudaMem();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::freeCudaMem() {
  Dynamics<LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>, LSTMDynamicsParams<S_DIM, C_DIM, K_DIM, layer_args...>, S_DIM, C_DIM>::freeCudaMem();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::CPUSetup() {
  // setup the CPU side values
  weights_ = new Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>[NUM_LAYERS-1];
  biases_ = new Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>[NUM_LAYERS-1];

  weighted_in_ = new Eigen::MatrixXf[NUM_LAYERS - 1];
  for(int i = 1; i < NUM_LAYERS; i++) {
    weighted_in_[i-1] = Eigen::MatrixXf::Zero(this->params_.net_structure[i], 1);
    weights_[i-1] = Eigen::MatrixXf::Zero(this->params_.net_structure[i], this->params_.net_structure[i-1]);
    biases_[i-1] = Eigen::MatrixXf::Zero(this->params_.net_structure[i], 1);
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::updateModel(std::vector<int> description,
        std::vector<float> data) {
  for(int i = 0; i < description.size(); i++) {
    if(description[i] != this->params_.net_structure[i]) {
      std::cerr << "Invalid model trying to to be set for NN" << std::endl;
      exit(0);
    }
  }
  for (int i = 0; i < NUM_LAYERS - 1; i++){
    for (int j = 0; j < this->params_.net_structure[i+1]; j++){
      for (int k = 0; k < this->params_.net_structure[i]; k++){
        weights_[i](j,k) = data[this->params_.stride_idcs[2*i] + j*this->params_.net_structure[i] + k];
        this->params_.theta[this->params_.stride_idcs[2*i] + j*this->params_.net_structure[i] + k] = data[this->params_.stride_idcs[2*i] + j*this->params_.net_structure[i] + k];
      }
    }
  }
  for (int i = 0; i < NUM_LAYERS - 1; i++){
    for (int j = 0; j < this->params_.net_structure[i+1]; j++){
      biases_[i](j,0) = data[this->params_.stride_idcs[2*i + 1] + j];
      this->params_.theta[this->params_.stride_idcs[2*i + 1] + j] = data[this->params_.stride_idcs[2*i + 1] + j];
    }
  }
  if(this->GPUMemStatus_) {
    paramsToDevice();
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::paramsToDevice() {
  // TODO copy to constant memory
  HANDLE_ERROR( hipMemcpyAsync(this->model_d_->control_rngs_,
                           this->control_rngs_,
                           NUM_PARAMS*sizeof(float), hipMemcpyHostToDevice,
                           this->stream_) );

  if (this->params_.copy_everything) {
    // Copy Weight Matrices
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_im,
                                  this->params_.W_im,
                                  this->params_.HIDDEN_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_fm,
                                  this->params_.W_fm,
                                  this->params_.HIDDEN_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_om,
                                  this->params_.W_om,
                                  this->params_.HIDDEN_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_cm,
                                  this->params_.W_cm,
                                  this->params_.HIDDEN_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_ii,
                                  this->params_.W_ii,
                                  this->params_.STATE_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_fi,
                                  this->params_.W_fi,
                                  this->params_.STATE_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_oi,
                                  this->params_.W_oi,
                                  this->params_.STATE_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_ci,
                                  this->params_.W_ci,
                                  this->params_.STATE_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.W_y,
                                  this->params_.W_y,
                                  this->params_.STATE_HIDDEN_SIZE*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    // Copy bias matrices
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.b_i,
                                  this->params_.b_i,
                                  this->params_.HIDDEN_DIM*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.b_f,
                                  this->params_.b_f,
                                  this->params_.HIDDEN_DIM*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.b_o,
                                  this->params_.b_o,
                                  this->params_.HIDDEN_DIM*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.b_c,
                                  this->params_.b_c,
                                  this->params_.HIDDEN_DIM*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.b_y,
                                  this->params_.b_y,
                                  this->params_.DYNAMICS_DIM*sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_) );
    this->params_.copy_everything = false;
  }
  if (this->params_.update_buffer) {
    this->params_.updateBuffer();
  }
  this->params_.updateInitialLSTMState();
  HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.initial_hidden,
                                this->params_.initial_hidden,
                                this->params_.HIDDEN_DIM*sizeof(float),
                                hipMemcpyHostToDevice, this->stream_) );
  HANDLE_ERROR( hipMemcpyAsync(this->model_d_->params_.initial_cell,
                                this->params_.initial_cell,
                                this->params_.HIDDEN_DIM*sizeof(float),
                                hipMemcpyHostToDevice, this->stream_) );
  HANDLE_ERROR(hipStreamSynchronize(this->stream_));
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::loadParams(const std::string& model_path) {
  int i,j,k;
  std::string bias_name = "";
  std::string weight_name = "";
  if (!fileExists(model_path)){
    std::cerr << "Could not load neural net model at path: " << model_path.c_str();
    exit(-1);
  }
  cnpy::npz_t param_dict = cnpy::npz_load(model_path);
  for (i = 0; i < NUM_LAYERS - 1; i++){
    // NN index from 1
    bias_name = "dynamics_b" + std::to_string(i + 1);
    weight_name = "dynamics_W" + std::to_string(i + 1);

    cnpy::NpyArray weight_i_raw = param_dict[weight_name];
    cnpy::NpyArray bias_i_raw = param_dict[bias_name];
    double* weight_i = weight_i_raw.data<double>();
    double* bias_i = bias_i_raw.data<double>();

    // copy over the weights
    for (j = 0; j < this->params_.net_structure[i + 1]; j++){
      for (k = 0; k < this->params_.net_structure[i]; k++){
        // TODO why i - 1?
        this->params_.theta[this->params_.stride_idcs[2*i] + j*this->params_.net_structure[i] + k] =
                (float)weight_i[j*this->params_.net_structure[i] + k];
        weights_[i](j,k) = (float)weight_i[j*this->params_.net_structure[i] + k];
      }
    }
    // copy over the bias
    for (j = 0; j < this->params_.net_structure[i+1]; j++){
      this->params_.theta[this->params_.stride_idcs[2*i + 1] + j] = (float)bias_i[j];
      biases_[i](j,0) = (float)bias_i[j];
    }
  }
  //Save parameters to GPU memory
  paramsToDevice();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
bool LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeGrad(const Eigen::Ref<const state_array>& state,
                                                                     const Eigen::Ref<const control_array>& control,
                                                                     Eigen::Ref<dfdx> A,
                                                                     Eigen::Ref<dfdu> B) {
  // TODO results are not returned
  Eigen::Matrix<float, S_DIM, S_DIM + C_DIM> jac;
  jac.setZero();

  //Start with the kinematic and physics model derivatives
  jac.row(0) << 0, 0, -sin(state(2))*state(4) - cos(state(2))*state(5), 0, cos(state(2)), -sin(state(2)), 0, 0, 0;
  jac.row(1) << 0, 0, cos(state(2))*state(4) - sin(state(2))*state(5), 0, sin(state(2)), cos(state(2)), 0, 0, 0;
  jac.row(2) << 0, 0, 0, 0, 0, 0, -1, 0, 0;

  state_array state_der;

  //First do the forward pass
  computeDynamics(state, control, state_der);

  //Start backprop
  Eigen::MatrixXf ip_delta = Eigen::MatrixXf::Identity(DYNAMICS_DIM, DYNAMICS_DIM);
  Eigen::MatrixXf temp_delta = Eigen::MatrixXf::Identity(DYNAMICS_DIM, DYNAMICS_DIM);

  //Main backprop loop
  for (int i = NUM_LAYERS-2; i > 0; i--){
    Eigen::MatrixXf zp = weighted_in_[i-1];
    for (int j = 0; j < this->params_.net_structure[i]; j++){
      zp(j) = MPPI_NNET_NONLINEARITY_DERIV(zp(j));
    }
    ip_delta =  ( (weights_[i]).transpose()*ip_delta).eval();
    for (int j = 0; j < DYNAMICS_DIM; j++){
      ip_delta.col(j) = ip_delta.col(j).array() * zp.array();
    }
  }
  //Finish the backprop loop
  ip_delta = ( ((weights_[0]).transpose())*ip_delta).eval();
  jac.bottomRightCorner(DYNAMICS_DIM, DYNAMICS_DIM + C_DIM) += ip_delta.transpose();
  A = jac.leftCols(S_DIM);
  B = jac.rightCols(C_DIM);
  return true;
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeKinematics(const Eigen::Ref<const state_array>& state,
        Eigen::Ref<state_array> state_der) {
  state_der(0) = cosf(state(2))*state(4) - sinf(state(2))*state(5);
  state_der(1) = sinf(state(2))*state(4) + cosf(state(2))*state(5);
  state_der(2) = -state(6); //Pose estimate actually gives the negative yaw derivative
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeDynamics(const Eigen::Ref<const state_array>& state,
        const Eigen::Ref<const control_array>& control, Eigen::Ref<state_array> state_der) {
  int i,j;
  Eigen::MatrixXf acts(this->params_.net_structure[0], 1);
  for (i = 0; i < DYNAMICS_DIM; i++){
    acts(i) = state(i + (S_DIM - DYNAMICS_DIM));
  }
  for (i = 0; i < C_DIM; i++){
    acts(DYNAMICS_DIM + i) = control(i);
  }
  for (i = 0; i < NUM_LAYERS - 1; i++){
    weighted_in_[i] = (weights_[i]*acts + biases_[i]).eval();
    acts = Eigen::MatrixXf::Zero(this->params_.net_structure[i+1], 1);
    if (i < NUM_LAYERS - 2) { //Last layer doesn't apply any non-linearity
      for (j = 0; j < this->params_.net_structure[i+1]; j++){
        acts(j) = MPPI_NNET_NONLINEARITY( (weighted_in_[i])(j) ); //Nonlinear component.
      }
    }
    else {
      for (j = 0; j < this->params_.net_structure[i+1]; j++){
        acts(j) = (weighted_in_[i])(j) ;
      }
    }
  }
  for (i = 0; i < DYNAMICS_DIM; i++){
    state_der(i + (S_DIM - DYNAMICS_DIM)) = acts(i);
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeKinematics(
        float* state, float* state_der) {
  state_der[0] = cosf(state[2])*state[4] - sinf(state[2])*state[5];
  state_der[1] = sinf(state[2])*state[4] + cosf(state[2])*state[5];
  state_der[2] = -state[6]; //Pose estimate actually gives the negative yaw derivative
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeDynamics(float* state, float* control, float* state_der, float* theta_s)
{
  float* curr_act;
  float* next_act;
  float* tmp_act;
  float tmp;
  float* W;
  float* b;
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  int tdz = threadIdx.z;
  int i,j,k;
  curr_act = &theta_s[(2*LARGEST_LAYER)*(blockDim.x*tdz + tdx)];
  next_act = &theta_s[(2*LARGEST_LAYER)*(blockDim.x*tdz + tdx) + LARGEST_LAYER];
  // iterate through the part of the state that should be an input to the NN
  for (i = tdy; i < DYNAMICS_DIM; i+= blockDim.y){
    curr_act[i] = state[i + (S_DIM - DYNAMICS_DIM)];
  }
  // iterate through the control to put into first layer
  for (i = tdy; i < C_DIM; i+= blockDim.y){
    curr_act[DYNAMICS_DIM + i] = control[i];
  }
  __syncthreads();
  // iterate through each layer
  for (i = 0; i < NUM_LAYERS - 1; i++){
    //Conditional compilation depending on if we're using a global constant memory array or not.
#if defined(MPPI_NNET_USING_CONSTANT_MEM__) //Use constant memory.
    W = &NNET_PARAMS[this->params_.stride_idcs[2*i]]; // weights
    b = &NNET_PARAMS[this->params_.stride_idcs[2*i + 1]]; // biases
#else //Use (slow) global memory.
    W = &this->params_.theta[this->params_.stride_idcs[2*i]]; // weights
    b = &this->params_.theta[this->params_.stride_idcs[2*i + 1]]; // biases
#endif
    // for first non input layer until last layer this thread deals with
    // calculates the next activation based on current
    for (j = tdy; j < this->params_.net_structure[i+1]; j += blockDim.y) {
      tmp = 0;
      // apply each neuron activation from current layer
      for (k = 0; k < this->params_.net_structure[i]; k++) {
        //No atomic add necessary.
        tmp += W[j*this->params_.net_structure[i] + k]*curr_act[k];
      }
      // add bias from next layer and neuron
      tmp += b[j];
      if (i < NUM_LAYERS - 2){
        tmp = MPPI_NNET_NONLINEARITY(tmp);
      }
      next_act[j] = tmp;
    }
    //Swap the two pointers
    tmp_act = curr_act;
    curr_act = next_act;
    next_act = tmp_act;
    __syncthreads();
  }
  // copies results back into state derivative
  for (i = tdy; i < DYNAMICS_DIM; i+= blockDim.y){
    state_der[i + (S_DIM - DYNAMICS_DIM)] = curr_act[i];
  }
  __syncthreads();
}

// x = v_k
// h = m_{k-1}
// c = c_{k-1}
template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void LSTMModel<S_DIM, C_DIM, K_DIM, layer_args...>::LSTMLayer(float* x, float* theta_s,
                                                                         float* c,
                                                                         float* h,
                                                                         float* next_output,
                                                                         float* next_cell_state,
                                                                         float* next_hidden_state) {
  // float* curr_act;
  // float* next_act;
  // float* tmp_act;
  float tmp;
  // Weights
  float* W_ii = &(this->params_.W_ii);
  float* W_im = &(this->params_.W_im);
  float* W_fi = &(this->params_.W_fi);
  float* W_fm = &(this->params_.W_fm);
  float* W_oi = &(this->params_.W_oi);
  float* W_om = &(this->params_.W_om);
  float* W_ci = &(this->params_.W_ci);
  float* W_cm = &(this->params_.W_cm);

  // float* W_im ; // hidden * state, hidden * hidden
  // float* W_fi, *W_fm; // hidden * state, hidden * hidden
  // float* W_oi, *W_om; // hidden * state, hidden * hidden
  // float* W_ci, *W_cm; // hidden * state, hidden * hidden
  // Biases
  float* b_i = &(this->params_.b_i); // hidden_size
  float* b_f = &(this->params_.b_f); // hidden_size
  float* b_o = &(this->params_.b_o); // hidden_size
  float* b_c = &(this->params_.b_c); // hidden_size
  // Intermediate outputs
  float* c = &theta_s[0];
  float* h = &theta_s[this->params_.HIDDEN_SIZE];
  float* next_cell_state = &theta_s[2 * this->params_.HIDDEN_SIZE];
  float* next_hidden_state = &theta_s[3 * this->params_.HIDDEN_SIZE];
  float* g_ki = &theta_s[4 * this->params_.HIDDEN_SIZE];
  float* g_kf = &theta_s[5 * this->params_.HIDDEN_SIZE];
  float* g_ko = &theta_s[6 * this->params_.HIDDEN_SIZE];
  float* cell_update = &theta_s[7 * this->params_.HIDDEN_SIZE];
  float* intermediate_y = &theta_s[8 * this->params_.HIDDEN_SIZE];
  // float* g_ki, *g_kf, *g_ko, *cell_update; // hidden_size

  float dt = 0.01;
  float* W_y,; // state * hidden
  float* b_y,; // state
  int i,j,k;
  int input_size, hidden_size;

  // float* intermediate_y;
  int index = 0;
  // input gate
  for (i = 0; i < hidden_size; i++) {
    g_ki[i] = 0;
    for (j = 0; j < input_size; j++) {
      g_ki[i] += W_ii[i * input_size + j] * x[j];
    }
    for (j = 0; j < hidden_size; j++) {
      index = i * hidden_size + j;
      g_ki[i] += W_im[index] * h[j];
    }
    g_ki[i] += b_i[i];
    g_ki[i] = SIGMOID(g_ki[i]);
  }
  // forget gate
  for (i = 0; i < hidden_size; i++) {
    g_kf[i] = 0;
    for (j = 0; j < input_size; j++) {
      g_kf[i] += W_fi[i * input_size + j] * x[j];
    }
    for (j = 0; j < hidden_size; j++) {
      index = i * hidden_size + j;
      g_kf[i] += W_fm[index] * h[j];
    }
    g_kf[i] += b_f[i];
    g_kf[i] = SIGMOID(g_kf[i]);
  }
  // output gate
  for (i = 0; i < hidden_size; i++) {
    g_ko[i] = 0;
    for (j = 0; j < input_size; j++) {
      g_ko[i] += W_oi[i * input_size + j] * x[j];
    }
    for (j = 0; j < hidden_size; j++) {
      index = i * hidden_size + j;
      g_ko[i] += W_om[index] * h[j];
    }
    g_ko[i] += b_o[i];
    g_ko[i] = SIGMOID(g_ko[i]);
  }
  // cell update
  for (i = 0; i < hidden_size; i++) {
    cell_update[i] = 0;
    for (j = 0; j < input_size; j++) {
      cell_update[i] += W_ci[i * input_size + j] * x[j];
    }
    for (j = 0; j < hidden_size; j++) {
      cell_update[i] += W_cm[i * hidden_size + j] * h[j];
    }
    cell_update[i] += b_c[i];
    cell_update[i] = RELU(cell_update[i]);
  }

  // outputs
  for (i = 0; i < hidden_size; i++) {
    next_cell_state[i] = g_ki[i] * cell_update[i] + g_kf[i] * c[j];
  }
  for (i = 0; i < hidden_size; i++) {
    next_hidden_state[i] = tanhf(next_cell_state[i]) * g_ko[i];
  }

  for (i = 0; i < input_size; i++) {
    intermediate_y = 0;
    for (j = 0; j < hidden_size; j++) {
      intermediate_y += W_y1[i * hidden_size + j] * next_hidden_state[j];
    }
    intermediate_y += b_y1[i];
    intermediate_y[i] = intermediate_y;
  }
}