#include "hip/hip_runtime.h"
#include <mppi/dynamics/double_integrator/di_dynamics.cuh>
#include <mppi/cost_functions/double_integrator/double_integrator_circle_cost.cuh>
#include <mppi/cost_functions/double_integrator/double_integrator_robust_cost.cuh>
#include <mppi/controllers/MPPI/mppi_controller.cuh>
#include <mppi/controllers/Tube-MPPI/tube_mppi_controller.cuh>
#include <mppi/controllers/R-MPPI/robust_mppi_controller.cuh>
#include <mppi/feedback_controllers/DDP/ddp.cuh>

#include <cnpy.h>
#include <random>  // Used to generate random noise for control trajectories

bool tubeFailure(float* s)
{
  float inner_path_radius2 = 1.675 * 1.675;
  float outer_path_radius2 = 2.325 * 2.325;
  float radial_position = s[0] * s[0] + s[1] * s[1];
  if ((radial_position < inner_path_radius2) || (radial_position > outer_path_radius2))
  {
    return true;
  }
  else
  {
    return false;
  }
}

using Dyn = DoubleIntegratorDynamics;
using SCost = DoubleIntegratorCircleCost;
using RCost = DoubleIntegratorRobustCost;
const int num_timesteps = 50;  // Optimization time horizon
const int total_time_horizon = 5000;
using Feedback = DDPFeedback<Dyn, num_timesteps>;
using Sampler = mppi::sampling_distributions::GaussianDistribution<Dyn::DYN_PARAMS_T>;

// Problem setup
const float dt = 0.02;   // Timestep of dynamics propagation
const int max_iter = 1;  // Maximum running iterations of optimization
const float lambda = 2;  // Learning rate parameter
const float alpha = 0.0;

typedef Eigen::Matrix<float, Dyn::STATE_DIM, num_timesteps> state_trajectory;

void saveTraj(const Eigen::Ref<const state_trajectory>& traj, int t, std::vector<float>& vec)
{
  for (int i = 0; i < num_timesteps; i++)
  {
    for (int j = 0; j < Dyn::STATE_DIM; j++)
    {
      vec[t * num_timesteps * Dyn::STATE_DIM + i * Dyn::STATE_DIM + j] = traj(j, i);
    }
  }
}

void saveState(const Eigen::Ref<const Dyn::state_array>& state, int t, std::vector<float>& vec)
{
  for (int j = 0; j < Dyn::STATE_DIM; j++)
  {
    vec[t * Dyn::STATE_DIM + j] = state(j);
  }
}

void runVanilla(const Eigen::Ref<const Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon>>& noise)
{
  // Set the initial state
  Dyn::state_array x;
  x << 2, 0, 0, 1;
  Dyn::state_array xdot;

  // control variance
  Sampler::SAMPLING_PARAMS_T sampler_params;
  for (int i = 0; i < Dyn::CONTROL_DIM; i++)
  {
    sampler_params.std_dev[i] = 1;
  }

  // Save actual trajectories, nominal_trajectory, free energy
  std::vector<float> van_trajectory(Dyn::STATE_DIM * total_time_horizon, 0);
  std::vector<float> van_nominal_traj(Dyn::STATE_DIM * num_timesteps * total_time_horizon, 0);
  std::vector<float> van_free_energy(total_time_horizon, 0);

  // Initialize the controllers
  Dyn model;
  SCost cost;
  Sampler sampler(sampler_params);
  // DDP cost parameters
  Feedback fb_controller(&model, dt);
  auto fb_params = fb_controller.getParams();
  fb_params.Q.diagonal() << 500, 500, 100, 100;
  fb_controller.setParams(fb_params);
  auto controller = VanillaMPPIController<Dyn, SCost, Feedback, num_timesteps, 1024, Sampler>(
      &model, &cost, &fb_controller, &sampler, dt, max_iter, lambda, alpha);
  auto controller_params = controller.getParams();
  controller_params.dynamics_rollout_dim_ = dim3(64, 1, 1);
  controller_params.cost_rollout_dim_ = dim3(64, 1, 1);
  controller.setParams(controller_params);
  controller.initFeedback();

  // Start the loop
  for (int t = 0; t < total_time_horizon; ++t)
  {
    /********************** Vanilla **********************/
    // Compute the control
    controller.computeControl(x, 1);

    // Compute the feedback gains
    controller.computeFeedback(x);

    // Propagate the feedback trajectory
    controller.computeFeedbackPropagatedStateSeq();

    auto nominal_trajectory = controller.getTargetStateSeq();
    auto nominal_control = controller.getControlSeq();
    auto fe_stat = controller.getFreeEnergyStatistics();

    // Save everything
    saveState(x, t, van_trajectory);
    saveTraj(nominal_trajectory, t, van_nominal_traj);
    van_free_energy[t] = fe_stat.real_sys.freeEnergyMean;

    // Get the open loop control
    DoubleIntegratorDynamics::control_array current_control = nominal_control.col(0);

    // Apply the feedback given the current state
    Dyn::control_array fb_control = controller.getFeedbackControl(x, nominal_trajectory.col(0), 0);
    current_control += fb_control;

    // Propagate the state forward
    model.computeDynamics(x, current_control, xdot);
    model.updateState(x, xdot, dt);

    // Add disturbance
    x += noise.col(t) * sqrt(model.getParams().system_noise) * dt;

    // Slide the control sequence
    controller.slideControlSequence(1);
  }
  /************* Save CNPY *********************/
  cnpy::npy_save("vanilla_state_trajectory.npy", van_trajectory.data(),
                 { total_time_horizon, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("vanilla_nominal_trajectory.npy", van_nominal_traj.data(),
                 { total_time_horizon, num_timesteps, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("vanilla_free_energy.npy", van_free_energy.data(), { total_time_horizon }, "w");
}

void runVanillaLarge(const Eigen::Ref<const Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon>>& noise)
{
  // Set the initial state
  DoubleIntegratorDynamics::state_array x;
  x << 2, 0, 0, 1;
  DoubleIntegratorDynamics::state_array xdot;

  // control variance
  Sampler::SAMPLING_PARAMS_T sampler_params;
  for (int i = 0; i < Dyn::CONTROL_DIM; i++)
  {
    sampler_params.std_dev[i] = 1;
  }

  // Save actual trajectories, nominal_trajectory, free energy
  std::vector<float> van_large_trajectory(Dyn::STATE_DIM * total_time_horizon, 0);
  std::vector<float> van_large_nominal_traj(Dyn::STATE_DIM * num_timesteps * total_time_horizon, 0);
  std::vector<float> van_large_free_energy(total_time_horizon, 0);

  // Initialize the controllers
  Dyn model(100);
  SCost cost;
  Sampler sampler(sampler_params);
  // DDP cost parameters
  Feedback fb_controller(&model, dt);
  auto fb_params = fb_controller.getParams();
  fb_params.Q.diagonal() << 500, 500, 100, 100;
  fb_controller.setParams(fb_params);
  auto controller = VanillaMPPIController<Dyn, SCost, Feedback, num_timesteps, 1024, Sampler>(
      &model, &cost, &fb_controller, &sampler, dt, max_iter, lambda, alpha);
  auto controller_params = controller.getParams();
  controller_params.dynamics_rollout_dim_ = dim3(64, 1, 1);
  controller_params.cost_rollout_dim_ = dim3(64, 1, 1);
  controller.setParams(controller_params);
  controller.initFeedback();

  // Start the loop
  for (int t = 0; t < total_time_horizon; ++t)
  {
    /********************** Vanilla Large **********************/
    // Compute the control
    controller.computeControl(x, 1);

    // Compute the feedback gains
    controller.computeFeedback(x);

    // Propagate the feedback trajectory
    controller.computeFeedbackPropagatedStateSeq();

    auto nominal_trajectory = controller.getTargetStateSeq();
    auto nominal_control = controller.getControlSeq();
    auto fe_stat = controller.getFreeEnergyStatistics();

    // Save everything
    saveState(x, t, van_large_trajectory);
    saveTraj(nominal_trajectory, t, van_large_nominal_traj);
    van_large_free_energy[t] = fe_stat.real_sys.freeEnergyMean;

    // Get the open loop control
    DoubleIntegratorDynamics::control_array current_control = nominal_control.col(0);

    // Apply the feedback given the current state
    Dyn::control_array fb_control = controller.getFeedbackControl(x, nominal_trajectory.col(0), 0);
    current_control += fb_control;

    // Propagate the state forward
    model.computeDynamics(x, current_control, xdot);
    model.updateState(x, xdot, dt);

    // Add disturbance
    x += noise.col(t) * sqrt(model.getParams().system_noise) * dt;

    // Slide the control sequence
    controller.slideControlSequence(1);
  }
  /************* Save CNPY *********************/
  cnpy::npy_save("vanilla_large_state_trajectory.npy", van_large_trajectory.data(),
                 { total_time_horizon, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("vanilla_large_nominal_trajectory.npy", van_large_nominal_traj.data(),
                 { total_time_horizon, num_timesteps, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("vanilla_large_free_energy.npy", van_large_free_energy.data(), { total_time_horizon }, "w");
}

void runVanillaLargeRC(const Eigen::Ref<const Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon>>& noise)
{
  // Set the initial state
  DoubleIntegratorDynamics::state_array x;
  x << 2, 0, 0, 1;
  DoubleIntegratorDynamics::state_array xdot;

  // control variance
  Sampler::SAMPLING_PARAMS_T sampler_params;
  for (int i = 0; i < Dyn::CONTROL_DIM; i++)
  {
    sampler_params.std_dev[i] = 1;
  }

  // Save actual trajectories, nominal_trajectory, free energy
  std::vector<float> van_large_trajectory(Dyn::STATE_DIM * total_time_horizon, 0);
  std::vector<float> van_large_nominal_traj(Dyn::STATE_DIM * num_timesteps * total_time_horizon, 0);
  std::vector<float> van_large_free_energy(total_time_horizon, 0);

  // Initialize the controllers
  Dyn model(100);

  RCost cost;
  Sampler sampler(sampler_params);
  auto params = cost.getParams();
  params.crash_cost = 100;
  cost.setParams(params);
  // DDP cost parameters
  Feedback fb_controller(&model, dt);
  auto fb_params = fb_controller.getParams();
  fb_params.Q.diagonal() << 500, 500, 100, 100;
  fb_controller.setParams(fb_params);

  auto controller = VanillaMPPIController<Dyn, RCost, Feedback, num_timesteps, 1024, Sampler>(
      &model, &cost, &fb_controller, &sampler, dt, max_iter, lambda, alpha);
  auto controller_params = controller.getParams();
  controller_params.dynamics_rollout_dim_ = dim3(64, 1, 1);
  controller_params.cost_rollout_dim_ = dim3(64, 1, 1);
  controller.setParams(controller_params);
  controller.initFeedback();

  // Start the loop
  for (int t = 0; t < total_time_horizon; ++t)
  {
    /********************** Vanilla Large **********************/
    // Compute the control
    controller.computeControl(x, 1);

    // Compute the feedback gains
    controller.computeFeedback(x);

    // Propagate the feedback trajectory
    controller.computeFeedbackPropagatedStateSeq();

    auto nominal_trajectory = controller.getTargetStateSeq();
    auto nominal_control = controller.getControlSeq();
    auto fe_stat = controller.getFreeEnergyStatistics();

    // Save everything
    saveState(x, t, van_large_trajectory);
    saveTraj(nominal_trajectory, t, van_large_nominal_traj);
    van_large_free_energy[t] = fe_stat.real_sys.freeEnergyMean;

    // Get the open loop control
    DoubleIntegratorDynamics::control_array current_control = nominal_control.col(0);

    // Apply the feedback given the current state
    Dyn::control_array fb_control = controller.getFeedbackControl(x, nominal_trajectory.col(0), 0);
    current_control += fb_control;

    // Propagate the state forward
    model.computeDynamics(x, current_control, xdot);
    model.updateState(x, xdot, dt);

    // Add disturbance
    x += noise.col(t) * sqrt(model.getParams().system_noise) * dt;

    // Slide the control sequence
    controller.slideControlSequence(1);
  }
  /************* Save CNPY *********************/
  cnpy::npy_save("vanilla_large_robust_state_trajectory.npy", van_large_trajectory.data(),
                 { total_time_horizon, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("vanilla_large_robust_nominal_trajectory.npy", van_large_nominal_traj.data(),
                 { total_time_horizon, num_timesteps, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("vanilla_large_robust_free_energy.npy", van_large_free_energy.data(), { total_time_horizon }, "w");
}

void runTube(const Eigen::Ref<const Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon>>& noise)
{
  // Set the initial state
  DoubleIntegratorDynamics::state_array x;
  x << 2, 0, 0, 1;
  DoubleIntegratorDynamics::state_array xdot;

  // control variance
  Sampler::SAMPLING_PARAMS_T sampler_params;
  for (int i = 0; i < Dyn::CONTROL_DIM; i++)
  {
    sampler_params.std_dev[i] = 1;
  }

  // Save actual trajectories, nominal_trajectory, free energy
  std::vector<float> tube_trajectory(Dyn::STATE_DIM * total_time_horizon, 0);
  std::vector<float> tube_nominal_traj(Dyn::STATE_DIM * num_timesteps * total_time_horizon, 0);
  std::vector<float> tube_nominal_free_energy(total_time_horizon, 0);
  std::vector<float> tube_real_free_energy(total_time_horizon, 0);
  std::vector<float> tube_nominal_state_used(total_time_horizon, 0);

  // Initialize the controllers
  Dyn model(100);
  SCost cost;
  Sampler sampler(sampler_params);
  // DDP cost parameters
  Feedback fb_controller(&model, dt);
  auto fb_params = fb_controller.getParams();
  fb_params.Q.diagonal() << 500, 500, 100, 100;
  fb_controller.setParams(fb_params);
  auto controller = TubeMPPIController<Dyn, SCost, Feedback, num_timesteps, 1024, Sampler>(
      &model, &cost, &fb_controller, &sampler, dt, max_iter, lambda, alpha);
  auto controller_params = controller.getParams();
  controller_params.dynamics_rollout_dim_ = dim3(64, 1, 1);
  controller_params.cost_rollout_dim_ = dim3(64, 1, 1);
  controller.setParams(controller_params);
  controller.setNominalThreshold(20);
  // Start the loop
  for (int t = 0; t < total_time_horizon; ++t)
  {
    /********************** Tube **********************/
    // Compute the control
    controller.computeControl(x, 1);

    // Compute the feedback gains
    controller.computeFeedback(x);

    // Propagate the feedback trajectory
    controller.computeFeedbackPropagatedStateSeq();

    auto nominal_trajectory = controller.getTargetStateSeq();
    auto nominal_control = controller.getControlSeq();
    auto fe_stat = controller.getFreeEnergyStatistics();

    // Save everything
    saveState(x, t, tube_trajectory);
    saveTraj(nominal_trajectory, t, tube_nominal_traj);
    tube_nominal_free_energy[t] = fe_stat.nominal_sys.freeEnergyMean;
    tube_real_free_energy[t] = fe_stat.real_sys.freeEnergyMean;
    tube_nominal_state_used[t] = fe_stat.nominal_state_used;

    // Get the open loop control
    DoubleIntegratorDynamics::control_array current_control = nominal_control.col(0);

    // Apply the feedback given the current state
    Dyn::control_array fb_control = controller.getFeedbackControl(x, nominal_trajectory.col(0), 0);
    current_control += fb_control;

    // Propagate the state forward
    model.computeDynamics(x, current_control, xdot);
    model.updateState(x, xdot, dt);
    controller.updateNominalState(current_control);

    // Add disturbance
    x += noise.col(t) * sqrt(model.getParams().system_noise) * dt;

    // Slide the control sequence
    controller.slideControlSequence(1);
  }
  /************* Save CNPY *********************/
  cnpy::npy_save("tube_state_trajectory.npy", tube_trajectory.data(),
                 { total_time_horizon, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("tube_nominal_trajectory.npy", tube_nominal_traj.data(),
                 { total_time_horizon, num_timesteps, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("tube_nominal_free_energy.npy", tube_nominal_free_energy.data(), { total_time_horizon }, "w");
  cnpy::npy_save("tube_real_free_energy.npy", tube_real_free_energy.data(), { total_time_horizon }, "w");
  cnpy::npy_save("tube_nominal_state_used.npy", tube_nominal_state_used.data(), { total_time_horizon }, "w");
}

void runTubeRC(const Eigen::Ref<const Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon>>& noise)
{
  // Set the initial state
  DoubleIntegratorDynamics::state_array x;
  x << 2, 0, 0, 1;
  DoubleIntegratorDynamics::state_array xdot;

  // control variance
  Sampler::SAMPLING_PARAMS_T sampler_params;
  for (int i = 0; i < Dyn::CONTROL_DIM; i++)
  {
    sampler_params.std_dev[i] = 1;
  }

  // Save actual trajectories, nominal_trajectory, free energy
  std::vector<float> tube_trajectory(Dyn::STATE_DIM * total_time_horizon, 0);
  std::vector<float> tube_nominal_traj(Dyn::STATE_DIM * num_timesteps * total_time_horizon, 0);
  std::vector<float> tube_nominal_free_energy(total_time_horizon, 0);
  std::vector<float> tube_real_free_energy(total_time_horizon, 0);
  std::vector<float> tube_nominal_state_used(total_time_horizon, 0);

  // Initialize the controllers
  Dyn model(100);
  RCost cost;
  Sampler sampler(sampler_params);
  auto params = cost.getParams();
  params.crash_cost = 100;
  cost.setParams(params);
  // DDP cost parameters
  Feedback fb_controller(&model, dt);
  auto fb_params = fb_controller.getParams();
  fb_params.Q.diagonal() << 500, 500, 100, 100;
  fb_controller.setParams(fb_params);
  auto controller = TubeMPPIController<Dyn, RCost, Feedback, num_timesteps, 1024>(
      &model, &cost, &fb_controller, &sampler, dt, max_iter, lambda, alpha);
  auto controller_params = controller.getParams();
  controller_params.dynamics_rollout_dim_ = dim3(64, 1, 1);
  controller_params.cost_rollout_dim_ = dim3(64, 1, 1);
  controller.setParams(controller_params);
  controller.setNominalThreshold(2);
  // Start the loop
  for (int t = 0; t < total_time_horizon; ++t)
  {
    /********************** Tube **********************/
    // Compute the control
    controller.computeControl(x, 1);

    // Compute the feedback gains
    controller.computeFeedback(x);

    // Propagate the feedback trajectory
    controller.computeFeedbackPropagatedStateSeq();

    auto nominal_trajectory = controller.getTargetStateSeq();
    auto nominal_control = controller.getControlSeq();
    auto fe_stat = controller.getFreeEnergyStatistics();

    // Save everything
    saveState(x, t, tube_trajectory);
    saveTraj(nominal_trajectory, t, tube_nominal_traj);
    tube_nominal_free_energy[t] = fe_stat.nominal_sys.freeEnergyMean;
    tube_real_free_energy[t] = fe_stat.real_sys.freeEnergyMean;
    tube_nominal_state_used[t] = fe_stat.nominal_state_used;

    // Get the open loop control
    DoubleIntegratorDynamics::control_array current_control = nominal_control.col(0);

    // Apply the feedback given the current state
    Dyn::control_array fb_control = controller.getFeedbackControl(x, nominal_trajectory.col(0), 0);
    current_control += fb_control;

    // Propagate the state forward
    model.computeDynamics(x, current_control, xdot);
    model.updateState(x, xdot, dt);
    controller.updateNominalState(current_control);

    // Add disturbance
    x += noise.col(t) * sqrt(model.getParams().system_noise) * dt;

    // Slide the control sequence
    controller.slideControlSequence(1);
  }
  /************* Save CNPY *********************/
  cnpy::npy_save("tube_robust_state_trajectory.npy", tube_trajectory.data(),
                 { total_time_horizon, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("tube_robust_nominal_trajectory.npy", tube_nominal_traj.data(),
                 { total_time_horizon, num_timesteps, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("tube_robust_nominal_free_energy.npy", tube_nominal_free_energy.data(), { total_time_horizon }, "w");
  cnpy::npy_save("tube_robust_real_free_energy.npy", tube_real_free_energy.data(), { total_time_horizon }, "w");
  cnpy::npy_save("tube_robust_nominal_state_used.npy", tube_nominal_state_used.data(), { total_time_horizon }, "w");
}

void runRobustSc(const Eigen::Ref<const Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon>>& noise)
{
  // Set the initial state
  DoubleIntegratorDynamics::state_array x;
  x << 2, 0, 0, 1;
  DoubleIntegratorDynamics::state_array xdot;

  // control variance
  Sampler::SAMPLING_PARAMS_T sampler_params;
  for (int i = 0; i < Dyn::CONTROL_DIM; i++)
  {
    sampler_params.std_dev[i] = 1;
  }

  // Save actual trajectories, nominal_trajectory, free energy
  std::vector<float> robust_sc_trajectory(Dyn::STATE_DIM * total_time_horizon, 0);
  std::vector<float> robust_sc_nominal_traj(Dyn::STATE_DIM * num_timesteps * total_time_horizon, 0);
  std::vector<float> robust_sc_nominal_free_energy(total_time_horizon, 0);
  std::vector<float> robust_sc_real_free_energy(total_time_horizon, 0);
  std::vector<float> robust_sc_nominal_free_energy_bound(total_time_horizon, 0);
  std::vector<float> robust_sc_real_free_energy_bound(total_time_horizon, 0);
  std::vector<float> robust_sc_real_free_energy_growth_bound(total_time_horizon, 0);
  std::vector<float> robust_sc_nominal_state_used(total_time_horizon, 0);

  // Initialize the controllers
  Dyn model(100);
  SCost cost;
  Sampler sampler(sampler_params);
  // DDP cost parameters
  Feedback fb_controller(&model, dt);
  auto fb_params = fb_controller.getParams();
  fb_params.Q.diagonal() << 500, 500, 100, 100;
  fb_controller.setParams(fb_params);
  // Value function threshold
  float value_function_threshold = 20.0;
  auto controller = RobustMPPIController<Dyn, SCost, Feedback, num_timesteps, 1024, Sampler>(
      &model, &cost, &fb_controller, &sampler, dt, max_iter, lambda, alpha, value_function_threshold);
  auto controller_params = controller.getParams();
  controller_params.dynamics_rollout_dim_ = dim3(64, 1, 1);
  controller_params.cost_rollout_dim_ = dim3(64, 1, 1);
  controller.setParams(controller_params);

  // Start the loop
  for (int t = 0; t < total_time_horizon; ++t)
  {
    /********************** Vanilla **********************/
    // Compute the control
    controller.updateImportanceSamplingControl(x, 1);
    controller.computeControl(x, 1);

    // Compute the feedback gains
    controller.computeFeedback(x);

    // Propagate the feedback trajectory
    controller.computeFeedbackPropagatedStateSeq();

    auto nominal_trajectory = controller.getTargetStateSeq();
    auto nominal_control = controller.getControlSeq();
    auto fe_stat = controller.getFreeEnergyStatistics();

    // Save everything
    saveState(x, t, robust_sc_trajectory);
    saveTraj(nominal_trajectory, t, robust_sc_nominal_traj);
    robust_sc_nominal_free_energy[t] = fe_stat.nominal_sys.freeEnergyMean;
    robust_sc_real_free_energy[t] = fe_stat.real_sys.freeEnergyMean;
    robust_sc_nominal_free_energy_bound[t] =
        value_function_threshold + 2 * fe_stat.nominal_sys.freeEnergyModifiedVariance;
    robust_sc_real_free_energy_bound[t] = 0;
    robust_sc_real_free_energy_growth_bound[t] = 0;
    robust_sc_nominal_state_used[t] = fe_stat.nominal_state_used;

    // Get the open loop control
    DoubleIntegratorDynamics::control_array current_control = nominal_control.col(0);

    // Apply the feedback given the current state
    Dyn::control_array fb_control = controller.getFeedbackControl(x, nominal_trajectory.col(0), 0);
    current_control += fb_control;

    // Propagate the state forward
    model.computeDynamics(x, current_control, xdot);
    model.updateState(x, xdot, dt);

    // Add disturbance
    x += noise.col(t) * sqrt(model.getParams().system_noise) * dt;

    // Slide the control sequence
    controller.slideControlSequence(1);
  }
  /************* Save CNPY *********************/
  cnpy::npy_save("robust_sc_state_trajectory.npy", robust_sc_trajectory.data(),
                 { total_time_horizon, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("robust_sc_nominal_trajectory.npy", robust_sc_nominal_traj.data(),
                 { total_time_horizon, num_timesteps, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("robust_sc_nominal_free_energy.npy", robust_sc_nominal_free_energy.data(), { total_time_horizon },
                 "w");
  cnpy::npy_save("robust_sc_real_free_energy.npy", robust_sc_real_free_energy.data(), { total_time_horizon }, "w");
  cnpy::npy_save("robust_sc_nominal_state_used.npy", robust_sc_nominal_state_used.data(), { total_time_horizon }, "w");
  cnpy::npy_save("robust_sc_real_free_energy_bound.npy", robust_sc_nominal_free_energy_bound.data(),
                 { total_time_horizon }, "w");
  cnpy::npy_save("robust_sc_nominal_free_energy_bound.npy", robust_sc_real_free_energy_bound.data(),
                 { total_time_horizon }, "w");
  cnpy::npy_save("robust_sc_real_free_energy_growth_bound.npy", robust_sc_real_free_energy_growth_bound.data(),
                 { total_time_horizon }, "w");
}

void runRobustRc(const Eigen::Ref<const Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon>>& noise)
{
  // Set the initial state
  DoubleIntegratorDynamics::state_array x;
  x << 2, 0, 0, 1;
  DoubleIntegratorDynamics::state_array xdot;

  // control variance
  Sampler::SAMPLING_PARAMS_T sampler_params;
  for (int i = 0; i < Dyn::CONTROL_DIM; i++)
  {
    sampler_params.std_dev[i] = 1;
  }

  // Save actual trajectories, nominal_trajectory, free energy
  std::vector<float> robust_rc_trajectory(Dyn::STATE_DIM * total_time_horizon, 0);
  std::vector<float> robust_rc_nominal_traj(Dyn::STATE_DIM * num_timesteps * total_time_horizon, 0);
  std::vector<float> robust_rc_nominal_free_energy(total_time_horizon, 0);
  std::vector<float> robust_rc_real_free_energy(total_time_horizon, 0);
  std::vector<float> robust_rc_nominal_free_energy_bound(total_time_horizon, 0);
  std::vector<float> robust_rc_real_free_energy_bound(total_time_horizon, 0);
  std::vector<float> robust_rc_real_free_energy_growth_bound(total_time_horizon, 0);
  std::vector<float> robust_rc_nominal_free_energy_growth(total_time_horizon, 0);
  std::vector<float> robust_rc_real_free_energy_growth(total_time_horizon, 0);
  std::vector<float> robust_rc_nominal_state_used(total_time_horizon, 0);

  // Initialize the controllers
  Dyn model(100);
  RCost cost;
  auto params = cost.getParams();
  params.crash_cost = 100;
  cost.setParams(params);
  Sampler sampler(sampler_params);

  // DDP cost parameters
  Feedback fb_controller(&model, dt);
  auto fb_params = fb_controller.getParams();
  fb_params.Q.diagonal() << 500, 500, 100, 100;
  fb_controller.setParams(fb_params);

  // Value function threshold
  float value_function_threshold = 20.0;
  auto controller = RobustMPPIController<Dyn, RCost, Feedback, num_timesteps, 1024, Sampler>(
      &model, &cost, &fb_controller, &sampler, dt, max_iter, lambda, alpha, value_function_threshold);
  auto controller_params = controller.getParams();
  controller_params.dynamics_rollout_dim_ = dim3(64, 1, 1);
  controller_params.cost_rollout_dim_ = dim3(64, 1, 1);
  controller.setParams(controller_params);

  // Start the loop
  for (int t = 0; t < total_time_horizon; ++t)
  {
    /********************** Robust Robust Cost **********************/
    // Compute the control
    controller.updateImportanceSamplingControl(x, 1);
    controller.computeControl(x, 1);

    // Compute the feedback gains
    controller.computeFeedback(x);

    // Propagate the feedback trajectory
    controller.computeFeedbackPropagatedStateSeq();

    auto nominal_trajectory = controller.getTargetStateSeq();
    auto nominal_control = controller.getControlSeq();
    auto fe_stat = controller.getFreeEnergyStatistics();

    // Save everything
    saveState(x, t, robust_rc_trajectory);
    saveTraj(nominal_trajectory, t, robust_rc_nominal_traj);
    robust_rc_nominal_free_energy[t] = fe_stat.nominal_sys.freeEnergyMean;
    robust_rc_real_free_energy[t] = fe_stat.real_sys.freeEnergyMean;
    robust_rc_nominal_free_energy_bound[t] =
        value_function_threshold + 2 * fe_stat.nominal_sys.freeEnergyModifiedVariance;
    robust_rc_real_free_energy_bound[t] = fe_stat.nominal_sys.freeEnergyMean +
                                          cost.getLipshitzConstantCost() * 1 * (x - nominal_trajectory.col(0)).norm();
    robust_rc_real_free_energy_growth_bound[t] = (value_function_threshold - fe_stat.nominal_sys.freeEnergyMean) +
                                                 cost.getLipshitzConstantCost() * 8 * 20 * controller.computeDF() +
                                                 2 * fe_stat.nominal_sys.freeEnergyModifiedVariance;
    robust_rc_nominal_free_energy_growth[t] = fe_stat.nominal_sys.increase;
    robust_rc_real_free_energy_growth[t] = fe_stat.real_sys.increase;
    robust_rc_nominal_state_used[t] = fe_stat.nominal_state_used;

    // Get the open loop control
    DoubleIntegratorDynamics::control_array current_control = nominal_control.col(0);

    // Apply the feedback given the current state
    Dyn::control_array fb_control = controller.getFeedbackControl(x, nominal_trajectory.col(0), 0);
    current_control += fb_control;

    // Propagate the state forward
    model.computeDynamics(x, current_control, xdot);
    model.updateState(x, xdot, dt);

    // Add disturbance
    x += noise.col(t) * sqrt(model.getParams().system_noise) * dt;

    // Slide the control sequence
    controller.slideControlSequence(1);
  }
  /************* Save CNPY *********************/
  cnpy::npy_save("robust_rc_state_trajectory.npy", robust_rc_trajectory.data(),
                 { total_time_horizon, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("robust_rc_nominal_trajectory.npy", robust_rc_nominal_traj.data(),
                 { total_time_horizon, num_timesteps, DoubleIntegratorDynamics::STATE_DIM }, "w");
  cnpy::npy_save("robust_rc_nominal_free_energy.npy", robust_rc_nominal_free_energy.data(), { total_time_horizon },
                 "w");
  cnpy::npy_save("robust_rc_real_free_energy.npy", robust_rc_real_free_energy.data(), { total_time_horizon }, "w");
  cnpy::npy_save("robust_rc_nominal_state_used.npy", robust_rc_nominal_state_used.data(), { total_time_horizon }, "w");
  cnpy::npy_save("robust_rc_real_free_energy_bound.npy", robust_rc_real_free_energy_bound.data(),
                 { total_time_horizon }, "w");
  cnpy::npy_save("robust_rc_nominal_free_energy_bound.npy", robust_rc_nominal_free_energy_bound.data(),
                 { total_time_horizon }, "w");
  cnpy::npy_save("robust_rc_real_free_energy_growth_bound.npy", robust_rc_real_free_energy_growth_bound.data(),
                 { total_time_horizon }, "w");
  cnpy::npy_save("robust_rc_real_free_energy_growth.npy", robust_rc_real_free_energy_growth.data(),
                 { total_time_horizon }, "w");
  cnpy::npy_save("robust_rc_nominal_free_energy_growth.npy", robust_rc_nominal_free_energy_growth.data(),
                 { total_time_horizon }, "w");
}

int main()
{
  // Run the double integrator example on all the controllers with the SAME noise 20 times.

  // Create a random number generator
  // Random number generator for system noise
  std::mt19937 gen;  // Standard mersenne_twister_engine which will be seeded
  std::normal_distribution<float> normal_distribution;
  gen.seed(7);  // Seed the 7, so everyone gets the same noise
  normal_distribution = std::normal_distribution<float>(0, 1);

  Eigen::Matrix<float, Dyn::STATE_DIM, total_time_horizon> universal_noise;
  universal_noise.setZero();

  // Create the noise for all systems
  for (int t = 0; t < total_time_horizon; ++t)
  {
    for (int i = 2; i < 4; ++i)
    {
      universal_noise(i, t) = normal_distribution(gen);
    }
  }

  runVanilla(universal_noise);
  std::cout << "Finished Vanilla" << std::endl;

  runVanillaLarge(universal_noise);
  std::cout << "Finished Vanilla Large" << std::endl;

  runVanillaLargeRC(universal_noise);
  std::cout << "Finished Vanilla Large with Robust Cost" << std::endl;

  runTube(universal_noise);
  std::cout << "Finished Tube with Standard Cost" << std::endl;

  runTubeRC(universal_noise);
  std::cout << "Finished Tube with Robust Cost" << std::endl;

  runRobustSc(universal_noise);
  std::cout << "Finished RMPPI with Standard Cost" << std::endl;

  runRobustRc(universal_noise);
  std::cout << "Finished RMPPI with Robust Cost" << std::endl;

  return 0;
}
