#include "hip/hip_runtime.h"
//
// Created by Bogdan on 12/26/21
//

#include <gtest/gtest.h>
#include <mppi/sampling_distributions/colored_noise/colored_noise.cuh>

#include <numeric>

void assert_float_rel_near(const float known, const float compute, float rel_err)
{
  float err = fabsf(compute - known) / known;
  ASSERT_NEAR(known, compute, rel_err) << "Relative error is " << err;
}

TEST(cuFFT, checkErrorCode)
{
  hipfftHandle plan;
  hipComplex* input_d;
  float* output_d;
  // As this call is intended to cause issues, disable compiler warning
  // src: https://stackoverflow.com/questions/14831051/how-to-disable-a-specific-nvcc-compiler-warnings
  // https://stackoverflow.com/questions/56193080/how-do-i-apply-a-flag-setting-nvcc-pragma-to-only-a-few-lines-of-code
  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#nv-diagnostic-pragmas
#pragma push
#pragma diag_suppress = used_before_set
  auto status = hipfftExecC2R(plan, input_d, output_d);
#pragma pop
  // cufftAssert(status, __FILE__, __LINE__);
  std::string error_string = cufftGetErrorString(status);
  // std::cout << error_string << std::endl;
  EXPECT_TRUE(error_string == "cuFFT was passed an invalid plan handle");
}

TEST(ColoredNoise, checkWhiteNoise)
{
  int NUM_TIMESTEPS = 50000;
  int NUM_ROLLOUTS = 1;
  int CONTROL_DIM = 1;
  std::vector<float> exponents(CONTROL_DIM, 0.0);
  int full_buffer_size = NUM_ROLLOUTS * NUM_TIMESTEPS * CONTROL_DIM;
  float* colored_noise_d;
  float colored_noise_output[full_buffer_size] = { 0 };
  HANDLE_ERROR(hipMalloc((void**)&colored_noise_d, sizeof(float) * full_buffer_size));
  hipStream_t stream;
  hiprandGenerator_t gen;
  hipStreamCreate(&stream);
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 42);
  hiprandSetStream(gen, stream);

  powerlaw_psd_gaussian(exponents, NUM_TIMESTEPS, NUM_ROLLOUTS, colored_noise_d, gen, stream);
  HANDLE_ERROR(hipMemcpyAsync(colored_noise_output, colored_noise_d, sizeof(float) * full_buffer_size,
                               hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));

  // Check percentages for 3 standard deviations
  std::vector<int> num_within_std_dev(3, 0);
  for (int i = 0; i < full_buffer_size; i++)
  {
    for (int j = 0; j < num_within_std_dev.size(); j++)
    {
      if (fabsf(colored_noise_output[i]) < j + 1.0)
      {
        num_within_std_dev[j]++;
        break;
      }
    }
  }

  float perc_within_n_std_dev[num_within_std_dev.size()];
  // Percentages from https://en.wikipedia.org/wiki/68%E2%80%9395%E2%80%9399.7_rule
  float known_percentages[3] = { 0.6827, 0.9545, 0.9973 };
  for (int i = 0; i < num_within_std_dev.size(); i++)
  {
    perc_within_n_std_dev[i] =
        std::accumulate(num_within_std_dev.begin(), num_within_std_dev.begin() + i + 1, 0.0) / full_buffer_size;
    assert_float_rel_near(known_percentages[i], perc_within_n_std_dev[i], 0.001);
  }
}

TEST(ColoredNoise, checkPinkNoise)
{
  int NUM_TIMESTEPS = 50000;
  int NUM_ROLLOUTS = 1;
  int CONTROL_DIM = 1;
  std::vector<float> exponents(CONTROL_DIM, 1.0);
  int full_buffer_size = NUM_ROLLOUTS * NUM_TIMESTEPS * CONTROL_DIM;
  float* colored_noise_d;
  float colored_noise_output[full_buffer_size] = { 0 };
  HANDLE_ERROR(hipMalloc((void**)&colored_noise_d, sizeof(float) * full_buffer_size));
  hipStream_t stream;
  hiprandGenerator_t gen;
  hipStreamCreate(&stream);
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 42);
  hiprandSetStream(gen, stream);

  powerlaw_psd_gaussian(exponents, NUM_TIMESTEPS, NUM_ROLLOUTS, colored_noise_d, gen, stream);
  HANDLE_ERROR(hipMemcpyAsync(colored_noise_output, colored_noise_d, sizeof(float) * full_buffer_size,
                               hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));
  int within_std_dev = 0;
  int within_2_std_dev = 0;
  for (int i = 0; i < full_buffer_size; i++)
  {
    if (fabsf(colored_noise_output[i]) < 1.0)
    {
      within_std_dev++;
    }
    else if (fabsf(colored_noise_output[i]) < 2.0)
    {
      within_2_std_dev++;
    }
  }
  float perc_one_std_dev = (float)within_std_dev / full_buffer_size;
  float perc_two_std_dev = (float)(within_std_dev + within_2_std_dev) / full_buffer_size;
  std::cout << "Percentage within 1 std dev: " << 100 * perc_one_std_dev << std::endl;
  std::cout << "Percentage within 2 std dev: " << 100 * perc_two_std_dev << std::endl;
  // assert_float_rel_near(0.6827, perc_one_std_dev, 0.001);
  // assert_float_rel_near(0.9545, perc_two_std_dev, 0.001);
}

TEST(ColoredNoise, checkRedNoise)
{
  int NUM_TIMESTEPS = 50000;
  int NUM_ROLLOUTS = 1;
  int CONTROL_DIM = 1;
  std::vector<float> exponents(CONTROL_DIM, 2.0);
  int full_buffer_size = NUM_ROLLOUTS * NUM_TIMESTEPS * CONTROL_DIM;
  float* colored_noise_d;
  float colored_noise_output[full_buffer_size] = { 0 };
  HANDLE_ERROR(hipMalloc((void**)&colored_noise_d, sizeof(float) * full_buffer_size));
  hipStream_t stream;
  hiprandGenerator_t gen;
  hipStreamCreate(&stream);
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 42);
  hiprandSetStream(gen, stream);

  powerlaw_psd_gaussian(exponents, NUM_TIMESTEPS, NUM_ROLLOUTS, colored_noise_d, gen, stream);
  HANDLE_ERROR(hipMemcpyAsync(colored_noise_output, colored_noise_d, sizeof(float) * full_buffer_size,
                               hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));

  // Check percentages for 3 standard deviations
  std::vector<int> num_within_std_dev(3, 0);
  for (int i = 0; i < full_buffer_size; i++)
  {
    for (int j = 0; j < num_within_std_dev.size(); j++)
    {
      if (fabsf(colored_noise_output[i]) < j + 1.0)
      {
        num_within_std_dev[j]++;
        break;
      }
    }
  }

  float perc_within_n_std_dev[num_within_std_dev.size()];
  // Percentages from https://en.wikipedia.org/wiki/68%E2%80%9395%E2%80%9399.7_rule
  float known_percentages[3] = { 0.6827, 0.9545, 0.9973 };
  for (int i = 0; i < num_within_std_dev.size(); i++)
  {
    perc_within_n_std_dev[i] =
        std::accumulate(num_within_std_dev.begin(), num_within_std_dev.begin() + i + 1, 0.0) / full_buffer_size;
    std::cout << "Percentage within " << i + 1 << " std dev: " << 100 * perc_within_n_std_dev[i] << std::endl;
    // assert_float_rel_near(known_percentages[i], perc_within_n_std_dev[i], 0.001);
  }
}

TEST(ColoredNoise, checkMultiNoise)
{
  int NUM_TIMESTEPS = 6000;
  int NUM_ROLLOUTS = 50;
  int CONTROL_DIM = 3;
  std::vector<float> exponents(CONTROL_DIM, 0.0);
  exponents[1] = 0.5;
  exponents[2] = 2.0;
  // exponents[3] = 1.25;
  // exponents[4] = 0.75;
  int full_buffer_size = NUM_ROLLOUTS * NUM_TIMESTEPS * CONTROL_DIM;
  float* colored_noise_d;
  float colored_noise_output[full_buffer_size] = { 0 };
  HANDLE_ERROR(hipMalloc((void**)&colored_noise_d, sizeof(float) * full_buffer_size));
  hipStream_t stream;
  hiprandGenerator_t gen;
  hipStreamCreate(&stream);
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 42);
  hiprandSetStream(gen, stream);

  powerlaw_psd_gaussian(exponents, NUM_TIMESTEPS, NUM_ROLLOUTS, colored_noise_d, gen, stream);
  HANDLE_ERROR(hipMemcpyAsync(colored_noise_output, colored_noise_d, sizeof(float) * full_buffer_size,
                               hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));

  // Check percentages for 3 standard deviations
  std::vector<int> num_within_std_dev(3, 0);
  float perc_within_n_std_dev[num_within_std_dev.size()];
  // Percentages from https://en.wikipedia.org/wiki/68%E2%80%9395%E2%80%9399.7_rule
  float known_percentages[3] = { 0.6827, 0.9545, 0.9973 };
  for (int c = 0; c < CONTROL_DIM; c++)
  {
    std::fill(num_within_std_dev.begin(), num_within_std_dev.end(), 0);
    for (int i = 0; i < NUM_TIMESTEPS * NUM_ROLLOUTS; i++)
    {
      for (int j = 0; j < num_within_std_dev.size(); j++)
      {
        if (fabsf(colored_noise_output[i * CONTROL_DIM + c]) < j + 1.0)
        {
          num_within_std_dev[j]++;
          break;
        }
      }
    }

    for (int i = 0; i < num_within_std_dev.size(); i++)
    {
      perc_within_n_std_dev[i] = std::accumulate(num_within_std_dev.begin(), num_within_std_dev.begin() + i + 1, 0.0) /
                                 (NUM_ROLLOUTS * NUM_TIMESTEPS);
      std::cout << "Colored Noise " << exponents[c] << " ";
      std::cout << "percent of samples within " << i + 1 << " std dev: " << 100 * perc_within_n_std_dev[i] << std::endl;
      if (exponents[c] == 0)
      {
        assert_float_rel_near(known_percentages[i], perc_within_n_std_dev[i], 0.001);
      }
    }
  }
}
