#include "hip/hip_runtime.h"
//
// Created by mgandhi3 on 3/4/20.
//
#include <gtest/gtest.h>
#include <mppi/dynamics/double_integrator/di_dynamics.cuh>
#include "di_dynamics_kernel_tests.cuh"
#include <memory>
#include <random>
#include <Eigen/Dense>

TEST(Miscellaneous, CompareModelSize) {
  auto model = std::make_shared<DoubleIntegratorDynamics>();

  model->GPUSetup();

  long* model_size_GPU_d;

  HANDLE_ERROR(hipMalloc((void**)&model_size_GPU_d, sizeof(long)));

  CheckModelSize<<<1,1>>>(model->model_d_, model_size_GPU_d);
  CudaCheckError();

  long model_size_GPU;
  HANDLE_ERROR(hipMemcpy(&model_size_GPU, model_size_GPU_d, sizeof(long), hipMemcpyDeviceToHost));

  ASSERT_EQ(sizeof(*model), model_size_GPU);

  std::cout << "Size of the shared pointer to the model:" << sizeof(model) << std::endl; // Should be the size of a pointer so 8 bytes for a 64 bit system

  std::cout << "Size of the model itself: " << sizeof(*model) << std::endl; // Should be bigger?

  std::cout << "Size of the model on the GPU: " << model_size_GPU << std::endl;

  std::cout << "Size of the control ranges in the model: " << sizeof(model->control_rngs_) << std::endl; // Should be 16 bytes ie. 4 floats!

  std::cout << "Size of the parameter structure of the model: " << sizeof(DoubleIntegratorParams) << std::endl;

  std::cout << "Size of the device pointer of the model: " << sizeof(model->model_d_) << std::endl;

  std::cout << "Size of the stream: " << sizeof(model->stream_) << std::endl;

  std::cout << "Size of GPU Memstatus: " << sizeof(model->GPUMemStatus_) << std::endl;

}

TEST(Miscellaneous, EigenNormalRandomVector) {
  std::random_device rd;
  std::mt19937 gen(rd());  //here you could also set a seed
  std::normal_distribution<float> dis(1, 2);

  //generate a matrix expression
  Eigen::MatrixXd M = Eigen::MatrixXd::NullaryExpr(100, 100, [&]() { return dis(gen); });

  EXPECT_NEAR(M.mean(), 1, 1e-1);

  EXPECT_NEAR(sqrtf((M.array()*M.array()).mean() - M.mean()*M.mean()), 2, 1e-1);
}

TEST(Miscellaneous, CreateRandomStateArray) {
  DoubleIntegratorDynamics::state_array X;
  DoubleIntegratorDynamics::state_array temp = DoubleIntegratorDynamics::state_array::Zero();

  std::random_device rd;
  std::mt19937 gen(rd());  //here you could also set a seed
  std::normal_distribution<float> dis(1, 2);

  //generate a matrix expression
  X = DoubleIntegratorDynamics::state_array::NullaryExpr([&]() { return dis(gen); });

  std::cout << temp + X*0.01 << std::endl;
}