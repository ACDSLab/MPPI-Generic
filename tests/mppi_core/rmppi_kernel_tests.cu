#include "hip/hip_runtime.h"
//
// Created by mgandhi on 5/23/20.
//
#include <gtest/gtest.h>
#include <mppi/dynamics/double_integrator/di_dynamics.cuh>
#include <mppi/cost_functions/double_integrator/double_integrator_circle_cost.cuh>
#include <mppi/core/rmppi_kernel_test.cuh>
#include <mppi/controllers/MPPI/mppi_controller.cuh>
#include <mppi/utils/test_helper.h>

class RMPPIKernels : public ::testing::Test {
public:
  using dynamics = DoubleIntegratorDynamics;
  using cost_function = DoubleIntegratorCircleCost;

  void SetUp() override {
    model = new dynamics(10);  // Initialize the double integrator dynamics
    cost = new cost_function;  // Initialize the cost function
  }

  void TearDown() override {
    delete model;
    delete cost;
  }

  dynamics* model;
  cost_function* cost;
};

TEST_F(RMPPIKernels, InitEvalRollout) {
  // Given the initial states, we need to roll out the number of samples.
  // 1.)  Generate the noise used to evaluate each sample.
  //

  const int num_candidates = 9;

  float dt = 0.01;

  Eigen::Matrix<float, 4, num_candidates> x0_candidates;
  x0_candidates << -4 , -3, -2, -1, 0, 1, 2, 3, 4, 0, 1, 2, 3, 4, 4, 4, 4, 4,
  0,  0,  0,  0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;

  // For each candidate, we want to estimate the free energy using a set number of samples.
  const int num_samples = 64;

  // We are going to propagate a trajectory for a given number of timesteps
  const int num_timesteps = 5;

  // Call the GPU setup functions of the model and cost
  model->GPUSetup();
  cost->GPUSetup();

  // Allocate and deallocate the CUDA memory
  int* strides_d;
  float* exploration_var_d;
  float* states_d;
  float* control_d;
  float* control_noise_d;
  float* costs_d;
  HANDLE_ERROR(hipMalloc((void**)&strides_d, sizeof(int)*num_candidates));
  HANDLE_ERROR(hipMalloc((void**)&exploration_var_d, sizeof(float)*dynamics::CONTROL_DIM));
  HANDLE_ERROR(hipMalloc((void**)&states_d, sizeof(float)*dynamics::STATE_DIM*num_candidates));
  HANDLE_ERROR(hipMalloc((void**)&control_d, sizeof(float)*dynamics::CONTROL_DIM*num_timesteps));
  HANDLE_ERROR(hipMalloc((void**)&control_noise_d, sizeof(float)*dynamics::CONTROL_DIM*num_candidates*num_timesteps*num_samples));
  HANDLE_ERROR(hipMalloc((void**)&costs_d, sizeof(float)*num_samples*num_candidates));

  // We need to generate a nominal trajectory for the control
  Eigen::Matrix<float, dynamics::CONTROL_DIM, num_timesteps> nominal_control = Eigen::MatrixXf::Random(dynamics::CONTROL_DIM, num_timesteps);

//  std::cout << "Nominal Control" << nominal_control << std::endl;

  // Exploration variance
  Eigen::Matrix<float, dynamics::CONTROL_DIM, 1> exploration_var;
  exploration_var << 2, 2;

//  std::cout << exploration_var << std::endl;

  // Generate noise to perturb the nominal control
  // Seed the PseudoRandomGenerator with the CPU time.
  hiprandGenerator_t gen_;
  hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  hiprandSetPseudoRandomGeneratorSeed(gen_, seed);
  hiprandGenerateNormal(gen_, control_noise_d,
                       num_samples*num_candidates*num_timesteps*dynamics::CONTROL_DIM,
                       0.0, 1.0);

  // Copy the noise back to the CPU so we can use it!
  Eigen::Matrix<float, dynamics::CONTROL_DIM, num_samples*num_candidates*num_timesteps> control_noise;

  std::vector<float> control_noise_data(num_samples*num_candidates*num_timesteps*dynamics::CONTROL_DIM);

  HANDLE_ERROR(hipMemcpy(control_noise_data.data(), control_noise_d, sizeof(float)*num_candidates*num_samples*num_timesteps*dynamics::CONTROL_DIM, hipMemcpyDeviceToHost));

  control_noise =  Eigen::Map<Eigen::Matrix<float,dynamics::CONTROL_DIM, num_candidates*num_samples*num_timesteps>>(control_noise_data.data());

//   std::cout << "Control Noise\n" << control_noise.col(num_samples*num_timesteps).cwiseProduct(exploration_var).transpose() << std::endl;
  int ctrl_stride = 2;

  Eigen::Matrix<int, 1, 9> strides;
  strides << 1, 2, 3, 4, 4, 4, 4, 4, 4;


  // Let us make temporary variables to hold the states and state derivatives and controls
  dynamics::state_array x_current, x_dot_current;
  dynamics::control_array u_current;

  Eigen::Matrix<float, 1, num_samples*9> cost_vector;

  float cost_current = 0.0;
  for (int i = 0; i < 9; ++i) { // Iterate through each candidate
    Eigen::Matrix<float, dynamics::CONTROL_DIM, num_timesteps> candidate_nominal_control;
    // For each candidate we want to slide the controls according to their own stride.
    for (int k = 0; k < num_timesteps; ++k) {
      if (k + strides(i) >= num_timesteps) {
        candidate_nominal_control.col(k) = nominal_control.col(num_timesteps-1);
      } else {
        candidate_nominal_control.col(k) = nominal_control.col(k+strides(i));
      }
    }
//    if (i == 0) {
//      std::cout << "Nominal_control:\n" << nominal_control << std::endl;
//      std::cout << "Candidate_control:\n" << candidate_nominal_control << std::endl;
//    }
    for (int j = 0; j < num_samples; ++j) {
      x_current = x0_candidates.col(i);  // The initial state of the rollout
      for (int k = 0; k < num_timesteps; ++k) {
        // compute the cost
        if (k > 0) {
          cost_current += (cost->computeStateCost(x_current) * dt - cost_current) / (1.0*k);
        }
        // get the control plus a disturbance
        if (j == 0 || k < ctrl_stride) { // First sample should always be noise free as should any timesteps that are below the control stride
          u_current = candidate_nominal_control.col(k);
        } else {
          u_current = candidate_nominal_control.col(k) +
                      control_noise.col(i * num_samples * num_timesteps + j * num_timesteps + k).cwiseProduct(
                              exploration_var);
        }

        // compute the next state_dot
        model->computeDynamics(x_current, u_current, x_dot_current);
        // update the state to the next
        model->updateState(x_current, x_dot_current, dt);
        }
      // compute the terminal cost -> this is the free energy estimate, save it!
      cost_vector.col(i*num_samples + j) << cost_current;
      cost_current = 0.0;
    }
  }


//  std::cout << "Eigen strides: " << strides << std::endl;


  // Copy the state candidates to GPU
  HANDLE_ERROR(hipMemcpy(states_d, x0_candidates.data(), sizeof(float)*dynamics::STATE_DIM*num_candidates, hipMemcpyHostToDevice));

  // Copy the control to the GPU
  HANDLE_ERROR(hipMemcpy(control_d, nominal_control.data(), sizeof(float)*dynamics::CONTROL_DIM*num_timesteps, hipMemcpyHostToDevice));

  // Copy the strides to the GPU
  HANDLE_ERROR(hipMemcpy(strides_d, strides.data(), sizeof(int)*num_candidates, hipMemcpyHostToDevice));

  // Copy exploration variance to GPU
  HANDLE_ERROR(hipMemcpy(exploration_var_d, exploration_var.data(), sizeof(float)*dynamics::CONTROL_DIM, hipMemcpyHostToDevice));

  // Run the GPU test kernel of the init eval kernel and get the output data
  // ();
  rmppi_kernels::launchInitEvalKernel<dynamics, cost_function, 64, 8, num_samples>(model->model_d_, cost->cost_d_,
          num_candidates, num_timesteps, ctrl_stride, dt,
          strides_d, exploration_var_d, states_d, control_d, control_noise_d, costs_d);
  CudaCheckError();

  Eigen::Matrix<float, 1, num_samples*num_candidates> cost_vector_GPU;
  // Compare with the CPU version
  HANDLE_ERROR(hipMemcpy(cost_vector_GPU.data(), costs_d, sizeof(float)*num_samples*num_candidates, hipMemcpyDeviceToHost));

//  std::cout <<  "Cost Vector CPU\n" << cost_vector.col(65) << std::endl;
//  std::cout << "Cost Vector GPU\n" << cost_vector_GPU.col(65) << std::endl;

//  std::cout << (cost_vector - cost_vector_GPU).transpose() << std::endl;

  EXPECT_LT((cost_vector - cost_vector_GPU).norm(), 1e-4);
}

TEST(RMPPITest, CPURolloutKernel) {
  using DYN = DoubleIntegratorDynamics;
  using COST = DoubleIntegratorCircleCost;
  DYN model;
  COST cost;

  const int state_dim = DYN::STATE_DIM;
  const int control_dim = DYN::CONTROL_DIM;

  float dt = 0.01;
  // int max_iter = 10;
  float lambda = 0.5;
  const int num_timesteps = 7;
  const int num_rollouts = 5;

  // float x[num_rollouts * state_dim * 2];
  // float x_dot[num_rollouts * state_dim * 2];
  // float u[num_rollouts * control_dim * 2];
  // float du[num_rollouts * control_dim * 2];
  float sigma_u[control_dim] = {0.5, 0.4}; // variance to sample noise from
  // float fb_u[num_rollouts * control_dim * state_dim];

  DYN::state_array x_init_act;
  x_init_act << 0, 0, 0, 0;
  DYN::state_array x_init_nom;

  // Generate control noise
  float sampled_noise[num_rollouts * num_timesteps * control_dim * 2];
  std::mt19937 rng_gen;
  std::vector<std::normal_distribution<float>> control_dist;
  for (int i = 0; i < control_dim; i++) {
    control_dist.push_back(std::normal_distribution<float>(0, sigma_u[i]));
  }

  for (int n = 0; n < num_rollouts; n++) {
    int n_ind = n * num_timesteps * control_dim;
    for (int t = 0; t < num_timesteps; t++) {
      int t_ind = t * control_dim;
      for (int j = 0; j < control_dim; j++) {
        sampled_noise[n_ind + t_ind + j] = control_dist[j](rng_gen);
      }
    }
  }
  // TODO: Figure out nonzero Initial control trajectory
  float u_traj[num_rollouts * num_timesteps * control_dim * 2] = {0};

  // TODO: fill the variance in with more reasonable numbers
  COST::control_matrix cost_variance = COST::control_matrix::Identity();

  // TODO: Generate feedback gain trajectories
  VanillaMPPIController<DYN, COST, 100, 512, 64, 8>::feedback_gain_trajectory feedback_gains;
  for (int i = 0; i < num_timesteps; i++) {
    feedback_gains.push_back(DYN::feedback_matrix::Random());
  }

  // Copy Feedback Gains into an array
  float feedback_array[num_timesteps * control_dim * state_dim];
  for (size_t i = 0; i < feedback_gains.size(); i++) {
    // std::cout << "Matrix " << i << ":\n";
    // std::cout << feedback_gains[i] << std::endl;
    int i_index = i * control_dim * state_dim;

    for (size_t j = 0; j < control_dim * state_dim; j++) {
      feedback_array[i_index + j] = feedback_gains[i].data()[j];
    }
  }

  for (int traj_i = 0; traj_i < num_rollouts; traj_i++)  {
    float cost_real_w_tracking = 0; // S^(V, x_0, x*_0) in Grady Thesis (8.24)
    float total_cost_real = 0; // S(V, x_0) with knowledge of tracking controller
    float state_cost_nom = 0; // S(V, x*_0)

    int traj_index = traj_i * num_rollouts;

    // Get all relevant values at time t in rollout i
    DYN::state_array x_t_nom = x_init_nom;
    DYN::state_array x_t_act = x_init_act;
    // Eigen::Map<DYN::state_array> x_t_act(x + traj_index * state_dim);
    // for (int state_i = 0; state_i < state_dim; state_i++) {
    //   x_t_act(state_i, 0) = x[traj_index * state_dim + state_i];
    //   x_t_nom(state_i, 0) = x[(traj_index + num_rollouts) * state_dim + state_i];
    // }

    for (int t = 0; t < num_timesteps - 1; t++){
      // Controls are read only so I can use Eigen::Map<const...>
      Eigen::Map<const DYN::control_array>
          u_t(u_traj + (traj_index * num_timesteps + t) * control_dim); // trajectory u at time t
      Eigen::Map<const DYN::control_array>
          eps_t(sampled_noise + (traj_index * num_timesteps + t) * control_dim); // Noise at time t
      Eigen::Map<const DYN::feedback_matrix>
          feedback_gains_t(feedback_array + t * control_dim * state_dim); // Feedback gains at time t
      // if (traj_i == 0) {
      //   std::cout << "feedback_gains_t " << traj_i << ", " << t << "s:\n" << feedback_gains_t << std::endl;
      // }


      // Create newly calculated values at time t in rollout i
      DYN::state_array x_dot_t_nom;
      DYN::state_array x_dot_t_act;
      DYN::control_array u_nom = u_t + eps_t;
      DYN::control_array fb_u_t = feedback_gains_t * (x_t_nom - x_t_act);
      DYN::control_array u_act = u_nom + fb_u_t;

      // Cost update
      DYN::control_array zero_u = DYN::control_array::Zero();
      state_cost_nom += cost.computeStateCost(x_t_nom);
      float state_cost_act = cost.computeStateCost(x_t_act);
      cost_real_w_tracking +=  state_cost_act +
                               cost.computeFeedbackCost(zero_u, zero_u, fb_u_t, cost_variance, lambda);

      total_cost_real += state_cost_act +
                         cost.computeLikelihoodRatioCost(u_t + fb_u_t, eps_t, cost_variance, lambda);

      // Dyanamics Update
      model.computeStateDeriv(x_t_nom, u_nom, x_dot_t_nom);
      model.computeStateDeriv(x_t_act, u_act, x_dot_t_act);

      model.updateState(x_t_act, x_dot_t_act, dt);
      model.updateState(x_t_nom, x_dot_t_nom, dt);
    }
    // cost_real_w_tracking += TERMINAL_COST(x_t_act);
    // state_cost_nom += TERMINAL_COST(x_t_nom);
    // total_cost_real += += TERMINAL_COST(x_t_act);
    // TODO Choose alpha better
    float alpha = 0.5;
    float cost_nom = 0.5 * state_cost_nom + 0.5 * std::max(std::min(cost_real_w_tracking, alpha), state_cost_nom);
    // std::cout << "for loop problems, I feel bad for you son" << std::endl;
    for (int t = 0; t < num_timesteps - 1; t++) {
      Eigen::Map<DYN::control_array>
          u_t(u_traj + (traj_index + num_timesteps) * control_dim); // trajectory u at time t
      Eigen::Map<DYN::control_array>
          eps_t(sampled_noise + (traj_index + num_timesteps) * control_dim); // Noise at time t
      cost_nom += cost.computeLikelihoodRatioCost(u_t, eps_t, cost_variance);
    }
  }
}