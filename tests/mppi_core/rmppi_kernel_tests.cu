#include "hip/hip_runtime.h"
//
// Created by mgandhi on 5/23/20.
//
#include <gtest/gtest.h>
#include <mppi/dynamics/double_integrator/di_dynamics.cuh>
#include <mppi/cost_functions/double_integrator/double_integrator_circle_cost.cuh>
#include <mppi/core/rmppi_kernel_test.cuh>
#include <mppi/controllers/MPPI/mppi_controller.cuh>
#include <mppi/utils/test_helper.h>
#include <vector>
#include <iostream>

class RMPPIKernels : public ::testing::Test {
public:
  using dynamics = DoubleIntegratorDynamics;
  using cost_function = DoubleIntegratorCircleCost;

  void SetUp() override {
    model = new dynamics(10);  // Initialize the double integrator dynamics
    cost = new cost_function;  // Initialize the cost function
  }

  void TearDown() override {
    delete model;
    delete cost;
  }

  dynamics* model;
  cost_function* cost;
};

TEST_F(RMPPIKernels, InitEvalRollout) {
  // Given the initial states, we need to roll out the number of samples.
  // 1.)  Generate the noise used to evaluate each sample.
  const int num_candidates = 9;

  float dt = 0.01;
  float lambda = 0.75;
  float alpha = 0.5;
  int crash_status[1] = {0};

  Eigen::Matrix<float, 4, num_candidates> x0_candidates;
  x0_candidates << -4 , -3, -2, -1, 0, 1, 2, 3, 4, 0, 1, 2, 3, 4, 4, 4, 4, 4,
  0,  0,  0,  0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;

  // For each candidate, we want to estimate the free energy using a set number of samples.
  const int num_samples = 64;

  // We are going to propagate a trajectory for a given number of timesteps
  const int num_timesteps = 5;

  // Call the GPU setup functions of the model and cost
  model->GPUSetup();
  cost->GPUSetup();

  // Allocate and deallocate the CUDA memory
  int* strides_d;
  float* exploration_var_d;
  float* states_d;
  float* control_d;
  float* control_noise_d;
  float* costs_d;

  HANDLE_ERROR(hipMalloc((void**)&strides_d, sizeof(int)*num_candidates));
  HANDLE_ERROR(hipMalloc((void**)&exploration_var_d, sizeof(float)*dynamics::CONTROL_DIM));
  HANDLE_ERROR(hipMalloc((void**)&states_d, sizeof(float)*dynamics::STATE_DIM*num_candidates));
  HANDLE_ERROR(hipMalloc((void**)&control_d, sizeof(float)*dynamics::CONTROL_DIM*num_timesteps));
  HANDLE_ERROR(hipMalloc((void**)&control_noise_d, sizeof(float)*dynamics::CONTROL_DIM*num_candidates*num_timesteps*num_samples));
  HANDLE_ERROR(hipMalloc((void**)&costs_d, sizeof(float)*num_samples*num_candidates));

  // We need to generate a nominal trajectory for the control
  Eigen::Matrix<float, dynamics::CONTROL_DIM, num_timesteps> nominal_control = Eigen::MatrixXf::Random(dynamics::CONTROL_DIM, num_timesteps);

//  std::cout << "Nominal Control" << nominal_control << std::endl;

  // Exploration variance
  Eigen::Matrix<float, dynamics::CONTROL_DIM, 1> exploration_var;
  exploration_var << 2, 2;

//  std::cout << exploration_var << std::endl;

  // Generate noise to perturb the nominal control
  // Seed the PseudoRandomGenerator with the CPU time.
  hiprandGenerator_t gen_;
  hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  hiprandSetPseudoRandomGeneratorSeed(gen_, seed);
  hiprandGenerateNormal(gen_, control_noise_d,
                       num_samples*num_candidates*num_timesteps*dynamics::CONTROL_DIM,
                       0.0, 1.0);

  // Copy the noise back to the CPU so we can use it!
  Eigen::Matrix<float, dynamics::CONTROL_DIM, num_samples*num_candidates*num_timesteps> control_noise;

  std::vector<float> control_noise_data(num_samples*num_candidates*num_timesteps*dynamics::CONTROL_DIM);

  HANDLE_ERROR(hipMemcpy(control_noise_data.data(), control_noise_d, sizeof(float)*num_candidates*num_samples*num_timesteps*dynamics::CONTROL_DIM, hipMemcpyDeviceToHost));

  control_noise =  Eigen::Map<Eigen::Matrix<float,dynamics::CONTROL_DIM, num_candidates*num_samples*num_timesteps>>(control_noise_data.data());

  int ctrl_stride = 2;

  Eigen::Matrix<int, 1, 9> strides;
  strides << 1, 2, 3, 4, 4, 4, 4, 4, 4;

  // Let us make temporary variables to hold the states and state derivatives and controls
  dynamics::state_array x_current, x_dot_current;
  dynamics::control_array u_current;
  dynamics::control_array noise_current;

  Eigen::Matrix<float, 1, num_samples*9> cost_vector;

  float cost_current = 0.0;
  for (int i = 0; i < 9; ++i) { // Iterate through each candidate
    Eigen::Matrix<float, dynamics::CONTROL_DIM, num_timesteps> candidate_nominal_control;
    // For each candidate we want to slide the controls according to their own stride.
    for (int k = 0; k < num_timesteps; ++k) {
      if (k + strides(i) >= num_timesteps) {
        candidate_nominal_control.col(k) = nominal_control.col(num_timesteps-1);
      } else {
        candidate_nominal_control.col(k) = nominal_control.col(k+strides(i));
      }
    }

    for (int j = 0; j < num_samples; ++j) {
      x_current = x0_candidates.col(i);  // The initial state of the rollout
      for (int k = 0; k < num_timesteps; ++k) {
        // get the control plus a disturbance
        if (j == 0 || k < ctrl_stride) { // First sample should always be noise free as should any timesteps that are below the control stride
          noise_current = dynamics::control_array::Zero();
        } else {
          noise_current = control_noise.col((i * num_samples + j) * num_timesteps + k).cwiseProduct(
                  exploration_var);
        }
        u_current = candidate_nominal_control.col(k) + noise_current;

        // enforce constraints
        model->enforceConstraints(x_current, u_current);

        // compute the cost
        if (k > 0) {
          cost_current += (cost->computeRunningCost(x_current, candidate_nominal_control.col(k), noise_current, exploration_var, lambda, alpha, k, crash_status) - cost_current) / (1.0f*k);
        }

        // compute the next state_dot
        model->computeStateDeriv(x_current, u_current, x_dot_current);
        // update the state to the next
        model->updateState(x_current, x_dot_current, dt);
        }
      // compute the terminal cost -> this is the free energy estimate, save it!
      cost_vector.col(i*num_samples + j) << cost_current;
      cost_current = 0.0;
    }
  }

  // Copy the state candidates to GPU
  HANDLE_ERROR(hipMemcpy(states_d, x0_candidates.data(), sizeof(float)*dynamics::STATE_DIM*num_candidates, hipMemcpyHostToDevice));

  // Copy the control to the GPU
  HANDLE_ERROR(hipMemcpy(control_d, nominal_control.data(), sizeof(float)*dynamics::CONTROL_DIM*num_timesteps, hipMemcpyHostToDevice));

  // Copy the strides to the GPU
  HANDLE_ERROR(hipMemcpy(strides_d, strides.data(), sizeof(int)*num_candidates, hipMemcpyHostToDevice));

  // Copy exploration variance to GPU
  HANDLE_ERROR(hipMemcpy(exploration_var_d, exploration_var.data(), sizeof(float)*dynamics::CONTROL_DIM, hipMemcpyHostToDevice));

  // Run the GPU test kernel of the init eval kernel and get the output data
  // ();
  rmppi_kernels::launchInitEvalKernel<dynamics, cost_function, 64, 8, num_samples>(model->model_d_, cost->cost_d_,
          num_candidates, num_timesteps, lambda, alpha, ctrl_stride, dt,
          strides_d, exploration_var_d, states_d, control_d, control_noise_d, costs_d, 0);

  CudaCheckError();

  Eigen::Matrix<float, 1, num_samples*num_candidates> cost_vector_GPU;
  // Compare with the CPU version
  HANDLE_ERROR(hipMemcpy(cost_vector_GPU.data(), costs_d, sizeof(float)*num_samples*num_candidates, hipMemcpyDeviceToHost));

  EXPECT_LT((cost_vector - cost_vector_GPU).norm(), 5e-3);
}

TEST(RMPPITest, RMPPIRolloutKernel_CPU_v_GPU) {
  using DYN = DoubleIntegratorDynamics;
  using COST = DoubleIntegratorCircleCost;
  DYN model;
  COST cost;

  const int state_dim = DYN::STATE_DIM;
  const int control_dim = DYN::CONTROL_DIM;

  float dt = 0.01;
  // int max_iter = 10;
  float lambda = 1.0;
  float alpha = 0.1;
  const int num_timesteps = 50;
  const int num_rollouts = 64;
  int optimization_stride = 1;

  float sigma_u[control_dim] = {0.5, 0.05}; // variance to sample noise from
  COST::control_matrix cost_variance = COST::control_matrix::Identity();
  for(int i = 0; i < control_dim; i++) {
    cost_variance(i, i) = sigma_u[i];
  }
  // float fb_u[num_rollouts * control_dim * state_dim];

  DYN::state_array x_init_act;
  x_init_act << 2, 0, 0, 0;
  DYN::state_array x_init_nom;
  x_init_nom << 2, 0, 0.1, 0;

  // Generate control noise
  float sampled_noise[num_rollouts * num_timesteps * control_dim];
  std::mt19937 rng_gen;
  std::vector<std::normal_distribution<float>> control_dist;
  for (int i = 0; i < control_dim; i++) {
    control_dist.push_back(std::normal_distribution<float>(0, 1));
  }

  for (int n = 0; n < num_rollouts; n++) {
    int n_ind = n * num_timesteps * control_dim;
    for (int t = 0; t < num_timesteps; t++) {
      int t_ind = t * control_dim;
      for (int j = 0; j < control_dim; j++) {
        sampled_noise[n_ind + t_ind + j] = control_dist[j](rng_gen);
      }
    }
  }
  // TODO: Figure out nonzero Initial control trajectory
  float u_traj[num_timesteps * control_dim] = {0};

  for (int i = 0; i < num_timesteps*control_dim; i++) {
    u_traj[i] = 2.0;
  }

  u_traj[0] = 1;
  u_traj[1] = 0.5;

  u_traj[10] = 1;
  u_traj[11] = 0.5;

  u_traj[14] = -1;
  u_traj[15] = 0.5;

  // TODO: Generate feedback gain trajectories
  VanillaMPPIController<DYN, COST, num_timesteps, num_rollouts, 64, 8>::feedback_gain_trajectory feedback_gains;
  for (int i = 0; i < num_timesteps; i++) {
    feedback_gains.push_back(DYN::feedback_matrix::Constant(-15));
  }

  // Copy Feedback Gains into an array
  float feedback_array[num_timesteps * control_dim * state_dim];
  for (size_t i = 0; i < feedback_gains.size(); i++) {
//     std::cout << "Matrix " << i << ":\n";
//     std::cout << feedback_gains[i] << std::endl;
    int i_index = i * control_dim * state_dim;

    for (size_t j = 0; j < control_dim * state_dim; j++) {
      feedback_array[i_index + j] = feedback_gains[i].data()[j];
    }
  }
  /**
   * Create vectors of data for GPU/CPU test
   */
  std::vector<float> x_init_act_vec, x_init_nom_vec, sigma_u_vec, u_traj_vec;
  x_init_act_vec.assign(x_init_act.data(), x_init_act.data() + state_dim);
  x_init_nom_vec.assign(x_init_nom.data(), x_init_nom.data() + state_dim);
  sigma_u_vec.assign(sigma_u, sigma_u + control_dim);
  u_traj_vec.assign(u_traj, u_traj + num_timesteps * control_dim);
  std::vector<float> feedback_gains_seq_vec, sampled_noise_vec;
  feedback_gains_seq_vec.assign(feedback_array, feedback_array +
    num_timesteps * control_dim * state_dim);
  int control_traj_size = num_rollouts * num_timesteps * control_dim;

  sampled_noise_vec.reserve(control_traj_size * 2);
  for(int i = 0; i < control_traj_size; i++) {
    sampled_noise_vec[i] = sampled_noise[i];
    sampled_noise_vec[control_traj_size + i] = sampled_noise_vec[i];
  }

  float value_func_threshold = 50000;

//  std::cout <<  "X_init_act_vec " << std::endl;
//  for (int i = 0; i < x_init_act_vec.size(); ++i) {
//    std::cout <<  " " << x_init_act_vec[i];
//  }
//  std::cout << std::endl;
//
//  std::cout <<  "X_init_nom_vec " << std::endl;
//  for (int i = 0; i < x_init_nom_vec.size(); ++i) {
//    std::cout <<  " " << x_init_nom_vec[i];
//  }
//  std::cout << std::endl;


  // Output Trajectory Costs
  std::array<float, num_rollouts> costs_act_GPU, costs_nom_GPU;
  std::array<float, num_rollouts> costs_act_CPU, costs_nom_CPU;
  launchRMPPIRolloutKernelGPU<DYN, COST, num_rollouts>(&model, &cost, dt,
    num_timesteps, optimization_stride, lambda, alpha, value_func_threshold, x_init_act_vec, x_init_nom_vec,
    sigma_u_vec, u_traj_vec, feedback_gains_seq_vec, sampled_noise_vec,
    costs_act_GPU, costs_nom_GPU);
  launchRMPPIRolloutKernelCPU<DYN, COST, num_rollouts>(&model, &cost, dt,
    num_timesteps, optimization_stride, lambda, alpha, value_func_threshold, x_init_act_vec, x_init_nom_vec,
    sigma_u_vec, u_traj_vec, feedback_gains_seq_vec, sampled_noise_vec,
    costs_act_CPU, costs_nom_CPU);

//  for (int i = 0; i < costs_nom_CPU.size(); ++i) {
//    std::cout << "Nominal Cost CPU: " << costs_nom_CPU[i] << std::endl;
//    std::cout << "Nominal Cost GPU: " << costs_nom_GPU[i] << std::endl;
//  }

  float max_diff_nom = -100;
  float max_diff_act = -100;
  int diff_nom_ind = -1;
  int diff_act_ind = -1;
  for (int i = 0; i < num_rollouts; i++) {
    // std::cout << i << ": GPU Nom: " << costs_nom_GPU[i] << ", CPU Nom: " << costs_nom_CPU[i] << std::endl;
    float diff_nom  = std::abs(costs_nom_CPU[i] - costs_nom_GPU[i]);
    float diff_act  = std::abs(costs_act_CPU[i] - costs_act_GPU[i]);
    if (diff_nom > max_diff_nom) {
      max_diff_nom = diff_nom;
      diff_nom_ind = i;
    }
    if (diff_act > max_diff_act) {
      max_diff_act = diff_act;
      diff_act_ind = i;
    }
  }
  std::cout << "Max Real Difference between CPU and GPU rollout "<< diff_act_ind
            << ": " << max_diff_act << std::endl;
  std::cout << "Max Nominal Difference between CPU and GPU rollout "<< diff_nom_ind
            << ": " << max_diff_nom << std::endl;
  array_assert_float_eq<num_rollouts>(costs_act_GPU, costs_act_CPU);
  std::cout << "Checking nominal systems differences between CPU and GPU" << std::endl;
  array_assert_float_eq<num_rollouts>(costs_nom_GPU, costs_nom_CPU);
}

TEST(RMPPITest, TwoSystemRolloutKernelComparison) {
  /**
   * If the nominal state and the real state are equal, and we are using the
   * same noise between the two, then the output result should be equal to the
   * standard rollout kernel.
   */
  using DYN = DoubleIntegratorDynamics;
  using COST = DoubleIntegratorCircleCost;
  DYN model;
  COST cost;

  model.GPUSetup();
  cost.GPUSetup();

  const int state_dim = DYN::STATE_DIM;
  const int control_dim = DYN::CONTROL_DIM;

  float dt = 0.01;
  float lambda = 4.2;
  float alpha = 0.05;
  const int num_timesteps = 100;
  const int num_rollouts = 256;
  int optimization_stride = 1;

  std::array<float, control_dim> sigma_u = {0.5, 1.5};

  std::array<float, state_dim> x_real = {2, 0, 1, 1};
  std::array<float, state_dim> x_nominal = {2, 0, 1, 1};

  std::array<float, control_dim*num_timesteps> u_init_trajectory{};
  std::default_random_engine generator(7.0);
  std::normal_distribution<float> distribution(0.0,1.0);
  for (auto & u_init : u_init_trajectory) {
    u_init = 2*distribution(generator);
  }

  std::array<float, num_timesteps*num_rollouts*control_dim> control_noise_array{};
  for (auto & noise : control_noise_array) {
    noise = distribution(generator);
  }


  // Create some random feedback gains
  VanillaMPPIController<DYN, COST, num_timesteps, num_rollouts, 64, 8>::feedback_gain_trajectory feedback_gains;
  feedback_gains.resize(num_timesteps);
  for (auto & feedback_gain : feedback_gains) {
    feedback_gain = Eigen::Matrix<float, control_dim, state_dim>::Random();
  }

  std::vector<float> feedback_gain_vector(num_timesteps*state_dim*control_dim);
  // Copy Feedback Gains into an array
  for (size_t i = 0; i < feedback_gains.size(); i++) {
    size_t i_index = i * DYN::STATE_DIM * DYN::CONTROL_DIM;
    for (size_t j = 0; j < DYN::CONTROL_DIM * DYN::STATE_DIM; j++) {
      feedback_gain_vector[i_index + j] = feedback_gains[i].data()[j];
    }
  }

  // Create objects that will hold the results
  std::array<float, 2*num_rollouts> rmppi_costs_out{};
  std::array<float, num_rollouts> mppi_costs_out{};

  // Launch the test kernel...
  launchComparisonRolloutKernelTest<DYN,COST, num_rollouts, num_timesteps, 64, 8>
          (&model, &cost, dt, lambda, alpha, x_real, x_nominal,
           feedback_gain_vector, u_init_trajectory, control_noise_array,
           sigma_u, rmppi_costs_out, mppi_costs_out, optimization_stride, 0);


  for (int i = 0; i < num_rollouts; i++) {
    ASSERT_FLOAT_EQ(rmppi_costs_out[num_rollouts+i], rmppi_costs_out[i])  <<  i;
  }

  for (int i = 0; i < num_rollouts; i++) {
    EXPECT_NEAR(rmppi_costs_out[i], mppi_costs_out[i], 1e-1) << i;
  }

  for (int i = 0; i < num_rollouts; i++) {
    EXPECT_NEAR(rmppi_costs_out[num_rollouts+i], mppi_costs_out[i], 1e-1)  << num_rollouts + i;
  }

}
