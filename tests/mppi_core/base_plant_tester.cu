#include "hip/hip_runtime.h"
//
// Created by jgibson37 on 2/24/20.
//

#include <gtest/gtest.h>
#include <gmock/gmock.h>
#include <mppi/utils/test_helper.h>
#include <random>
#include <algorithm>
#include <numeric>

#include <mppi/core/base_plant.hpp>
#include <mppi/instantiations/cartpole_mppi/cartpole_mppi.cuh>
#include <mppi_test/mock_classes/mock_dynamics.h>
#include <mppi_test/mock_classes/mock_controller.h>
#include <mppi_test/mock_classes/mock_costs.h>

template <class CONTROLLER_T>
class TestPlant : public BasePlant<CONTROLLER_T>
{
public:
  double time_ = 0.0;

  int pubControlCalled = 0;
  int pubNominalStateCalled = 0;

  using c_array = typename CONTROLLER_T::control_array;
  using c_traj = typename CONTROLLER_T::control_trajectory;

  using s_array = typename CONTROLLER_T::state_array;
  using s_traj = typename CONTROLLER_T::state_trajectory;

  using DYN_T = typename CONTROLLER_T::TEMPLATED_DYNAMICS;
  using DYN_PARAMS_T = typename DYN_T::DYN_PARAMS_T;
  using COST_T = typename CONTROLLER_T::TEMPLATED_COSTS;
  using COST_PARAMS_T = typename COST_T::COST_PARAMS_T;
  double timestamp_;
  double loop_speed_;

  TestPlant(std::shared_ptr<MockController> controller, int hz = 20, int opt_stride = 1)
    : BasePlant<CONTROLLER_T>(controller, hz, opt_stride)
  {
  }

  void pubControl(const c_array& u) override
  {
    pubControlCalled++;
  }

  void pubNominalState(const s_array& s) override
  {
    pubNominalStateCalled++;
  }

  void pubFreeEnergyStatistics(MPPIFreeEnergyStatistics& fe_stats) override
  {
  }

  void incrementTime()
  {
    time_ += 0.05;
  }

  void incrementTime(double dt)
  {
    time_ += dt;
  }

  int checkStatus() override
  {
    return 1;
  }

  double getCurrentTime()
  {
    return time_ + 0.3421;
  }

  double getPoseTime()
  {
    return time_;
  }

  // accessors for protected members
  int getNumIter()
  {
    return this->num_iter_;
  }
  double getLastUsedPoseUpdateTime()
  {
    return this->last_used_state_update_time_;
  }
  int getStatus()
  {
    return this->status_;
  }
  bool getDebugMode()
  {
    return this->debug_mode_;
  }
  double getOptimizationDuration()
  {
    return this->optimization_duration_;
  }
  double getOptimizationAvg()
  {
    return this->avg_optimize_time_ms_;
  }
  double getLoopDuration()
  {
    return this->optimize_loop_duration_;
  }
  double getLoopAvg()
  {
    return this->avg_loop_time_ms_;
  }
  double getFeedbackDuration()
  {
    return this->feedback_duration_;
  }
  double getFeedbackAvg()
  {
    return this->avg_feedback_time_ms_;
  }
  void setLastTime(double time)
  {
    time_ = time;
  }
  double getSleepTimeAvg()
  {
    return this->avg_sleep_time_ms_;
  }
};

typedef TestPlant<MockController> MockTestPlant;

class BasePlantTest : public ::testing::Test
{
protected:
  void SetUp() override
  {
    EXPECT_CALL(mockCost, getParams()).Times(1);
    EXPECT_CALL(mockDynamics, getParams()).Times(1);

    mockController = std::make_shared<MockController>();
    EXPECT_CALL(*mockController, getDt()).WillRepeatedly(testing::Return(0.05));
    mockFeedback = new FEEDBACK_T(&mockDynamics, mockController->getDt());
    mockController->cost_ = &mockCost;
    mockController->model_ = &mockDynamics;
    mockController->fb_controller_ = mockFeedback;

    plant = std::make_shared<MockTestPlant>(mockController);
  }

  void TearDown() override
  {
    plant = nullptr;
    mockController = nullptr;
    delete mockFeedback;
  }
  MockDynamics mockDynamics;
  MockCost mockCost;
  FEEDBACK_T* mockFeedback;
  std::shared_ptr<MockController> mockController;
  std::shared_ptr<MockTestPlant> plant;

  const float SMALL_TIME_MS = 8;
};

TEST_F(BasePlantTest, Constructor)
{
  EXPECT_EQ(plant->controller_, mockController);
  EXPECT_EQ(plant->getHz(), 20);
  EXPECT_EQ(plant->getTargetOptimizationStride(), 1);
  EXPECT_EQ(plant->getNumIter(), 0);
  EXPECT_EQ(plant->getLastUsedPoseUpdateTime(), -1);
  EXPECT_EQ(plant->getStatus(), 1);
  EXPECT_EQ(mockController->getFeedbackEnabled(), false);
  EXPECT_EQ(plant->hasNewCostParams(), false);
  EXPECT_EQ(plant->hasNewDynamicsParams(), false);
}

TEST_F(BasePlantTest, getAndSetState)
{
  // check initial state is zerod

  MockController::state_array state = plant->getState();
  for (int i = 0; i < 1; i++)
  {
    EXPECT_EQ(state(i), 0.0);
  }

  MockController::state_array new_state;
  for (int i = 0; i < 1; i++)
  {
    new_state(i) = i;
  }
  plant->setState(new_state);
  state = plant->getState();
  for (int i = 0; i < 1; i++)
  {
    EXPECT_EQ(state(i), i);
  }
}

TEST_F(BasePlantTest, getSetOptimizationStride)
{
  int optimization_stride = plant->getTargetOptimizationStride();

  EXPECT_EQ(optimization_stride, 1);

  plant->setTargetOptimizationStride(5);
  optimization_stride = plant->getTargetOptimizationStride();

  EXPECT_EQ(optimization_stride, 5);
}

TEST_F(BasePlantTest, getSetDynamicsParams)
{
  EXPECT_EQ(plant->hasNewDynamicsParams(), false);
  MockTestPlant::DYN_PARAMS_T params;

  params.test = 3;

  plant->setDynamicsParams(params);
  EXPECT_EQ(plant->hasNewDynamicsParams(), true);

  MockTestPlant::DYN_PARAMS_T new_params = plant->getNewDynamicsParams();
  EXPECT_EQ(plant->hasNewDynamicsParams(), false);
  EXPECT_EQ(new_params.test, params.test);
}

TEST_F(BasePlantTest, getSetCostParams)
{
  EXPECT_EQ(plant->hasNewCostParams(), false);

  MockTestPlant::COST_PARAMS_T params;
  params.test = 100;

  plant->setCostParams(params);
  EXPECT_EQ(plant->hasNewCostParams(), true);

  auto new_params = plant->getNewCostParams();
  EXPECT_EQ(plant->hasNewCostParams(), false);
  EXPECT_EQ(params.test, new_params.test);
}

TEST_F(BasePlantTest, updateParametersAllFalse)
{
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(0);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(0);

  plant->updateParameters();
}

TEST_F(BasePlantTest, updateParametersAllTrue)
{
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(1);

  plant->setDebugMode(true);
  plant->setDynamicsParams(MockDynamics::DYN_PARAMS_T());
  plant->setCostParams(MockCost::COST_PARAMS_T());

  plant->updateParameters();
}

TEST_F(BasePlantTest, updateStateOutsideTimeTest)
{
  mockController->setDt(0.05);
  plant->setLastTime(0);

  EXPECT_CALL(*mockController, getCurrentControl(testing::_, testing::_, testing::_, testing::_, testing::_)).Times(0);

  MockController::state_array state = MockController::state_array::Zero();
  plant->updateState(state, mockController->getDt() * mockController->getNumTimesteps() + 0.01);
  EXPECT_EQ(plant->getState(), state);

  plant->setLastTime(100);
  plant->updateState(state, 99.99);
  EXPECT_EQ(plant->getState(), state);
  EXPECT_EQ(plant->pubControlCalled, 0);
  EXPECT_EQ(plant->pubNominalStateCalled, 0);
}

TEST_F(BasePlantTest, updateStateTest)
{
  mockController->setDt(0.05);
  plant->setLastTime(0);

  MockController::state_array state = MockController::state_array::Zero();
  EXPECT_CALL(*mockController, getCurrentControl(testing::_, testing::_, testing::_, testing::_, testing::_)).Times(1);
  plant->updateState(state, mockController->getDt());
  EXPECT_EQ(plant->getState(), state);

  EXPECT_EQ(plant->pubControlCalled, 1);
  EXPECT_EQ(plant->pubNominalStateCalled, 0);

  // EXPECT_CALL(*mockController, getCurrentControl(state, mockController->getDt()+100)).Times(1);
  // plant->setLastTime(100);
  // plant->updateState(state, 100+mockController->getDt());
  // EXPECT_EQ(plant->getState(), state);
}

TEST_F(BasePlantTest, runControlIterationStoppedTest)
{
  EXPECT_CALL(*mockController, slideControlSequence(testing::_)).Times(0);
  EXPECT_CALL(*mockController, computeControl(testing::_, testing::_)).Times(0);

  std::atomic<bool> is_alive(false);
  plant->runControlIteration(&is_alive);
}

// TODO speed up to make tests run faster
TEST_F(BasePlantTest, runControlIterationDebugFalseNoFeedbackTest)
{
  double init_time = 100;
  plant->setLastTime(init_time);

  for (int i = 0; i < 2; i++)
  {
    double wait_ms = 50 * i;

    auto wait_function = [wait_ms](const Eigen::Ref<const MockController::state_array>& state,
                                   int optimization_stride = 0) { usleep(wait_ms * 1e3); };

    int expect_opt_stride = i > 0 ? 1 : 0;

    EXPECT_CALL(*mockController, slideControlSequence(expect_opt_stride)).Times(i > 0 ? 1 : 0);
    EXPECT_CALL(*mockController, computeControl(testing::_, testing::_))
        .Times(1)
        .WillRepeatedly(testing::Invoke(wait_function));
    MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
    EXPECT_CALL(*mockController, getControlSeq()).Times(1).WillRepeatedly(testing::Return(control_seq));
    MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
    EXPECT_CALL(*mockController, getTargetStateSeq()).Times(1).WillRepeatedly(testing::Return(state_seq));

    EXPECT_CALL(*mockController, computeFeedback(testing::_)).Times(0);
    EXPECT_CALL(*mockController, getFeedbackControl(testing::_, testing::_, testing::_)).Times(0);
    EXPECT_CALL(*mockController, computeFeedbackPropagatedStateSeq()).Times(1);
    EXPECT_CALL(*mockController, calculateSampledStateTrajectories()).Times(0);

    EXPECT_EQ(plant->getDebugMode(), false);

    std::atomic<bool> is_alive(true);
    plant->runControlIteration(&is_alive);
    plant->incrementTime();

    EXPECT_EQ(plant->checkStatus(), 1);
    EXPECT_EQ(plant->getStateTraj(), state_seq);
    EXPECT_EQ(plant->getControlTraj(), control_seq);
    MockController::TEMPLATED_FEEDBACK_STATE feedback = plant->getFeedbackState();
    MockController::state_array state = MockController::state_array::Ones();
    for (int j = 0; j < 100; j++)
    {
      // TODO check that feedback is correct
      // auto result = feedback[i] * state;
      // float sum = feedback[i] * state;
      // EXPECT_FLOAT_EQ(result(0), 0);
    }

    // check last pose update
    EXPECT_FLOAT_EQ(plant->getLastUsedPoseUpdateTime(), 0.05 * i + init_time);
    EXPECT_EQ(plant->getNumIter(), i + 1);
    EXPECT_EQ(plant->getLastOptimizationStride(), expect_opt_stride);

    EXPECT_THAT(plant->getOptimizationDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
    EXPECT_LT(plant->getOptimizationAvg(), wait_ms + SMALL_TIME_MS);
    EXPECT_THAT(plant->getLoopDuration(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
    EXPECT_LT(plant->getLoopAvg(), wait_ms + SMALL_TIME_MS);
    EXPECT_LE(plant->getFeedbackDuration(), SMALL_TIME_MS);
    EXPECT_LE(plant->getFeedbackAvg(), SMALL_TIME_MS);
  }
}

TEST_F(BasePlantTest, runControlIterationDebugFalseFeedbackTest)
{
  mockController->initFeedback();

  double init_time = 51789;
  plant->setLastTime(init_time);

  for (int i = 0; i < 10; i++)
  {
    double wait_ms = 50 * i;

    auto wait_function = [wait_ms](const Eigen::Ref<const MockController::state_array>& state,
                                   int optimization_stride = 0) { usleep(wait_ms * 1e3); };

    int expect_opt_stride = i > 0 ? 1 : 0;

    EXPECT_CALL(*mockController, slideControlSequence(expect_opt_stride)).Times(i > 0 ? 1 : 0);
    EXPECT_CALL(*mockController, computeControl(testing::_, testing::_))
        .Times(1)
        .WillRepeatedly(testing::Invoke(wait_function));
    MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
    EXPECT_CALL(*mockController, getControlSeq()).Times(1).WillRepeatedly(testing::Return(control_seq));
    MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
    EXPECT_CALL(*mockController, getTargetStateSeq()).Times(1).WillRepeatedly(testing::Return(state_seq));

    EXPECT_CALL(*mockController, computeFeedback(testing::_)).Times(1).WillRepeatedly(testing::Invoke(wait_function));
    MockController::TEMPLATED_FEEDBACK_STATE feedback;
    EXPECT_CALL(*mockController, getFeedbackState()).Times(1).WillRepeatedly(testing::Return(feedback));
    EXPECT_CALL(*mockController, computeFeedbackPropagatedStateSeq()).Times(1);
    EXPECT_CALL(*mockController, calculateSampledStateTrajectories()).Times(0);

    EXPECT_EQ(plant->getDebugMode(), false);

    std::atomic<bool> is_alive(true);
    plant->runControlIteration(&is_alive);
    plant->incrementTime();

    EXPECT_EQ(plant->checkStatus(), 1);
    EXPECT_EQ(plant->getStateTraj(), state_seq);
    EXPECT_EQ(plant->getControlTraj(), control_seq);
    EXPECT_EQ(plant->getFeedbackState(), feedback);

    // check last pose update
    EXPECT_FLOAT_EQ(plant->getLastUsedPoseUpdateTime(), 0.05 * i + init_time);
    EXPECT_EQ(plant->getNumIter(), i + 1);
    EXPECT_EQ(plant->getLastOptimizationStride(), expect_opt_stride);

    EXPECT_THAT(plant->getOptimizationDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
    EXPECT_LT(plant->getOptimizationAvg(), wait_ms + SMALL_TIME_MS);
    EXPECT_THAT(plant->getFeedbackDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le((wait_ms + SMALL_TIME_MS) * 2)));
    // TODO should be range as well
    EXPECT_LT(plant->getFeedbackAvg(), wait_ms + SMALL_TIME_MS);
    EXPECT_THAT(plant->getLoopDuration(),
                testing::AllOf(testing::Ge(wait_ms * 2), testing::Le((wait_ms + SMALL_TIME_MS) * 2)));
    EXPECT_LT(plant->getLoopAvg(), (wait_ms + SMALL_TIME_MS) * 2);
  }
}

TEST_F(BasePlantTest, runControlIterationDebugFalseFeedbackAvgTest)
{
  mockController->initFeedback();

  double init_time = 51531;
  plant->setLastTime(init_time);

  for (int i = 0; i < 10; i++)
  {
    double wait_ms = 50;

    auto wait_function = [wait_ms](const Eigen::Ref<const MockController::state_array>& state,
                                   int optimization_stride = 0) { usleep(wait_ms * 1e3); };

    int expect_opt_stride = i > 0 ? 1 : 0;

    EXPECT_CALL(*mockController, slideControlSequence(expect_opt_stride)).Times(i > 0 ? 1 : 0);
    EXPECT_CALL(*mockController, computeControl(testing::_, testing::_))
        .Times(1)
        .WillRepeatedly(testing::Invoke(wait_function));
    MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
    EXPECT_CALL(*mockController, getControlSeq()).Times(1).WillRepeatedly(testing::Return(control_seq));
    MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
    EXPECT_CALL(*mockController, getTargetStateSeq()).Times(1).WillRepeatedly(testing::Return(state_seq));

    EXPECT_CALL(*mockController, computeFeedback(testing::_)).Times(1).WillRepeatedly(testing::Invoke(wait_function));
    MockController::TEMPLATED_FEEDBACK_STATE feedback;
    EXPECT_CALL(*mockController, getFeedbackState()).Times(1).WillRepeatedly(testing::Return(feedback));
    EXPECT_CALL(*mockController, computeFeedbackPropagatedStateSeq()).Times(1);
    EXPECT_CALL(*mockController, calculateSampledStateTrajectories()).Times(0);

    EXPECT_EQ(plant->getDebugMode(), false);

    std::atomic<bool> is_alive(true);
    plant->runControlIteration(&is_alive);
    plant->incrementTime();

    EXPECT_EQ(plant->checkStatus(), 1);
    EXPECT_EQ(plant->getStateTraj(), state_seq);
    EXPECT_EQ(plant->getControlTraj(), control_seq);
    EXPECT_EQ(plant->getFeedbackState(), feedback);

    // check last pose update
    EXPECT_FLOAT_EQ(plant->getLastUsedPoseUpdateTime(), 0.05 * i + init_time);
    EXPECT_EQ(plant->getNumIter(), i + 1);
    EXPECT_EQ(plant->getLastOptimizationStride(), expect_opt_stride);

    EXPECT_THAT(plant->getOptimizationDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
    EXPECT_THAT(plant->getOptimizationAvg(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
    EXPECT_THAT(plant->getLoopDuration(),
                testing::AllOf(testing::Ge(wait_ms * 2), testing::Le((wait_ms + SMALL_TIME_MS) * 2)));
    EXPECT_THAT(plant->getLoopAvg(),
                testing::AllOf(testing::Ge((wait_ms)*2), testing::Le((wait_ms + SMALL_TIME_MS) * 2)));
    EXPECT_THAT(plant->getFeedbackDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
    EXPECT_THAT(plant->getFeedbackAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  }
}

TEST_F(BasePlantTest, runControlLoopRegular)
{
  mockController->initFeedback();

  int hz = plant->getHz();
  double test_duration = 1.0;  // in seconds for how long to run the test

  int init_time = 78;
  plant->setLastTime(init_time);

  // setup mock expected calls
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(0);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(0);
  EXPECT_CALL(*mockController, resetControls()).Times(1);

  double wait_s =
      (1.0 / hz) / 3;  // divide by 3 since wait is evenly split across computeFeedback, computeControl, and waiting

  auto wait_function = [wait_s](const Eigen::Ref<const MockController::state_array>& state,
                                int optimization_stride = 0) { usleep(wait_s * 1e6); };
  int iterations = int(std::round((hz * 1.0) / (test_duration)));  // number of times the method will be called
  // slide control sequence is skipped on the first iteration
  EXPECT_CALL(*mockController, slideControlSequence(1)).Times(iterations / 2 - 1);
  EXPECT_CALL(*mockController, computeControl(testing::_, testing::_))
      .Times(iterations / 2)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
  EXPECT_CALL(*mockController, getControlSeq()).Times(iterations / 2).WillRepeatedly(testing::Return(control_seq));
  MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
  EXPECT_CALL(*mockController, getTargetStateSeq()).Times(iterations / 2).WillRepeatedly(testing::Return(state_seq));
  EXPECT_CALL(*mockController, computeFeedback(testing::_))
      .Times(iterations / 2)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::TEMPLATED_FEEDBACK_STATE feedback;
  EXPECT_CALL(*mockController, getFeedbackState()).Times(iterations / 2).WillRepeatedly(testing::Return(feedback));
  EXPECT_CALL(*mockController, computeFeedbackPropagatedStateSeq()).Times(iterations / 2);
  EXPECT_CALL(*mockController, calculateSampledStateTrajectories()).Times(0);

  std::atomic<bool> is_alive(true);
  std::thread optimizer(&MockTestPlant::runControlLoop, plant.get(), &is_alive);

  std::chrono::steady_clock::time_point loop_start = std::chrono::steady_clock::now();
  std::chrono::duration<double, std::milli> loop_duration = std::chrono::steady_clock::now() - loop_start;
  // counter is number of dts
  for (int counter = 0; loop_duration.count() < test_duration * 1e3; counter++)
  {
    // wait until the correct hz has passed to tick the time
    // state at 100 Hz
    while (loop_duration.count() < (test_duration / 100) * 1e3 * counter)
    {
      usleep(50);
      loop_duration = std::chrono::steady_clock::now() - loop_start;
    }
    if (counter / 5 > iterations / 2)
    {  // this forces it to block
      plant->incrementTime(0.01);
    }
  }
  is_alive.store(false);
  optimizer.join();

  // check all the things
  EXPECT_EQ(plant->checkStatus(), 1);
  EXPECT_EQ(plant->getStateTraj(), state_seq);
  EXPECT_EQ(plant->getControlTraj(), control_seq);
  EXPECT_EQ(plant->getFeedbackState(), feedback);

  // check last pose update
  EXPECT_NE(plant->getLastUsedPoseUpdateTime(), 0.0);
  EXPECT_EQ(plant->getNumIter(), iterations / 2);
  EXPECT_EQ(plant->getLastOptimizationStride(), 1);

  double wait_ms = wait_s * 1e3;
  EXPECT_THAT(plant->getOptimizationDuration(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getOptimizationAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopDuration(),
              testing::AllOf(testing::Ge(wait_ms * 2), testing::Le(wait_ms * 2 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopAvg(), testing::AllOf(testing::Ge(wait_ms * 2), testing::Le(wait_ms * 2 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackDuration(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  // 10 iters of just waiting, 10 iters of waiting for correct time
  double expected_avg_wait = ((wait_ms * 3 * 10) + wait_ms * 10) / 10;
  EXPECT_THAT(plant->getSleepTimeAvg(),
              testing::AllOf(testing::Gt(expected_avg_wait), testing::Le(expected_avg_wait + SMALL_TIME_MS * 4)));
}

TEST_F(BasePlantTest, runControlLoopSlowed)
{
  mockController->initFeedback();

  int hz = plant->getHz();
  double test_duration = 1.0;  // in seconds for how long to run the test

  int init_time = 78;
  plant->setLastTime(init_time);

  MockTestPlant::COST_PARAMS_T cost_params;
  MockTestPlant::DYN_PARAMS_T dyn_params;

  double wait_s =
      (1.0 / hz) / 2;  // divide by 3 since wait is evenly split across computeFeedback, computeControl, and waiting

  auto wait_function = [wait_s](const Eigen::Ref<const MockController::state_array>& state,
                                int optimization_stride = 0) { usleep(wait_s * 1e6); };
  auto wait_function2 = [wait_s](MockTestPlant::COST_PARAMS_T) { usleep(wait_s * 1e6); };
  auto wait_function3 = [wait_s](MockTestPlant::DYN_PARAMS_T) { usleep(wait_s * 1e6); };

  int iterations = int(std::round((hz * 1.0) / (test_duration)));  // number of times the method will be called

  // setup mock expected calls
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(16).WillRepeatedly(testing::Invoke(wait_function2));
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(16).WillRepeatedly(testing::Invoke(wait_function3));
  EXPECT_CALL(*mockController, resetControls()).Times(1);

  // slide control sequence is skipped on the first iteration
  int expected_iters = iterations / 4;
  // EXPECT_CALL(*mockController, slideControlSequence(1)).Times(1);
  EXPECT_CALL(*mockController, slideControlSequence(testing::_)).Times(expected_iters - 1);
  EXPECT_CALL(*mockController, computeControl(testing::_, testing::_))
      .Times(expected_iters)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
  EXPECT_CALL(*mockController, getControlSeq()).Times(expected_iters).WillRepeatedly(testing::Return(control_seq));
  MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
  EXPECT_CALL(*mockController, getTargetStateSeq()).Times(expected_iters).WillRepeatedly(testing::Return(state_seq));
  EXPECT_CALL(*mockController, computeFeedback(testing::_))
      .Times(expected_iters)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::TEMPLATED_FEEDBACK_STATE feedback;
  EXPECT_CALL(*mockController, getFeedbackState()).Times(expected_iters).WillRepeatedly(testing::Return(feedback));
  EXPECT_CALL(*mockController, computeFeedbackPropagatedStateSeq()).Times(expected_iters);
  EXPECT_CALL(*mockController, calculateSampledStateTrajectories()).Times(0);

  std::atomic<bool> is_alive(true);
  std::thread optimizer(&MockTestPlant::runControlLoop, plant.get(), &is_alive);

  std::chrono::steady_clock::time_point loop_start = std::chrono::steady_clock::now();
  std::chrono::duration<double, std::milli> loop_duration = std::chrono::steady_clock::now() - loop_start;
  // counter is number of dts
  for (int counter = 0; loop_duration.count() < test_duration * 1e3; counter++)
  {
    // wait until the correct hz has passed to tick the time
    // state at 100 Hz
    while (loop_duration.count() < (test_duration / 100) * 1e3 * counter)
    {
      usleep(50);
      loop_duration = std::chrono::steady_clock::now() - loop_start;
    }
    if (counter / 5 > iterations / 2)
    {  // this forces it to block
      plant->incrementTime(0.01);
    }

    plant->setCostParams(cost_params);
    plant->setDynamicsParams(dyn_params);
  }
  is_alive.store(false);
  optimizer.join();

  // check all the things
  EXPECT_EQ(plant->checkStatus(), 1);
  EXPECT_EQ(plant->getStateTraj(), state_seq);
  EXPECT_EQ(plant->getControlTraj(), control_seq);
  EXPECT_EQ(plant->getFeedbackState(), feedback);

  // check last pose update
  EXPECT_NE(plant->getLastUsedPoseUpdateTime(), 0.0);
  EXPECT_EQ(plant->getNumIter(), expected_iters);
  EXPECT_EQ(plant->getLastOptimizationStride(), 2);

  double wait_ms = wait_s * 1e3;
  EXPECT_THAT(plant->getOptimizationDuration(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getOptimizationAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopDuration(),
              testing::AllOf(testing::Ge(wait_ms * 4), testing::Le(wait_ms * 4 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopAvg(), testing::AllOf(testing::Ge(wait_ms * 4), testing::Le(wait_ms * 4 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackDuration(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  // 10 iters of just waiting
  double expected_avg_wait = ((wait_ms * 2 * 10)) / 6;
  // EXPECT_THAT(plant->getSleepTimeAvg(), testing::AllOf(testing::Gt(expected_avg_wait - SMALL_TIME_MS),
  //                                                     testing::Le(expected_avg_wait + SMALL_TIME_MS * 4)));
}

TEST_F(BasePlantTest, runControlLoopRegularRealTime)
{
  mockController->initFeedback();

  int hz = plant->getHz();
  double test_duration = 1.0;  // in seconds for how long to run the test

  int init_time = 78;
  plant->setLastTime(init_time);
  plant->setUseRealTimeTiming(true);

  // setup mock expected calls
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(0);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(0);
  EXPECT_CALL(*mockController, resetControls()).Times(1);

  double wait_s =
      (1.0 / hz) / 3;  // divide by 3 since wait is evenly split across computeFeedback, computeControl, and waiting

  auto wait_function = [wait_s](const Eigen::Ref<const MockController::state_array>& state,
                                int optimization_stride = 0) { usleep(wait_s * 1e6); };
  int iterations = int(std::round((hz * 1.0) / (test_duration)));  // number of times the method will be called
  // slide control sequence is skipped on the first iteration
  EXPECT_CALL(*mockController, slideControlSequence(1)).Times(iterations / 2 - 1);
  EXPECT_CALL(*mockController, computeControl(testing::_, testing::_))
      .Times(iterations / 2)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
  EXPECT_CALL(*mockController, getControlSeq()).Times(iterations / 2).WillRepeatedly(testing::Return(control_seq));
  MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
  EXPECT_CALL(*mockController, getTargetStateSeq()).Times(iterations / 2).WillRepeatedly(testing::Return(state_seq));
  EXPECT_CALL(*mockController, computeFeedback(testing::_))
      .Times(iterations / 2)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::TEMPLATED_FEEDBACK_STATE feedback;
  EXPECT_CALL(*mockController, getFeedbackState()).Times(iterations / 2).WillRepeatedly(testing::Return(feedback));
  EXPECT_CALL(*mockController, computeFeedbackPropagatedStateSeq()).Times(iterations / 2);
  EXPECT_CALL(*mockController, calculateSampledStateTrajectories()).Times(0);

  std::atomic<bool> is_alive(true);
  std::thread optimizer(&MockTestPlant::runControlLoop, plant.get(), &is_alive);

  std::chrono::steady_clock::time_point loop_start = std::chrono::steady_clock::now();
  std::chrono::duration<double, std::milli> loop_duration = std::chrono::steady_clock::now() - loop_start;
  // counter is number of dts
  for (int counter = 0; loop_duration.count() < test_duration * 1e3; counter++)
  {
    // wait until the correct hz has passed to tick the time
    // state at 100 Hz
    while (loop_duration.count() < (test_duration / 100) * 1e3 * counter)
    {
      usleep(50);
      loop_duration = std::chrono::steady_clock::now() - loop_start;
    }
    if (counter / 5 > iterations / 2)
    {  // this forces it to block
      plant->incrementTime(0.01);
    }
  }
  is_alive.store(false);
  optimizer.join();

  // check all the things
  EXPECT_EQ(plant->checkStatus(), 1);
  EXPECT_EQ(plant->getStateTraj(), state_seq);
  EXPECT_EQ(plant->getControlTraj(), control_seq);
  EXPECT_EQ(plant->getFeedbackState(), feedback);

  // check last pose update
  EXPECT_NE(plant->getLastUsedPoseUpdateTime(), 0.0);
  EXPECT_EQ(plant->getNumIter(), iterations / 2);
  EXPECT_EQ(plant->getLastOptimizationStride(), 1);

  double wait_ms = wait_s * 1e3;
  EXPECT_THAT(plant->getOptimizationDuration(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getOptimizationAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopDuration(),
              testing::AllOf(testing::Ge(wait_ms * 2), testing::Le(wait_ms * 2 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopAvg(), testing::AllOf(testing::Ge(wait_ms * 2), testing::Le(wait_ms * 2 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackDuration(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  // 10 iters of just waiting, 10 iters of waiting for correct time
  double expected_avg_wait = ((wait_ms * 3 * 10) + wait_ms * 10) / 10;
  EXPECT_THAT(plant->getSleepTimeAvg(),
              testing::AllOf(testing::Gt(expected_avg_wait), testing::Le(expected_avg_wait + SMALL_TIME_MS * 4)));
}

TEST_F(BasePlantTest, runControlLoopRegularDelayed)
{
  EXPECT_CALL(*mockController, getDt()).WillRepeatedly(testing::Return(0.025));
  mockController->initFeedback();

  int hz = plant->getHz();
  double test_duration = 1.0;  // in seconds for how long to run the test

  int init_time = 78;
  plant->setLastTime(init_time);
  plant->setUseRealTimeTiming(true);

  // setup mock expected calls
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(0);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(0);
  EXPECT_CALL(*mockController, resetControls()).Times(1);

  double wait_s =
      (1.0 / hz) / 3;  // divide by 3 since wait is evenly split across computeFeedback, computeControl, and waiting

  auto wait_function = [wait_s](const Eigen::Ref<const MockController::state_array>& state,
                                int optimization_stride = 0) { usleep(wait_s * 1e6); };
  int iterations = int(std::round((hz * 1.0) / (test_duration)));  // number of times the method will be called
  // slide control sequence is skipped on the first iteration
  EXPECT_CALL(*mockController, slideControlSequence(2)).Times(iterations / 2 - 1);
  EXPECT_CALL(*mockController, computeControl(testing::_, testing::_))
      .Times(iterations / 2)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
  EXPECT_CALL(*mockController, getControlSeq()).Times(iterations / 2).WillRepeatedly(testing::Return(control_seq));
  MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
  EXPECT_CALL(*mockController, getTargetStateSeq()).Times(iterations / 2).WillRepeatedly(testing::Return(state_seq));
  EXPECT_CALL(*mockController, computeFeedback(testing::_))
      .Times(iterations / 2)
      .WillRepeatedly(testing::Invoke(wait_function));
  MockController::TEMPLATED_FEEDBACK_STATE feedback;
  EXPECT_CALL(*mockController, getFeedbackState()).Times(iterations / 2).WillRepeatedly(testing::Return(feedback));
  EXPECT_CALL(*mockController, computeFeedbackPropagatedStateSeq()).Times(iterations / 2);
  EXPECT_CALL(*mockController, calculateSampledStateTrajectories()).Times(0);

  std::atomic<bool> is_alive(true);
  std::thread optimizer(&MockTestPlant::runControlLoop, plant.get(), &is_alive);

  std::chrono::steady_clock::time_point loop_start = std::chrono::steady_clock::now();
  std::chrono::duration<double, std::milli> loop_duration = std::chrono::steady_clock::now() - loop_start;
  // counter is number of dts
  for (int counter = 0; loop_duration.count() < test_duration * 1e3; counter++)
  {
    // wait until the correct hz has passed to tick the time
    // state at 100 Hz
    while (loop_duration.count() < (test_duration / 100) * 1e3 * counter)
    {
      usleep(50);
      loop_duration = std::chrono::steady_clock::now() - loop_start;
    }
    if (counter / 5 > iterations / 2)
    {  // this forces it to block
      plant->incrementTime(0.01);
    }
  }
  is_alive.store(false);
  optimizer.join();

  // check all the things
  EXPECT_EQ(plant->checkStatus(), 1);
  EXPECT_EQ(plant->getStateTraj(), state_seq);
  EXPECT_EQ(plant->getControlTraj(), control_seq);
  EXPECT_EQ(plant->getFeedbackState(), feedback);

  // check last pose update
  EXPECT_NE(plant->getLastUsedPoseUpdateTime(), 0.0);
  EXPECT_EQ(plant->getNumIter(), iterations / 2);
  EXPECT_EQ(plant->getLastOptimizationStride(), 2);

  double wait_ms = wait_s * 1e3;
  EXPECT_THAT(plant->getOptimizationDuration(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getOptimizationAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopDuration(),
              testing::AllOf(testing::Ge(wait_ms * 2), testing::Le(wait_ms * 2 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getLoopAvg(), testing::AllOf(testing::Ge(wait_ms * 2), testing::Le(wait_ms * 2 + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackDuration(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  EXPECT_THAT(plant->getFeedbackAvg(), testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + SMALL_TIME_MS)));
  // 10 iters of just waiting, 10 iters of waiting for correct time
  double expected_avg_wait = ((wait_ms * 3 * 10) + wait_ms * 10) / 10;
  EXPECT_THAT(plant->getSleepTimeAvg(),
              testing::AllOf(testing::Gt(expected_avg_wait), testing::Le(expected_avg_wait + SMALL_TIME_MS * 4)));
}
