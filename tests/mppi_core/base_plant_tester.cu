#include "hip/hip_runtime.h"
//
// Created by jgibson37 on 2/24/20.
//


#include <gtest/gtest.h>
#include <gmock/gmock.h>
#include <mppi/utils/test_helper.h>
#include <random>
#include <algorithm>
#include <numeric>
#include <boost/thread.hpp>

#include <mppi/core/base_plant.hpp>
#include <mppi/instantiations/cartpole_mppi/cartpole_mppi.cuh>
#include <mppi_test/mock_classes/mock_dynamics.h>
#include <mppi_test/mock_classes/mock_controller.h>
#include <mppi_test/mock_classes/mock_costs.h>

template <class CONTROLLER_T>
class TestPlant : public BasePlant<CONTROLLER_T> {
public:
  double time_ = 0.0;

  double avgDurationMs_ = 0;
  double avgTickDuration_ = 0;
  double avgSleepTime_ = 0;

  using c_array = typename CONTROLLER_T::control_array;
  using c_traj = typename CONTROLLER_T::control_trajectory;

  using s_array = typename CONTROLLER_T::state_array;
  using s_traj = typename CONTROLLER_T::state_trajectory;
  using K_mat = typename CONTROLLER_T::feedback_gain_trajectory;

  using DYN_T = typename CONTROLLER_T::TEMPLATED_DYNAMICS;
  using DYN_PARAMS_T = typename DYN_T::DYN_PARAMS_T;
  using COST_T = typename CONTROLLER_T::TEMPLATED_COSTS;
  using COST_PARAMS_T = typename COST_T::COST_PARAMS_T;
  double timestamp_;
  double loop_speed_;

  TestPlant(std::shared_ptr<MockController> controller, int hz = 20, int opt_stride=1)
        : BasePlant<CONTROLLER_T>(controller,hz,opt_stride) {}


  void pubControl(const c_array& u) override {

  }

  void incrementTime() {
    time_ += 0.05;
  }

  void setTimingInfo(double avg_duration_ms, double avg_tick_duration, double avg_sleep_time) override {
    avgDurationMs_ = avg_duration_ms;
    avgTickDuration_ = avg_tick_duration;
    avgSleepTime_ = avg_sleep_time;
  }

  int checkStatus() override {
    return 1;
  }

  double getCurrentTime() {
    return time_;
  }

  // accessors for protected members
  int getNumIter() {return this->num_iter_;}
  double getLastUsedPoseUpdateTime() {return this->last_used_pose_update_time_;}
  int getStatus() {return this->status_;}
  bool getDebugMode() {return this->debug_mode_;}
  double getOptimizationDuration() {return this->optimization_duration_;}
  double getOptimizationAvg() {return this->avg_optimize_time_ms_;}
  double getLoopDuration() {return this->optimize_loop_duration_;}
  double getLoopAvg() {return this->avg_loop_time_ms_;}
  double getFeedbackDuration() {return this->feedback_duration_;}
  double getFeedbackAvg() {return this->avg_feedback_time_ms_;}
  void setLastTime(double time) {this->last_used_pose_update_time_ = time;}
};

typedef TestPlant<MockController> MockTestPlant;

TEST(BasePlant, Constructor) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant plant(mockController);

  EXPECT_EQ(plant.controller_, mockController);
  EXPECT_EQ(plant.getHz(), 20);
  EXPECT_EQ(plant.getTargetOptimizationStride(), 1);
  EXPECT_EQ(plant.getNumIter(), 0);
  EXPECT_EQ(plant.getLastUsedPoseUpdateTime(), -1);
  EXPECT_EQ(plant.getStatus(), 1);

  EXPECT_EQ(plant.hasNewCostParams(), false);
  EXPECT_EQ(plant.hasNewDynamicsParams(), false);
  EXPECT_EQ(plant.hasNewModel(), false);
  EXPECT_EQ(plant.hasNewCostmap(), false);
  EXPECT_EQ(plant.hasNewObstacles(), false);
}

TEST(BasePlant, getAndSetState) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant plant(mockController);
  // check initial state is zerod

  MockController::state_array state = plant.getState();
  for(int i = 0; i < 1; i++) {
    EXPECT_EQ(state(i), 0.0);
  }

  MockController::state_array new_state;
  for(int i = 0; i < 1; i++) {
    new_state(i) = i;
  }
  plant.setState(new_state);
  state = plant.getState();
  for(int i = 0; i < 1; i++) {
    EXPECT_EQ(state(i), i);
  }
}

TEST(BasePlant, getSetOptimizationStride) {

  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant plant(mockController);
  int optimization_stride = plant.getTargetOptimizationStride();

  EXPECT_EQ(optimization_stride, 1);

  plant.setTargetOptimizationStride(5);
  optimization_stride = plant.getTargetOptimizationStride();

  EXPECT_EQ(optimization_stride, 5);
}

TEST(BasePlant, getSetDynamicsParams) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant plant(mockController);

  EXPECT_EQ(plant.hasNewDynamicsParams(), false);
  MockTestPlant::DYN_PARAMS_T params;

  params.test = 3;

  plant.setDynamicsParams(params);
  EXPECT_EQ(plant.hasNewDynamicsParams(), true);

  MockTestPlant::DYN_PARAMS_T new_params = plant.getNewDynamicsParams();
  EXPECT_EQ(plant.hasNewDynamicsParams(), false);
  EXPECT_EQ(new_params.test, params.test);
}

TEST(BasePlant, getSetCostParams) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant plant(mockController);
  EXPECT_EQ(plant.hasNewCostParams(), false);

  MockTestPlant::COST_PARAMS_T params;
  params.test = 100;

  plant.setCostParams(params);
  EXPECT_EQ(plant.hasNewCostParams(), true);

  auto new_params = plant.getNewCostParams();
  EXPECT_EQ(plant.hasNewCostParams(), false);
  EXPECT_EQ(params.test, new_params.test);
}

TEST(BasePlant, updateParametersAllFalse) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant testPlant(mockController);

  EXPECT_CALL(mockCost, getDebugDisplayEnabled()).Times(0);
  EXPECT_CALL(mockCost, getDebugDisplay(testing::_)).Times(0);
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(0);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(0);
  EXPECT_CALL(mockCost, updateCostmap(testing::_, testing::_)).Times(0);

  MockDynamics::state_array state = MockDynamics::state_array::Zero();
  testPlant.updateParameters(mockController.get(), state);
}


TEST(BasePlant, updateParametersAllTrue) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant testPlant(mockController);

  EXPECT_CALL(mockCost, getDebugDisplayEnabled()).Times(1).WillRepeatedly(testing::Return(true));
  EXPECT_CALL(mockCost, getDebugDisplay(testing::_)).Times(1);
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(1);
  // TODO implement updating costmap
  EXPECT_CALL(mockCost, updateCostmap(testing::_, testing::_)).Times(0);

  testPlant.setDebugMode(true);
  testPlant.setDynamicsParams(MockDynamics::DYN_PARAMS_T());
  testPlant.setCostParams(MockCost::COST_PARAMS_T());

  MockDynamics::state_array state = MockDynamics::state_array::Zero();
  testPlant.updateParameters(mockController.get(), state);
}

TEST(BasePlant, updateStateOutsideTimeTest) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  mockController->setDt(0.05);

  MockTestPlant testPlant(mockController);
  testPlant.setLastTime(0);

  EXPECT_CALL(*mockController, getCurrentControl(testing::_, testing::_)).Times(0);

  MockController::state_array state = MockController::state_array::Zero();
  testPlant.updateState(state, mockController->getDt() * mockController->getNumTimesteps() + 0.01);
  EXPECT_EQ(testPlant.getState(), state);

  testPlant.setLastTime(100);
  testPlant.updateState(state, 99.99);
  EXPECT_EQ(testPlant.getState(), state);
}


TEST(BasePlant, updateStateTest) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  mockController->setDt(0.05);

  MockTestPlant testPlant(mockController);
  testPlant.setLastTime(0);

  MockController::state_array state = MockController::state_array::Zero();
  EXPECT_CALL(*mockController, getCurrentControl(state, mockController->getDt())).Times(1);
  testPlant.updateState(state, mockController->getDt());
  EXPECT_EQ(testPlant.getState(), state);

  //EXPECT_CALL(*mockController, getCurrentControl(state, mockController->getDt()+100)).Times(1);
  //testPlant.setLastTime(100);
  //testPlant.updateState(state, 100+mockController->getDt());
  //EXPECT_EQ(testPlant.getState(), state);
}

TEST(BasePlant, runControlIterationStoppedTest) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant testPlant(mockController);

  EXPECT_CALL(*mockController, slideControlSequence(testing::_)).Times(0);
  EXPECT_CALL(*mockController, computeControl(testing::_)).Times(0);

  std::atomic<bool> is_alive(false);
  testPlant.runControlIteration(mockController.get(), &is_alive);
}

// TODO speed up to make tests run faster
TEST(BasePlant, runControlIterationDebugFalseNoFeedbackTest) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant testPlant(mockController);

  for(int i = 0; i < 2; i++) {
    double wait_ms = 50*i;

    auto wait_function = [wait_ms]() {
      usleep(wait_ms*1e3);
    };

    int expect_opt_stride = i > 0 ? 1 : 0;

    EXPECT_CALL(*mockController, slideControlSequence(expect_opt_stride)).Times(i > 0 ? 1 : 0);
    EXPECT_CALL(*mockController, computeControl(testing::_)).Times(1).WillRepeatedly(testing::Invoke(wait_function));
    MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
    EXPECT_CALL(*mockController, getControlSeq()).Times(1).WillRepeatedly(testing::Return(control_seq));
    MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
    EXPECT_CALL(*mockController, getStateSeq()).Times(1).WillRepeatedly(testing::Return(state_seq));

    EXPECT_CALL(*mockController, computeFeedbackGains(testing::_)).Times(0);
    EXPECT_CALL(*mockController, getFeedbackGains()).Times(0);

    EXPECT_EQ(testPlant.getDebugMode(), false);

    std::atomic<bool> is_alive(true);
    testPlant.runControlIteration(mockController.get(), &is_alive);
    testPlant.incrementTime();

    EXPECT_EQ(testPlant.checkStatus(), 1);
    EXPECT_EQ(testPlant.getStateTraj(), state_seq);
    EXPECT_EQ(testPlant.getControlTraj(), control_seq);
    MockController::feedback_gain_trajectory feedback = testPlant.getFeedbackGains();
    MockController::state_array state = MockController::state_array::Ones();
    for(int j = 0; j < 100; j++) {
      // TODO check that feedback is correct
      //auto result = feedback[i] * state;
      //float sum = feedback[i] * state;
      //EXPECT_FLOAT_EQ(result(0), 0);
    }

    // check last pose update
    EXPECT_FLOAT_EQ(testPlant.getLastUsedPoseUpdateTime(), 0.05*i);
    EXPECT_EQ(testPlant.getNumIter(), i+1);
    EXPECT_EQ(testPlant.getLastOptimizationStride(), expect_opt_stride);

    double small_time_ms = 2; // how long we should expect a non delayed call to take
    EXPECT_THAT(testPlant.getOptimizationDuration(),
            testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
    EXPECT_LT(testPlant.getOptimizationAvg(), wait_ms + small_time_ms);
    EXPECT_THAT(testPlant.getLoopDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
    EXPECT_LT(testPlant.getLoopAvg(), wait_ms + small_time_ms);
    EXPECT_LE(testPlant.getFeedbackDuration(), small_time_ms);
    EXPECT_LE(testPlant.getFeedbackAvg(), small_time_ms);
  }

}

TEST(BasePlant, runControlIterationDebugFalseFeedbackTest) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  mockController->setFeedbackController(true);
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant testPlant(mockController);

  for(int i = 0; i < 10; i++) {
    double wait_ms = 50*i;

    auto wait_function = [wait_ms]() {
      usleep(wait_ms*1e3);
    };

    int expect_opt_stride = i > 0 ? 1 : 0;

    EXPECT_CALL(*mockController, slideControlSequence(expect_opt_stride)).Times(i > 0 ? 1 : 0);
    EXPECT_CALL(*mockController, computeControl(testing::_)).Times(1).WillRepeatedly(testing::Invoke(wait_function));
    MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
    EXPECT_CALL(*mockController, getControlSeq()).Times(1).WillRepeatedly(testing::Return(control_seq));
    MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
    EXPECT_CALL(*mockController, getStateSeq()).Times(1).WillRepeatedly(testing::Return(state_seq));

    EXPECT_CALL(*mockController, computeFeedbackGains(testing::_)).Times(1).WillRepeatedly(testing::Invoke(wait_function));
    MockController::feedback_gain_trajectory feedback;
    EXPECT_CALL(*mockController, getFeedbackGains()).Times(1).WillRepeatedly(testing::Return(feedback));

    EXPECT_EQ(testPlant.getDebugMode(), false);

    std::atomic<bool> is_alive(true);
    testPlant.runControlIteration(mockController.get(), &is_alive);
    testPlant.incrementTime();

    EXPECT_EQ(testPlant.checkStatus(), 1);
    EXPECT_EQ(testPlant.getStateTraj(), state_seq);
    EXPECT_EQ(testPlant.getControlTraj(), control_seq);
    EXPECT_EQ(testPlant.getFeedbackGains(), feedback);

    // check last pose update
    EXPECT_FLOAT_EQ(testPlant.getLastUsedPoseUpdateTime(), 0.05*i);
    EXPECT_EQ(testPlant.getNumIter(), i+1);
    EXPECT_EQ(testPlant.getLastOptimizationStride(), expect_opt_stride);

    double small_time_ms = 10; // how long we should expect a non delayed call to take
    EXPECT_THAT(testPlant.getOptimizationDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
    EXPECT_LT(testPlant.getOptimizationAvg(), wait_ms + small_time_ms);
    EXPECT_THAT(testPlant.getFeedbackDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
    // TODO should be range as well
    EXPECT_LT(testPlant.getFeedbackAvg(), wait_ms + small_time_ms);
    EXPECT_THAT(testPlant.getLoopDuration(),
                testing::AllOf(testing::Ge(wait_ms*2), testing::Le((wait_ms + small_time_ms)*2)));
    EXPECT_LT(testPlant.getLoopAvg(), (wait_ms + small_time_ms)*2);
  }

}

TEST(BasePlant, runControlIterationDebugFalseFeedbackAvgTest) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  mockController->setFeedbackController(true);
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant testPlant(mockController);

  for(int i = 0; i < 10; i++) {
    double wait_ms = 50;

    auto wait_function = [wait_ms]() {
      usleep(wait_ms*1e3);
    };

    int expect_opt_stride = i > 0 ? 1 : 0;

    EXPECT_CALL(*mockController, slideControlSequence(expect_opt_stride)).Times(i > 0 ? 1 : 0);
    EXPECT_CALL(*mockController, computeControl(testing::_)).Times(1).WillRepeatedly(testing::Invoke(wait_function));
    MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
    EXPECT_CALL(*mockController, getControlSeq()).Times(1).WillRepeatedly(testing::Return(control_seq));
    MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
    EXPECT_CALL(*mockController, getStateSeq()).Times(1).WillRepeatedly(testing::Return(state_seq));

    EXPECT_CALL(*mockController, computeFeedbackGains(testing::_)).Times(1).WillRepeatedly(testing::Invoke(wait_function));
    MockController::feedback_gain_trajectory feedback;
    EXPECT_CALL(*mockController, getFeedbackGains()).Times(1).WillRepeatedly(testing::Return(feedback));

    EXPECT_EQ(testPlant.getDebugMode(), false);

    std::atomic<bool> is_alive(true);
    testPlant.runControlIteration(mockController.get(), &is_alive);
    testPlant.incrementTime();

    EXPECT_EQ(testPlant.checkStatus(), 1);
    EXPECT_EQ(testPlant.getStateTraj(), state_seq);
    EXPECT_EQ(testPlant.getControlTraj(), control_seq);
    EXPECT_EQ(testPlant.getFeedbackGains(), feedback);

    // check last pose update
    EXPECT_FLOAT_EQ(testPlant.getLastUsedPoseUpdateTime(), 0.05*i);
    EXPECT_EQ(testPlant.getNumIter(), i+1);
    EXPECT_EQ(testPlant.getLastOptimizationStride(), expect_opt_stride);

    double small_time_ms = 10; // how long we should expect a non delayed call to take
    EXPECT_THAT(testPlant.getOptimizationDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
    EXPECT_THAT(testPlant.getOptimizationAvg(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
    EXPECT_THAT(testPlant.getLoopDuration(),
                testing::AllOf(testing::Ge(wait_ms*2), testing::Le((wait_ms + small_time_ms)*2)));
    EXPECT_THAT(testPlant.getLoopAvg(),
                testing::AllOf(testing::Ge((wait_ms)*2), testing::Le((wait_ms + small_time_ms)*2)));
    EXPECT_THAT(testPlant.getFeedbackDuration(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
    EXPECT_THAT(testPlant.getFeedbackAvg(),
                testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
  }

}

TEST(BasePlant, runControlLoop) {
  std::shared_ptr<MockController> mockController = std::make_shared<MockController>();
  mockController->setFeedbackController(true);
  MockCost mockCost;
  MockDynamics mockDynamics;
  mockController->cost_ = &mockCost;
  mockController->model_ = &mockDynamics;

  MockTestPlant testPlant(mockController);
  int hz = testPlant.getHz();
  double time = 1.0; // in seconds

  // setup mock expected calls
  EXPECT_CALL(mockCost, getDebugDisplayEnabled()).Times(0);
  EXPECT_CALL(mockCost, setParams(testing::_)).Times(0);
  EXPECT_CALL(mockDynamics, setParams(testing::_)).Times(0);
  EXPECT_CALL(*mockController, resetControls()).Times(1);

  double wait_s = (1.0/hz)/2; // divide by 2 since wait is evenly split across computeFeedbackGains and computeControl

  auto wait_function = [wait_s]() {
    usleep(wait_s*1e6);
  };
  int iterations = int(std::round((hz*1.0) / (time * 1.0))); // number of times the method will be called
  EXPECT_CALL(*mockController, slideControlSequence(1)).Times(iterations/2);
  EXPECT_CALL(*mockController, computeControl(testing::_)).Times(iterations/2).WillRepeatedly(testing::Invoke(wait_function));
  MockController::control_trajectory control_seq = MockController::control_trajectory::Zero();
  EXPECT_CALL(*mockController, getControlSeq()).Times(iterations/2).WillRepeatedly(testing::Return(control_seq));
  MockController::state_trajectory state_seq = MockController::state_trajectory::Zero();
  EXPECT_CALL(*mockController, getStateSeq()).Times(iterations/2).WillRepeatedly(testing::Return(state_seq));
  EXPECT_CALL(*mockController, computeFeedbackGains(testing::_)).Times(iterations/2).WillRepeatedly(testing::Invoke(wait_function));
  MockController::feedback_gain_trajectory feedback;
  EXPECT_CALL(*mockController, getFeedbackGains()).Times(iterations/2).WillRepeatedly(testing::Return(feedback));

  std::atomic<bool> is_alive(true);
  boost::thread optimizer;
  optimizer = boost::thread(boost::bind(&MockTestPlant::runControlLoop, &testPlant, mockController.get(), &is_alive));

  std::chrono::steady_clock::time_point loop_start = std::chrono::steady_clock::now();
  std::chrono::duration<double, std::milli> loop_duration = std::chrono::steady_clock::now() - loop_start;
  int counter = 0;
  while(loop_duration.count() < time*1e3) {
    counter++;
    while(loop_duration.count() < (time/hz)*1e3*counter) {
      usleep(50);
      loop_duration = std::chrono::steady_clock::now() - loop_start;
    }
    if(counter >= iterations / 2) { // this forces it to block
      testPlant.incrementTime();
    }
  }
  is_alive.store(false);
  optimizer.join();

  // check all the things
  EXPECT_EQ(testPlant.checkStatus(), 1);

  EXPECT_EQ(testPlant.checkStatus(), 1);
  EXPECT_EQ(testPlant.getStateTraj(), state_seq);
  EXPECT_EQ(testPlant.getControlTraj(), control_seq);
  EXPECT_EQ(testPlant.getFeedbackGains(), feedback);

  // check last pose update
  EXPECT_NE(testPlant.getLastUsedPoseUpdateTime(), 0.0);
  EXPECT_EQ(testPlant.getNumIter(), iterations/2);
  EXPECT_EQ(testPlant.getLastOptimizationStride(), 1);

  double small_time_ms = 10; // how long we should expect a non delayed call to take
  double wait_ms = wait_s*1e3;
  EXPECT_THAT(testPlant.getOptimizationDuration(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
  EXPECT_THAT(testPlant.getOptimizationAvg(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
  EXPECT_THAT(testPlant.getLoopDuration(),
              testing::AllOf(testing::Ge(wait_ms*2), testing::Le((wait_ms + small_time_ms)*2)));
  EXPECT_THAT(testPlant.getLoopAvg(),
              testing::AllOf(testing::Ge((wait_ms)*2), testing::Le((wait_ms + small_time_ms)*2)));
  EXPECT_THAT(testPlant.getFeedbackDuration(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
  EXPECT_THAT(testPlant.getFeedbackAvg(),
              testing::AllOf(testing::Ge(wait_ms), testing::Le(wait_ms + small_time_ms)));
}
