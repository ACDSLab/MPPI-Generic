#include <gtest/gtest.h>
#include <mppi/controllers/R-MPPI/robust_mppi_controller.cuh>
#include <mppi/core/rmppi_kernel_test.cuh>
#include <mppi/cost_functions/double_integrator/double_integrator_circle_cost.cuh>
#include <mppi/dynamics/double_integrator/di_dynamics.cuh>
#include <mppi/feedback_controllers/CCM/ccm.h>
#include <mppi/utils/test_helper.h>
#include <cnpy.h>

const int NUM_TIMESTEPS = 50;
const int NUM_ROLLOUTS_CONST = 1024;

// Might be simpler to create a new Controller CLass from RMPPI
template<class DYN_T = DoubleIntegratorDynamics, class COST_T = DoubleIntegratorCircleCost,
         int MAX_TIMESTEPS = NUM_TIMESTEPS, int NUM_ROLLOUTS = NUM_ROLLOUTS_CONST,
         int B_X = 64, int B_Y = 1, int S = 1>
class RMPPICCMDoubleIntegratorController : public RobustMPPIController<DYN_T, COST_T,
    MAX_TIMESTEPS, NUM_ROLLOUTS, B_X, B_Y, S> {

protected:
  std::mt19937 rng_gen_;
  std::normal_distribution<float> control_dist_;
  ccm::LinearCCM<DYN_T> CCM_feedback_controller_;
public:
  using Q_MAT = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                              NUM_ROLLOUTS, B_X, B_Y,
                                              S>::StateCostWeight;

  using Qf_MAT = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                               NUM_ROLLOUTS, B_X, B_Y,
                                               S>::Hessian;

  using R_MAT = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                              NUM_ROLLOUTS, B_X, B_Y,
                                              S>::ControlCostWeight;

  using control_array = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                                      NUM_ROLLOUTS, B_X, B_Y,
                                                      S>::control_array;

  using control_trajectory = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                                      NUM_ROLLOUTS, B_X, B_Y,
                                                      S>::control_trajectory;

  using state_array = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                                      NUM_ROLLOUTS, B_X, B_Y,
                                                      S>::state_array;

  // Constructor... Yeah It ain't pretty
  RMPPICCMDoubleIntegratorController(DYN_T* model, COST_T* cost, float dt, float lambda,
      float alpha, float value_function_threshold,
      const Eigen::Ref<const control_array>& control_std_dev,
      int num_timesteps = MAX_TIMESTEPS,
      const Eigen::Ref<const control_trajectory>& init_control_traj = control_trajectory::Zero(),
      int num_candidate_nominal_states = 9, int optimization_stride = 1,
      hipStream_t stream = nullptr) : RobustMPPIController<DYN_T, COST_T,
      MAX_TIMESTEPS, NUM_ROLLOUTS, 64, 1, 1>(model, cost, dt, 1, lambda, alpha,
      value_function_threshold, Q_MAT::Zero(), Qf_MAT::Zero(), R_MAT::Zero(),
      control_std_dev, num_timesteps, init_control_traj,
      num_candidate_nominal_states, optimization_stride, stream) {

    control_dist_ = std::normal_distribution<float>(0, 1);
    CCM_feedback_controller_ = ccm::LinearCCM<DYN_T>(model);
    // Q_MAT M_new = Q_MAT::Identity();
    // CCM_feedback_controller_.setM(M_new);
  }

  void ptrToVec(const float* input, int num, std::vector<float>& output) {
    output.assign(input, input + num);
    if (output.size() != num) {
      output.assign(num, 0.0);
      for(int i = 0; i < num; i++) {
        output[i] = input[i];
      }
    }

  }

  void computeControl(const Eigen::Ref<const state_array>& state, int optimization_stride=1) override {
    std::cout << "Candidate chosen: " << this->best_index_ << std::endl;
    // Rewrite computeControl using the CCM Rollout Kernel
    int c_dim = DYN_T::CONTROL_DIM;
    int s_dim = DYN_T::STATE_DIM;
    int single_control_traj_size = this->num_timesteps_ * c_dim;
    int multi_control_traj_size = NUM_ROLLOUTS * single_control_traj_size;

    // Handy dandy pointers to nominal data
    float * trajectory_costs_nominal_d = this->trajectory_costs_d_ + NUM_ROLLOUTS;
    float * initial_state_nominal_d = this->initial_state_d_ + s_dim;
    float * control_noise_nominal_d = this->control_noise_d_ + multi_control_traj_size;
    float * control_nominal_d = this->control_d_ + single_control_traj_size;
    for (int opt_iter = 0; opt_iter < this->num_iters_; opt_iter++) {
      // Create noise for trajectories
      std::vector<float> control_noise_vec(multi_control_traj_size * 2, 0);
      for(int i = 0; i < multi_control_traj_size; i++) {
        control_noise_vec[i] = control_dist_(rng_gen_);
        control_noise_vec[multi_control_traj_size + i] = control_noise_vec[i];
      }
      std::vector<float> x_init_act_vec, x_init_nom_vec, u_traj_vec;
      std::vector<float> control_std_dev_vec;
      ptrToVec(state.data(), s_dim, x_init_act_vec);
      ptrToVec(this->nominal_state_.data(), s_dim, x_init_nom_vec);
      ptrToVec(this->nominal_control_trajectory_.data(),
               single_control_traj_size, u_traj_vec);
      ptrToVec(this->control_std_dev_.data(), c_dim, control_std_dev_vec);


      // Launch rollout kernel using CCM
      // TODO pass in alpha
      std::array<float, NUM_ROLLOUTS> costs_act_CPU, costs_nom_CPU;
      launchRMPPIRolloutKernelCCMCPU<DYN_T, COST_T, NUM_ROLLOUTS>(this->model_,
        this->cost_, &CCM_feedback_controller_, this->dt_,
        this->num_timesteps_, optimization_stride, this->lambda_, this->alpha_,
        this->value_function_threshold_, x_init_nom_vec, x_init_act_vec,
        control_std_dev_vec, u_traj_vec, control_noise_vec,
        costs_act_CPU, costs_nom_CPU);

      for(int i = 0; i < NUM_ROLLOUTS; i++) {
        this->trajectory_costs_(i) = costs_act_CPU[i];
        this->trajectory_costs_nominal_(i) = costs_nom_CPU[i];
      }
      // Control noise should be modified to contain u + noise
      this->baseline_ = mppi_common::computeBaselineCost(
          this->trajectory_costs_.data(), NUM_ROLLOUTS);
      this->baseline_nominal_ = mppi_common::computeBaselineCost(
          this->trajectory_costs_nominal_.data(), NUM_ROLLOUTS);

      // Copy data over to GPU
      HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_d_,
                                  this->trajectory_costs_.data(),
                                  NUM_ROLLOUTS * sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_));

      HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nominal_d,
                                  this->trajectory_costs_nominal_.data(),
                                  NUM_ROLLOUTS * sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_));

      HANDLE_ERROR(hipMemcpyAsync(this->control_noise_d_,
                                  control_noise_vec.data(),
                                  multi_control_traj_size * sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_));

      HANDLE_ERROR(hipMemcpyAsync(control_noise_nominal_d,
                                  control_noise_vec.data() + multi_control_traj_size,
                                  multi_control_traj_size * sizeof(float),
                                  hipMemcpyHostToDevice, this->stream_));

      // After rollout kernel, control_d_ and nominal_control_d are written to
      // and not read from so there is nothing to copy to them
      // HANDLE_ERROR(hipMemcpyAsync(this->control_d_,
      //                              this->nominal_control_trajectory_.data(),
      //                              single_control_traj_size * sizeof(float),
      //                              hipMemcpyHostToDevice, this->stream_));

      // // TODO Not done in RMPPI RolloutKernel but I think it should be
      // HANDLE_ERROR(hipMemcpyAsync(control_nominal_d,
      //                              this->nominal_control_trajectory_.data(),
      //                              single_control_traj_size * sizeof(float),
      //                              hipMemcpyHostToDevice, this->stream_));

      HANDLE_ERROR(hipStreamSynchronize(this->stream_));

      // In this case this->gamma = 1 / lambda
      mppi_common::launchNormExpKernel(NUM_ROLLOUTS, B_X,
                                      this->trajectory_costs_d_, 1.0 / this->lambda_,
                                      this->baseline_, this->stream_);
      mppi_common::launchNormExpKernel(NUM_ROLLOUTS, B_X,
                                      trajectory_costs_nominal_d, 1.0 / this->lambda_,
                                      this->baseline_nominal_, this->stream_);

      HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(),
                                  this->trajectory_costs_d_,
                                  NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost,
                                  this->stream_));
      HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_nominal_.data(),
                                  trajectory_costs_nominal_d,
                                  NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost,
                                  this->stream_));
      HANDLE_ERROR(hipStreamSynchronize(this->stream_));

      this->normalizer_ = mppi_common::computeNormalizer(
          this->trajectory_costs_.data(), NUM_ROLLOUTS);
      this->normalizer_nominal_ = mppi_common::computeNormalizer(
          this->trajectory_costs_nominal_.data(), NUM_ROLLOUTS);


      mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, B_X>(
              this->trajectory_costs_d_, this->control_noise_d_, this->control_d_,
              this->normalizer_, this->num_timesteps_, this->stream_);
      mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, B_X>(
              trajectory_costs_nominal_d,
              control_noise_nominal_d, control_nominal_d,
              this->normalizer_nominal_, this->num_timesteps_, this->stream_);
      // Transfer the new control to the host
      HANDLE_ERROR( hipMemcpyAsync(this->control_.data(), this->control_d_,
                                    sizeof(float) * single_control_traj_size,
                                    hipMemcpyDeviceToHost, this->stream_));
      HANDLE_ERROR( hipMemcpyAsync(this->nominal_control_trajectory_.data(),
                                    control_nominal_d,
                                    sizeof(float) * single_control_traj_size,
                                    hipMemcpyDeviceToHost, this->stream_));
      hipStreamSynchronize(this->stream_);
    }
  }

  void computeNominalFeedbackGains(const Eigen::Ref<const state_array>& state) override {}

  control_array getCCMFeedbackGains(const Eigen::Ref<const state_array>& x_act,
                                 const Eigen::Ref<const state_array>& x_nom,
                                 const Eigen::Ref<const control_array>& u_nom) {
    control_array fb_u = CCM_feedback_controller_.u_feedback(x_act, x_nom, u_nom);
    std::cout << "Act: " << x_act.transpose() << std::endl;
    std::cout << "nom: " << x_nom.transpose() << std::endl;
    std::cout << "U: " << u_nom.transpose() << std::endl;
    std::cout << "Feedback: " << fb_u.transpose() << std::endl;
    return fb_u;

  }
};

bool tubeFailure(float *s) {
  float inner_path_radius2 = 1.675*1.675;
  float outer_path_radius2 = 2.325*2.325;
  float radial_position = s[0]*s[0] + s[1]*s[1];
  if ((radial_position < inner_path_radius2) || (radial_position > outer_path_radius2)) {
    return true;
  } else {
    return false;
  }
}

TEST(CCMTest, CCMFeedbackTest) {
  using DYN = DoubleIntegratorDynamics;
  using COST = DoubleIntegratorCircleCost;
  DYN model(100);
  ccm::LinearCCM<DYN> fb_controller(&model);
  float dt = 0.02;
  int mission_length = int(10/dt);

  DYN::state_array x, x_nom, x_dot;
  x << 4, 0, 0, 1;
  x_nom << -3, 2, 0, 0;
  DYN::control_array current_control;

  float two_percent_settle_time = -1;

  for (int t = 0; t < mission_length; t++) {
    current_control = fb_controller.u_feedback(x, x_nom, DYN::control_array::Zero());
    model.computeDynamics(x, current_control, x_dot);
    model.updateState(x, x_dot, dt);

    DYN::state_array abs_diff = x - x_nom;
    for (int i = 0; i < DYN::STATE_DIM; i++) {
      if (x_nom(i) >= 1) {
        abs_diff(i) /= x_nom(i);
      }
    }
    abs_diff = abs_diff.cwiseAbs();
    if (abs_diff.block<2,1>(0,0).maxCoeff() < 0.02 && two_percent_settle_time < 0) {
      two_percent_settle_time = t * dt;
    }

    if (t % 5 == 0) {
      std::cout << "State at t = " << t * dt << ": " << x.transpose() << std::endl;
    }
  }
  std::cout << "2% settling time is " << two_percent_settle_time << " secs" << std::endl;
}

TEST(CCMTest, RMPPIRolloutKernel) {
  using DYN = DoubleIntegratorDynamics;
  using COST = DoubleIntegratorCircleCost;
  DYN model(100);
  COST cost;
  const int num_timesteps = NUM_TIMESTEPS;
  const int num_rollouts = NUM_ROLLOUTS_CONST;

  using CONTROLLER = RMPPICCMDoubleIntegratorController<DYN, COST,
                                                        num_timesteps,
                                                        num_rollouts>;

  const int state_dim = DYN::STATE_DIM;
  const int control_dim = DYN::CONTROL_DIM;

  float dt = 0.02;
  // int max_iter = 10;
  float lambda = 4;
  float alpha = 0;
  float value_func_threshold = 10;


  int mission_length = int(100 / dt); // 100 seconds

  std::vector<float> actual_trajectory_save(num_timesteps*mission_length*DYN::STATE_DIM);
  std::vector<float> nominal_trajectory_save(num_timesteps*mission_length*DYN::STATE_DIM);
  std::string file_prefix = "/data/bvlahov3/RMPPI_CCM_control_trajectories_CoRL2020/";

  CONTROLLER::control_array control_std_dev = CONTROLLER::control_array::Constant(1.0);
  CONTROLLER::control_trajectory u_traj_eigen = CONTROLLER::control_trajectory::Zero();
  // Set first control to 1 across entire time
  u_traj_eigen.row(0) = CONTROLLER::cost_trajectory::Constant(1.0);

  CONTROLLER rmppi_controller = CONTROLLER(&model, &cost, dt, lambda, alpha,
                                           value_func_threshold,
                                           control_std_dev, num_timesteps,
                                           u_traj_eigen);


  // float x[num_rollouts * state_dim * 2];
  // float x_dot[num_rollouts * state_dim * 2];
  // float u[num_rollouts * control_dim * 2];
  // float du[num_rollouts * control_dim * 2];
  // float sigma_u[control_dim] = {0.5, 0.05}; // variance to sample noise from

  // COST::control_matrix cost_variance = COST::control_matrix::Identity();
  // for(int i = 0; i < control_dim; i++) {
  //   cost_variance(i, i) = sigma_u[i];
  // }
  // float fb_u[num_rollouts * control_dim * state_dim];

  DYN::state_array x_init_act, x_dot;
  x_init_act << 2, 0, 0, 1;
  DYN::state_array x_init_nom;
  x_init_nom << 0, 0, 0.1, 0;

  // rmppi_controller.computeControl(x_init_act);
  DYN::state_array x = x_init_act;

  for(int t = 0; t < mission_length; t++) {
    std::string act_file_name = file_prefix + "robust_large_actual_CCM_t_" +
                                std::to_string(t) + ".npy";
    std::string nom_file_name = file_prefix + "robust_large_nominal_CCM_t_" +
                                std::to_string(t) + ".npy";

    // if (cost.computeStateCost(x) > 1000) {
    //   std::cout << "State Cost is " << cost.computeStateCost(x) << std::endl;
    //   std::cout << "State was " << x.transpose() << std::endl;
    //   FAIL();
    // }

    if (tubeFailure(x.data())) {
      cnpy::npy_save(act_file_name, actual_trajectory_save.data(),
                     {mission_length, num_timesteps, DYN::STATE_DIM},"w");
      cnpy::npy_save(nom_file_name, nominal_trajectory_save.data(),
                     {mission_length, num_timesteps, DYN::STATE_DIM},"w");
      printf("Current Time: %f    ", t * dt);
      model.printState(x.data());
      std::cout << "\tCandidate Free Energies: "
                << rmppi_controller.getCandidateFreeEnergy().transpose() << std::endl;
      std::cout << "Tube failure!!" << std::endl;
      FAIL() << "Visualize the trajectories by running scripts/double_integrator/plot_DI_test_trajectories; "
                "the argument to this python file is the build directory of MPPI-Generic";
    }

    if (t % 2 == 0) {
      printf("Current Time: %5.2f    ", t * dt);
      model.printState(x.data());
      std::cout << "\tCandidate Free Energies: "
                << rmppi_controller.getCandidateFreeEnergy().transpose()
                << std::endl;
    }
    rmppi_controller.updateImportanceSamplingControl(x, 1);
    rmppi_controller.computeControl(x);

    auto nominal_trajectory = rmppi_controller.getStateSeq();

    for (int i = 0; i < num_timesteps; i++) {
      for (int j = 0; j < DYN::STATE_DIM; j++) {
        actual_trajectory_save[t * num_timesteps * DYN::STATE_DIM +
                               i*DYN::STATE_DIM + j] = x(j);
        nominal_trajectory_save[t * num_timesteps * DYN::STATE_DIM +
                                i*DYN::STATE_DIM + j] = nominal_trajectory(j, i);
      }
    }
    DYN::state_array x_nom = rmppi_controller.getStateSeq().col(0);
    DYN::control_array current_control = rmppi_controller.getControlSeq().col(0);

    current_control += rmppi_controller.getCCMFeedbackGains(x, x_nom, current_control);
    model.computeDynamics(x, current_control, x_dot);
    model.updateState(x, x_dot, dt);

    model.computeStateDisturbance(dt, x);
    rmppi_controller.slideControlSequence(1);

    if (t % 50 == 0) {
      cnpy::npy_save(act_file_name, actual_trajectory_save.data(),
                     {mission_length, num_timesteps, DYN::STATE_DIM},"w");
      cnpy::npy_save(nom_file_name, nominal_trajectory_save.data(),
                     {mission_length, num_timesteps, DYN::STATE_DIM},"w");
    }
  }
}
