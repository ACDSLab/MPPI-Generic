#include <gtest/gtest.h>
#include <mppi/dynamics/double_integrator/di_dynamics.cuh>
#include <mppi/cost_functions/double_integrator/double_integrator_circle_cost.cuh>
#include <mppi/core/rmppi_kernel_test.cuh>
#include <mppi/controllers/R-MPPI/robust_mppi_controller.cuh>
#include <mppi/utils/test_helper.h>

const int NUM_TIMESTEPS = 50;
const int NUM_ROLLOUTS_CONST = 64;

// Might be simpler to create a new Controller CLass from RMPPI
template<class DYN_T = DoubleIntegratorDynamics, class COST_T = DoubleIntegratorCircleCost,
         int MAX_TIMESTEPS = NUM_TIMESTEPS, int NUM_ROLLOUTS = NUM_ROLLOUTS_CONST,
         int B_X = 64, int B_Y = 1, int S  = 1>
class RMPPICCMDoubleIntegratorController : public RobustMPPIController<DYN_T, COST_T,
    MAX_TIMESTEPS, NUM_ROLLOUTS, B_X, B_Y, S> {
public:
  using Q_MAT = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                              NUM_ROLLOUTS, B_X, B_Y,
                                              S>::StateCostWeight;

  using Qf_MAT = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                               NUM_ROLLOUTS, B_X, B_Y,
                                               S>::Hessian;

  using R_MAT = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                              NUM_ROLLOUTS, B_X, B_Y,
                                              S>::ControlCostWeight;

  using control_array = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                                      NUM_ROLLOUTS, B_X, B_Y,
                                                      S>::control_array;

  using control_trajectory = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                                      NUM_ROLLOUTS, B_X, B_Y,
                                                      S>::control_trajectory;

  using state_array = typename RobustMPPIController<DYN_T, COST_T, MAX_TIMESTEPS,
                                                      NUM_ROLLOUTS, B_X, B_Y,
                                                      S>::state_array;

  // Constructor... Yeah It ain't pretty
  RMPPICCMDoubleIntegratorController(DYN_T* model, COST_T* cost, float dt, float lambda,
      float alpha, float value_function_threshold,
      const Eigen::Ref<const control_array>& control_std_dev,
      int num_timesteps = MAX_TIMESTEPS,
      const Eigen::Ref<const control_trajectory>& init_control_traj = control_trajectory::Zero(),
      int num_candidate_nominal_states = 9, int optimization_stride = 1,
      hipStream_t stream = nullptr) : RobustMPPIController<DYN_T, COST_T,
      MAX_TIMESTEPS, NUM_ROLLOUTS, 64, 1, 1>(model, cost, dt, 1, lambda, alpha,
      value_function_threshold, Q_MAT::Zero(), Qf_MAT::Zero(), R_MAT::Zero(),
      control_std_dev, num_timesteps, init_control_traj,
      num_candidate_nominal_states, optimization_stride, stream) {
    control_dist_ = std::normal_distribution<float>(0, 1);
  }

  std::vector<float> ptrToVec(float* input, int num) {
    std::vector<float> output;
    output.assign(input, input + num);
    return output;
  }

  void computeControl(const Eigen::Ref<const state_array>& state) override {
    std::cout << "PPPBBBBBBBBBTTTTTTTTTTTTTT" << std::endl;
    // Rewrite computeControl using the CCM Rollout Kernel
    int c_dim = DYN_T::CONTROL_DIM;
    int s_dim = s_dim;
    int single_control_traj_size = this->num_timesteps_ * c_dim;
    int multi_control_traj_size = NUM_ROLLOUTS * single_control_traj_size;

    // Handy dandy pointers to nominal data
    float * trajectory_costs_nominal_d = this->trajectory_costs_d_ + NUM_ROLLOUTS;
    float * initial_state_nominal_d = this->initial_state_d_ + s_dim;
    float * control_noise_nominal_d = this->control_noise_d_ + multi_control_traj_size;
    float * control_nominal_d = this->control_d_ + single_control_traj_size;
    for (int opt_iter = 0; opt_iter < this->num_iters_; opt_iter++) {
      // Create noise for trajectories
      std::vector<float> control_noise_vec(multi_control_traj_size * 2, 0);
      for(int i = 0; i < multi_control_traj_size; i++) {
        control_noise_vec[i] = control_dist_(rng_gen_);
        control_noise_vec[multi_control_traj_size + i] = control_noise_vec[i];
      }
      auto x_init_act_vec = ptrToVec(state.data(), s_dim);
      auto x_init_nom_vec = ptrToVec(this->nominal_state_.data(), s_dim);
      auto u_traj_vec = ptrToVec(this->nominal_control_trajectory_.data(),
                                 single_control_traj_size);
      auto control_std_dev_vec = ptrToVec(this->control_std_dev_.data(), c_dim);


      // Launch rollout kernel using CCM
      // TODO pass in alpha
      std::array<float, NUM_ROLLOUTS> costs_act_CPU, costs_nom_CPU;
      launchRMPPIRolloutKernelCCMCPU<DYN_T, COST_T, NUM_ROLLOUTS>(this->model_,
        this->cost_, this->dt, this->num_timesteps, this->lambda, this->alpha,
        this->value_func_threshold_, x_init_nom_vec, x_init_act_vec,
        control_std_dev_vec, u_traj_vec, control_noise_vec,
        costs_act_CPU, costs_nom_CPU);

      for(int i = 0; i < multi_control_traj_size; i++) {
        this->trajectory_costs_(i) = costs_act_CPU[i];
        this->trajectory_costs_nominal_(i) = costs_nom_CPU[i];
      }
      // Control noise should be modified to contain u + noise
      this->baseline_ = mppi_common::computeBaselineCost(
          this->trajectory_costs_.data(), NUM_ROLLOUTS);
      this->baseline_nominal_ = mppi_common::computeBaselineCost(
          this->trajectory_costs_nominal_.data(), NUM_ROLLOUTS);

    // Copy data over to GPU
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_d_,
                                 this->trajectory_costs_.data(),
                                 NUM_ROLLOUTS * sizeof(float),
                                 hipMemcpyHostToDevice, this->stream_));

    HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nominal_d,
                                 this->trajectory_costs_nominal_.data(),
                                 NUM_ROLLOUTS * sizeof(float),
                                 hipMemcpyHostToDevice, this->stream_));

    HANDLE_ERROR(hipMemcpyAsync(this->control_noise_d_,
                                 control_noise_vec.data(),
                                 multi_control_traj_size * sizeof(float),
                                 hipMemcpyHostToDevice, this->stream_));

    HANDLE_ERROR(hipMemcpyAsync(control_noise_nominal_d,
                                 control_noise_vec.data() + multi_control_traj_size,
                                 multi_control_traj_size * sizeof(float),
                                 hipMemcpyHostToDevice, this->stream_));

    // After rollout kernel, control_d_ and nominal_control_d are written to
    // and not read from so there is nothing to copy to them
    // HANDLE_ERROR(hipMemcpyAsync(this->control_d_,
    //                              this->nominal_control_trajectory_.data(),
    //                              single_control_traj_size * sizeof(float),
    //                              hipMemcpyHostToDevice, this->stream_));

    // // TODO Not done in RMPPI RolloutKernel but I think it should be
    // HANDLE_ERROR(hipMemcpyAsync(control_nominal_d,
    //                              this->nominal_control_trajectory_.data(),
    //                              single_control_traj_size * sizeof(float),
    //                              hipMemcpyHostToDevice, this->stream_));

    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    // In this case this->gamma = 1 / lambda
    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, B_X,
                                     this->trajectory_costs_d_, this->lambda_,
                                     this->baseline_, this->stream_);
    mppi_common::launchNormExpKernel(NUM_ROLLOUTS, B_X,
                                     trajectory_costs_nominal_d, this->lambda_,
                                     this->baseline_nominal_, this->stream_);

    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_.data(),
                                 this->trajectory_costs_d_,
                                 NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost,
                                 this->stream_));
    HANDLE_ERROR(hipMemcpyAsync(this->trajectory_costs_nominal_.data(),
                                 trajectory_costs_nominal_d,
                                 NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost,
                                 this->stream_));
    HANDLE_ERROR(hipStreamSynchronize(this->stream_));

    this->normalizer_ = mppi_common::computeNormalizer(
        this->trajectory_costs_.data(), NUM_ROLLOUTS);
    this->normalizer_nominal_ = mppi_common::computeNormalizer(
        this->trajectory_costs_nominal_.data(), NUM_ROLLOUTS);


    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, B_X>(
            this->trajectory_costs_d_, this->control_noise_d_, this->control_d_,
            this->normalizer_, this->num_timesteps_, this->stream_);
    mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, B_X>(
            trajectory_costs_nominal_d,
            control_noise_nominal_d, control_nominal_d,
            this->normalizer_nominal_, this->num_timesteps_, this->stream_);

    // Transfer the new control to the host
    HANDLE_ERROR( hipMemcpyAsync(this->control_.data(), this->control_d_,
                                  sizeof(float) * single_control_traj_size,
                                  hipMemcpyDeviceToHost, this->stream_));
    HANDLE_ERROR( hipMemcpyAsync(this->nominal_control_trajectory_.data(),
                                  control_nominal_d,
                                  sizeof(float) * single_control_traj_size,
                                  hipMemcpyDeviceToHost, this->stream_));
    hipStreamSynchronize(this->stream_);

    }
  }
protected:
  std::mt19937 rng_gen_;
  std::normal_distribution<float> control_dist_;
};

TEST(CCMTest, RMPPIRolloutKernel) {
  using DYN = DoubleIntegratorDynamics;
  using COST = DoubleIntegratorCircleCost;
  DYN model;
  COST cost;
  const int num_timesteps = 50;
  const int num_rollouts = 64;

  using CONTROLLER = RMPPICCMDoubleIntegratorController<DYN, COST,
                                                        num_timesteps,
                                                        num_rollouts>;

  // Todo create RMPPI controller to use updateImportanceSamplingmethods

  const int state_dim = DYN::STATE_DIM;
  const int control_dim = DYN::CONTROL_DIM;

  float dt = 0.01;
  // int max_iter = 10;
  float lambda = 0.1;
  float alpha = 0;

  // float x[num_rollouts * state_dim * 2];
  // float x_dot[num_rollouts * state_dim * 2];
  // float u[num_rollouts * control_dim * 2];
  // float du[num_rollouts * control_dim * 2];
  float sigma_u[control_dim] = {0.5, 0.05}; // variance to sample noise from
  CONTROLLER::control_array control_std_dev = CONTROLLER::control_array::Constant(0.5);
  // COST::control_matrix cost_variance = COST::control_matrix::Identity();
  // for(int i = 0; i < control_dim; i++) {
  //   cost_variance(i, i) = sigma_u[i];
  // }
  // float fb_u[num_rollouts * control_dim * state_dim];

  DYN::state_array x_init_act;
  x_init_act << 4, 0, 0, 0;
  DYN::state_array x_init_nom;
  x_init_nom << 0, 0, 0.1, 0;

  // Generate control noise
  float sampled_noise[num_rollouts * num_timesteps * control_dim];
  std::mt19937 rng_gen;
  std::vector<std::normal_distribution<float>> control_dist;
  for (int i = 0; i < control_dim; i++) {
    control_dist.push_back(std::normal_distribution<float>(0, 1));
  }

  for (int n = 0; n < num_rollouts; n++) {
    int n_ind = n * num_timesteps * control_dim;
    for (int t = 0; t < num_timesteps; t++) {
      int t_ind = t * control_dim;
      for (int j = 0; j < control_dim; j++) {
        sampled_noise[n_ind + t_ind + j] = control_dist[j](rng_gen);
      }
    }
  }
  // TODO: Figure out nonzero Initial control trajectory
  float u_traj[num_timesteps * control_dim] = {0};
  for (int i = 0; i < num_timesteps; i++) {
    u_traj[i * control_dim] = 1;
  }
  // u_traj[0] = 1;
  // u_traj[1] = 0.5;

  // u_traj[10] = 1;
  // u_traj[11] = 0.5;

  // u_traj[14] = -1;
  // u_traj[15] = 0.5;

  // TODO: Generate feedback gain trajectories
  // VanillaMPPIController<DYN, COST, 100, 512, 64, 8>::feedback_gain_trajectory feedback_gains;
  // for (int i = 0; i < num_timesteps; i++) {
  //   feedback_gains.push_back(DYN::feedback_matrix::Constant(-15));
  // }

  // // Copy Feedback Gains into an array
  // float feedback_array[num_timesteps * control_dim * state_dim];
  // for (size_t i = 0; i < feedback_gains.size(); i++) {
  //   // std::cout << "Matrix " << i << ":\n";
  //   // std::cout << feedback_gains[i] << std::endl;
  //   int i_index = i * control_dim * state_dim;

  //   for (size_t j = 0; j < control_dim * state_dim; j++) {
  //     feedback_array[i_index + j] = feedback_gains[i].data()[j];
  //   }
  // }
  /**
   * Create vectors of data for GPU/CPU test
   */
  std::vector<float> x_init_act_vec, x_init_nom_vec, sigma_u_vec, u_traj_vec;
  x_init_act_vec.assign(x_init_act.data(), x_init_act.data() + state_dim);
  x_init_nom_vec.assign(x_init_nom.data(), x_init_nom.data() + state_dim);
  sigma_u_vec.assign(sigma_u, sigma_u + control_dim);
  u_traj_vec.assign(u_traj, u_traj + num_timesteps * control_dim);
  std::vector<float> feedback_gains_seq_vec, sampled_noise_vec;
  // feedback_gains_seq_vec.assign(feedback_array, feedback_array +
  //   num_timesteps * control_dim * state_dim);
  sampled_noise_vec.assign(sampled_noise, sampled_noise +
    num_rollouts * num_timesteps * control_dim);

  float value_func_threshold = 50000;


  // ============= Entire Sim loop ================
  // for (int i = 0; i < num_timesteps; i++) {
    // UpdateImportanceSamplingControl with new state
    // =============== computeControl ===============
    // Generate noise for new control sequences
    // Call Rollout kernel with new state and noise
    // computeBaseline for nominal and actual

    // Launch NormExpKernel for nominal and actual
    // Compute Normalizer for norminal and actual
    // Launch weighted reduction kernel for nominal and actual
    // Optional: Smooth trajectory

    // ============== Update State ======================
    // Get current control
    // Get current feedback
    // ComputeDynamics
    // Update State

  // }
  // Output Trajectory Costs
  std::array<float, num_rollouts> costs_act_CPU, costs_nom_CPU;
  launchRMPPIRolloutKernelCCMCPU<DYN, COST, num_rollouts>(&model, &cost, dt,
    num_timesteps, lambda, alpha, value_func_threshold, x_init_nom_vec,
    x_init_act_vec, sigma_u_vec, u_traj_vec, sampled_noise_vec,
    costs_act_CPU, costs_nom_CPU);
}
