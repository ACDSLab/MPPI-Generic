#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <mppi/utils/test_helper.h>
#include <random>
#include <algorithm>
#include <math.h>

#include <mppi/utils/nn_helpers/lstm_helper.cuh>
// Auto-generated header file
#include <autorally_test_network.h>
#include <mppi/utils/network_helper_kernel_test.cuh>
#include <mppi/utils/math_utils.h>
#include <unsupported/Eigen/NumericalDiff>
#include "mppi/ddp/ddp_dynamics.h"

class LSTMHelperTest : public testing::Test
{
protected:
  void SetUp() override
  {
    generator = std::default_random_engine(7.0);
    distribution = std::normal_distribution<float>(0.0, 1.0);
  }

  void TearDown() override
  {
  }

  std::default_random_engine generator;
  std::normal_distribution<float> distribution;
};

TEST_F(LSTMHelperTest, ParamsConstructor1)
{
  const int hidden_dim = LSTMParams<5, 25>::HIDDEN_DIM;
  const int shared_mem_grd = LSTMHelper<LSTMParams<5, 25>, FNNParams<28, 3>>::SHARED_MEM_REQUEST_GRD;
  const int shared_mem_blk = LSTMHelper<LSTMParams<5, 25>, FNNParams<28, 3>>::SHARED_MEM_REQUEST_BLK;
  const int input_dim = LSTMParams<5, 25>::INPUT_DIM;
  const int hidden_hidden_dim = LSTMParams<5, 25>::HIDDEN_HIDDEN_SIZE;
  const int input_hidden_dim = LSTMParams<5, 25>::INPUT_HIDDEN_SIZE;
  EXPECT_EQ(shared_mem_grd, sizeof(LSTMParams<5, 25>) + sizeof(FNNParams<28, 3>));
  EXPECT_EQ(shared_mem_blk, 25 * 8 + 5 + 29 * 2);
  EXPECT_EQ(input_dim, 5);
  EXPECT_EQ(hidden_hidden_dim, 25 * 25);
  EXPECT_EQ(input_hidden_dim, 5 * 25);

  LSTMParams<5, 25> params;

  auto W_im = params.W_im;
  auto W_fm = params.W_fm;
  auto W_om = params.W_om;
  auto W_cm = params.W_cm;
  for (int i = 0; i < 25 * 25; i++)
  {
    EXPECT_FLOAT_EQ(W_im[i], 0.0f);
    EXPECT_FLOAT_EQ(W_fm[i], 0.0f);
    EXPECT_FLOAT_EQ(W_cm[i], 0.0f);
    EXPECT_FLOAT_EQ(W_om[i], 0.0f);
  }

  auto W_ii = params.W_ii;
  auto W_fi = params.W_fi;
  auto W_oi = params.W_oi;
  auto W_ci = params.W_ci;
  for (int i = 0; i < 5 * 25; i++)
  {
    EXPECT_FLOAT_EQ(W_ii[i], 0.0f);
    EXPECT_FLOAT_EQ(W_fi[i], 0.0f);
    EXPECT_FLOAT_EQ(W_oi[i], 0.0f);
    EXPECT_FLOAT_EQ(W_ci[i], 0.0f);
  }

  auto b_i = params.b_i;
  auto b_f = params.b_f;
  auto b_o = params.b_o;
  auto b_c = params.b_c;
  for (int i = 0; i < 25; i++)
  {
    EXPECT_FLOAT_EQ(b_i[i], 0.0f);
    EXPECT_FLOAT_EQ(b_f[i], 0.0f);
    EXPECT_FLOAT_EQ(b_o[i], 0.0f);
    EXPECT_FLOAT_EQ(b_c[i], 0.0f);
  }

  auto init_hidden = params.initial_hidden;
  auto init_cell = params.initial_cell;
  for (int i = 0; i < 25; i++)
  {
    EXPECT_FLOAT_EQ(init_hidden[i], 0.0f);
    EXPECT_FLOAT_EQ(init_cell[i], 0.0f);
  }
}

TEST_F(LSTMHelperTest, ParamsConstructor2)
{
  int total_amount = 0;
  // delay model
  total_amount += LSTMHelper<LSTMParams<1, 1>, FNNParams<2, 10, 1>>::SHARED_MEM_REQUEST_GRD;
  total_amount += LSTMHelper<LSTMParams<1, 1>, FNNParams<2, 10, 1>>::SHARED_MEM_REQUEST_BLK * 8;
  // terra model
  total_amount += LSTMHelper<LSTMParams<8, 10>, FNNParams<18, 10, 3>>::SHARED_MEM_REQUEST_GRD;
  total_amount += LSTMHelper<LSTMParams<8, 10>, FNNParams<18, 10, 3>>::SHARED_MEM_REQUEST_BLK * 8;
  // engine model
  total_amount += LSTMHelper<LSTMParams<4, 5>, FNNParams<9, 10, 1>>::SHARED_MEM_REQUEST_GRD;
  total_amount += LSTMHelper<LSTMParams<4, 5>, FNNParams<9, 10, 1>>::SHARED_MEM_REQUEST_BLK * 8;
  // steering model
  total_amount += LSTMHelper<LSTMParams<7, 5>, FNNParams<12, 20, 1>>::SHARED_MEM_REQUEST_GRD;
  total_amount += LSTMHelper<LSTMParams<7, 5>, FNNParams<12, 20, 1>>::SHARED_MEM_REQUEST_BLK * 8;

  std::cout << "total amount: " << total_amount * 4 << std::endl;
  EXPECT_LT(total_amount * 4, 49152);
}

TEST_F(LSTMHelperTest, BindStream)
{
  hipStream_t stream;
  HANDLE_ERROR(hipStreamCreate(&stream));

  LSTMHelper<LSTMParams<5, 25>, FNNParams<25, 30, 3>> helper(stream);

  EXPECT_EQ(helper.stream_, stream);
}

using LSTM = LSTMHelper<LSTMParams<8, 20>, FNNParams<28, 3>>;
TEST_F(LSTMHelperTest, GPUSetupAndParamsCheck)
{
  LSTM model;

  std::vector<float> theta_vec(87);
  for (int i = 0; i < 87; i++)
  {
    theta_vec[i] = distribution(generator);
  }
  model.updateOutputModel({ 28, 3 }, theta_vec);

  int grid_dim = 5;

  std::vector<LSTM::LSTM_PARAMS_T> lstm_params(grid_dim);
  std::vector<LSTM::LSTM_PARAMS_T> shared_lstm_params(grid_dim);
  std::vector<LSTM::OUTPUT_FNN_T::NN_PARAMS_T> fnn_params(grid_dim);
  std::vector<LSTM::OUTPUT_FNN_T::NN_PARAMS_T> shared_fnn_params(grid_dim);

  EXPECT_EQ(model.GPUMemStatus_, false);
  EXPECT_EQ(model.network_d_, nullptr);
  EXPECT_NE(model.getOutputModel(), nullptr);

  model.GPUSetup();

  EXPECT_EQ(model.GPUMemStatus_, true);
  EXPECT_NE(model.network_d_, nullptr);

  // launch kernel
  launchParameterCheckTestKernel<LSTM>(model, lstm_params, shared_lstm_params, fnn_params, shared_fnn_params);

  for (int grid = 0; grid < grid_dim; grid++)
  {
    // ensure that the output nn matches
    for (int i = 0; i < 87; i++)
    {
      EXPECT_FLOAT_EQ(fnn_params[grid].theta[i], theta_vec[i]) << "at grid " << grid << " at index " << i;
      EXPECT_FLOAT_EQ(shared_fnn_params[grid].theta[i], theta_vec[i]) << "at grid " << grid << "at index " << i;
    }
    EXPECT_EQ(fnn_params[grid].stride_idcs[0], 0);
    EXPECT_EQ(fnn_params[grid].stride_idcs[1], 84);
    EXPECT_EQ(shared_fnn_params[grid].stride_idcs[0], 0) << "at grid " << grid;
    EXPECT_EQ(shared_fnn_params[grid].stride_idcs[1], 84) << "at grid " << grid;

    EXPECT_EQ(fnn_params[grid].net_structure[0], 28);
    EXPECT_EQ(fnn_params[grid].net_structure[1], 3);
    EXPECT_EQ(shared_fnn_params[grid].net_structure[0], 28) << "at grid " << grid;
    EXPECT_EQ(shared_fnn_params[grid].net_structure[1], 3) << "at grid " << grid;

    for (int i = 0; i < 20 * 20; i++)
    {
      EXPECT_FLOAT_EQ(lstm_params[grid].W_im[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_fm[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_cm[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_om[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_im[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_fm[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_cm[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_om[i], 0.0f);
    }

    for (int i = 0; i < 8 * 20; i++)
    {
      EXPECT_FLOAT_EQ(lstm_params[grid].W_ii[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_fi[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_oi[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_ci[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_ii[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_fi[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_oi[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_ci[i], 0.0f);
    }

    for (int i = 0; i < 20; i++)
    {
      EXPECT_FLOAT_EQ(lstm_params[grid].b_i[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].b_f[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].b_o[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].b_c[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_i[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_f[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_o[i], 0.0f);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_c[i], 0.0f);
      EXPECT_FLOAT_EQ(lstm_params[grid].initial_hidden[i], 0.0f) << "at index " << i;
      EXPECT_FLOAT_EQ(lstm_params[grid].initial_cell[i], 0.0f) << "at index " << i;
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].initial_hidden[i], 0.0f) << "at index " << i;
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].initial_cell[i], 0.0f) << "at index " << i;
    }
  }
}

TEST_F(LSTMHelperTest, UpdateModel)
{
  LSTM model;

  int grid_dim = 5;

  std::vector<float> theta_vec(87);
  for (int i = 0; i < theta_vec.size(); i++)
  {
    theta_vec[i] = distribution(generator);
  }
  model.updateOutputModel({ 28, 3 }, theta_vec);

  auto params = model.getLSTMParams();
  for (int i = 0; i < 20 * 20; i++)
  {
    params.W_im[i] = distribution(generator);
    params.W_fm[i] = distribution(generator);
    params.W_om[i] = distribution(generator);
    params.W_cm[i] = distribution(generator);
  }
  for (int i = 0; i < 8 * 20; i++)
  {
    params.W_ii[i] = distribution(generator);
    params.W_fi[i] = distribution(generator);
    params.W_oi[i] = distribution(generator);
    params.W_ci[i] = distribution(generator);
  }
  for (int i = 0; i < 20; i++)
  {
    params.b_i[i] = distribution(generator);
    params.b_f[i] = distribution(generator);
    params.b_o[i] = distribution(generator);
    params.b_c[i] = distribution(generator);
    params.initial_hidden[i] = distribution(generator);
    params.initial_cell[i] = distribution(generator);
  }
  model.setLSTMParams(params);

  for (int i = 0; i < 20; i++)
  {
    EXPECT_FLOAT_EQ(model.getHiddenState()[i], params.initial_hidden[i]);
    EXPECT_FLOAT_EQ(model.getCellState()[i], params.initial_cell[i]);
  }

  std::vector<LSTM::LSTM_PARAMS_T> lstm_params(grid_dim);
  std::vector<LSTM::LSTM_PARAMS_T> shared_lstm_params(grid_dim);
  std::vector<LSTM::OUTPUT_FNN_T::NN_PARAMS_T> fnn_params(grid_dim);
  std::vector<LSTM::OUTPUT_FNN_T::NN_PARAMS_T> shared_fnn_params(grid_dim);

  EXPECT_EQ(model.GPUMemStatus_, false);
  EXPECT_EQ(model.network_d_, nullptr);
  EXPECT_NE(model.getOutputModel(), nullptr);

  model.GPUSetup();

  EXPECT_EQ(model.GPUMemStatus_, true);
  EXPECT_NE(model.network_d_, nullptr);

  // launch kernel
  launchParameterCheckTestKernel<LSTM>(model, lstm_params, shared_lstm_params, fnn_params, shared_fnn_params);

  for (int grid = 0; grid < grid_dim; grid++)
  {
    // ensure that the output nn matches
    for (int i = 0; i < 87; i++)
    {
      EXPECT_FLOAT_EQ(fnn_params[grid].theta[i], theta_vec[i]) << "at index " << i;
      EXPECT_FLOAT_EQ(shared_fnn_params[grid].theta[i], theta_vec[i]) << "at index " << i;
    }
    EXPECT_EQ(fnn_params[grid].stride_idcs[0], 0);
    EXPECT_EQ(fnn_params[grid].stride_idcs[1], 84);
    EXPECT_EQ(shared_fnn_params[grid].stride_idcs[0], 0) << "at grid " << grid;
    EXPECT_EQ(shared_fnn_params[grid].stride_idcs[1], 84) << "at grid " << grid;

    EXPECT_EQ(fnn_params[grid].net_structure[0], 28);
    EXPECT_EQ(fnn_params[grid].net_structure[1], 3);
    EXPECT_EQ(shared_fnn_params[grid].net_structure[0], 28) << "at grid " << grid;
    EXPECT_EQ(shared_fnn_params[grid].net_structure[1], 3) << "at grid " << grid;

    for (int i = 0; i < 20 * 20; i++)
    {
      EXPECT_FLOAT_EQ(lstm_params[grid].W_im[i], params.W_im[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_fm[i], params.W_fm[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_cm[i], params.W_cm[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_om[i], params.W_om[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_im[i], params.W_im[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_fm[i], params.W_fm[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_cm[i], params.W_cm[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_om[i], params.W_om[i]);
    }

    for (int i = 0; i < 8 * 20; i++)
    {
      EXPECT_FLOAT_EQ(lstm_params[grid].W_ii[i], params.W_ii[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_fi[i], params.W_fi[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_oi[i], params.W_oi[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].W_ci[i], params.W_ci[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_ii[i], params.W_ii[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_fi[i], params.W_fi[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_oi[i], params.W_oi[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].W_ci[i], params.W_ci[i]);
    }

    for (int i = 0; i < 20; i++)
    {
      EXPECT_FLOAT_EQ(lstm_params[grid].b_i[i], params.b_i[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].b_f[i], params.b_f[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].b_o[i], params.b_o[i]);
      EXPECT_FLOAT_EQ(lstm_params[grid].b_c[i], params.b_c[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_i[i], params.b_i[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_f[i], params.b_f[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_o[i], params.b_o[i]);
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].b_c[i], params.b_c[i]);
    }

    for (int i = 0; i < 20; i++)
    {
      EXPECT_FLOAT_EQ(lstm_params[grid].initial_hidden[i], params.initial_hidden[i]) << "at index " << i;
      EXPECT_FLOAT_EQ(lstm_params[grid].initial_cell[i], params.initial_cell[i]) << "at index " << i;
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].initial_hidden[i], params.initial_hidden[i]) << "at index " << i;
      EXPECT_FLOAT_EQ(shared_lstm_params[grid].initial_cell[i], params.initial_cell[i]) << "at index " << i;
    }
  }
}

TEST_F(LSTMHelperTest, LoadModelPathTest)
{
  using LSTM = LSTMHelper<LSTMParams<3, 25>, FNNParams<28, 30, 30, 2>>;
  LSTM model;
  model.GPUSetup();

  std::string path = mppi::tests::test_lstm_network;
  if (!fileExists(path))
  {
    std::cerr << "Could not load neural net model at path: " << path.c_str();
    exit(-1);
  }
  model.loadParams(path);

  path = mppi::tests::test_lstm_input_output;

  cnpy::npz_t input_outputs = cnpy::npz_load(path);
  double* inputs = input_outputs.at("input").data<double>();
  double* outputs = input_outputs.at("output").data<double>();
  double* hidden = input_outputs.at("hidden").data<double>();
  double* cell = input_outputs.at("cell").data<double>();

  double tol = 1e-5;

  LSTM::input_array input;
  LSTM::output_array output;

  for (int point = 0; point < 1000; point++)
  {
    for (int i = 0; i < 3; i++)
    {
      input(i) = inputs[i + 3 * point];
    }
    model.resetHiddenCellCPU();
    model.forward(input, output);

    for (int i = 0; i < 2; i++)
    {
      EXPECT_NEAR(output[i], outputs[i + 2 * point], tol) << "point " << point << " at dim " << i;
    }
    for (int i = 0; i < 25; i++)
    {
      EXPECT_NEAR(model.getHiddenState()[i], hidden[i + 25 * point], tol) << "point " << point << " at dim " << i;
      EXPECT_NEAR(model.getCellState()[i], cell[i + 25 * point], tol) << "point " << point << " at dim " << i;
    }
  }
}

TEST_F(LSTMHelperTest, LoadModelNPZTest)
{
  using LSTM = LSTMHelper<LSTMParams<3, 25>, FNNParams<28, 30, 30, 2>>;
  LSTM model;
  model.GPUSetup();

  std::string path = mppi::tests::test_lstm_network;
  if (!fileExists(path))
  {
    std::cerr << "Could not load neural net model at path: " << path.c_str();
    exit(-1);
  }
  cnpy::npz_t param_dict = cnpy::npz_load(path);
  model.loadParams(param_dict);

  path = mppi::tests::test_lstm_input_output;

  cnpy::npz_t input_outputs = cnpy::npz_load(path);
  double* inputs = input_outputs.at("input").data<double>();
  double* outputs = input_outputs.at("output").data<double>();
  double* hidden = input_outputs.at("hidden").data<double>();
  double* cell = input_outputs.at("cell").data<double>();

  double tol = 1e-5;

  LSTM::input_array input;
  LSTM::output_array output;

  for (int point = 0; point < 1000; point++)
  {
    for (int i = 0; i < 3; i++)
    {
      input(i) = inputs[i + 3 * point];
    }
    model.resetHiddenCellCPU();
    model.forward(input, output);

    for (int i = 0; i < 2; i++)
    {
      EXPECT_NEAR(output[i], outputs[i + 2 * point], tol) << "point " << point << " at dim " << i;
    }
    for (int i = 0; i < 25; i++)
    {
      EXPECT_NEAR(model.getHiddenState()[i], hidden[i + 25 * point], tol) << "point " << point << " at dim " << i;
      EXPECT_NEAR(model.getCellState()[i], cell[i + 25 * point], tol) << "point " << point << " at dim " << i;
    }
  }
}

TEST_F(LSTMHelperTest, forwardCPU)
{
  LSTM model;

  std::vector<float> theta_vec(87);
  for (int i = 0; i < theta_vec.size(); i++)
  {
    theta_vec[i] = 1.0;
  }
  model.updateOutputModel({ 28, 3 }, theta_vec);

  auto params = model.getLSTMParams();
  params.setAllValues(1.0f);
  model.setLSTMParams(params);

  LSTM::input_array input = LSTM::input_array::Ones();
  LSTM::output_array output;

  model.forward(input, output);
  EXPECT_FLOAT_EQ(output[0], 28.28055);
  EXPECT_FLOAT_EQ(output[1], 28.28055);
  EXPECT_FLOAT_EQ(output[2], 28.28055);

  model.forward(input, output);
  EXPECT_FLOAT_EQ(output[0], 28.901096);
  EXPECT_FLOAT_EQ(output[1], 28.901096);
  EXPECT_FLOAT_EQ(output[2], 28.901096);

  model.forward(input, output);
  EXPECT_FLOAT_EQ(output[0], 28.986588);
  EXPECT_FLOAT_EQ(output[1], 28.986588);
  EXPECT_FLOAT_EQ(output[2], 28.986588);

  model.forward(input, output);
  EXPECT_FLOAT_EQ(output[0], 28.998184);
  EXPECT_FLOAT_EQ(output[1], 28.998184);
  EXPECT_FLOAT_EQ(output[2], 28.998184);

  model.forward(input, output);
  EXPECT_FLOAT_EQ(output[0], 28.999756);
  EXPECT_FLOAT_EQ(output[1], 28.999756);
  EXPECT_FLOAT_EQ(output[2], 28.999756);
}

using LSTM2 = LSTMHelper<LSTMParams<8, 20>, FNNParams<28, 3>, false>;
TEST_F(LSTMHelperTest, forwardGPU)
{
  const int num_rollouts = 1000;

  LSTM2 model;
  model.GPUSetup();

  std::vector<float> theta(87);
  std::fill(theta.begin(), theta.end(), 1);
  model.updateOutputModel({ 28, 3 }, theta);

  auto params = model.getLSTMParams();
  for (int i = 0; i < 20 * 20; i++)
  {
    params.W_im[i] = 1.0;
    params.W_fm[i] = 1.0;
    params.W_om[i] = 1.0;
    params.W_cm[i] = 1.0;
  }
  for (int i = 0; i < 8 * 20; i++)
  {
    params.W_ii[i] = 1.0;
    params.W_fi[i] = 1.0;
    params.W_oi[i] = 1.0;
    params.W_ci[i] = 1.0;
  }
  for (int i = 0; i < 20; i++)
  {
    params.b_i[i] = 1.0;
    params.b_f[i] = 1.0;
    params.b_o[i] = 1.0;
    params.b_c[i] = 1.0;
    params.initial_hidden[i] = 1.0;
    params.initial_cell[i] = 1.0;
  }
  model.setLSTMParams(params);

  Eigen::Matrix<float, LSTM2::INPUT_DIM, num_rollouts> inputs;
  inputs = Eigen::Matrix<float, LSTM2::INPUT_DIM, num_rollouts>::Ones();
  LSTM2::output_array output;

  std::array<float, 5> true_vals = { 28.28055, 28.901096, 28.986588, 28.998184, 28.999756 };

  std::vector<std::array<float, LSTM2::INPUT_DIM>> input_arr(num_rollouts);
  std::vector<std::array<float, 3>> output_arr(num_rollouts);

  for (int y_dim = 1; y_dim < 16; y_dim++)
  {
    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < input_arr[0].size(); dim++)
      {
        input_arr[state_index][dim] = inputs.col(state_index)(dim);
      }
    }

    for (int step = 1; step < 6; step++)
    {
      launchForwardTestKernel<LSTM2, 32>(model, input_arr, output_arr, y_dim, step);
      for (int point = 0; point < num_rollouts; point++)
      {
        model.resetHiddenCellCPU();
        LSTM2::input_array input = inputs.col(point);
        LSTM2::output_array output;

        for (int cpu_step = 0; cpu_step < step; cpu_step++)
        {
          model.forward(input, output);
        }
        for (int dim = 0; dim < LSTM2::INPUT_DIM; dim++)
        {
          EXPECT_NEAR(input(dim), input_arr[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        }
        for (int dim = 0; dim < LSTM2::OUTPUT_DIM; dim++)
        {
          EXPECT_NEAR(output(dim), output_arr[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
          EXPECT_TRUE(isfinite(output_arr[point][dim]));
          EXPECT_FLOAT_EQ(output(dim), true_vals[step - 1]) << "at dim " << dim << " step " << step;
        }
      }
    }
  }
}

TEST_F(LSTMHelperTest, forwardGPUCompareNoShared)
{
  const int num_rollouts = 1000;

  LSTM2 model;
  model.GPUSetup();

  std::vector<float> theta(223);
  std::fill(theta.begin(), theta.end(), 1);
  model.updateOutputModel({ 28, 3 }, theta);

  auto params = model.getLSTMParams();
  for (int i = 0; i < 20 * 20; i++)
  {
    params.W_im[i] = distribution(generator);
    params.W_fm[i] = distribution(generator);
    params.W_om[i] = distribution(generator);
    params.W_cm[i] = distribution(generator);
  }
  for (int i = 0; i < 8 * 20; i++)
  {
    params.W_ii[i] = distribution(generator);
    params.W_fi[i] = distribution(generator);
    params.W_oi[i] = distribution(generator);
    params.W_ci[i] = distribution(generator);
  }
  for (int i = 0; i < 20; i++)
  {
    params.b_i[i] = distribution(generator);
    params.b_f[i] = distribution(generator);
    params.b_o[i] = distribution(generator);
    params.b_c[i] = distribution(generator);
    params.initial_hidden[i] = distribution(generator);
    params.initial_cell[i] = distribution(generator);
  }
  model.setLSTMParams(params);

  Eigen::Matrix<float, LSTM2::INPUT_DIM, num_rollouts> inputs;
  inputs = Eigen::Matrix<float, LSTM2::INPUT_DIM, num_rollouts>::Random();
  LSTM2::output_array output;

  std::vector<std::array<float, LSTM2::INPUT_DIM>> input_arr(num_rollouts);
  std::vector<std::array<float, 3>> output_arr(num_rollouts);

  for (int y_dim = 1; y_dim < 16; y_dim++)
  {
    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < input_arr[0].size(); dim++)
      {
        input_arr[state_index][dim] = inputs.col(state_index)(dim);
      }
    }
    for (int step = 1; step < 6; step++)
    {
      launchForwardTestKernel<LSTM2, 32>(model, input_arr, output_arr, y_dim, step);
      for (int point = 0; point < num_rollouts; point++)
      {
        model.resetHiddenCellCPU();
        LSTM2::input_array input = inputs.col(point);
        LSTM2::output_array output;

        for (int cpu_step = 0; cpu_step < step; cpu_step++)
        {
          model.forward(input, output);
        }
        for (int dim = 0; dim < LSTM2::INPUT_DIM; dim++)
        {
          EXPECT_NEAR(input(dim), input_arr[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        }
        for (int dim = 0; dim < LSTM2::OUTPUT_DIM; dim++)
        {
          EXPECT_NEAR(output(dim), output_arr[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
          EXPECT_TRUE(isfinite(output_arr[point][dim]));
        }
      }
    }
  }
}

using LSTM3 = LSTMHelper<LSTMParams<8, 10>, FNNParams<18, 10, 3>>;
TEST_F(LSTMHelperTest, forwardGPUCompareShared)
{
  const int num_rollouts = 1000;

  LSTM3 model;
  model.GPUSetup();

  std::vector<float> theta(223);
  std::fill(theta.begin(), theta.end(), 1);
  model.updateOutputModel({ 18, 10, 3 }, theta);

  auto params = model.getLSTMParams();
  for (int i = 0; i < 10 * 10; i++)
  {
    params.W_im[i] = distribution(generator);
    params.W_fm[i] = distribution(generator);
    params.W_om[i] = distribution(generator);
    params.W_cm[i] = distribution(generator);
  }
  for (int i = 0; i < 8 * 10; i++)
  {
    params.W_ii[i] = distribution(generator);
    params.W_fi[i] = distribution(generator);
    params.W_oi[i] = distribution(generator);
    params.W_ci[i] = distribution(generator);
  }
  for (int i = 0; i < 10; i++)
  {
    params.b_i[i] = distribution(generator);
    params.b_f[i] = distribution(generator);
    params.b_o[i] = distribution(generator);
    params.b_c[i] = distribution(generator);
    params.initial_hidden[i] = distribution(generator);
    params.initial_cell[i] = distribution(generator);
  }
  model.setLSTMParams(params);

  Eigen::Matrix<float, LSTM3::INPUT_DIM, num_rollouts> inputs;
  inputs = Eigen::Matrix<float, LSTM3::INPUT_DIM, num_rollouts>::Random();
  LSTM3::output_array output;

  std::vector<std::array<float, LSTM3::INPUT_DIM>> input_arr(num_rollouts);
  std::vector<std::array<float, 3>> output_arr(num_rollouts);

  for (int y_dim = 1; y_dim < 16; y_dim++)
  {
    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < input_arr[0].size(); dim++)
      {
        input_arr[state_index][dim] = inputs.col(state_index)(dim);
      }
    }
    for (int step = 1; step < 6; step++)
    {
      launchForwardTestKernel<LSTM3, 32>(model, input_arr, output_arr, y_dim, step);
      for (int point = 0; point < num_rollouts; point++)
      {
        model.resetHiddenCellCPU();
        LSTM3::input_array input = inputs.col(point);
        LSTM3::output_array output;

        for (int cpu_step = 0; cpu_step < step; cpu_step++)
        {
          model.forward(input, output);
        }
        for (int dim = 0; dim < LSTM3::INPUT_DIM; dim++)
        {
          EXPECT_NEAR(input(dim), input_arr[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        }
        for (int dim = 0; dim < LSTM3::OUTPUT_DIM; dim++)
        {
          EXPECT_NEAR(output(dim), output_arr[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
          EXPECT_TRUE(isfinite(output_arr[point][dim]));
        }
      }
    }
  }
}

using LSTM3Global = LSTMHelper<LSTMParams<8, 10>, FNNParams<18, 10, 3>, false>;
TEST_F(LSTMHelperTest, forwardGPUSpeedTest)
{
  const int num_rollouts = 3000;

  LSTM3 shared_model;
  LSTM3Global global_model;

  shared_model.GPUSetup();
  global_model.GPUSetup();

  std::vector<float> theta(223);
  std::fill(theta.begin(), theta.end(), 1);
  shared_model.updateOutputModel({ 18, 10, 3 }, theta);
  global_model.updateOutputModel({ 18, 10, 3 }, theta);

  auto shared_params = shared_model.getLSTMParams();
  auto global_params = global_model.getLSTMParams();
  for (int i = 0; i < 10 * 10; i++)
  {
    shared_params.W_im[i] = distribution(generator);
    shared_params.W_fm[i] = distribution(generator);
    shared_params.W_om[i] = distribution(generator);
    shared_params.W_cm[i] = distribution(generator);
    global_params.W_im[i] = shared_params.W_im[i];
    global_params.W_fm[i] = shared_params.W_fm[i];
    global_params.W_om[i] = shared_params.W_om[i];
    global_params.W_cm[i] = shared_params.W_cm[i];
  }
  for (int i = 0; i < 8 * 10; i++)
  {
    shared_params.W_ii[i] = distribution(generator);
    shared_params.W_fi[i] = distribution(generator);
    shared_params.W_oi[i] = distribution(generator);
    shared_params.W_ci[i] = distribution(generator);
    global_params.W_ii[i] = shared_params.W_ii[i];
    global_params.W_fi[i] = shared_params.W_fi[i];
    global_params.W_oi[i] = shared_params.W_oi[i];
    global_params.W_ci[i] = shared_params.W_ci[i];
  }
  for (int i = 0; i < 10; i++)
  {
    shared_params.b_i[i] = distribution(generator);
    shared_params.b_f[i] = distribution(generator);
    shared_params.b_o[i] = distribution(generator);
    shared_params.b_c[i] = distribution(generator);
    shared_params.initial_hidden[i] = distribution(generator);
    shared_params.initial_cell[i] = distribution(generator);

    global_params.b_i[i] = shared_params.b_i[i];
    global_params.b_f[i] = shared_params.b_f[i];
    global_params.b_o[i] = shared_params.b_o[i];
    global_params.b_c[i] = shared_params.b_c[i];
    global_params.initial_hidden[i] = shared_params.initial_hidden[i];
    global_params.initial_cell[i] = shared_params.initial_cell[i];
  }
  shared_model.setLSTMParams(shared_params);
  global_model.setLSTMParams(global_params);

  Eigen::Matrix<float, LSTM3::INPUT_DIM, num_rollouts> inputs;
  inputs = Eigen::Matrix<float, LSTM3::INPUT_DIM, num_rollouts>::Random();
  LSTM3::output_array output;

  std::vector<std::array<float, LSTM3::INPUT_DIM>> input_arr(num_rollouts);
  std::vector<std::array<float, 3>> output_arr(num_rollouts);

  for (int state_index = 0; state_index < num_rollouts; state_index++)
  {
    for (int dim = 0; dim < input_arr[0].size(); dim++)
    {
      input_arr[state_index][dim] = inputs.col(state_index)(dim);
    }
  }

  for (int y_dim = 1; y_dim < 16; y_dim++)
  {
    auto shared_start = std::chrono::steady_clock::now();
    launchForwardTestKernel<LSTM3, 32>(shared_model, input_arr, output_arr, y_dim, 2000);
    auto shared_stop = std::chrono::steady_clock::now();

    auto global_start = std::chrono::steady_clock::now();
    launchForwardTestKernel<LSTM3Global, 32>(global_model, input_arr, output_arr, y_dim, 2000);
    auto global_stop = std::chrono::steady_clock::now();

    float shared_time_ms = mppi::math::timeDiffms(shared_stop, shared_start);
    float global_time_ms = mppi::math::timeDiffms(global_stop, global_start);
    std::cout << "for y dim " << y_dim << " got shared: " << shared_time_ms << std::endl;
    std::cout << "for y dim " << y_dim << " got global: " << global_time_ms << std::endl;
  }
}

// TEST_F(LSTMHelperTest, TestComputeGradComputationFinite)
// {
//   LSTMHelper<LSTMParams<6, 32, 32, 4>> model;
//   std::vector<float> theta(1412);
//   for (int i = 0; i < 1412; i++)
//   {
//     theta[i] = distribution(generator);
//   }
//   model.updateModel({ 6, 32, 32, 4 }, theta);
//
//   LSTMHelper<LSTMParams<6, 32, 32, 4>>::dfdx numeric_jac;
//   LSTMHelper<LSTMParams<6, 32, 32, 4>>::dfdx analytic_jac;
//
//   for (int i = 0; i < 1000; i++)
//   {
//     LSTMHelper<LSTMParams<6, 32, 32, 4>>::input_array input;
//     input = LSTMHelper<LSTMParams<6, 32, 32, 4>>::input_array::Random();
//
//     model.computeGrad(input, analytic_jac);
//     EXPECT_TRUE(analytic_jac.allFinite());
//   }
// }
//
// TEST_F(LSTMHelperTest, TestComputeGradComputationCompare)
// {
//   GTEST_SKIP();
//   LSTMHelper<LSTMParams<6, 32, 32, 4>> model;
//   std::vector<float> theta(1412);
//   for (int i = 0; i < 1412; i++)
//   {
//     theta[i] = distribution(generator);
//   }
//   model.updateModel({ 6, 32, 32, 4 }, theta);
//
//   LSTMHelper<LSTMParams<6, 32, 32, 4>>::dfdx numeric_jac;
//   LSTMHelper<LSTMParams<6, 32, 32, 4>>::dfdx analytic_jac;
//
//   LSTMHelper<LSTMParams<6, 32, 32, 4>>::input_array input;
//   input << 1, 2, 3, 4, 5, 6;
//
//   model.computeGrad(input, analytic_jac);
//
//   // numeric_jac = num_diff.df(input, numeric_jac);
//
//   ASSERT_LT((numeric_jac - analytic_jac).norm(), 1e-3) << "Numeric Jacobian\n"
//                                                        << numeric_jac << "\nAnalytic Jacobian\n"
//                                                        << analytic_jac;
// }
