#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <mppi/utils/test_helper.h>
#include <random>
#include <algorithm>
#include <math.h>
#include <cmath>

#include <mppi/utils/activation_functions.cuh>
#include <mppi/utils/gpu_err_chk.cuh>
#include "mppi/utils/math_utils.h"

class ActivationFunctionTest : public testing::Test
{
protected:
  void SetUp() override
  {
    generator = std::default_random_engine(7.0);
    distribution = std::normal_distribution<float>(0.0, 2.0);
  }

  void TearDown() override
  {
  }

  std::default_random_engine generator;
  std::normal_distribution<float> distribution;
};

TEST_F(ActivationFunctionTest, TanhCPU)
{
  for (int i = 0; i < 1e5; i++)
  {
    float num = distribution(generator);
    EXPECT_FLOAT_EQ(mppi::nn::tanh(num), std::tanh(num));
  }
}

__global__ void tanhTestKernel(float* input, int num, int times)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num)
  {
    for (int i = 0; i < times; i++)
    {
      input[tid] = mppi::nn::tanh(input[tid]);
    }
  }
}

__global__ void tanhStableTestKernel(float* input, int num, int times)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num)
  {
    for (int i = 0; i < times; i++)
    {
      input[tid] = mppi::nn::tanh_accurate(input[tid]);
    }
  }
}

template <int BLOCKDIM_X = 128>
void launchTanhTestKernel(std::vector<float>& input, int times = 100)
{
  float* input_d;
  float* input_stable_d;
  int count = input.size();
  HANDLE_ERROR(hipMalloc((void**)&input_d, sizeof(float) * count));
  HANDLE_ERROR(hipMalloc((void**)&input_stable_d, sizeof(float) * count));
  HANDLE_ERROR(hipMemcpy(input_d, input.data(), sizeof(float) * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(input_stable_d, input.data(), sizeof(float) * count, hipMemcpyHostToDevice));

  const int gridsize_x = (count - 1) / BLOCKDIM_X + 1;
  dim3 threadsPerBlock(BLOCKDIM_X);
  dim3 numBlocks(gridsize_x, 1);

  hipStream_t stream;
  HANDLE_ERROR(hipStreamCreate(&stream));

  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  std::cout << "\n===== TANH ======\n";

  hipEventRecord(start, stream);
  tanhTestKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(input_d, count, times);
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "time to compute fast " << time << std::endl;

  hipEventRecord(start, stream);
  tanhStableTestKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(input_stable_d, count, times);
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "time to compute stable " << time << std::endl;

  HANDLE_ERROR(hipMemcpy(input.data(), input_d, sizeof(float) * count, hipMemcpyDeviceToHost));
  hipFree(input_d);
  hipFree(input_stable_d);
}

TEST_F(ActivationFunctionTest, TanhGPU)
{
  std::vector<float> vec(1e8);
  for (int i = 0; i < vec.size(); i++)
  {
    vec[i] = distribution(generator);
  }
  std::vector<float> output_vec = vec;
  launchTanhTestKernel(output_vec, 1);
  for (int i = 0; i < vec.size(); i++)
  {
    EXPECT_NEAR(output_vec[i], std::tanh(vec[i]), 2.0e-7);
  }

  for (int i = 0; i < 10; i++)
  {
    launchTanhTestKernel(output_vec);
  }
}

TEST_F(ActivationFunctionTest, SigmoidCPU)
{
  for (int i = 0; i < 1e5; i++)
  {
    float num = distribution(generator);
    EXPECT_FLOAT_EQ(mppi::nn::sigmoid(num), (1.0f / (1.0f + std::exp(-num))));
  }
}

__global__ void sigmoidTestKernel(float* input, int num, int times)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num)
  {
    for (int i = 0; i < times; i++)
    {
      input[tid] = mppi::nn::sigmoid(input[tid]);
    }
  }
}

__global__ void sigmoidStableTestKernel(float* input, int num, int times)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num)
  {
    for (int i = 0; i < times; i++)
    {
      input[tid] = mppi::nn::sigmoid_accurate(input[tid]);
    }
  }
}

template <int BLOCKDIM_X = 128>
void launchSigmoidTestKernel(std::vector<float>& input, int times = 100)
{
  float* input_d;
  float* input_stable_d;
  int count = input.size();
  HANDLE_ERROR(hipMalloc((void**)&input_d, sizeof(float) * count));
  HANDLE_ERROR(hipMalloc((void**)&input_stable_d, sizeof(float) * count));
  HANDLE_ERROR(hipMemcpy(input_d, input.data(), sizeof(float) * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(input_stable_d, input.data(), sizeof(float) * count, hipMemcpyHostToDevice));

  const int gridsize_x = (count - 1) / BLOCKDIM_X + 1;
  dim3 threadsPerBlock(BLOCKDIM_X);
  dim3 numBlocks(gridsize_x, 1);

  hipStream_t stream;
  HANDLE_ERROR(hipStreamCreate(&stream));

  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  std::cout << "\n===== SIGMOID ======\n";

  hipEventRecord(start, stream);
  sigmoidTestKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(input_d, count, times);
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "time to compute fast " << time << std::endl;

  hipEventRecord(start, stream);
  sigmoidStableTestKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(input_stable_d, count, times);
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "time to compute stable " << time << std::endl;

  HANDLE_ERROR(hipMemcpy(input.data(), input_d, sizeof(float) * count, hipMemcpyDeviceToHost));
  hipFree(input_d);
  hipFree(input_stable_d);
}

TEST_F(ActivationFunctionTest, SigmoidGPU)
{
  std::vector<float> vec(1e8);
  for (int i = 0; i < vec.size(); i++)
  {
    vec[i] = distribution(generator);
  }
  std::vector<float> output_vec = vec;
  launchSigmoidTestKernel(output_vec, 1);
  for (int i = 0; i < vec.size(); i++)
  {
    EXPECT_NEAR(output_vec[i], (1.0f / (1.0f + std::exp(-vec[i]))), 2.0e-7);
  }
  for (int i = 0; i < 10; i++)
  {
    launchSigmoidTestKernel(output_vec);
  }
}
