#include <gtest/gtest.h>
#include <instantiations/cartpole_mppi/cartpole_mppi.cuh>


class Cartpole_VanillaMPPI: public ::testing::Test {
public:
    CartpoleDynamics model = CartpoleDynamics(0.1, 1.0, 1.0, 1.0);
    CartpoleQuadraticCost cost;
    float dt = 0.01;
    int max_iter = 10;
    float gamma = 0.5;
};



TEST_F(Cartpole_VanillaMPPI, BindToStream) {
    const int num_timesteps = 100;
    const int num_rollouts = 256;

    std::array<float, CartpoleDynamics::CONTROL_DIM> control_var = {2.5};
    std::array<float, CartpoleDynamics::CONTROL_DIM * num_timesteps> init_control = {0};
    hipStream_t stream;

    HANDLE_ERROR(hipStreamCreate(&stream));

    auto CartpoleController = VanillaMPPIController<CartpoleDynamics, CartpoleQuadraticCost, num_timesteps, num_rollouts, 64, 8>(&model, &cost,
                                                                                                                                 dt, max_iter, gamma, num_timesteps, control_var, init_control, stream);

    EXPECT_EQ(CartpoleController.stream_, CartpoleController.model_->stream_)
                        << "Stream bind to dynamics failure";
    EXPECT_EQ(CartpoleController.stream_, CartpoleController.cost_->stream_)
                        << "Stream bind to cost failure";
    HANDLE_ERROR(hipStreamDestroy(stream));
}

TEST_F(Cartpole_VanillaMPPI, UpdateNoiseVariance) {
    const int num_timesteps = 150;
    const int num_rollouts = 512;
    std::array<float, CartpoleDynamics::CONTROL_DIM> control_var = {1.5};
    std::array<float, CartpoleDynamics::CONTROL_DIM> new_control_var = {3.5};

    auto CartpoleController = VanillaMPPIController<CartpoleDynamics, CartpoleQuadraticCost, num_timesteps, num_rollouts, 64, 8>(&model, &cost,
                                                                                                                                 dt, max_iter, gamma, num_timesteps, control_var);

    CartpoleController.updateControlNoiseVariance(new_control_var);

    EXPECT_FLOAT_EQ(new_control_var[0], CartpoleController.getControlVariance()[0]);
}

TEST_F(Cartpole_VanillaMPPI, SwingUpTest) {
    cartpoleQuadraticCostParams new_params;
    new_params.cart_position_coeff = 100;
    new_params.pole_angle_coeff = 200;
    new_params.cart_velocity_coeff = 10;
    new_params.pole_angular_velocity_coeff = 20;
    new_params.control_force_coeff = 1;
    new_params.terminal_cost_coeff = 0;
    new_params.desired_terminal_state[0] = -20;
    new_params.desired_terminal_state[1] = 0;
    new_params.desired_terminal_state[2] = M_PI;
    new_params.desired_terminal_state[3] = 0;

    cost.setParams(new_params);


    float dt = 0.01;
    int max_iter = 1;
    float gamma = 0.25;
    int num_timesteps = 100;

    std::array<float, CartpoleDynamics::CONTROL_DIM> control_var = {5.0};

    auto controller = VanillaMPPIController<CartpoleDynamics, CartpoleQuadraticCost, 100, 2048, 64, 8>(&model, &cost,
                                                                                                       dt, max_iter, gamma, num_timesteps, control_var);
    Eigen::MatrixXf current_state(4, 1);
    int time_horizon = 1000;

    //float xdot[CartpoleDynamics::STATE_DIM];
    Eigen::MatrixXf xdot(4, 1);

    auto time_start = std::chrono::system_clock::now();
    for (int i =0; i < time_horizon; ++i) {
        if (i % 50 == 0) {
            printf("Current Time: %f    ", i * dt);
            printf("Current Baseline Cost: %f    ", controller.getBaselineCost());
            model.printState(current_state.data());
        }

        // Compute the control
        controller.computeControl(current_state);

        // Increment the state
        model.xDot(current_state.data(), &controller.getControlSeq()[0], xdot);
        model.updateState(current_state.data(), xdot, dt);

        controller.slideControlSequence(1);

    }

    EXPECT_LT(controller.getBaselineCost(), 1.0);
}


