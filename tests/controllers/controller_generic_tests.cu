#include "hip/hip_runtime.h"
#include <mppi/controllers/controller.cuh>
#include <mppi_test/mock_classes/mock_costs.h>
#include <mppi_test/mock_classes/mock_dynamics.h>

#include <gtest/gtest.h>
#include <gmock/gmock.h>
#include <mppi/utils/test_helper.h>
#include <random>
#include <algorithm>
#include <numeric>


static const int number_rollouts = 1200;

class TestController : public Controller<MockDynamics, MockCost, 100, number_rollouts, 1, 2>{
public:
  TestController(MockDynamics* model, MockCost* cost, float dt, int max_iter, float gamma,
                 const Eigen::Ref<const control_array>& control_variance,
                 int num_timesteps = 100,
                 const Eigen::Ref<const control_trajectory>& init_control_traj = control_trajectory::Zero(),
                 hipStream_t stream = nullptr) : Controller<MockDynamics, MockCost, 100, number_rollouts, 1, 2>(
                         model, cost, dt, max_iter, gamma, control_variance, num_timesteps,
                         init_control_traj, stream) {

    // Allocate CUDA memory for the controller
    allocateCUDAMemoryHelper(0);

    // Copy the noise variance to the device
    this->copyControlStdDevToDevice();
  }

  virtual void computeControl(const Eigen::Ref<const state_array>& state) override {

  }

  void computeControl(const Eigen::Ref<const state_array>& state,
                      const std::array<control_trajectory, number_rollouts> noise) {
    int trajectory_size = control_trajectory().size();
    for (int i = 0; i < number_rollouts; i++) {
      HANDLE_ERROR(hipMemcpyAsync(control_noise_d_ + i * trajectory_size,
                                   noise[i].data(),
                                   sizeof(float)*trajectory_size,
                                   hipMemcpyHostToDevice, stream_));
    }
    HANDLE_ERROR(hipStreamSynchronize(stream_));
    // Normally rolloutKernel would be called here and would transform
    //  control_noise_d_ from u to u + noise

    // Instead we just get back noise in this test
    this->copySampledControlFromDevice();
  }

  virtual void slideControlSequence(int steps) override {

  }

  float getDt() {return dt_;}
  int getNumIter() {return num_iters_;}
  float getGamma() {return gamma_;}
  float getNumTimesteps() {return num_timesteps_;}
  hipStream_t getStream() {return stream_;}

  void setFeedbackGains(TestController::feedback_gain_trajectory traj) {
    this->result_.feedback_gain = traj;
  }
};

TEST(Controller, ConstructorDestructor) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);
  int num_timesteps = 10;
  hipStream_t stream;
  HANDLE_ERROR(hipStreamCreate(&stream));

  TestController::control_trajectory init_control_trajectory = TestController::control_trajectory::Ones();

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(stream)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(stream)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController* controller = new TestController(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var, num_timesteps,
          init_control_trajectory, stream);

  EXPECT_EQ(controller->model_, &mockDynamics);
  EXPECT_EQ(controller->cost_, &mockCost);
  EXPECT_EQ(controller->getDt(), dt);
  EXPECT_EQ(controller->getNumIter(), max_iter);
  EXPECT_EQ(controller->getGamma(), gamma);
  EXPECT_EQ(controller->getNumTimesteps(), num_timesteps);
  EXPECT_EQ(controller->getControlStdDev(), control_var);
  EXPECT_EQ(controller->getControlSeq(), init_control_trajectory);
  EXPECT_EQ(controller->getStream(), stream);
  EXPECT_EQ(controller->getFeedbackEnabled(), false);

  // TODO check that a random seed was set and stream was set
  //EXPECT_NE(controller->getRandomSeed(), 0);

  // TODO check for correct defaults
  delete controller;
}

TEST(Controller, setNumTimesteps) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);

  controller.setNumTimesteps(10);
  EXPECT_EQ(controller.getNumTimesteps(), 10);

  controller.setNumTimesteps(1000);
  EXPECT_EQ(controller.getNumTimesteps(), 100);
}


TEST(Controller, updateControlNoiseStdDev) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);

  TestController::control_array new_control_var = TestController::control_array::Ones();

  controller.updateControlNoiseStdDev(new_control_var);

  EXPECT_EQ(controller.getControlStdDev(), new_control_var);
  // TODO verify copied to GPU correctly
}

TEST(Controller, slideControlSequenceHelper) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);
  TestController::control_trajectory u;
  for(int i = 0; i < controller.num_timesteps_; i++) {
    TestController::control_array control = TestController::control_array::Ones();
    control = control * i;
    u.col(i) = control.transpose();
  }

  controller.slideControlSequenceHelper(1, u);
  for(int i = 0; i < controller.num_timesteps_; i++) {
    for(int j = 0; j < MockDynamics::CONTROL_DIM; j++) {
      int val = std::min(i + 1, controller.num_timesteps_ - 1);
      EXPECT_FLOAT_EQ(u(j, i), val);
    }
  }

  controller.slideControlSequenceHelper(10, u);
  for(int i = 0; i < controller.num_timesteps_; i++) {
    for(int j = 0; j < MockDynamics::CONTROL_DIM; j++) {
      int val = std::min(i + 11, controller.num_timesteps_ - 1);
      EXPECT_FLOAT_EQ(u(j, i), val);
    }
  }
}

TEST(Controller, computeStateTrajectoryHelper) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);

  TestController::state_array x = TestController::state_array::Ones();
  TestController::state_array xdot = TestController::state_array::Ones();
  EXPECT_CALL(mockDynamics, computeStateDeriv(testing::_, testing::_, testing::_)).Times(controller.num_timesteps_ - 1);
  EXPECT_CALL(mockDynamics, updateState(testing::_, testing::_, dt)).Times(controller.num_timesteps_ - 1);
  
  TestController::state_trajectory result = TestController::state_trajectory::Ones();
  TestController::control_trajectory u = TestController::control_trajectory::Zero();
  controller.computeStateTrajectoryHelper(result, x, u);

  for(int i = 0; i < controller.num_timesteps_; i++) {
    for(int j = 0; j < MockDynamics::STATE_DIM; j++){
      EXPECT_FLOAT_EQ(result(j, i), 1.0);
    }
  }
}

TEST(Controller, interpolateControl) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);
  TestController::control_trajectory traj;
  for(int i = 0; i < controller.getNumTimesteps(); i++) {
    traj.col(i) = TestController::control_array::Ones() * i;
  }
  controller.updateImportanceSampler(traj);

  for(double i = 0; i < controller.getNumTimesteps() - 1; i+= 0.25) {
    TestController::control_array result = controller.interpolateControls(i*controller.getDt());
    EXPECT_FLOAT_EQ(result(0), i) << i;
  }
}

TEST(Controller, interpolateFeedback) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);

  controller.setFeedbackController(true);
  TestController::feedback_gain_trajectory feedback_traj = TestController::feedback_gain_trajectory(controller.getNumTimesteps());
  for(int i = 0; i < controller.getNumTimesteps(); i++) {
    feedback_traj[i] = Eigen::Matrix<float, 1, 1>::Ones() * i;
  }
  controller.setFeedbackGains(feedback_traj);

  TestController::state_array state = TestController::state_array::Ones();
  for(double i = 0; i < controller.getNumTimesteps() - 1; i += 0.25) {
    TestController::control_array result = controller.interpolateFeedback(state, i*controller.getDt());
    EXPECT_FLOAT_EQ(result(0), i);
  }
}


TEST(Controller, getCurrentControlTest) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);

  EXPECT_CALL(mockDynamics, enforceConstraints(testing::_, testing::_)).Times(4 * (controller.getNumTimesteps() - 1));

  controller.setFeedbackController(true);
  TestController::feedback_gain_trajectory feedback_traj = TestController::feedback_gain_trajectory(controller.getNumTimesteps());
  TestController::control_trajectory traj;
  for(int i = 0; i < controller.getNumTimesteps(); i++) {
    feedback_traj[i] = Eigen::Matrix<float, 1, 1>::Ones() * i;
    traj.col(i) = TestController::control_array::Ones() * i;
  }
  controller.setFeedbackGains(feedback_traj);
  controller.updateImportanceSampler(traj);

  TestController::state_array state = TestController::state_array::Ones();
  for(double i = 0; i < controller.getNumTimesteps() - 1; i += 0.25) {
    TestController::control_array result = controller.getCurrentControl(state, i*controller.getDt());
    EXPECT_FLOAT_EQ(result(0), i*2);
  }
}

TEST(Controller, getSampledControlTrajectories) {
  // Create controller
  // Use computeControl with noise passed in
  // Inside computeControl copySampledControlFromDevice is used
  // Get sampled control sequence
  // Compare to original noise
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  MockDynamics::control_array control_var;
  control_var = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_var);

  // Create noisy trajectories./
  std::array<TestController::control_trajectory, number_rollouts> noise;
  for(int i = 0; i < number_rollouts; i++) {
    noise[i] = TestController::control_trajectory::Random();
  }
  // Save back a percentage of trajectories
  controller.setPercentageSampledControlTrajectories(0.3);

  TestController::state_array x = TestController::state_array::Ones();
  controller.computeControl(x, noise);
  std::vector<TestController::control_trajectory> sampled_controls = controller.getSampledControlSeq();
  int j;
  float total_difference;
  for (int i = 0; i < sampled_controls.size(); i++) {
    float diff = -1;
    // Need to find which noise trajectory the current sample matches
    for (j = 0; j < number_rollouts; j++){
      diff = std::abs((noise[j] - sampled_controls[i]).norm());
      if (diff == 0) {
        break;
      }
    }
    total_difference += diff;
  }
  EXPECT_FLOAT_EQ(0, total_difference);
}

TEST(Controller, saveControlHistoryHelper_1) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  int steps = 1;
  MockDynamics::control_array control_std_dev;
  control_std_dev = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_std_dev);

  TestController::control_trajectory u = TestController::control_trajectory::Random();
  Eigen::Matrix<float, MockDynamics::CONTROL_DIM, 2> u_history;
  u_history.setOnes();

  controller.saveControlHistoryHelper(steps, u, u_history);

  for (int i = 0; i < MockDynamics::CONTROL_DIM; ++i) {
    EXPECT_FLOAT_EQ(u_history(i, 0), 1.0f) << "History column 0 failed";
    EXPECT_FLOAT_EQ(u_history(i, 1), u(i, steps-1)) << "History column 1 failed";
  }
}

TEST(Controller, saveControlHistoryHelper_2) {
  MockCost mockCost;
  MockDynamics mockDynamics;

  float dt = 0.1;
  int max_iter = 1;
  float gamma = 1.2;
  int steps = 4;
  MockDynamics::control_array control_std_dev;
  control_std_dev = MockDynamics::control_array::Constant(1.0);

  // expect double check rebind
  EXPECT_CALL(mockCost, bindToStream(testing::_)).Times(1);
  EXPECT_CALL(mockDynamics, bindToStream(testing::_)).Times(1);

  // expect GPU setup called again
  EXPECT_CALL(mockCost, GPUSetup()).Times(1);
  EXPECT_CALL(mockDynamics, GPUSetup()).Times(1);

  TestController controller(&mockDynamics, &mockCost, dt, max_iter, gamma, control_std_dev);

  TestController::control_trajectory u = TestController::control_trajectory::Random();
  Eigen::Matrix<float, MockDynamics::CONTROL_DIM, 2> u_history;
  u_history.setOnes();

  controller.saveControlHistoryHelper(steps, u, u_history);

  for (int i = 0; i < MockDynamics::CONTROL_DIM; ++i) {
    EXPECT_FLOAT_EQ(u_history(i, 0), u(i, steps-2)) << "History column 0 failed";
    EXPECT_FLOAT_EQ(u_history(i, 1), u(i, steps-1)) << "History column 1 failed";
  }
}
