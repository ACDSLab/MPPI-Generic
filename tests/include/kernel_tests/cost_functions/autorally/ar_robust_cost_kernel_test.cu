#include "hip/hip_runtime.h"
//
// Created by jgibson37 on 2/7/20.
//

template <class CLASS_T>
__global__ void getCostmapCostTestKernel(CLASS_T* cost, float* test_xu, float* cost_results, int num_points)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num_points)
  {
    float* state = &test_xu[tid];
    cost_results[tid] = cost->getCostmapCost(state);
  }
}

template <class CLASS_T>
void launchGetCostmapCostTestKernel(CLASS_T& cost, std::vector<std::array<float, 9>>& test_xu,
                                    std::vector<float>& cost_results)
{
  int num_test_points = test_xu.size();
  cost_results.resize(num_test_points * 9);

  float* cost_results_d;
  float* test_xu_d;
  HANDLE_ERROR(hipMalloc((void**)&cost_results_d, sizeof(float) * num_test_points))
  HANDLE_ERROR(hipMalloc((void**)&test_xu_d, sizeof(float) * 9 * num_test_points))

  for (int i = 0; i < num_test_points; i++)
  {
    for (int j = 0; j < 9; j++)
    {
      cost_results[9 * i + j] = test_xu[i][j];
    }
  }

  HANDLE_ERROR(hipMemcpy(test_xu_d, test_xu.data(), sizeof(float) * 9 * num_test_points, hipMemcpyHostToDevice));

  // TODO amount should depend on the number of query points
  dim3 threadsPerBlock(num_test_points, 1);
  dim3 numBlocks(1, 1);
  getCostmapCostTestKernel<CLASS_T>
      <<<numBlocks, threadsPerBlock>>>(static_cast<CLASS_T*>(cost.cost_d_), test_xu_d, cost_results_d, num_test_points);
  CudaCheckError();
  hipDeviceSynchronize();

  // Copy the memory back to the host
  HANDLE_ERROR(
      hipMemcpy(cost_results.data(), cost_results_d, sizeof(float) * num_test_points, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  hipFree(cost_results_d);
  hipFree(test_xu_d);
}
