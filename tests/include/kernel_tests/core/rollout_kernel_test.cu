#include "hip/hip_runtime.h"
#include "rollout_kernel_test.cuh"

#include <mppi/dynamics/cartpole/cartpole_dynamics.cuh>
#include <mppi/cost_functions/cartpole/cartpole_quadratic_cost.cuh>

// const int STATE_DIM = 12;
// const int CONTROL_DIM = 3;
// const int NUM_ROLLOUTS = 100; // .99 times this number has to be an integer... TODO fix how brittle this is
// const int BLOCKSIZE_X = 32;
// const int BLOCKSIZE_Y = 8; // Blocksize_y has to be greater than the control dim TODO fix how we step through the
// controls

template <int BLOCKSIZE_Z>
__global__ void loadGlobalToShared_KernelTest(float* x0_device, float* x_thread_device, float* xdot_thread_device,
                                              float* u_thread_device)
{
  const int STATE_DIM = 12;
  const int CONTROL_DIM = 3;
  const int NUM_ROLLOUTS = 100;  // .99 times this number has to be an integer... TODO fix how brittle this is
  const int BLOCKSIZE_X = 32;
  const int BLOCKSIZE_Y =
      8;  // Blocksize_y has to be greater than the control dim TODO fix how we step through the controls

  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = threadIdx.x + block_idx * blockDim.x;

  // Create shared arrays which hold state and control data
  __shared__ float x_shared[BLOCKSIZE_X * STATE_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * CONTROL_DIM * BLOCKSIZE_Z];

  float* x_thread;
  float* xdot_thread;

  float* u_thread;
  float* du_thread;

  if (global_idx < NUM_ROLLOUTS)
  {
    x_thread = &x_shared[(blockDim.x * thread_idz + thread_idx) * STATE_DIM];
    xdot_thread = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * STATE_DIM];
    u_thread = &u_shared[(blockDim.x * thread_idz + thread_idx) * CONTROL_DIM];
  }
  __syncthreads();
  mppi::kernels::loadGlobalToShared<STATE_DIM, CONTROL_DIM>(NUM_ROLLOUTS, BLOCKSIZE_Y, global_idx, thread_idy,
                                                            thread_idz, x0_device, x_thread, xdot_thread, u_thread);
  __syncthreads();

  // Check if on the first rollout the correct values were copied over
  // Prevent y threads from all writing to the same memory
  if (global_idx == 1 && thread_idy == 0)
  {
    for (int i = 0; i < STATE_DIM; ++i)
    {
      int ind = i + thread_idz * STATE_DIM;
      int ind_thread = i + thread_idz * STATE_DIM * blockDim.x;
      x_thread_device[ind] = x_shared[ind_thread];
      xdot_thread_device[ind] = xdot_shared[ind_thread];
    }

    for (int i = 0; i < CONTROL_DIM; ++i)
    {
      int ind = i + thread_idz * CONTROL_DIM;
      int ind_thread = i + thread_idz * CONTROL_DIM * blockDim.x;
      u_thread_device[ind] = u_shared[ind_thread];
    }
    __syncthreads();
  }

  // To test what the results are, we have to return them back to the host.
}

void launchGlobalToShared_KernelTest(const std::vector<float>& x0_host, std::vector<float>& x_thread_host,
                                     std::vector<float>& xdot_thread_host, std::vector<float>& u_thread_host)
{
  const int STATE_DIM = 12;
  const int CONTROL_DIM = 3;
  const int NUM_ROLLOUTS = 100;  // .99 times this number has to be an integer... TODO fix how brittle this is
  const int BLOCKSIZE_X = 32;
  const int BLOCKSIZE_Y =
      8;  // Blocksize_y has to be greater than the control dim TODO fix how we step through the controls

  // Define the initial condition x0_device and the exploration variance in global device memory
  float* x0_device;
  HANDLE_ERROR(hipMalloc((void**)&x0_device, sizeof(float) * STATE_DIM));
  HANDLE_ERROR(hipMemcpy(x0_device, x0_host.data(), sizeof(float) * STATE_DIM, hipMemcpyHostToDevice));

  // Define the return arguments in global device memory
  float* x_thread_device;
  float* xdot_thread_device;
  float* u_thread_device;

  HANDLE_ERROR(hipMalloc((void**)&x_thread_device, sizeof(float) * STATE_DIM));
  HANDLE_ERROR(hipMalloc((void**)&xdot_thread_device, sizeof(float) * STATE_DIM));
  HANDLE_ERROR(hipMalloc((void**)&u_thread_device, sizeof(float) * CONTROL_DIM));

  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y);
  dim3 dimGrid(2048);

  loadGlobalToShared_KernelTest<<<dimGrid, dimBlock>>>(x0_device, x_thread_device, xdot_thread_device, u_thread_device);
  CudaCheckError();

  // Copy the data back to the host
  HANDLE_ERROR(hipMemcpy(x_thread_host.data(), x_thread_device, sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(xdot_thread_host.data(), xdot_thread_device, sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(u_thread_host.data(), u_thread_device, sizeof(float) * CONTROL_DIM, hipMemcpyDeviceToHost));

  // Free the cuda memory that we allocated
  hipFree(x0_device);

  hipFree(x_thread_device);
  hipFree(xdot_thread_device);
  hipFree(u_thread_device);
}

/**
 * Tube-MPPI versions of the kernel tests
 */

// This is to test tube-mppi calls to the kernel
void launchGlobalToShared_KernelTest_nom_act(
    const std::vector<float>& x0_host_act, std::vector<float>& x_thread_host_act,
    std::vector<float>& xdot_thread_host_act, std::vector<float>& u_thread_host_act,
    const std::vector<float>& x0_host_nom, std::vector<float>& x_thread_host_nom,
    std::vector<float>& xdot_thread_host_nom, std::vector<float>& u_thread_host_nom)
{
  const int STATE_DIM = 12;
  const int CONTROL_DIM = 3;
  const int NUM_ROLLOUTS = 100;  // .99 times this number has to be an integer... TODO fix how brittle this is
  const int BLOCKSIZE_X = 32;
  const int BLOCKSIZE_Y =
      8;  // Blocksize_y has to be greater than the control dim TODO fix how we step through the controls

  // Define the initial condition x0_device and the exploration variance in global device memory
  // Need twice as much memory for tube-mppi
  float* x0_device;
  HANDLE_ERROR(hipMalloc((void**)&x0_device, sizeof(float) * STATE_DIM * 2));

  // Copy both act and nominal initial state
  HANDLE_ERROR(hipMemcpy(x0_device, x0_host_act.data(), sizeof(float) * STATE_DIM, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(x0_device + STATE_DIM, x0_host_nom.data(), sizeof(float) * STATE_DIM, hipMemcpyHostToDevice));

  // Define the return arguments in global device memory
  float* x_thread_device;
  float* xdot_thread_device;
  float* u_thread_device;

  HANDLE_ERROR(hipMalloc((void**)&x_thread_device, sizeof(float) * STATE_DIM * 2));
  HANDLE_ERROR(hipMalloc((void**)&xdot_thread_device, sizeof(float) * STATE_DIM * 2));
  HANDLE_ERROR(hipMalloc((void**)&u_thread_device, sizeof(float) * CONTROL_DIM * 2));

  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 2);
  dim3 dimGrid(100);

  loadGlobalToShared_KernelTest<2>
      <<<dimGrid, dimBlock>>>(x0_device, x_thread_device, xdot_thread_device, u_thread_device);
  CudaCheckError();

  // Copy the initial_state for actual and nominal
  HANDLE_ERROR(
      hipMemcpy(x_thread_host_act.data(), x_thread_device, sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(x_thread_host_nom.data(), x_thread_device + STATE_DIM, sizeof(float) * STATE_DIM,
                          hipMemcpyDeviceToHost));
  // Copy the xdot for actual and nominal
  HANDLE_ERROR(
      hipMemcpy(xdot_thread_host_act.data(), xdot_thread_device, sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(xdot_thread_host_nom.data(), xdot_thread_device + STATE_DIM, sizeof(float) * STATE_DIM,
                          hipMemcpyDeviceToHost));
  // copy the initial u for actual and nominal
  HANDLE_ERROR(
      hipMemcpy(u_thread_host_act.data(), u_thread_device, sizeof(float) * CONTROL_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(u_thread_host_nom.data(), u_thread_device + CONTROL_DIM, sizeof(float) * CONTROL_DIM,
                          hipMemcpyDeviceToHost));

  // Free the cuda memory that we allocated
  hipFree(x0_device);

  hipFree(x_thread_device);
  hipFree(xdot_thread_device);
  hipFree(u_thread_device);
}

template <class COST_T>
__global__ void computeAndSaveCostAllRollouts_KernelTest(COST_T* cost, int state_dim, int num_rollouts,
                                                         float* running_costs, float* terminal_state,
                                                         float* cost_rollout_device)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;  // index on rollouts
                                                    //    if (tid == 0) {
  //        printf("Current state [%f, %f, %f, %f]\n", terminal_state[state_dim * tid],
  //               terminal_state[state_dim * tid + 1], terminal_state[state_dim * tid + 2],
  //               terminal_state[state_dim * tid + 3]);
  //        printf("Current cost [%f]\n", running_costs[tid]);
  //    }
  mppi_common::computeAndSaveCost(num_rollouts, 2, tid, cost, &terminal_state[state_dim * tid], running_costs[tid],
                                  nullptr, cost_rollout_device);
  //    if (tid == 0) {
  //        printf("Total cost [%f]\n", cost_rollout_device[tid]);
  //    }
}

template <class COST_T, int STATE_DIM, int NUM_ROLLOUTS>
void launchComputeAndSaveCostAllRollouts_KernelTest(COST_T& cost,
                                                    const std::array<float, NUM_ROLLOUTS>& cost_all_rollouts,
                                                    const std::array<float, STATE_DIM * NUM_ROLLOUTS>& terminal_states,
                                                    std::array<float, NUM_ROLLOUTS>& cost_compute)
{
  const int BLOCKSIZE_X = 32;
  const int BLOCKSIZE_Y = 8;

  // Allocate CUDA memory
  float* cost_all_rollouts_device;
  float* terminal_states_device;
  float* cost_compute_device;

  HANDLE_ERROR(hipMalloc((void**)&cost_all_rollouts_device, sizeof(float) * cost_all_rollouts.size()));
  HANDLE_ERROR(hipMalloc((void**)&terminal_states_device, sizeof(float) * terminal_states.size()));
  HANDLE_ERROR(hipMalloc((void**)&cost_compute_device, sizeof(float) * cost_compute.size()));

  // Copy Host side data to the Device
  HANDLE_ERROR(hipMemcpy(cost_all_rollouts_device, cost_all_rollouts.data(), sizeof(float) * cost_all_rollouts.size(),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(terminal_states_device, terminal_states.data(), sizeof(float) * terminal_states.size(),
                          hipMemcpyHostToDevice));

  // Launch kernel
  dim3 blocksize(BLOCKSIZE_X, 1);
  dim3 gridsize((NUM_ROLLOUTS + (BLOCKSIZE_X - 1)) / BLOCKSIZE_X, 1);
  computeAndSaveCostAllRollouts_KernelTest<<<blocksize, gridsize>>>(
      cost.cost_d_, STATE_DIM, NUM_ROLLOUTS, cost_all_rollouts_device, terminal_states_device, cost_compute_device);
  CudaCheckError();

  // Copy Device side data to the host
  HANDLE_ERROR(hipMemcpy(cost_compute.data(), cost_compute_device, sizeof(float) * cost_compute.size(),
                          hipMemcpyDeviceToHost));

  // free cuda Memory
  hipFree(cost_all_rollouts_device);
  hipFree(terminal_states_device);
  hipFree(cost_compute_device);
}

template <class DYN_T, class COST_T, class SAMPLER_T>
void launchRolloutKernel_nom_act(DYN_T* dynamics, COST_T* costs, SAMPLER_T* sampler, float dt, const int num_timesteps,
                                 const int num_rollouts, float lambda, float alpha, const std::vector<float>& x0,
                                 const std::vector<float>& nom_control_seq, std::vector<float>& trajectory_costs_act,
                                 std::vector<float>& trajectory_costs_nom, hipStream_t stream)
{
  float* initial_state_d;
  float* trajectory_costs_d;

  const int BLOCKSIZE_X = 16;
  const int BLOCKSIZE_Y = 8;

  /**
   * Ensure dynamics, costs, and sampler exist on GPU
   */
  dynamics->bindToStream(stream);
  costs->bindToStream(stream);
  sampler->bindToStream(stream);
  // Call the GPU setup functions of the dynamics, costs, and sampler
  dynamics->GPUSetup();
  costs->GPUSetup();
  sampler->GPUSetup();

  sampler->setNumTimesteps(num_timesteps);
  sampler->setNumRollouts(num_rollouts);
  sampler->setNumDistributions(2);

  // Create x init cuda array
  HANDLE_ERROR(hipMalloc((void**)&initial_state_d, sizeof(float) * DYN_T::STATE_DIM * 2));
  // Create cost trajectory cuda array
  HANDLE_ERROR(hipMalloc((void**)&trajectory_costs_d, sizeof(float) * num_rollouts * 2));
  // Create random noise generator
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetStream(gen, stream);

  /**
   * Fill in GPU arrays
   */
  HANDLE_ERROR(
      hipMemcpyAsync(initial_state_d, x0.data(), sizeof(float) * DYN_T::STATE_DIM, hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(initial_state_d + DYN_T::STATE_DIM, x0.data(), sizeof(float) * DYN_T::STATE_DIM,
                               hipMemcpyHostToDevice, stream));

  sampler->copyImportanceSamplerToDevice(nom_control_seq.data(), 0, false);
  sampler->copyImportanceSamplerToDevice(nom_control_seq.data(), 1, false);
  // Generate samples and do stream synchronize
  sampler->generateSamples(1, 0, gen, true);
  dim3 threadsPerBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 2);
  // Launch rollout kernel
  mppi::kernels::launchRolloutKernel<DYN_T, COST_T, SAMPLER_T>(dynamics, costs, sampler, dt, num_timesteps,
                                                               num_rollouts, lambda, alpha, initial_state_d,
                                                               trajectory_costs_d, threadsPerBlock, stream, false);

  // Copy the costs back to the host
  HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_act.data(), trajectory_costs_d, num_rollouts * sizeof(float),
                               hipMemcpyDeviceToHost, stream));

  HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nom.data(), trajectory_costs_d + num_rollouts,
                               num_rollouts * sizeof(float), hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));

  hipFree(initial_state_d);
  hipFree(trajectory_costs_d);
}

template <class DYNAMICS_T, class COSTS_T, int NUM_ROLLOUTS, int BLOCKSIZE_X, int BLOCKSIZE_Y>
__global__ void autorallyRolloutKernel(int num_timesteps, float* state_d, float* U_d, float* du_d, float* nu_d,
                                       float* costs_d, DYNAMICS_T* dynamics_model, COSTS_T* mppi_costs, int opt_delay,
                                       float lambda, float alpha, float dt)
{
  int i, j;
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  int bdx = blockIdx.x;

  // Initialize the local state, controls, and noise
  float* s;
  float* s_der;
  float* u;
  float* nu;
  float* du;
  int* crash;

  const int STATE_DIM = DYNAMICS_T::STATE_DIM;
  const int CONTROL_DIM = DYNAMICS_T::CONTROL_DIM;
  const int SHARED_MEM_REQUEST_GRD = DYNAMICS_T::SHARED_MEM_REQUEST_GRD;
  const int SHARED_MEM_REQUEST_BLK = DYNAMICS_T::SHARED_MEM_REQUEST_BLK;

  // Create shared arrays for holding state and control data.
  __shared__ float state_shared[BLOCKSIZE_X * STATE_DIM];
  __shared__ float state_der_shared[BLOCKSIZE_X * STATE_DIM];
  __shared__ float control_shared[BLOCKSIZE_X * CONTROL_DIM];
  __shared__ float control_var_shared[BLOCKSIZE_X * CONTROL_DIM];
  __shared__ float exploration_variance[BLOCKSIZE_X * CONTROL_DIM];
  __shared__ int crash_status[BLOCKSIZE_X];
  // Create a shared array for the dynamics model to use
  __shared__ float theta[SHARED_MEM_REQUEST_GRD / sizeof(float) + 1 + SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X];
  __shared__ float theta_c[COSTS_T::SHARED_MEM_REQUEST_GRD + COSTS_T::SHARED_MEM_REQUEST_BLK * BLOCKSIZE_X];
  __shared__ float y[DYNAMICS_T::OUTPUT_DIM];

  // Initialize trajectory cost
  float running_cost = 0;

  int global_idx = BLOCKSIZE_X * bdx + tdx;
  if (global_idx < NUM_ROLLOUTS)
  {
    // Portion of the shared array belonging to each x-thread index.
    s = &state_shared[tdx * STATE_DIM];
    s_der = &state_der_shared[tdx * STATE_DIM];
    u = &control_shared[tdx * CONTROL_DIM];
    du = &control_var_shared[tdx * CONTROL_DIM];
    nu = &exploration_variance[tdx * CONTROL_DIM];
    crash = &crash_status[tdx];
    // Load the initial state, nu, and zero the noise
    for (i = tdy; i < STATE_DIM; i += blockDim.y)
    {
      s[i] = state_d[i];
      s_der[i] = 0;
    }
    // Load nu
    for (i = tdy; i < CONTROL_DIM; i += blockDim.y)
    {
      u[i] = 0;
      du[i] = 0;
      nu[i] = nu_d[i];
    }
    crash[0] = 0;
  }
  __syncthreads();
  /*<----Start of simulation loop-----> */
  dynamics_model->initializeDynamics(s, u, y, theta, 0.0, dt);
  mppi_costs->initializeCosts(s, u, theta_c, 0.0, dt);
  for (i = 0; i < num_timesteps; i++)
  {
    if (global_idx < NUM_ROLLOUTS)
    {
      for (j = tdy; j < CONTROL_DIM; j += blockDim.y)
      {
        // Noise free rollout
        if (global_idx == 0 || i < opt_delay)
        {  // Don't optimize variables that are already being executed
          du[j] = 0.0;
          u[j] = U_d[i * CONTROL_DIM + j];
        }
        else if (global_idx >= .99 * NUM_ROLLOUTS)
        {
          du[j] = du_d[CONTROL_DIM * num_timesteps * (BLOCKSIZE_X * bdx + tdx) + i * CONTROL_DIM + j] * nu[j];
          u[j] = du[j];
        }
        else
        {
          du[j] = du_d[CONTROL_DIM * num_timesteps * (BLOCKSIZE_X * bdx + tdx) + i * CONTROL_DIM + j] * nu[j];
          u[j] = U_d[i * CONTROL_DIM + j] + du[j];
        }
        du_d[CONTROL_DIM * num_timesteps * (BLOCKSIZE_X * bdx + tdx) + i * CONTROL_DIM + j] = u[j];
      }
    }
    __syncthreads();
    dynamics_model->enforceConstraints(s, &du_d[(global_idx * num_timesteps + i) * CONTROL_DIM]);
    if (tdy == 0 && global_idx < NUM_ROLLOUTS)
    {
      dynamics_model->enforceConstraints(s, u);
    }
    __syncthreads();
    // Compute the cost of the being in the current state
    if (tdy == 0 && global_idx < NUM_ROLLOUTS && i > 0 && crash[0] > -1)
    {
      // Running average formula
      running_cost +=
          (mppi_costs->computeRunningCost(s, u, du, nu, lambda, alpha, i, theta_c, crash) - running_cost) / (1.0 * i);
      //      printf("AutoRa Current State rollout %i, timestep: %i: [%f, %f, %f, %f]\n", global_idx, i, s[0], s[1],
      //      s[2], s[3]); printf("AutoRa Running Cost rollout %i, timestep: %i: %f\n", global_idx, i, running_cost);
    }
    // Compute the dynamics
    if (global_idx < NUM_ROLLOUTS)
    {
      dynamics_model->computeStateDeriv(s, u, s_der, theta);
    }
    __syncthreads();
    // Update the state
    if (global_idx < NUM_ROLLOUTS)
    {
      dynamics_model->updateState(s, s_der, dt);
    }
    //    //Check to see if the rollout will result in a (physical) crash.
    //    if (tdy == 0 && global_idx < NUM_ROLLOUTS) {
    //      mppi_costs.getCrash(s, crash);
    //    }
  }
  /* <------- End of the simulation loop ----------> */
  if (global_idx < NUM_ROLLOUTS && tdy == 0)
  {  // Write cost results back to global memory.
    costs_d[(BLOCKSIZE_X)*bdx + tdx] = running_cost + mppi_costs->terminalCost(s, theta_c);
  }
}

template <class DYNAMICS_T, class COSTS_T, int NUM_ROLLOUTS, int NUM_TIMESTEPS, int BLOCKSIZE_X, int BLOCKSIZE_Y>
void launchAutorallyRolloutKernelTest(
    DYNAMICS_T* dynamics, COSTS_T* costs, float dt, float lambda, float alpha,
    std::array<float, DYNAMICS_T::STATE_DIM> state_array,
    std::array<float, NUM_TIMESTEPS * DYNAMICS_T::CONTROL_DIM> control_array,
    std::array<float, NUM_TIMESTEPS * NUM_ROLLOUTS * DYNAMICS_T::CONTROL_DIM> control_noise_array,
    std::array<float, DYNAMICS_T::CONTROL_DIM> sigma_u, std::array<float, NUM_ROLLOUTS>& costs_out,
    std::array<float, NUM_TIMESTEPS * NUM_ROLLOUTS * DYNAMICS_T::CONTROL_DIM>& control_noise_out, int opt_delay,
    hipStream_t stream)
{
  float* state_d;
  float* U_d;
  float* du_d;
  float* nu_d;
  float* costs_d;

  // Allocate CUDA memory for the rollout
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * state_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&U_d, sizeof(float) * control_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&du_d, sizeof(float) * control_noise_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&nu_d, sizeof(float) * sigma_u.size()));
  HANDLE_ERROR(hipMalloc((void**)&costs_d, sizeof(float) * costs_out.size()));

  // Copy the initial values
  HANDLE_ERROR(
      hipMemcpyAsync(state_d, state_array.data(), sizeof(float) * state_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(
      hipMemcpyAsync(U_d, control_array.data(), sizeof(float) * control_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(du_d, control_noise_array.data(), sizeof(float) * control_noise_array.size(),
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(nu_d, sigma_u.data(), sizeof(float) * sigma_u.size(), hipMemcpyHostToDevice, stream));

  const int GRIDSIZE_X = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;

  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
  dim3 dimGrid(GRIDSIZE_X, 1, 1);

  autorallyRolloutKernel<DYNAMICS_T, COSTS_T, NUM_ROLLOUTS, BLOCKSIZE_X, BLOCKSIZE_Y>
      <<<dimGrid, dimBlock, 0, stream>>>(NUM_TIMESTEPS, state_d, U_d, du_d, nu_d, costs_d, dynamics->model_d_,
                                         costs->cost_d_, opt_delay, lambda, alpha, dt);

  CudaCheckError();

  // Copy data back
  HANDLE_ERROR(
      hipMemcpyAsync(costs_out.data(), costs_d, sizeof(float) * costs_out.size(), hipMemcpyDeviceToHost, stream));

  // Copy the noise back
  HANDLE_ERROR(hipMemcpyAsync(control_noise_out.data(), du_d, sizeof(float) * control_noise_out.size(),
                               hipMemcpyDeviceToHost, stream));

  // Deallocate CUDA Memory
  HANDLE_ERROR(hipFree(state_d));
  HANDLE_ERROR(hipFree(U_d));
  HANDLE_ERROR(hipFree(du_d));
  HANDLE_ERROR(hipFree(nu_d));
  HANDLE_ERROR(hipFree(costs_d));
}

template <class DYNAMICS_T, class COSTS_T, int NUM_ROLLOUTS, int NUM_TIMESTEPS, int BLOCKSIZE_X, int BLOCKSIZE_Y>
void launchGenericRolloutKernelTest(
    DYNAMICS_T* dynamics, COSTS_T* costs, float dt, float lambda, float alpha,
    std::array<float, DYNAMICS_T::STATE_DIM> state_array,
    std::array<float, NUM_TIMESTEPS * DYNAMICS_T::CONTROL_DIM> control_array,
    std::array<float, NUM_TIMESTEPS * NUM_ROLLOUTS * DYNAMICS_T::CONTROL_DIM> control_noise_array,
    std::array<float, DYNAMICS_T::CONTROL_DIM> sigma_u, std::array<float, NUM_ROLLOUTS>& costs_out,
    std::array<float, NUM_TIMESTEPS * NUM_ROLLOUTS * DYNAMICS_T::CONTROL_DIM>& control_noise_out, int opt_delay,
    hipStream_t stream)
{
  float* state_d;
  float* U_d;
  float* du_d;
  float* nu_d;
  float* costs_d;

  // Allocate CUDA memory for the rollout
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * state_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&U_d, sizeof(float) * control_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&du_d, sizeof(float) * control_noise_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&nu_d, sizeof(float) * sigma_u.size()));
  HANDLE_ERROR(hipMalloc((void**)&costs_d, sizeof(float) * costs_out.size()));

  // Copy the initial values
  HANDLE_ERROR(
      hipMemcpyAsync(state_d, state_array.data(), sizeof(float) * state_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(
      hipMemcpyAsync(U_d, control_array.data(), sizeof(float) * control_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(du_d, control_noise_array.data(), sizeof(float) * control_noise_array.size(),
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(nu_d, sigma_u.data(), sizeof(float) * sigma_u.size(), hipMemcpyHostToDevice, stream));

  const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
  dim3 dimGrid(gridsize_x, 1, 1);
  mppi_common::rolloutKernel<DYNAMICS_T, COSTS_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS, 1>
      <<<dimGrid, dimBlock, 0, stream>>>(dynamics->model_d_, costs->cost_d_, dt, NUM_TIMESTEPS, opt_delay, lambda,
                                         alpha, state_d, U_d, du_d, nu_d, costs_d);
  CudaCheckError();

  // Copy data back
  HANDLE_ERROR(
      hipMemcpyAsync(costs_out.data(), costs_d, sizeof(float) * costs_out.size(), hipMemcpyDeviceToHost, stream));

  // Copy the noise back
  HANDLE_ERROR(hipMemcpyAsync(control_noise_out.data(), du_d, sizeof(float) * control_noise_out.size(),
                               hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));

  // Deallocate CUDA Memory
  HANDLE_ERROR(hipFree(state_d));
  HANDLE_ERROR(hipFree(U_d));
  HANDLE_ERROR(hipFree(du_d));
  HANDLE_ERROR(hipFree(nu_d));
  HANDLE_ERROR(hipFree(costs_d));
}

template <class DYNAMICS_T, class COSTS_T, int NUM_ROLLOUTS, int NUM_TIMESTEPS, int BLOCKSIZE_X, int BLOCKSIZE_Y>
void launchFastRolloutKernelTest(
    DYNAMICS_T* dynamics, COSTS_T* costs, float dt, float lambda, float alpha,
    std::array<float, DYNAMICS_T::STATE_DIM>& state_array,
    std::array<float, NUM_TIMESTEPS * DYNAMICS_T::CONTROL_DIM>& control_array,
    std::array<float, NUM_TIMESTEPS * NUM_ROLLOUTS * DYNAMICS_T::CONTROL_DIM>& control_noise_array,
    std::array<float, DYNAMICS_T::CONTROL_DIM> sigma_u, std::array<float, NUM_ROLLOUTS>& costs_out,
    std::array<float, NUM_TIMESTEPS * NUM_ROLLOUTS * DYNAMICS_T::CONTROL_DIM>& control_noise_out, int opt_delay,
    int state_traj_array_size, hipStream_t stream)
{
  float* state_d;
  float* U_d;
  float* du_d;
  float* nu_d;
  float* costs_d;
  float* x_d;

  // Allocate CUDA memory for the rollout
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * state_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&U_d, sizeof(float) * control_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&du_d, sizeof(float) * control_noise_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&nu_d, sizeof(float) * sigma_u.size()));
  HANDLE_ERROR(hipMalloc((void**)&costs_d, sizeof(float) * costs_out.size()));
  HANDLE_ERROR(hipMalloc((void**)&x_d, sizeof(float) * state_traj_array_size));

  // Copy the initial values
  HANDLE_ERROR(
      hipMemcpyAsync(state_d, state_array.data(), sizeof(float) * state_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(
      hipMemcpyAsync(U_d, control_array.data(), sizeof(float) * control_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(du_d, control_noise_array.data(), sizeof(float) * control_noise_array.size(),
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(nu_d, sigma_u.data(), sizeof(float) * sigma_u.size(), hipMemcpyHostToDevice, stream));

  mppi_common::launchFastRolloutKernel<DYNAMICS_T, COSTS_T, NUM_ROLLOUTS, BLOCKSIZE_X, BLOCKSIZE_Y>(
      dynamics, costs, dt, NUM_TIMESTEPS, opt_delay, lambda, alpha, state_d, x_d, U_d, du_d, nu_d, costs_d, stream,
      true);

  // Copy data back
  HANDLE_ERROR(
      hipMemcpyAsync(costs_out.data(), costs_d, sizeof(float) * costs_out.size(), hipMemcpyDeviceToHost, stream));

  // Copy the noise back
  HANDLE_ERROR(hipMemcpyAsync(control_noise_out.data(), du_d, sizeof(float) * control_noise_out.size(),
                               hipMemcpyDeviceToHost, stream));
  // HANDLE_ERROR(hipMemcpyAsync(state_traj_array.data(), x_d, sizeof(float) * state_traj_array.size(),
  //                              hipMemcpyDeviceToHost, stream));

  HANDLE_ERROR(hipStreamSynchronize(stream));

  // Deallocate CUDA Memory
  HANDLE_ERROR(hipFree(state_d));
  HANDLE_ERROR(hipFree(x_d));
  HANDLE_ERROR(hipFree(U_d));
  HANDLE_ERROR(hipFree(du_d));
  HANDLE_ERROR(hipFree(nu_d));
  HANDLE_ERROR(hipFree(costs_d));
}

template <class DYN_T, class COST_T, class SAMPLER_T>
void launchCPURolloutKernel(DYN_T* model, COST_T* cost, SAMPLER_T* sampler, float dt, const int num_timesteps,
                            const int num_rollouts, float lambda, float alpha,
                            const Eigen::Ref<const typename DYN_T::state_array>& x0,
                            Eigen::Ref<Eigen::MatrixXf> trajectory_costs, hipStream_t stream)
{
  using state_array = typename DYN_T::state_array;
  using output_array = typename DYN_T::output_array;
  using control_array = typename DYN_T::control_array;

  Eigen::MatrixXf control_noise = Eigen::MatrixXf::Zero(DYN_T::CONTROL_DIM, num_rollouts * num_timesteps);
  Eigen::MatrixXi crash_status = Eigen::MatrixXi::Zero(num_rollouts, 1);
  HANDLE_ERROR(hipMemcpy(control_noise.data(), sampler->getControlSample(0, 0, 0),
                          sizeof(float) * DYN_T::CONTROL_DIM * num_rollouts * num_timesteps, hipMemcpyDeviceToHost));

  state_array curr_x, next_x, x_der;
  control_array u;
  output_array y;
  for (int sample_idx = 0; sample_idx < num_rollouts; sample_idx++)
  {
    curr_x = x0;
    model->initializeDynamics(curr_x, u, y, 0, dt);
    cost->initializeCosts(y, u, 0, dt);
    float& running_cost = trajectory_costs(sample_idx, 0);
    running_cost = 0.0f;
    for (int t = 0; t < num_timesteps; t++)
    {
      u = control_noise.col(t + num_timesteps * sample_idx);
      model->enforceConstraints(curr_x, u);
      model->step(curr_x, next_x, x_der, u, y, t, dt);
      running_cost += cost->computeRunningCost(y, u, t, &crash_status(sample_idx));
      running_cost += sampler->computeLikelihoodRatioCost(u, t, 0, lambda, alpha);
      curr_x = next_x;
    }
    running_cost += cost->terminalCost(y);
    running_cost /= num_timesteps;
  }
}
