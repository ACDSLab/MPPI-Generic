#include "hip/hip_runtime.h"
#include <mppi/core/mppi_common_new.cuh>

template <typename CLASS_T, typename PARAMS_T>
__global__ void parameterTestKernel(CLASS_T* class_t, PARAMS_T& params)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0)
  {
    params = class_t->getParams();
  }
}

template <typename CLASS_T, typename PARAMS_T>
void launchParameterTestKernel(CLASS_T& class_t, PARAMS_T& params)
{
  PARAMS_T* params_d;
  HANDLE_ERROR(hipMalloc((void**)&params_d, sizeof(PARAMS_T)))

  parameterTestKernel<CLASS_T, PARAMS_T><<<1, 1>>>(static_cast<CLASS_T*>(class_t.model_d_), *params_d);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(&params, params_d, sizeof(PARAMS_T), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(params_d);
}

template <typename DYNAMICS_T, int C_DIM>
__global__ void controlRangesTestKernel(DYNAMICS_T* dynamics, float2* control_rngs)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0)
  {
    float2* raw_ptr = dynamics->getControlRangesRaw();
    for (int i = 0; i < C_DIM; i++)
    {
      control_rngs[i].x = raw_ptr[i].x;
      control_rngs[i].y = raw_ptr[i].y;
    }
  }
}

template <typename DYNAMICS_T, int C_DIM>
void launchControlRangesTestKernel(DYNAMICS_T& dynamics, std::array<float2, C_DIM>& control_rngs)
{
  float2* ranges_d;
  HANDLE_ERROR(hipMalloc((void**)&ranges_d, sizeof(float2) * control_rngs.size()))

  controlRangesTestKernel<DYNAMICS_T, C_DIM><<<1, 1>>>(static_cast<DYNAMICS_T*>(dynamics.model_d_), ranges_d);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(control_rngs.data(), ranges_d, sizeof(float2) * control_rngs.size(), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(ranges_d);
}

template <typename DYNAMICS_T, int S_DIM, int C_DIM>
__global__ void enforceConstraintTestKernel(DYNAMICS_T* dynamics, float* state, float* control, int num)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num)
  {
    dynamics->enforceConstraints(&state[tid * S_DIM], &control[tid * C_DIM]);
  }
}

template <typename DYNAMICS_T, int S_DIM, int C_DIM>
void launchEnforceConstraintTestKernel(DYNAMICS_T& dynamics, std::vector<std::array<float, S_DIM>>& state,
                                       std::vector<std::array<float, C_DIM>>& control, int dim_y)
{
  int count = state.size();
  float* state_d;
  float* control_d;
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * S_DIM * state.size()))
  HANDLE_ERROR(hipMalloc((void**)&control_d, sizeof(float) * C_DIM * control.size()))

  HANDLE_ERROR(hipMemcpy(state_d, state.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(control_d, control.data(), sizeof(float) * C_DIM * count, hipMemcpyHostToDevice));

  dim3 threadsPerBlock(count, dim_y);
  dim3 numBlocks(1, 1);
  enforceConstraintTestKernel<DYNAMICS_T, S_DIM, C_DIM>
      <<<numBlocks, threadsPerBlock>>>(static_cast<DYNAMICS_T*>(dynamics.model_d_), state_d, control_d, count);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(state.data(), state_d, sizeof(float) * S_DIM * state.size(), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(control.data(), control_d, sizeof(float) * C_DIM * control.size(), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(state_d);
  hipFree(control_d);
}

template <typename DYNAMICS_T, int S_DIM>
__global__ void updateStateTestKernel(DYNAMICS_T* dynamics, float* state, float* state_der, float dt, int num)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num)
  {
    dynamics->updateState(state + (tid * S_DIM), state_der + (tid * S_DIM), dt);
  }
}

template <typename DYNAMICS_T, int S_DIM>
void launchUpdateStateTestKernel(DYNAMICS_T& dynamics, std::vector<std::array<float, S_DIM>>& state,
                                 std::vector<std::array<float, S_DIM>>& state_der, float dt, int dim_y)
{
  int count = state.size();
  float* state_d;
  float* state_der_d;
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * S_DIM * count))
  HANDLE_ERROR(hipMalloc((void**)&state_der_d, sizeof(float) * S_DIM * count))

  HANDLE_ERROR(hipMemcpy(state_d, state.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(state_der_d, state_der.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));

  dim3 threadsPerBlock(count, dim_y);
  dim3 numBlocks(1, 1);
  updateStateTestKernel<DYNAMICS_T, S_DIM>
      <<<numBlocks, threadsPerBlock>>>(static_cast<DYNAMICS_T*>(dynamics.model_d_), state_d, state_der_d, dt, count);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(state.data(), state_d, sizeof(float) * S_DIM * state.size(), hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(state_der.data(), state_der_d, sizeof(float) * S_DIM * state_der.size(), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(state_d);
  hipFree(state_der_d);
}

template <typename DYNAMICS_T, int S_DIM>
__global__ void computeKinematicsTestKernel(DYNAMICS_T* dynamics, float* state, float* state_der, int num)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num && threadIdx.y == 0)
  {
    dynamics->computeKinematics(state + (tid * S_DIM), state_der + (tid * S_DIM));
  }
}

template <typename DYNAMICS_T, int S_DIM>
void launchComputeKinematicsTestKernel(DYNAMICS_T& dynamics, std::vector<std::array<float, S_DIM>>& state,
                                       std::vector<std::array<float, S_DIM>>& state_der, int dim_y)
{
  int count = state.size();
  float* state_d;
  float* state_der_d;
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * S_DIM * state.size()))
  HANDLE_ERROR(hipMalloc((void**)&state_der_d, sizeof(float) * S_DIM * state_der.size()))

  HANDLE_ERROR(hipMemcpy(state_d, state.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(state_der_d, state_der.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));

  dim3 threadsPerBlock(count, dim_y);
  dim3 numBlocks(1, 1);
  computeKinematicsTestKernel<DYNAMICS_T, S_DIM>
      <<<numBlocks, threadsPerBlock>>>(static_cast<DYNAMICS_T*>(dynamics.model_d_), state_d, state_der_d, count);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(state.data(), state_d, sizeof(float) * S_DIM * state.size(), hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(state_der.data(), state_der_d, sizeof(float) * S_DIM * state_der.size(), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(state_d);
  hipFree(state_der_d);
}

template <class DYNAMICS_T, int S_DIM, int C_DIM, int BLOCKDIM_X>
__global__ void computeDynamicsTestKernel(DYNAMICS_T* model, float* state, float* control, float* state_der, int count)
{
  extern __shared__ float entire_buffer[];

  float* output = entire_buffer;
  float* theta = &output[mppi::math::nearest_multiple_4(blockDim.x * DYNAMICS_T::OUTPUT_DIM)];

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  model->initializeDynamics(state, control, output, theta, 0.0f, 0.0f);
  if (tid < count)
  {
    model->computeDynamics(state + (tid * S_DIM), control + (tid * C_DIM), state_der + (tid * S_DIM), theta);
  }
}

template <class DYNAMICS_T, int S_DIM, int C_DIM, int BLOCKDIM_X>
void launchComputeDynamicsTestKernel(DYNAMICS_T& dynamics, std::vector<std::array<float, S_DIM>>& state,
                                     std::vector<std::array<float, C_DIM>>& control,
                                     std::vector<std::array<float, S_DIM>>& state_der, int dim_y)
{
  int count = state.size();
  float* state_d;
  float* state_der_d;
  float* control_d;

  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * S_DIM * count))
  HANDLE_ERROR(hipMalloc((void**)&state_der_d, sizeof(float) * S_DIM * count))
  HANDLE_ERROR(hipMalloc((void**)&control_d, sizeof(float) * C_DIM * count))

  HANDLE_ERROR(hipMemcpy(state_d, state.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(state_der_d, state_der.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(control_d, control.data(), sizeof(float) * C_DIM * count, hipMemcpyHostToDevice));

  // make sure you cannot use invalid inputs
  const int gridsize_x = (count - 1) / BLOCKDIM_X + 1;
  dim3 threadsPerBlock(BLOCKDIM_X, dim_y);
  dim3 numBlocks(gridsize_x, 1);
  unsigned shared_mem = mppi::kernels::calcDynamicsSharedMemSize(&dynamics, threadsPerBlock) +
                        mppi::math::nearest_multiple_4(threadsPerBlock.x * DYNAMICS_T::OUTPUT_DIM);
  // launch kernel
  computeDynamicsTestKernel<DYNAMICS_T, S_DIM, C_DIM, BLOCKDIM_X>
      <<<numBlocks, threadsPerBlock, shared_mem>>>(dynamics.model_d_, state_d, control_d, state_der_d, count);
  CudaCheckError();

  HANDLE_ERROR(hipMemcpy(state.data(), state_d, sizeof(float) * S_DIM * count, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(state_der.data(), state_der_d, sizeof(float) * S_DIM * count, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(control.data(), control_d, sizeof(float) * C_DIM * count, hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(state_d);
  hipFree(state_der_d);
  hipFree(control_d);
}

template <typename DYNAMICS_T, int S_DIM, int C_DIM, int BLOCKDIM_X>
__global__ void computeStateDerivTestKernel(DYNAMICS_T* dynamics, float* state, float* control, float* state_der,
                                            int num)
{
  extern __shared__ float entire_buffer[];

  float* output = entire_buffer;
  float* theta = &output[mppi::math::nearest_multiple_4(blockDim.x * DYNAMICS_T::OUTPUT_DIM)];

  dynamics->initializeDynamics(state, control, output, theta, 0.0f, 0.0f);

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num)
  {
    // printf("calling on thread %d, %d\n", tid, threadIdx.y);
    dynamics->computeStateDeriv(state + (tid * S_DIM), control + (tid * C_DIM), state_der + (tid * S_DIM), theta);
  }
}

template <typename DYNAMICS_T, int S_DIM, int C_DIM, int BLOCKDIM_X>
void launchComputeStateDerivTestKernel(DYNAMICS_T& dynamics, std::vector<std::array<float, S_DIM>>& state,
                                       std::vector<std::array<float, C_DIM>>& control,
                                       std::vector<std::array<float, S_DIM>>& state_der, int dim_y)
{
  int count = state.size();
  float* state_d;
  float* control_d;
  float* state_der_d;
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * S_DIM * state.size()))
  HANDLE_ERROR(hipMalloc((void**)&state_der_d, sizeof(float) * S_DIM * state_der.size()))
  HANDLE_ERROR(hipMalloc((void**)&control_d, sizeof(float) * C_DIM * control.size()))

  HANDLE_ERROR(hipMemcpy(state_d, state.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(state_der_d, state_der.data(), sizeof(float) * S_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(control_d, control.data(), sizeof(float) * C_DIM * count, hipMemcpyHostToDevice));

  const int gridsize_x = (count - 1) / BLOCKDIM_X + 1;
  dim3 threadsPerBlock(BLOCKDIM_X, dim_y);
  dim3 numBlocks(gridsize_x, 1);

  unsigned shared_mem = mppi::kernels::calcDynamicsSharedMemSize(&dynamics, threadsPerBlock) +
                        mppi::math::nearest_multiple_4(threadsPerBlock.x * DYNAMICS_T::OUTPUT_DIM);
  computeStateDerivTestKernel<DYNAMICS_T, S_DIM, C_DIM, BLOCKDIM_X><<<numBlocks, threadsPerBlock, shared_mem>>>(
      static_cast<DYNAMICS_T*>(dynamics.model_d_), state_d, control_d, state_der_d, count);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(state.data(), state_d, sizeof(float) * S_DIM * state.size(), hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(state_der.data(), state_der_d, sizeof(float) * S_DIM * state_der.size(), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(control.data(), control_d, sizeof(float) * C_DIM * control.size(), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(state_d);
  hipFree(state_der_d);
  hipFree(control_d);
}

template <typename DYNAMICS_T, int BLOCKDIM_X>
__global__ void stepTestKernel(DYNAMICS_T* dynamics, float* state, float* control, float* state_der, float* next_state,
                               float* output, int t, float dt, int num)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  extern __shared__ float entire_buffer[];
  float* theta = entire_buffer;

  // float* theta = reinterpret_cast<float*>(theta_s4);
  float* x = state + (tid * DYNAMICS_T::STATE_DIM);
  float* x_dot = state_der + (tid * DYNAMICS_T::STATE_DIM);
  float* x_next = next_state + (tid * DYNAMICS_T::STATE_DIM);
  float* u = control + (tid * DYNAMICS_T::CONTROL_DIM);
  float* y = output + (tid * DYNAMICS_T::OUTPUT_DIM);

  dynamics->initializeDynamics(state, control, output, theta, 0.0f, dt);

  if (tid < num)
  {
    float* x = state + (tid * DYNAMICS_T::STATE_DIM);
    float* x_dot = state_der + (tid * DYNAMICS_T::STATE_DIM);
    float* x_next = next_state + (tid * DYNAMICS_T::STATE_DIM);
    float* u = control + (tid * DYNAMICS_T::CONTROL_DIM);
    float* y = output + (tid * DYNAMICS_T::OUTPUT_DIM);

    dynamics->initializeDynamics(state, control, output, theta, 0.0f, dt);

    // printf("calling on thread %d, %d\n", tid, threadIdx.y);
    dynamics->enforceConstraints(x, u);
    dynamics->step(x, x_next, x_dot, u, y, theta, t, dt);
    // dynamics->computeStateDeriv(state + (tid * S_DIM), control + (tid * C_DIM), state_der + (tid * S_DIM), theta);
  }
}

template <typename DYNAMICS_T, int BLOCKDIM_X = 32>
void launchStepTestKernel(DYNAMICS_T& dynamics, std::vector<std::array<float, DYNAMICS_T::STATE_DIM>>& state,
                          std::vector<std::array<float, DYNAMICS_T::CONTROL_DIM>>& control,
                          std::vector<std::array<float, DYNAMICS_T::STATE_DIM>>& state_der,
                          std::vector<std::array<float, DYNAMICS_T::STATE_DIM>>& next_state, int t, float dt, int dim_y)
{
  if (state.size() != control.size())
  {
    std::cerr << "Num States doesn't match num controls" << std::endl;
    return;
  }
  if (state.size() != state_der.size())
  {
    std::cerr << "Num States doesn't match num state_ders" << std::endl;
    return;
  }
  if (state.size() != next_state.size())
  {
    std::cerr << "Num States doesn't match num next_states" << std::endl;
    return;
  }
  int count = state.size();
  float* state_d;
  float* control_d;
  float* state_der_d;
  float* next_state_d;
  float* output_d;
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float) * DYNAMICS_T::STATE_DIM * count));
  HANDLE_ERROR(hipMalloc((void**)&state_der_d, sizeof(float) * DYNAMICS_T::STATE_DIM * count));
  HANDLE_ERROR(hipMalloc((void**)&next_state_d, sizeof(float) * DYNAMICS_T::STATE_DIM * count));
  HANDLE_ERROR(hipMalloc((void**)&control_d, sizeof(float) * DYNAMICS_T::CONTROL_DIM * count));
  HANDLE_ERROR(hipMalloc((void**)&output_d, sizeof(float) * DYNAMICS_T::OUTPUT_DIM * count));

  HANDLE_ERROR(
      hipMemcpy(state_d, state.data(), sizeof(float) * DYNAMICS_T::STATE_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(state_der_d, state_der.data(), sizeof(float) * DYNAMICS_T::STATE_DIM * count, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(next_state_d, next_state.data(), sizeof(float) * DYNAMICS_T::STATE_DIM * count,
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(control_d, control.data(), sizeof(float) * DYNAMICS_T::CONTROL_DIM * count, hipMemcpyHostToDevice));

  const int gridsize_x = (count - 1) / BLOCKDIM_X + 1;
  dim3 threadsPerBlock(BLOCKDIM_X, dim_y);
  dim3 numBlocks(gridsize_x, 1);

  unsigned shared_mem = mppi::kernels::calcDynamicsSharedMemSize(&dynamics, threadsPerBlock);
  stepTestKernel<DYNAMICS_T, BLOCKDIM_X><<<numBlocks, threadsPerBlock, shared_mem>>>(
      dynamics.model_d_, state_d, control_d, state_der_d, next_state_d, output_d, t, dt, count);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(
      hipMemcpy(state.data(), state_d, sizeof(float) * DYNAMICS_T::STATE_DIM * count, hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(state_der.data(), state_der_d, sizeof(float) * DYNAMICS_T::STATE_DIM * count, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(next_state.data(), next_state_d, sizeof(float) * DYNAMICS_T::STATE_DIM * count,
                          hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(control.data(), control_d, sizeof(float) * DYNAMICS_T::CONTROL_DIM * count, hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(state_d);
  hipFree(state_der_d);
  hipFree(control_d);
  hipFree(next_state_d);
  hipFree(output_d);
}
