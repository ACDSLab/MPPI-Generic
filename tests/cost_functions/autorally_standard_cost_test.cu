#include "hip/hip_runtime.h"
//
// Created by jason on 1/7/20.
//

#include <gtest/gtest.h>
#include <cost_functions/autorally/ar_standard_cost.cuh>
#include <cost_functions/ar_standard_cost_kernel_test.cuh>

TEST(ARStandardCost, Constructor) {
  ARStandardCost cost;
}

TEST(ARStandardCost, BindStream) {
  hipStream_t stream;

  HANDLE_ERROR(hipStreamCreate(&stream));

  ARStandardCost cost(stream);

  EXPECT_EQ(cost.stream_, stream) << "Stream binding failure.";

  HANDLE_ERROR(hipStreamDestroy(stream));
}

TEST(ARStandardCost, SetGetParamsHost) {
  ARStandardCost::ARStandardCostParams params;
  params.desired_speed = 25;
  params.num_timesteps = 100;
  params.r_c1.x = 0;
  params.r_c1.y = 1;
  params.r_c1.z = 2;
  ARStandardCost cost;

  cost.setParams(params);
  ARStandardCost::ARStandardCostParams result_params = cost.getParams();

  EXPECT_FLOAT_EQ(params.desired_speed, result_params.desired_speed);
  EXPECT_EQ(params.num_timesteps, result_params.num_timesteps);
  EXPECT_FLOAT_EQ(params.r_c1.x, result_params.r_c1.x);
  EXPECT_FLOAT_EQ(params.r_c1.y, result_params.r_c1.y);
  EXPECT_FLOAT_EQ(params.r_c1.z, result_params.r_c1.z);
}

TEST(ARStandardCost, GPUSetupAndParamsToDeviceTest) {
  ARStandardCost::ARStandardCostParams params;
  ARStandardCost cost;
  params.desired_speed = 25;
  params.num_timesteps = 100;
  params.r_c1.x = 0;
  params.r_c1.y = 1;
  params.r_c1.z = 2;
  cost.setParams(params);

  EXPECT_EQ(cost.GPUMemStatus_, false);

  cost.GPUSetup();

  EXPECT_EQ(cost.GPUMemStatus_, true);

  float desired_speed;
  int num_timesteps, height, width;
  float3 r_c1;
  launchParameterTestKernel(cost, desired_speed, num_timesteps, r_c1, width, height);

  EXPECT_FLOAT_EQ(desired_speed, 25);
  EXPECT_EQ(num_timesteps, 100);
  EXPECT_FLOAT_EQ(r_c1.x, 0);
  EXPECT_FLOAT_EQ(r_c1.y, 1);
  EXPECT_FLOAT_EQ(r_c1.z, 2);
  // neither should be set by this sequence
  EXPECT_EQ(width, -1);
  EXPECT_EQ(height, -1);

  params.desired_speed = 5;
  params.num_timesteps = 50;
  params.r_c1.x = 4;
  params.r_c1.y = 5;
  params.r_c1.z = 6;
  cost.setParams(params);
  cost.paramsToDevice();

  launchParameterTestKernel(cost, desired_speed, num_timesteps, r_c1, height, width);

  EXPECT_FLOAT_EQ(desired_speed, 5);
  EXPECT_EQ(num_timesteps, 50);
  EXPECT_FLOAT_EQ(r_c1.x, 4);
  EXPECT_FLOAT_EQ(r_c1.y, 5);
  EXPECT_FLOAT_EQ(r_c1.z, 6);

  // neither should be set by this sequence
  EXPECT_EQ(width, -1);
  EXPECT_EQ(height, -1);
}

TEST(ARStandardCost, clearCostmapCPUTestValidInputs) {
  ARStandardCost cost;
  cost.clearCostmapCPU(4, 8);

  EXPECT_EQ(cost.getWidth(), 4);
  EXPECT_EQ(cost.getHeight(), 8);

  for(int i = 0; i < 4 * 8; i++) {
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).x, 0);
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).y, 0);
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).z, 0);
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).w, 0);
  }
}

TEST(ARStandardCost, clearCostmapCPUTestDefault) {
  ARStandardCost cost;
  cost.clearCostmapCPU(4, 8);
  cost.clearCostmapCPU();

  EXPECT_EQ(cost.getWidth(), 4);
  EXPECT_EQ(cost.getHeight(), 8);

  for(int i = 0; i < 4 * 8; i++) {
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).x, 0);
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).y, 0);
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).z, 0);
    EXPECT_FLOAT_EQ(cost.getTrackCostCPU().at(i).w, 0);
  }
}


TEST(ARStandardCost, clearCostmapCPUTestDefaultFail) {
  ARStandardCost cost;
  cost.clearCostmapCPU();

  EXPECT_EQ(cost.getWidth(), -1);
  EXPECT_EQ(cost.getHeight(), -1);
}

TEST(ARStandardCost, LoadTrackDataTest) {
  ARStandardCost::ARStandardCostParams params;
  ARStandardCost cost;
  // TODO set parameters for cost map
  cost.setParams(params);
  cost.GPUSetup();
  float desired_speed;
  int num_timesteps, height, width;
  float3 r_c1;
  launchParameterTestKernel(cost, desired_speed, num_timesteps, r_c1, width, height);
  std::string test_location = "/home/jason/Documents/research/MPPI-Generic/resource/autorally/test/test_map.npz";
  Eigen::Matrix3f R;
  Eigen::Array3f trs;

  // load
  std::vector<float4> costmap = cost.loadTrackData(test_location, R, trs);

  EXPECT_FLOAT_EQ(costmap.at(0).x, 0);
  EXPECT_FLOAT_EQ(costmap.at(0).y, 0);
  EXPECT_FLOAT_EQ(costmap.at(0).z, 0);
  EXPECT_FLOAT_EQ(costmap.at(0).w, 0);
  EXPECT_FLOAT_EQ(costmap.at(1).x, 1);
  EXPECT_FLOAT_EQ(costmap.at(1).y, 10);
  EXPECT_FLOAT_EQ(costmap.at(1).z, 100);
  EXPECT_FLOAT_EQ(costmap.at(1).w, 1000);

  // check transformation, should not have a rotation
  EXPECT_FLOAT_EQ(R(0,0), 1.0 / (10));
  EXPECT_FLOAT_EQ(R(1,1), 1.0 / (20));
  EXPECT_FLOAT_EQ(R(2,2), 1.0);

  EXPECT_FLOAT_EQ(R(0, 1), 0);
  EXPECT_FLOAT_EQ(R(0, 2), 0);
  EXPECT_FLOAT_EQ(R(1, 0), 0);
  EXPECT_FLOAT_EQ(R(1, 2), 0);
  EXPECT_FLOAT_EQ(R(2, 0), 0);
  EXPECT_FLOAT_EQ(R(2, 1), 0);

  EXPECT_FLOAT_EQ(trs(0), 0.5);
  EXPECT_FLOAT_EQ(trs(1), 0.5);
  EXPECT_FLOAT_EQ(trs(2), 1);

  for(int i = 0; i < 2 * 10; i++) {
    for(int j = 0; j < 2 * 20; j++) {
      EXPECT_FLOAT_EQ(costmap.at(i*2*20 + j).x, i*2*20 + j);
      EXPECT_FLOAT_EQ(costmap.at(i*2*20 + j).y, (i*2*20 + j) * 10);
      EXPECT_FLOAT_EQ(costmap.at(i*2*20 + j).z, (i*2*20 + j) * 100);
      EXPECT_FLOAT_EQ(costmap.at(i*2*20 + j).w, (i*2*20 + j) * 1000);
    }
  }

}

