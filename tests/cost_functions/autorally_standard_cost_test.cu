#include "hip/hip_runtime.h"
//
// Created by jason on 1/7/20.
//

#include <gtest/gtest.h>
#include <cost_functions/autorally/ar_standard_cost.cuh>
#include <cost_functions/ar_standard_cost_kernel_test.cuh>

TEST(ARStandardCost, Constructor) {
  ARStandardCost cost(4, 5);

  EXPECT_EQ(cost.getWidth(), 4);
  EXPECT_EQ(cost.getHeight(), 5);
}

TEST(ARStandardCost, BindStream) {
  hipStream_t stream;

  HANDLE_ERROR(hipStreamCreate(&stream));

  ARStandardCost cost(1, 2, stream);

  EXPECT_EQ(cost.stream_, stream) << "Stream binding failure.";

  HANDLE_ERROR(hipStreamDestroy(stream));
}

TEST(ARStandardCost, SetGetParamsHost) {
  ARStandardCost::ARStandardCostParams params;
  params.desired_speed = 25;
  params.num_timesteps = 100;
  params.r_c1.x = 0;
  params.r_c1.y = 1;
  params.r_c1.z = 2;
  ARStandardCost cost(4, 5);

  cost.setParams(params);
  ARStandardCost::ARStandardCostParams result_params = cost.getParams();

  EXPECT_FLOAT_EQ(params.desired_speed, result_params.desired_speed);
  EXPECT_EQ(params.num_timesteps, result_params.num_timesteps);
  EXPECT_FLOAT_EQ(params.r_c1.x, result_params.r_c1.x);
  EXPECT_FLOAT_EQ(params.r_c1.y, result_params.r_c1.y);
  EXPECT_FLOAT_EQ(params.r_c1.z, result_params.r_c1.z);
}

/*
 * __global__ void objectAllocationTestKernel(ARStandardCost* cost) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  printf("Entering the kernel!\n");
  printf("The thread id is: %i\n", tid);
  if (tid == 0) {
    printf("The cart mass is: %f\n", cost->getParams().desired_speed);
  }
}
 */

TEST(ARStandardCost, GPUSetupAndParamsToDeviceTest) {
  ARStandardCost::ARStandardCostParams params;
  ARStandardCost cost(4,8);
  params.desired_speed = 25;
  params.num_timesteps = 100;
  params.r_c1.x = 0;
  params.r_c1.y = 1;
  params.r_c1.z = 2;
  cost.setParams(params);

  EXPECT_EQ(cost.GPUMemStatus_, false);

  cost.GPUSetup();

  EXPECT_EQ(cost.GPUMemStatus_, true);

  float desired_speed;
  int num_timesteps, height, width;
  float3 r_c1;
  launchParameterTestKernel(cost, desired_speed, num_timesteps, r_c1, width, height);

  EXPECT_FLOAT_EQ(desired_speed, 25);
  EXPECT_EQ(num_timesteps, 100);
  EXPECT_FLOAT_EQ(r_c1.x, 0);
  EXPECT_FLOAT_EQ(r_c1.y, 1);
  EXPECT_FLOAT_EQ(r_c1.z, 2);
  EXPECT_EQ(width, 4);
  EXPECT_EQ(height, 8);

  params.desired_speed = 5;
  params.num_timesteps = 50;
  params.r_c1.x = 4;
  params.r_c1.y = 5;
  params.r_c1.z = 6;
  cost.setParams(params);
  cost.paramsToDevice();

  launchParameterTestKernel(cost, desired_speed, num_timesteps, r_c1, height, width);

  EXPECT_FLOAT_EQ(desired_speed, 5);
  EXPECT_EQ(num_timesteps, 50);
  EXPECT_FLOAT_EQ(r_c1.x, 4);
  EXPECT_FLOAT_EQ(r_c1.y, 5);
  EXPECT_FLOAT_EQ(r_c1.z, 6);
}

