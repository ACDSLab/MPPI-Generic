#include "hip/hip_runtime.h"
//
// Created by jason on 1/7/20.
//

#include <gtest/gtest.h>
#include <cost_functions/autorally/ar_standard_cost.cuh>

TEST(ARStandardCost, Constructor) {
  ARStandardCost cost(4, 5);
}

TEST(ARStandardCost, BindStream) {
  hipStream_t stream;

  HANDLE_ERROR(hipStreamCreate(&stream));

  ARStandardCost cost(1, 2, stream);

  EXPECT_EQ(cost.stream_, stream) << "Stream binding failure.";

  HANDLE_ERROR(hipStreamDestroy(stream));
}

TEST(ARStandardCost, SetGetParamsHost) {
  ARStandardCost::ARStandardCostParams params;
  params.desired_speed = 25;
  params.num_timesteps = 100;
  params.r_c1.x = 0;
  params.r_c1.y = 1;
  params.r_c1.z = 2;
  ARStandardCost cost(4, 5);

  cost.setParams(params);
  ARStandardCost::ARStandardCostParams result_params = cost.getParams();

  EXPECT_FLOAT_EQ(params.desired_speed, result_params.desired_speed);
  EXPECT_EQ(params.num_timesteps, result_params.num_timesteps);
  EXPECT_FLOAT_EQ(params.r_c1.x, result_params.r_c1.x);
  EXPECT_FLOAT_EQ(params.r_c1.y, result_params.r_c1.y);
  EXPECT_FLOAT_EQ(params.r_c1.z, result_params.r_c1.z);
}

/*
 * __global__ void objectAllocationTestKernel(ARStandardCost* cost) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  printf("Entering the kernel!\n");
  printf("The thread id is: %i\n", tid);
  if (tid == 0) {
    printf("The cart mass is: %f\n", cost->getParams().desired_speed);
  }
}
 */

TEST(ARStandardCost, GPUSetup_Test) {
  ARStandardCost::ARStandardCostParams params;
  ARStandardCost cost(4,5);
  params.desired_speed = 25;
  params.num_timesteps = 100;
  params.r_c1.x = 0;
  params.r_c1.y = 1;
  params.r_c1.z = 2;
  cost.setParams(params);
  cost.GPUSetup();
  float desired_speed;
  int num_timesteps;
  float3 r_c1;
  launchParameterTestKernel(cost, desired_speed, num_timesteps, r_c1);

  EXPECT_FLOAT_EQ(desired_speed, 25);
  EXPECT_EQ(num_timesteps, 100);
  EXPECT_FLOAT_EQ(r_c1.x, 0);
  EXPECT_FLOAT_EQ(r_c1.y, 1);
  EXPECT_FLOAT_EQ(r_c1.z, 2);
}

