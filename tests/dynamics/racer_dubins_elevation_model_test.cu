#include "hip/hip_runtime.h"
#include <Eigen/Dense>
#include <gtest/gtest.h>
#include <mppi/dynamics/racer_dubins/racer_dubins_elevation.cuh>
#include <kernel_tests/dynamics/dynamics_generic_kernel_tests.cuh>
#include <mppi/ddp/ddp_model_wrapper.h>
#include <hip/hip_runtime.h>

class RacerDubinsElevationTest : public ::testing::Test
{
public:
  hipStream_t stream;

  void SetUp() override
  {
    CudaCheckError();
    HANDLE_ERROR(hipStreamCreate(&stream));
  }

  void TearDown() override
  {
    CudaCheckError();
    HANDLE_ERROR(hipStreamDestroy(stream));
  }
};

TEST_F(RacerDubinsElevationTest, Template)
{
  auto dynamics = RacerDubinsElevation();
  EXPECT_EQ(19, RacerDubinsElevation::STATE_DIM);
  EXPECT_EQ(2, RacerDubinsElevation::CONTROL_DIM);
  EXPECT_NE(dynamics.getTextureHelper(), nullptr);
}

TEST_F(RacerDubinsElevationTest, BindStream)
{
  auto dynamics = RacerDubinsElevation(stream);

  EXPECT_EQ(dynamics.stream_, stream) << "Stream binding failure.";
  EXPECT_NE(dynamics.getTextureHelper(), nullptr);
  EXPECT_EQ(dynamics.getTextureHelper()->stream_, stream);
}

/*
float c_t = 1.3;
float c_b = 2.5;
float c_v = 3.7;
float c_0 = 4.9;
float wheel_base = 0.3;
 */

// TEST_F(RacerDubinsElevationTest, ComputeDynamics)
// {
//   RacerDubinsElevation dynamics = RacerDubinsElevation();
//   auto params = dynamics.getParams();
//   RacerDubinsElevation::state_array x = RacerDubinsElevation::state_array::Zero();
//   RacerDubinsElevation::control_array u = RacerDubinsElevation::control_array::Zero();

//   // computeDynamics should not touch the roll/pitch element
//   RacerDubinsElevation::state_array next_x = RacerDubinsElevation::state_array::Ones() * 0.153;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), 0);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 1, M_PI_2, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 2.6 - 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 1);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 1, M_PI_2, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 3.5 - 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 1);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -1, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 4.7 + 2.6 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -1);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -1, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.5 + 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -1);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.9 - 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), 7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.9 + 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 4.5 - 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), 7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 4.5 + 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 1, M_PI_2, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 0, 1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (1 / .3) * tan(0));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 1);
//   EXPECT_FLOAT_EQ(next_x(4), 5 * 0.6);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -1, M_PI_2, 0, 3, 5.0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, -1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 4.7 + 3.5 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (-1 / .3) * tan(5.0 / -10.2));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), -1);
//   EXPECT_FLOAT_EQ(next_x(4), -5);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -0.4, M_PI_2, 0, 3, 5.0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, -1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.7 * 0.4 + 2.5 * 0.4 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (-0.4 / .3) * tan(5.0 / -9.1));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), -0.4);
//   EXPECT_FLOAT_EQ(next_x(4), -5);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 0.4, M_PI_2, 0, 3, 5.0, 0.5, -0.5, 0.0, 0.0;
//   u << 0.1, -1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 3.7 * 0.4 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (0.4 / .3) * tan(5.0 / -9.1));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 0.4);
//   EXPECT_FLOAT_EQ(next_x(4), -5);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);
// }

// TEST_F(RacerDubinsElevationTest, TestModelGPU)
// {
//   RacerDubinsElevation dynamics = RacerDubinsElevation();
//   dynamics.GPUSetup();
//   CudaCheckError();

//   Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, 100> control_trajectory;
//   control_trajectory = Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, 100>::Random();
//   Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, 100> state_trajectory;
//   state_trajectory = Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, 100>::Random();

//   std::vector<std::array<float, 9>> s(100);
//   std::vector<std::array<float, 9>> s_der(100);
//   // steering, throttle
//   std::vector<std::array<float, 2>> u(100);
//   for (int state_index = 0; state_index < s.size(); state_index++)
//   {
//     for (int dim = 0; dim < s[0].size(); dim++)
//     {
//       s[state_index][dim] = state_trajectory.col(state_index)(dim);
//     }
//     for (int dim = 0; dim < u[0].size(); dim++)
//     {
//       u[state_index][dim] = control_trajectory.col(state_index)(dim);
//     }
//   }

//   // These variables will be changed so initialized to the right size only

//   // Run dynamics on dynamicsU
//   // Run dynamics on GPU
//   for (int y_dim = 1; y_dim <= 4; y_dim++)
//   {
//     launchComputeDynamicsTestKernel<RacerDubinsElevation, 9, 2>(dynamics, s, u, s_der, y_dim);
//     for (int point = 0; point < 100; point++)
//     {
//       RacerDubinsElevation::state_array state = state_trajectory.col(point);
//       RacerDubinsElevation::control_array control = control_trajectory.col(point);
//       RacerDubinsElevation::state_array state_der_cpu = RacerDubinsElevation::state_array::Zero();

//       dynamics.computeDynamics(state, control, state_der_cpu);
//       for (int dim = 0; dim < 6; dim++)
//       {
//         EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-5)
//             << "at point " << point << " dim " << dim << " with y_dim " << y_dim;
//         EXPECT_TRUE(isfinite(s_der[point][dim]));
//       }
//     }
//   }

//   dynamics.freeCudaMem();
//   CudaCheckError();
// }

// TEST_F(RacerDubinsElevationTest, TestUpdateState)
// {
//   CudaCheckError();
//   RacerDubinsElevation dynamics = RacerDubinsElevation();
//   RacerDubinsElevation::state_array state;
//   RacerDubinsElevation::state_array next_state;
//   RacerDubinsElevation::state_array state_der;

//   // TODO add in the elevation map

//   state << 0, 0, 0, 0, 0, -0.5, 0.5;
//   state_der << 1, 1, 1, 1, 1, 0, 0;
//   dynamics.updateState(state, next_state, state_der, 0.1);
//   EXPECT_TRUE(state_der != RacerDubinsElevation::state_array::Zero());
//   EXPECT_FLOAT_EQ(next_state(0), 0.1);
//   EXPECT_FLOAT_EQ(next_state(1), 0.1);
//   EXPECT_FLOAT_EQ(next_state(2), 0.1);
//   EXPECT_FLOAT_EQ(next_state(3), 0.1);
//   EXPECT_FLOAT_EQ(next_state(4), 0.1);
//   EXPECT_FLOAT_EQ(next_state(5), 0.0);
//   EXPECT_FLOAT_EQ(next_state(6), 0.0);

//   state << 0, M_PI - 0.1, 0, 0, 0, -0.5, 0.5;
//   state_der << 1, 1, 1, 1, 1;
//   dynamics.updateState(state, next_state, state_der, 1.0);
//   EXPECT_TRUE(state_der != RacerDubinsElevation::state_array::Zero());
//   EXPECT_FLOAT_EQ(next_state(0), 1.0);
//   EXPECT_FLOAT_EQ(next_state(1), 1.0 - M_PI - 0.1);
//   EXPECT_FLOAT_EQ(next_state(2), 1.0);
//   EXPECT_FLOAT_EQ(next_state(3), 1.0);
//   EXPECT_FLOAT_EQ(next_state(4), 0.5);
//   EXPECT_FLOAT_EQ(next_state(5), 0.0);
//   EXPECT_FLOAT_EQ(next_state(6), 0.0);

//   state << 0, -M_PI + 0.1, 0, 0, 0, -0.5, 0.5;
//   state_der << 1, -1, 1, 1, 1;
//   dynamics.updateState(state, next_state, state_der, 1.0);
//   EXPECT_TRUE(state_der != RacerDubinsElevation::state_array::Zero());
//   EXPECT_FLOAT_EQ(next_state(0), 1.0);
//   EXPECT_FLOAT_EQ(next_state(1), M_PI + 0.1 - 1.0);
//   EXPECT_FLOAT_EQ(next_state(2), 1.0);
//   EXPECT_FLOAT_EQ(next_state(3), 1.0);
//   EXPECT_FLOAT_EQ(next_state(4), 0.5);
//   EXPECT_FLOAT_EQ(next_state(5), 0.0);
//   EXPECT_FLOAT_EQ(next_state(6), 0.0);

//   CudaCheckError();
// }

TEST_F(RacerDubinsElevationTest, TestStep)
{
  CudaCheckError();
  using DYN = RacerDubinsElevation;
  const float tol = 1e-6;
  DYN dynamics = DYN();
  auto params = dynamics.getParams();
  params.c_0 = 0;
  params.c_b[0] = 1;
  params.c_b[1] = 10;
  params.c_b[2] = 100;
  params.c_v[0] = 0.25;
  params.c_v[1] = 0.5;
  params.c_v[2] = 0.75;
  params.c_t[0] = 2;
  params.c_t[1] = 20;
  params.c_t[2] = 200;
  params.low_min_throttle = 0.2;
  params.steer_command_angle_scale = 0.5;
  params.steering_constant = 0.5;
  params.wheel_base = 0.5;
  params.max_steer_rate = 5;
  dynamics.setParams(params);
  DYN::state_array state;
  DYN::state_array next_state;
  DYN::state_array state_der = DYN::state_array::Zero();
  DYN::control_array control;
  DYN::output_array output;
  float dt = 0.1;
  // TODO add in the elevation map

  // Basic initial state and no movement should stay still
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_TRUE(state_der == DYN::state_array::Zero());
  EXPECT_NEAR(next_state(0), 0.0, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), 0.0, tol);

  // check the first index of throttle
  state << 0.54, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0.21, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -0.115, tol);
  EXPECT_NEAR(next_state(0), 0.5285, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.054, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), -0.115, tol);

  // check the start of second index of throttle
  state << 0.56, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0.01, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -0.08, tol);
  EXPECT_NEAR(next_state(0), 0.552, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.056, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), -0.08, tol);

  // check the start of second index of throttle with different dt
  state << 0.56, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0.21, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, 0.2);
  EXPECT_NEAR(state_der(0), -0.12, tol);
  EXPECT_NEAR(next_state(0), 0.536, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.112, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), -0.12, tol);

  // check the end of second index of throttle
  state << 2.99, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0.01, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -1.295, tol);
  EXPECT_NEAR(next_state(0), 2.8605, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.299, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), -1.295, tol);

  // check the end of second index of throttle
  state << 3.01, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0.01, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -0.2575, tol);
  EXPECT_NEAR(next_state(0), 2.98425, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.301, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), -0.2575, tol);

  // Apply full throttle from zero state
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 1.6, tol);
  EXPECT_NEAR(next_state(0), 0.16, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), 1.6, tol);

  // Apply throttle to a state with positive velocity
  state << 1, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 5.5, tol);
  EXPECT_NEAR(next_state(0), 1.55, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.1, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), 5.5, tol);

  // Apply full throttle and half left turn to origin state
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 1.6, tol);
  EXPECT_NEAR(next_state(0), 0.16, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), powf(0.5, 3) * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), powf(0.5, 3), tol);
  EXPECT_NEAR(output(14), 1.6, tol);

  // Apply full throttle and half left turn to a moving state oriented 30 degrees to the left
  float yaw = M_PI / 6;
  state << 1.0, yaw, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 5.5, tol);
  EXPECT_NEAR(next_state(0), 1.55, tol);
  EXPECT_NEAR(next_state(1), yaw, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), powf(0.5, 3) * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), powf(0.5, 3), tol);
  EXPECT_NEAR(output(14), 5.5, tol);

  // Apply full throttle and half left turn to a moving state oriented 30 degrees to the left which is already turning
  float steer_angle = M_PI / 8;
  state << 1.0, yaw, 0, 0, steer_angle, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 5.5, tol);
  EXPECT_NEAR(state_der(1), -0.086361105, tol);
  EXPECT_NEAR(next_state(0), 1.55, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), 5.5, tol);

  // Apply full brake and half left turn to a moving state oriented 30 degrees to the left which is already turning
  state << 1.0, yaw, 0, 0, steer_angle, 1.0, 0.0, 0, 0;
  control << -1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -5.5, tol);
  EXPECT_NEAR(next_state(0), 1 - 5.5 * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), -5.5, tol);

  /**
   * Apply full brake and half left turn to a moving state oriented 30 degrees to the left which is already turning
   * and on a downward facing hill
   */
  float pitch = 20 * M_PI / 180;
  state << 1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), 1 + (-5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), (-5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half left turn to a backwards moving state oriented 30 degrees to the left which is already
   * turning and on a downward facing hill
   */
  state << -1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(state_der(1), 0.086361105, tol);
  EXPECT_NEAR(next_state(1), yaw + 0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), (5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half right turn to a backwards moving state oriented 30 degrees to the left which is already
   * turning and on a downward facing hill
   */
  state << -1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, -0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + 0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (-0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (-0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), (5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half right turn to a backwards moving state with a huge steering angle to test max steer
   * angle and steering rate. We are also on a downward facing hill and are already oriented 30 degrees to the left
   */
  steer_angle *= 100;
  state << -1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, -0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + tan(steer_angle / -9.1) * dt * -2, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), params.max_steer_angle, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), -params.max_steer_rate, tol);
  EXPECT_NEAR(output(14), (5.5 + 9.81 * sinf(pitch)), tol);
}

TEST_F(RacerDubinsElevationTest, TestStepGPUvsCPU)
{
  const int num_rollouts = 1000;
  const float dt = 0.1f;
  CudaCheckError();
  RacerDubinsElevation dynamics = RacerDubinsElevation();

  hipExtent extent = make_hipExtent(10, 20, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(10 * 20);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(1, 2, 3));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  CudaCheckError();
  dynamics.GPUSetup();
  CudaCheckError();

  Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, num_rollouts>::Random();

  std::vector<std::array<float, RacerDubinsElevation::STATE_DIM>> s(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevation::STATE_DIM>> s_next(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevation::STATE_DIM>> s_der(num_rollouts);
  // steering, throttle
  std::vector<std::array<float, RacerDubinsElevation::CONTROL_DIM>> u(num_rollouts);

  RacerDubinsElevation::state_array state;
  RacerDubinsElevation::state_array next_state_cpu;
  RacerDubinsElevation::control_array control;
  RacerDubinsElevation::output_array output;
  RacerDubinsElevation::state_array state_der_cpu = RacerDubinsElevation::state_array::Zero();

  // Run dynamics on dynamicsU
  // Run dynamics on GPU
  for (int y_dim = 1; y_dim <= 16; y_dim++)
  {
    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < s[0].size(); dim++)
      {
        s[state_index][dim] = state_trajectory.col(state_index)(dim);
      }
      for (int dim = 0; dim < u[0].size(); dim++)
      {
        u[state_index][dim] = control_trajectory.col(state_index)(dim);
      }
    }

    launchStepTestKernel<RacerDubinsElevation>(dynamics, s, u, s_der, s_next, 0, dt, y_dim);
    for (int point = 0; point < num_rollouts; point++)
    {
      state = state_trajectory.col(point);
      control = control_trajectory.col(point);
      state_der_cpu = RacerDubinsElevation::state_array::Zero();

      dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
      for (int dim = 0; dim < RacerDubinsElevation::STATE_DIM; dim++)
      {
        EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        // EXPECT_NEAR(state(dim), s[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        EXPECT_NEAR(next_state_cpu(dim), s_next[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        EXPECT_TRUE(isfinite(s_next[point][dim]));
      }
    }
  }
  dynamics.freeCudaMem();
}

TEST_F(RacerDubinsElevationTest, TestStepReverse)
{
  CudaCheckError();
  using DYN = RacerDubinsElevation;
  const float tol = 1e-6;
  DYN dynamics = DYN();
  auto params = dynamics.getParams();
  params.c_0 = 0;
  params.c_b[0] = 1;
  params.c_b[1] = 10;
  params.c_b[2] = 100;
  params.c_v[0] = 0.25;
  params.c_v[1] = 0.5;
  params.c_v[2] = 0.75;
  params.c_t[0] = 2;
  params.c_t[1] = 20;
  params.c_t[2] = 200;
  params.low_min_throttle = 0.2;
  params.steer_command_angle_scale = 0.5;
  params.steering_constant = 0.5;
  params.wheel_base = 0.5;
  params.max_steer_rate = 5;
  params.gear_sign = -1;
  dynamics.setParams(params);
  DYN::state_array state;
  DYN::state_array next_state;
  DYN::state_array state_der = DYN::state_array::Zero();
  DYN::control_array control;
  DYN::output_array output;
  float dt = 0.1;
  // TODO add in the elevation map

  // Basic initial state and no movement should stay still
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_TRUE(state_der == DYN::state_array::Zero());
  EXPECT_NEAR(next_state(0), 0.0, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), 0.0, tol);

  // Apply full throttle from zero state
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -1.6, tol);
  EXPECT_NEAR(next_state(0), -0.16, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), -1.6, tol);

  // Apply throttle to a state with positive velocity
  state << 1, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -5.5, tol);
  EXPECT_NEAR(next_state(0), 0.45, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.1, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 0.0, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 0.0, tol);
  EXPECT_NEAR(output(14), -5.5, tol);

  // Apply full throttle and half left turn to origin state
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -1.6, tol);
  EXPECT_NEAR(next_state(0), -0.16, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), powf(0.5, 3) * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), powf(0.5, 3), tol);
  EXPECT_NEAR(output(14), -1.6, tol);

  // Apply full throttle and half left turn to a moving state oriented 30 degrees to the left
  float yaw = M_PI / 6;
  state << 1.0, yaw, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -5.5, tol);
  EXPECT_NEAR(next_state(0), 0.45, tol);
  EXPECT_NEAR(next_state(1), yaw, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), powf(0.5, 3) * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), powf(0.5, 3), tol);
  EXPECT_NEAR(output(14), -5.5, tol);

  // Apply full throttle and half left turn to a moving state oriented 30 degrees to the left which is already turning
  float steer_angle = M_PI / 8;
  state << 1.0, yaw, 0, 0, steer_angle, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -5.5, tol);
  EXPECT_NEAR(next_state(0), 0.45, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), -5.5, tol);

  // Apply full brake and half left turn to a moving state oriented 30 degrees to the left which is already turning
  state << 1.0, yaw, 0, 0, steer_angle, 1.0, 0, 0.0, 0, 0;
  control << -1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -5.5, tol);
  EXPECT_NEAR(next_state(0), 1 - 5.5 * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), -5.5, tol);

  /**
   * Apply full brake and half left turn to a moving state oriented 30 degrees to the left which is already turning
   * and on a downward facing hill
   */
  float pitch = 20 * M_PI / 180;
  state << 1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), 1 + (-5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), (-5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half left turn to a backwards moving state oriented 30 degrees to the left which is already
   * turning and on a downward facing hill
   */
  state << -1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, 0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + 0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), (5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half right turn to a backwards moving state oriented 30 degrees to the left which is already
   * turning and on a downward facing hill
   */
  state << -1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, -0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + 0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + (-0.25 - steer_angle) * 0.5 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), (-0.25 - steer_angle) * 0.5, tol);
  EXPECT_NEAR(output(14), (5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half right turn to a backwards moving state with a huge steering angle to test max steer
   * angle and steering rate. We are also on a downward facing hill and are already oriented 30 degrees to the left
   */
  steer_angle *= 100;
  state << -1.0, yaw, 0, 0, steer_angle, 1.0, 0, pitch, 0, 0;
  control << -1, -0.5;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + tan(steer_angle / -9.1) * dt * -2, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), params.max_steer_angle, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), -params.max_steer_rate, tol);
  EXPECT_NEAR(output(14), (5.5 + 9.81 * sinf(pitch)), tol);
}

TEST_F(RacerDubinsElevationTest, TestStepGPUvsCPUReverse)
{
  const int num_rollouts = 1000;
  const float dt = 0.1f;
  CudaCheckError();
  RacerDubinsElevation dynamics = RacerDubinsElevation();
  auto params = dynamics.getParams();
  params.gear_sign = -1;
  dynamics.setParams(params);

  hipExtent extent = make_hipExtent(10, 20, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(10 * 20);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(1, 2, 3));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  CudaCheckError();
  dynamics.GPUSetup();
  CudaCheckError();

  Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, num_rollouts>::Random();

  std::vector<std::array<float, RacerDubinsElevation::STATE_DIM>> s(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevation::STATE_DIM>> s_next(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevation::STATE_DIM>> s_der(num_rollouts);
  // steering, throttle
  std::vector<std::array<float, RacerDubinsElevation::CONTROL_DIM>> u(num_rollouts);

  RacerDubinsElevation::state_array state;
  RacerDubinsElevation::state_array next_state_cpu;
  RacerDubinsElevation::control_array control;
  RacerDubinsElevation::output_array output;
  RacerDubinsElevation::state_array state_der_cpu = RacerDubinsElevation::state_array::Zero();

  // Run dynamics on dynamicsU
  // Run dynamics on GPU
  for (int y_dim = 1; y_dim <= 16; y_dim++)
  {
    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < s[0].size(); dim++)
      {
        s[state_index][dim] = state_trajectory.col(state_index)(dim);
      }
      for (int dim = 0; dim < u[0].size(); dim++)
      {
        u[state_index][dim] = control_trajectory.col(state_index)(dim);
      }
    }

    launchStepTestKernel<RacerDubinsElevation>(dynamics, s, u, s_der, s_next, 0, dt, y_dim);
    for (int point = 0; point < num_rollouts; point++)
    {
      state = state_trajectory.col(point);
      control = control_trajectory.col(point);
      state_der_cpu = RacerDubinsElevation::state_array::Zero();

      dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
      for (int dim = 0; dim < RacerDubinsElevation::STATE_DIM; dim++)
      {
        EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        // EXPECT_NEAR(state(dim), s[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        EXPECT_NEAR(next_state_cpu(dim), s_next[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        EXPECT_TRUE(isfinite(s_next[point][dim]));
      }
    }
  }
  dynamics.freeCudaMem();
}

TEST_F(RacerDubinsElevationTest, ComputeStateTrajectoryFiniteTest)
{
  RacerDubinsElevation dynamics = RacerDubinsElevation();
  using PARAMS = RacerDubinsElevation::DYN_PARAMS_T;
  PARAMS params;
  params.c_t[0] = 3.0;
  params.c_b[0] = 0.2;
  params.c_v[0] = 0.2;
  params.c_0 = 0.2;
  params.wheel_base = 3.0;
  params.steering_constant = 1.0;
  dynamics.setParams(params);

  Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, 500> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, 500>::Zero();
  Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, 500> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, 500>::Zero();
  RacerDubinsElevation::state_array state_der;
  RacerDubinsElevation::state_array x, x_next;
  RacerDubinsElevation::output_array output;
  x << 0, 1.46919e-6, 0.0140179, 1.09739e-8, -0.000735827;

  for (int i = 0; i < 500; i++)
  {
    RacerDubinsElevation::control_array u = control_trajectory.col(i);
    dynamics.step(x, x_next, state_der, u, output, i, 0.02);
    dynamics.computeDynamics(x, u, state_der);
    EXPECT_TRUE(x.allFinite());
    EXPECT_TRUE(x_next.allFinite());
    EXPECT_TRUE(state_der.allFinite());
    EXPECT_TRUE(u.allFinite());
    EXPECT_TRUE(state_der != RacerDubinsElevation::state_array::Zero());
    x = x_next;
  }
  params.steering_constant = 0.5;
  dynamics.setParams(params);

  x << 0, 1.46919e-6, 0.0140179, 1.09739e-8, -1.0;
  for (int i = 0; i < 500; i++)
  {
    RacerDubinsElevation::control_array u = control_trajectory.col(i);
    dynamics.step(x, x_next, state_der, u, output, i, 0.02);
    dynamics.computeDynamics(x, u, state_der);
    EXPECT_TRUE(x.allFinite());
    EXPECT_TRUE(x_next.allFinite());
    EXPECT_TRUE(state_der.allFinite());
    EXPECT_TRUE(u.allFinite());
    EXPECT_TRUE(state_der != RacerDubinsElevation::state_array::Zero());
    x = x_next;
  }
}

class LinearDummy : public RacerDubinsElevation
{
public:
  bool computeGrad(const Eigen::Ref<const state_array>& state, const Eigen::Ref<const control_array>& control,
                   Eigen::Ref<dfdx> A, Eigen::Ref<dfdu> B)
  {
    return false;
  };
};

TEST_F(RacerDubinsElevationTest, TestComputeGradComputation)
{
  GTEST_SKIP();
  Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM,
                RacerDubinsElevation::STATE_DIM + RacerDubinsElevation::CONTROL_DIM>
      numeric_jac;
  Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM,
                RacerDubinsElevation::STATE_DIM + RacerDubinsElevation::CONTROL_DIM>
      analytic_jac;

  const int num_rollouts = 100;
  Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevation::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevation::STATE_DIM, num_rollouts>::Random();

  // TODO properly scale the random values

  auto analytic_grad_model = RacerDubinsElevation();

  RacerDubinsElevation::dfdx A_analytic = RacerDubinsElevation::dfdx::Zero();
  RacerDubinsElevation::dfdu B_analytic = RacerDubinsElevation::dfdu::Zero();

  auto numerical_grad_model = LinearDummy();

  std::shared_ptr<ModelWrapperDDP<LinearDummy>> ddp_model =
      std::make_shared<ModelWrapperDDP<LinearDummy>>(&numerical_grad_model);

  auto params = analytic_grad_model.getParams();
  // params.c_t[0] = 3.0;
  // params.c_b[0] = 0.2;
  // params.c_v[0] = 0.2;
  params.c_0 = 0.0;
  params.wheel_base = 3.0;
  params.steering_constant = 1.1;
  params.low_min_throttle = 0.0;
  params.max_brake_rate_pos = 10.0;
  analytic_grad_model.setParams(params);
  numerical_grad_model.setParams(params);

  auto limits = analytic_grad_model.getControlRanges();
  limits[0].x = -0.3;
  limits[0].y = 1.0;
  limits[1].x = -1.0;
  limits[1].y = 1.0;
  analytic_grad_model.setControlRanges(limits);
  numerical_grad_model.setControlRanges(limits);

  state_trajectory.col(0) << 5, M_PI_4, 1, 1, 3, 0, 0, 0, 2;
  control_trajectory.col(0) << 0.5, 1;

  state_trajectory.col(1) << 5, M_PI_4, 1, 1, 3, 0, 0, 0, 2;
  control_trajectory.col(1) << -1.0, 1;

  // state_trajectory.col(5) = state_trajectory.col(5).cwiseAbs();

  for (int i = 0; i < num_rollouts; i++)
  {
    RacerDubinsElevation::state_array state = state_trajectory.col(i);
    RacerDubinsElevation::control_array control = control_trajectory.col(i);

    if (abs(state(0)) < 1)
    {
      state(0) = state(0) * 10;
      state(1) = state(1) * M_PI;
      state(2) = state(2) * 100;
      state(3) = state(3) * 100;
      state(4) = state(4) * 5;
      state(6) = state(6) * M_PI_2;
      state(7) = state(7) * M_PI_2;
      state(8) = state(8) * 10;
    }
    state(5) = min(abs(state(5) / 0.33f), 0.3f);

    // std::cout << "iteration " << i << std::endl;
    // std::cout << "state: " << state.transpose() << std::endl;
    // std::cout << "control: " << control.transpose() << std::endl;

    bool analytic_grad = analytic_grad_model.computeGrad(state, control, A_analytic, B_analytic);
    EXPECT_TRUE(analytic_grad);

    analytic_jac.leftCols<RacerDubinsElevation::STATE_DIM>() = A_analytic;
    analytic_jac.rightCols<RacerDubinsElevation::CONTROL_DIM>() = B_analytic;
    numeric_jac = ddp_model->df(state, control);

    EXPECT_LT((numeric_jac - analytic_jac).norm(), 5e-2)
        << "at index " << i << "\nstate: " << state.transpose() << "\ncontrol " << control.transpose()
        << "\nNumeric Jacobian\n"
        << numeric_jac << "\nAnalytic Jacobian\n"
        << analytic_jac;
  }
}
