#include "hip/hip_runtime.h"
#include <Eigen/Dense>
#include <gtest/gtest.h>
#include <mppi/dynamics/racer_dubins/racer_dubins_elevation_lstm_steering.cuh>
#include <kernel_tests/dynamics/dynamics_generic_kernel_tests.cuh>
#include <mppi/ddp/ddp_model_wrapper.h>
#include <racer_test_networks.h>
#include <hip/hip_runtime.h>

class RacerDubinsElevationLSTMSteeringTest : public ::testing::Test
{
public:
  hipStream_t stream;

  void SetUp() override
  {
    CudaCheckError();
    HANDLE_ERROR(hipStreamCreate(&stream));
  }

  void TearDown() override
  {
    CudaCheckError();
    HANDLE_ERROR(hipStreamDestroy(stream));
  }

  std::vector<int> init_output_layers = { 23, 100, 8 };
  std::vector<int> output_layers = { 8, 20, 1 };
};

TEST_F(RacerDubinsElevationLSTMSteeringTest, Template)
{
  auto dynamics = RacerDubinsElevationLSTMSteering(3, 20, init_output_layers, 4, 4, output_layers, 11);
  EXPECT_EQ(19, RacerDubinsElevationLSTMSteering::STATE_DIM);
  EXPECT_EQ(2, RacerDubinsElevationLSTMSteering::CONTROL_DIM);
  EXPECT_TRUE(dynamics.checkRequiresBuffer());
  EXPECT_NE(dynamics.getTextureHelper(), nullptr);
  EXPECT_EQ(dynamics.getBlkSharedSizeBytes(), 432);
  EXPECT_EQ(dynamics.getGrdSharedSizeBytes(), 1408);
}

TEST_F(RacerDubinsElevationLSTMSteeringTest, BindStream)
{
  auto dynamics = RacerDubinsElevationLSTMSteering(3, 20, init_output_layers, 4, 4, output_layers, 11);
  dynamics.bindToStream(stream);

  EXPECT_EQ(dynamics.stream_, stream) << "Stream binding failure.";
  EXPECT_NE(dynamics.getTextureHelper(), nullptr);
  EXPECT_EQ(dynamics.getTextureHelper()->stream_, stream);
  EXPECT_NE(dynamics.getHelper(), nullptr);
  EXPECT_EQ(dynamics.getHelper()->getLSTMModel()->stream_, stream);

  auto dynamics2 = RacerDubinsElevationLSTMSteering(3, 20, init_output_layers, 4, 4, output_layers, 11, stream);

  EXPECT_EQ(dynamics2.stream_, stream) << "Stream binding failure.";
  EXPECT_NE(dynamics2.getTextureHelper(), nullptr);
  EXPECT_EQ(dynamics2.getTextureHelper()->stream_, stream);
  EXPECT_NE(dynamics2.getHelper(), nullptr);
  EXPECT_EQ(dynamics2.getHelper()->getLSTMModel()->stream_, stream);
}

/*
float c_t = 1.3;
float c_b = 2.5;
float c_v = 3.7;
float c_0 = 4.9;
float wheel_base = 0.3;
 */

// TEST_F(RacerDubinsElevationLSTMSteeringTest, ComputeDynamics)
// {
//   RacerDubinsElevationLSTMSteering dynamics = RacerDubinsElevationLSTMSteering();
//   auto params = dynamics.getParams();
//   RacerDubinsElevationLSTMSteering::state_array x = RacerDubinsElevationLSTMSteering::state_array::Zero();
//   RacerDubinsElevationLSTMSteering::control_array u = RacerDubinsElevationLSTMSteering::control_array::Zero();

//   // computeDynamics should not touch the roll/pitch element
//   RacerDubinsElevationLSTMSteering::state_array next_x = RacerDubinsElevationLSTMSteering::state_array::Ones() *
//   0.153; dynamics.computeDynamics(x, u, next_x); EXPECT_FLOAT_EQ(next_x(0), 4.9); EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), 0);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 1, M_PI_2, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 2.6 - 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 1);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 1, M_PI_2, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 3.5 - 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 1);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -1, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 4.7 + 2.6 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -1);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -1, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.5 + 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -1);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.9 - 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), 7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.9 + 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 4.5 - 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), 7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -7, 0, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, 0;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 4.5 + 5.7 * 7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), 0);
//   EXPECT_FLOAT_EQ(next_x(2), -7);
//   EXPECT_FLOAT_EQ(next_x(3), 0);
//   EXPECT_FLOAT_EQ(next_x(4), 0);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 1, M_PI_2, 0, 3, 0, 0.5, -0.5, 0.0, 0.0;
//   u << 0, 1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 4.7 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (1 / .3) * tan(0));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 1);
//   EXPECT_FLOAT_EQ(next_x(4), 5 * 0.6);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -1, M_PI_2, 0, 3, 5.0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, -1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 4.7 + 3.5 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (-1 / .3) * tan(5.0 / -10.2));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), -1);
//   EXPECT_FLOAT_EQ(next_x(4), -5);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << -0.4, M_PI_2, 0, 3, 5.0, 0.5, -0.5, 0.0, 0.0;
//   u << -1, -1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 + 3.7 * 0.4 + 2.5 * 0.4 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (-0.4 / .3) * tan(5.0 / -9.1));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), -0.4);
//   EXPECT_FLOAT_EQ(next_x(4), -5);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);

//   x << 0.4, M_PI_2, 0, 3, 5.0, 0.5, -0.5, 0.0, 0.0;
//   u << 0.1, -1;
//   dynamics.computeDynamics(x, u, next_x);
//   EXPECT_FLOAT_EQ(next_x(0), 4.9 - 3.7 * 0.4 - sinf(-0.5) * -9.81);
//   EXPECT_FLOAT_EQ(next_x(1), (0.4 / .3) * tan(5.0 / -9.1));
//   EXPECT_NEAR(next_x(2), 0, 1e-7);
//   EXPECT_FLOAT_EQ(next_x(3), 0.4);
//   EXPECT_FLOAT_EQ(next_x(4), -5);
//   EXPECT_FLOAT_EQ(next_x(5), 0.153);
//   EXPECT_FLOAT_EQ(next_x(6), 0.153);
// }

// TEST_F(RacerDubinsElevationLSTMSteeringTest, TestModelGPU)
// {
//   RacerDubinsElevationLSTMSteering dynamics = RacerDubinsElevationLSTMSteering();
//   dynamics.GPUSetup();
//   CudaCheckError();

//   Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, 100> control_trajectory;
//   control_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, 100>::Random();
//   Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, 100> state_trajectory;
//   state_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, 100>::Random();

//   std::vector<std::array<float, 9>> s(100);
//   std::vector<std::array<float, 9>> s_der(100);
//   // steering, throttle
//   std::vector<std::array<float, 2>> u(100);
//   for (int state_index = 0; state_index < s.size(); state_index++)
//   {
//     for (int dim = 0; dim < s[0].size(); dim++)
//     {
//       s[state_index][dim] = state_trajectory.col(state_index)(dim);
//     }
//     for (int dim = 0; dim < u[0].size(); dim++)
//     {
//       u[state_index][dim] = control_trajectory.col(state_index)(dim);
//     }
//   }

//   // These variables will be changed so initialized to the right size only

//   // Run dynamics on dynamicsU
//   // Run dynamics on GPU
//   for (int y_dim = 1; y_dim <= 4; y_dim++)
//   {
//     launchComputeDynamicsTestKernel<RacerDubinsElevationLSTMSteering, 9, 2>(dynamics, s, u, s_der, y_dim);
//     for (int point = 0; point < 100; point++)
//     {
//       RacerDubinsElevationLSTMSteering::state_array state = state_trajectory.col(point);
//       RacerDubinsElevationLSTMSteering::control_array control = control_trajectory.col(point);
//       RacerDubinsElevationLSTMSteering::state_array state_der_cpu =
//       RacerDubinsElevationLSTMSteering::state_array::Zero();

//       dynamics.computeDynamics(state, control, state_der_cpu);
//       for (int dim = 0; dim < 6; dim++)
//       {
//         EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-5)
//             << "at point " << point << " dim " << dim << " with y_dim " << y_dim;
//         EXPECT_TRUE(isfinite(s_der[point][dim]));
//       }
//     }
//   }

//   dynamics.freeCudaMem();
//   CudaCheckError();
// }

// TEST_F(RacerDubinsElevationLSTMSteeringTest, TestUpdateState)
// {
//   CudaCheckError();
//   RacerDubinsElevationLSTMSteering dynamics = RacerDubinsElevationLSTMSteering();
//   RacerDubinsElevationLSTMSteering::state_array state;
//   RacerDubinsElevationLSTMSteering::state_array next_state;
//   RacerDubinsElevationLSTMSteering::state_array state_der;

//   // TODO add in the elevation map

//   state << 0, 0, 0, 0, 0, -0.5, 0.5;
//   state_der << 1, 1, 1, 1, 1, 0, 0;
//   dynamics.updateState(state, next_state, state_der, 0.1);
//   EXPECT_TRUE(state_der != RacerDubinsElevationLSTMSteering::state_array::Zero());
//   EXPECT_FLOAT_EQ(next_state(0), 0.1);
//   EXPECT_FLOAT_EQ(next_state(1), 0.1);
//   EXPECT_FLOAT_EQ(next_state(2), 0.1);
//   EXPECT_FLOAT_EQ(next_state(3), 0.1);
//   EXPECT_FLOAT_EQ(next_state(4), 0.1);
//   EXPECT_FLOAT_EQ(next_state(5), 0.0);
//   EXPECT_FLOAT_EQ(next_state(6), 0.0);

//   state << 0, M_PI - 0.1, 0, 0, 0, -0.5, 0.5;
//   state_der << 1, 1, 1, 1, 1;
//   dynamics.updateState(state, next_state, state_der, 1.0);
//   EXPECT_TRUE(state_der != RacerDubinsElevationLSTMSteering::state_array::Zero());
//   EXPECT_FLOAT_EQ(next_state(0), 1.0);
//   EXPECT_FLOAT_EQ(next_state(1), 1.0 - M_PI - 0.1);
//   EXPECT_FLOAT_EQ(next_state(2), 1.0);
//   EXPECT_FLOAT_EQ(next_state(3), 1.0);
//   EXPECT_FLOAT_EQ(next_state(4), 0.5);
//   EXPECT_FLOAT_EQ(next_state(5), 0.0);
//   EXPECT_FLOAT_EQ(next_state(6), 0.0);

//   state << 0, -M_PI + 0.1, 0, 0, 0, -0.5, 0.5;
//   state_der << 1, -1, 1, 1, 1;
//   dynamics.updateState(state, next_state, state_der, 1.0);
//   EXPECT_TRUE(state_der != RacerDubinsElevationLSTMSteering::state_array::Zero());
//   EXPECT_FLOAT_EQ(next_state(0), 1.0);
//   EXPECT_FLOAT_EQ(next_state(1), M_PI + 0.1 - 1.0);
//   EXPECT_FLOAT_EQ(next_state(2), 1.0);
//   EXPECT_FLOAT_EQ(next_state(3), 1.0);
//   EXPECT_FLOAT_EQ(next_state(4), 0.5);
//   EXPECT_FLOAT_EQ(next_state(5), 0.0);
//   EXPECT_FLOAT_EQ(next_state(6), 0.0);

//   CudaCheckError();
// }

TEST_F(RacerDubinsElevationLSTMSteeringTest, TestStep)
{
  GTEST_SKIP() << "Skipping test since they have not been updated to accel ";
  CudaCheckError();
  using DYN = RacerDubinsElevationLSTMSteering;
  const float tol = 1e-6;
  auto dynamics = RacerDubinsElevationLSTMSteering(3, 20, init_output_layers, 4, 4, output_layers, 11);
  auto params = dynamics.getParams();
  params.c_0 = 0;
  params.c_b[0] = 1;
  params.c_b[1] = 10;
  params.c_b[2] = 100;
  params.c_v[0] = 0.25;
  params.c_v[1] = 0.5;
  params.c_v[2] = 0.75;
  params.c_t[0] = 2;
  params.c_t[1] = 20;
  params.c_t[2] = 200;
  params.low_min_throttle = 0.2;
  params.steer_command_angle_scale = 0.5;
  params.steering_constant = 0.5;
  params.wheel_base = 0.5;
  params.max_steer_rate = 5;
  params.max_steer_angle = 5;
  dynamics.setParams(params);
  DYN::state_array state;
  DYN::state_array next_state;
  DYN::state_array state_der = DYN::state_array::Zero();
  DYN::control_array control;
  DYN::output_array output;
  float dt = 0.1;
  // TODO add in the elevation map

  auto model = dynamics.getHelper();
  model->getLSTMModel()->getOutputModel()->setAllWeights(0.1f);
  model->getInitModel()->getOutputModel()->setAllWeights(0.01f);
  model->getLSTMModel()->setAllValues(0.3f);
  model->getInitModel()->setAllValues(0.01f);

  Eigen::MatrixXf buffer = model->getEmptyBufferMatrix();
  buffer.setOnes();
  buffer = buffer * 0.01f;

  // Basic initial state and no movement should stay still
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 0, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), 0.0, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 4.1500520706176758 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 4.1500520706176758, tol);
  EXPECT_NEAR(output(23), 0.0, tol);

  // Apply full throttle from zero state
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0;
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 1.6, tol);
  EXPECT_NEAR(next_state(0), 0.16, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 5.2751355171203613 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 5.2751355171203613, tol);
  EXPECT_NEAR(output(23), 1.6, tol);

  // Apply throttle to a state with positive velocity
  state << 1, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 5.5, tol);
  EXPECT_NEAR(next_state(0), 1.55, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.1, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 7.1901092529296875 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 7.1901092529296875, tol);
  EXPECT_NEAR(output(23), 5.5, tol);

  // Apply full throttle and half left turn to origin state
  state << 0, 0, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 1.6, tol);
  EXPECT_NEAR(next_state(0), 0.16, tol);
  EXPECT_NEAR(next_state(1), 0.0, tol);
  EXPECT_NEAR(next_state(2), 0.0, tol);
  EXPECT_NEAR(next_state(3), 0.0, tol);
  EXPECT_NEAR(next_state(4), 6.1967658996582031 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 6.1967658996582031, tol);
  EXPECT_NEAR(output(23), 1.6, tol);

  // Apply full throttle and half left turn to a moving state oriented 30 degrees to the left
  float yaw = M_PI / 6;
  state << 1.0, yaw, 0, 0, 0, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 5.5, tol);
  EXPECT_NEAR(next_state(0), 1.55, tol);
  EXPECT_NEAR(next_state(1), yaw, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), 9.0641689300537109 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 9.0641689300537109, tol);
  EXPECT_NEAR(output(23), 5.5, tol);

  // Apply full throttle and half left turn to a moving state oriented 30 degrees to the left which is already turning
  float steer_angle = M_PI / 8;
  state << 1.0, yaw, 0, 0, steer_angle, -0.0, 0.0, 0, 0;
  control << 1, 0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), 5.5, tol);
  EXPECT_NEAR(next_state(0), 1.55, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + 9.3808889389038086 * dt, tol);
  EXPECT_NEAR(next_state(5), 0.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 9.3808889389038086, tol);
  EXPECT_NEAR(output(23), 5.5, tol);

  // Apply full brake and half left turn to a moving state oriented 30 degrees to the left which is already turning
  state << 1.0, yaw, 0, 0, steer_angle, 1.0, -0.0, 0.0, 0, 0;
  control << -1, 0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(state_der(0), -5.5, tol);
  EXPECT_NEAR(next_state(0), 1 - 5.5 * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + 9.3808889389038086 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 9.3808889389038086, tol);
  EXPECT_NEAR(output(23), -5.5, tol);

  /**
   * Apply full brake and half left turn to a moving state oriented 30 degrees to the left which is already turning
   * and on a downward facing hill
   */
  float pitch = 20 * M_PI / 180;
  state << 1.0, yaw, 0, 0, steer_angle, 1, -0.0, pitch, 0, 0;
  control << -1, 0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), 1 + (-5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + -0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), 1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), 1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + 9.3808889389038086 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 9.3808889389038086, tol);
  EXPECT_NEAR(output(23), (-5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half left turn to a backwards moving state oriented 30 degrees to the left which is already
   * turning and on a downward facing hill
   */
  state << -1.0, yaw, 0, 0, steer_angle, 1, -0.0, pitch, 0, 0;
  control << -1, 0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + 0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + 3.5283551216125488 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 3.5283551216125488, tol);
  EXPECT_NEAR(output(23), (5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half right turn to a backwards moving state oriented 30 degrees to the left which is already
   * turning and on a downward facing hill
   */
  state << -1.0, yaw, 0, 0, steer_angle, 1, -0.0, pitch, 0, 0;
  control << -1, -0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + 0.086361105 * dt, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), steer_angle + -0.32771033048629761 * dt, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), -0.32771033048629761, tol);
  EXPECT_NEAR(output(23), (5.5 + 9.81 * sinf(pitch)), tol);

  /**
   * Apply full brake and half right turn to a backwards moving state with a huge steering angle to test max steer
   * angle and steering rate. We are also on a downward facing hill and are already oriented 30 degrees to the left
   */
  steer_angle *= 100;
  state << -1.0, yaw, 0, 0, steer_angle, 1, -0.0, pitch, 0, 0;
  control << -1, -0.5;
  model->initializeLSTM(buffer);
  dynamics.step(state, next_state, state_der, control, output, 0, dt);
  EXPECT_NEAR(next_state(0), -1 + (5.5 + 9.81 * sinf(pitch)) * dt, tol);
  EXPECT_NEAR(next_state(1), yaw + tan(steer_angle / -9.1) * dt * -2, tol);
  EXPECT_NEAR(next_state(2), -1 * cos(yaw) * dt, tol);
  EXPECT_NEAR(next_state(3), -1 * sin(yaw) * dt, tol);
  EXPECT_NEAR(next_state(4), params.max_steer_angle, tol);
  EXPECT_NEAR(next_state(5), 1.0, tol);
  EXPECT_NEAR(next_state(6), 0.0, tol);
  EXPECT_NEAR(next_state(7), 0.0, tol);
  EXPECT_NEAR(next_state(8), 15.97845268249511, tol);
  EXPECT_NEAR(output(23), (5.5 + 9.81 * sinf(pitch)), tol);
}

TEST_F(RacerDubinsElevationLSTMSteeringTest, TestStepGPUvsCPUNoNetwork)
{
  const int num_rollouts = 1000;
  const float dt = 0.1f;
  CudaCheckError();
  using DYN = RacerDubinsElevationLSTMSteering;
  RacerDubinsElevationLSTMSteering dynamics =
      RacerDubinsElevationLSTMSteering(3, 20, init_output_layers, 4, 4, output_layers, 11);

  hipExtent extent = make_hipExtent(10, 20, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(10 * 20);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(1, 2, 3));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  CudaCheckError();
  dynamics.GPUSetup();
  CudaCheckError();

  EXPECT_NE(dynamics.getHelper()->getLSTMDevicePtr(), nullptr);
  EXPECT_NE(dynamics.network_d_, nullptr);
  EXPECT_EQ(dynamics.network_d_, dynamics.getHelper()->getLSTMDevicePtr());

  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts>::Random();

  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s_next(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s_der(num_rollouts);
  // steering, throttle
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM>> u(num_rollouts);

  RacerDubinsElevationLSTMSteering::state_array state;
  RacerDubinsElevationLSTMSteering::state_array next_state_cpu;
  RacerDubinsElevationLSTMSteering::control_array control;
  RacerDubinsElevationLSTMSteering::output_array output;
  RacerDubinsElevationLSTMSteering::state_array state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

  // Run dynamics on dynamicsU
  // Run dynamics on GPU
  for (int y_dim = 1; y_dim <= 16; y_dim++)
  {
    DYN::buffer_trajectory buffer;
    buffer["VEL_X"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE_RATE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_CMD"] = Eigen::VectorXf::Random(51);

    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < s[0].size(); dim++)
      {
        s[state_index][dim] = state_trajectory.col(state_index)(dim);
      }
      for (int dim = 0; dim < u[0].size(); dim++)
      {
        u[state_index][dim] = control_trajectory.col(state_index)(dim);
      }
    }
    dynamics.updateFromBuffer(buffer);
    launchStepTestKernel<RacerDubinsElevationLSTMSteering, 16>(dynamics, s, u, s_der, s_next, 0, dt, y_dim);
    for (int point = 0; point < num_rollouts; point++)
    {
      dynamics.initializeDynamics(state, control, output, 0, 0);
      state = state_trajectory.col(point);
      control = control_trajectory.col(point);
      state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

      dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
      // for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::STATE_DIM; dim++)
      for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::STATE_DIM; dim++)
      {
        EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        // EXPECT_NEAR(state(dim), s[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        EXPECT_NEAR(next_state_cpu(dim), s_next[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        EXPECT_TRUE(isfinite(s_next[point][dim]));
      }
    }
  }
  dynamics.freeCudaMem();
}

TEST_F(RacerDubinsElevationLSTMSteeringTest, TestStepGPUvsCPU)
{
  const int num_rollouts = 60;
  const float dt = 0.1f;
  CudaCheckError();
  using DYN = RacerDubinsElevationLSTMSteering;
  RacerDubinsElevationLSTMSteering dynamics = RacerDubinsElevationLSTMSteering(mppi::tests::steering_lstm);
  EXPECT_FLOAT_EQ(dynamics.getParams().max_steer_rate, 17.590296);
  EXPECT_FLOAT_EQ(dynamics.getParams().steering_constant, 3.286375);
  EXPECT_FLOAT_EQ(dynamics.getParams().steer_accel_constant, 9.301527);
  EXPECT_FLOAT_EQ(dynamics.getParams().steer_accel_drag_constant, -0.60327667);

  hipExtent extent = make_hipExtent(10, 20, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(10 * 20);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(1, 2, 3));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  CudaCheckError();
  dynamics.GPUSetup();
  CudaCheckError();

  EXPECT_NE(dynamics.getHelper()->getLSTMDevicePtr(), nullptr);
  EXPECT_NE(dynamics.network_d_, nullptr);
  EXPECT_EQ(dynamics.network_d_, dynamics.getHelper()->getLSTMDevicePtr());

  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts>::Random();

  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s_next(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s_der(num_rollouts);
  // steering, throttle
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM>> u(num_rollouts);

  RacerDubinsElevationLSTMSteering::state_array state;
  RacerDubinsElevationLSTMSteering::state_array next_state_cpu;
  RacerDubinsElevationLSTMSteering::control_array control;
  RacerDubinsElevationLSTMSteering::output_array output;
  RacerDubinsElevationLSTMSteering::state_array state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

  // Run dynamics on dynamicsU
  // Run dynamics on GPU
  for (int y_dim = 1; y_dim <= 16; y_dim++)
  {
    DYN::buffer_trajectory buffer;
    buffer["VEL_X"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE_RATE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_CMD"] = Eigen::VectorXf::Random(51);

    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < s[0].size(); dim++)
      {
        s[state_index][dim] = state_trajectory.col(state_index)(dim);
      }
      for (int dim = 0; dim < u[0].size(); dim++)
      {
        u[state_index][dim] = control_trajectory.col(state_index)(dim);
      }
    }
    dynamics.updateFromBuffer(buffer);
    launchStepTestKernel<RacerDubinsElevationLSTMSteering, 16>(dynamics, s, u, s_der, s_next, 0, dt, y_dim);
    for (int point = 0; point < num_rollouts; point++)
    {
      dynamics.initializeDynamics(state, control, output, 0, 0);
      state = state_trajectory.col(point);
      control = control_trajectory.col(point);
      state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

      dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
      for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::STATE_DIM; dim++)
      {
        EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        // EXPECT_NEAR(state(dim), s[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        EXPECT_NEAR(next_state_cpu(dim), s_next[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        EXPECT_TRUE(isfinite(s_next[point][dim]));
      }
    }
  }
  dynamics.freeCudaMem();
}

// TODO assert they are different when a network is loaded, check params are the same

TEST_F(RacerDubinsElevationLSTMSteeringTest, TestStepGPUvsCPUReverse)
{
  using DYN = RacerDubinsElevationLSTMSteering;

  const int num_rollouts = 1000;
  const float dt = 0.1f;
  CudaCheckError();
  RacerDubinsElevationLSTMSteering dynamics = RacerDubinsElevationLSTMSteering(mppi::tests::steering_lstm);
  auto params = dynamics.getParams();
  params.gear_sign = -1;
  dynamics.setParams(params);
  EXPECT_FLOAT_EQ(dynamics.getParams().max_steer_rate, 17.590296);
  EXPECT_FLOAT_EQ(dynamics.getParams().steering_constant, 3.286375);
  EXPECT_FLOAT_EQ(dynamics.getParams().steer_accel_constant, 9.301527);
  EXPECT_FLOAT_EQ(dynamics.getParams().steer_accel_drag_constant, -0.60327667);

  hipExtent extent = make_hipExtent(10, 20, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(10 * 20);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(1, 2, 3));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  CudaCheckError();
  dynamics.GPUSetup();
  CudaCheckError();

  EXPECT_NE(dynamics.getHelper()->getLSTMDevicePtr(), nullptr);
  EXPECT_NE(dynamics.network_d_, nullptr);
  EXPECT_EQ(dynamics.network_d_, dynamics.getHelper()->getLSTMDevicePtr());

  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts>::Random();

  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s_next(num_rollouts);
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::STATE_DIM>> s_der(num_rollouts);
  // steering, throttle
  std::vector<std::array<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM>> u(num_rollouts);

  RacerDubinsElevationLSTMSteering::state_array state;
  RacerDubinsElevationLSTMSteering::state_array next_state_cpu;
  RacerDubinsElevationLSTMSteering::control_array control;
  RacerDubinsElevationLSTMSteering::output_array output;
  RacerDubinsElevationLSTMSteering::state_array state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

  // Run dynamics on dynamicsU
  // Run dynamics on GPU
  for (int y_dim = 1; y_dim <= 16; y_dim++)
  {
    DYN::buffer_trajectory buffer;
    buffer["VEL_X"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE_RATE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_CMD"] = Eigen::VectorXf::Random(51);

    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < s[0].size(); dim++)
      {
        s[state_index][dim] = state_trajectory.col(state_index)(dim);
      }
      for (int dim = 0; dim < u[0].size(); dim++)
      {
        u[state_index][dim] = control_trajectory.col(state_index)(dim);
      }
    }
    dynamics.updateFromBuffer(buffer);
    launchStepTestKernel<RacerDubinsElevationLSTMSteering, 16>(dynamics, s, u, s_der, s_next, 0, dt, y_dim);
    for (int point = 0; point < num_rollouts; point++)
    {
      dynamics.initializeDynamics(state, control, output, 0, 0);
      state = state_trajectory.col(point);
      control = control_trajectory.col(point);
      state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

      dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
      // for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::STATE_DIM; dim++)
      for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::STATE_DIM; dim++)
      {
        EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        // EXPECT_NEAR(state(dim), s[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
        EXPECT_NEAR(next_state_cpu(dim), s_next[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        EXPECT_TRUE(isfinite(s_next[point][dim]));
      }
    }
  }
  dynamics.freeCudaMem();
}

TEST_F(RacerDubinsElevationLSTMSteeringTest, compareToElevationWithoutSteering)
{
  // by default the network will output zeros and not effect any states
  using DYN = RacerDubinsElevationLSTMSteering;

  const int num_rollouts = 1000;
  const float dt = 0.1f;
  CudaCheckError();
  auto dynamics = RacerDubinsElevationLSTMSteering(3, 20, init_output_layers, 4, 4, output_layers, 11);
  RacerDubinsElevation dynamics2 = RacerDubinsElevation();
  auto params = dynamics.getParams();
  dynamics.setParams(params);
  dynamics2.setParams(params);

  hipExtent extent = make_hipExtent(10, 20, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(10 * 20);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(1, 2, 3));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  TwoDTextureHelper<float>* helper2 = dynamics2.getTextureHelper();
  helper2->setExtent(0, extent);

  helper2->updateRotation(0, new_rot_mat);
  helper2->updateOrigin(0, make_float3(1, 2, 3));

  data_vec.resize(10 * 20);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }
  helper2->updateTexture(0, data_vec);
  helper2->updateResolution(0, 10);
  helper2->enableTexture(0);
  helper2->copyToDevice(true);

  CudaCheckError();
  dynamics.GPUSetup();
  dynamics2.GPUSetup();
  CudaCheckError();

  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, num_rollouts>::Random();

  RacerDubinsElevationLSTMSteering::state_array state;
  RacerDubinsElevationLSTMSteering::state_array next_state_cpu;
  RacerDubinsElevationLSTMSteering::control_array control;
  RacerDubinsElevationLSTMSteering::output_array output;
  RacerDubinsElevationLSTMSteering::state_array state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

  RacerDubinsElevationLSTMSteering::state_array state2;
  RacerDubinsElevationLSTMSteering::state_array next_state_cpu2;
  RacerDubinsElevationLSTMSteering::control_array control2;
  RacerDubinsElevationLSTMSteering::output_array output2;
  RacerDubinsElevationLSTMSteering::state_array state_der_cpu2 = RacerDubinsElevationLSTMSteering::state_array::Zero();

  DYN::buffer_trajectory buffer;
  buffer["VEL_X"] = Eigen::VectorXf::Random(51);
  buffer["STEER_ANGLE"] = Eigen::VectorXf::Random(51);
  buffer["STEER_ANGLE_RATE"] = Eigen::VectorXf::Random(51);
  buffer["STEER_CMD"] = Eigen::VectorXf::Random(51);

  dynamics.updateFromBuffer(buffer);
  for (int point = 0; point < num_rollouts; point++)
  {
    dynamics.initializeDynamics(state, control, output, 0, 0);
    state = state_trajectory.col(point);
    control = control_trajectory.col(point);
    state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

    dynamics2.initializeDynamics(state2, control2, output2, 0, 0);
    state2 = state_trajectory.col(point);
    control2 = control_trajectory.col(point);
    state_der_cpu2 = RacerDubinsElevationLSTMSteering::state_array::Zero();

    dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
    dynamics2.step(state2, next_state_cpu2, state_der_cpu2, control2, output2, 0, dt);

    for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::STATE_DIM; dim++)
    {
      if (dim == S_IND(params, STEER_ANGLE) or dim == S_IND(params, STEER_ANGLE_RATE))
      {  // this is done since the steering wheel setup is different, accel version
        continue;
      }
      EXPECT_NEAR(state_der_cpu(dim), state_der_cpu2(dim), 1e-4) << "state der at index " << point << " dim " << dim;
      EXPECT_NEAR(next_state_cpu(dim), next_state_cpu2(dim), 1e-4) << "next state at index " << point << " dim " << dim;
    }
    for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::OUTPUT_DIM; dim++)
    {
      if (dim == O_IND(params, STEER_ANGLE) or dim == O_IND(params, STEER_ANGLE_RATE) or
          dim == O_IND(params, WHEEL_FORCE_UP_MAX) or dim == O_IND(params, WHEEL_FORCE_FWD_MAX) or
          dim == O_IND(params, WHEEL_FORCE_SIDE_MAX) or dim == O_IND(params, FILLER_1))
      {  // this is done since the steering wheel setup is different, accel version
        continue;
      }
      EXPECT_NEAR(output(dim), output2(dim), 1e-4) << "output at index " << point << " dim " << dim;
    }
  }

  params.gear_sign = -1;
  dynamics.setParams(params);
  dynamics2.setParams(params);

  // check in reverse as well
  for (int point = 0; point < num_rollouts; point++)
  {
    dynamics.initializeDynamics(state, control, output, 0, 0);
    state = state_trajectory.col(point);
    control = control_trajectory.col(point);
    state_der_cpu = RacerDubinsElevationLSTMSteering::state_array::Zero();

    dynamics2.initializeDynamics(state2, control2, output2, 0, 0);
    state2 = state_trajectory.col(point);
    control2 = control_trajectory.col(point);
    state_der_cpu2 = RacerDubinsElevationLSTMSteering::state_array::Zero();

    dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
    dynamics2.step(state2, next_state_cpu2, state_der_cpu2, control2, output2, 0, dt);

    for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::STATE_DIM; dim++)
    {
      if (dim == S_IND(params, STEER_ANGLE) or dim == S_IND(params, STEER_ANGLE_RATE))
      {  // this is done since the steering wheel setup is different, accel version
        continue;
      }
      EXPECT_NEAR(state_der_cpu(dim), state_der_cpu2(dim), 1e-4) << "at index " << point << " dim " << dim;
      EXPECT_NEAR(next_state_cpu(dim), next_state_cpu2(dim), 1e-4) << "at index " << point << " dim " << dim;
    }
    for (int dim = 0; dim < RacerDubinsElevationLSTMSteering::OUTPUT_DIM; dim++)
    {
      if (dim == O_IND(params, STEER_ANGLE) or dim == O_IND(params, STEER_ANGLE_RATE) or
          dim == O_IND(params, WHEEL_FORCE_UP_MAX) or dim == O_IND(params, WHEEL_FORCE_FWD_MAX) or
          dim == O_IND(params, WHEEL_FORCE_SIDE_MAX) or dim == O_IND(params, FILLER_1))
      {  // this is done since the steering wheel setup is different, accel version
        continue;
      }
      EXPECT_NEAR(output(dim), output2(dim), 1e-4) << "at index " << point << " dim " << dim;
    }
  }
  dynamics.freeCudaMem();
}

/*
class LinearDummy : public RacerDubinsElevationLSTMSteering {
public:
  bool computeGrad(const Eigen::Ref<const state_array> & state,
                   const Eigen::Ref<const control_array>& control,
                   Eigen::Ref<dfdx> A,
                   Eigen::Ref<dfdu> B) {
    return false;
  };
};

TEST_F(RacerDubinsElevationLSTMSteeringTest, TestComputeGradComputation) {
  Eigen::Matrix<float, RacerDubinsElevationLSTMSteering::STATE_DIM, RacerDubinsElevationLSTMSteering::STATE_DIM +
RacerDubinsElevationLSTMSteering::CONTROL_DIM> numeric_jac; Eigen::Matrix<float,
RacerDubinsElevationLSTMSteering::STATE_DIM, RacerDubinsElevationLSTMSteering::STATE_DIM +
RacerDubinsElevationLSTMSteering::CONTROL_DIM> analytic_jac; RacerDubinsElevationLSTMSteering::state_array state; state
<< 1, 2, 3, 4; RacerDubinsElevationLSTMSteering::control_array control; control << 5;

  auto analytic_grad_model = RacerDubinsElevationLSTMSteering();

  RacerDubinsElevationLSTMSteering::dfdx A_analytic = RacerDubinsElevationLSTMSteering::dfdx::Zero();
  RacerDubinsElevationLSTMSteering::dfdu B_analytic = RacerDubinsElevationLSTMSteering::dfdu::Zero();

  analytic_grad_model.computeGrad(state, control, A_analytic, B_analytic);

  auto numerical_grad_model = LinearDummy();

  std::shared_ptr<ModelWrapperDDP<LinearDummy>> ddp_model =
std::make_shared<ModelWrapperDDP<LinearDummy>>(&numerical_grad_model);

  analytic_jac.leftCols<RacerDubinsElevationLSTMSteering::STATE_DIM>() = A_analytic;
  analytic_jac.rightCols<RacerDubinsElevationLSTMSteering::CONTROL_DIM>() = B_analytic;
  numeric_jac = ddp_model->df(state, control);

  ASSERT_LT((numeric_jac - analytic_jac).norm(), 1e-3) << "Numeric Jacobian\n" << numeric_jac << "\nAnalytic Jacobian\n"
<< analytic_jac;
}

*/
