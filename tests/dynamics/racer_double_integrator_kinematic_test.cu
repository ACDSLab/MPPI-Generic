#include "hip/hip_runtime.h"
#include <Eigen/Dense>
#include <gtest/gtest.h>
#include <mppi/dynamics/bicycle_slip/racer_double_integrator_kinematic.cuh>
#include <mppi/dynamics/dynamics_generic_kernel_tests.cuh>
#include <mppi/ddp/ddp_model_wrapper.h>
#include <hip/hip_runtime.h>
#include <racer_test_networks.h>

class RacerDoubleIntegratorKinematicTest : public ::testing::Test
{
public:
  hipStream_t stream;

  void SetUp() override
  {
    CudaCheckError();
    HANDLE_ERROR(hipStreamCreate(&stream));
  }

  void TearDown() override
  {
    CudaCheckError();
    HANDLE_ERROR(hipStreamDestroy(stream));
  }
};

const double tol = 1e-5;

TEST_F(RacerDoubleIntegratorKinematicTest, Template)
{
  auto dynamics = RacerDoubleIntegratorKinematic();
  EXPECT_EQ(11, RacerDoubleIntegratorKinematic::STATE_DIM);
  EXPECT_EQ(2, RacerDoubleIntegratorKinematic::CONTROL_DIM);
  EXPECT_TRUE(dynamics.checkRequiresBuffer());
  EXPECT_NE(dynamics.getTextureHelper(), nullptr);

  EXPECT_NE(dynamics.getSteerHelper(), nullptr);
  EXPECT_NE(dynamics.getDelayHelper(), nullptr);
  EXPECT_NE(dynamics.getTerraHelper(), nullptr);
}

TEST_F(RacerDoubleIntegratorKinematicTest, BindStream)
{
  auto dynamics = RacerDoubleIntegratorKinematic(stream);

  EXPECT_EQ(dynamics.stream_, stream) << "Stream binding failure.";
  EXPECT_NE(dynamics.getTextureHelper(), nullptr);
  EXPECT_EQ(dynamics.getTextureHelper()->stream_, stream);

  EXPECT_NE(dynamics.getSteerHelper(), nullptr);
  EXPECT_EQ(dynamics.getSteerHelper()->getLSTMModel()->stream_, stream);
  EXPECT_NE(dynamics.getDelayHelper(), nullptr);
  EXPECT_EQ(dynamics.getDelayHelper()->getLSTMModel()->stream_, stream);
  EXPECT_NE(dynamics.getTerraHelper(), nullptr);
  EXPECT_EQ(dynamics.getTerraHelper()->getLSTMModel()->stream_, stream);
}

TEST_F(RacerDoubleIntegratorKinematicTest, computeDynamicsCPUZeroNetworks)
{
  auto dynamics = RacerDoubleIntegratorKinematic();

  auto params = dynamics.getParams();
  params.max_steer_angle = 5.0;
  params.wheel_base = 2.981;
  dynamics.setParams(params);

  RacerDoubleIntegratorKinematic::state_array x = RacerDoubleIntegratorKinematic::state_array::Zero();
  RacerDoubleIntegratorKinematic::control_array u = RacerDoubleIntegratorKinematic::control_array::Zero();
  RacerDoubleIntegratorKinematic::output_array output = RacerDoubleIntegratorKinematic::output_array::Zero();
  dynamics.initializeDynamics(x, u, output, 0, 0);

  // computeDynamics should not touch the roll/pitch element
  RacerDoubleIntegratorKinematic::state_array state_der = RacerDoubleIntegratorKinematic::state_array::Ones() * 0.153;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);   // x
  EXPECT_FLOAT_EQ(state_der(1), 0);   // y
  EXPECT_FLOAT_EQ(state_der(2), 0);   // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);   // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);   // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);   // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);   // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);   // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);   // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);  // steer angle rate

  u << 1, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);   // x
  EXPECT_FLOAT_EQ(state_der(1), 0);   // y
  EXPECT_FLOAT_EQ(state_der(2), 0);   // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);   // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);   // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);   // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);   // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);   // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);   // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);  // steer angle rate

  u << -1, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);     // x
  EXPECT_FLOAT_EQ(state_der(1), 0);     // y
  EXPECT_FLOAT_EQ(state_der(2), 0);     // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);     // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0.33);  // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);     // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);     // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);     // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);     // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);     // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);    // steer angle rate

  x << 0, 0, 0, 0, 1.0, 0, 0, 0, 0, 0, 0;
  u << 1, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);     // x
  EXPECT_FLOAT_EQ(state_der(1), 0);     // y
  EXPECT_FLOAT_EQ(state_der(2), 0);     // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);     // steer angle
  EXPECT_FLOAT_EQ(state_der(4), -0.9);  // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);     // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);     // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);     // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);     // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);     // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);    // steer angle rate

  x << 0, 0, 0, 0, 1.0, 0, 0, 0, 0, 0, 0;
  u << -0.9, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);      // x
  EXPECT_FLOAT_EQ(state_der(1), 0);      // y
  EXPECT_FLOAT_EQ(state_der(2), 0);      // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), -0.66);  // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);      // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);      // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);      // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);      // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);      // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);     // steer angle rate

  x << 0, 0, 0, 0, 1.0, 0, 0, 0, 0, 0, 0;
  u << -0.9, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);      // x
  EXPECT_FLOAT_EQ(state_der(1), 0);      // y
  EXPECT_FLOAT_EQ(state_der(2), 0);      // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), -0.66);  // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);      // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);      // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);      // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);      // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);      // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);     // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << -0.01, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);      // x
  EXPECT_FLOAT_EQ(state_der(1), 0);      // y
  EXPECT_FLOAT_EQ(state_der(2), 0);      // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0.066);  // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);      // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);      // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);      // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);      // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);      // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);     // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, 1.0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);    // x
  EXPECT_FLOAT_EQ(state_der(1), 0);    // y
  EXPECT_FLOAT_EQ(state_der(2), 0);    // yaw
  EXPECT_FLOAT_EQ(state_der(3), 3.0);  // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);    // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);    // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);    // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);    // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);    // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);    // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);   // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, -1.0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);   // x
  EXPECT_FLOAT_EQ(state_der(1), 0);   // y
  EXPECT_FLOAT_EQ(state_der(2), 0);   // yaw
  EXPECT_FLOAT_EQ(state_der(3), -3);  // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);   // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);   // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);   // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);   // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);   // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);  // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, 0.5;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);    // x
  EXPECT_FLOAT_EQ(state_der(1), 0);    // y
  EXPECT_FLOAT_EQ(state_der(2), 0);    // yaw
  EXPECT_FLOAT_EQ(state_der(3), 1.5);  // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);    // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);    // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);    // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);    // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);    // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);    // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);   // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, -0.1;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);     // x
  EXPECT_FLOAT_EQ(state_der(1), 0);     // y
  EXPECT_FLOAT_EQ(state_der(2), 0);     // yaw
  EXPECT_FLOAT_EQ(state_der(3), -0.3);  // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);     // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);     // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);     // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);     // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);     // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);     // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);    // steer angle rate

  x << 0, 0, 0, -2.5, 0, 0, 0, 0, 0, 0, 0;
  u << 0, -0.5;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);   // x
  EXPECT_FLOAT_EQ(state_der(1), 0);   // y
  EXPECT_FLOAT_EQ(state_der(2), 0);   // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);   // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);   // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);   // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);   // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);   // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);   // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);  // steer angle rate

  x << 0, 0, 0, 2.5, 0, 0, 0, 0, 0, 0, 0;
  u << 0, 0.5;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);   // x
  EXPECT_FLOAT_EQ(state_der(1), 0);   // y
  EXPECT_FLOAT_EQ(state_der(2), 0);   // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);   // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);   // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);   // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);   // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);   // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);   // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);  // steer angle rate

  x << 0, 0, 0, 2.5, 0, 1.0, 1.0, 1.0, 0, 0, 0;
  u << 0, 0.5;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 1);    // x
  EXPECT_FLOAT_EQ(state_der(1), 1);    // y
  EXPECT_FLOAT_EQ(state_der(2), 1.0);  // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);    // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);    // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);    // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);    // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);    // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);    // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);    // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);   // steer angle rate

  x << 0, 0, M_PI_4f32, 2.5, 0, 1.0, 1.0, 1.0, 0, 0, 0;
  u << 0, 0.5;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                    // x
  EXPECT_FLOAT_EQ(state_der(1), cosf(M_PI_4f32) * 2);  // y
  EXPECT_FLOAT_EQ(state_der(2), 1.0);                  // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);                    // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);                    // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);                    // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);                    // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);                    // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                    // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                    // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                   // steer angle rate

  x << 0, 0, -M_PI_4f32, 2.5, 0, 1.0, 0.0, 1.0, 0, 0, 0;
  u << 0, 0.5;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), cosf(M_PI_4f32));   // x
  EXPECT_FLOAT_EQ(state_der(1), -cosf(M_PI_4f32));  // y
  EXPECT_FLOAT_EQ(state_der(2), 1.0);               // yaw
  EXPECT_FLOAT_EQ(state_der(3), 0);                 // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0);                 // brake state
  EXPECT_FLOAT_EQ(state_der(5), 0);                 // vel x
  EXPECT_FLOAT_EQ(state_der(6), 0);                 // vel y
  EXPECT_FLOAT_EQ(state_der(7), 0);                 // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                 // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                 // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                // steer angle rate
}

TEST_F(RacerDoubleIntegratorKinematicTest, computeDynamicsCPUFakeNetworks)
{
  auto dynamics = RacerDoubleIntegratorKinematic();

  auto params = dynamics.getParams();
  params.max_steer_angle = 5.0;
  params.wheel_base = 2.981;
  dynamics.setParams(params);

  RacerDoubleIntegratorKinematic::state_array x = RacerDoubleIntegratorKinematic::state_array::Zero();
  RacerDoubleIntegratorKinematic::control_array u = RacerDoubleIntegratorKinematic::control_array::Zero();
  RacerDoubleIntegratorKinematic::output_array output = RacerDoubleIntegratorKinematic::output_array::Zero();
  dynamics.initializeDynamics(x, u, output, 0, 0);

  // force brake output
  auto brake_params = dynamics.getDelayHelper()->getOutputModel()->getParams();
  std::vector<float> brake_theta(RacerDoubleIntegratorKinematic::DELAY_LSTM::OUTPUT_PARAMS_T::NUM_PARAMS);
  brake_theta[brake_params.stride_idcs[3]] = 1.0;
  dynamics.getDelayHelper()->getOutputModel()->updateModel({ 8, 10, 1 }, brake_theta);

  auto steer_params = dynamics.getSteerHelper()->getOutputModel()->getParams();
  std::vector<float> steer_theta(RacerDoubleIntegratorKinematic::STEER_LSTM::OUTPUT_PARAMS_T::NUM_PARAMS);
  steer_theta[steer_params.stride_idcs[3]] = 2.0;
  dynamics.getSteerHelper()->getOutputModel()->updateModel({ 10, 5, 1 }, steer_theta);

  auto terra_params = dynamics.getTerraHelper()->getOutputModel()->getParams();
  std::vector<float> terra_theta(RacerDoubleIntegratorKinematic::TERRA_LSTM::OUTPUT_PARAMS_T::NUM_PARAMS);
  terra_theta[terra_params.stride_idcs[3]] = 4.0;
  terra_theta[terra_params.stride_idcs[3] + 1] = 10.0;
  terra_theta[terra_params.stride_idcs[3] + 2] = 6.0;
  terra_theta[terra_params.stride_idcs[3] + 3] = 0.07;
  dynamics.getTerraHelper()->getOutputModel()->updateModel({ 20, 20, 4 }, terra_theta);

  float delta = 0;
  // computeDynamics should not touch the roll/pitch element
  RacerDoubleIntegratorKinematic::state_array state_der = RacerDoubleIntegratorKinematic::state_array::Ones() * 0.153;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                                      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                                                     // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  u << 1, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                                      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                                                     // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  u << -1, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                                      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.33);                                                    // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  // change the initial brake state
  x << 0, 0, 0, 0, 1.0, 0, 0, 0, 0, 0, 0;
  u << 1, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                                      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), -0.9 + 1);                                                // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, 0, 1.0, 0, 0, 0, 0, 0, 0;
  u << -0.9, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                                      // steer angle
  EXPECT_NEAR(state_der(4), -0.66 + 1, tol);                                              // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, 0, 1.0, 0, 0, 0, 0, 0, 0;
  u << -0.9, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                                      // steer angle
  EXPECT_NEAR(state_der(4), -0.66 + 1, tol);                                              // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << -0.01, 0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                                      // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 0.066 + 1);                                               // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, 1.0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 3.0 + 20);                                                // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                                                     // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, -1.0;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), -3 + 20);                                                 // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                                                     // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, 0.5;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), 1.5 + 20);                                                // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                                                     // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0;
  u << 0, -0.1;
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                                       // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                                       // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                                       // yaw
  EXPECT_FLOAT_EQ(state_der(3), -0.3 + 20);                                               // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                                                     // brake state
  EXPECT_FLOAT_EQ(state_der(5), cosf(0.0f + 0.07f) * 40 - 50 * sinf(0.0f + 0.07f) + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), sinf(0.0f + 0.07f) * 40 + 50 * cosf(0.0f + 0.07f) + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                                                   // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                                       // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                                       // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                                      // steer angle rate

  x << 0, 0, 0, -2.5, 0, 0, 0, 0, 0, 0, 0;
  u << 0, -0.1;
  delta = tanf(x(3) / -9.2);
  float c_delta = cosf(delta + 0.07);
  float s_delta = sinf(delta + 0.07);
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                 // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                 // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                 // yaw
  EXPECT_FLOAT_EQ(state_der(3), 1.2 + 20);                          // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                               // brake state
  EXPECT_FLOAT_EQ(state_der(5), 40 * c_delta - 50 * s_delta + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), 40 * s_delta + 50 * c_delta + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                             // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                 // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                 // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                // steer angle rate

  x << 0, 0, 0, 2.5, 0, 0, 0, 0, 0, 0, 0;
  u << 0, 0.5;
  delta = tanf(x(3) / -9.2);
  c_delta = cosf(delta + 0.07);
  s_delta = sinf(delta + 0.07);
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 0);                                 // x
  EXPECT_FLOAT_EQ(state_der(1), 0);                                 // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                 // yaw
  EXPECT_FLOAT_EQ(state_der(3), 20);                                // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                               // brake state
  EXPECT_FLOAT_EQ(state_der(5), 40 * c_delta - 50 * s_delta + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), 40 * s_delta + 50 * c_delta + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                             // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                 // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                 // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                // steer angle rate

  x << 0, 0, 0, 0, 0, 1.0, 2.0, 0, 0, 0, 0;
  u << 0, 0.5;
  delta = tanf(x(3) / -9.2);
  c_delta = cosf(delta + 0.07);
  s_delta = sinf(delta + 0.07);
  dynamics.computeDynamics(x, u, state_der);
  EXPECT_FLOAT_EQ(state_der(0), 1.0);                               // x
  EXPECT_FLOAT_EQ(state_der(1), 2.0);                               // y
  EXPECT_FLOAT_EQ(state_der(2), 0);                                 // yaw
  EXPECT_FLOAT_EQ(state_der(3), 21.5);                              // steer angle
  EXPECT_FLOAT_EQ(state_der(4), 1.0);                               // brake state
  EXPECT_FLOAT_EQ(state_der(5), 40 * c_delta - 50 * s_delta + 40);  // vel x
  EXPECT_FLOAT_EQ(state_der(6), 40 * s_delta + 50 * c_delta + 50);  // vel y
  EXPECT_FLOAT_EQ(state_der(7), 30.0f);                             // omega z
  EXPECT_FLOAT_EQ(state_der(8), 0);                                 // roll
  EXPECT_FLOAT_EQ(state_der(9), 0);                                 // pitch
  EXPECT_FLOAT_EQ(state_der(10), 0);                                // steer angle rate
}

TEST_F(RacerDoubleIntegratorKinematicTest, updateState)
{
  auto dynamics = RacerDoubleIntegratorKinematic();

  auto params = dynamics.getParams();
  params.max_steer_angle = 5.0;
  dynamics.setParams(params);

  auto limits = dynamics.getControlRanges();
  limits[0].x = -1.0;
  dynamics.setControlRanges(limits);

  RacerDoubleIntegratorKinematic::state_array s = RacerDoubleIntegratorKinematic::state_array::Zero();
  RacerDoubleIntegratorKinematic::state_array s_next = RacerDoubleIntegratorKinematic::state_array::Zero();
  RacerDoubleIntegratorKinematic::state_array s_der = RacerDoubleIntegratorKinematic::state_array::Zero();

  s << 1, 2, 3, 4, 0.55, 6, 7, 8, 9, 10, 11;
  dynamics.updateState(s, s_next, s_der, 0.1);
  EXPECT_FLOAT_EQ(s_next(0), 1);     // x
  EXPECT_FLOAT_EQ(s_next(1), 2);     // y
  EXPECT_FLOAT_EQ(s_next(2), 3);     // yaw
  EXPECT_FLOAT_EQ(s_next(3), 4);     // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 0.55);  // brake state
  EXPECT_FLOAT_EQ(s_next(5), 6);     // vel x
  EXPECT_FLOAT_EQ(s_next(6), 7);     // vel y
  EXPECT_FLOAT_EQ(s_next(7), 8);     // omega z
  EXPECT_FLOAT_EQ(s_next(8), 9);     // roll
  EXPECT_FLOAT_EQ(s_next(9), 10);    // pitch
  EXPECT_FLOAT_EQ(s_next(10), 0);    // steer angle rate

  s = RacerDoubleIntegratorKinematic::state_array::Ones() * 10;
  dynamics.updateState(s, s_next, s_der, 0.1);
  EXPECT_FLOAT_EQ(s_next(0), 10);                                 // x
  EXPECT_FLOAT_EQ(s_next(1), 10);                                 // y
  EXPECT_FLOAT_EQ(s_next(2), angle_utils::normalizeAngle(s(2)));  // yaw
  EXPECT_FLOAT_EQ(s_next(3), 5);                                  // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 1.0);                                // brake state
  EXPECT_FLOAT_EQ(s_next(5), 10);                                 // vel x
  EXPECT_FLOAT_EQ(s_next(6), 10);                                 // vel y
  EXPECT_FLOAT_EQ(s_next(7), 10);                                 // omega z
  EXPECT_FLOAT_EQ(s_next(8), 10);                                 // roll
  EXPECT_FLOAT_EQ(s_next(9), 10);                                 // pitch
  EXPECT_FLOAT_EQ(s_next(10), 0);                                 // steer angle rate

  s = RacerDoubleIntegratorKinematic::state_array::Ones() * -10;
  dynamics.updateState(s, s_next, s_der, 0.1);
  EXPECT_FLOAT_EQ(s_next(0), -10);                                // x
  EXPECT_FLOAT_EQ(s_next(1), -10);                                // y
  EXPECT_FLOAT_EQ(s_next(2), angle_utils::normalizeAngle(s(2)));  // yaw
  EXPECT_FLOAT_EQ(s_next(3), -5);                                 // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 0.0);                                // brake state
  EXPECT_FLOAT_EQ(s_next(5), -10);                                // vel x
  EXPECT_FLOAT_EQ(s_next(6), -10);                                // vel y
  EXPECT_FLOAT_EQ(s_next(7), -10);                                // omega z
  EXPECT_FLOAT_EQ(s_next(8), -10);                                // roll
  EXPECT_FLOAT_EQ(s_next(9), -10);                                // pitch
  EXPECT_FLOAT_EQ(s_next(10), 0);                                 // steer angle rate

  s = RacerDoubleIntegratorKinematic::state_array::Zero();
  s_der = RacerDoubleIntegratorKinematic::state_array::Ones();
  dynamics.updateState(s, s_next, s_der, 0.1);
  EXPECT_FLOAT_EQ(s_next(0), 0.1);   // x
  EXPECT_FLOAT_EQ(s_next(1), 0.1);   // y
  EXPECT_NEAR(s_next(2), 0.1, tol);  // yaw
  EXPECT_FLOAT_EQ(s_next(3), 0.1);   // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 0.1);   // brake state
  EXPECT_FLOAT_EQ(s_next(5), 0.1);   // vel x
  EXPECT_FLOAT_EQ(s_next(6), 0.1);   // vel y
  EXPECT_FLOAT_EQ(s_next(7), 0.1);   // omega z
  EXPECT_FLOAT_EQ(s_next(8), 0.1);   // roll
  EXPECT_FLOAT_EQ(s_next(9), 0.1);   // pitch
  EXPECT_FLOAT_EQ(s_next(10), 1.0);  // steer angle rate

  s = RacerDoubleIntegratorKinematic::state_array::Zero();
  s_der = RacerDoubleIntegratorKinematic::state_array::Ones() * -1;
  dynamics.updateState(s, s_next, s_der, 0.1);
  EXPECT_FLOAT_EQ(s_next(0), -0.1);   // x
  EXPECT_FLOAT_EQ(s_next(1), -0.1);   // y
  EXPECT_NEAR(s_next(2), -0.1, tol);  // yaw
  EXPECT_FLOAT_EQ(s_next(3), -0.1);   // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 0.0);    // brake state
  EXPECT_FLOAT_EQ(s_next(5), -0.1);   // vel x
  EXPECT_FLOAT_EQ(s_next(6), -0.1);   // vel y
  EXPECT_FLOAT_EQ(s_next(7), -0.1);   // omega z
  EXPECT_FLOAT_EQ(s_next(8), -0.1);   // roll
  EXPECT_FLOAT_EQ(s_next(9), -0.1);   // pitch
  EXPECT_FLOAT_EQ(s_next(10), -1.0);  // steer angle rate

  s = RacerDoubleIntegratorKinematic::state_array::Zero();
  s_der = RacerDoubleIntegratorKinematic::state_array::Ones() * 10;
  dynamics.updateState(s, s_next, s_der, 1);
  EXPECT_FLOAT_EQ(s_next(0), 10);                                   // x
  EXPECT_FLOAT_EQ(s_next(1), 10);                                   // y
  EXPECT_NEAR(s_next(2), angle_utils::normalizeAngle(10.0f), tol);  // yaw
  EXPECT_FLOAT_EQ(s_next(3), 5);                                    // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 1);                                    // brake state
  EXPECT_FLOAT_EQ(s_next(5), 10);                                   // vel x
  EXPECT_FLOAT_EQ(s_next(6), 10);                                   // vel y
  EXPECT_FLOAT_EQ(s_next(7), 10);                                   // omega z
  EXPECT_FLOAT_EQ(s_next(8), 10);                                   // roll
  EXPECT_FLOAT_EQ(s_next(9), 10);                                   // pitch
  EXPECT_FLOAT_EQ(s_next(10), 10.0);                                // steer angle rate

  s = RacerDoubleIntegratorKinematic::state_array::Zero();
  s_der = RacerDoubleIntegratorKinematic::state_array::Ones() * -10;
  dynamics.updateState(s, s_next, s_der, 1.0);
  EXPECT_FLOAT_EQ(s_next(0), -10.0);                                 // x
  EXPECT_FLOAT_EQ(s_next(1), -10.0);                                 // y
  EXPECT_NEAR(s_next(2), angle_utils::normalizeAngle(-10.0f), tol);  // yaw
  EXPECT_FLOAT_EQ(s_next(3), -5.0);                                  // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 0);                                     // brake state
  EXPECT_FLOAT_EQ(s_next(5), -10.0);                                 // vel x
  EXPECT_FLOAT_EQ(s_next(6), -10.0);                                 // vel y
  EXPECT_FLOAT_EQ(s_next(7), -10.0);                                 // omega z
  EXPECT_FLOAT_EQ(s_next(8), -10);                                   // roll
  EXPECT_FLOAT_EQ(s_next(9), -10);                                   // pitch
  EXPECT_FLOAT_EQ(s_next(10), -10.0);                                // steer angle rate
}

TEST_F(RacerDoubleIntegratorKinematicTest, stepCPU)
{
  auto dynamics = RacerDoubleIntegratorKinematic();

  auto params = dynamics.getParams();
  params.max_steer_angle = 5.0;
  params.wheel_base = 2.981;
  dynamics.setParams(params);

  auto limits = dynamics.getControlRanges();
  limits[0].x = -1.0;
  dynamics.setControlRanges(limits);

  hipExtent extent = make_hipExtent(10, 20, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(10 * 20);
  for (int i = 1; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 1.0f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(0, 0, 0));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  // force brake output
  auto brake_params = dynamics.getDelayHelper()->getOutputModel()->getParams();
  std::vector<float> brake_theta(RacerDoubleIntegratorKinematic::DELAY_LSTM::OUTPUT_PARAMS_T::NUM_PARAMS);
  brake_theta[brake_params.stride_idcs[3]] = 1.0;
  dynamics.getDelayHelper()->getOutputModel()->updateModel({ 8, 10, 1 }, brake_theta);

  auto steer_params = dynamics.getSteerHelper()->getOutputModel()->getParams();
  std::vector<float> steer_theta(RacerDoubleIntegratorKinematic::STEER_LSTM::OUTPUT_PARAMS_T::NUM_PARAMS);
  steer_theta[steer_params.stride_idcs[3]] = 2.0;
  dynamics.getSteerHelper()->getOutputModel()->updateModel({ 10, 5, 1 }, steer_theta);

  auto terra_params = dynamics.getTerraHelper()->getOutputModel()->getParams();
  std::vector<float> terra_theta(RacerDoubleIntegratorKinematic::TERRA_LSTM::OUTPUT_PARAMS_T::NUM_PARAMS);
  terra_theta[terra_params.stride_idcs[3]] = 4.0;
  terra_theta[terra_params.stride_idcs[3] + 1] = 10.0;
  terra_theta[terra_params.stride_idcs[3] + 2] = 6.0;
  dynamics.getTerraHelper()->getOutputModel()->updateModel({ 20, 20, 4 }, terra_theta);

  RacerDoubleIntegratorKinematic::state_array s = RacerDoubleIntegratorKinematic::state_array::Ones();
  RacerDoubleIntegratorKinematic::control_array u = RacerDoubleIntegratorKinematic::control_array::Ones();
  RacerDoubleIntegratorKinematic::state_array s_next = RacerDoubleIntegratorKinematic::state_array::Zero();
  RacerDoubleIntegratorKinematic::state_array s_der = RacerDoubleIntegratorKinematic::state_array::Ones();
  RacerDoubleIntegratorKinematic::output_array output = RacerDoubleIntegratorKinematic::output_array::Zero();
  s(0) = 5;
  s(1) = 5;

  dynamics.step(s, s_next, s_der, u, output, 0, 0.1);
  EXPECT_FLOAT_EQ(s_der(0), -0.30116868);  // x
  EXPECT_FLOAT_EQ(s_der(1), 1.3817732);    // y
  EXPECT_FLOAT_EQ(s_der(2), 1.0);          // yaw
  EXPECT_FLOAT_EQ(s_der(3), 22.4);         // steer angle
  EXPECT_FLOAT_EQ(s_der(4), 0.1);          // brake state
  EXPECT_FLOAT_EQ(s_der(5), 85.207535);    // vel x
  EXPECT_FLOAT_EQ(s_der(6), 95.346207);    // vel y
  EXPECT_FLOAT_EQ(s_der(7), 30);           // omega z
  EXPECT_FLOAT_EQ(s_der(8), 0);            // roll
  EXPECT_FLOAT_EQ(s_der(9), 0);            // pitch
  EXPECT_FLOAT_EQ(s_der(10), 0);           // steer angle rate

  EXPECT_FLOAT_EQ(s_next(0), 4.96988314);   // x
  EXPECT_FLOAT_EQ(s_next(1), 5.1381773);    // y
  EXPECT_FLOAT_EQ(s_next(2), 1.1f);         // yaw
  EXPECT_FLOAT_EQ(s_next(3), 3.24);         // steer angle
  EXPECT_FLOAT_EQ(s_next(4), 1.0);          // brake state
  EXPECT_FLOAT_EQ(s_next(5), 9.5207539);    // vel x
  EXPECT_FLOAT_EQ(s_next(6), 10.534621);    // vel y
  EXPECT_FLOAT_EQ(s_next(7), 4);            // omega z
  EXPECT_FLOAT_EQ(s_next(8), -0.7060864);   // roll
  EXPECT_FLOAT_EQ(s_next(9), -0.44172257);  // pitch
  EXPECT_FLOAT_EQ(s_next(10), 22.4);        // steer angle rate

  EXPECT_FLOAT_EQ(output(0), 9.5207539);    // x vel
  EXPECT_FLOAT_EQ(output(1), 10.534621);    // y vel
  EXPECT_FLOAT_EQ(output(2), 0);            // z vel
  EXPECT_FLOAT_EQ(output(3), 4.96988314);   // x pos
  EXPECT_FLOAT_EQ(output(4), 5.1381773);    // y pos
  EXPECT_FLOAT_EQ(output(5), 0);            // z pos
  EXPECT_FLOAT_EQ(output(6), 1.1000001);    // yaw
  EXPECT_FLOAT_EQ(output(7), -0.7060864);   // roll
  EXPECT_FLOAT_EQ(output(8), -0.44172257);  // pitch
  EXPECT_FLOAT_EQ(output(9), 3.24);         // steer angle
  EXPECT_FLOAT_EQ(output(10), 22.4);        // steer angle rate
  // EXPECT_FLOAT_EQ(output(11), 5.6652329);   // fl wheel x
  // EXPECT_FLOAT_EQ(output(12), 8.1291666);   // fl wheel y
  // EXPECT_FLOAT_EQ(output(13), 6.9788725);   // fr wheel x
  // EXPECT_FLOAT_EQ(output(14), 7.460566);    // fr wheel y
  // EXPECT_FLOAT_EQ(output(15), 4.31306332);  // bl wheel x
  // EXPECT_FLOAT_EQ(output(16), 5.4724776);   // bl wheel y
  // EXPECT_FLOAT_EQ(output(17), 5.626703);    // br wheel x
  // EXPECT_FLOAT_EQ(output(18), 4.803877);    // br wheel y
  EXPECT_FLOAT_EQ(output(19), 10000);      // wheel f fl
  EXPECT_FLOAT_EQ(output(20), 10000);      // wheel f fr
  EXPECT_FLOAT_EQ(output(21), 10000);      // wheel f bl
  EXPECT_FLOAT_EQ(output(22), 10000);      // wheel f br
  EXPECT_FLOAT_EQ(output(23), 85.207535);  // accel x
  EXPECT_FLOAT_EQ(output(24), 95.346207);  // accel y
  EXPECT_FLOAT_EQ(output(25), 1);          // omega z
}

TEST_F(RacerDoubleIntegratorKinematicTest, TestPythonComparison)
{
  const int num_points = 1;
  const float dt = 0.02f;
  const int T = 250;
  const int init_T = 51;
  const int state_dim = 12;
  const int output_dim = 5;
  CudaCheckError();
  using DYN = RacerDoubleIntegratorKinematic;
  RacerDoubleIntegratorKinematic dynamics = RacerDoubleIntegratorKinematic(mppi::tests::bicycle_test);

  auto limits = dynamics.getControlRanges();
  limits[0].x = -1.0;
  dynamics.setControlRanges(limits);

  auto params = dynamics.getParams();
  params.max_steer_angle = 5.0;
  params.wheel_base = 2.981;
  dynamics.setParams(params);

  cnpy::npz_t input_outputs = cnpy::npz_load(mppi::tests::bicycle_test);
  double* inputs = input_outputs.at("input").data<double>();
  double* outputs = input_outputs.at("output").data<double>();
  double* init_inputs = input_outputs.at("init_input").data<double>();
  double* delay_init_hidden = input_outputs.at("delay_init_hidden").data<double>();
  double* delay_init_cell = input_outputs.at("delay_init_cell").data<double>();
  double* steer_init_hidden = input_outputs.at("steer_init_hidden").data<double>();
  double* steer_init_cell = input_outputs.at("steer_init_cell").data<double>();
  double* terra_init_hidden = input_outputs.at("terra_init_hidden").data<double>();
  double* terra_init_cell = input_outputs.at("terra_init_cell").data<double>();

  // steering model params
  EXPECT_FLOAT_EQ(dynamics.getParams().max_steer_rate, 4.04);
  EXPECT_FLOAT_EQ(dynamics.getParams().steering_constant, 2.03);

  // delay model params
  EXPECT_FLOAT_EQ(dynamics.getParams().brake_delay_constant, 6.6);
  EXPECT_FLOAT_EQ(dynamics.getParams().max_brake_rate_neg, 0.9);
  EXPECT_FLOAT_EQ(dynamics.getParams().max_brake_rate_pos, 0.33);

  // rest params
  EXPECT_FLOAT_EQ(dynamics.getParams().gravity, -9.81);
  EXPECT_FLOAT_EQ(dynamics.getParams().wheel_angle_scale, -9.2);

  std::map<std::string, Eigen::VectorXf> buffer;
  buffer["VEL_X"] = Eigen::VectorXf::Random(51);
  buffer["VEL_Y"] = Eigen::VectorXf::Random(51);
  buffer["STEER_ANGLE"] = Eigen::VectorXf::Random(51);
  buffer["STEER_ANGLE_RATE"] = Eigen::VectorXf::Random(51);
  buffer["STEER_CMD"] = Eigen::VectorXf::Random(51);
  buffer["BRAKE_STATE"] = Eigen::VectorXf::Random(51);
  buffer["BRAKE_CMD"] = Eigen::VectorXf::Random(51);
  buffer["THROTTLE_CMD"] = Eigen::VectorXf::Random(51);
  buffer["OMEGA_Z"] = Eigen::VectorXf::Random(51);
  buffer["ROLL"] = Eigen::VectorXf::Random(51);
  buffer["PITCH"] = Eigen::VectorXf::Random(51);

  RacerDoubleIntegratorKinematic::state_array state;
  RacerDoubleIntegratorKinematic::state_array next_state_cpu;
  RacerDoubleIntegratorKinematic::control_array control;
  RacerDoubleIntegratorKinematic::output_array output;
  RacerDoubleIntegratorKinematic::state_array state_der = RacerDoubleIntegratorKinematic::state_array::Zero();

  for (int point = 0; point < num_points; point++)
  {
    for (int t = 0; t < init_T; t++)
    {
      buffer["VEL_X"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 0];
      buffer["VEL_Y"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 1];
      buffer["OMEGA_Z"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 2];
      buffer["THROTTLE_CMD"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 3];
      buffer["BRAKE_STATE"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 4];
      buffer["STEER_ANGLE"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 5];
      buffer["STEER_ANGLE_RATE"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 6];
      buffer["PITCH"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 7];
      buffer["ROLL"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 8];
      buffer["BRAKE_CMD"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 9];
      buffer["STEER_CMD"](t) = init_inputs[point * init_T * state_dim + t * state_dim + 10];
    }
    dynamics.updateFromBuffer(buffer);

    for (int i = 0; i < 5; i++)
    {
      EXPECT_NEAR(dynamics.getDelayHelper()->getLSTMModel()->getHiddenState()(i), delay_init_hidden[5 * point + i], tol)
          << "at point " << point << " index " << i;
      EXPECT_NEAR(dynamics.getDelayHelper()->getLSTMModel()->getCellState()(i), delay_init_cell[5 * point + i], tol)
          << "at point " << point << " index " << i;
    }
    for (int i = 0; i < 5; i++)
    {
      EXPECT_NEAR(dynamics.getSteerHelper()->getLSTMModel()->getHiddenState()(i), steer_init_hidden[5 * point + i], tol)
          << "at point " << point << " index " << i;
      EXPECT_NEAR(dynamics.getSteerHelper()->getLSTMModel()->getCellState()(i), steer_init_cell[5 * point + i], tol)
          << "at point " << point << " index " << i;
    }
    for (int i = 0; i < 10; i++)
    {
      EXPECT_NEAR(dynamics.getTerraHelper()->getLSTMModel()->getHiddenState()(i), terra_init_hidden[10 * point + i], tol)
          << "at point " << point << " index " << i;
      EXPECT_NEAR(dynamics.getTerraHelper()->getLSTMModel()->getCellState()(i), terra_init_cell[10 * point + i], tol)
          << "at point " << point << " index " << i;
    }

    RacerDoubleIntegratorKinematic::state_array state;
    for (int t = 0; t < T; t++)
    {
      state = RacerDoubleIntegratorKinematic::state_array::Zero();
      state_der = RacerDoubleIntegratorKinematic::state_array::Zero();
      state(3) = inputs[point * T * state_dim + t * state_dim + 5];   // STEER_ANGLE
      state(4) = inputs[point * T * state_dim + t * state_dim + 4];   // BRAKE_STATE
      state(5) = inputs[point * T * state_dim + t * state_dim + 0];   // VX
      state(6) = inputs[point * T * state_dim + t * state_dim + 1];   // VY
      state(7) = inputs[point * T * state_dim + t * state_dim + 2];   // OMEGA_Z
      state(8) = inputs[point * T * state_dim + t * state_dim + 8];   // ROLL
      state(9) = inputs[point * T * state_dim + t * state_dim + 7];   // PITCH
      state(10) = inputs[point * T * state_dim + t * state_dim + 6];  // STEER_ANGLE_RATE
      control(0) = inputs[point * T * state_dim + t * state_dim + 3] -
                   inputs[point * T * state_dim + t * state_dim + 9];   // THROTTLE/BRAKE
      control(1) = inputs[point * T * state_dim + t * state_dim + 10];  // STEER_CMD

      dynamics.step(state, next_state_cpu, state_der, control, output, 0, dt);

      EXPECT_NEAR(state_der[5], outputs[point * T * output_dim + t * output_dim + 0], tol)
          << "point " << point << " at dim ACCEL_X at time " << t;
      EXPECT_NEAR(state_der[6], outputs[point * T * output_dim + t * output_dim + 1], tol)
          << "point " << point << " at dim ACCEL_Y"
          << " at time " << t;
      EXPECT_NEAR(state_der[7], outputs[point * T * output_dim + t * output_dim + 2], tol)
          << "point " << point << " at dim OMEGA_Z"
          << " at time " << t;
      EXPECT_NEAR(state_der[4], outputs[point * T * output_dim + t * output_dim + 3], tol)
          << "point " << point << " at dim BRAKE_STATE"
          << " at time " << t;
      EXPECT_NEAR(state_der[3], outputs[point * T * output_dim + t * output_dim + 4], tol)
          << "point " << point << " at dim STEER_ANGLE"
          << " at time " << t;
      // for (int i = 0; i < 25; i++)
      // {
      //   EXPECT_NEAR(dynamics.getLSTMModel()->getHiddenState()[i], hidden[point * T * 25 + 25 * t + i], tol)
      //                 << "point " << point << " at dim " << i;
      //   EXPECT_NEAR(dynamics.getLSTMModel()->getCellState()[i], cell[point * T * 25 + 25 * t + i], tol)
      //                 << "point " << point << " at dim " << i;
      // }
    }
  }
}

TEST_F(RacerDoubleIntegratorKinematicTest, TestStepGPUvsCPU)
{
  const int num_rollouts = 2000;
  const float dt = 0.1f;
  CudaCheckError();
  using DYN = RacerDoubleIntegratorKinematic;
  RacerDoubleIntegratorKinematic dynamics = RacerDoubleIntegratorKinematic(mppi::tests::bicycle_test);

  auto params = dynamics.getParams();
  params.max_steer_angle = 5.0;
  params.wheel_base = 2.981;
  dynamics.setParams(params);
  // steering model params
  EXPECT_FLOAT_EQ(dynamics.getParams().max_steer_rate, 4.04);
  EXPECT_FLOAT_EQ(dynamics.getParams().steering_constant, 2.03);

  // delay model params
  EXPECT_FLOAT_EQ(dynamics.getParams().brake_delay_constant, 6.6);
  EXPECT_FLOAT_EQ(dynamics.getParams().max_brake_rate_neg, 0.9);
  EXPECT_FLOAT_EQ(dynamics.getParams().max_brake_rate_pos, 0.33);

  // rest params
  EXPECT_FLOAT_EQ(dynamics.getParams().gravity, -9.81);
  EXPECT_FLOAT_EQ(dynamics.getParams().wheel_angle_scale, -9.2);

  // ensure that the network values are not nan
  auto terra_init_params = dynamics.getTerraHelper()->getInitLSTMParams();
  for (int i = 0; i < LSTMParams<10, 200>::HIDDEN_HIDDEN_SIZE; i++)
  {
    EXPECT_TRUE(isfinite(terra_init_params.W_im[i]));
    EXPECT_TRUE(isfinite(terra_init_params.W_fm[i]));
    EXPECT_TRUE(isfinite(terra_init_params.W_om[i]));
    EXPECT_TRUE(isfinite(terra_init_params.W_cm[i]));
  }
  for (int i = 0; i < LSTMParams<10, 200>::INPUT_HIDDEN_SIZE; i++)
  {
    EXPECT_TRUE(isfinite(terra_init_params.W_ii[i]));
    EXPECT_TRUE(isfinite(terra_init_params.W_fi[i]));
    EXPECT_TRUE(isfinite(terra_init_params.W_oi[i]));
    EXPECT_TRUE(isfinite(terra_init_params.W_ci[i]));
  }
  for (int i = 0; i < LSTMParams<10, 200>::HIDDEN_DIM; i++)
  {
    EXPECT_TRUE(isfinite(terra_init_params.b_i[i]));
    EXPECT_TRUE(isfinite(terra_init_params.b_f[i]));
    EXPECT_TRUE(isfinite(terra_init_params.b_o[i]));
    EXPECT_TRUE(isfinite(terra_init_params.b_c[i]));
  }

  auto terra_params = dynamics.getTerraHelper()->getLSTMParams();
  for (int i = 0; i < LSTMParams<10, 200>::HIDDEN_HIDDEN_SIZE; i++)
  {
    EXPECT_TRUE(isfinite(terra_params.W_im[i]));
    EXPECT_TRUE(isfinite(terra_params.W_fm[i]));
    EXPECT_TRUE(isfinite(terra_params.W_om[i]));
    EXPECT_TRUE(isfinite(terra_params.W_cm[i]));
  }
  for (int i = 0; i < LSTMParams<10, 200>::INPUT_HIDDEN_SIZE; i++)
  {
    EXPECT_TRUE(isfinite(terra_params.W_ii[i]));
    EXPECT_TRUE(isfinite(terra_params.W_fi[i]));
    EXPECT_TRUE(isfinite(terra_params.W_oi[i]));
    EXPECT_TRUE(isfinite(terra_params.W_ci[i]));
  }
  for (int i = 0; i < LSTMParams<10, 200>::HIDDEN_DIM; i++)
  {
    EXPECT_TRUE(isfinite(terra_params.b_i[i]));
    EXPECT_TRUE(isfinite(terra_params.b_f[i]));
    EXPECT_TRUE(isfinite(terra_params.b_o[i]));
    EXPECT_TRUE(isfinite(terra_params.b_c[i]));
  }

  hipExtent extent = make_hipExtent(100, 200, 0);
  TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
  helper->setExtent(0, extent);

  std::vector<float> data_vec;
  data_vec.resize(100 * 200);
  for (int i = 0; i < data_vec.size(); i++)
  {
    data_vec[i] = i * 0.1f;
  }

  std::array<float3, 3> new_rot_mat{};
  new_rot_mat[0] = make_float3(0, 1, 0);
  new_rot_mat[1] = make_float3(1, 0, 0);
  new_rot_mat[2] = make_float3(0, 0, 1);
  helper->updateRotation(0, new_rot_mat);
  helper->updateOrigin(0, make_float3(0, 0, 0));

  helper->updateTexture(0, data_vec);
  helper->updateResolution(0, 10);
  helper->enableTexture(0);
  helper->copyToDevice(true);

  CudaCheckError();
  dynamics.GPUSetup();
  CudaCheckError();

  EXPECT_NE(dynamics.getSteerHelper()->getLSTMDevicePtr(), nullptr);
  EXPECT_NE(dynamics.steer_network_d_, nullptr);
  EXPECT_EQ(dynamics.steer_network_d_, dynamics.getSteerHelper()->getLSTMDevicePtr());

  EXPECT_NE(dynamics.getDelayHelper()->getLSTMDevicePtr(), nullptr);
  EXPECT_NE(dynamics.delay_network_d_, nullptr);
  EXPECT_EQ(dynamics.delay_network_d_, dynamics.getDelayHelper()->getLSTMDevicePtr());

  EXPECT_NE(dynamics.getTerraHelper()->getLSTMDevicePtr(), nullptr);
  EXPECT_NE(dynamics.terra_network_d_, nullptr);
  EXPECT_EQ(dynamics.terra_network_d_, dynamics.getTerraHelper()->getLSTMDevicePtr());

  Eigen::Matrix<float, RacerDoubleIntegratorKinematic::CONTROL_DIM, num_rollouts> control_trajectory;
  control_trajectory = Eigen::Matrix<float, RacerDoubleIntegratorKinematic::CONTROL_DIM, num_rollouts>::Random();
  Eigen::Matrix<float, RacerDoubleIntegratorKinematic::STATE_DIM, num_rollouts> state_trajectory;
  state_trajectory = Eigen::Matrix<float, RacerDoubleIntegratorKinematic::STATE_DIM, num_rollouts>::Random();

  std::vector<std::array<float, RacerDoubleIntegratorKinematic::STATE_DIM>> s(num_rollouts);
  std::vector<std::array<float, RacerDoubleIntegratorKinematic::STATE_DIM>> s_next(num_rollouts);
  std::vector<std::array<float, RacerDoubleIntegratorKinematic::STATE_DIM>> s_der(num_rollouts);
  // steering, throttle
  std::vector<std::array<float, RacerDoubleIntegratorKinematic::CONTROL_DIM>> u(num_rollouts);

  RacerDoubleIntegratorKinematic::state_array state;
  RacerDoubleIntegratorKinematic::state_array next_state_cpu;
  RacerDoubleIntegratorKinematic::control_array control;
  RacerDoubleIntegratorKinematic::output_array output;
  RacerDoubleIntegratorKinematic::state_array state_der_cpu = RacerDoubleIntegratorKinematic::state_array::Zero();

  // Run dynamics on dynamicsU
  // Run dynamics on GPU
  for (int y_dim = 1; y_dim <= 16; y_dim++)
  {
    DYN::buffer_trajectory buffer;
    buffer["VEL_X"] = Eigen::VectorXf::Random(51);
    buffer["VEL_Y"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_ANGLE_RATE"] = Eigen::VectorXf::Random(51);
    buffer["STEER_CMD"] = Eigen::VectorXf::Random(51);
    buffer["BRAKE_STATE"] = Eigen::VectorXf::Random(51);
    buffer["BRAKE_CMD"] = Eigen::VectorXf::Random(51);
    buffer["THROTTLE_CMD"] = Eigen::VectorXf::Random(51);
    buffer["OMEGA_Z"] = Eigen::VectorXf::Random(51);
    buffer["ROLL"] = Eigen::VectorXf::Random(51);
    buffer["PITCH"] = Eigen::VectorXf::Random(51);

    for (int state_index = 0; state_index < num_rollouts; state_index++)
    {
      for (int dim = 0; dim < s[0].size(); dim++)
      {
        s[state_index][dim] = state_trajectory.col(state_index)(dim);
      }
      for (int dim = 0; dim < u[0].size(); dim++)
      {
        u[state_index][dim] = control_trajectory.col(state_index)(dim);
      }
    }
    dynamics.updateFromBuffer(buffer);
    launchStepTestKernel<RacerDoubleIntegratorKinematic, 16>(dynamics, s, u, s_der, s_next, 0, dt, y_dim);
    for (int point = 0; point < num_rollouts; point++)
    {
      dynamics.initializeDynamics(state, control, output, 0, 0);
      state = state_trajectory.col(point);
      control = control_trajectory.col(point);
      state_der_cpu = RacerDoubleIntegratorKinematic::state_array::Zero();

      dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
      for (int dim = 0; dim < RacerDoubleIntegratorKinematic::STATE_DIM; dim++)
      {
        EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        EXPECT_NEAR(next_state_cpu(dim), s_next[point][dim], 1e-4)
            << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
        EXPECT_TRUE(isfinite(s_next[point][dim]));
      }
    }
  }
  dynamics.freeCudaMem();
}

// TEST_F(RacerDoubleIntegratorKinematicTest, TestStepGPUvsCPUReverse)
// {
//   using DYN = RacerDoubleIntegratorKinematic;
//
//   const int num_rollouts = 2000;
//   const float dt = 0.1f;
//   CudaCheckError();
//   RacerDoubleIntegratorKinematic dynamics = RacerDoubleIntegratorKinematic(mppi::tests::steering_lstm);
//   auto params = dynamics.getParams();
//   params.gear_sign = -1;
//   dynamics.setParams(params);
//   EXPECT_FLOAT_EQ(dynamics.getParams().max_steer_rate, 3.9760568141937256);
//   EXPECT_FLOAT_EQ(dynamics.getParams().steering_constant, 2.1222121715545654);
//
//   hipExtent extent = make_hipExtent(10, 20, 0);
//   TwoDTextureHelper<float>* helper = dynamics.getTextureHelper();
//   helper->setExtent(0, extent);
//
//   std::vector<float> data_vec;
//   data_vec.resize(10 * 20);
//   for (int i = 0; i < data_vec.size(); i++)
//   {
//     data_vec[i] = i * 1.0f;
//   }
//
//   std::array<float3, 3> new_rot_mat{};
//   new_rot_mat[0] = make_float3(0, 1, 0);
//   new_rot_mat[1] = make_float3(1, 0, 0);
//   new_rot_mat[2] = make_float3(0, 0, 1);
//   helper->updateRotation(0, new_rot_mat);
//   helper->updateOrigin(0, make_float3(1, 2, 3));
//
//   helper->updateTexture(0, data_vec);
//   helper->updateResolution(0, 10);
//   helper->enableTexture(0);
//   helper->copyToDevice(true);
//
//   CudaCheckError();
//   dynamics.GPUSetup();
//   CudaCheckError();
//
//   EXPECT_NE(dynamics.getHelper()->getLSTMDevicePtr(), nullptr);
//   EXPECT_NE(dynamics.network_d_, nullptr);
//   EXPECT_EQ(dynamics.network_d_, dynamics.getHelper()->getLSTMDevicePtr());
//
//   Eigen::Matrix<float, RacerDoubleIntegratorKinematic::CONTROL_DIM, num_rollouts> control_trajectory;
//   control_trajectory = Eigen::Matrix<float, RacerDoubleIntegratorKinematic::CONTROL_DIM, num_rollouts>::Random();
//   Eigen::Matrix<float, RacerDoubleIntegratorKinematic::STATE_DIM, num_rollouts> state_trajectory;
//   state_trajectory = Eigen::Matrix<float, RacerDoubleIntegratorKinematic::STATE_DIM, num_rollouts>::Random();
//
//   std::vector<std::array<float, RacerDoubleIntegratorKinematic::STATE_DIM>> s(num_rollouts);
//   std::vector<std::array<float, RacerDoubleIntegratorKinematic::STATE_DIM>> s_next(num_rollouts);
//   std::vector<std::array<float, RacerDoubleIntegratorKinematic::STATE_DIM>> s_der(num_rollouts);
//   // steering, throttle
//   std::vector<std::array<float, RacerDoubleIntegratorKinematic::CONTROL_DIM>> u(num_rollouts);
//
//   RacerDoubleIntegratorKinematic::state_array state;
//   RacerDoubleIntegratorKinematic::state_array next_state_cpu;
//   RacerDoubleIntegratorKinematic::control_array control;
//   RacerDoubleIntegratorKinematic::output_array output;
//   RacerDoubleIntegratorKinematic::state_array state_der_cpu = RacerDoubleIntegratorKinematic::state_array::Zero();
//
//   // Run dynamics on dynamicsU
//   // Run dynamics on GPU
//   for (int y_dim = 1; y_dim <= 16; y_dim++)
//   {
//     DYN::buffer_trajectory buffer;
//     buffer["VEL_X"] = Eigen::VectorXf::Random(51);
//     buffer["VEL_Y"] = Eigen::VectorXf::Random(51);
//     buffer["STEER_ANGLE"] = Eigen::VectorXf::Random(51);
//     buffer["STEER_ANGLE_RATE"] = Eigen::VectorXf::Random(51);
//     buffer["STEER_CMD"] = Eigen::VectorXf::Random(51);
//
//     for (int state_index = 0; state_index < num_rollouts; state_index++)
//     {
//       for (int dim = 0; dim < s[0].size(); dim++)
//       {
//         s[state_index][dim] = state_trajectory.col(state_index)(dim);
//       }
//       for (int dim = 0; dim < u[0].size(); dim++)
//       {
//         u[state_index][dim] = control_trajectory.col(state_index)(dim);
//       }
//     }
//     dynamics.updateFromBuffer(buffer);
//     launchStepTestKernel<RacerDoubleIntegratorKinematic>(dynamics, s, u, s_der, s_next, 0, dt, y_dim);
//     for (int point = 0; point < num_rollouts; point++)
//     {
//       dynamics.initializeDynamics(state, control, output, 0, 0);
//       state = state_trajectory.col(point);
//       control = control_trajectory.col(point);
//       state_der_cpu = RacerDoubleIntegratorKinematic::state_array::Zero();
//
//       dynamics.step(state, next_state_cpu, state_der_cpu, control, output, 0, dt);
//       // for (int dim = 0; dim < RacerDoubleIntegratorKinematic::STATE_DIM; dim++)
//       for (int dim = 0; dim < RacerDoubleIntegratorKinematic::STATE_DIM; dim++)
//       {
//         EXPECT_NEAR(state_der_cpu(dim), s_der[point][dim], 1e-4)
//             << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
//         // EXPECT_NEAR(state(dim), s[point][dim], 1e-4) << "at index " << point << " with y_dim " << y_dim;
//         EXPECT_NEAR(next_state_cpu(dim), s_next[point][dim], 1e-4)
//             << "at index " << point << " with y_dim " << y_dim << " dim " << dim;
//         EXPECT_TRUE(isfinite(s_next[point][dim]));
//       }
//     }
//   }
//   dynamics.freeCudaMem();
// }
// /*
// class LinearDummy : public RacerDoubleIntegratorKinematic {
// public:
//   bool computeGrad(const Eigen::Ref<const state_array> & state,
//                    const Eigen::Ref<const control_array>& control,
//                    Eigen::Ref<dfdx> A,
//                    Eigen::Ref<dfdu> B) {
//     return false;
//   };
// };
//
// TEST_F(RacerDoubleIntegratorKinematicTest, TestComputeGradComputation) {
//   Eigen::Matrix<float, RacerDoubleIntegratorKinematic::STATE_DIM, RacerDoubleIntegratorKinematic::STATE_DIM +
// RacerDoubleIntegratorKinematic::CONTROL_DIM> numeric_jac; Eigen::Matrix<float,
// RacerDoubleIntegratorKinematic::STATE_DIM, RacerDoubleIntegratorKinematic::STATE_DIM +
// RacerDoubleIntegratorKinematic::CONTROL_DIM> analytic_jac; RacerDoubleIntegratorKinematic::state_array state;
// state
// << 1, 2, 3, 4; RacerDoubleIntegratorKinematic::control_array control; control << 5;
//
//   auto analytic_grad_model = RacerDoubleIntegratorKinematic();
//
//   RacerDoubleIntegratorKinematic::dfdx A_analytic = RacerDoubleIntegratorKinematic::dfdx::Zero();
//   RacerDoubleIntegratorKinematic::dfdu B_analytic = RacerDoubleIntegratorKinematic::dfdu::Zero();
//
//   analytic_grad_model.computeGrad(state, control, A_analytic, B_analytic);
//
//   auto numerical_grad_model = LinearDummy();
//
//   std::shared_ptr<ModelWrapperDDP<LinearDummy>> ddp_model =
// std::make_shared<ModelWrapperDDP<LinearDummy>>(&numerical_grad_model);
//
//   analytic_jac.leftCols<RacerDoubleIntegratorKinematic::STATE_DIM>() = A_analytic;
//   analytic_jac.rightCols<RacerDoubleIntegratorKinematic::CONTROL_DIM>() = B_analytic;
//   numeric_jac = ddp_model->df(state, control);
//
//   ASSERT_LT((numeric_jac - analytic_jac).norm(), 1e-3) << "Numeric Jacobian\n" << numeric_jac << "\nAnalytic
//   Jacobian\n"
// << analytic_jac;
// }
//
// */
