#include "hip/hip_runtime.h"
//
// Created by jgibson37 on 1/13/20.
//

#include <gtest/gtest.h>
#include <mppi/dynamics/autorally/ar_nn_model.cuh>
#include <mppi/dynamics/autorally/ar_nn_dynamics_kernel_test.cuh>
#include <stdio.h>
#include <math.h>

// Auto-generated header file
#include <autorally_test_network.h>
#include "mppi/ddp/ddp_model_wrapper.h"

/**
 * Note: the analytical solution for the test NN is outlined in the python script
 */

TEST(ARNeuralNetDynamics, verifyTemplateParamters)
{
  int state_dim = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::STATE_DIM;
  EXPECT_EQ(state_dim, 7);

  int control_dim = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::CONTROL_DIM;
  EXPECT_EQ(control_dim, 2);

  int dynamics_dim = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::DYNAMICS_DIM;
  EXPECT_EQ(dynamics_dim, 7 - 3);

  int num_layers = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::NUM_LAYERS;
  EXPECT_EQ(num_layers, 4);

  int prime_padding = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::PRIME_PADDING;
  EXPECT_EQ(prime_padding, 1);

  int largest_layer = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::LARGEST_LAYER;
  EXPECT_EQ(largest_layer, 32 + 1);

  int num_params = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::NUM_PARAMS;
  EXPECT_EQ(num_params, (6 + 1) * 32 + (32 + 1) * 32 + (32 + 1) * 4);

  int shared_mem_request_grd = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::SHARED_MEM_REQUEST_GRD_BYTES;
  EXPECT_EQ(shared_mem_request_grd, sizeof(FNNParams<6, 32, 32, 4>));

  int shared_mem_request_blk = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::SHARED_MEM_REQUEST_BLK_BYTES;
  EXPECT_EQ(shared_mem_request_blk, (32 + 1) * 2);

  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;
  std::array<int, 4> net_structure = model.getNetStructure();

  EXPECT_EQ(net_structure[0], 6);
  EXPECT_EQ(net_structure[1], 32);
  EXPECT_EQ(net_structure[2], 32);
  EXPECT_EQ(net_structure[3], 4);
}

TEST(ARNeuralNetDynamics, BindStreamControlRanges)
{
  hipStream_t stream;

  HANDLE_ERROR(hipStreamCreate(&stream));

  std::array<float2, 2> u_constraint = {};
  u_constraint[0].x = -1.0;
  u_constraint[0].y = 1.0;

  u_constraint[1].x = -2.0;
  u_constraint[1].y = 2.0;
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model(u_constraint, stream);

  EXPECT_EQ(model.stream_, stream) << "Stream binding failure.";

  HANDLE_ERROR(hipStreamDestroy(stream));
}

TEST(ARNeuralNetDynamics, BindStreamDefaultArgRanges)
{
  hipStream_t stream;

  HANDLE_ERROR(hipStreamCreate(&stream));

  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model(stream);

  EXPECT_EQ(model.stream_, stream) << "Stream binding failure.";

  HANDLE_ERROR(hipStreamDestroy(stream));
}

TEST(ARNeuralNetDynamics, ControlRangesSetDefaultCPU)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  std::array<float2, 2> ranges = model.getControlRanges();
  for (int i = 0; i < 2; i++)
  {
    EXPECT_FLOAT_EQ(ranges[0].x, -FLT_MAX);
    EXPECT_FLOAT_EQ(ranges[0].y, FLT_MAX);
  }
}

TEST(ARNeuralNetDynamics, ControlRangesSetCPU)
{
  std::array<float2, 2> u_constraint = {};
  u_constraint[0].x = -1.0;
  u_constraint[0].y = 1.0;

  u_constraint[1].x = -2.0;
  u_constraint[1].y = 2.0;
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model(u_constraint);

  std::array<float2, 2> ranges = model.getControlRanges();
  EXPECT_FLOAT_EQ(ranges[0].x, -1.0);
  EXPECT_FLOAT_EQ(ranges[0].y, 1.0);

  EXPECT_FLOAT_EQ(ranges[1].x, -2.0);
  EXPECT_FLOAT_EQ(ranges[1].y, 2.0);
}

TEST(ARNeuralNetDynamics, stideIdcsSetDefault)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  std::array<int, 6> result = model.getStideIdcs();

  EXPECT_EQ(result[0], 0);
  EXPECT_EQ(result[1], 192);
  EXPECT_EQ(result[2], 224);
  EXPECT_EQ(result[3], 1248);
  EXPECT_EQ(result[4], 1280);
  EXPECT_EQ(result[5], 1408);
}

TEST(ARNeuralNetDynamics, GPUSetupAndParamsCheck)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  std::array<float, 1412> theta = model.getTheta();
  std::array<int, 6> stride = model.getStideIdcs();
  std::array<int, 4> net_structure = model.getNetStructure();

  std::array<float, 1412> theta_result = {};
  std::array<int, 6> stride_result = {};
  std::array<int, 4> net_structure_result = {};

  EXPECT_EQ(model.GPUMemStatus_, false);
  EXPECT_EQ(model.model_d_, nullptr);
  EXPECT_NE(model.getHelperPtr(), nullptr);

  model.GPUSetup();

  EXPECT_EQ(model.GPUMemStatus_, true);
  EXPECT_NE(model.model_d_, nullptr);

  // launch kernel
  launchParameterCheckTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 1412, 6, 4>(model, theta_result, stride_result,
                                                                                    net_structure_result);

  for (int i = 0; i < 1412; i++)
  {
    // these are a bunch of mostly random values and nan != nan
    if (!isnan(theta[i]))
    {
      EXPECT_FLOAT_EQ(theta_result[i], theta[i]);
    }
  }
  for (int i = 0; i < 6; i++)
  {
    EXPECT_EQ(stride_result[i], stride[i]);
  }

  for (int i = 0; i < 4; i++)
  {
    EXPECT_EQ(net_structure[i], net_structure_result[i]);
  }
}

TEST(ARNeuralNetDynamics, UpdateModelTest)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  std::array<float, 1412> theta = model.getTheta();
  std::array<int, 6> stride = model.getStideIdcs();
  std::array<int, 4> net_structure = model.getNetStructure();

  std::array<float, 1412> theta_result = {};
  std::array<int, 6> stride_result = {};
  std::array<int, 4> net_structure_result = {};

  model.GPUSetup();

  std::vector<float> theta_vec(1412);
  srand(time(NULL));
  for (int i = 0; i < 1412; i++)
  {
    theta_vec[i] = rand();
  }

  model.updateModel({ 6, 32, 32, 4 }, theta_vec);

  // check CPU
  for (int i = 0; i < 1412; i++)
  {
    // these are a bunch of mostly random values and nan != nan
    if (!isnan(theta_vec[i]))
    {
      EXPECT_FLOAT_EQ(model.getTheta()[i], theta_vec[i]);
    }
  }

  // launch kernel
  launchParameterCheckTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 1412, 6, 4>(model, theta_result, stride_result,
                                                                                    net_structure_result);

  for (int i = 0; i < 1412; i++)
  {
    // these are a bunch of mostly random values and nan != nan
    if (!isnan(theta_vec[i]))
    {
      EXPECT_FLOAT_EQ(theta_result[i], theta_vec[i]) << "failed at index " << i;
    }
  }
  for (int i = 0; i < 6; i++)
  {
    EXPECT_EQ(stride_result[i], stride[i]);
  }

  for (int i = 0; i < 4; i++)
  {
    EXPECT_EQ(net_structure[i], net_structure_result[i]);
  }
}

TEST(ARNeuralNetDynamics, LoadModelTest)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;
  model.GPUSetup();

  // TODO procedurally generate a NN in python and save and run like costs
  std::string path = mppi::tests::test_load_nn_file;
  model.loadParams(path);

  // check CPU
  for (int i = 0; i < 1412; i++)
  {
    EXPECT_FLOAT_EQ(model.getTheta()[i], i) << "failed at index " << i;
  }

  std::array<float, 1412> theta_result = {};
  std::array<int, 6> stride_result = {};
  std::array<int, 4> net_structure_result = {};

  // launch kernel
  launchParameterCheckTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 1412, 6, 4>(model, theta_result, stride_result,
                                                                                    net_structure_result);

  for (int i = 0; i < 1412; i++)
  {
    EXPECT_FLOAT_EQ(theta_result[i], i) << "failed at index " << i;
  }
}

TEST(ARNeuralNetDynamics, computeKinematicsTestCPU)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::state_array s(7, 1);
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::state_array s_der(7, 1);

  s(2) = 0.0;  // yaw
  s(4) = 1.0;  // body frame vx
  s(5) = 2.0;  // body frame vy
  s(6) = 0.0;  // yaw dot

  model.computeKinematics(s, s_der);

  EXPECT_FLOAT_EQ(s_der(0), 1.0);
  EXPECT_FLOAT_EQ(s_der(1), 2.0);
  EXPECT_FLOAT_EQ(s_der(2), 0.0);

  s(2) = M_PI / 2;  // yaw
  s(4) = 3.0;       // body frame vx
  s(5) = 5.0;       // body frame vy
  s(6) = 1.0;       // yaw dot

  model.computeKinematics(s, s_der);

  EXPECT_FLOAT_EQ(s_der(0), -5);
  EXPECT_FLOAT_EQ(s_der(1), 3.0);
  EXPECT_FLOAT_EQ(s_der(2), -1.0);
}

TEST(ARNeuralNetDynamics, computeKinematicsTestGPU)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  std::vector<std::array<float, 7>> s(1);
  std::vector<std::array<float, 7>> s_der(1);

  model.GPUSetup();

  for (int y_dim = 1; y_dim < 17; y_dim++)
  {
    s[0] = { 0.0 };
    s[0][2] = 0.0;  // yaw
    s[0][4] = 1.0;  // body frame vx
    s[0][5] = 2.0;  // body frame vy
    s[0][6] = 0.0;  // yaw dot

    s_der[0] = { 0.0 };

    launchComputeKinematicsTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7>(model, s, s_der, y_dim);

    EXPECT_FLOAT_EQ(s_der[0][0], 1.0);
    EXPECT_FLOAT_EQ(s_der[0][1], 2.0);
    EXPECT_FLOAT_EQ(s_der[0][2], 0.0);

    s[0][2] = M_PI / 2;  // yaw
    s[0][4] = 3.0;       // body frame vx
    s[0][5] = 5.0;       // body frame vy
    s[0][6] = 1.0;       // yaw dot

    launchComputeKinematicsTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7>(model, s, s_der, y_dim);

    EXPECT_FLOAT_EQ(s_der[0][0], -5);
    EXPECT_FLOAT_EQ(s_der[0][1], 3.0);
    EXPECT_FLOAT_EQ(s_der[0][2], -1.0);
  }
}

TEST(ARNeuralNetDynamics, updateStateGPUTest)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  std::vector<std::array<float, 7>> s(1);
  // x_dot, y_dot, theta_dot
  std::vector<std::array<float, 7>> s_der(1);

  model.GPUSetup();

  for (int j = 1; j < 17; j++)
  {
    s[0] = { 0.0 };
    s[0][2] = 0.0;  // yaw
    s[0][4] = 1.0;  // body frame vx
    s[0][5] = 2.0;  // body frame vy
    s[0][6] = 0.0;  // yaw dot

    s_der[0] = { 0.0 };
    s_der[0][0] = 1.0;
    s_der[0][1] = 2.0;
    s_der[0][2] = 3.0;

    launchUpdateStateTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7>(model, s, s_der, 0.1, j);

    EXPECT_FLOAT_EQ(s_der[0][0], 1);
    EXPECT_FLOAT_EQ(s_der[0][1], 2);
    EXPECT_FLOAT_EQ(s_der[0][2], 3);
    EXPECT_FLOAT_EQ(s_der[0][3], 0);
    EXPECT_FLOAT_EQ(s_der[0][4], 0);
    EXPECT_FLOAT_EQ(s_der[0][5], 0);
    EXPECT_FLOAT_EQ(s_der[0][6], 0);

    EXPECT_FLOAT_EQ(s[0][0], 0.1);
    EXPECT_FLOAT_EQ(s[0][1], 0.2);
    EXPECT_FLOAT_EQ(s[0][2], 0.3);
    EXPECT_FLOAT_EQ(s[0][3], 0.0);
    EXPECT_FLOAT_EQ(s[0][4], 1.0);
    EXPECT_FLOAT_EQ(s[0][5], 2.0);
    EXPECT_FLOAT_EQ(s[0][6], 0.0);
  }
}

/**
 *
 * @tparam CLASS_T
 * @param model
 * @param s
 * @param ds
 * @param u
 * @param du
 */
template <class CLASS_T>
void compareFiniteDifferenceGradient(CLASS_T& model, Eigen::MatrixXf& s, Eigen::MatrixXf& ds, Eigen::MatrixXf& u,
                                     Eigen::MatrixXf& du)
{
  Eigen::MatrixXf s_2(7, 1);
  s_2 = s + ds;
  Eigen::MatrixXf u_2(2, 1);
  u_2 = u + du;
  Eigen::MatrixXf s_der(7, 1);
  Eigen::MatrixXf s_der_2(7, 1);
  s_der.setZero();
  s_der_2.setZero();

  Eigen::MatrixXf calculated_A(7, 7);
  Eigen::MatrixXf calculated_B(7, 2);

  model.computeDynamics(s_2, u_2, s_der_2);
  model.computeDynamics(s, u, s_der);
  std::cout << "s_der\n" << s_der << std::endl;
  std::cout << "s_der_2\n" << s_der_2 << std::endl;
  std::cout << "s_der_2 - s_der\n" << (s_der_2 - s_der) << std::endl;

  Eigen::MatrixXf A(7, 7);
  Eigen::MatrixXf B(7, 2);

  model.computeGrad(s, u, A, B);
  std::cout << "A = \n" << A << std::endl;
  std::cout << "B = \n" << B << std::endl;

  // compare A
  for (int i = 0; i < 7; i++)
  {
    for (int j = 0; j < 7; j++)
    {
      EXPECT_NEAR(calculated_A(i, j), A(i, j), 0.01) << "failed at index = " << i << ", " << j;
    }
  }

  // compare B
  for (int i = 0; i < 7; i++)
  {
    for (int j = 0; j < 2; j++)
    {
      EXPECT_NEAR(calculated_B(i, j), B(i, j), 0.01) << "failed at index = " << i << ", " << j;
    }
  }
}

/*
// Note math for analytical solution is in the python script
TEST(ARNeuralNetDynamics, computeGrad) {
  GTEST_SKIP();
  NeuralNetModel<7,2,3,6,32,32,4> model;

  Eigen::MatrixXf s(7, 1);
  Eigen::MatrixXf ds(7, 1);
  Eigen::MatrixXf u(2, 1);
  Eigen::MatrixXf du(2, 1);
  s.setZero();
  ds.setZero();
  u.setZero();
  du.setZero();
  ds << 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0;

  std::vector<float> theta(1412);

  std::fill(theta.begin(), theta.end(), 1);
  model.updateModel({6, 32, 32, 4}, theta);

  compareFiniteDifferenceGradient(model, s, ds, u, du);

}
 */

TEST(ARNeuralNetDynamics, computeDynamicsCPU)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::state_array s;
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::control_array u;
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::state_array s_der;
  s.setZero();
  s_der.setZero();
  u << 1, -1;

  std::vector<float> theta(1412);

  std::fill(theta.begin(), theta.end(), 1);
  model.updateModel({ 6, 32, 32, 4 }, theta);

  model.computeDynamics(s, u, s_der);

  EXPECT_FLOAT_EQ(s(0), 0);
  EXPECT_FLOAT_EQ(s(1), 0);
  EXPECT_FLOAT_EQ(s(2), 0);
  EXPECT_FLOAT_EQ(s(3), 0);
  EXPECT_FLOAT_EQ(s(4), 0);
  EXPECT_FLOAT_EQ(s(5), 0);
  EXPECT_FLOAT_EQ(s(6), 0);

  EXPECT_FLOAT_EQ(s_der(0), 0);
  EXPECT_FLOAT_EQ(s_der(1), 0);
  EXPECT_FLOAT_EQ(s_der(2), 0);
  EXPECT_FLOAT_EQ(s_der(3), 33);
  EXPECT_FLOAT_EQ(s_der(4), 33);
  EXPECT_FLOAT_EQ(s_der(5), 33);
  EXPECT_FLOAT_EQ(s_der(6), 33);

  EXPECT_FLOAT_EQ(u(0), 1.0);
  EXPECT_FLOAT_EQ(u(1), -1.0);
}

TEST(ARNeuralNetDynamics, computeDynamicsGPU)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;

  std::vector<std::array<float, 7>> s(1);
  // x_dot, y_dot, theta_dot
  std::vector<std::array<float, 7>> s_der(1);
  // steering, throttle
  std::vector<std::array<float, 2>> u(1);

  std::vector<float> theta(1412);
  model.GPUSetup();

  std::fill(theta.begin(), theta.end(), 1);
  model.updateModel({ 6, 32, 32, 4 }, theta);

  for (int y_dim = 1; y_dim < 17; y_dim++)
  {
    s[0] = { 0 };
    s_der[0] = { 0 };
    u[0] = { 0 };
    u[0][0] = 1.0;
    u[0][1] = -1.0;

    launchComputeDynamicsTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7, 2>(model, s, u, s_der, y_dim);

    EXPECT_FLOAT_EQ(s[0][0], 0);
    EXPECT_FLOAT_EQ(s[0][1], 0);
    EXPECT_FLOAT_EQ(s[0][2], 0);
    EXPECT_FLOAT_EQ(s[0][3], 0);
    EXPECT_FLOAT_EQ(s[0][4], 0);
    EXPECT_FLOAT_EQ(s[0][5], 0);
    EXPECT_FLOAT_EQ(s[0][6], 0);

    EXPECT_FLOAT_EQ(s_der[0][0], 0);
    EXPECT_FLOAT_EQ(s_der[0][1], 0);
    EXPECT_FLOAT_EQ(s_der[0][2], 0);
    EXPECT_FLOAT_EQ(s_der[0][3], 33) << "at y_dim " << y_dim;
    EXPECT_FLOAT_EQ(s_der[0][4], 33) << "at y_dim " << y_dim;
    EXPECT_FLOAT_EQ(s_der[0][5], 33) << "at y_dim " << y_dim;
    EXPECT_FLOAT_EQ(s_der[0][6], 33) << "at y_dim " << y_dim;

    EXPECT_FLOAT_EQ(u[0][0], 1.0);
    EXPECT_FLOAT_EQ(u[0][1], -1.0);
  }
}

// TODO compute state deriv CPU
TEST(ARNeuralNetDynamics, computeStateDerivCPU)
{
}

TEST(ARNeuralNetDynamics, computeStateDerivGPU)
{
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model;
  model.GPUSetup();

  std::vector<std::array<float, 7>> s(1);
  // x_dot, y_dot, theta_dot
  std::vector<std::array<float, 7>> s_der(1);
  // steering, throttle
  std::vector<std::array<float, 2>> u(1);

  std::vector<float> theta(1412);
  std::fill(theta.begin(), theta.end(), 1);
  model.updateModel({ 6, 32, 32, 4 }, theta);

  for (int j = 1; j < 17; j++)
  {
    s[0] = { 0.0 };
    s[0][4] = 1;
    s[0][5] = 2;
    s[0][6] = 3;

    s_der[0] = { 0.0 };
    u[0] = { 0.0 };

    launchComputeStateDerivTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7, 2>(model, s, u, s_der, j);

    EXPECT_FLOAT_EQ(s[0][0], 0);
    EXPECT_FLOAT_EQ(s[0][1], 0);
    EXPECT_FLOAT_EQ(s[0][2], 0);
    EXPECT_FLOAT_EQ(s[0][3], 0);
    EXPECT_FLOAT_EQ(s[0][4], 1);
    EXPECT_FLOAT_EQ(s[0][5], 2);
    EXPECT_FLOAT_EQ(s[0][6], 3);

    EXPECT_FLOAT_EQ(s_der[0][0], 1);
    EXPECT_FLOAT_EQ(s_der[0][1], 2);
    EXPECT_FLOAT_EQ(s_der[0][2], -3);
    EXPECT_FLOAT_EQ(s_der[0][3], 33);
    EXPECT_FLOAT_EQ(s_der[0][4], 33);
    EXPECT_FLOAT_EQ(s_der[0][5], 33);
    EXPECT_FLOAT_EQ(s_der[0][6], 33);

    EXPECT_FLOAT_EQ(u[0][0], 0);
    EXPECT_FLOAT_EQ(u[0][1], 0);
  }
}

TEST(ARNeuralNetDynamics, full)
{
  std::array<float2, 2> u_constraint = {};
  u_constraint[0].x = -1.0;
  u_constraint[0].y = 1.0;
  u_constraint[1].x = -2.0;
  u_constraint[1].y = 2.0;

  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model(u_constraint);

  model.GPUSetup();

  std::vector<std::array<float, 7>> s(1);
  // x_dot, y_dot, theta_dot
  std::vector<std::array<float, 7>> s_der(1);
  // steering, throttle
  std::vector<std::array<float, 2>> u(1);

  std::vector<float> theta(1412);

  for (int y_dim = 1; y_dim < 17; y_dim++)
  {
    std::fill(theta.begin(), theta.end(), 0);
    model.updateModel({ 6, 32, 32, 4 }, theta);

    s[0] = { 0.0 };
    s_der[0] = { 0.0 };
    u[0] = { 0.0 };

    launchFullARNNTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7, 2, 1, 1>(model, s, u, s_der, 0.1, y_dim);

    EXPECT_FLOAT_EQ(s[0][0], 0);
    EXPECT_FLOAT_EQ(s[0][1], 0);
    EXPECT_FLOAT_EQ(s[0][2], 0);
    EXPECT_FLOAT_EQ(s[0][3], 0);
    EXPECT_FLOAT_EQ(s[0][4], 0);
    EXPECT_FLOAT_EQ(s[0][5], 0);
    EXPECT_FLOAT_EQ(s[0][6], 0);

    EXPECT_FLOAT_EQ(s_der[0][0], 0);
    EXPECT_FLOAT_EQ(s_der[0][1], 0);
    EXPECT_FLOAT_EQ(s_der[0][2], 0);
    EXPECT_FLOAT_EQ(s_der[0][3], 0);
    EXPECT_FLOAT_EQ(s_der[0][4], 0);
    EXPECT_FLOAT_EQ(s_der[0][5], 0);
    EXPECT_FLOAT_EQ(s_der[0][6], 0);

    EXPECT_FLOAT_EQ(u[0][0], 0);
    EXPECT_FLOAT_EQ(u[0][1], 0);

    u[0][0] = 100;
    u[0][1] = -20;

    std::fill(theta.begin(), theta.end(), 1);
    model.updateModel({ 6, 32, 32, 4 }, theta);

    launchFullARNNTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7, 2, 1>(model, s, u, s_der, 0.1, y_dim);

    EXPECT_FLOAT_EQ(s[0][0], 0);
    EXPECT_FLOAT_EQ(s[0][1], 0);
    EXPECT_FLOAT_EQ(s[0][2], 0);
    EXPECT_FLOAT_EQ(s[0][3], 2.5371017) << "y_dim " << y_dim;
    EXPECT_FLOAT_EQ(s[0][4], 2.5371017);
    EXPECT_FLOAT_EQ(s[0][5], 2.5371017);
    EXPECT_FLOAT_EQ(s[0][6], 2.5371017);

    EXPECT_FLOAT_EQ(s_der[0][0], 0);
    EXPECT_FLOAT_EQ(s_der[0][1], 0);
    EXPECT_FLOAT_EQ(s_der[0][2], 0);
    EXPECT_FLOAT_EQ(s_der[0][3], 25.371017);
    EXPECT_FLOAT_EQ(s_der[0][4], 25.371017);
    EXPECT_FLOAT_EQ(s_der[0][5], 25.371017);
    EXPECT_FLOAT_EQ(s_der[0][6], 25.371017);

    EXPECT_FLOAT_EQ(u[0][0], 1.0);
    EXPECT_FLOAT_EQ(u[0][1], -2.0);
  }
}

void parseTextIntoDataPointHelper(std::string text, std::array<float, 7>& state, std::array<float, 7>& state_result,
                                  std::array<float, 7>& state_der, std::array<float, 2>& control)
{
  size_t line_pos = 0;
  size_t prev_line_pos = 1;
  int what_var = 0;
  text.append(" *");
  while ((line_pos = text.find("*", prev_line_pos)) != std::string::npos)
  {
    std::string line = text.substr(prev_line_pos, line_pos - prev_line_pos);
    line.append(" ");
    size_t value_pos = 0;
    size_t prev_value_pos = 0;
    int counter = 0;
    while ((value_pos = line.find(" ", prev_value_pos)) != std::string::npos)
    {
      std::string value = line.substr(prev_value_pos, value_pos - prev_value_pos);
      // makes sure it is a number
      if (!value.empty())
      {
        float number = 0;  // = std::stoi(value.substr(0, std::string::npos));
        if (value[0] == '-' && isdigit(value[1]) || isdigit(value[0]))
        {
          number = std::stof(value);
        }
        else
        {
          prev_value_pos = value_pos + 1;
          continue;
        }
        if (what_var == 0)
        {
          state[counter++] = number;
        }
        else if (what_var == 1)
        {
          control[counter++] = number;
        }
        else if (what_var == 2)
        {
          state_der[counter++] = number;
        }
        else if (what_var == 3)
        {
          state_result[counter++] = number;
        }
      }
      prev_value_pos = std::min(value_pos + 1, line.length());
    }
    what_var++;
    prev_line_pos = std::min(line_pos + 1, text.length());
  }
}

/**
 * values grabbed 7/7/20 from melodic-devel in gazebo with
 * model = <param name="model_path" value="$(env AR_MPPI_PARAMS_PATH)/models/autorally_nnet_09_12_2018.npz" />
 *
 * input state 4.264431 -30.974377 -0.955451 -0.028595 3.346700 0.048521 0.315486
 * input control -0.221381 0.089168
 * output state_der 1.971473 -2.704820 -0.315486 0.136986 -0.877249 0.713279 -2.542408
 * output state 4.303861 -31.028473 -0.961760 -0.025856 3.329155 0.062786 0.264637
 *
 * input state 17.818813 -33.751003 2.603801 0.046806 3.690995 -0.162579 0.046056
 * input control 0.247343 0.521264
 * output state_der -3.086702 2.030306 -0.046056 -0.170345 5.059716 -0.795022 1.001305
 * output state 17.757080 -33.710396 2.602880 0.043399 3.792189 -0.178480 0.066082
 *
 * input state 29.102535 -28.311907 0.324347 0.030684 4.354917 -0.250158 -0.181747
 * input control -0.465928 0.304528
 * output state_der 4.207570 1.150753 0.181747 0.285242 0.759887 1.638221 -3.125369
 * output state 29.186687 -28.288891 0.327982 0.036389 4.370114 -0.217393 -0.244254
 *
 * input state -2.603741 -15.197432 2.065675 0.001336 4.745344 0.066507 0.039070
 * input control 0.212821 0.477029
 * output state_der -2.312211 4.144441 -0.039070 -0.049963 4.542547 -1.031745 1.590496
 * output state -2.649985 -15.114543 2.064894 0.000337 4.836195 0.045872 0.070880
 *
 * input state -2.460799 -15.449237 2.066902 0.001312 4.874847 0.114099 0.022956
 * input control -0.126225 -0.288144
 * output state_der -2.420791 4.232839 -0.022956 0.106002 -2.117054 -0.294837 -1.255360
 * output state -2.509215 -15.364580 2.066442 0.003432 4.832506 0.108203 -0.002152
 *
 * input state -9.680823 -7.434339 11.683491 0.428528 -1.655267 -3.769846 -2.194825
 * input control -0.003010 0.147335
 * output state_der -3.963449 -1.114773 2.194825 -0.100577 3.684401 7.052448 2.626748
 * output state -9.760093 -7.456634 11.727387 0.426516 -1.581579 -3.628797 -2.142290
 *
 * input state 20.899069 -37.733856 0.468771 0.109863 6.329141 -0.916158 -0.136938
 * input control 0.281979 0.775381
 * output state_der 6.060292 2.042114 0.136938 -0.325823 4.262759 -1.176095 0.928013
 * output state 21.020275 -37.693012 0.471509 0.103346 6.414396 -0.939680 -0.118377
 */
TEST(ARNeuralNetDynamics, fullComparedToAutoRallyImpl)
{
  std::array<float2, 2> u_constraint = {};
  u_constraint[0].x = -0.99;
  u_constraint[0].y = 0.65;
  u_constraint[1].x = -0.99;
  u_constraint[1].y = 0.99;

  NeuralNetModel<7, 2, 3, 6, 32, 32, 4> model(u_constraint);

  model.GPUSetup();

  int number_queries = 7;

  std::vector<std::array<float, 7>> s(number_queries);
  // x_dot, y_dot, theta_dot
  std::vector<std::array<float, 7>> s_der(number_queries);
  // steering, throttle
  std::vector<std::array<float, 2>> u(number_queries);

  std::vector<std::array<float, 7>> s_result(number_queries);
  // x_dot, y_dot, theta_dot
  std::vector<std::array<float, 7>> s_der_result(number_queries);
  // steering, throttle
  std::vector<std::array<float, 2>> u_result(number_queries);

  parseTextIntoDataPointHelper(
      "* input state 4.264431 -30.974377 -0.955451 -0.028595 3.346700 0.048521 0.315486"
      "* input control -0.221381 0.089168"
      "* output state_der 1.971473 -2.704820 -0.315486 0.136986 -0.877249 0.713279 -2.542408"
      "* output state 4.303861 -31.028473 -0.961760 -0.025856 3.329155 0.062786 0.264637",
      s[0], s_result[0], s_der_result[0], u[0]);
  parseTextIntoDataPointHelper(
      "* input state 17.818813 -33.751003 2.603801 0.046806 3.690995 -0.162579 0.046056"
      "* input control 0.247343 0.521264"
      "* output state_der -3.086702 2.030306 -0.046056 -0.170345 5.059716 -0.795022 1.001305"
      "* output state 17.757080 -33.710396 2.602880 0.043399 3.792189 -0.178480 0.066082",
      s[1], s_result[1], s_der_result[1], u[1]);
  parseTextIntoDataPointHelper(
      "* input state 29.102535 -28.311907 0.324347 0.030684 4.354917 -0.250158 -0.181747"
      "* input control -0.465928 0.304528"
      "* output state_der 4.207570 1.150753 0.181747 0.285242 0.759887 1.638221 -3.125369"
      "* output state 29.186687 -28.288891 0.327982 0.036389 4.370114 -0.217393 -0.244254",
      s[2], s_result[2], s_der_result[2], u[2]);
  parseTextIntoDataPointHelper(
      "* input state -2.603741 -15.197432 2.065675 0.001336 4.745344 0.066507 0.039070"
      "* input control 0.212821 0.477029"
      "* output state_der -2.312211 4.144441 -0.039070 -0.049963 4.542547 -1.031745 1.590496"
      "* output state -2.649985 -15.114543 2.064894 0.000337 4.836195 0.045872 0.070880",
      s[3], s_result[3], s_der_result[3], u[3]);
  parseTextIntoDataPointHelper(
      "* input state -2.460799 -15.449237 2.066902 0.001312 4.874847 0.114099 0.022956"
      "* input control -0.126225 -0.288144"
      "* output state_der -2.420791 4.232839 -0.022956 0.106002 -2.117054 -0.294837 -1.255360"
      "* output state -2.509215 -15.364580 2.066442 0.003432 4.832506 0.108203 -0.002152",
      s[4], s_result[4], s_der_result[4], u[4]);
  parseTextIntoDataPointHelper(
      "* input state -9.680823 -7.434339 11.683491 0.428528 -1.655267 -3.769846 -2.194825"
      "* input control -0.003010 0.147335"
      "* output state_der -3.963449 -1.114773 2.194825 -0.100577 3.684401 7.052448 2.626748"
      "* output state -9.760093 -7.456634 11.727387 0.426516 -1.581579 -3.628797 -2.142290",
      s[5], s_result[5], s_der_result[5], u[5]);
  parseTextIntoDataPointHelper(
      "* input state 20.899069 -37.733856 0.468771 0.109863 6.329141 -0.916158 -0.136938"
      "* input control 0.281979 0.775381"
      "* output state_der 6.060292 2.042114 0.136938 -0.325823 4.262759 -1.176095 0.928013"
      "* output state 21.020275 -37.693012 0.471509 0.103346 6.414396 -0.939680 -0.118377",
      s[6], s_result[6], s_der_result[6], u[6]);

  std::copy(u.begin(), u.end(), u_result.begin());

  std::string path = mppi::tests::old_autorally_network_file;
  model.loadParams(path);

  for (int y_dim = 1; y_dim < 33; y_dim++)
  {
    std::vector<std::array<float, 7>> s_temp(s);
    std::vector<std::array<float, 7>> s_der_temp(s_der);
    std::vector<std::array<float, 2>> u_temp(u);

    launchFullARNNTestKernel<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>, 7, 2, 7>(model, s_temp, u_temp, s_der_temp,
                                                                             1.0 / 50, y_dim);

    for (int point_index = 0; point_index < 1; point_index++)
    {
      for (int dim = 0; dim < 7; dim++)
      {
        EXPECT_NEAR(s_temp[point_index][dim], s_result[point_index][dim], 0.001)
            << "point_index: " << point_index << " dim: " << dim << " y_dim: " << y_dim
            << " diff = " << std::abs(s_temp[point_index][dim] - s_result[point_index][dim]);
        EXPECT_NEAR(s_der_temp[point_index][dim], s_der_result[point_index][dim], 0.001)
            << "point_index: " << point_index << " dim: " << dim << " y_dim: " << y_dim
            << " diff = " << std::abs(s_der_temp[point_index][dim] - s_der_result[point_index][dim]);
      }
      for (int dim = 0; dim < 2; dim++)
      {
        EXPECT_NEAR(u_temp[point_index][dim], u_result[point_index][dim], 0.001)
            << "point_index: " << point_index << " dpoint_indexm: " << dim;
      }
    }
  }
}

class DynamicsDummy : public NeuralNetModel<7, 2, 3, 6, 32, 32, 4>
{
public:
  bool computeGrad(const Eigen::Ref<const state_array>& state, const Eigen::Ref<const control_array>& control,
                   Eigen::Ref<dfdx> A, Eigen::Ref<dfdu> B)
  {
    return false;
  };
};

TEST(ARNeuralNetDynamics, computeGradTest)
{
  std::string path = mppi::tests::old_autorally_network_file;

  Eigen::Matrix<float, NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::STATE_DIM,
                NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::STATE_DIM + NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::CONTROL_DIM>
      numeric_jac;
  Eigen::Matrix<float, NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::STATE_DIM,
                NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::STATE_DIM + NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::CONTROL_DIM>
      analytic_jac;
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::state_array state;
  state << 4.264431, -30.974377, -0.955451, -0.028595, 3.346700, 0.048521, 0.315486;
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::control_array control;
  control << -0.221381, 0.089168;

  auto analytic_grad_model = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>();
  analytic_grad_model.loadParams(path);

  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::dfdx A_analytic = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::dfdx::Zero();
  NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::dfdu B_analytic = NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::dfdu::Zero();

  analytic_grad_model.computeGrad(state, control, A_analytic, B_analytic);

  auto numerical_grad_model = DynamicsDummy();
  numerical_grad_model.loadParams(path);

  std::shared_ptr<ModelWrapperDDP<DynamicsDummy>> ddp_model =
      std::make_shared<ModelWrapperDDP<DynamicsDummy>>(&numerical_grad_model);

  analytic_jac.leftCols<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::STATE_DIM>() = A_analytic;
  analytic_jac.rightCols<NeuralNetModel<7, 2, 3, 6, 32, 32, 4>::CONTROL_DIM>() = B_analytic;
  numeric_jac = ddp_model->df(state, control);

  ASSERT_LT((numeric_jac - analytic_jac).norm(), 1e-1) << "Numeric Jacobian\n"
                                                       << numeric_jac << "\nAnalytic Jacobian\n"
                                                       << analytic_jac;
}
