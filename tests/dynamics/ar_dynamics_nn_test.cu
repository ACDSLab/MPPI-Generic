#include "hip/hip_runtime.h"
//
// Created by jgibson37 on 1/13/20.
//

#include <gtest/gtest.h>
#include <dynamics/autorally/ar_nn_model.cuh>
#include <dynamics/autorally/ar_nn_dynamics_kernel_test.cuh>
#include <stdio.h>
#include <math.h>

TEST(ARNeuralNetDynamics, verifyTemplateParamters) {
  int state_dim = NeuralNetModel<7,2,3,6,32,32,4>::STATE_DIM;
  EXPECT_EQ(state_dim, 7);

  int control_dim = NeuralNetModel<7,2,3,6,32,32,4>::CONTROL_DIM;
  EXPECT_EQ(control_dim, 2);

  int dynamics_dim = NeuralNetModel<7,2,3,6,32,32,4>::DYNAMICS_DIM;
  EXPECT_EQ(dynamics_dim, 7-3);

  int num_layers = NeuralNetModel<7,2,3,6,32,32,4>::NUM_LAYERS;
  EXPECT_EQ(num_layers, 4);

  int prime_padding = NeuralNetModel<7,2,3,6,32,32,4>::PRIME_PADDING;
  EXPECT_EQ(prime_padding, 1);

  int largest_layer = NeuralNetModel<7,2,3,6,32,32,4>::LARGEST_LAYER;
  EXPECT_EQ(largest_layer, 32+1);

  int num_params = NeuralNetModel<7,2,3,6,32,32,4>::NUM_PARAMS;
  EXPECT_EQ(num_params, (6+1)*32+(32+1)*32+(32+1)*4);

  int shared_mem_request_grd = NeuralNetModel<7,2,3,6,32,32,4>::SHARED_MEM_REQUEST_GRD;
  EXPECT_EQ(shared_mem_request_grd, 0);

  int shared_mem_request_blk = NeuralNetModel<7,2,3,6,32,32,4>::SHARED_MEM_REQUEST_BLK;
  EXPECT_EQ(shared_mem_request_blk, (32+1)*2);

  NeuralNetModel<7,2,3,6,32,32,4> model(0.1);
  std::array<int, 4> net_structure = model.getNetStructure();

  EXPECT_EQ(net_structure[0], 6);
  EXPECT_EQ(net_structure[1], 32);
  EXPECT_EQ(net_structure[2], 32);
  EXPECT_EQ(net_structure[3], 4);
}

TEST(ARNeuralNetDynamics, BindStreamControlRanges) {
  hipStream_t stream;

  HANDLE_ERROR(hipStreamCreate(&stream));

  std::array<float2, 2> u_constraint = {};
  u_constraint[0].x = -1.0;
  u_constraint[0].y = 1.0;

  u_constraint[1].x = -2.0;
  u_constraint[1].y = 2.0;
  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt, u_constraint, stream);

  EXPECT_EQ(model.stream_, stream) << "Stream binding failure.";

  HANDLE_ERROR(hipStreamDestroy(stream));
}


TEST(ARNeuralNetDynamics, BindStreamDefaultArgRanges) {
  hipStream_t stream;

  HANDLE_ERROR(hipStreamCreate(&stream));

  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt, stream);

  EXPECT_EQ(model.stream_, stream) << "Stream binding failure.";

  HANDLE_ERROR(hipStreamDestroy(stream));
}

TEST(ARNeuralNetDynamics, ControlRangesSetDefaultCPU) {
  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt);

 std::array<float2, 2> ranges = model.getControlRanges();
 EXPECT_FLOAT_EQ(model.dt_, 0.1);
 for(int i = 0; i < 2; i++) {
   EXPECT_FLOAT_EQ(ranges[0].x, -FLT_MAX);
   EXPECT_FLOAT_EQ(ranges[0].y, FLT_MAX);
 }
}


TEST(ARNeuralNetDynamics, ControlRangesSetCPU) {
  std::array<float2, 2> u_constraint = {};
  u_constraint[0].x = -1.0;
  u_constraint[0].y = 1.0;

  u_constraint[1].x = -2.0;
  u_constraint[1].y = 2.0;
  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt, u_constraint);

  EXPECT_FLOAT_EQ(model.dt_, 0.1);

  std::array<float2, 2> ranges = model.getControlRanges();
  EXPECT_FLOAT_EQ(ranges[0].x, -1.0);
  EXPECT_FLOAT_EQ(ranges[0].y, 1.0);

  EXPECT_FLOAT_EQ(ranges[1].x, -2.0);
  EXPECT_FLOAT_EQ(ranges[1].y, 2.0);
}

TEST(ARNeuralNetDynamics, stideIdcsSetDefault) {
  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt);

  std::array<int, 6> result = model.getStideIdcs();

  EXPECT_EQ(result[0], 0);
  EXPECT_EQ(result[1], 192);
  EXPECT_EQ(result[2], 224);
  EXPECT_EQ(result[3], 1248);
  EXPECT_EQ(result[4], 1280);
  EXPECT_EQ(result[5], 1408);
}

TEST(ARNeuralNetDynamics, GPUSetupAndParamsCheck) {
  NeuralNetModel<7,2,3,6,32,32,4> model(0.1);

  std::array<float, 1412> theta = model.getTheta();
  std::array<int, 6> stride = model.getStideIdcs();
  std::array<int, 4> net_structure = model.getNetStructure();

  std::array<float, 1412> theta_result = {};
  std::array<int, 6> stride_result = {};
  std::array<int, 4> net_structure_result = {};

  EXPECT_EQ(model.GPUMemStatus_, false);
  EXPECT_EQ(model.model_d_, nullptr);

  model.GPUSetup();

  EXPECT_EQ(model.GPUMemStatus_, true);
  EXPECT_NE(model.model_d_, nullptr);

  //launch kernel
  launchParameterCheckTestKernel<NeuralNetModel<7,2,3,6,32,32,4>, 1412, 6, 4>
          (model, theta_result, stride_result, net_structure_result);

  for(int i = 0; i < 1412; i++) {
    // these are a bunch of mostly random values and nan != nan
    if(!isnan(theta[i])) {
      EXPECT_FLOAT_EQ(theta_result[i], theta[i]);
    }
  }
  for(int i = 0; i < 6; i++) {
    EXPECT_EQ(stride_result[i], stride[i]);
  }

  for(int i = 0; i < 4; i++) {
    EXPECT_EQ(net_structure[i], net_structure_result[i]);
  }
}

TEST(ARNeuralNetDynamics, UpdateModelTest) {
  NeuralNetModel<7,2,3,6,32,32,4> model(0.1);

  std::array<float, 1412> theta = model.getTheta();
  std::array<int, 6> stride = model.getStideIdcs();
  std::array<int, 4> net_structure = model.getNetStructure();

  std::array<float, 1412> theta_result = {};
  std::array<int, 6> stride_result = {};
  std::array<int, 4> net_structure_result = {};

  model.GPUSetup();

  std::vector<float> theta_vec(1412);
  srand (time(NULL));
  for(int i = 0; i < 1412; i++) {
    theta_vec[i] = rand();
  }

  model.updateModel({6, 32, 32, 4}, theta_vec);

  // check CPU
  for(int i = 0; i < 1412; i++) {
    // these are a bunch of mostly random values and nan != nan
    if(!isnan(theta_vec[i])) {
      EXPECT_FLOAT_EQ(model.getTheta()[i], theta_vec[i]);
    }
  }

  //launch kernel
  launchParameterCheckTestKernel<NeuralNetModel<7,2,3,6,32,32,4>, 1412, 6, 4>
          (model, theta_result, stride_result, net_structure_result);

  for(int i = 0; i < 1412; i++) {
    // these are a bunch of mostly random values and nan != nan
    if(!isnan(theta_vec[i])) {
      EXPECT_FLOAT_EQ(theta_result[i], theta_vec[i]) << "failed at index " << i;
    }
  }
  for(int i = 0; i < 6; i++) {
    EXPECT_EQ(stride_result[i], stride[i]);
  }

  for(int i = 0; i < 4; i++) {
    EXPECT_EQ(net_structure[i], net_structure_result[i]);
  }
}

TEST(ARNeuralNetDynamics, LoadModelTest) {
  NeuralNetModel<7,2,3,6,32,32,4> model(0.1);
  model.GPUSetup();

  // TODO procedurally generate a NN in python and save and run like costs
  std::string path = "/home/mgandhi3/git/MPPI-Generic/test_nn.npz";
  model.loadParams(path);

  // check CPU
  for(int i = 0; i < 1412; i++) {
    EXPECT_FLOAT_EQ(model.getTheta()[i], i) << "failed at index " << i;
  }

  std::array<float, 1412> theta_result = {};
  std::array<int, 6> stride_result = {};
  std::array<int, 4> net_structure_result = {};

  //launch kernel
  launchParameterCheckTestKernel<NeuralNetModel<7,2,3,6,32,32,4>, 1412, 6, 4>
          (model, theta_result, stride_result, net_structure_result);

  for(int i = 0; i < 1412; i++) {
    EXPECT_FLOAT_EQ(theta_result[i], i) << "failed at index " << i;
  }
}

TEST(ARNeuralNetDynamics, enforceConstraintsTest) {
  std::array<float2, 2> u_constraint = {};
  u_constraint[0].x = -1.0;
  u_constraint[0].y = 1.0;

  u_constraint[1].x = -2.0;
  u_constraint[1].y = 2.0;
  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt, u_constraint);

  float s[7];
  float u[2];

  u[0] = 10;
  u[1] = 1000;

  model.enforceConstraints(s, u);

  EXPECT_FLOAT_EQ(u[0], 1);
  EXPECT_FLOAT_EQ(u[1], 2);

  u[0] = -124;
  u[1] = -512789;

  model.enforceConstraints(s, u);

  EXPECT_FLOAT_EQ(u[0], -1);
  EXPECT_FLOAT_EQ(u[1], -2);

  u[0] = 0.5;
  u[1] = 1;

  model.enforceConstraints(s, u);

  EXPECT_FLOAT_EQ(u[0], 0.5);
  EXPECT_FLOAT_EQ(u[1], 1.0);
}

TEST(ARNeuralNetDynamics, computeKinematicsTest) {
  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt);

  float s[7];
  // x_dot, y_dot, theta_dot
  float s_der[3];

  s[2] = 0.0; // yaw
  s[4] = 1.0; // body frame vx
  s[5] = 2.0; // body frame vy
  s[6] = 0.0; // yaw dot

  model.computeKinematics(s, s_der);

  EXPECT_FLOAT_EQ(s_der[0], 1.0);
  EXPECT_FLOAT_EQ(s_der[1], 2.0);
  EXPECT_FLOAT_EQ(s_der[2], 0.0);

  s[2] = M_PI/2; // yaw
  s[4] = 3.0; // body frame vx
  s[5] = 5.0; // body frame vy
  s[6] = 1.0; // yaw dot

  model.computeKinematics(s, s_der);

  EXPECT_FLOAT_EQ(s_der[0], -5);
  EXPECT_FLOAT_EQ(s_der[1], 3.0);
  EXPECT_FLOAT_EQ(s_der[2], -1.0);
}

TEST(ARNeuralNetDynamics, incrementState) {
  float dt = 0.1;
  NeuralNetModel<7,2,3,6,32,32,4> model(dt);

  std::array<float, 7> s = {0.0};
  // x_dot, y_dot, theta_dot
  std::array<float, 7> s_der = {0.0};

  s[2] = 0.0; // yaw
  s[4] = 1.0; // body frame vx
  s[5] = 2.0; // body frame vy
  s[6] = 0.0; // yaw dot

  s_der[0] = 1.0;
  s_der[1] = 2.0;
  s_der[2] = 3.0;

  model.GPUSetup();

  launchIncrementStateTestKernel<NeuralNetModel<7,2,3,6,32,32,4>, 1, 7>(model, s, s_der);

  EXPECT_FLOAT_EQ(s_der[0], 0);
  EXPECT_FLOAT_EQ(s_der[1], 0);
  EXPECT_FLOAT_EQ(s_der[2], 0);

  EXPECT_FLOAT_EQ(s[0], 0.1);
  EXPECT_FLOAT_EQ(s[1], 0.2);
  EXPECT_FLOAT_EQ(s[2], 0.3);
}

TEST(ARNeuralNetDynamics, computeDynamicsTest) {

}
