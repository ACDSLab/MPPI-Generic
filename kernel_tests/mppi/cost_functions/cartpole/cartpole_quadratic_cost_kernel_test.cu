#include "hip/hip_runtime.h"
__global__ void parameterTestKernel(CartpoleQuadraticCost* cost_d, CartpoleQuadraticCostParams& params_d) {
    // The parameters have been set outside of the kernel on the device, copy the current values of the parameters to params_d
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid == 0) {
        params_d = cost_d->getParams();
    }
}

void launchParameterTestKernel(const CartpoleQuadraticCost& cost, CartpoleQuadraticCostParams& param_check) {
    // Allocate memory for the device side parameter structure
    CartpoleQuadraticCostParams* param_d = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&param_d, sizeof(CartpoleQuadraticCostParams)))

    parameterTestKernel<<<1,1>>>(cost.cost_d_, *param_d);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(&param_check, param_d, sizeof(param_check), hipMemcpyDeviceToHost))

    hipFree(param_d);
}


