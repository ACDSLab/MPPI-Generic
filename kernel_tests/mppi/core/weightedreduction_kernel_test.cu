#include "hip/hip_runtime.h"
#include "weightedreduction_kernel_test.cuh"

__global__ void setInitialControlToZero_KernelTest(int control_dim, float* u_d, float* u_intermediate) {
    int thread_idx = blockDim.x*blockIdx.x + threadIdx.x;
    mppi_common::setInitialControlToZero(control_dim, thread_idx, u_d, u_intermediate);
}

template<int num_threads, int control_dim>
void launchSetInitialControlToZero_KernelTest(std::array<float, control_dim>& u_host,
                                              std::array<float, num_threads*control_dim>& u_intermediate_host) {

    float* u_dev;
    float* u_intermediate_dev;

    // Allocate Memory
    HANDLE_ERROR(hipMalloc((void**)&u_dev, sizeof(float)*u_host.size()))
    HANDLE_ERROR(hipMalloc((void**)&u_intermediate_dev, sizeof(float)*u_intermediate_host.size()))

    setInitialControlToZero_KernelTest<<<1,num_threads>>>(control_dim, u_dev, u_intermediate_dev);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(u_host.data(), u_dev, sizeof(float)*u_host.size(), hipMemcpyDeviceToHost))
    HANDLE_ERROR(hipMemcpy(u_intermediate_host.data(), u_intermediate_dev, sizeof(float)*u_intermediate_host.size(), hipMemcpyDeviceToHost))

    hipFree(u_dev);
    hipFree(u_intermediate_dev);
}

template<int control_dim>
__global__ void strideControlWeightReduction_KernelTest(int num_rollouts, int num_timesteps, int sum_stride,
        float* exp_costs_d, float normalizer, float* du_d, float* u_intermediate) {
    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;

    float u_thread[control_dim];
    float* u_intermediate_thread = &u_intermediate[block_idx*control_dim*((num_rollouts - 1) / sum_stride + 1)];

    mppi_common::strideControlWeightReduction(num_rollouts, num_timesteps, sum_stride,
            thread_idx, block_idx, control_dim, exp_costs_d, normalizer, du_d,
                                              u_thread, u_intermediate_thread);
}

template<int control_dim, int num_rollouts, int num_timesteps, int sum_stride>
void launchStrideControlWeightReduction_KernelTest(float normalizer,
                                                   const std::array<float, num_rollouts>& exp_costs_host,
                                                   const std::array<float, num_rollouts*num_timesteps*control_dim>& du_host,
                                                   std::array<float, num_timesteps*control_dim*((num_rollouts - 1) / sum_stride + 1)>& u_intermediate_host) {
    float* exp_costs_dev;
    float* du_dev;
    float* u_intermediate_dev;

    // Allocate Memory
    HANDLE_ERROR(hipMalloc((void**)&exp_costs_dev, sizeof(float)*exp_costs_host.size()));
    HANDLE_ERROR(hipMalloc((void**)&du_dev, sizeof(float)*du_host.size()))
    HANDLE_ERROR(hipMalloc((void**)&u_intermediate_dev, sizeof(float)*u_intermediate_host.size()));

    HANDLE_ERROR(hipMemcpy(exp_costs_dev, exp_costs_host.data(), sizeof(float)*exp_costs_host.size(), hipMemcpyHostToDevice))
    HANDLE_ERROR(hipMemcpy(du_dev, du_host.data(), sizeof(float)*du_host.size(), hipMemcpyHostToDevice))

    dim3 blockdim((num_rollouts - 1) / sum_stride + 1, 1, 1);
    dim3 griddim(num_timesteps, 1, 1);

    strideControlWeightReduction_KernelTest<control_dim><<<griddim,blockdim>>>(num_rollouts, num_timesteps, sum_stride,
    exp_costs_dev, normalizer, du_dev, u_intermediate_dev);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(u_intermediate_host.data(), u_intermediate_dev, sizeof(float)*u_intermediate_host.size(), hipMemcpyDeviceToHost))

    hipFree(exp_costs_dev);
    hipFree(du_dev);
    hipFree(u_intermediate_dev);
}

template<int control_dim>
__global__ void rolloutWeightReductionAndSaveControl_KernelTest(int num_rollouts, int num_timesteps, int sum_stride,
        float* u_intermediate, float* du_new_d) {
    int thread_idx = threadIdx.x; // Current cell
    int block_idx = blockIdx.x; // Current timestep

    float u[control_dim];
    float* u_intermediate_thread = &u_intermediate[block_idx*control_dim*((num_rollouts - 1) / sum_stride + 1)];
    mppi_common::rolloutWeightReductionAndSaveControl(thread_idx, block_idx, num_rollouts, num_timesteps, control_dim, sum_stride,
            u, u_intermediate_thread, du_new_d);
}

template<int control_dim, int num_rollouts, int num_timesteps, int sum_stride>
void launchRolloutWeightReductionAndSaveControl_KernelTest(const std::array<float, num_timesteps*control_dim*((num_rollouts - 1) / sum_stride + 1)>& u_intermediate_host,
        std::array<float, num_timesteps*control_dim>& du_new_host) {
    float* u_intermediate_dev;
    float* du_new_dev;

    // Allocate Memory
    HANDLE_ERROR(hipMalloc((void**)&u_intermediate_dev, sizeof(float)*u_intermediate_host.size()))
    HANDLE_ERROR(hipMalloc((void**)&du_new_dev, sizeof(float)*du_new_host.size()))

    HANDLE_ERROR(hipMemcpy(u_intermediate_dev, u_intermediate_host.data(), sizeof(float)*u_intermediate_host.size(), hipMemcpyHostToDevice))
    dim3 blockdim((num_rollouts - 1) / sum_stride + 1, 1, 1);
    dim3 griddim(num_timesteps, 1, 1);

    rolloutWeightReductionAndSaveControl_KernelTest<control_dim><<<griddim, blockdim>>>(num_rollouts, num_timesteps,
            sum_stride, u_intermediate_dev, du_new_dev);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(du_new_host.data(), du_new_dev, sizeof(float)*du_new_host.size(), hipMemcpyDeviceToHost))

    hipFree(u_intermediate_dev);
    hipFree(du_new_dev);
}
