#include "hip/hip_runtime.h"
//
// Created by mgandhi on 5/23/20.
//

#include "rmppi_kernel_test.cuh"

const int BLOCKSIZE_X = 32;
const int BLOCKSIZE_Y = 8;

template<class DYN_T, class COST_T, class FB_T, int NUM_ROLLOUTS>
void launchRMPPIRolloutKernelGPU(DYN_T* dynamics, COST_T* costs,
                                 FB_T* fb_controller, float dt,
                                 int num_timesteps,
                                 int optimization_stride,
                                 float lambda,
                                 float alpha,
                                 float value_func_threshold,
                                 const std::vector<float>& x0_nom,
                                 const std::vector<float>& x0_act,
                                 const std::vector<float>& sigma_u,
                                 const std::vector<float>& nom_control_seq,
                                 const std::vector<float>& sampled_noise,
                                 std::array<float, NUM_ROLLOUTS>& trajectory_costs_act,
                                 std::array<float, NUM_ROLLOUTS>& trajectory_costs_nom,
                                 hipStream_t stream) {
  float* initial_state_d;
  float* trajectory_costs_d;
  float* control_noise_d; // du
  float* control_std_dev_d;
  float* control_d;

  /**
   * Ensure dynamics and costs exist on GPU
   */
  dynamics->bindToStream(stream);
  costs->bindToStream(stream);
  fb_controller->bindToStream(stream);
  // Call the GPU setup functions of the model and cost
  dynamics->GPUSetup();
  costs->GPUSetup();
  fb_controller->GPUSetup();

  int control_noise_size = NUM_ROLLOUTS * num_timesteps * DYN_T::CONTROL_DIM;
  // Create x init cuda array
  HANDLE_ERROR(hipMalloc((void**)&initial_state_d,
                          sizeof(float) * DYN_T::STATE_DIM * 2));
  // Create control variance cuda array
  HANDLE_ERROR(hipMalloc((void**)&control_std_dev_d,
                          sizeof(float) * DYN_T::CONTROL_DIM));
  // create control u trajectory cuda array
  HANDLE_ERROR(hipMalloc((void**)&control_d,
                          sizeof(float) * DYN_T::CONTROL_DIM *
                          num_timesteps * 2));
  // Create cost trajectory cuda array
  HANDLE_ERROR(hipMalloc((void**)&trajectory_costs_d,
                          sizeof(float) * NUM_ROLLOUTS * 2));
  // Create zero-mean noise cuda array
  HANDLE_ERROR(hipMalloc((void**)&control_noise_d,
                          sizeof(float) * DYN_T::CONTROL_DIM *
                          num_timesteps * NUM_ROLLOUTS * 2));
  // Create random noise generator
  // hiprandGenerator_t gen;
  // hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  /**
   * Fill in GPU arrays
   */
  HANDLE_ERROR(hipMemcpyAsync(initial_state_d, x0_act.data(),
                               sizeof(float) * DYN_T::STATE_DIM,
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(initial_state_d + DYN_T::STATE_DIM, x0_nom.data(),
                               sizeof(float) * DYN_T::STATE_DIM,
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(control_std_dev_d, sigma_u.data(),
                               sizeof(float) * DYN_T::CONTROL_DIM,
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(control_d, nom_control_seq.data(),
                               sizeof(float) * DYN_T::CONTROL_DIM * num_timesteps,
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(control_d + num_timesteps * DYN_T::CONTROL_DIM,
                               nom_control_seq.data(),
                               sizeof(float) * DYN_T::CONTROL_DIM * num_timesteps,
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(control_noise_d,
                               sampled_noise.data(),
                               sizeof(float) * control_noise_size,
                               hipMemcpyHostToDevice, stream));

 HANDLE_ERROR(hipMemcpyAsync(control_noise_d + control_noise_size,
                               sampled_noise.data(),
                               sizeof(float) * control_noise_size,
                               hipMemcpyHostToDevice, stream));

  // hiprandGenerateNormal(gen, control_noise_d, control_noise_size, 0.0, 1.0);
  // HANDLE_ERROR(hipMemcpyAsync(control_noise_d + control_noise_size,
  //                              control_noise_d,
  //                              control_noise_size * sizeof(float),
  //                              hipMemcpyDeviceToDevice, stream));
  // Ensure copying finishes?
  HANDLE_ERROR(hipStreamSynchronize(stream));
  // Launch rollout kernel
  rmppi_kernels::launchRMPPIRolloutKernel<DYN_T, COST_T, typename FB_T::TEMPLATED_GPU_FEEDBACK, NUM_ROLLOUTS, BLOCKSIZE_X,
    BLOCKSIZE_Y, 2>(dynamics->model_d_, costs->cost_d_, fb_controller->getDevicePointer(), dt, num_timesteps,
                    optimization_stride, lambda, alpha, value_func_threshold, initial_state_d, control_d,
                    control_noise_d, control_std_dev_d,
                    trajectory_costs_d, stream);


  HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_act.data(),
                               trajectory_costs_d,
                               NUM_ROLLOUTS * sizeof(float),
                               hipMemcpyDeviceToHost, stream));

  HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nom.data(),
                               trajectory_costs_d + NUM_ROLLOUTS,
                               NUM_ROLLOUTS * sizeof(float),
                               hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));

  hipFree(initial_state_d);
  hipFree(control_std_dev_d);
  hipFree(control_d);
  hipFree(trajectory_costs_d);
  hipFree(control_noise_d);
}

template<class DYN_T, class COST_T, class FB_T, int NUM_ROLLOUTS>
void launchRMPPIRolloutKernelCPU(DYN_T* model, COST_T* costs,
                                 FB_T* fb_controller,
                                 float dt,
                                 int num_timesteps,
                                 int optimization_stride,
                                 float lambda,
                                 float alpha,
                                 float value_func_threshold,
                                 const std::vector<float>& x0_nom,
                                 const std::vector<float>& x0_act,
                                 const std::vector<float>& sigma_u,
                                 const std::vector<float>& nom_control_seq,
                                 std::vector<float>& sampled_noise,
                                 std::array<float, NUM_ROLLOUTS>& trajectory_costs_act,
                                 std::array<float, NUM_ROLLOUTS>& trajectory_costs_nom) {
  // Crate Eigen items
  const int state_dim = DYN_T::STATE_DIM;
  const int control_dim = DYN_T::CONTROL_DIM;
  using control_matrix = typename COST_T::control_matrix;
  using control_array = typename DYN_T::control_array;
  using state_array = typename DYN_T::state_array;
  using feedback_matrix = typename DYN_T::feedback_matrix;
  Eigen::Map<const state_array> x_init_nom(x0_nom.data());
  Eigen::Map<const state_array> x_init_act(x0_act.data());

  control_array cost_std_dev;
  for(int i = 0; i < control_dim; i++) {
    cost_std_dev(i) = sigma_u[i];
  }

  int control_traj_size = NUM_ROLLOUTS * num_timesteps * control_dim;

  // Start rollouts
  for (int traj_i = 0; traj_i < NUM_ROLLOUTS; traj_i++)  {
    float cost_real_w_tracking = 0; // S^(V, x_0, x*_0) in Grady Thesis (8.24)
    float state_cost_nom = 0; // S(V, x*_0)
    float running_state_cost_real = 0;
    float running_control_cost_real = 0;
    int crash_status_nom[1] = {0};
    int crash_status_act[1] = {0};

    int traj_index = traj_i * num_timesteps;

    // Get all relevant values at time t in rollout i
    state_array x_t_nom = x_init_nom;
    state_array x_t_act = x_init_act;

    for (int t = 0; t < num_timesteps; t++){
      // Controls are read only so I can use Eigen::Map<const...>
      Eigen::Map<const control_array>
          u_t(nom_control_seq.data() + t * control_dim); // trajectory u at time t
      Eigen::Map<control_array>
          pure_noise_act(sampled_noise.data() + (traj_index + t) * control_dim); // Noise at time t
      Eigen::Map<control_array>
          pure_noise_nom(sampled_noise.data() + control_traj_size +
                         (traj_index + t) * control_dim); // ptr to noise for nominal
      control_array eps_t = cost_std_dev.cwiseProduct(pure_noise_act);

      // Create newly calculated values at time t in rollout i
      state_array x_dot_t_nom;
      state_array x_dot_t_act;
      control_array u_nom;
      if (traj_i == 0 || t < optimization_stride) {
        eps_t = control_array::Zero();
        u_nom = u_t;
      } else if (traj_i >= 0.99 * NUM_ROLLOUTS) {
        u_nom = eps_t;
      } else {
         u_nom = u_t + eps_t;
      }

      control_array fb_u_t = fb_controller->k(x_t_act, x_t_nom, t);
      control_array u_act = u_nom + fb_u_t;

      // Cost update
      if (t > 0) {
        control_array zero_u = control_array::Zero();
        state_cost_nom += costs->computeStateCost(x_t_nom, t, crash_status_nom);
        float state_cost_act = costs->computeStateCost(x_t_act, t,
                                                       crash_status_act);
        cost_real_w_tracking += state_cost_act +
                                costs->computeFeedbackCost(fb_u_t, cost_std_dev,
                                                           lambda, alpha);

        running_state_cost_real += state_cost_act;
        running_control_cost_real +=
                costs->computeLikelihoodRatioCost(u_t + fb_u_t, eps_t,
                                                  cost_std_dev, lambda, alpha);
      }
      model->enforceConstraints(x_t_nom, u_nom);
      model->enforceConstraints(x_t_act, u_act);

      // Dyanamics Update
      model->computeStateDeriv(x_t_nom, u_nom, x_dot_t_nom);
      model->computeStateDeriv(x_t_act, u_act, x_dot_t_act);

      model->updateState(x_t_act, x_dot_t_act, dt);
      model->updateState(x_t_nom, x_dot_t_nom, dt);
    }

    // Compute average cost per timestep
    state_cost_nom /= ((float)num_timesteps-1);
    cost_real_w_tracking /= ((float)num_timesteps-1);
    running_state_cost_real /= ((float)num_timesteps-1);

    state_cost_nom += costs->terminalCost(x_t_nom);
    cost_real_w_tracking += costs->terminalCost(x_t_act);
    running_state_cost_real += costs->terminalCost(x_t_act);

    float cost_nom = 0.5 * state_cost_nom + 0.5 *
      std::max(std::min(cost_real_w_tracking, value_func_threshold), state_cost_nom);
    // Figure out control costs for the nominal trajectory
    float cost_nom_control = 0;
    for (int t = 1; t < num_timesteps; t++) {
      Eigen::Map<const control_array>
          u_nom(nom_control_seq.data() + t * control_dim); // trajectory u at time t
      Eigen::Map<const control_array>
          pure_noise(sampled_noise.data() + (traj_index + t) * control_dim); // Noise at time t
      control_array eps_t = cost_std_dev.cwiseProduct(pure_noise);
      control_array u_t = u_nom;
      if (traj_i == 0 || t < optimization_stride) {
        eps_t = control_array::Zero();
      } else if (traj_i >= 0.99 * NUM_ROLLOUTS) {
        u_t = control_array::Zero();;
      }
      cost_nom_control += costs->computeLikelihoodRatioCost(u_t, eps_t, cost_std_dev, lambda, alpha);
    }

    // Compute average cost per timestep
    cost_nom_control /= ((float)num_timesteps-1);
    running_control_cost_real /= ((float)num_timesteps-1);

    cost_nom += cost_nom_control;
    trajectory_costs_nom[traj_i] = cost_nom;
    trajectory_costs_act[traj_i] = running_state_cost_real + running_control_cost_real;
  }
}

template<class DYNAMICS_T, class COSTS_T, class FB_T, int NUM_ROLLOUTS, int NUM_TIMESTEPS, int BLOCKSIZE_X, int BLOCKSIZE_Y>
void launchComparisonRolloutKernelTest(DYNAMICS_T* dynamics, COSTS_T* costs, FB_T* fb_controller,
                                       float dt, float lambda, float alpha,
                                       std::array<float, DYNAMICS_T::STATE_DIM> state_array,
                                       std::array<float, DYNAMICS_T::STATE_DIM> state_array_nominal,
                                       std::array<float, NUM_TIMESTEPS*DYNAMICS_T::CONTROL_DIM> control_array,
                                       std::array<float, NUM_TIMESTEPS*NUM_ROLLOUTS*DYNAMICS_T::CONTROL_DIM> control_noise_array,
                                       std::array<float, DYNAMICS_T::CONTROL_DIM> sigma_u,
                                       std::array<float, 2*NUM_ROLLOUTS>& rmppi_costs_out,
                                       std::array<float, NUM_ROLLOUTS>& mppi_costs_out,
                                       int opt_delay, hipStream_t stream) {

  /*************************** MPPI ******************************************/
  float* state_d;
  float* U_d;
  float* du_d;
  float* nu_d;
  float* costs_d;

  // Allocate CUDA memory for the rollout
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float)*state_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&U_d, sizeof(float)*control_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&du_d, sizeof(float)*DYNAMICS_T::CONTROL_DIM*NUM_TIMESTEPS*NUM_ROLLOUTS));
  HANDLE_ERROR(hipMalloc((void**)&nu_d, sizeof(float)*sigma_u.size()));
  HANDLE_ERROR(hipMalloc((void**)&costs_d, sizeof(float)*mppi_costs_out.size()));

  // Copy the initial values
  HANDLE_ERROR(hipMemcpyAsync(state_d, state_array.data(),
                               sizeof(float) * state_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(U_d, control_array.data(),
                               sizeof(float) * control_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(du_d, control_noise_array.data(),
                               sizeof(float) * control_noise_array.size(), hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(nu_d, sigma_u.data(),
                               sizeof(float) * sigma_u.size(), hipMemcpyHostToDevice, stream));

  const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
  dim3 dimGrid(gridsize_x, 1, 1);
  mppi_common::rolloutKernel<DYNAMICS_T, COSTS_T, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS, 1>
  <<<dimGrid, dimBlock, 0, stream>>>
                           (dynamics->model_d_, costs->cost_d_, dt, NUM_TIMESTEPS, opt_delay, lambda, alpha, state_d, U_d, du_d, nu_d, costs_d);
  CudaCheckError();

  // Copy data back
  HANDLE_ERROR(hipMemcpyAsync(mppi_costs_out.data(), costs_d,
                               sizeof(float) * mppi_costs_out.size(), hipMemcpyDeviceToHost, stream));

  // Deallocate CUDA Memory
  HANDLE_ERROR(hipFree(state_d));
  HANDLE_ERROR(hipFree(U_d));
  HANDLE_ERROR(hipFree(du_d));
  HANDLE_ERROR(hipFree(nu_d));
  HANDLE_ERROR(hipFree(costs_d));

  /*************************** RMPPI ******************************************/

  // Allocate CUDA memory for the rollout
  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float)*2*state_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&U_d, sizeof(float)*2*control_array.size()));
  HANDLE_ERROR(hipMalloc((void**)&du_d, sizeof(float)*2*DYNAMICS_T::CONTROL_DIM*NUM_TIMESTEPS*NUM_ROLLOUTS));
  HANDLE_ERROR(hipMalloc((void**)&nu_d, sizeof(float)*sigma_u.size()));
  HANDLE_ERROR(hipMalloc((void**)&costs_d, sizeof(float)*rmppi_costs_out.size()));


  // Copy the initial values
  HANDLE_ERROR(hipMemcpyAsync(state_d, state_array.data(),
                               sizeof(float) * state_array.size(),
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(state_d + DYNAMICS_T::STATE_DIM, state_array_nominal.data(),
                               sizeof(float) * DYNAMICS_T::STATE_DIM,
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(nu_d, sigma_u.data(),
                               sizeof(float) * sigma_u.size(),
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(U_d, control_array.data(),
                               sizeof(float) * DYNAMICS_T::CONTROL_DIM * NUM_TIMESTEPS,
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(U_d + NUM_TIMESTEPS * DYNAMICS_T::CONTROL_DIM,
                               control_array.data(),
                               sizeof(float) * DYNAMICS_T::CONTROL_DIM * NUM_TIMESTEPS,
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(du_d, control_noise_array.data(),
                               sizeof(float) * control_noise_array.size(),
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(du_d + control_noise_array.size(),
                               control_noise_array.data(),
                               sizeof(float) * control_noise_array.size(),
                               hipMemcpyHostToDevice, stream));

  dimBlock = dim3(BLOCKSIZE_X, BLOCKSIZE_Y, 2);
  dimGrid = dim3(gridsize_x, 1, 1);

  rmppi_kernels::RMPPIRolloutKernel<DYNAMICS_T, COSTS_T, typename FB_T::TEMPLATED_GPU_FEEDBACK, BLOCKSIZE_X, BLOCKSIZE_Y, NUM_ROLLOUTS,
          2><<<dimGrid, dimBlock, 0, stream>>>(
                  dynamics->model_d_, costs->cost_d_, fb_controller->getDevicePointer(), dt, NUM_TIMESTEPS,
                  opt_delay, lambda, alpha, 10, state_d, U_d, du_d,  nu_d,
                  costs_d);

  // Copy data back
  HANDLE_ERROR(hipMemcpyAsync(rmppi_costs_out.data(), costs_d,
                               sizeof(float) * rmppi_costs_out.size(), hipMemcpyDeviceToHost, stream));

  // Deallocate CUDA Memory
  HANDLE_ERROR(hipFree(state_d));
  HANDLE_ERROR(hipFree(U_d));
  HANDLE_ERROR(hipFree(du_d));
  HANDLE_ERROR(hipFree(nu_d));
  HANDLE_ERROR(hipFree(costs_d));
}


template<class DYN_T, class COST_T, int NUM_TIMESTEPS, int NUM_ROLLOUTS>
void launchRMPPIRolloutKernelCCMCPU(DYN_T* model, COST_T* costs,
                                    ccm::LinearCCM<DYN_T, NUM_TIMESTEPS>* fb_controller,
                                    float dt,
                                    int num_timesteps,
                                    int optimization_stride,
                                    float lambda,
                                    float alpha,
                                    float value_func_threshold,
                                    const std::vector<float>& x0_nom,
                                    const std::vector<float>& x0_act,
                                    const std::vector<float>& sigma_u,
                                    const std::vector<float>& nom_control_seq,
                                    std::vector<float>& sampled_noise,
                                    std::array<float, NUM_ROLLOUTS>& trajectory_costs_act,
                                    std::array<float, NUM_ROLLOUTS>& trajectory_costs_nom) {
  // Crate Eigen items
  const int state_dim = DYN_T::STATE_DIM;
  const int control_dim = DYN_T::CONTROL_DIM;
  using control_matrix = typename COST_T::control_matrix;
  using control_array = typename DYN_T::control_array;
  using state_array = typename DYN_T::state_array;
  using feedback_matrix = typename DYN_T::feedback_matrix;
  using dfdx = typename DYN_T::dfdx;
  Eigen::Map<const state_array> x_init_nom(x0_nom.data());
  Eigen::Map<const state_array> x_init_act(x0_act.data());

  // CCM Initialization
  // ccm::Vectorf<7> pts, weights;
  // std::tie(pts, weights) = ccm::chebyshevPts<7>();
  // auto CCM_Controller = ccm::LinearCCM<DYN_T>(model);
  // dfdx M_new = dfdx::Identity();
  // M_new(3,3) = 0.01;
  // M_new(2,2) = 0.01;
  // CCM_Controller.setM(M_new);

  control_array cost_std_dev;
  for(int i = 0; i < control_dim; i++) {
    cost_std_dev(i) = sigma_u[i];
  }

  int control_traj_size = NUM_ROLLOUTS * num_timesteps * control_dim;

  // Start rollouts
  for (int traj_i = 0; traj_i < NUM_ROLLOUTS; traj_i++)  {
    float cost_real_w_tracking = 0; // S^(V, x_0, x*_0) in Grady Thesis (8.24)
    float state_cost_nom = 0; // S(V, x*_0)
    float running_state_cost_real = 0;
    float running_control_cost_real = 0;
    float running_control_cost_nom = 0;

    int traj_index = traj_i * num_timesteps;

    // Get all relevant values at time t in rollout i
    state_array x_t_nom = x_init_nom;
    state_array x_t_act = x_init_act;

    for (int t = 0; t < num_timesteps; t++){
      // Controls are read only so I can use Eigen::Map<const...>
      Eigen::Map<const control_array>
          u_t(nom_control_seq.data() + t * control_dim); // trajectory u at time t
      /**
       * Get the noise at time t for nominal and actual systems
       * Note that they are assumed to be the same
       * They both need to exist to save different controls to
       * later on.
       */
      Eigen::Map<control_array>
          pure_noise_act(sampled_noise.data() + (traj_index + t) * control_dim); // Noise at time t
      Eigen::Map<control_array>
          pure_noise_nom(sampled_noise.data() + control_traj_size +
                         (traj_index + t) * control_dim); // ptr to noise for nominal
      control_array eps_t = cost_std_dev.cwiseProduct(pure_noise_act);
      // Eigen::Map<const feedback_matrix>
      //     feedback_gains_t(feedback_gains_seq.data() + t * control_dim * state_dim); // Feedback gains at time t

      // Create newly calculated values at time t in rollout i
      state_array x_dot_t_nom;
      state_array x_dot_t_act;
      control_array u_nom;
      if (traj_i == 0 || t < optimization_stride) {
        eps_t = control_array::Zero();
        u_nom = u_t;
      } else if (traj_i >= 0.99 * NUM_ROLLOUTS) {
        u_nom = eps_t;
      } else {
         u_nom = u_t + eps_t;
      }
      bool debug = false;
      // if (traj_i == 0) {
      //   debug = true;
      // }
      // control_array fb_u_t = feedback_gains_t * (x_t_act - x_t_nom);
      control_array fb_u_t = fb_controller->u_feedback(x_t_act, x_t_nom, u_nom, debug);
      if (traj_i == -1) {
        std::cout << "Feedback at t = " << t << ": " << fb_u_t.transpose() << std::endl;
        std::cout << "\tu_nominl: " << u_nom.transpose() << std::endl;
        std::cout << "\tx_actual: " << x_t_act.transpose() << std::endl;
        std::cout << "\tx_nominl: " << x_t_nom.transpose() << std::endl;
        std::cout << std::endl;
      }

      control_array u_act = u_nom + fb_u_t;


      model->enforceConstraints(x_t_nom, u_nom);
      model->enforceConstraints(x_t_act, u_act);

      /**
       * Copy controls back into noise vecotrs
       * This is where the noise pointing to different locations
       * for nominal and actual matter.
       */
      pure_noise_act = u_act;
      pure_noise_nom = u_nom;

      // Cost update
      state_cost_nom += costs->computeStateCost(x_t_nom);
      float state_cost_act = costs->computeStateCost(x_t_act);
      cost_real_w_tracking += state_cost_act +
                              costs->computeFeedbackCost(fb_u_t, cost_std_dev,
                                                         lambda, alpha);

      running_state_cost_real += state_cost_act;
      running_control_cost_real +=
        costs->computeLikelihoodRatioCost(u_nom - eps_t + fb_u_t, eps_t, cost_std_dev,
                                          lambda, alpha);
      running_control_cost_nom +=
        costs->computeLikelihoodRatioCost(u_nom - eps_t, eps_t, cost_std_dev, lambda, alpha);

      // Dyanamics Update
      model->computeStateDeriv(x_t_nom, u_nom, x_dot_t_nom);
      model->computeStateDeriv(x_t_act, u_act, x_dot_t_act);

      model->updateState(x_t_act, x_dot_t_act, dt);
      model->updateState(x_t_nom, x_dot_t_nom, dt);
    }

    // Compute average cost per timestep
    state_cost_nom /= ((float)num_timesteps-1);
    cost_real_w_tracking /= ((float)num_timesteps-1);
    running_state_cost_real /= ((float)num_timesteps-1);

    state_cost_nom += costs->terminalCost(x_t_nom);
    cost_real_w_tracking += costs->terminalCost(x_t_act);
    running_state_cost_real += costs->terminalCost(x_t_act);

    float cost_nom = 0.5 * state_cost_nom + 0.5 *
      std::max(std::min(cost_real_w_tracking, value_func_threshold), state_cost_nom);
    // Figure out control costs for the nominal trajectory
    // float cost_nom_control = 0;
    // for (int t = 0; t < num_timesteps - 1; t++) {
    //   Eigen::Map<const control_array>
    //       u_nom(nom_control_seq.data() + t * control_dim); // trajectory u at time t
    //   Eigen::Map<const control_array>
    //       eps_t(sampled_noise.data() + control_traj_size +
    //                  (traj_index + t) * control_dim); // U + noise Noise at time t
    //   // control_array eps_t = cost_std_dev.cwiseProduct(pure_noise);
    //   // control_array u_t = u_nom;
    //   // if (traj_i == 0) {
    //   //   eps_t = control_array::Zero();
    //   // } else if (traj_i >= 0.99 * NUM_ROLLOUTS) {
    //   //   u_t = control_array::Zero();;
    //   // }
    //   cost_nom_control += costs->computeLikelihoodRatioCost(u_nom, eps_t - u_nom, cost_std_dev,
    //                                                         lambda, alpha);
    // }
    // Compute average cost per timestep

    running_control_cost_nom /= (float) (num_timesteps - 1);
    running_control_cost_real /= (float)(num_timesteps - 1);

    cost_nom += running_control_cost_nom;
    trajectory_costs_nom[traj_i] = cost_nom;
    trajectory_costs_act[traj_i] = running_state_cost_real + running_control_cost_real;
  }
}
