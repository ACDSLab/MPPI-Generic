#include "normexp_kernel_test.cuh"

template<int NUM_ROLLOUTS>
void launchNormExp_KernelTest(std::array<float, NUM_ROLLOUTS>& trajectory_costs_host, float gamma, float baseline, std::array<float, NUM_ROLLOUTS>& normalized_compute) {
    // Allocate CUDA memory
    float* trajectory_costs_d;
    HANDLE_ERROR(hipMalloc((void**)&trajectory_costs_d, sizeof(float)*trajectory_costs_host.size()))

    HANDLE_ERROR(hipMemcpy(trajectory_costs_d, trajectory_costs_host.data(), sizeof(float)*trajectory_costs_host.size(), hipMemcpyHostToDevice))

    mppi_common::normExpKernel<<<1,NUM_ROLLOUTS>>>(NUM_ROLLOUTS, trajectory_costs_d, gamma, baseline);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(normalized_compute.data(), trajectory_costs_d, sizeof(float)*trajectory_costs_host.size(), hipMemcpyDeviceToHost))

    hipFree(trajectory_costs_d);
}

template<int NUM_ROLLOUTS, int BLOCKSIZE_X>
void launchGenericNormExpKernelTest(std::array<float, NUM_ROLLOUTS> trajectory_costs_host,
                                  float gamma, float baseline, std::array<float, NUM_ROLLOUTS>& normalized_compute) {
  // Allocate CUDA memory
  float* trajectory_costs_d;
  HANDLE_ERROR(hipMalloc((void**)&trajectory_costs_d, sizeof(float)*trajectory_costs_host.size()));

  HANDLE_ERROR(hipMemcpy(trajectory_costs_d, trajectory_costs_host.data(), sizeof(float)*trajectory_costs_host.size(), hipMemcpyHostToDevice));

  dim3 dimBlock(BLOCKSIZE_X, 1, 1);
  dim3 dimGrid((NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1, 1, 1);

  mppi_common::normExpKernel<<<dimGrid,dimBlock>>>(NUM_ROLLOUTS, trajectory_costs_d, gamma, baseline);
  CudaCheckError();

  HANDLE_ERROR(hipMemcpy(normalized_compute.data(), trajectory_costs_d, sizeof(float)*trajectory_costs_host.size(), hipMemcpyDeviceToHost));

  hipFree(trajectory_costs_d);
}