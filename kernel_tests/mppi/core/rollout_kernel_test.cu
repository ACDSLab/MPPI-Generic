#include "hip/hip_runtime.h"
#include "rollout_kernel_test.cuh"

#include <mppi/dynamics/cartpole/cartpole_dynamics.cuh>
#include <mppi/cost_functions/cartpole/cartpole_quadratic_cost.cuh>

const int STATE_DIM = 12;
const int CONTROL_DIM = 3;
const int NUM_ROLLOUTS = 100; // .99 times this number has to be an integer... TODO fix how brittle this is
const int BLOCKSIZE_X = 32;
const int BLOCKSIZE_Y = 8; // Blocksize_y has to be greater than the control dim TODO fix how we step through the controls

template<int BLOCKSIZE_Z>
__global__ void loadGlobalToShared_KernelTest(float* x0_device,
                                              float* sigma_u_device,
                                              float* x_thread_device,
                                              float* xdot_thread_device,
                                              float* u_thread_device,
                                              float* du_thread_device,
                                              float* sigma_u_thread_device) {
  int thread_idx = threadIdx.x;
  int thread_idy = threadIdx.y;
  int thread_idz = threadIdx.z;
  int block_idx = blockIdx.x;
  int global_idx = threadIdx.x + block_idx*blockDim.x;

  //Create shared arrays which hold state and control data
  __shared__ float x_shared[BLOCKSIZE_X * STATE_DIM * BLOCKSIZE_Z];
  __shared__ float xdot_shared[BLOCKSIZE_X * STATE_DIM * BLOCKSIZE_Z];
  __shared__ float u_shared[BLOCKSIZE_X * CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float du_shared[BLOCKSIZE_X * CONTROL_DIM * BLOCKSIZE_Z];
  __shared__ float sigma_u_thread[CONTROL_DIM];

  float* x_thread;
  float* xdot_thread;

  float* u_thread;
  float* du_thread;

  if (global_idx < NUM_ROLLOUTS) {
    x_thread = &x_shared[(blockDim.x * thread_idz + thread_idx) * STATE_DIM];
    xdot_thread = &xdot_shared[(blockDim.x * thread_idz + thread_idx) * STATE_DIM];
    u_thread = &u_shared[(blockDim.x * thread_idz + thread_idx) * CONTROL_DIM];
    du_thread = &du_shared[(blockDim.x * thread_idz + thread_idx) * CONTROL_DIM];
  }
  __syncthreads();
  mppi_common::loadGlobalToShared(STATE_DIM, CONTROL_DIM, NUM_ROLLOUTS,
                                  BLOCKSIZE_Y, global_idx,
                                  thread_idy, thread_idz,
                                  x0_device, sigma_u_device, x_thread,
                                  xdot_thread, u_thread, du_thread, sigma_u_thread);
  __syncthreads();

  // Check if on the first rollout the correct values were coped over
  // Prevent y threads from all writing to the same memory
  if (global_idx == 1 && thread_idy == 0) {
    for (int i = 0; i < STATE_DIM; ++i) {
      int ind = i + thread_idz * STATE_DIM;
      int ind_thread = i + thread_idz * STATE_DIM * blockDim.x;
      x_thread_device[ind] = x_shared[ind_thread];
      xdot_thread_device[ind] = xdot_shared[ind_thread];
    }

    for (int i = 0; i < CONTROL_DIM; ++i) {
      int ind = i + thread_idz * CONTROL_DIM;
      int ind_thread = i + thread_idz * CONTROL_DIM * blockDim.x;
      u_thread_device[ind] = u_shared[ind_thread];
      du_thread_device[ind] = du_shared[ind_thread];
      // There is only control_dim
      sigma_u_thread_device[i] = sigma_u_thread[i];
    }
    __syncthreads();
  }



  // To test what the results are, we have to return them back to the host.
}

void launchGlobalToShared_KernelTest(const std::vector<float>& x0_host,
                                     const std::vector<float>& u_var_host,
                                     std::vector<float>& x_thread_host,
                                     std::vector<float>& xdot_thread_host,
                                     std::vector<float>& u_thread_host,
                                     std::vector<float>& du_thread_host,
                                     std::vector<float>& sigma_u_thread_host) {

  // Define the initial condition x0_device and the exploration variance in global device memory
  float* x0_device;
  float* u_var_device;
  HANDLE_ERROR(hipMalloc((void**)&x0_device, sizeof(float)*STATE_DIM));
  HANDLE_ERROR(hipMalloc((void**)&u_var_device, sizeof(float)*CONTROL_DIM));

  HANDLE_ERROR(hipMemcpy(x0_device, x0_host.data(), sizeof(float)*STATE_DIM, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(u_var_device, u_var_host.data(), sizeof(float)*CONTROL_DIM, hipMemcpyHostToDevice));


  // Define the return arguments in global device memory
  float* x_thread_device;
  float* xdot_thread_device;
  float* u_thread_device;
  float* du_thread_device;
  float* sigma_u_thread_device;

  HANDLE_ERROR(hipMalloc((void**)&x_thread_device, sizeof(float)*STATE_DIM));
  HANDLE_ERROR(hipMalloc((void**)&xdot_thread_device, sizeof(float)*STATE_DIM));
  HANDLE_ERROR(hipMalloc((void**)&u_thread_device, sizeof(float)*CONTROL_DIM));
  HANDLE_ERROR(hipMalloc((void**)&du_thread_device, sizeof(float)*CONTROL_DIM));
  HANDLE_ERROR(hipMalloc((void**)&sigma_u_thread_device, sizeof(float)*CONTROL_DIM));

  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y);
  dim3 dimGrid(2048);

  loadGlobalToShared_KernelTest<<<dimGrid,dimBlock>>>(x0_device, u_var_device,
      x_thread_device, xdot_thread_device, u_thread_device, du_thread_device, sigma_u_thread_device);
  CudaCheckError();

  // Copy the data back to the host
  HANDLE_ERROR(hipMemcpy(x_thread_host.data(), x_thread_device, sizeof(float)*STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(xdot_thread_host.data(), xdot_thread_device, sizeof(float)*STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(u_thread_host.data(), u_thread_device, sizeof(float)*CONTROL_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(du_thread_host.data(), du_thread_device, sizeof(float)*CONTROL_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(sigma_u_thread_host.data(), sigma_u_thread_device, sizeof(float)*CONTROL_DIM, hipMemcpyDeviceToHost));

  // Free the cuda memory that we allocated
  hipFree(x0_device);
  hipFree(u_var_device);

  hipFree(x_thread_device);
  hipFree(xdot_thread_device);
  hipFree(u_thread_device);
  hipFree(du_thread_device);
  hipFree(sigma_u_thread_device);
}

/**
 * Tube-MPPI versions of the kernel tests
 */

// This is to test tube-mppi calls to the kernel
void launchGlobalToShared_KernelTest_nom_act(const std::vector<float>& x0_host_act,
                                             const std::vector<float>& u_var_host,
                                             std::vector<float>& x_thread_host_act,
                                             std::vector<float>& xdot_thread_host_act,
                                             std::vector<float>& u_thread_host_act,
                                             std::vector<float>& du_thread_host_act,
                                             const std::vector<float>& x0_host_nom,
                                             std::vector<float>& x_thread_host_nom,
                                             std::vector<float>& xdot_thread_host_nom,
                                             std::vector<float>& u_thread_host_nom,
                                             std::vector<float>& du_thread_host_nom,
                                             std::vector<float>& sigma_u_thread_host) {

  // Define the initial condition x0_device and the exploration variance in global device memory
  // Need twice as much memory for tube-mppi
  float* x0_device;
  float* u_var_device;
  HANDLE_ERROR(hipMalloc((void**)&x0_device, sizeof(float) * STATE_DIM * 2));
  HANDLE_ERROR(hipMalloc((void**)&u_var_device, sizeof(float) * CONTROL_DIM));

  // Copy both act and nominal initial state
  HANDLE_ERROR(hipMemcpy(x0_device, x0_host_act.data(), sizeof(float) * STATE_DIM, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(x0_device + STATE_DIM, x0_host_nom.data(), sizeof(float) * STATE_DIM, hipMemcpyHostToDevice));
  // Copy both act and nominal control variance
  HANDLE_ERROR(hipMemcpy(u_var_device, u_var_host.data(), sizeof(float) * CONTROL_DIM, hipMemcpyHostToDevice));


  // Define the return arguments in global device memory
  float* x_thread_device;
  float* xdot_thread_device;
  float* u_thread_device;
  float* du_thread_device;
  float* sigma_u_thread_device;

  HANDLE_ERROR(hipMalloc((void**)&x_thread_device, sizeof(float)*STATE_DIM * 2));
  HANDLE_ERROR(hipMalloc((void**)&xdot_thread_device, sizeof(float)*STATE_DIM * 2));
  HANDLE_ERROR(hipMalloc((void**)&u_thread_device, sizeof(float)*CONTROL_DIM * 2));
  HANDLE_ERROR(hipMalloc((void**)&du_thread_device, sizeof(float)*CONTROL_DIM * 2));
  HANDLE_ERROR(hipMalloc((void**)&sigma_u_thread_device, sizeof(float)*CONTROL_DIM));

  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 2);
  dim3 dimGrid(100);

  loadGlobalToShared_KernelTest<2><<<dimGrid,dimBlock>>>(x0_device, u_var_device,
      x_thread_device, xdot_thread_device, u_thread_device, du_thread_device,
      sigma_u_thread_device);
  CudaCheckError();

  // Copy the initial_state for actual and nominal
  HANDLE_ERROR(hipMemcpy(x_thread_host_act.data(), x_thread_device,
                          sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(x_thread_host_nom.data(), x_thread_device + STATE_DIM,
                          sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  // Copy the xdot for actual and nominal
  HANDLE_ERROR(hipMemcpy(xdot_thread_host_act.data(), xdot_thread_device,
                          sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(xdot_thread_host_nom.data(), xdot_thread_device + STATE_DIM,
                          sizeof(float) * STATE_DIM, hipMemcpyDeviceToHost));
  // copy the initial u for actual and nominal
  HANDLE_ERROR(hipMemcpy(u_thread_host_act.data(), u_thread_device,
                          sizeof(float) * CONTROL_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(u_thread_host_nom.data(), u_thread_device + CONTROL_DIM,
                          sizeof(float) * CONTROL_DIM, hipMemcpyDeviceToHost));
  // copy the du for actual and nominal
  HANDLE_ERROR(hipMemcpy(du_thread_host_act.data(), du_thread_device,
                          sizeof(float) * CONTROL_DIM, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(du_thread_host_nom.data(), du_thread_device + CONTROL_DIM,
                          sizeof(float) * CONTROL_DIM, hipMemcpyDeviceToHost));
  // copy the control variance for actual and nominal
  HANDLE_ERROR(hipMemcpy(sigma_u_thread_host.data(), sigma_u_thread_device,
                          sizeof(float) * CONTROL_DIM, hipMemcpyDeviceToHost));

  // Free the cuda memory that we allocated
  hipFree(x0_device);
  hipFree(u_var_device);

  hipFree(x_thread_device);
  hipFree(xdot_thread_device);
  hipFree(u_thread_device);
  hipFree(du_thread_device);
  hipFree(sigma_u_thread_device);
}

__global__ void  injectControlNoiseOnce_KernelTest(int num_rollouts, int num_timesteps, int timestep, float* u_traj_device, float* ep_v_device, float* sigma_u_device, float* control_compute_device) {
  int global_idx = threadIdx.x + blockDim.x*blockIdx.x;
  int thread_idy = threadIdx.y;
  float u_thread[CONTROL_DIM];
  float du_thread[CONTROL_DIM];

  if (global_idx < num_rollouts) {
    mppi_common::injectControlNoise(CONTROL_DIM, BLOCKSIZE_Y, num_rollouts, num_timesteps, timestep, global_idx, thread_idy, u_traj_device, ep_v_device, sigma_u_device, u_thread, du_thread);
    if (thread_idy < CONTROL_DIM) {
      control_compute_device[global_idx * CONTROL_DIM + thread_idy] = u_thread[thread_idy];
    }
  }
}

void launchInjectControlNoiseOnce_KernelTest(const std::vector<float>& u_traj_host,
                                             const int num_rollouts,
                                             const int num_timesteps,
                                             std::vector<float>& ep_v_host,
                                             std::vector<float>& sigma_u_host,
                                             std::vector<float>& control_compute) {

  // Timestep
  int timestep = 0;

  // Declare variables for device memory
  float* u_traj_device;
  float* ep_v_device;
  float* sigma_u_device;
  float* control_compute_device;

  // Allocate cuda memory
  HANDLE_ERROR(hipMalloc((void**)&u_traj_device, sizeof(float)*u_traj_host.size()));
  HANDLE_ERROR(hipMalloc((void**)&ep_v_device, sizeof(float)*ep_v_host.size()));
  HANDLE_ERROR(hipMalloc((void**)&sigma_u_device, sizeof(float)*sigma_u_host.size()));
  HANDLE_ERROR(hipMalloc((void**)&control_compute_device, sizeof(float)*control_compute.size()));

  // Copy the control trajectory and the control variance to the device
  HANDLE_ERROR(hipMemcpy(u_traj_device, u_traj_host.data(), sizeof(float)*u_traj_host.size(), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(sigma_u_device, sigma_u_host.data(), sizeof(float)*sigma_u_host.size(), hipMemcpyHostToDevice));

  // Generate the noise
  hiprandGenerator_t gen_;
  hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen_, 1234ULL);
  hiprandGenerateNormal(gen_, ep_v_device, ep_v_host.size(), 0.0, 1.0);

  // Copy the noise back to the host
  HANDLE_ERROR(hipMemcpy(ep_v_host.data(), ep_v_device, sizeof(float)*ep_v_host.size(), hipMemcpyDeviceToHost));

  // Create the block and grid dimensions
  dim3 block_size(BLOCKSIZE_X, BLOCKSIZE_Y);
  dim3 grid_size(num_rollouts, 1);

  // Launch the test kernel
  injectControlNoiseOnce_KernelTest<<<grid_size,block_size>>>(num_rollouts, num_timesteps, timestep, u_traj_device, ep_v_device, sigma_u_device, control_compute_device);
  CudaCheckError();

  // Copy the result back to the host
  HANDLE_ERROR(hipMemcpy(control_compute.data(), control_compute_device, sizeof(float)*control_compute.size(), hipMemcpyDeviceToHost));

  // Free cuda memory
  hipFree(u_traj_device);
  hipFree(ep_v_device);
  hipFree(control_compute_device);
  hipFree(sigma_u_device);
  hiprandDestroyGenerator(gen_);
}

template<int control_dim, int blocksize_y>
__global__ void injectControlNoiseCheckControlV_KernelTest(int num_rollouts, int num_timesteps, int timestep,
    float* u_traj_device, float* ep_v_device, float* sigma_u_device) {
  int global_idx = threadIdx.x + blockDim.x*blockIdx.x;
  int thread_idy = threadIdx.y;
  float u_thread[control_dim];
  float du_thread[control_dim];

  if (global_idx < num_rollouts) {
    mppi_common::injectControlNoise(control_dim, blocksize_y, num_rollouts, num_timesteps, timestep, global_idx, thread_idy, u_traj_device, ep_v_device, sigma_u_device, u_thread, du_thread);
    __syncthreads();
  }
}

template<int num_rollouts, int num_timesteps, int control_dim, int blocksize_x, int blocksize_y, int gridsize_x>
void launchInjectControlNoiseCheckControlV_KernelTest(const std::array<float, num_timesteps*control_dim>& u_traj_host,
    std::array<float, num_rollouts*num_timesteps*control_dim>& ep_v_host, const std::array<float, control_dim>& sigma_u_host) {

  // Declare variables for device memory
  float* u_traj_device;
  float* ep_v_device;
  float* sigma_u_device;

  // Allocate cuda memory
  HANDLE_ERROR(hipMalloc((void**)&u_traj_device, sizeof(float)*u_traj_host.size()));
  HANDLE_ERROR(hipMalloc((void**)&ep_v_device, sizeof(float)*ep_v_host.size()));
  HANDLE_ERROR(hipMalloc((void**)&sigma_u_device, sizeof(float)*sigma_u_host.size()));

  // Copy to device
  HANDLE_ERROR(hipMemcpy(u_traj_device, u_traj_host.data(), sizeof(float)*u_traj_host.size(), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(ep_v_device, ep_v_host.data(), sizeof(float)*ep_v_host.size(), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(sigma_u_device, sigma_u_host.data(), sizeof(float)*sigma_u_host.size(), hipMemcpyHostToDevice));


  dim3 dimBlock(blocksize_x, blocksize_y, 1);
  dim3 dimGrid(gridsize_x, 1, 1);

  for (int i = 0; i < num_timesteps; ++i) {
    injectControlNoiseCheckControlV_KernelTest<control_dim, blocksize_y><<<dimGrid, dimBlock>>>(num_rollouts, num_timesteps, i, u_traj_device, ep_v_device, sigma_u_device);
    CudaCheckError();
  }

  // Copy to the host
  HANDLE_ERROR(hipMemcpy(ep_v_host.data(), ep_v_device, sizeof(float)*ep_v_host.size(), hipMemcpyDeviceToHost));


  hipFree(u_traj_device);
  hipFree(ep_v_device);
  hipFree(sigma_u_device);
}

template<class COST_T>
__global__ void computeAndSaveCostAllRollouts_KernelTest(COST_T* cost, int state_dim, int num_rollouts, float* running_costs, float* terminal_state, float* cost_rollout_device) {
  int tid = blockDim.x*blockIdx.x + threadIdx.x; // index on rollouts
//    if (tid == 0) {
//        printf("Current state [%f, %f, %f, %f]\n", terminal_state[state_dim * tid],
//               terminal_state[state_dim * tid + 1], terminal_state[state_dim * tid + 2],
//               terminal_state[state_dim * tid + 3]);
//        printf("Current cost [%f]\n", running_costs[tid]);
//    }
  mppi_common::computeAndSaveCost(num_rollouts, tid, cost, &terminal_state[state_dim*tid], running_costs[tid], cost_rollout_device);
//    if (tid == 0) {
//        printf("Total cost [%f]\n", cost_rollout_device[tid]);
//    }
}

template<class COST_T, int STATE_DIM, int NUM_ROLLOUTS>
void launchComputeAndSaveCostAllRollouts_KernelTest(COST_T cost,
    const std::array<float, NUM_ROLLOUTS>& cost_all_rollouts,
    const std::array<float, STATE_DIM*NUM_ROLLOUTS>& terminal_states,
    std::array<float, NUM_ROLLOUTS>& cost_compute) {

  // Allocate CUDA memory
  float* cost_all_rollouts_device;
  float* terminal_states_device;
  float* cost_compute_device;

  HANDLE_ERROR(hipMalloc((void**)&cost_all_rollouts_device, sizeof(float)*cost_all_rollouts.size()));
  HANDLE_ERROR(hipMalloc((void**)&terminal_states_device, sizeof(float)*terminal_states.size()));
  HANDLE_ERROR(hipMalloc((void**)&cost_compute_device, sizeof(float)*cost_compute.size()));

  // Copy Host side data to the Device
  HANDLE_ERROR(hipMemcpy(cost_all_rollouts_device, cost_all_rollouts.data(), sizeof(float)*cost_all_rollouts.size(), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(terminal_states_device, terminal_states.data(), sizeof(float)*terminal_states.size(), hipMemcpyHostToDevice));

  // Launch kernel
  dim3 blocksize(BLOCKSIZE_X, 1);
  dim3 gridsize((NUM_ROLLOUTS + (BLOCKSIZE_X - 1)) / BLOCKSIZE_X, 1);
  computeAndSaveCostAllRollouts_KernelTest<<<blocksize, gridsize>>>(cost.cost_d_, STATE_DIM, NUM_ROLLOUTS, cost_all_rollouts_device, terminal_states_device, cost_compute_device);
  CudaCheckError();

  // Copy Device side data to the host
  HANDLE_ERROR(hipMemcpy(cost_compute.data(), cost_compute_device, sizeof(float)*cost_compute.size(), hipMemcpyDeviceToHost));

  // free cuda Memory
  hipFree(cost_all_rollouts_device);
  hipFree(terminal_states_device);
  hipFree(cost_compute_device);
}

template<class DYN_T, class COST_T, int NUM_ROLLOUTS>
void launchRolloutKernel_nom_act(DYN_T* dynamics, COST_T* costs,
                                 float dt,
                                 int num_timesteps,
                                 const std::vector<float>& x0,
                                 const std::vector<float>& sigma_u,
                                 const std::vector<float>& nom_control_seq,
                                 std::vector<float>& trajectory_costs_act,
                                 std::vector<float>& trajectory_costs_nom,
                                 hipStream_t stream) {
  float * initial_state_d;
  float * trajectory_costs_d;
  float * control_noise_d; // du
  float * control_variance_d;
  float * control_d;

  /**
   * Ensure dynamics and costs exist on GPU
   */
  dynamics->bindToStream(stream);
  costs->bindToStream(stream);
  // Call the GPU setup functions of the model and cost
  dynamics->GPUSetup();
  costs->GPUSetup();

  int control_noise_size = NUM_ROLLOUTS * num_timesteps * DYN_T::CONTROL_DIM;
  // Create x init cuda array
  HANDLE_ERROR(hipMalloc((void**)&initial_state_d,
                          sizeof(float) * DYN_T::STATE_DIM * 2));
  // Create control variance cuda array
  HANDLE_ERROR(hipMalloc((void**)&control_variance_d,
                          sizeof(float) * DYN_T::CONTROL_DIM));
  // create control u trajectory cuda array
  HANDLE_ERROR(hipMalloc((void**)&control_d,
                          sizeof(float) * DYN_T::CONTROL_DIM *
                          num_timesteps * 2));
  // Create cost trajectory cuda array
  HANDLE_ERROR(hipMalloc((void**)&trajectory_costs_d,
                          sizeof(float) * NUM_ROLLOUTS * 2));
  // Create zero-mean noise cuda array
  HANDLE_ERROR(hipMalloc((void**)&control_noise_d,
                          sizeof(float) * DYN_T::CONTROL_DIM *
                          num_timesteps * NUM_ROLLOUTS * 2));
  // Create random noise generator
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  /**
   * Fill in GPU arrays
   */
  HANDLE_ERROR(hipMemcpyAsync(initial_state_d, x0.data(),
                               sizeof(float) * DYN_T::STATE_DIM,
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(initial_state_d + DYN_T::STATE_DIM, x0.data(),
                               sizeof(float) * DYN_T::STATE_DIM,
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(control_variance_d, sigma_u.data(),
                               sizeof(float) * DYN_T::CONTROL_DIM,
                               hipMemcpyHostToDevice, stream));

  HANDLE_ERROR(hipMemcpyAsync(control_d, nom_control_seq.data(),
                               sizeof(float) * DYN_T::CONTROL_DIM * num_timesteps,
                               hipMemcpyHostToDevice, stream));
  HANDLE_ERROR(hipMemcpyAsync(control_d + num_timesteps * DYN_T::CONTROL_DIM,
                               nom_control_seq.data(),
                               sizeof(float) * DYN_T::CONTROL_DIM * num_timesteps,
                               hipMemcpyHostToDevice, stream));

  hiprandGenerateNormal(gen, control_noise_d, control_noise_size, 0.0, 1.0);
  HANDLE_ERROR(hipMemcpyAsync(control_noise_d + control_noise_size,
                               control_noise_d,
                               control_noise_size * sizeof(float),
                               hipMemcpyDeviceToDevice, stream));
  // Ensure copying finishes?
  HANDLE_ERROR(hipStreamSynchronize(stream));
  // Launch rollout kernel
  mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BLOCKSIZE_X,
    BLOCKSIZE_Y, 2>(dynamics->model_d_, costs->cost_d_, dt, num_timesteps,
                    initial_state_d, control_d, control_noise_d,
                    control_variance_d, trajectory_costs_d, stream);

  // Copy the costs back to the host
  HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_act.data(),
                               trajectory_costs_d,
                               NUM_ROLLOUTS * sizeof(float),
                               hipMemcpyDeviceToHost, stream));

  HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nom.data(),
                               trajectory_costs_d + NUM_ROLLOUTS,
                               NUM_ROLLOUTS * sizeof(float),
                               hipMemcpyDeviceToHost, stream));
  HANDLE_ERROR(hipStreamSynchronize(stream));

  hipFree(initial_state_d);
  hipFree(control_variance_d);
  hipFree(control_d);
  hipFree(trajectory_costs_d);
  hipFree(control_noise_d);
}

/**
 * Cartpole Compute and Save cost all rollouts instantiations
 */
const int num_rollouts_cs = 1234;
template void launchComputeAndSaveCostAllRollouts_KernelTest<CartpoleQuadraticCost, CartpoleDynamics::STATE_DIM, num_rollouts_cs>(CartpoleQuadraticCost cost,
                          const std::array<float, num_rollouts_cs>& cost_all_rollouts,
                          const std::array<float, CartpoleDynamics::STATE_DIM*num_rollouts_cs>& terminal_states,
                          std::array<float, num_rollouts_cs>& cost_compute);
