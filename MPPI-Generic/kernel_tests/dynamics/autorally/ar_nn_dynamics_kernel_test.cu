#include "hip/hip_runtime.h"
template <class NETWORK_T, int THETA_SIZE, int STRIDE_SIZE, int NUM_LAYERS>
__global__ void parameterCheckTestKernel(NETWORK_T* model,  float* theta, int* stride, int* net_structure) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid == 0) {
    for(int i = 0; i < THETA_SIZE; i++) {
      theta[i] = model->getThetaPtr()[i];
    }
    for(int i = 0; i < STRIDE_SIZE; i++) {
      stride[i] = model->getStrideIdcsPtr()[i];
    }
    for(int i = 0; i < NUM_LAYERS; i++) {
      net_structure[i] = model->getNetStructurePtr()[i];
    }
  }
}

template<class NETWORK_T, int THETA_SIZE, int STRIDE_SIZE, int NUM_LAYERS>
void launchParameterCheckTestKernel(NETWORK_T& model, std::array<float, THETA_SIZE>& theta, std::array<int, STRIDE_SIZE>& stride,
                                    std::array<int, NUM_LAYERS>& net_structure) {
  float* theta_d;
  int* stride_d;
  int* net_structure_d;

  HANDLE_ERROR(hipMalloc((void**)&theta_d, sizeof(float)*theta.size()))
  HANDLE_ERROR(hipMalloc((void**)&stride_d, sizeof(int)*stride.size()))
  HANDLE_ERROR(hipMalloc((void**)&net_structure_d, sizeof(int)*net_structure.size()))

  dim3 threadsPerBlock(1, 1);
  dim3 numBlocks(1, 1);
  parameterCheckTestKernel<NETWORK_T, THETA_SIZE, STRIDE_SIZE, NUM_LAYERS><<<numBlocks,threadsPerBlock>>>(model.model_d_, theta_d, stride_d, net_structure_d);
  CudaCheckError();

  HANDLE_ERROR(hipMemcpy(theta.data(), theta_d, sizeof(float)*theta.size(), hipMemcpyDeviceToHost))
  HANDLE_ERROR(hipMemcpy(stride.data(), stride_d, sizeof(int)*stride.size(), hipMemcpyDeviceToHost))
  HANDLE_ERROR(hipMemcpy(net_structure.data(), net_structure_d, sizeof(int)*net_structure.size(), hipMemcpyDeviceToHost))
  hipDeviceSynchronize();

  hipFree(theta_d);
  hipFree(stride_d);
  hipFree(net_structure_d);
}


template<class NETWORK_T, int S_DIM, int C_DIM>
__global__ void fullARNNTestKernel(NETWORK_T* model, float* state, float* control, float* state_der, float dt) {
  __shared__ float theta[NETWORK_T::SHARED_MEM_REQUEST_GRD + NETWORK_T::SHARED_MEM_REQUEST_BLK];
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  // calls enforce constraints -> compute state derivative -> increment state

  //printf("before enforceConstraints %d, %d\n", threadIdx.x, threadIdx.y);
  //printf("enforceConstraints %d, %d\n", threadIdx.x, threadIdx.y);
  //printf("before enforce Constraints %f, %f\n", (control+(tid*C_DIM))[0], (control+(tid*C_DIM))[1]);
  model->enforceConstraints(state+(tid*S_DIM), control+(tid*C_DIM));
  //printf("after enforce Constraints %f, %f\n", (control+(tid*C_DIM))[0], (control+(tid*C_DIM))[1]);
  model->computeStateDeriv(state+(tid*S_DIM), control+(tid*C_DIM), state_der+(tid*S_DIM), theta);
  // TODO generalize
  model->updateState(state+(tid*S_DIM), state_der+(tid*S_DIM), dt);
}

template<class NETWORK_T, int S_DIM, int C_DIM>
void launchFullARNNTestKernel(NETWORK_T& model, std::vector< std::array<float, S_DIM>>& state,
                              std::vector< std::array<float, C_DIM>>& control, std::vector< std::array<float, S_DIM>>& state_der,
                              float dt, int dim_y) {
  float* state_d;
  float* state_der_d;
  float* control_d;

  HANDLE_ERROR(hipMalloc((void**)&state_d, sizeof(float)*S_DIM))
  HANDLE_ERROR(hipMalloc((void**)&state_der_d, sizeof(float)*S_DIM))
  HANDLE_ERROR(hipMalloc((void**)&control_d, sizeof(float)*C_DIM))

  HANDLE_ERROR(hipMemcpy(state_d, state.data(), sizeof(float)*S_DIM, hipMemcpyHostToDevice))
  HANDLE_ERROR(hipMemcpy(state_der_d, state_der.data(), sizeof(float)*S_DIM, hipMemcpyHostToDevice))
  HANDLE_ERROR(hipMemcpy(control_d, control.data(), sizeof(float)*C_DIM, hipMemcpyHostToDevice))

  // make sure you cannot use invalid inputs
  dim3 threadsPerBlock(1, dim_y);
  dim3 numBlocks(1, 1);
  // launch kernel
  fullARNNTestKernel<NETWORK_T, S_DIM, C_DIM><<<numBlocks,threadsPerBlock>>>(model.model_d_, state_d, control_d, state_der_d, dt);
  CudaCheckError();

  HANDLE_ERROR(hipMemcpy(state.data(), state_d, sizeof(float)*S_DIM, hipMemcpyDeviceToHost))
  HANDLE_ERROR(hipMemcpy(state_der.data(), state_der_d, sizeof(float)*S_DIM, hipMemcpyDeviceToHost))
  HANDLE_ERROR(hipMemcpy(control.data(), control_d, sizeof(float)*C_DIM, hipMemcpyDeviceToHost))
  hipDeviceSynchronize();

  hipFree(state_d);
  hipFree(state_der_d);
  hipFree(control_d);
}


