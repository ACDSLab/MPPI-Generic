#include "hip/hip_runtime.h"
#include "ar_nn_dynamics_kernel_test.cuh"


template <class NETWORK_T>
__global__ void parameterCheckTestKernel(NETWORK_T* model,  float* theta, int* stride, int* net_structure) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid == 0) {
    for(int i = 0; i < 1412; i++) {
      theta[i] = model->getThetaPtr()[i];
    }
    for(int i = 0; i < 6; i++) {
      stride[i] = model->getStrideIdcsPtr()[i];
    }
    for(int i = 0; i < 4; i++) {
      net_structure[i] = model->getNetStructurePtr()[i];
    }
  }
}

template<class NETWORK_T, int THETA_SIZE, int STRIDE_SIZE, int NUM_LAYERS>
void launchParameterCheckTestKernel(NETWORK_T& model, std::array<float, THETA_SIZE>& theta, std::array<int, STRIDE_SIZE>& stride,
                                    std::array<int, NUM_LAYERS>& net_structure) {
  float* theta_d;
  int* stride_d;
  int* net_structure_d;

  HANDLE_ERROR(hipMalloc((void**)&theta_d, sizeof(float)*theta.size()))
  HANDLE_ERROR(hipMalloc((void**)&stride_d, sizeof(int)*stride.size()))
  HANDLE_ERROR(hipMalloc((void**)&net_structure_d, sizeof(int)*net_structure.size()))

  dim3 threadsPerBlock(1, 1);
  dim3 numBlocks(1, 1);
  parameterCheckTestKernel<NETWORK_T><<<numBlocks,threadsPerBlock>>>(model.model_d_, theta_d, stride_d, net_structure_d);
  CudaCheckError();

  HANDLE_ERROR(hipMemcpy(theta.data(), theta_d, sizeof(float)*theta.size(), hipMemcpyDeviceToHost))
  HANDLE_ERROR(hipMemcpy(stride.data(), stride_d, sizeof(int)*stride.size(), hipMemcpyDeviceToHost))
  HANDLE_ERROR(hipMemcpy(net_structure.data(), net_structure_d, sizeof(int)*net_structure.size(), hipMemcpyDeviceToHost))
  hipDeviceSynchronize();

  hipFree(theta_d);
  hipFree(stride_d);
  hipFree(net_structure_d);
}


// explicit instantiation
template void launchParameterCheckTestKernel<NeuralNetModel<7,2,3,6,32,32,4>, 1412, 6, 4>(NeuralNetModel<7,2,3,6,32,32,4>& model, std::array<float, 1412>& theta, std::array<int, 6>& stride,
                                                                                          std::array<int, 4>& net_structure);

// explicit instantiation
template __global__ void parameterCheckTestKernel<NeuralNetModel<7,2,3,6,32,32,4>>(NeuralNetModel<7,2,3,6,32,32,4>* model,  float* theta, int* stride, int* net_structure);
