#include "hip/hip_runtime.h"
//
// Created by mgandhi3 on 1/7/20.
//
#include <kernel_tests/dynamics/cartpole/cartpole_kernel_test.cuh>

__global__ void CartMassTestKernel(Cartpole* CP, float& mass_check) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    // printf("\nEntering the kernel!\n");
    // printf("The thread id is: %i\n", tid);
    if (tid == 0) {
        // printf("This is gravity: %f\n", CP->getGravity());
        // printf("This is the mass of the cart: %f\n", CP->getCartMass());
        // printf("This is the mass of the pole: %f\n", CP->getPoleMass());
        // printf("This is the length of the pole: %f\n", CP->getPoleLength());
        // printf("This is the value of GPUMemstatus on the GPU: %d\n", CP->GPUMemStatus_);
        // printf("This is the value of CP_device on the GPU: %d\n", CP->CP_device);
        mass_check = CP->getCartMass();
    }
}

__global__ void DyanmicsTestKernel(Cartpole* CP, float* state, float* control, float* state_der) {
    // int tid = blockIdx.x*blockDim.x + threadIdx.x;
    /**
     * This will probably do stupid things because of parallelization
     * Fix later
     */
    CP->xDot(state, control, state_der);
}

void launchCartMassTestKernel(const Cartpole& CP, float& mass_check) {
    // Allocate memory on the CPU for checking the mass
    float* mass_check_device;
    HANDLE_ERROR(hipMalloc((void**)&mass_check_device, sizeof(float)));

    CartMassTestKernel<<<1,1>>>(CP.CP_device, *mass_check_device);
    CudaCheckError();

    // Copy the memory back to the host
    HANDLE_ERROR(hipMemcpy(&mass_check, mass_check_device, sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    hipFree(mass_check_device);
}

void launchDyanmicsTestKernel(const Cartpole& CP, float* state_cpu,
                              float* control_cpu, float* state_der_cpu) {
    // Allocate memory on the CPU for checking the mass
    float* state_gpu;
    float* control_gpu;
    float* state_der_gpu;

    HANDLE_ERROR(hipMalloc((void**)&state_gpu, sizeof(float) * CP.STATE_DIM));
    HANDLE_ERROR(hipMalloc((void**)&control_gpu, sizeof(float) * CP.CONTROL_DIM));
    HANDLE_ERROR(hipMalloc((void**)&state_der_gpu, sizeof(float) * CP.STATE_DIM));

    HANDLE_ERROR(hipMemcpy(state_gpu, state_cpu, sizeof(float)* CP.STATE_DIM, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(control_gpu, control_cpu, sizeof(float)* CP.CONTROL_DIM, hipMemcpyHostToDevice));

    DyanmicsTestKernel<<<1,1>>>(CP.CP_device, state_gpu, control_gpu, state_der_gpu);
    CudaCheckError();

    // Copy the memory back to the host
    HANDLE_ERROR(hipMemcpy(state_der_cpu, state_der_gpu, sizeof(float)* CP.STATE_DIM, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    hipFree(state_gpu);
    hipFree(control_gpu);
    hipFree(state_der_gpu);
}
