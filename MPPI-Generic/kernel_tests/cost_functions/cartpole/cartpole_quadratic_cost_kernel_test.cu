#include "hip/hip_runtime.h"
#include "cartpole_quadratic_cost_kernel_test.cuh"

__global__ void parameterTestKernel(CartPoleQuadraticCost* cost_d, CartPoleQuadraticCost::CartPoleQuadraticCostParams& params_d) {
    // The parameters have been set outside of the kernel on the device, copy the current values of the parameters to params_d
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid == 0) {
        params_d = cost_d->getParams();
    }
}

void launchParameterTestKernel(const CartPoleQuadraticCost& cost, CartPoleQuadraticCost::CartPoleQuadraticCostParams& param_check) {
    // Allocate memory for the device side parameter structure
    CartPoleQuadraticCost::CartPoleQuadraticCostParams* param_d = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&param_d, sizeof(CartPoleQuadraticCost::CartPoleQuadraticCostParams)))

    parameterTestKernel<<<1,1>>>(cost.cost_d_, *param_d);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(&param_check, param_d, sizeof(param_check), hipMemcpyDeviceToHost))

    hipFree(param_d);
}


