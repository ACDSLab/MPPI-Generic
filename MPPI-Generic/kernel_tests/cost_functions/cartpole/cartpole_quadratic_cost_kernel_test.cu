#include "hip/hip_runtime.h"
#include "cartpole_quadratic_cost_kernel_test.cuh"

__global__ void parameterTestKernel(CartpoleQuadraticCost* cost_d, CartpoleQuadraticCost::Params& params_d) {
    // The parameters have been set outside of the kernel on the device, copy the current values of the parameters to params_d
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid == 0) {
        params_d = cost_d->getParams();
    }
}

void launchParameterTestKernel(const CartpoleQuadraticCost& cost, CartpoleQuadraticCost::Params& param_check) {
    // Allocate memory for the device side parameter structure
    CartpoleQuadraticCost::Params* param_d = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&param_d, sizeof(CartpoleQuadraticCost::Params)))

    parameterTestKernel<<<1,1>>>(cost.cost_d_, *param_d);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(&param_check, param_d, sizeof(param_check), hipMemcpyDeviceToHost))

    hipFree(param_d);
}


