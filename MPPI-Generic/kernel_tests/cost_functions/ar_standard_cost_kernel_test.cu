#include "hip/hip_runtime.h"
#include <cost_functions/autorally/ar_standard_cost.cuh>

__global__ void ParameterTestKernel(ARStandardCost* cost, ARStandardCost::ARStandardCostParams& params, int& width, int& height) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  //printf("\nEntering the kernel!\n");
  //printf("The thread id is: %i\n", tid);
  if (tid == 0) {
    //printf("")
    params.desired_speed = cost->getParams().desired_speed;
    params.speed_coeff = cost->getParams().speed_coeff;
    params.track_coeff = cost->getParams().track_coeff;
    params.max_slip_ang = cost->getParams().max_slip_ang;
    params.slip_penalty = cost->getParams().slip_penalty;
    params.track_slop = cost->getParams().track_slop;
    params.crash_coeff = cost->getParams().crash_coeff;
    params.steering_coeff = cost->getParams().steering_coeff;
    params.throttle_coeff = cost->getParams().throttle_coeff;
    params.boundary_threshold = cost->getParams().boundary_threshold;
    params.discount = cost->getParams().discount;
    params.num_timesteps = cost->getParams().num_timesteps;
    params.grid_res = cost->getParams().grid_res;

    params.r_c1 = cost->getParams().r_c1;
    params.r_c2 = cost->getParams().r_c2;
    params.trs = cost->getParams().trs;

    width = cost->getWidth();
    height = cost->getHeight();
  }
}

void launchParameterTestKernel(const ARStandardCost& cost, ARStandardCost::ARStandardCostParams& params, int& width, int& height) {
  // Allocate memory on the CPU for checking the mass
  ARStandardCost::ARStandardCostParams* params_d;
  int* width_d;
  int* height_d;
  HANDLE_ERROR(hipMalloc((void**)&params_d, sizeof(ARStandardCost::ARStandardCostParams)))
  HANDLE_ERROR(hipMalloc((void**)&width_d, sizeof(float)))
  HANDLE_ERROR(hipMalloc((void**)&height_d, sizeof(float)))

  ParameterTestKernel<<<1,1>>>(cost.cost_d_, *params_d, *width_d, *height_d);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(&params, params_d, sizeof(ARStandardCost::ARStandardCostParams), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(&width, width_d, sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(&height, height_d, sizeof(float), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(params_d);
}

// TODO actually check texture
__global__ void checkCudaArrayKernel(float4* result_arr, hipArray* array, int number) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  //printf("\nEntering the kernel!\n");
  //printf("The thread id is: %i\n", tid);
  if(tid < number) {
    //printf("The thread id is: %i\n", tid);
    result_arr[tid].x = 0.0;
    result_arr[tid].y = 0.0;
    result_arr[tid].z = 0.0;
    result_arr[tid].w = 0.0;
    //result_arr[tid] = array[tid];
    //printf(array[tid]);
  }
}

void launchCheckCudaArray(std::vector<float4>& result_arr, hipArray* array, int number) {
  float4* results_d;
  HANDLE_ERROR(hipMalloc((void**)&results_d, sizeof(float4)*number));

  result_arr.resize(number);

  dim3 threadsPerBlock(4, 1);
  dim3 numBlocks(1, 1);
  checkCudaArrayKernel<<<numBlocks,threadsPerBlock>>>(results_d, array, number);
  CudaCheckError();
  hipDeviceSynchronize();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(result_arr.data(), results_d, sizeof(float4)*number, hipMemcpyDeviceToHost));

  hipFree(results_d);
}

__global__ void transformTestKernel(float3* results, ARStandardCost* cost) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  //printf("\nEntering the kernel!\n");
  //printf("The thread id is: %i\n", tid);
  if (tid == 0) {
    //printf("")
    results[0] = cost->getParams().r_c1;
    results[1] = cost->getParams().r_c2;
    results[2] = cost->getParams().trs;
  }
}

void launchTransformTestKernel(std::vector<float3>& result, const ARStandardCost& cost) {
  result.resize(3);

  // Allocate memory on the CPU for checking the mass
  float3* results_d;
  HANDLE_ERROR(hipMalloc((void**)&results_d, sizeof(float3) * 3))

  transformTestKernel<<<1,1>>>(results_d, cost.cost_d_);
  CudaCheckError();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(result.data(), results_d, sizeof(float3)*3, hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  hipFree(results_d);
}



__global__ void textureTestKernel(const ARStandardCost& cost, float4* test_results, float2* test_indexes, int num_points) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  //printf("\nEntering the kernel!\n");
  //printf("The thread id is: %i\n", tid);
  if(tid < num_points) {
    //printf("thread id: %i went to check texture at index %i, %i\n", tid, test_indexes[tid].x, test_indexes[tid].y);

    // query texture
    float4 track_params_back = cost.queryTexture(test_indexes[tid].x, test_indexes[tid].y);
    // put result in array
    //printf("thread id: %i got texture point (%f, %f, %f, %f)\n", tid, track_params_back.x, track_params_back.y, track_params_back.z, track_params_back.w);
    test_results[tid] = track_params_back;
    //test_results[tid].x = 1;
  }
}

void launchTextureTestKernel(const ARStandardCost& cost, std::vector<float4>& test_results, std::vector<float2>& test_indexes) {
  int num_test_points = test_indexes.size();
  test_results.resize(num_test_points);

  float4* tex_results_d;
  float2* tex_test_indexes_d;
  HANDLE_ERROR(hipMalloc((void**)&tex_results_d, sizeof(float4)*num_test_points))
  HANDLE_ERROR(hipMalloc((void**)&tex_test_indexes_d, sizeof(float2)*num_test_points))

  HANDLE_ERROR(hipMemcpy(tex_test_indexes_d, test_indexes.data(), sizeof(float2)*num_test_points, hipMemcpyHostToDevice));

  // TODO amount should depend on the number of query points
  dim3 threadsPerBlock(num_test_points, 1);
  dim3 numBlocks(1, 1);
  textureTestKernel<<<numBlocks,threadsPerBlock>>>(*cost.cost_d_, tex_results_d, tex_test_indexes_d, num_test_points);
  CudaCheckError();
  hipDeviceSynchronize();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(test_results.data(), tex_results_d, sizeof(float4)*num_test_points, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  hipFree(tex_results_d);
  hipFree(tex_test_indexes_d);
}

__global__ void textureTransformTestKernel(ARStandardCost& cost, float4* test_results, float2* test_indexes, int num_points) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  //printf("\nEntering the kernel!\n");
  //printf("The thread id is: %i\n", tid);
  if(tid < num_points) {
    // query texture
    float4 track_params_back = cost.queryTextureTransformed(test_indexes[tid].x, test_indexes[tid].y);
    // put result in array
    test_results[tid] = track_params_back;
    //test_results[tid].x = 1;
  }
}

void launchTextureTransformTestKernel(const ARStandardCost& cost, std::vector<float4>& test_results, std::vector<float2>& test_indexes) {
  int num_test_points = test_indexes.size();
  test_results.resize(num_test_points);

  float4* tex_results_d;
  float2* tex_test_indexes_d;
  HANDLE_ERROR(hipMalloc((void**)&tex_results_d, sizeof(float4)*num_test_points))
  HANDLE_ERROR(hipMalloc((void**)&tex_test_indexes_d, sizeof(float2)*num_test_points))

  HANDLE_ERROR(hipMemcpy(tex_test_indexes_d, test_indexes.data(), sizeof(float2)*num_test_points, hipMemcpyHostToDevice));

  // TODO amount should depend on the number of query points
  dim3 threadsPerBlock(num_test_points, 1);
  dim3 numBlocks(1, 1);
  textureTransformTestKernel<<<numBlocks,threadsPerBlock>>>(*cost.cost_d_, tex_results_d, tex_test_indexes_d, num_test_points);
  CudaCheckError();
  hipDeviceSynchronize();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(test_results.data(), tex_results_d, sizeof(float4)*num_test_points, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  hipFree(tex_results_d);
  hipFree(tex_test_indexes_d);
}

__global__ void trackCostTestKernel(ARStandardCost& cost, float3* test_indexes, int num_points,
                                    float* cost_results, int* crash_results) {

  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid < num_points) {
    float state[7];
    int crash = 0;
    state[0] = test_indexes[tid].x;
    state[1] = test_indexes[tid].y;
    state[2] = test_indexes[tid].z;
    printf("got test indexes %d, state %f, %f, %f\n", tid, state[0], state[1], state[2]);
    cost_results[tid] = cost.getTrackCost(state, &crash);
    printf("set results %d\n", tid);
    crash_results[tid] = crash;
    printf("set crash results %d\n", tid);
  }
}

void launchTrackCostTestKernel(const ARStandardCost& cost, std::vector<float3>& test_indexes,
                               std::vector<float>& cost_results, std::vector<int>& crash_results) {

  int num_test_points = test_indexes.size();
  crash_results.resize(num_test_points);
  cost_results.resize(num_test_points);

  float* cost_results_d;
  int* crash_results_d;
  float3* test_indexes_d;
  HANDLE_ERROR(hipMalloc((void**)&cost_results_d, sizeof(float)*num_test_points))
  HANDLE_ERROR(hipMalloc((void**)&crash_results_d, sizeof(int)*num_test_points))
  HANDLE_ERROR(hipMalloc((void**)&test_indexes_d, sizeof(float3)*num_test_points))

  HANDLE_ERROR(hipMemcpy(test_indexes_d, test_indexes.data(), sizeof(float3)*num_test_points, hipMemcpyHostToDevice));

  // TODO amount should depend on the number of query points
  dim3 threadsPerBlock(num_test_points, 1);
  dim3 numBlocks(1, 1);
  trackCostTestKernel<<<numBlocks,threadsPerBlock>>>(*cost.cost_d_, test_indexes_d, num_test_points, cost_results_d, crash_results_d);
  CudaCheckError();
  hipDeviceSynchronize();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(cost_results.data(), cost_results_d, sizeof(float)*num_test_points, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(crash_results.data(), crash_results_d, sizeof(int)*num_test_points, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  hipFree(cost_results_d);
  hipFree(crash_results_d);
  hipFree(test_indexes_d);
}

__global__ void computeCostTestKernel(ARStandardCost& cost, float* test_xu, float* cost_results, int num_points) {

  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid < num_points) {
    float* state = &test_xu[tid];
    float* control = &test_xu[tid+7];
    int crash = 0;
    float vars[2] = {1,1};
    float du[2] = {0,0};
    cost_results[tid] = cost.computeCost(state, control, du, vars, &crash, tid);
  }
}

void launchComputeCostTestKernel(const ARStandardCost& cost, std::vector<std::array<float, 9>>& test_xu, std::vector<float>& cost_results) {

  int num_test_points = test_xu.size();
  cost_results.resize(num_test_points*9);

  float* cost_results_d;
  float* test_xu_d;
  HANDLE_ERROR(hipMalloc((void**)&cost_results_d, sizeof(float)*num_test_points))
  HANDLE_ERROR(hipMalloc((void**)&test_xu_d, sizeof(float)*9*num_test_points))

  for(int i = 0; i < num_test_points; i++) {
    for(int j = 0; j < 9; j++) {
      cost_results[9*i+j] = test_xu[i][j];
    }
  }

  HANDLE_ERROR(hipMemcpy(test_xu_d, test_xu.data(), sizeof(float)*9*num_test_points, hipMemcpyHostToDevice));

  // TODO amount should depend on the number of query points
  dim3 threadsPerBlock(num_test_points, 1);
  dim3 numBlocks(1, 1);
  computeCostTestKernel<<<numBlocks,threadsPerBlock>>>(*cost.cost_d_, test_xu_d, cost_results_d, num_test_points);
  CudaCheckError();
  hipDeviceSynchronize();

  // Copy the memory back to the host
  HANDLE_ERROR(hipMemcpy(cost_results.data(), cost_results_d, sizeof(float)*num_test_points, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  hipFree(cost_results_d);
  hipFree(test_xu_d);
}

