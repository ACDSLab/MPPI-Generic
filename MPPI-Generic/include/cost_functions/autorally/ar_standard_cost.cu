#include <cost_functions/autorally/ar_standard_cost.cuh>

ARStandardCost::ARStandardCost(int width, int height, hipStream_t stream) {
  this->width_ = width;
  this->height_ = height;

  bindToStream(stream);
}

ARStandardCost::~ARStandardCost() {

}

void ARStandardCost::GPUSetup() {
  if (!GPUMemStatus_) {
    cost_device_ = Managed::GPUSetup(this);
  } else {
    std::cout << "GPU Memory already set." << std::endl;
  }
  // load track data
  // update transform
  // update params
  // allocate texture memory
  // convert costmap to texture
  paramsToDevice();
}

void ARStandardCost::freeCudaMem() {
  hipFree(cost_device_);
}

void ARStandardCost::paramsToDevice() {
  HANDLE_ERROR( hipMemcpyAsync(&cost_device_->params_, &params_, sizeof(ARStandardCostParams), hipMemcpyHostToDevice, stream_));
  HANDLE_ERROR( hipMemcpyAsync(&cost_device_->width_, &width_, sizeof(float), hipMemcpyHostToDevice, stream_));
  HANDLE_ERROR( hipMemcpyAsync(&cost_device_->height_, &height_, sizeof(float), hipMemcpyHostToDevice, stream_));
  HANDLE_ERROR( hipStreamSynchronize(stream_));
}

