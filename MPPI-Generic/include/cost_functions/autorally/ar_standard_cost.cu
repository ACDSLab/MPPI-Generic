#include "hip/hip_runtime.h"
#include <cost_functions/autorally/ar_standard_cost.cuh>

ARStandardCost::ARStandardCost(hipStream_t stream) {

  bindToStream(stream);
}

ARStandardCost::~ARStandardCost() {
  if(GPUMemStatus_) {
    freeCudaMem();
  }
}

void ARStandardCost::setParams(ARStandardCostParams params) {
  this->params_ = params;
  if(GPUMemStatus_) {
    paramsToDevice();
  }
}

void ARStandardCost::GPUSetup() {
  if (!GPUMemStatus_) {
    cost_d_ = Managed::GPUSetup(this);
  } else {
    std::cout << "GPU Memory already set." << std::endl;
  }
  // load track data
  // update transform
  // update params
  // allocate texture memory
  // convert costmap to texture
  paramsToDevice();
}

void ARStandardCost::freeCudaMem() {
  // TODO free everything
  hipFree(cost_d_);
}

void ARStandardCost::paramsToDevice() {
  HANDLE_ERROR( hipMemcpyAsync(&cost_d_->params_, &params_, sizeof(ARStandardCostParams), hipMemcpyHostToDevice, stream_));
  HANDLE_ERROR( hipMemcpyAsync(&cost_d_->width_, &width_, sizeof(float), hipMemcpyHostToDevice, stream_));
  HANDLE_ERROR( hipMemcpyAsync(&cost_d_->height_, &height_, sizeof(float), hipMemcpyHostToDevice, stream_));
  HANDLE_ERROR( hipStreamSynchronize(stream_));
}

bool ARStandardCost::changeCostmapSize(int width, int height) {
  // TODO set flag at top that indicates memory allocation changes
  if(height < 0 && width < 0) {
    std::cerr << "ERROR: cannot resize costmap to size less than 1" << std::endl;
    return false;
  }
  if(height != height_ || width != width_) {
    track_costs_.resize(width * height);

    //Allocate memory for the cuda array which is bound the costmap_tex_
    // has been allocated in the past, must be freed
    if(height_ > 0 && width_ > 0) {
      HANDLE_ERROR(hipFreeArray(costmapArray_d_));
    }
    // 4 floats of size 32 bits
    channelDesc_ = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    HANDLE_ERROR(hipMallocArray(&costmapArray_d_, &channelDesc_, width, height));

    // set all of the elements in the array to be zero
    std::vector<float4> zero_array(width_*height_);
    zero_array.resize(0, make_float4(0,0,0,0));
    HANDLE_ERROR(hipMemcpyToArray(costmapArray_d_, 0, 0, zero_array.data(), width*height*sizeof(float4), hipMemcpyHostToDevice));
  }

  width_ = width;
  height_ = height;
  return true;
}

void ARStandardCost::clearCostmapCPU(int width, int height) {
  changeCostmapSize(width, height);

  if(width_ < 0 && height_ < 0) {
    return;
  }

  for (int i = 0; i < width_*height_; i++){
    track_costs_[i].x = 0;
    track_costs_[i].y = 0;
    track_costs_[i].z = 0;
    track_costs_[i].w = 0;
  }
}

std::vector<float4> ARStandardCost::loadTrackData(std::string map_path) {
  // check if file exists
  if(!fileExists(map_path)) {
    std::cerr << "ERROR: map path invalid, " << map_path << std::endl;
    return std::vector<float4>();
  }

  // load the npz file
  cnpy::npz_t map_dict = cnpy::npz_load(map_path);
  float x_min, x_max, y_min, y_max, ppm;
  float* xBounds = map_dict["xBounds"].data<float>();
  float* yBounds = map_dict["yBounds"].data<float>();
  float* pixelsPerMeter = map_dict["pixelsPerMeter"].data<float>();
  x_min = xBounds[0];
  x_max = xBounds[1];
  y_min = yBounds[0];
  y_max = yBounds[1];
  ppm = pixelsPerMeter[0];

  int width = int((x_max - x_min)*ppm);
  int height = int((y_max - y_min)*ppm);

  if(!changeCostmapSize(width, height)) {
    std::cerr << "ERROR: load track has invalid sizes" << std::endl;
    return std::vector<float4>();
  }

  float* channel0 = map_dict["channel0"].data<float>();
  float* channel1 = map_dict["channel1"].data<float>();
  float* channel2 = map_dict["channel2"].data<float>();
  float* channel3 = map_dict["channel3"].data<float>();

  // copy the track data into CPU side storage
  for (int i = 0; i < width_*height_; i++){
    track_costs_[i].x = channel0[i];
    track_costs_[i].y = channel1[i];
    track_costs_[i].z = channel2[i];
    track_costs_[i].w = channel3[i];
  }

  Eigen::Matrix3f R;
  Eigen::Array3f trs;

  //Save the scaling and offset
  R << 1./(x_max - x_min), 0,                  0,
          0,                  1./(y_max - y_min), 0,
          0,                  0,                  1;
  trs << -x_min/(x_max - x_min), -y_min/(y_max - y_min), 1;

  updateTransform(R, trs);
  costmapToTexture();

  return track_costs_;
}

void ARStandardCost::costmapToTexture() {
  if(width_ < 0 || height_ < 0) {
    std::cerr << "ERROR: cannot allocate texture with zero size" << std::endl;
    return;
  }

  // transfer CPU version of costmap to GPU
  float4* costmap_ptr = track_costs_.data();
  HANDLE_ERROR(hipMemcpyToArray(costmapArray_d_ , 0, 0, costmap_ptr, width_*height_*sizeof(float4), hipMemcpyHostToDevice));
  hipStreamSynchronize(stream_);

  //Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = costmapArray_d_;

  //Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  //Destroy current texture and create new texture object
  HANDLE_ERROR(hipDestroyTextureObject(costmap_tex_d_));
  HANDLE_ERROR(hipCreateTextureObject(&costmap_tex_d_, &resDesc, &texDesc, NULL) );

  // copy over pointers setup up on CPU code to GPU
  HANDLE_ERROR( hipMemcpyAsync(&cost_d_->costmapArray_d_, &costmapArray_d_, sizeof(hipArray*), hipMemcpyHostToDevice, stream_));
  HANDLE_ERROR( hipMemcpyAsync(&cost_d_->costmap_tex_d_, &costmap_tex_d_, sizeof(hipTextureObject_t), hipMemcpyHostToDevice, stream_));
  hipStreamSynchronize(stream_);
}

inline __device__ float4 ARStandardCost::queryTexture(float x, float y) const {
  //printf("\nquerying point (%f, %f)", x, y);
  return tex2D<float4>(costmap_tex_d_, x, y);
}

void ARStandardCost::updateTransform(Eigen::MatrixXf m, Eigen::ArrayXf trs) {
  params_.r_c1.x = m(0,0);
  params_.r_c1.y = m(1,0);
  params_.r_c1.z = m(2,0);
  params_.r_c2.x = m(0,1);
  params_.r_c2.y = m(1,1);
  params_.r_c2.z = m(2,1);
  params_.trs.x = trs(0);
  params_.trs.y = trs(1);
  params_.trs.z = trs(2);
  //Move the updated parameters to gpu memory
  if(GPUMemStatus_) {
    paramsToDevice();
  }
}

__host__ __device__ void ARStandardCost::coorTransform(float x, float y, float* u, float* v, float* w) {
  ////Compute a projective transform of (x, y, 0, 1)
  //printf("coordiante transform %f, %f, %f\n", params_.r_c1.x, params_.r_c2.x, params_.trs.x);
  u[0] = params_.r_c1.x*x + params_.r_c2.x*y + params_.trs.x;
  v[0] = params_.r_c1.y*x + params_.r_c2.y*y + params_.trs.y;
  w[0] = params_.r_c1.z*x + params_.r_c2.z*y + params_.trs.z;
}

__device__ float4 ARStandardCost::queryTextureTransformed(float x, float y) {
  float u, v, w;
  coorTransform(x, y, &u, &v, &w);
  //printf("\ninput coordinates: %f, %f", x, y);
  //printf("\nu = %f, v = %f, w = %f", u, v, w);
  //printf("\ntransformed coordinates %f, %f\n", u/w, v/w);
  return tex2D<float4>(costmap_tex_d_, u/w, v/w);
}

Eigen::Matrix3f ARStandardCost::getRotation() {
  Eigen::Matrix3f m;
  m(0,0) = params_.r_c1.x;
  m(1,0) = params_.r_c1.y;
  m(2,0) = params_.r_c1.z;
  m(0,1) = params_.r_c2.x;
  m(1,1) = params_.r_c2.y;
  m(2,1) = params_.r_c2.z;
  m(0,2) = 0.0;
  m(1,2) = 0.0;
  m(2,2) = 1.0;
  return m;
}

Eigen::Array3f ARStandardCost::getTranslation() {
  Eigen::Array3f array;
  array(0) = params_.trs.x;
  array(1) = params_.trs.y;
  array(2) = params_.trs.z;
  return array;
}

inline __host__ __device__ float ARStandardCost::getTerminalCost(float *s) {
  return 0.0;
}

inline __host__ __device__ float ARStandardCost::getControlCost(float *u, float *du, float *vars) {
  float control_cost = 0.0;
  //printf("du %f, %f\n", du[0], du[1]);
//printf("vars %f, %f\n", vars[0], vars[1]);
  //printf("vars %f, %f\n", u[0], u[1]);
  control_cost += params_.steering_coeff*du[0]*(u[0] - du[0])/(vars[0]*vars[0]);
  control_cost += params_.throttle_coeff*du[1]*(u[1] - du[1])/(vars[1]*vars[1]);
  return control_cost;
}

inline __host__ __device__ float ARStandardCost::getSpeedCost(float *s, int *crash) {
  float cost = 0;
  float error = s[4] - params_.desired_speed;
  if (l1_cost_){
    cost = fabs(error);
  }
  else {
    cost = error*error;
  }
  return (params_.speed_coeff*cost);
}

inline __host__ __device__ float ARStandardCost::getStabilizingCost(float *s) {
  float stabilizing_cost = 0;
  if (fabs(s[4]) > 0.001) {
    float slip = -atan(s[5]/fabs(s[4]));
    stabilizing_cost = params_.slip_penalty*powf(slip,2);
    if (fabs(-atan(s[5]/fabs(s[4]))) > params_.max_slip_ang) {
      //If the slip angle is above the max slip angle kill the trajectory.
      stabilizing_cost += params_.crash_coeff;
    }
  }
  return stabilizing_cost;
}

inline __host__ __device__ float ARStandardCost::getCrashCost(float *s, int *crash, int num_timestep) {
  float crash_cost = 0;
  if (crash[0] > 0) {
    crash_cost = params_.crash_coeff;
  }
  return crash_cost;
}

inline __device__ float ARStandardCost::getTrackCost(float *s, int *crash) {
  float track_cost = 0;

  //Compute a transformation to get the (x,y) positions of the front and back of the car.
  float x_front = s[0] + FRONT_D*__cosf(s[2]);
  float y_front = s[1] + FRONT_D*__sinf(s[2]);
  float x_back = s[0] + BACK_D*__cosf(s[2]);
  float y_back = s[1] + BACK_D*__sinf(s[2]);

  //Cost of front of the car
  float track_cost_front = queryTextureTransformed(x_front, y_front).x;
  //Cost for back of the car
  float track_cost_back = queryTextureTransformed(x_back, y_back).x;

  track_cost = (fabs(track_cost_front) + fabs(track_cost_back) )/2.0;
  if (fabs(track_cost) < params_.track_slop) {
    track_cost = 0;
  }
  else {
    track_cost = params_.track_coeff*track_cost;
  }
  if (track_cost_front >= params_.boundary_threshold || track_cost_back >= params_.boundary_threshold) {
    crash[0] = 1;
  }
  return track_cost;
}

inline __device__ float ARStandardCost::computeCost(float *s, float *u, float *du, float *vars, int *crash, int timestep) {
  float control_cost = getControlCost(u, du, vars);
  float track_cost = getTrackCost(s, crash);
  float speed_cost = getSpeedCost(s, crash);
  float crash_cost = powf(params_.discount, timestep)*getCrashCost(s, crash, timestep);
  float stabilizing_cost = getStabilizingCost(s);
  float cost = control_cost + speed_cost + crash_cost + track_cost + stabilizing_cost;
  if (cost > MAX_COST_VALUE || isnan(cost)) {
    cost = MAX_COST_VALUE;
  }
  return cost;
}
