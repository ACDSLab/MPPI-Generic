#include "hip/hip_runtime.h"
#include "mppi_core/mppi_common.cuh"

//#define STATE_DIM DYN_T::STATE_DIM;

namespace mppi_common {
    // Kernel functions
    template<class DYN_T, class COST_T, int BLOCKSIZE_X, int BLOCKSIZE_Y, int NUM_ROLLOUTS>
    __global__ void rolloutKernel(DYN_T* dynamics, COST_T* costs, float dt,
                                    int num_timesteps, float* x_d, float* u_d, float* du_d, float* sigma_u_d) {
        //Get thread and block id
        int thread_idx = threadIdx.x;
        int thread_idy = threadIdx.y;
        int block_idx = blockIdx.x;
        int global_idx = BLOCKSIZE_X * block_idx + thread_idx;

        //Create shared state and control arrays
        __shared__ float x_shared[BLOCKSIZE_X * DYN_T::STATE_DIM];
        __shared__ float xdot_shared[BLOCKSIZE_X * DYN_T::STATE_DIM];
        __shared__ float u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM];
        __shared__ float du_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM];
        __shared__ float sigma_u_shared[BLOCKSIZE_X * DYN_T::CONTROL_DIM];

        //Create local state, state dot and controls
        float* x;
        float* xdot;
        float* u;
        float* du;
        float* sigma_u;

        //Initialize running cost and total cost
        float running_cost = 0;
        float cost[NUM_ROLLOUTS];

        //Load global array to shared array
        if (global_idx < NUM_ROLLOUTS) {
            x = &x_shared[thread_idx * DYN_T::STATE_DIM];
            xdot = &xdot_shared[thread_idx * DYN_T::STATE_DIM];
            u = &u_shared[thread_idx * DYN_T::CONTROL_DIM];
            du = &du_shared[thread_idx * DYN_T::CONTROL_DIM];
            sigma_u = &sigma_u_shared[thread_idx * DYN_T::CONTROL_DIM];
        }
        __syncthreads();
        loadGlobalToShared(DYN_T::STATE_DIM, DYN_T::CONTROL_DIM, NUM_ROLLOUTS, BLOCKSIZE_Y,global_idx, thread_idy, x_d, sigma_u_d,
                            x, xdot, u, du, sigma_u);
        __syncthreads();

        /*<----Start of simulation loop-----> */
        for (int t = 0; t < num_timesteps; t++) {
            if (global_idx < NUM_ROLLOUTS) {
                //Load noise trajectories scaled by the exploration factor
                injectControlNoise(DYN_T::CONTROL_DIM, BLOCKSIZE_Y, NUM_ROLLOUTS, num_timesteps,
                                   t, global_idx, thread_idy, u_d, du_d, u, du, sigma_u);
                __syncthreads();

                //Accumulate running cost
                computeRunningCostAllRollouts(costs, x, u, running_cost);
                __syncthreads();

                //Compute state derivatives
                computeStateDerivAllRollouts(dynamics, x, u, xdot);
                __syncthreads();

                //Increment states
                incrementStateAllRollouts(DYN_T::STATE_DIM, BLOCKSIZE_Y, thread_idy, dt, x, xdot);
                __syncthreads();
            }
        }

        //Compute terminal cost and the final cost for each thread
        computeAndSaveCost(global_idx, costs, x, running_cost, cost);
        __syncthreads();
    }

    // Launch functions

    // RolloutKernel Helpers -------------------------------------------------------------------------------------------

    __device__ void loadGlobalToShared(int state_dim, int control_dim, int num_rollouts, int blocksize_y, int global_idx, int thread_idy,
                                        const float* x_device, const float* sigma_u_device, float* x_thread,
                                        float* xdot_thread, float* u_thread, float* du_thread, float* sigma_u_thread) {
        //Transfer to shared memory
        int i;
        if (global_idx < num_rollouts) {
            for (i = thread_idy; i < state_dim; i += blocksize_y) {
                x_thread[i] = x_device[i];
                xdot_thread[i] = 0;
            }
            for (i = thread_idy; i < control_dim; i += blocksize_y) {
                u_thread[i] = 0;
                du_thread[i] = 0;
                sigma_u_thread[i] = sigma_u_device[i];
            }
        }
    }

    __device__ void injectControlNoise(int control_dim, int blocksize_y, int num_rollouts, int num_timesteps,
            int current_timestep, int global_idx, int thread_idy,
            const float* u_traj_device, const float* ep_v_device, const float* sigma_u_thread,
            float* u_thread, float* du_thread) {
        //Load the noise trajectory scaled by the exploration factor
        // The prior loop already guarantees that the global index is less than the number of rollouts
        for (int i = thread_idy; i < control_dim; i += blocksize_y) {
            //Keep one noise free trajectory
            if (global_idx == 0){
                du_thread[i] = 0;
                u_thread[i] = u_traj_device[current_timestep * control_dim + i];
            }
            //Generate 1% zero control trajectory
            else if (global_idx >= 0.99*num_rollouts) {
                du_thread[i] = ep_v_device[global_idx*control_dim*num_timesteps + current_timestep * control_dim + i] * sigma_u_thread[i];
                u_thread[i] = du_thread[i];
            }
            else {
                du_thread[i] = ep_v_device[global_idx*control_dim*num_timesteps + current_timestep * control_dim + i] * sigma_u_thread[i];
                u_thread[i] = u_traj_device[current_timestep * control_dim + i] + du_thread[i];
            }
        }
    }

    template<class COST_T>
    __device__ void computeRunningCostAllRollouts(COST_T* costs, float* x_thread, float* u_thread, float& running_cost) {
        // The prior loop already guarantees that the global index is less than the number of rollouts
        running_cost += costs->computeRunningCost(x_thread, u_thread);
    }

    template<class DYN_T>
    __device__ void computeStateDerivAllRollouts(DYN_T* dynamics, float* x_thread, float* u_thread, float* xdot_thread) {
        // The prior loop already guarantees that the global index is less than the number of rollouts
        dynamics->xDot(x_thread, u_thread, xdot_thread);
    }

    __device__ void incrementStateAllRollouts(int state_dim, int blocksize_y, int thread_idy, float dt,
                                                float* x_thread, float* xdot_thread) {
        // The prior loop already guarantees that the global index is less than the number of rollouts
        //Implementing simple first order Euler for now, more complex scheme can be added later
        for (int i = thread_idy; i < state_dim; i += blocksize_y) {
            x_thread[i] += xdot_thread[i] * dt;
        }
    }

    template<class COST_T>
    __device__ void computeAndSaveCost(int num_rollouts, int global_idx, COST_T* costs, float* x_thread,
                                        float running_cost, float* cost_rollouts_device) {
        if (global_idx < num_rollouts) {
            cost_rollouts_device[global_idx] = running_cost + costs->computeTerminalCost(x_thread);
        }
    }

    // End of rollout kernel helpers -----------------------------------------------------------------------------------

    __global__ void normExpKernel(int blocksize_x, int num_rollouts, float* trajectory_costs_d, float gamma, float baseline) {
        int thread_idx = threadIdx.x;
        int block_idx = blockIdx.x;
        int global_idx = blocksize_x * block_idx + thread_idx;

        if (global_idx < num_rollouts) {
            float cost_dif = trajectory_costs_d[global_idx] - baseline;
            trajectory_costs_d[global_idx] = expf(-gamma*cost_dif);
        }
    }

    template<int CONTROL_DIM, int NUM_ROLLOUTS, int SUM_STRIDE>
    __global__ void weightedReductionKernel(int blocksize_x, int num_rollouts,float*  exp_costs_d, float* du_d, float* sigma_u_d, float* du_new_d,
            float normalizer, int num_timesteps) {
        int thread_idx = threadIdx.x;
        int block_idx = blockIdx.x;

        //Create a shared array for intermediate sums
        __shared__ float u_intermediate[CONTROL_DIM * ((NUM_ROLLOUTS - 1) / SUM_STRIDE) + 1];

        float u[CONTROL_DIM];
        for (int i = 0; i < CONTROL_DIM; i++) {
            u[i] = 0;
            u_intermediate[thread_idx * CONTROL_DIM + i] = 0;
        }

        __syncthreads();

        //Sum the weighted control variations at a desired stride
        if (thread_idx * SUM_STRIDE < NUM_ROLLOUTS) {
            float weight = 0;
            for (int i = 0; i < SUM_STRIDE; i++) {
                weight = exp_costs_d[thread_idx * SUM_STRIDE + i] / normalizer;
                for (int j = 0; j < CONTROL_DIM; j++) {
                    u[j] = du_d[(thread_idx * SUM_STRIDE + i) * (num_timesteps * CONTROL_DIM) + block_idx * CONTROL_DIM + j] * sigma_u_d[j];
                    u_intermediate[thread_idx * CONTROL_DIM + j] += weight * u[j];
                }
            }
        }

        __syncthreads();

        //Sum all weighted control variations
        if (thread_idx == 0 && block_idx < num_timesteps) {
            for (int i = 0; i < CONTROL_DIM; i++) {
                u[i] = 0;
            }
            for (int i = 0; i < ((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1); i++) {
                for (int j = 0; j < CONTROL_DIM; j++) {
                    u[j] += u_intermediate[i * CONTROL_DIM + j];
                }
            }
            for (int i = 0; i < CONTROL_DIM; i++) {
                du_new_d[block_idx * CONTROL_DIM + i] = u[i];
            }
        }
    }

    template<class DYN_T, int NUM_ROLLOUTS, int SUM_STRIDE >
    void launchWeightedReductionKernel(float* exp_costs_d, float* du_d, float* sigma_u_d, float* du_new_d, float normalizer, int num_timesteps) {
        dim3 dimBlock((NUM_ROLLOUTS - 1) / SUM_STRIDE + 1, 1, 1);
        dim3 dimGrid(num_timesteps, 1, 1);
        weightedReductionKernel<DYN_T::CONTROL_DIM, NUM_ROLLOUTS, SUM_STRIDE><<<dimGrid, dimBlock>>>(exp_costs_d, du_d, sigma_u_d, du_new_d, normalizer, num_timesteps);
        CudaCheckError();
        HANDLE_ERROR( hipDeviceSynchronize() );
    }

    void launchNormExpKernel(int num_rollouts, int blocksize_x, float* trajectory_costs_d, float gamma, float baseline) {
        dim3 dimBlock(blocksize_x, 1, 1);
        dim3 dimGrid((num_rollouts - 1) / blocksize_x + 1, 1, 1);
        normExpKernel<<<dimGrid, dimBlock>>>(blocksize_x, num_rollouts, trajectory_costs_d, gamma, baseline);
        CudaCheckError();
        HANDLE_ERROR( hipDeviceSynchronize() );
    }

    template<class DYN_T, class COST_T, int NUM_ROLLOUTS, int BLOCKSIZE_X, int BLOCKSIZE_Y>
    void launchRolloutKernel(DYN_T* dynamics, COST_T* costs, float dt, int num_timesteps, float* x_d, float* u_d, float* du_d, float* sigma_u_d) {
        const int gridsize_x = (NUM_ROLLOUTS - 1) / BLOCKSIZE_X + 1;
        dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
        dim3 dimGrid(gridsize_x, 1, 1);
        rolloutKernel<DYN_T, COST_T><<<dimGrid, dimBlock>>>(dynamics, costs, dt,
                num_timesteps, x_d, u_d, du_d, sigma_u_d);
        CudaCheckError();
        HANDLE_ERROR( hipDeviceSynchronize() );
    }

}
