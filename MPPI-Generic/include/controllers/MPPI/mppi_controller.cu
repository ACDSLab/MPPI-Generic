#include "hip/hip_runtime.h"
#include <controllers/MPPI/mppi_controller.cuh>
#include <mppi_core/mppi_common.cuh>

#define VanillaMPPI VanillaMPPIController<DYN_T, COST_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
VanillaMPPI::VanillaMPPIController(DYN_T* model, COST_T* cost,
                                   float dt,
                                   int max_iter,
                                   float gamma,
                                   int num_timesteps,
                                   const control_array& control_variance,
                                   const control_trajectory& init_control_traj,
                                   hipStream_t stream) :
dt_(dt), num_iters_(max_iter), gamma_(gamma), stream_(stream) {
    this->model_ = model;
    this->cost_ = cost;

    control_variance_ = control_variance;
    nominal_control_ = init_control_traj;
    setNumTimesteps(num_timesteps);

    // Create the random number generator
    createAndSeedCUDARandomNumberGen();

    // Bind the model and control to the given stream
    setCUDAStream(stream);

    // Call the GPU setup functions of the model and cost
    this->model_->GPUSetup();
    this->cost_->GPUSetup();


    // Allocate CUDA memory for the controller
    allocateCUDAMemory();

    // Copy the noise variance to the device
    copyControlVarianceToDevice();
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
VanillaMPPI::~VanillaMPPIController() {
    // Free the CUDA memory of every object
    this->model_->freeCudaMem();
    this->cost_->freeCudaMem();

    // Free the CUDA memory of the controller
    deallocateCUDAMemory();
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::computeControl(const state_array& state) {

    // Send the initial condition to the device
    HANDLE_ERROR( hipMemcpyAsync(initial_state_d_, state.data(),
        DYN_T::STATE_DIM*sizeof(float), hipMemcpyHostToDevice, stream_));

    for (int opt_iter = 0; opt_iter < num_iters_; opt_iter++) {
        // Send the nominal control to the device
        copyNominalControlToDevice();

        //Generate noise data
        hiprandGenerateNormal(gen_, control_noise_d_,
                             NUM_ROLLOUTS*num_timesteps_*DYN_T::CONTROL_DIM,
                             0.0, 1.0);

        //Launch the rollout kernel
        mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y>(
            this->model_->model_d_, this->cost_->cost_d_, dt_, num_timesteps_,
            initial_state_d_, nominal_control_d_, control_noise_d_,
            control_variance_d_, trajectory_costs_d_, stream_);

        // Copy the costs back to the host
        HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_.data(),
            trajectory_costs_d_,
            NUM_ROLLOUTS*sizeof(float),
            hipMemcpyDeviceToHost, stream_));
        HANDLE_ERROR( hipStreamSynchronize(stream_) );

        baseline_ = mppi_common::computeBaselineCost(trajectory_costs_.data(),
            NUM_ROLLOUTS);

        // Launch the norm exponential kernel
        mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X,
            trajectory_costs_d_, gamma_, baseline_, stream_);
        HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_.data(),
            trajectory_costs_d_,
            NUM_ROLLOUTS*sizeof(float),
            hipMemcpyDeviceToHost, stream_));
        HANDLE_ERROR(hipStreamSynchronize(stream_));

        // Compute the normalizer
        normalizer_ = mppi_common::computeNormalizer(trajectory_costs_.data(),
            NUM_ROLLOUTS);

        // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
        mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
            trajectory_costs_d_, control_noise_d_, nominal_control_d_,
            normalizer_, num_timesteps_, stream_);

        // Transfer the new control to the host
        HANDLE_ERROR( hipMemcpyAsync(nominal_control_.data(), nominal_control_d_,
                sizeof(float)*num_timesteps_*DYN_T::CONTROL_DIM,
                hipMemcpyDeviceToHost, stream_));
        hipStreamSynchronize(stream_);

        // TODO Add SavitskyGolay?
        // TODO Add nominal state computation
        computeNominalStateTrajectory(state);
    }

}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::allocateCUDAMemory() {
    HANDLE_ERROR(hipMalloc((void**)&initial_state_d_,
                            sizeof(float)*DYN_T::STATE_DIM));
    HANDLE_ERROR(hipMalloc((void**)&nominal_control_d_,
                            sizeof(float)*DYN_T::CONTROL_DIM*num_timesteps_));
    HANDLE_ERROR(hipMalloc((void**)&nominal_state_d_,
                            sizeof(float)*DYN_T::STATE_DIM*num_timesteps_));
    HANDLE_ERROR(hipMalloc((void**)&trajectory_costs_d_,
                            sizeof(float)*NUM_ROLLOUTS));
    HANDLE_ERROR(hipMalloc((void**)&control_variance_d_,
                            sizeof(float)*DYN_T::CONTROL_DIM));
    HANDLE_ERROR(hipMalloc((void**)&control_noise_d_,
                            sizeof(float)*DYN_T::CONTROL_DIM*num_timesteps_*NUM_ROLLOUTS));
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::deallocateCUDAMemory() {
    hipFree(nominal_control_d_);
    hipFree(nominal_state_d_);
    hipFree(trajectory_costs_d_);
    hipFree(control_variance_d_);
    hipFree(control_noise_d_);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void
VanillaMPPI::setCUDAStream(hipStream_t stream) {
    stream_ = stream;
    this->model_->bindToStream(stream);
    this->cost_->bindToStream(stream);
    hiprandSetStream(gen_, stream); // requires the generator to be created!
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::updateControlNoiseVariance(const control_array &sigma_u) {
    control_variance_ = sigma_u;
    copyControlVarianceToDevice();
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::copyControlVarianceToDevice() {
    HANDLE_ERROR(hipMemcpyAsync(control_variance_d_, control_variance_.data(), sizeof(float)*control_variance_.size(), hipMemcpyHostToDevice, stream_));
    hipStreamSynchronize(stream_);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::copyNominalControlToDevice() {
    HANDLE_ERROR(hipMemcpyAsync(nominal_control_d_, nominal_control_.data(), sizeof(float)*nominal_control_.size(), hipMemcpyHostToDevice, stream_));
    HANDLE_ERROR(hipStreamSynchronize(stream_));
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::computeNominalStateTrajectory(const state_array &x0) {
  nominal_state_.col(0) = x0;
  state_array xdot;
//  for (int i =0; i < num_timesteps_ - 1; ++i) {
//    nominal_state_.col(i+1) = nominal_state_.col(i);
//    state_array state = nominal_state_.col(i+1);
//    control_array control = nominal_control_.col(i);
//    this->model_->computeStateDeriv(state, control, xdot);
//    this->model_->updateState(state, xdot, dt_);
//    nominal_state_.col(i+1) = state;
//    }
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void
VanillaMPPI::setNumTimesteps(int num_timesteps) {
    if ((num_timesteps <= MAX_TIMESTEPS) && (num_timesteps > 0)) {
        num_timesteps_ = num_timesteps;
    } else {
        num_timesteps_ = MAX_TIMESTEPS;
        printf("You must give a number of timesteps between [0, %d]\n", MAX_TIMESTEPS);
    }
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void
VanillaMPPI::createAndSeedCUDARandomNumberGen() {
    // Seed the PseudoRandomGenerator with the CPU time.
    hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    hiprandSetPseudoRandomGeneratorSeed(gen_, seed);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void VanillaMPPI::slideControlSequence(int steps) {
    for (int i = 0; i < num_timesteps_; ++i) {
        for (int j = 0; j < DYN_T::CONTROL_DIM; j++) {
            if (i + steps < num_timesteps_) {
                nominal_control_(j,i) = nominal_control_(j,i + steps);
            } else {
                nominal_control_(j,i) = nominal_control_(j,num_timesteps_-1);
            }
        }
    }
}

//template<class DYN_T, class COST_T, int NUM_TIMESTEPS, int NUM_ROLLOUTS, int BDIM_X, int BDIM_Y>
//void
//VanillaMPPI::computeNominalStateTrajectory(const state_array& x0) {
//    // Increment the system forward
//    for (int i = 0; i < DYN_T::STATE_DIM; i++) {
//        nominal_state_[i] = x0[i];
//    }
//    for (int i = 1; i < NUM_TIMESTEPS; i++) {
//        for (int j = 0; j < DYN_T::STATE_DIM; j++) {
//            nominal_state_[i*DYN_T::STATE_DIM + j] = model_
//        }
//
//    }
//
//}


#undef VanillaMPPI
