#include "hip/hip_runtime.h"

#include <mppi_core/mppi_common.cuh>

#define TubeMPPI TubeMPPIController<DYN_T, COST_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>
#define VanillaMPPI VanillaMPPIController<DYN_T, COST_T, MAX_TIMESTEPS, NUM_ROLLOUTS, BDIM_X, BDIM_Y>

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
        int BDIM_X, int BDIM_Y>
TubeMPPI::TubeMPPIController(DYN_T* model, COST_T* cost, float dt, int max_iter,
                             float gamma, int num_timesteps,
                             const StateCostWeight& Q,
                             const Hessian& Qf,
                             const ControlCostWeight& R,
                             const control_array& control_variance,
                             const control_trajectory& init_control_traj,
                             hipStream_t stream) :
dt_(dt), num_iters_(max_iter), gamma_(gamma),
actual_control_(init_control_traj),
nominal_control_(init_control_traj), stream_(stream) {
    this->model_ = model;
    this->cost_ = cost;

    this->control_variance_ = control_variance;
    this->num_timesteps_ = num_timesteps;

    this->setNumTimesteps(num_timesteps);

    // Create the random number generator
    this->createAndSeedCUDARandomNumberGen();

    // Bind the model and control to the given stream
    this->setCUDAStream(stream);

    // Call the GPU setup functions of the model and cost
    this->model_->GPUSetup();
    this->cost_->GPUSetup();


    // Allocate CUDA memory for the controller
    allocateCUDAMemory();

    // Copy the noise variance to the device
    this->copyControlVarianceToDevice();

    // TODO: CREATE DDP STUFF HERE
    initDDP(Q, Qf, R);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void TubeMPPI::computeControl(const state_array& state) {
    if (!nominalStateInit_){
      for (int i = 0; i < DYN_T::STATE_DIM; i++){
        nominal_state_(i, 0) = state(i);
      }
      nominalStateInit_ = true;
    }

    // Handy reference pointers
    float * trajectory_costs_nominal_d = trajectory_costs_d_ + NUM_ROLLOUTS;
    float * initial_state_nominal_d = initial_state_d_ + DYN_T::STATE_DIM;

    float * control_noise_nominal_d = control_noise_d_ + NUM_ROLLOUTS *
                                      this->num_timesteps_ * DYN_T::CONTROL_DIM;
    float * control_nominal_d = control_d_ + this->num_timesteps_ * DYN_T::CONTROL_DIM;

    // Send the initial condition to the device

    HANDLE_ERROR( hipMemcpyAsync(initial_state_d_, state.data(),
        DYN_T::STATE_DIM*sizeof(float), hipMemcpyHostToDevice, stream_));

    HANDLE_ERROR( hipMemcpyAsync(initial_state_nominal_d, nominal_state_.data(),
        DYN_T::STATE_DIM*sizeof(float), hipMemcpyHostToDevice, stream_));

    for (int opt_iter = 0; opt_iter < num_iters_; opt_iter++) {
        // Send the nominal control to the device
        copyControlToDevice();

        //Generate noise data
        hiprandGenerateNormal(this->gen_, control_noise_d_,
                             NUM_ROLLOUTS*this->num_timesteps_*DYN_T::CONTROL_DIM,
                             0.0, 1.0);
        HANDLE_ERROR( hipMemcpyAsync(control_noise_nominal_d, control_noise_d_,
                     NUM_ROLLOUTS*this->num_timesteps_*DYN_T::CONTROL_DIM * sizeof(float),
                     hipMemcpyDeviceToDevice,
                     stream_) );

        //Launch the rollout kernel
        mppi_common::launchRolloutKernel<DYN_T, COST_T, NUM_ROLLOUTS, BDIM_X, BDIM_Y, 2>(
            this->model_->model_d_, this->cost_->cost_d_, dt_, this->num_timesteps_,
            initial_state_d_, control_d_, control_noise_d_,
            this->control_variance_d_, trajectory_costs_d_, stream_);

        // Copy the costs back to the host
        HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_actual_.data(),
            trajectory_costs_d_,
            NUM_ROLLOUTS*sizeof(float),
            hipMemcpyDeviceToHost, stream_));

        HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nominal_.data(),
            trajectory_costs_nominal_d,
            NUM_ROLLOUTS*sizeof(float),
            hipMemcpyDeviceToHost, stream_));
        HANDLE_ERROR( hipStreamSynchronize(stream_) );

        baseline_actual_ = mppi_common::computeBaselineCost(
            trajectory_costs_actual_.data(),
            NUM_ROLLOUTS);

        baseline_nominal_ = mppi_common::computeBaselineCost(
            trajectory_costs_nominal_.data(),
            NUM_ROLLOUTS);

        // Launch the norm exponential kernel for both actual and nominal
        mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X,
            trajectory_costs_d_, gamma_, baseline_actual_, stream_);

        mppi_common::launchNormExpKernel(NUM_ROLLOUTS, BDIM_X,
            trajectory_costs_nominal_d, gamma_, baseline_nominal_, stream_);

        HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_actual_.data(),
            trajectory_costs_d_,
            NUM_ROLLOUTS*sizeof(float),
            hipMemcpyDeviceToHost, stream_));
        HANDLE_ERROR(hipMemcpyAsync(trajectory_costs_nominal_.data(),
            trajectory_costs_nominal_d,
            NUM_ROLLOUTS*sizeof(float),
            hipMemcpyDeviceToHost, stream_));
        HANDLE_ERROR(hipStreamSynchronize(stream_));

        // Compute the normalizer
        normalizer_actual_ = mppi_common::computeNormalizer(
            trajectory_costs_actual_.data(), NUM_ROLLOUTS);
        normalizer_nominal_ = mppi_common::computeNormalizer(
            trajectory_costs_nominal_.data(), NUM_ROLLOUTS);

        // Compute the cost weighted average //TODO SUM_STRIDE is BDIM_X, but should it be its own parameter?
        mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
            trajectory_costs_d_, control_noise_d_, control_d_,
            normalizer_actual_, this->num_timesteps_, stream_);
        mppi_common::launchWeightedReductionKernel<DYN_T, NUM_ROLLOUTS, BDIM_X>(
            trajectory_costs_nominal_d,
            control_noise_nominal_d, control_nominal_d,
            normalizer_nominal_, this->num_timesteps_, stream_);

        // Transfer the new control to the host
        HANDLE_ERROR( hipMemcpyAsync(actual_control_.data(), control_d_,
                sizeof(float)*this->num_timesteps_*DYN_T::CONTROL_DIM,
                hipMemcpyDeviceToHost, stream_));
        HANDLE_ERROR( hipMemcpyAsync(nominal_control_.data(), control_nominal_d,
                sizeof(float)*this->num_timesteps_*DYN_T::CONTROL_DIM,
                hipMemcpyDeviceToHost, stream_));
        hipStreamSynchronize(stream_);

        if (baseline_actual_ < baseline_nominal_ + nominal_threshold_) {
            use_nominal_state_ = false;
            // reset nominal to actual

            // std::copy(actual_state_.begin(), actual_state_.end(), nominal_state_.begin());
            // std::copy(actual_control_.begin(), actual_control_.end(), nominal_control_.begin());
            nominal_state_ = actual_state_;
            nominal_control_ = actual_control_;
        }



        // TODO Add SavitskyGolay?

        // TODO Add nominal state computation

    }

}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void TubeMPPI::copyControlToDevice() {
    HANDLE_ERROR(hipMemcpyAsync(control_d_, actual_control_.data(),
                                 sizeof(float)*actual_control_.size(),
                                 hipMemcpyHostToDevice, stream_));

    HANDLE_ERROR(hipMemcpyAsync(control_d_ + nominal_control_.size(),
                                 nominal_control_.data(),
                                 sizeof(float)*nominal_control_.size(),
                                 hipMemcpyHostToDevice, stream_));
    HANDLE_ERROR(hipStreamSynchronize(stream_));
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void TubeMPPI::allocateCUDAMemory() {
    HANDLE_ERROR(hipMalloc((void**)&initial_state_d_,
                            sizeof(float) * DYN_T::STATE_DIM * 2));
    HANDLE_ERROR(hipMalloc((void**)&control_d_,
                            sizeof(float) * DYN_T::CONTROL_DIM *
                            this->num_timesteps_ * 2));
    HANDLE_ERROR(hipMalloc((void**)&state_d_,
                            sizeof(float) * DYN_T::STATE_DIM *
                            this->num_timesteps_ * 2));
    HANDLE_ERROR(hipMalloc((void**)&trajectory_costs_d_,
                            sizeof(float) * NUM_ROLLOUTS * 2));
    HANDLE_ERROR(hipMalloc((void**)&this->control_variance_d_,
                            sizeof(float) * DYN_T::CONTROL_DIM));
    HANDLE_ERROR(hipMalloc((void**)&control_noise_d_,
                            sizeof(float) * DYN_T::CONTROL_DIM *
                            this->num_timesteps_ * NUM_ROLLOUTS * 2));
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void TubeMPPI::deallocateCUDAMemory() {
    hipFree(control_d_);
    hipFree(state_d_);
    hipFree(trajectory_costs_d_);
    hipFree(this->control_variance_d_);
    hipFree(control_noise_d_);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void TubeMPPI::initDDP(const StateCostWeight& q_mat,
                       const Hessian& q_f_mat,
                       const ControlCostWeight& r_mat) {
    util::DefaultLogger logger;
    bool verbose = false;
    ddp_model_  = std::make_shared<ModelWrapperDDP<DYN_T>>(this->model_);
    ddp_solver_ = std::make_shared< DDP<ModelWrapperDDP<DYN_T>>>(dt_,
            this->num_timesteps_, 1, &logger, verbose);
    Q_ = q_mat;
    Qf_ = q_f_mat;
    R_ = r_mat;

    for (int i = 0; i < DYN_T::CONTROL_DIM; i++) {
        control_min_(i) = this->model_->control_rngs_[i].x;
        control_max_(i) = this->model_->control_rngs_[i].y;
    }

    run_cost_ = std::make_shared<TrackingCostDDP<ModelWrapperDDP<DYN_T>>>(Q_,
        R_, this->num_timesteps_);
    terminal_cost_ = std::make_shared<TrackingTerminalCost<ModelWrapperDDP<DYN_T>>>(Qf_);
}

template<class DYN_T, class COST_T, int MAX_TIMESTEPS, int NUM_ROLLOUTS,
         int BDIM_X, int BDIM_Y>
void TubeMPPI::computeFeedbackGains(const state_array& state) {
  Eigen::MatrixXf control_traj = Eigen::MatrixXf::Zero(DYN_T::CONTROL_DIM,
                                                       this->num_timesteps_);
  // replace with transpose?
  for (int t = 0; t < this->num_timesteps_; t++){
    for (int i = 0; i < DYN_T::CONTROL_DIM; i++){
      control_traj(i,t) = nominal_control_[DYN_T::CONTROL_DIM*t + i];
    }
  }
  run_cost_->setTargets(nominal_state_.data(), nominal_control_.data(),
    this->num_timesteps_);
  // Convert state_array to eigen
  Eigen::Matrix<float, DYN_T::STATE_DIM, 1> s;
  for (int i = 0; i < DYN_T::STATE_DIM; i++) {
    s(i) = state[i];
  }
  terminal_cost_->xf = run_cost_->traj_target_x_.col(this->num_timesteps_ - 1);
  result_ = ddp_solver_->run(s, control_traj,
                             *ddp_model_, *run_cost_, *terminal_cost_,
                             control_min_, control_max_);
}
